#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/cast_op.h"
#include "caffe2/utils/conversions.h"

namespace caffe2 {

template <typename DstType, typename SrcType>
__global__ void CastKernel(const int N, const SrcType* X, DstType* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    // Y[i] = static_cast<DstType>(X[i]);
    Y[i] = convert::To<SrcType, DstType>(X[i]);
  }
}

template <>
template <typename DstType, typename SrcType>
bool CastOp<HIPContext>::DoRunWithType() {
  auto& input = Input(0);

  auto* output = Output(0, input.sizes(), at::dtype<DstType>());
  const auto* data = input.template data<SrcType>();
  auto* out = output->template mutable_data<DstType>();
  DCHECK(input.numel() < INT_MAX);
  int N = input.numel();
  if (N == 0) {
    // skip the rest of the computation if input is empty
    return true;
  }
  CastKernel<DstType, SrcType>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(N, data, out);
  return true;
}

template <>
template <typename DstType>
bool CastOp<HIPContext>::DoRunWithDstType() {
  return DispatchHelper<
      TensorTypes<
          float,
          int32_t,
          bool,
          uint8_t,
          int8_t,
          uint16_t,
          int16_t,
          int64_t,
          double>,
      DstType>::call(this, Input(0));
}

// specific version that allows for casting to fp16
template <>
template <>
bool CastOp<HIPContext>::DoRunWithDstType<float>() {
  return DispatchHelper<
      TensorTypes<
          float,
          at::Half,
          int32_t,
          bool,
          uint8_t,
          int8_t,
          uint16_t,
          int16_t,
          int64_t,
          double>,
      float /* DstType */>::call(this, Input(0));
}

// specific version for casting _from_ fp16
template <>
template <>
bool CastOp<HIPContext>::DoRunWithDstType<at::Half>() {
  return DispatchHelper<
      TensorTypes<
          float,
          at::Half>,
      at::Half /* DstType */>::call(this, Input(0));
}
template <>
void CastOp<HIPContext>::SetBody(TensorProto_DataType to) {
  switch (to) {
    case TensorProto_DataType_FLOAT:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<float>;
      break;
    case TensorProto_DataType_INT32:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<int>;
      break;
    case TensorProto_DataType_BYTE:
      LOG(FATAL) << "BYTE is deprecated";
      break;
    case TensorProto_DataType_STRING:
      CAFFE_THROW("Casting to and from strings is not supported yet");
      // break;
    case TensorProto_DataType_BOOL:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<bool>;
      break;
    case TensorProto_DataType_UINT8:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<uint8_t>;
      break;
    case TensorProto_DataType_INT8:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<int8_t>;
      break;
    case TensorProto_DataType_UINT16:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<uint16_t>;
      break;
    case TensorProto_DataType_INT16:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<int16_t>;
      break;
    case TensorProto_DataType_INT64:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<int64_t>;
      break;
    case TensorProto_DataType_FLOAT16:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<at::Half>;
      break;
    case TensorProto_DataType_DOUBLE:
      body_ = &CastOp<HIPContext>::DoRunWithDstType<double>;
      break;
    case TensorProto_DataType_UNDEFINED:
      CAFFE_THROW("Cast op must have 'to' argument of type DataType");
      // break;
    default:
      CAFFE_THROW("Unexpected 'to' argument value: ", to);
  }
}

REGISTER_HIP_OPERATOR(Cast, CastOp<HIPContext>);

}  // namespace caffe2
