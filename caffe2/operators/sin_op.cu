#include "hip/hip_runtime.h"
#include "caffe2/operators/sin_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void
SinGradientCUDAKernel(const int N, const T* dY, const T* X, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    dX[i] = __ldg(dY + i) * cos(__ldg(X + i));
#else
    dX[i] = dY[i] * cos(X[i]);
#endif
  }
}

} // namespace

template <>
template <typename T>
bool SinGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& X_dims,
    const std::vector<int>& /* dY_dims */,
    const T* X,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      X_dims.cbegin(), X_dims.cend(), 1, std::multiplies<int>());
  SinGradientCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(size, dY, X, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    Sin,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        SinFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    SinGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        SinGradientFunctor<HIPContext>>);

} // namespace caffe2
