#include "hip/hip_runtime.h"
#include "caffe2/operators/rms_norm_op.h"

#include <vector>

#include <thrust/tuple.h>

#include "c10/cuda/CUDAMathCompat.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"
#include "caffe2/utils/math/reduce.cuh"
#include "caffe2/utils/math/utils.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void RowwiseRMSCUDAKernel(int64_t N, T eps, const T* X, T* rrms) {
  __shared__ typename BlockReduce<T>::TempStorage rms_storage;
  const int64_t i = blockIdx.x;
  T sum = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    sum += X[index] * X[index];
  }
  sum = BlockReduce<T>(rms_storage).Sum(sum);
  if (threadIdx.x == 0) {
    rrms[i] =
        c10::cuda::compat::rsqrt(sum / static_cast<T>(N) + static_cast<T>(eps));
  }
}

template <typename T>
__global__ void RMSNormForwardCUDAKernel(
    int64_t N,
    const T* X,
    const T* gamma,
    const T* beta,
    const T* rrms,
    T* Y) {
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    Y[index] = rrms[i] * X[index] * gamma[j] + beta[j];
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    const T* rrms,
    T* c2) {
  __shared__ typename BlockReduce<T>::TempStorage ds_storage;
  const int64_t i = blockIdx.x;
  T ds = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int index = i * N + j;
    ds += dY[index] * X[index] * gamma[j];
  }
  ds = BlockReduce<T>(ds_storage).Sum(ds);
  if (threadIdx.x == 0) {
    c2[i] = -ds * math::utils::Cube<T>(rrms[i]) / static_cast<T>(N);
  }
}

template <typename T>
__global__ void RMSNormBackwardCUDAKernel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    const T* c1,
    const T* c2,
    T* dX) {
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    dX[index] = c1[i] * dY[index] * gamma[j] + c2[i] * X[index];
  }
}

// Assume the batch size will not be very large, direct implementation is the
// most efficient one.
template <typename T>
__global__ void GammaBetaBackwardCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T* rrms,
    T* dg,
    T* db) {
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N) {
    T sum1 = 0;
    T sum2 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dY[index] * X[index] * rrms[i];
      sum2 += dY[index];
    }
    dg[j] = sum1;
    db[j] = sum2;
  }
}

} // namespace

template <>
template <typename T>
bool RMSNormOp<HIPContext>::DoRunWithType() {
  const auto& X = Input(0);
  const auto& gamma = Input(1);
  const auto& beta = Input(2);
  auto* Y = Output(0, X.sizes(), at::dtype<T>());
  CAFFE_ENFORCE_GE(X.dim(), 2, "RMSNorm requires input dim >= 2.");
  const int canonical_axis = X.canonical_axis_index(axis_);
  const std::vector<int64_t> rms_dims(
      X.sizes().cbegin(), X.sizes().cbegin() + canonical_axis);
  auto* rrms = Output(1, rms_dims, at::dtype<T>());
  const int64_t M = X.size_to_dim(canonical_axis);
  const int64_t N = X.size_from_dim(canonical_axis);
  CAFFE_ENFORCE_EQ(gamma.numel(), N);
  CAFFE_ENFORCE_EQ(beta.numel(), N);

  const T* X_data = X.template data<T>();
  const T* gamma_data = gamma.template data<T>();
  const T* beta_data = beta.template data<T>();
  T* Y_data = Y->template data<T>();
  T* rrms_data = rrms->template data<T>();

  if (M > 0) {
    RowwiseRMSCUDAKernel<T>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            N, static_cast<T>(eps_), X_data, rrms_data);
    RMSNormForwardCUDAKernel<T>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            N, X_data, gamma_data, beta_data, rrms_data, Y_data);
  }

  return true;
}

template <>
template <typename T>
void RMSNormGradientOp<HIPContext>::RMSNormBackward(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    const T* rrms,
    T* dX) {
  ReinitializeTensor(
      &c2_, {M}, at::dtype<T>().device(HIPContext::GetDeviceType()));
  T* c2_data = c2_.mutable_data<T>();
  ComputeInternalGradientsCUDAKernel<T>
      <<<M, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          N, dY, X, gamma, rrms, c2_data);
  RMSNormBackwardCUDAKernel<T>
      <<<M, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          N, dY, X, gamma, rrms, c2_data, dX);
}

template <>
template <typename T>
void RMSNormGradientOp<HIPContext>::GammaBetaBackward(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T* rrms,
    T* dgamma,
    T* dbeta) {
  const int64_t B = math::DivUp<int64_t>(N, CAFFE_CUDA_NUM_THREADS);
  GammaBetaBackwardCUDAKernel<T>
      <<<B, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          M, N, dY, X, rrms, dgamma, dbeta);
}

REGISTER_HIP_OPERATOR(RMSNorm, RMSNormOp<HIPContext>);
REGISTER_HIP_OPERATOR(RMSNormGradient, RMSNormGradientOp<HIPContext>);

} // namespace caffe2
