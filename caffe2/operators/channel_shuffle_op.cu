#include "hip/hip_runtime.h"
#include "caffe2/operators/channel_shuffle_op.h"

#include <array>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

template <typename T, bool kNFirst>
__global__ void ChannelShuffleNCHWKernel(
    const int G,
    const int K,
    const int HxW,
    const T* X,
    T* Y) {
  const int C = G * K;
  const int n = kNFirst ? blockIdx.x : blockIdx.y;
  const int s = kNFirst ? blockIdx.y : blockIdx.x;
  const int g = blockIdx.z % G;
  const int k = blockIdx.z / G;
  const int offset = s * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (offset < HxW) {
#if __CUDA_ARCH__ >= 350
    Y[(n * C + blockIdx.z) * HxW + offset] =
        __ldg(X + (n * C + g * K + k) * HxW + offset);
#else
    Y[(n * C + blockIdx.z) * HxW + offset] =
        X[(n * C + g * K + k) * HxW + offset];
#endif
  }
}

template <typename T, int kSharedSize>
__global__ void
ChannelShuffleNHWCKernel(const int G, const int K, const T* X, T* Y) {
  __shared__ T sdata[kSharedSize];
  const int C = G * K;
  const int offset = blockIdx.x * C;
  for (int i = threadIdx.x; i < C; i += blockDim.x) {
#if __CUDA_ARCH__ >= 350
    sdata[i] = __ldg(X + offset + i);
#else
    sdata[i] = X[offset + i];
#endif
  }
  __syncthreads();
  for (int i = threadIdx.x; i < C; i += blockDim.x) {
    const int g = i % G;
    const int k = i / G;
    Y[offset + i] = sdata[g * K + k];
  }
}

template <>
bool ChannelShuffleOp<float, HIPContext>::RunOnDeviceWithOrderNCHW() {
  const auto& X = Input(0);
  
  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  const int N = X.dim32(0);
  const int C = X.dim32(1);
  const int G = this->group_;
  CAFFE_ENFORCE_EQ(C % G, 0);
  if (X.numel() == 0) {
    return true;
  }
  const int K = C / G;
  const int HxW = X.numel() / (N * C);
  const int S = (HxW + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
  const float* X_data = X.data<float>();
  float* Y_data = Y->mutable_data<float>();
  if (N <= kCUDAGridDimMaxY) {
    const dim3 dim_grid(S, N, C);
    ChannelShuffleNCHWKernel<float, false>
        <<<dim_grid, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            G, K, HxW, X_data, Y_data);
  } else {
    const dim3 dim_grid(N, S, C);
    ChannelShuffleNCHWKernel<float, true>
        <<<dim_grid, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            G, K, HxW, X_data, Y_data);
  }
  return true;
}

template <>
bool ChannelShuffleOp<float, HIPContext>::RunOnDeviceWithOrderNHWC() {
  const auto& X = Input(0);
  
  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  const int ndim = X.dim();
  const int N = X.dim32(0);
  const int C = X.dim32(ndim - 1);
  const int G = this->group_;
  CAFFE_ENFORCE_EQ(C % G, 0);
  if (X.numel() == 0) {
    return true;
  }
  const int K = C / G;
  const int HxW = X.numel() / (N * C);
  const int outer_size = N * HxW;
  const float* X_data = X.data<float>();
  float* Y_data = Y->mutable_data<float>();
  if (C <= 32) {
    ChannelShuffleNHWCKernel<float, 32>
        <<<outer_size, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            G, K, X_data, Y_data);
  } else if (C <= 128) {
    ChannelShuffleNHWCKernel<float, 128>
        <<<outer_size, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            G, K, X_data, Y_data);
  } else if (C <= 512) {
    ChannelShuffleNHWCKernel<float, 512>
        <<<outer_size, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            G, K, X_data, Y_data);
  } else {
    const std::array<std::int64_t, 3> dims = {N * HxW, G, K};
    const std::array<std::int32_t, 3> axes = {0, 2, 1};
    math::Transpose<std::int64_t, float, HIPContext>(
        3, dims.data(), axes.data(), X_data, Y_data, &context_);
  }
  return true;
}

template <>
bool ChannelShuffleGradientOp<float, HIPContext>::RunOnDeviceWithOrderNCHW() {
  const auto& dY = Input(0);
  
  auto* dX = Output(0, dY.sizes(), at::dtype<float>());
  const int N = dY.dim32(0);
  const int C = dY.dim32(1);
  const int G = this->group_;
  CAFFE_ENFORCE_EQ(C % G, 0);
  if (dY.numel() == 0) {
    return true;
  }
  const int K = C / G;
  const int HxW = dY.numel() / (N * C);
  const int S = (HxW + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
  const float* dY_data = dY.data<float>();
  float* dX_data = dX->mutable_data<float>();
  if (N <= kCUDAGridDimMaxY) {
    const dim3 dim_grid(S, N, C);
    ChannelShuffleNCHWKernel<float, false>
        <<<dim_grid, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            K, G, HxW, dY_data, dX_data);
  } else {
    const dim3 dim_grid(N, S, C);
    ChannelShuffleNCHWKernel<float, true>
        <<<dim_grid, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            K, G, HxW, dY_data, dX_data);
  }
  return true;
}

template <>
bool ChannelShuffleGradientOp<float, HIPContext>::RunOnDeviceWithOrderNHWC() {
  const auto& dY = Input(0);
  
  auto* dX = Output(0, dY.sizes(), at::dtype<float>());
  const int ndim = dY.dim();
  const int N = dY.dim32(0);
  const int C = dY.dim32(ndim - 1);
  const int G = this->group_;
  CAFFE_ENFORCE_EQ(C % G, 0);
  if (dY.numel() == 0) {
    return true;
  }
  const int K = C / G;
  const int HxW = dY.numel() / (N * C);
  const int outer_size = N * HxW;
  const float* dY_data = dY.data<float>();
  float* dX_data = dX->mutable_data<float>();
  if (C <= 32) {
    ChannelShuffleNHWCKernel<float, 32>
        <<<outer_size, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            K, G, dY_data, dX_data);
  } else if (C <= 128) {
    ChannelShuffleNHWCKernel<float, 128>
        <<<outer_size, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            K, G, dY_data, dX_data);
  } else if (C <= 512) {
    ChannelShuffleNHWCKernel<float, 512>
        <<<outer_size, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
            K, G, dY_data, dX_data);
  } else {
    const std::array<std::int64_t, 3> dims = {N * HxW, K, G};
    const std::array<std::int32_t, 3> axes = {0, 2, 1};
    math::Transpose<std::int64_t, float, HIPContext>(
        3, dims.data(), axes.data(), dY_data, dX_data, &context_);
  }
  return true;
}

REGISTER_HIP_OPERATOR(ChannelShuffle, ChannelShuffleOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    ChannelShuffleGradient,
    ChannelShuffleGradientOp<float, HIPContext>);

} // namespace caffe2
