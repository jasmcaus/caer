#include "hip/hip_runtime.h"
#include "caffe2/operators/tile_op.h"

#include <array>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void TileCopyCUDAKernel(
    const int total_size,
    const int inner_size,
    const int tiles,
    const T* X,
    T* Y) {
  const int x = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (x < total_size) {
    const int r = x / inner_size / tiles;
    const int c = x % inner_size;
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
    Y[x] = __ldg(X + r * inner_size + c);
#else
    Y[x] = X[r * inner_size + c];
#endif
  }
}

} // namespace

template <>
template <typename T>
bool TileOp<HIPContext>::DoTile(
    const int outer_size,
    const int inner_size,
    const T* X,
    T* Y) {
  const std::int64_t total_size = static_cast<std::int64_t>(outer_size) *
      static_cast<std::int64_t>(tiles_) * static_cast<std::int64_t>(inner_size);
  const int M = math::DivUp<std::int64_t>(total_size, CAFFE_CUDA_NUM_THREADS);
  TileCopyCUDAKernel<T>
      <<<M, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          total_size, inner_size, tiles_, X, Y);
  return true;
}

template <>
template <typename T>
bool TileGradientOp<HIPContext>::DoTileGradient(
    const int outer_size,
    const int inner_size,
    const T* dY,
    T* dX) {
  const std::array<int, 3> dY_dims = {outer_size, tiles_, inner_size};
  const std::array<int, 3> dX_dims = {outer_size, 1, inner_size};
  math::ReduceSum<T, HIPContext>(
      3, dY_dims.data(), dX_dims.data(), T(1), dY, dX, &context_);
  return true;
}

template <>
template <>
bool TileGradientOp<HIPContext>::DoTileGradient<float>(
    const int outer_size,
    const int inner_size,
    const float* dY,
    float* dX) {
  if (inner_size == 1) {
    const std::array<int, 2> dY_dims = {outer_size, tiles_};
    const std::array<int, 2> dX_dims = {outer_size, 1};
    math::ReduceSum<float, HIPContext>(
        2, dY_dims.data(), dX_dims.data(), 1.0f, dY, dX, &context_);
  } else {
    ReinitializeTensor(&ones_, tiles_, at::dtype<float>().device(CUDA));
    math::Set<float, HIPContext>(
        tiles_, 1.0f, ones_.template mutable_data<float>(), &context_);
    math::GemmStridedBatched<float, HIPContext>(
        CblasTrans,
        CblasNoTrans,
        outer_size,
        inner_size,
        1,
        tiles_,
        1.0f,
        dY,
        tiles_ * inner_size,
        ones_.template data<float>(),
        0,
        0.0f,
        dX,
        inner_size,
        &context_);
  }
  return true;
}

REGISTER_HIP_OPERATOR(Tile, TileOp<HIPContext>);
REGISTER_HIP_OPERATOR(TileGradient, TileGradientOp<HIPContext>);

} // namespace caffe2
