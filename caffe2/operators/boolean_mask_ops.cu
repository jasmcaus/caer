#include "hip/hip_runtime.h"
#include <algorithm>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/boolean_mask_ops.h"

#include <hipcub/hipcub.hpp>

namespace caffe2 {

namespace {
__global__ void BooleanMaskCopyKernel(
    const int64_t numOfOutput,
    const int64_t numBytes,
    const int64_t* indices,
    const uint8_t* src,
    uint8_t* dest) {
  for (int64_t i = blockIdx.x; i < numOfOutput; i += gridDim.x) {
    const auto srcBase = indices[i] * numBytes;
    const auto destBase = i * numBytes;
    for (int64_t j = threadIdx.x; j < numBytes; j += blockDim.x) {
      dest[destBase + j] = src[srcBase + j];
    }
  }
}
}

template <>
class BooleanMaskOp<HIPContext> final : public Operator<HIPContext> {
 public:
  BooleanMaskOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws) {}

  bool RunOnDevice() override {
    const auto& src = Input(0);
    const auto& mask = Input(1);
    auto* dest = Output(0);

    CAFFE_ENFORCE(src.dim() >= 1);
    CAFFE_ENFORCE_EQ(mask.dim(), 1);
    CAFFE_ENFORCE(src.size(0) == mask.size(0));

    const auto* maskData = mask.data<bool>();
    const auto outerSize = mask.size(0);
    ReinitializeTensor(
        &indices_, {outerSize}, at::dtype<int64_t>().device(CUDA));
    auto* indicesData = indices_.mutable_data<int64_t>();

    size_t numBytes = 0;
    hipcub::CountingInputIterator<int> itr(0);
    hipcub::DeviceSelect::Flagged(
        nullptr,
        numBytes,
        itr,
        maskData,
        indicesData,
        static_cast<int64_t*>(nullptr),
        outerSize,
        context_.hip_stream());

    auto numint64_t =
        static_cast<int64_t>((numBytes + sizeof(int64_t) - 1) / sizeof(int64_t));
    // allocate one more int64_t at the end of scratch for storing numOfOutput
    ReinitializeTensor(
        &scratch_, {numint64_t + 1}, at::dtype<int64_t>().device(CUDA));
    auto* scratchData = scratch_.mutable_data<int64_t>();
    auto* numOfOutputData = scratchData + numint64_t;

    hipcub::DeviceSelect::Flagged(
        static_cast<void*>(scratchData),
        numBytes,
        itr,
        maskData,
        indicesData,
        numOfOutputData,
        outerSize,
        context_.hip_stream());

    // Copy numOfOutput from gpu to cpu
    int64_t numOfOutput;
    context_.CopyToCPU(1, numOfOutputData, &numOfOutput);

    indices_.Resize(numOfOutput);
    std::vector<int64_t> dims = src.sizes().vec();
    dims[0] = numOfOutput;
    dest->Resize(dims);
    auto* destData = (uint8_t*)dest->raw_mutable_data(src.meta());
    const auto* srcData = (uint8_t*)src.raw_data();
    if (OutputSize() == 2) {

      auto* indicesOut = Output(1, {numOfOutput}, at::dtype<int64_t>());
      indicesOut->template mutable_data<int64_t>();
    }

    if (numOfOutput > 0) {
      BooleanMaskCopyKernel<<<
          std::min(numOfOutput, static_cast<int64_t>(CAFFE_MAXIMUM_NUM_BLOCKS)),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          numOfOutput,
          src.size_from_dim(1) * src.meta().itemsize(),
          indicesData,
          srcData,
          destData);

      if (OutputSize() == 2) {
        Output(1)->CopyFrom(indices_, /* async */ true);
      }
    }

    return true;
  }

 private:
  Tensor indices_;
  Tensor scratch_;
};

REGISTER_HIP_OPERATOR(BooleanMask, BooleanMaskOp<HIPContext>);

namespace {

#define minf (-1.0f * std::numeric_limits<float>::infinity())

template <typename T>
__global__ void sequenceMaskKernel(
    int N,
    int M,
    int B,
    const T* in,
    const int* seq_lengths,
    T fill_val,
    T* out) {
  if (B >= 0) {
    CUDA_1D_KERNEL_LOOP(index, B * N * M) {
      int k = index % M;
      int j = (index - k) / M % N;
      int i = (index - M * j - k) / (N * M);

      int ind = N * M * i + M * j + k;
      out[ind] = (k >= seq_lengths[j] ? fill_val : in[ind]);
    }
  } else {
    CUDA_1D_KERNEL_LOOP(index, N * M) {
      int i = index / M;
      int j = index % M;

      out[index] = (j >= seq_lengths[i] ? fill_val : in[index]);
    }
  }
}

template <typename T>
__global__ void repeatedSequenceMaskKernel(
    int N,
    int M,
    int D,
    const T* in,
    const int* seq_lengths,
    T fill_val,
    T* out) {
  CUDA_1D_KERNEL_LOOP(index, N * M * D) {
    int i = index / (D * M);
    int j = (index / D) % M;

    out[index] = (j >= seq_lengths[i] ? fill_val : in[index]);
  }
}

template <typename T>
__global__ void windowMaskKernel(
    int N,
    int M,
    int B,
    const T* in,
    const int* window_centers,
    const int radius,
    T fill_val,
    T* out) {
  if (B >= 0) {
    CUDA_1D_KERNEL_LOOP(index, B * N * M) {
      int k = index % M;
      int j = (index - k) / M % N;
      int i = (index - M * j - k) / (N * M);

      int ind = N * M * i + M * j + k;
      out[ind] =
          (k < window_centers[j] - radius || k > window_centers[j] + radius
               ? fill_val
               : in[ind]);
    }
  } else {
    CUDA_1D_KERNEL_LOOP(index, N * M) {
      int i = index / M;
      int j = index % M;

      out[index] =
          (j < window_centers[i] - radius || j > window_centers[i] + radius
               ? fill_val
               : in[index]);
    }
  }
}

template <typename T>
__global__ void
upperMaskKernel(int N, int M, int B, const T* in, T fill_val, T* out) {
  if (B >= 0) {
    CUDA_1D_KERNEL_LOOP(index, B * N * M) {
      int k = index % M;
      int j = (index - k) / M % N;
      int i = (index - M * j - k) / (N * M);

      int ind = N * M * i + M * j + k;
      out[ind] = (k > j ? fill_val : in[ind]);
    }
  } else {
    CUDA_1D_KERNEL_LOOP(index, N * M) {
      int i = index / M;
      int j = index % M;

      out[index] = (j > i ? fill_val : in[index]);
    }
  }
}

template <typename T>
__global__ void
lowerMaskKernel(int N, int M, int B, const T* in, T fill_val, T* out) {
  if (B >= 0) {
    CUDA_1D_KERNEL_LOOP(index, B * N * M) {
      int k = index % M;
      int j = (index - k) / M % N;
      int i = (index - M * j - k) / (N * M);

      int ind = N * M * i + M * j + k;
      out[ind] = (k < j ? fill_val : in[ind]);
    }
  } else {
    CUDA_1D_KERNEL_LOOP(index, N * M) {
      int i = index / M;
      int j = index % M;

      out[index] = (j < i ? fill_val : in[index]);
    }
  }
}

template <typename T>
__global__ void
upperDiagMaskKernel(int N, int M, int B, const T* in, T fill_val, T* out) {
  if (B >= 0) {
    CUDA_1D_KERNEL_LOOP(index, B * N * M) {
      int k = index % M;
      int j = (index - k) / M % N;
      int i = (index - M * j - k) / (N * M);

      int ind = N * M * i + M * j + k;
      out[ind] = (k >= j ? fill_val : in[ind]);
    }
  } else {
    CUDA_1D_KERNEL_LOOP(index, N * M) {
      int i = index / M;
      int j = index % M;

      out[index] = (j >= i ? fill_val : in[index]);
    }
  }
}

template <typename T>
__global__ void
lowerDiagMaskKernel(int N, int M, int B, const T* in, T fill_val, T* out) {
  if (B >= 0) {
    CUDA_1D_KERNEL_LOOP(index, B * N * M) {
      int k = index % M;
      int j = (index - k) / M % N;
      int i = (index - M * j - k) / (N * M);

      int ind = N * M * i + M * j + k;
      out[ind] = (k <= j ? fill_val : in[ind]);
    }
  } else {
    CUDA_1D_KERNEL_LOOP(index, N * M) {
      int i = index / M;
      int j = index % M;

      out[index] = (j <= i ? fill_val : in[index]);
    }
  }
}

} // namespace

template <>
bool SequenceMaskOp<HIPContext>::RunOnDevice() {
    return DispatchHelper<TensorTypes<at::Half, float>>::call(this, Input(0));
}

template <>
template <class T>
bool SequenceMaskOp<HIPContext>::DoRunWithType() {
  const Tensor* input = &Input(0);
  const Tensor* sequence_lengths = nullptr;
  const Tensor* window_centers = nullptr;

  if (mode_ == "sequence") {
    sequence_lengths = &Input(1);
  } else if (mode_ == "window") {
    window_centers = &Input(1);
  }

  auto* output = Output(0, input->sizes(), at::dtype<T>());

  const auto canonical_axis = input->canonical_axis_index(axis_);

  // canonical_batch is non-negative if batching, -1 otherwise
  int canonical_batch = -1;
  if ((HasArgument("batch"))) {
    canonical_batch = input->canonical_axis_index(batch_);
  }

  // make sure batch < axis
  if (canonical_batch >= 0) {
    CAFFE_ENFORCE_LT(canonical_batch, canonical_axis);
  }

  // if no batch, then left is product of dims up to axis
  // otherwise, left is product of dims between batch and axis
  const int left =
      (canonical_batch >= 0
           ? input->size_between_dim(canonical_batch, canonical_axis)
           : input->size_to_dim(canonical_axis));
  const int right = input->size_from_dim(canonical_axis);

  // product of dims from 1 to batch
  const int batch_dim =
      (canonical_batch >= 0
           ? input->size_to_dim(canonical_batch) * input->dim(canonical_batch)
           : -1);

  T fill_val = convert::To<float, T>(grad_ ? 0.0f : fill_val_);
  if (mode_ == "sequence") {
    if (HasArgument("repeat_from_axis")) {
      const int canonical_repeat_from =
          input->canonical_axis_index(repeat_from_);
      const int repeated_dims = input->size_from_dim(canonical_repeat_from);
      const int masked_dims = right / repeated_dims;
      repeatedSequenceMaskKernel<<<
          CAFFE_GET_BLOCKS(left * right),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          left,
          masked_dims,
          repeated_dims,
          input->data<T>(),
          sequence_lengths->data<int>(),
          fill_val,
          output->template mutable_data<T>());
    } else {
      sequenceMaskKernel<<<
          CAFFE_GET_BLOCKS(left * right),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          left,
          right,
          batch_dim,
          input->data<T>(),
          sequence_lengths->data<int>(),
          fill_val,
          output->template mutable_data<T>());
    }
  } else if (mode_ == "window") {
    windowMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        batch_dim,
        input->data<T>(),
        window_centers->data<int>(),
        radius_,
        fill_val,
        output->template mutable_data<T>());
  } else if (mode_ == "upper") {
    upperMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        batch_dim,
        input->data<T>(),
        fill_val,
        output->template mutable_data<T>());
  } else if (mode_ == "lower") {
    lowerMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        batch_dim,
        input->data<T>(),
        fill_val,
        output->template mutable_data<T>());
  } else if (mode_ == "upperdiag") {
    upperDiagMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        batch_dim,
        input->data<T>(),
        fill_val,
        output->template mutable_data<T>());
  } else if (mode_ == "lowerdiag") {
    lowerDiagMaskKernel<<<
        CAFFE_GET_BLOCKS(left * right),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        left,
        right,
        batch_dim,
        input->data<T>(),
        fill_val,
        output->template mutable_data<T>());
  } else {
    CAFFE_ENFORCE(false, "Unsupported mode for SequenceMaskOp!");
  }

  return true;
}

REGISTER_HIP_OPERATOR(SequenceMask, SequenceMaskOp<HIPContext>);

} // namespace caffe2
