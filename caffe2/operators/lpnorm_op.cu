#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/lpnorm_op.h"
#include "caffe2/operators/operator_fallback_gpu.h"

namespace caffe2 {

REGISTER_HIP_OPERATOR(LpNorm, GPUFallbackOp);
REGISTER_HIP_OPERATOR(LpNormGradient, GPUFallbackOp);

} // namespace caffe2
