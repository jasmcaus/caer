#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/bucketize_op.h"

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>

namespace caffe2 {

__global__ void BucketizeOpKernel(
    const int N,
    const int M,
    const float* bounds,
    const float* X,
    int32_t* out) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int32_t low = -1, high = M;
    while (high - low > 1) {
      const int32_t median = low + (high - low) / 2;
      if (bounds[median] < X[i]) {
        low = median;
      } else {
        high = median;
      }
    }
    out[i] = high;
  }
}

template <>
bool BucketizeOp<HIPContext>::RunOnDevice() {
  auto& input = Input(X);
  CAFFE_ENFORCE_GE(input.dim(), 1);

  auto N = input.numel();
  auto* output = Output(INDICES, input.sizes(), at::dtype<int32_t>());
  const auto* input_data = input.template data<float>();
  auto* output_data = output->template mutable_data<int32_t>();

  BucketizeOpKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N,
      boundaries_device_.numel(),
      boundaries_device_.data<float>(),
      input_data,
      output_data);

  return true;
};

REGISTER_HIP_OPERATOR(Bucketize, BucketizeOp<HIPContext>);
} // namespace caffe2

using BucketizeCUDA = caffe2::BucketizeOp<caffe2::HIPContext>;

C10_EXPORT_CAFFE2_OP_TO_C10_CUDA(
    Bucketize,
    BucketizeCUDA);
