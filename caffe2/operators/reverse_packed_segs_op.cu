#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/reverse_packed_segs_op.h"

namespace caffe2 {

namespace {

template <typename T, typename LengthType>
__global__
void ReversePackedSegments_kernel(
      size_t max_length,
      size_t batch_size,
      size_t block_size,
      const LengthType* lengths_ptr,
      const T* data_ptr,
      T* rev_data_ptr) {

  const int block_id = blockIdx.x;

  // index into [0, batch_size)
  const int batch = block_id / max_length;
  // index into [0, segment)
  const int segment = block_id % max_length;

  if (batch >= batch_size || segment >= max_length) return;

  const int seg_length = lengths_ptr[batch];

  // unique data pointer for this CTA
  const T* local_data_ptr = data_ptr + (segment * batch_size + batch) * block_size;

  // unique pointer for result
  T* local_rev_data_ptr;
  if (segment < seg_length) {
    local_rev_data_ptr = rev_data_ptr + ((seg_length - 1 - segment) * batch_size + batch) * block_size;
  } else {
    local_rev_data_ptr = rev_data_ptr + (segment * batch_size + batch) * block_size;
  }

  // copy using 1 element / thread for now
  for (int idx = threadIdx.x; idx < block_size; idx+=blockDim.x) {
    local_rev_data_ptr[idx] = local_data_ptr[idx];
  }
}

} // namespace

// specialization of DoRunWithLengthType
template <>
template <typename T, typename LengthType>
void ReversePackedSegsOp<HIPContext>::DoRunWithLengthType() {
  const auto& data = Input(DATA);
  const auto& lengths = Input(LENGTHS);

  CAFFE_ENFORCE(
      data.dim() == 3,
      "DATA should be 3-D tensor <lengths, "
      "segments, embeddings>");
  CAFFE_ENFORCE(lengths.dim() == 1, "LENGTH should be 1-D");

  auto* output = Output(0, data.sizes(), at::dtype<T>());

  const auto max_length = data.size(0);
  const auto batch_size = data.size(1);
  const auto block_size = data.size(2);
  CAFFE_ENFORCE(
      lengths.sizes()[0] == batch_size,
      "lenths size should be"
      " equal to batch size");

  const T* data_ptr = data.template data<T>();
  const LengthType* lengths_ptr = lengths.template data<LengthType>();

  // reversed data
  T* rev_data_ptr = output->template mutable_data<T>();

  const int grid = max_length * batch_size;

  ReversePackedSegments_kernel<T,LengthType><<<grid, 512, 0, context_.hip_stream()>>>(
        max_length,
        batch_size,
        block_size,
        lengths_ptr,
        data_ptr,
        rev_data_ptr);
}

REGISTER_HIP_OPERATOR(ReversePackedSegs, ReversePackedSegsOp<HIPContext>);
} // namespace caffe2
