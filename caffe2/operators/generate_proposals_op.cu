#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "caffe2/core/context.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/generate_proposals_op.h"
#include "caffe2/operators/generate_proposals_op_util_boxes.h" // BBOX_XFORM_CLIP_DEFAULT
#include "caffe2/operators/generate_proposals_op_util_nms.h"
#include "caffe2/operators/generate_proposals_op_util_nms_gpu.h"

#ifdef __HIP_PLATFORM_HCC__
#include <cfloat>
#endif

using caffe2::utils::RotatedBox;

namespace caffe2 {
namespace {
__global__ void GeneratePreNMSUprightBoxesKernel(
    const int* d_sorted_scores_keys,
    const int nboxes_to_generate,
    const float* d_bbox_deltas,
    const float4* d_anchors,
    const int H,
    const int W,
    const int A,
    const float feat_stride,
    const float min_size,
    const float* d_img_info_vec,
    const int num_images,
    const float bbox_xform_clip,
    const bool legacy_plus_one,
    float4* d_out_boxes,
    const int prenms_nboxes, // leading dimension of out_boxes
    float* d_inout_scores,
    char* d_boxes_keep_flags) {
  const int K = H * W;
  const int KA = K * A;
  CUDA_2D_KERNEL_LOOP(ibox, nboxes_to_generate, image_index, num_images) {
    // box_conv_index : # of the same box, but indexed in
    // the scores from the conv layer, of shape (A,H,W)
    // the num_images dimension was already removed
    // box_conv_index = a*K + h*W + w
    const int box_conv_index = d_sorted_scores_keys[image_index * KA + ibox];

    // We want to decompose box_conv_index in (a,h,w)
    // such as box_conv_index = a*K + h*W + w
    // (avoiding modulos in the process)
    int remaining = box_conv_index;
    const int dA = K; // stride of A
    const int a = remaining / dA;
    remaining -= a * dA;
    const int dH = W; // stride of H
    const int h = remaining / dH;
    remaining -= h * dH;
    const int w = remaining; // dW = 1

    // Loading the anchor a
    // float4 is a struct with float x,y,z,w
    const float4 anchor = d_anchors[a];
    // x1,y1,x2,y2 :coordinates of anchor a, shifted for position (h,w)
    const float shift_w = feat_stride * w;
    float x1 = shift_w + anchor.x;
    float x2 = shift_w + anchor.z;
    const float shift_h = feat_stride * h;
    float y1 = shift_h + anchor.y;
    float y2 = shift_h + anchor.w;

    // TODO use fast math when possible

    // Deltas for that box
    // Deltas of shape (num_images,4*A,K)
    // We're going to compute 4 scattered reads
    // better than the alternative, ie transposing the complete deltas
    // array first
    int deltas_idx = image_index * (KA * 4) + a * 4 * K + h * W + w;
    const float dx = d_bbox_deltas[deltas_idx];
    // Stride of K between each dimension
    deltas_idx += K;
    const float dy = d_bbox_deltas[deltas_idx];
    deltas_idx += K;
    float dw = d_bbox_deltas[deltas_idx];
    deltas_idx += K;
    float dh = d_bbox_deltas[deltas_idx];

    // Upper bound on dw,dh
    dw = fmin(dw, bbox_xform_clip);
    dh = fmin(dh, bbox_xform_clip);

    // Applying the deltas
    float width = x2 - x1 + float(int(legacy_plus_one));
    const float ctr_x = x1 + 0.5f * width;
    const float pred_ctr_x = ctr_x + width * dx; // TODO fuse madd
    const float pred_w = width * expf(dw);
    x1 = pred_ctr_x - 0.5f * pred_w;
    x2 = pred_ctr_x + 0.5f * pred_w - float(int(legacy_plus_one));

    float height = y2 - y1 + float(int(legacy_plus_one));
    const float ctr_y = y1 + 0.5f * height;
    const float pred_ctr_y = ctr_y + height * dy;
    const float pred_h = height * expf(dh);
    y1 = pred_ctr_y - 0.5f * pred_h;
    y2 = pred_ctr_y + 0.5f * pred_h - float(int(legacy_plus_one));

    // Clipping box to image
    const float img_height = d_img_info_vec[3 * image_index + 0];
    const float img_width = d_img_info_vec[3 * image_index + 1];
    const float min_size_scaled =
        min_size * d_img_info_vec[3 * image_index + 2];
    x1 = fmax(fmin(x1, img_width - float(int(legacy_plus_one))), 0.0f);
    y1 = fmax(fmin(y1, img_height - float(int(legacy_plus_one))), 0.0f);
    x2 = fmax(fmin(x2, img_width - float(int(legacy_plus_one))), 0.0f);
    y2 = fmax(fmin(y2, img_height - float(int(legacy_plus_one))), 0.0f);

    // Filter boxes
    // Removing boxes with one dim < min_size
    // (center of box is in image, because of previous step)
    width = x2 - x1 + float(int(legacy_plus_one)); // may have changed
    height = y2 - y1 + float(int(legacy_plus_one));
    bool keep_box = fmin(width, height) >= min_size_scaled;

    // We are not deleting the box right now even if !keep_box
    // we want to keep the relative order of the elements stable
    // we'll do it in such a way later
    // d_boxes_keep_flags size: (num_images,prenms_nboxes)
    // d_out_boxes size: (num_images,prenms_nboxes)
    const int out_index = image_index * prenms_nboxes + ibox;
    d_boxes_keep_flags[out_index] = keep_box;
    d_out_boxes[out_index] = {x1, y1, x2, y2};

    // d_inout_scores size: (num_images,KA)
    if (!keep_box)
      d_inout_scores[image_index * KA + ibox] = FLT_MIN; // for NMS
  }
}

__global__ void GeneratePreNMSRotatedBoxesKernel(
    const int* d_sorted_scores_keys,
    const int nboxes_to_generate,
    const float* d_bbox_deltas,
    const RotatedBox* d_anchors,
    const int H,
    const int W,
    const int A,
    const float feat_stride,
    const float min_size,
    const float* d_img_info_vec,
    const int num_images,
    const float bbox_xform_clip,
    const bool legacy_plus_one,
    const bool angle_bound_on,
    const int angle_bound_lo,
    const int angle_bound_hi,
    const bool clip_angle_thresh,
    RotatedBox* d_out_boxes,
    const int prenms_nboxes, // leading dimension of out_boxes
    float* d_inout_scores,
    char* d_boxes_keep_flags) {
  constexpr float PI = 3.14159265358979323846;
  const int K = H * W;
  const int KA = K * A;
  CUDA_2D_KERNEL_LOOP(ibox, nboxes_to_generate, image_index, num_images) {
    // box_conv_index : # of the same box, but indexed in
    // the scores from the conv layer, of shape (A,H,W)
    // the num_images dimension was already removed
    // box_conv_index = a*K + h*W + w
    const int box_conv_index = d_sorted_scores_keys[image_index * KA + ibox];

    // We want to decompose box_conv_index in (a,h,w)
    // such as box_conv_index = a*K + h*W + w
    // (avoiding modulos in the process)
    int remaining = box_conv_index;
    const int dA = K; // stride of A
    const int a = remaining / dA;
    remaining -= a * dA;
    const int dH = W; // stride of H
    const int h = remaining / dH;
    remaining -= h * dH;
    const int w = remaining; // dW = 1

    // Loading the anchor a and applying shifts.
    // RotatedBox in [ctr_x, ctr_y, w, h, angle] format.
    // Zero shift for width, height and angle.
    RotatedBox box = d_anchors[a];
    box.x_ctr += feat_stride * w; // x_ctr shifted for w
    box.y_ctr += feat_stride * h; // y_ctr shifted for h

    // TODO use fast math when possible

    // Deltas for that box
    // Deltas of shape (num_images,5*A,K)
    // We're going to compute 5 scattered reads
    // better than the alternative, ie transposing the complete deltas
    // array first
    int deltas_idx = image_index * (KA * 5) + a * 5 * K + h * W + w;
    // Stride of K between each dimension
    RotatedBox delta;
    delta.x_ctr = d_bbox_deltas[deltas_idx + K * 0];
    delta.y_ctr = d_bbox_deltas[deltas_idx + K * 1];
    delta.w = d_bbox_deltas[deltas_idx + K * 2];
    delta.h = d_bbox_deltas[deltas_idx + K * 3];
    delta.a = d_bbox_deltas[deltas_idx + K * 4];

    // Upper bound on dw,dh
    delta.w = fmin(delta.w, bbox_xform_clip);
    delta.h = fmin(delta.h, bbox_xform_clip);

    // Convert back to degrees
    delta.a *= 180.f / PI;

    // Applying the deltas
    box.x_ctr += delta.x_ctr * box.w;
    box.y_ctr += delta.y_ctr * box.h;
    box.w *= expf(delta.w);
    box.h *= expf(delta.h);
    box.a += delta.a;

    if (angle_bound_on) {
      // Normalize angle to be within [angle_bound_lo, angle_bound_hi].
      // Deltas are guaranteed to be <= period / 2 while computing training
      // targets by bbox_transform_inv.
      const float period = angle_bound_hi - angle_bound_lo;
      // CAFFE_ENFORCE(period > 0 && period % 180 == 0);
      if (box.a < angle_bound_lo) {
        box.a += period;
      } else if (box.a > angle_bound_hi) {
        box.a -= period;
      }
    }

    // Clipping box to image.
    // Only clip boxes that are almost upright (with a tolerance of
    // clip_angle_thresh) for backward compatibility with horizontal boxes.
    const float img_height = d_img_info_vec[3 * image_index + 0];
    const float img_width = d_img_info_vec[3 * image_index + 1];
    const float min_size_scaled =
        min_size * d_img_info_vec[3 * image_index + 2];
    if (fabs(box.a) <= clip_angle_thresh) {
      // Convert from [x_ctr, y_ctr, w, h] to [x1, y1, x2, y2]
      float x1 = box.x_ctr - (box.w - float(int(legacy_plus_one))) / 2.f;
      float y1 = box.y_ctr - (box.h - float(int(legacy_plus_one))) / 2.f;
      float x2 = x1 + box.w - float(int(legacy_plus_one));
      float y2 = y1 + box.h - float(int(legacy_plus_one));

      // Clip
      x1 = fmax(fmin(x1, img_width - float(int(legacy_plus_one))), 0.0f);
      y1 = fmax(fmin(y1, img_height - float(int(legacy_plus_one))), 0.0f);
      x2 = fmax(fmin(x2, img_width - float(int(legacy_plus_one))), 0.0f);
      y2 = fmax(fmin(y2, img_height - float(int(legacy_plus_one))), 0.0f);

      // Convert back to [x_ctr, y_ctr, w, h]
      box.x_ctr = (x1 + x2) / 2.f;
      box.y_ctr = (y1 + y2) / 2.f;
      box.w = x2 - x1 + float(int(legacy_plus_one));
      box.h = y2 - y1 + float(int(legacy_plus_one));
    }

    // Filter boxes.
    // Removing boxes with one dim < min_size or center outside the image.
    bool keep_box = (fmin(box.w, box.h) >= min_size_scaled) &&
        (box.x_ctr < img_width) && (box.y_ctr < img_height);

    // We are not deleting the box right now even if !keep_box
    // we want to keep the relative order of the elements stable
    // we'll do it in such a way later
    // d_boxes_keep_flags size: (num_images,prenms_nboxes)
    // d_out_boxes size: (num_images,prenms_nboxes)
    const int out_index = image_index * prenms_nboxes + ibox;
    d_boxes_keep_flags[out_index] = keep_box;
    d_out_boxes[out_index] = box;

    // d_inout_scores size: (num_images,KA)
    if (!keep_box) {
      d_inout_scores[image_index * KA + ibox] = FLT_MIN; // for NMS
    }
  }
}

__global__ void WriteUprightBoxesOutput(
    const float4* d_image_boxes,
    const float* d_image_scores,
    const int* d_image_boxes_keep_list,
    const int nboxes,
    const int image_index,
    float* d_image_out_rois,
    float* d_image_out_rois_probs) {
  CUDA_1D_KERNEL_LOOP(i, nboxes) {
    const int ibox = d_image_boxes_keep_list[i];
    const float4 box = d_image_boxes[ibox];
    const float score = d_image_scores[ibox];
    // Scattered memory accesses
    // postnms_nboxes is small anyway
    d_image_out_rois_probs[i] = score;
    const int base_idx = 5 * i;
    d_image_out_rois[base_idx + 0] = image_index;
    d_image_out_rois[base_idx + 1] = box.x;
    d_image_out_rois[base_idx + 2] = box.y;
    d_image_out_rois[base_idx + 3] = box.z;
    d_image_out_rois[base_idx + 4] = box.w;
  }
}

__global__ void WriteRotatedBoxesOutput(
    const RotatedBox* d_image_boxes,
    const float* d_image_scores,
    const int* d_image_boxes_keep_list,
    const int nboxes,
    const int image_index,
    float* d_image_out_rois,
    float* d_image_out_rois_probs) {
  CUDA_1D_KERNEL_LOOP(i, nboxes) {
    const int ibox = d_image_boxes_keep_list[i];
    const RotatedBox box = d_image_boxes[ibox];
    const float score = d_image_scores[ibox];
    // Scattered memory accesses
    // postnms_nboxes is small anyway
    d_image_out_rois_probs[i] = score;
    const int base_idx = 6 * i;
    d_image_out_rois[base_idx + 0] = image_index;
    d_image_out_rois[base_idx + 1] = box.x_ctr;
    d_image_out_rois[base_idx + 2] = box.y_ctr;
    d_image_out_rois[base_idx + 3] = box.w;
    d_image_out_rois[base_idx + 4] = box.h;
    d_image_out_rois[base_idx + 5] = box.a;
  }
}

__global__ void InitializeDataKernel(
    const int num_images,
    const int KA,
    int* d_image_offsets,
    int* d_boxes_keys_iota) {
  CUDA_2D_KERNEL_LOOP(box_idx, KA, img_idx, num_images) {
    d_boxes_keys_iota[img_idx * KA + box_idx] = box_idx;

    // One 1D line sets the 1D data
    if (box_idx == 0) {
      d_image_offsets[img_idx] = KA * img_idx;
      // One thread sets the last+1 offset
      if (img_idx == 0)
        d_image_offsets[num_images] = KA * num_images;
    }
  }
}

} // namespace

template <>
bool GenerateProposalsOp<HIPContext>::RunOnDevice() {
  const auto& scores = Input(0);
  const auto& bbox_deltas = Input(1);
  const auto& im_info_tensor = Input(2);
  const auto& anchors = Input(3);
  auto* out_rois = Output(0);
  auto* out_rois_probs = Output(1);

  CAFFE_ENFORCE_EQ(scores.ndim(), 4, scores.ndim());
  CAFFE_ENFORCE(scores.template IsType<float>(), scores.meta().name());

  const auto num_images = scores.dim(0);
  const auto A = scores.dim(1);
  const auto H = scores.dim(2);
  const auto W = scores.dim(3);
  const auto box_dim = anchors.dim(1);

  CAFFE_ENFORCE(box_dim == 4 || box_dim == 5);

  const int K = H * W;
  const int conv_layer_nboxes = K * A;
  // Getting data members ready

  // We'll sort the scores
  // we want to remember their original indexes,
  // ie their indexes in the tensor of shape (num_images,A,K)
  // from the conv layer
  // each row of d_conv_layer_indexes is at first initialized to 1..A*K
  dev_conv_layer_indexes_.Resize(num_images, conv_layer_nboxes);
  int* d_conv_layer_indexes =
      dev_conv_layer_indexes_.template mutable_data<int>();

  // d_image_offset[i] = i*K*A for i from 1 to num_images+1
  // Used by the segmented sort to only sort scores within one image
  dev_image_offset_.Resize(num_images + 1);
  int* d_image_offset = dev_image_offset_.template mutable_data<int>();

  // The following calls to CUB primitives do nothing
  // (because the first arg is nullptr)
  // except setting cub_*_temp_storage_bytes
  size_t cub_sort_temp_storage_bytes = 0;
  float* flt_ptr = nullptr;
  int* int_ptr = nullptr;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      nullptr,
      cub_sort_temp_storage_bytes,
      flt_ptr,
      flt_ptr,
      int_ptr,
      int_ptr,
      num_images * conv_layer_nboxes,
      num_images,
      int_ptr,
      int_ptr,
      0,
      8 * sizeof(float), // sort all bits
      context_.hip_stream());

  // Allocate temporary storage for CUB
  dev_cub_sort_buffer_.Resize(cub_sort_temp_storage_bytes);
  void* d_cub_sort_temp_storage =
      dev_cub_sort_buffer_.template mutable_data<char>();

  size_t cub_select_temp_storage_bytes = 0;
  char* char_ptr = nullptr;
  hipcub::DeviceSelect::Flagged(
      nullptr,
      cub_select_temp_storage_bytes,
      flt_ptr,
      char_ptr,
      flt_ptr,
      int_ptr,
      K * A,
      context_.hip_stream());

  // Allocate temporary storage for CUB
  dev_cub_select_buffer_.Resize(cub_select_temp_storage_bytes);
  void* d_cub_select_temp_storage =
      dev_cub_select_buffer_.template mutable_data<char>();

  // Initialize :
  // - each row of dev_conv_layer_indexes to 1..K*A
  // - each d_nboxes to 0
  // - d_image_offset[i] = K*A*i for i 1..num_images+1
  // 2D grid
  InitializeDataKernel<<<
      (CAFFE_GET_BLOCKS(A * K), num_images),
      CAFFE_CUDA_NUM_THREADS, // blockDim.y == 1
      0,
      context_.hip_stream()>>>(
      num_images, conv_layer_nboxes, d_image_offset, d_conv_layer_indexes);

  // Sorting input scores
  dev_sorted_conv_layer_indexes_.Resize(num_images, conv_layer_nboxes);
  dev_sorted_scores_.Resize(num_images, conv_layer_nboxes);
  const float* d_in_scores = scores.data<float>();
  int* d_sorted_conv_layer_indexes =
      dev_sorted_conv_layer_indexes_.template mutable_data<int>();
  float* d_sorted_scores = dev_sorted_scores_.template mutable_data<float>();
  ;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(
      d_cub_sort_temp_storage,
      cub_sort_temp_storage_bytes,
      d_in_scores,
      d_sorted_scores,
      d_conv_layer_indexes,
      d_sorted_conv_layer_indexes,
      num_images * conv_layer_nboxes,
      num_images,
      d_image_offset,
      d_image_offset + 1,
      0,
      8 * sizeof(float), // sort all bits
      context_.hip_stream());

  // Keeping only the topN pre_nms
  const int nboxes_to_generate = std::min(conv_layer_nboxes, rpn_pre_nms_topN_);

  // Generating the boxes associated to the topN pre_nms scores
  dev_boxes_.Resize(num_images, box_dim * nboxes_to_generate);
  dev_boxes_keep_flags_.Resize(num_images, nboxes_to_generate);
  const float* d_bbox_deltas = bbox_deltas.data<float>();
  const float* d_anchors = anchors.data<float>();
  const float* d_im_info_vec = im_info_tensor.data<float>();
  float* d_boxes = dev_boxes_.template mutable_data<float>();
  ;
  char* d_boxes_keep_flags =
      dev_boxes_keep_flags_.template mutable_data<char>();

  if (box_dim == 4) {
    GeneratePreNMSUprightBoxesKernel<<<
        (CAFFE_GET_BLOCKS(nboxes_to_generate), num_images),
        CAFFE_CUDA_NUM_THREADS, // blockDim.y == 1
        0,
        context_.hip_stream()>>>(
        d_sorted_conv_layer_indexes,
        nboxes_to_generate,
        d_bbox_deltas,
        reinterpret_cast<const float4*>(d_anchors),
        H,
        W,
        A,
        feat_stride_,
        rpn_min_size_,
        d_im_info_vec,
        num_images,
        utils::BBOX_XFORM_CLIP_DEFAULT,
        legacy_plus_one_,
        reinterpret_cast<float4*>(d_boxes),
        nboxes_to_generate,
        d_sorted_scores,
        d_boxes_keep_flags);
  } else {
    GeneratePreNMSRotatedBoxesKernel<<<
        (CAFFE_GET_BLOCKS(nboxes_to_generate), num_images),
        CAFFE_CUDA_NUM_THREADS, // blockDim.y == 1
        0,
        context_.hip_stream()>>>(
        d_sorted_conv_layer_indexes,
        nboxes_to_generate,
        d_bbox_deltas,
        reinterpret_cast<const RotatedBox*>(d_anchors),
        H,
        W,
        A,
        feat_stride_,
        rpn_min_size_,
        d_im_info_vec,
        num_images,
        utils::BBOX_XFORM_CLIP_DEFAULT,
        legacy_plus_one_,
        angle_bound_on_,
        angle_bound_lo_,
        angle_bound_hi_,
        clip_angle_thresh_,
        reinterpret_cast<RotatedBox*>(d_boxes),
        nboxes_to_generate,
        d_sorted_scores,
        d_boxes_keep_flags);
  }
  const int nboxes_generated = nboxes_to_generate;
  dev_image_prenms_boxes_.Resize(box_dim * nboxes_generated);
  float* d_image_prenms_boxes =
      dev_image_prenms_boxes_.template mutable_data<float>();
  dev_image_prenms_scores_.Resize(nboxes_generated);
  float* d_image_prenms_scores =
      dev_image_prenms_scores_.template mutable_data<float>();
  dev_image_boxes_keep_list_.Resize(nboxes_generated);
  int* d_image_boxes_keep_list =
      dev_image_boxes_keep_list_.template mutable_data<int>();

  const int roi_cols = box_dim + 1;
  const int max_postnms_nboxes = std::min(nboxes_generated, rpn_post_nms_topN_);
  dev_postnms_rois_.Resize(roi_cols * num_images * max_postnms_nboxes);
  dev_postnms_rois_probs_.Resize(num_images * max_postnms_nboxes);
  float* d_postnms_rois = dev_postnms_rois_.template mutable_data<float>();
  float* d_postnms_rois_probs =
      dev_postnms_rois_probs_.template mutable_data<float>();

  dev_prenms_nboxes_.Resize(num_images);
  host_prenms_nboxes_.Resize(num_images);
  int* d_prenms_nboxes = dev_prenms_nboxes_.template mutable_data<int>();
  int* h_prenms_nboxes = host_prenms_nboxes_.template mutable_data<int>();

  int nrois_in_output = 0;
  for (int image_index = 0; image_index < num_images; ++image_index) {
    // Sub matrices for current image
    const float* d_image_boxes =
        &d_boxes[image_index * nboxes_generated * box_dim];
    const float* d_image_sorted_scores = &d_sorted_scores[image_index * K * A];
    char* d_image_boxes_keep_flags =
        &d_boxes_keep_flags[image_index * nboxes_generated];

    float* d_image_postnms_rois = &d_postnms_rois[roi_cols * nrois_in_output];
    float* d_image_postnms_rois_probs = &d_postnms_rois_probs[nrois_in_output];

    // Moving valid boxes (ie the ones with d_boxes_keep_flags[ibox] == true)
    // to the output tensors

    if (box_dim == 4) {
      hipcub::DeviceSelect::Flagged(
          d_cub_select_temp_storage,
          cub_select_temp_storage_bytes,
          reinterpret_cast<const float4*>(d_image_boxes),
          d_image_boxes_keep_flags,
          reinterpret_cast<float4*>(d_image_prenms_boxes),
          d_prenms_nboxes,
          nboxes_generated,
          context_.hip_stream());
    } else {
      hipcub::DeviceSelect::Flagged(
          d_cub_select_temp_storage,
          cub_select_temp_storage_bytes,
          reinterpret_cast<const RotatedBox*>(d_image_boxes),
          d_image_boxes_keep_flags,
          reinterpret_cast<RotatedBox*>(d_image_prenms_boxes),
          d_prenms_nboxes,
          nboxes_generated,
          context_.hip_stream());
    }

    hipcub::DeviceSelect::Flagged(
        d_cub_select_temp_storage,
        cub_select_temp_storage_bytes,
        d_image_sorted_scores,
        d_image_boxes_keep_flags,
        d_image_prenms_scores,
        d_prenms_nboxes,
        nboxes_generated,
        context_.hip_stream());

    host_prenms_nboxes_.CopyFrom(dev_prenms_nboxes_);

    // We know prenms_boxes <= topN_prenms, because nboxes_generated <=
    // topN_prenms. Calling NMS on the generated boxes
    const int prenms_nboxes = *h_prenms_nboxes;
    int nkeep;
    utils::nms_gpu(
        d_image_prenms_boxes,
        prenms_nboxes,
        rpn_nms_thresh_,
        legacy_plus_one_,
        d_image_boxes_keep_list,
        &nkeep,
        dev_nms_mask_,
        host_nms_mask_,
        &context_,
        box_dim);

    // All operations done after previous sort were keeping the relative order
    // of the elements the elements are still sorted keep topN <=> truncate the
    // array
    const int postnms_nboxes = std::min(nkeep, rpn_post_nms_topN_);

    // Moving the out boxes to the output tensors,
    // adding the image_index dimension on the fly
    if (box_dim == 4) {
      WriteUprightBoxesOutput<<<
          CAFFE_GET_BLOCKS(postnms_nboxes),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          reinterpret_cast<const float4*>(d_image_prenms_boxes),
          d_image_prenms_scores,
          d_image_boxes_keep_list,
          postnms_nboxes,
          image_index,
          d_image_postnms_rois,
          d_image_postnms_rois_probs);
    } else {
      WriteRotatedBoxesOutput<<<
          CAFFE_GET_BLOCKS(postnms_nboxes),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          reinterpret_cast<const RotatedBox*>(d_image_prenms_boxes),
          d_image_prenms_scores,
          d_image_boxes_keep_list,
          postnms_nboxes,
          image_index,
          d_image_postnms_rois,
          d_image_postnms_rois_probs);
    }

    nrois_in_output += postnms_nboxes;
  }

  // Using a buffer because we cannot call ShrinkTo
  out_rois->Resize(nrois_in_output, roi_cols);
  out_rois_probs->Resize(nrois_in_output);
  float* d_out_rois = out_rois->template mutable_data<float>();
  float* d_out_rois_probs = out_rois_probs->template mutable_data<float>();

  CUDA_CHECK(hipMemcpyAsync(
      d_out_rois,
      d_postnms_rois,
      nrois_in_output * roi_cols * sizeof(float),
      hipMemcpyDeviceToDevice,
      context_.hip_stream()));
  CUDA_CHECK(hipMemcpyAsync(
      d_out_rois_probs,
      d_postnms_rois_probs,
      nrois_in_output * sizeof(float),
      hipMemcpyDeviceToDevice,
      context_.hip_stream()));

  return true;
}

REGISTER_HIP_OPERATOR(GenerateProposals, GenerateProposalsOp<HIPContext>);
} // namespace caffe2

C10_EXPORT_CAFFE2_OP_TO_C10_CUDA(
    GenerateProposals,
    caffe2::GenerateProposalsOp<caffe2::HIPContext>);
