#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/system/cuda/execution_policy.h>

#include "caffe2/operators/summarize_op.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

// structure used to accumulate the moments and other statistical properties
// encountered so far.
template <typename T>
struct SummaryStatsData {
  T n;
  T min;
  T max;
  T mean;
  T M2;

  // initialize to the identity element
  void initialize() {
    n = mean = M2 = 0;
    min = std::numeric_limits<T>::max();
    max = std::numeric_limits<T>::min();
  }

  T variance() { return (n == 1 ? 0 : M2 / (n - 1)); }
};

// stats_unary_op is a functor that takes in a value x and
// returns a variace_data whose mean value is initialized to x.
template <typename T>
struct summary_stats_unary_op {
  __host__ __device__ SummaryStatsData<T> operator()(const T& x) const {
     SummaryStatsData<T> result;
     result.n    = 1;
     result.min  = x;
     result.max  = x;
     result.mean = x;
     result.M2   = 0;
     return result;
  }
};

// summary_stats_binary_op is a functor that accepts two SummaryStatsData
// structs and returns a new SummaryStatsData which are an
// approximation to the summary_stats for
// all values that have been aggregated so far
template <typename T>
struct summary_stats_binary_op
    : public thrust::binary_function<const SummaryStatsData<T>&,
                                     const SummaryStatsData<T>&,
                                           SummaryStatsData<T> > {
  __host__ __device__ SummaryStatsData<T> operator()(
      const SummaryStatsData<T>& x, const SummaryStatsData <T>& y) const {
    SummaryStatsData<T> result;
    T n  = x.n + y.n;
    T delta  = y.mean - x.mean;
    T delta2 = delta  * delta;
    result.n   = n;
    result.min = thrust::min(x.min, y.min);
    result.max = thrust::max(x.max, y.max);
    result.mean = x.mean + delta * y.n / n;
    result.M2  = x.M2 + y.M2;
    result.M2 += delta2 * x.n * y.n / n;
    return result;
  }
};

}  // namespace

template<>
bool SummarizeOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  const int N = X.numel();
  DCHECK_GT(N, 0);

  // TODO(Yangqing): Any better way to avoid having to const cast?
  thrust::device_ptr<float> Xdata(const_cast<float*>(X.data<float>()));
  summary_stats_unary_op<float> unary_op;
  summary_stats_binary_op<float> binary_op;
  SummaryStatsData<float> init;
  init.initialize();
  // compute summary statistics
  SummaryStatsData<float> result = thrust::transform_reduce(
#if THRUST_VERSION >= 100800
      thrust::cuda::par.on(context_.hip_stream()),
#endif  // THRUST_VERSION >= 100800
      Xdata, Xdata + N, unary_op, init, binary_op);
  float standard_deviation = std::sqrt(result.variance());
  if (to_file_) {
    (*log_file_) << result.min << " " << result.max << " " << result.mean << " "
                 << standard_deviation << std::endl;
  }
  if (OutputSize()) {
    auto* Y = Output(0, {4}, at::dtype<float>());
    float output_buffer[NUM_STATS] = {result.min, result.max, result.mean,
                               standard_deviation};
    context_.CopyFromCPU<float>(
        NUM_STATS, output_buffer, Y->template mutable_data<float>());
  }
  return true;
}

REGISTER_HIP_OPERATOR(Summarize, SummarizeOp<float, HIPContext>);
}  // namespace caffe2
