#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/loss_op.h"

namespace caffe2 {
REGISTER_HIP_OPERATOR(AveragedLoss, AveragedLoss<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    AveragedLossGradient,
    AveragedLossGradient<float, HIPContext>);
}  // namespace caffe2
