#include "hip/hip_runtime.h"
#include "caffe2/operators/affine_channel_op.h"

#include <cub/block/block_reduce.cuh>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
using BlockReduce = hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS>;

template <typename T, StorageOrder kOrder>
__global__ void AffineChannelScaleBiasBackwardCUDAKernel(
    const int N,
    const int C,
    const int HxW,
    const T* dY,
    const T* X,
    T* dscale,
    T* dbias) {
  const int outer_size = C;
  const int inner_size = N * HxW;
  __shared__ typename BlockReduce<T>::TempStorage ds_storage;
  __shared__ typename BlockReduce<T>::TempStorage db_storage;
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    T ds_sum = 0;
    T db_sum = 0;
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int index = kOrder == StorageOrder::NCHW
          ? (j / HxW * C + i) * HxW + j % HxW
          : j * outer_size + i;
#if __CUDA_ARCH__ >= 350
      ds_sum += __ldg(dY + index) * __ldg(X + index);
      db_sum += __ldg(dY + index);
#else
      ds_sum += dY[index] * X[index];
      db_sum += dY[index];
#endif
    }
    ds_sum = BlockReduce<T>(ds_storage).Reduce(ds_sum, hipcub::Sum());
    db_sum = BlockReduce<T>(db_storage).Reduce(db_sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      dscale[i] = ds_sum;
      dbias[i] = db_sum;
    }
    __syncthreads();
  }
}

} // namespace

template <>
bool AffineChannelGradientOp<float, HIPContext>::RunOnDeviceWithOrderNCHW() {
  const auto& dY = Input(0);
  const auto& scale = is_learnable_ ? Input(2) : Input(1);
  
  auto* dX = Output(0, dY.sizes(), at::dtype<float>());
  const int N = dY.dim32(0);
  const int C = dY.dim32(1);
  const int HxW = dY.numel() / (N * C);
  const float* dY_data = dY.data<float>();
  const float* scale_data = scale.data<float>();
  const std::array<int, 3> X_dims = {N, C, HxW};
  const std::array<int, 3> scale_dims = {1, C, 1};
  math::Mul<float, HIPContext>(
      3,
      X_dims.data(),
      3,
      scale_dims.data(),
      dY_data,
      scale_data,
      dX->template mutable_data<float>(),
      &context_);
  if (is_learnable_) {
    const auto& X = Input(1);
    const float* X_data = X.data<float>();
    
    
    auto* dscale = Output(1, scale.sizes(), at::dtype<float>());
    auto* dbias = Output(2, scale.sizes(), at::dtype<float>());
    const int outer_size = N * HxW;
    AffineChannelScaleBiasBackwardCUDAKernel<float, StorageOrder::NCHW>
        <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            N,
            C,
            HxW,
            dY_data,
            X_data,
            dscale->template mutable_data<float>(),
            dbias->template mutable_data<float>());
  }
  return true;
}

template <>
bool AffineChannelGradientOp<float, HIPContext>::RunOnDeviceWithOrderNHWC() {
  const auto& dY = Input(0);
  const auto& scale = is_learnable_ ? Input(2) : Input(1);
  
  auto* dX = Output(0, dY.sizes(), at::dtype<float>());
  const int ndim = dY.dim();
  const int C = dY.dim32(ndim - 1);
  const int rows = dY.numel() / C;
  const int cols = C;
  const float* dY_data = dY.data<float>();
  const float* scale_data = scale.data<float>();
  math::RowwiseMul<float, HIPContext>(
      rows,
      cols,
      dY_data,
      scale_data,
      dX->template mutable_data<float>(),
      &context_);
  if (is_learnable_) {
    const auto& X = Input(1);
    const float* X_data = X.data<float>();
    const int N = X.dim32(0);
    const int HxW = rows / N;
    
    
    auto* dscale = Output(1, scale.sizes(), at::dtype<float>());
    auto* dbias = Output(2, scale.sizes(), at::dtype<float>());
    AffineChannelScaleBiasBackwardCUDAKernel<float, StorageOrder::NHWC>
        <<<std::min(rows, CAFFE_MAXIMUM_NUM_BLOCKS),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            N,
            C,
            HxW,
            dY_data,
            X_data,
            dscale->template mutable_data<float>(),
            dbias->template mutable_data<float>());
  }
  return true;
}

REGISTER_HIP_OPERATOR(AffineChannel, AffineChannelOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    AffineChannelGradient,
    AffineChannelGradientOp<float, HIPContext>);

} // namespace caffe2
