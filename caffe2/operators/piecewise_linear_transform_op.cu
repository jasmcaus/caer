#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/piecewise_linear_transform_op.h"

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>

namespace caffe2 {

namespace {
__global__ void PieceWiseLinearTransformGeneralKernel(
    const int N,
    const int M,
    const int num_grp,
    const int num_fnc_per_grp,
    const float* bounds,
    const float* slopes,
    const float* intercepts,
    const float* X,
    float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N * M) {
    int col = i % M;
    const float* bounds_group = bounds + (col * (num_fnc_per_grp + 1));
    const float* slopes_group = slopes + (col * num_fnc_per_grp);
    const float* intercepts_group = intercepts + (col * num_fnc_per_grp);

    if (X[i] <= bounds_group[0]) {
      Y[i] = slopes_group[0] * bounds_group[0] + intercepts_group[0];
    } else if (X[i] >= bounds_group[num_fnc_per_grp]) {
      Y[i] = slopes_group[num_fnc_per_grp - 1] * bounds_group[num_fnc_per_grp] +
          intercepts_group[num_fnc_per_grp - 1];
    } else {
      auto low_bound = thrust::lower_bound(
          thrust::device,
          bounds_group,
          bounds_group + num_fnc_per_grp + 1,
          X[i]);
      int bounds_idx = low_bound - bounds_group - 1;
      Y[i] = slopes_group[bounds_idx] * X[i] + intercepts_group[bounds_idx];
    }
  }
}

} // namespace

namespace {
__global__ void PieceWiseLinearTransformBinaryKernel1(
    const int N,
    const int M,
    const int num_grp,
    const int num_fnc_per_grp,
    const float* bounds,
    const float* slopes,
    const float* intercepts,
    const float* X,
    float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    if (X[i] <= bounds[0]) {
      Y[i] = slopes[0] * bounds[0] + intercepts[0];
    } else if (X[i] >= bounds[num_fnc_per_grp]) {
      Y[i] = slopes[num_fnc_per_grp - 1] * bounds[num_fnc_per_grp] +
          intercepts[num_fnc_per_grp - 1];
    } else {
      auto low_bound = thrust::lower_bound(
          thrust::device, bounds, bounds + num_fnc_per_grp + 1, X[i]);
      int bounds_idx = low_bound - bounds - 1;
      Y[i] = slopes[bounds_idx] * X[i] + intercepts[bounds_idx];
    }
  }
}
} // namespace

namespace {
__global__ void PieceWiseLinearTransformBinaryKernel2(
    const int N,
    const int M,
    const int num_grp,
    const int num_fnc_per_grp,
    const float* bounds,
    const float* slopes,
    const float* intercepts,
    const float* X,
    float* Y) {
  // N*M/2 = N as M=2
  CUDA_1D_KERNEL_LOOP(i, N) {
    int index = i * M;
    if (X[index + 1] <= bounds[0]) {
      Y[index + 1] = slopes[0] * bounds[0] + intercepts[0];
    } else if (X[index + 1] >= bounds[num_fnc_per_grp]) {
      Y[index + 1] = slopes[num_fnc_per_grp - 1] * bounds[num_fnc_per_grp] +
          intercepts[num_fnc_per_grp - 1];
    } else {
      auto low_bound = thrust::lower_bound(
          thrust::device, bounds, bounds + num_fnc_per_grp + 1, X[index + 1]);
      int bounds_idx = low_bound - bounds - 1;
      Y[index + 1] = slopes[bounds_idx] * X[index + 1] + intercepts[bounds_idx];
    }
    Y[index] = 1.0f - Y[index + 1];
  }
}
} // namespace

template <>
void PiecewiseLinearTransformOp<float, HIPContext>::setUpTensors(
    int64_t& num_func_per_group,
    int64_t& num_group,
    int64_t M) {
  if (transform_param_from_arg_) {
    if (!gpu_copied_) {
      int64_t num_bounds;
      int64_t num_slopes;
      int64_t num_intercepts;

      CAFFE_ENFORCE_EQ(InputSize(), 1);

      const float* bounds;
      const float* slopes;
      const float* intercepts;
      bounds = bounds_from_arg_.data();
      slopes = slopes_from_arg_.data();
      intercepts = intercepts_from_arg_.data();
      num_bounds = bounds_from_arg_.size();
      num_slopes = slopes_from_arg_.size();
      num_intercepts = intercepts_from_arg_.size();
      InferNumFunctionsPerGroup(
          num_bounds,
          num_slopes,
          num_intercepts,
          &num_func_per_group,
          &num_group);

      if (binary_) {
        CAFFE_ENFORCE_EQ(num_group, 1);
      } else {
        CAFFE_ENFORCE_EQ(num_group, M);
      }

      int length = num_group * num_func_per_group;
      Tensor bounds_host{CPU};
      bounds_host.Resize(length + num_group);
      memcpy(
          bounds_host.mutable_data<float>(),
          bounds,
          (length + num_group) * sizeof(float));

      Tensor intercepts_host{CPU};
      intercepts_host.Resize(length);
      memcpy(
          intercepts_host.mutable_data<float>(),
          intercepts,
          (length) * sizeof(float));
      Tensor slopes_host{CPU};
      slopes_host.Resize(length);
      memcpy(
          slopes_host.mutable_data<float>(), slopes, (length) * sizeof(float));

      bounds_device_.CopyFrom(bounds_host);
      intercepts_device_.CopyFrom(intercepts_host);
      slopes_device_.CopyFrom(slopes_host);

      gpu_copied_ = true;
    }
  } else {
    int64_t num_bounds;
    int64_t num_slopes;
    int64_t num_intercepts;
    CAFFE_ENFORCE_EQ(InputSize(), 4);
    auto& bounds_input = Input(BOUNDS);
    auto& slopes_input = Input(SLOPES);
    auto& intercepts_input = Input(INTERCEPTS);
    num_bounds = bounds_input.numel();
    num_slopes = slopes_input.numel();
    num_intercepts = intercepts_input.numel();
    InferNumFunctionsPerGroup(
        num_bounds,
        num_slopes,
        num_intercepts,
        &num_func_per_group,
        &num_group);

    if (binary_) {
      CAFFE_ENFORCE_EQ(num_group, 1);
    } else {
      CAFFE_ENFORCE_EQ(num_group, M);
    }

    bounds_device_.CopyFrom(bounds_input);
    slopes_device_.CopyFrom(slopes_input);
    intercepts_device_.CopyFrom(intercepts_input);
  }
}

template <>
bool PiecewiseLinearTransformOp<float, HIPContext>::TransformGeneral() {
  auto& X = Input(0);

  CAFFE_ENFORCE_EQ(X.dim(), 2);
  int64_t N = X.dim32(0);
  int64_t M = X.dim32(1);
  auto* Y = Output(0, X.sizes(), at::dtype<float>());

  int64_t num_func_per_group;
  int64_t num_group;

  setUpTensors(num_func_per_group, num_group, M);

  PieceWiseLinearTransformGeneralKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N,
      M,
      num_group,
      num_func_per_group,
      bounds_device_.data<float>(),
      slopes_device_.data<float>(),
      intercepts_device_.data<float>(),
      X.data<float>(),
      Y->template mutable_data<float>());

  return true;
}

template <>
bool PiecewiseLinearTransformOp<float, HIPContext>::TransformBinary() {
  auto& X = Input(0);

  CAFFE_ENFORCE(X.dim() == 1 || X.dim() == 2);
  int64_t N = X.dim32(0);
  int64_t M = X.dim() == 2 ? X.dim32(1) : 1;
  CAFFE_ENFORCE(
      M == 1 || M == 2,
      "If binary is set to true, the input must be Nx2 or Nx1 tensor");
  auto* Y = Output(0, X.sizes(), at::dtype<float>());

  int64_t num_func_per_group;
  int64_t num_group;

  setUpTensors(num_func_per_group, num_group, M);

  if (M == 1) {
    PieceWiseLinearTransformBinaryKernel1<<<
        CAFFE_GET_BLOCKS(X.numel()),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        N,
        M,
        num_group,
        num_func_per_group,
        bounds_device_.data<float>(),
        slopes_device_.data<float>(),
        intercepts_device_.data<float>(),
        X.data<float>(),
        Y->template mutable_data<float>());
  } else {
    // don't want N*M threads, only N*M/2
    PieceWiseLinearTransformBinaryKernel2<<<
        CAFFE_GET_BLOCKS(X.numel() / 2),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        N,
        M,
        num_group,
        num_func_per_group,
        bounds_device_.data<float>(),
        slopes_device_.data<float>(),
        intercepts_device_.data<float>(),
        X.data<float>(),
        Y->template mutable_data<float>());
  }

  return true;
}

REGISTER_HIP_OPERATOR(
    PiecewiseLinearTransform,
    PiecewiseLinearTransformOp<float, HIPContext>);

} // namespace caffe2

using PiecewiseLinearTransformOpFloatCUDA =
    caffe2::PiecewiseLinearTransformOp<float, caffe2::HIPContext>;

C10_EXPORT_CAFFE2_OP_TO_C10_CUDA(
    PiecewiseLinearTransform,
    PiecewiseLinearTransformOpFloatCUDA);
