#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/floor_op.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

template <typename T>
__global__ void FloorKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = std::floor(X[i]);
  }
}

template <>
bool FloorOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  CAFFE_ENFORCE_GT(X.numel(), 0);
  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  FloorKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.numel(), X.data<float>(), Y->template mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(Floor, FloorOp<float, HIPContext>);

} // namespace caffe2
