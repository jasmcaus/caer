#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/replace_nan_op.h"

namespace caffe2 {

namespace {
template <typename T>
__global__ void
replace_nan_kernel(const T value, const int64_t size, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, size) {
    if (isnan(X[i])) {
      Y[i] = value;
    } else {
      Y[i] = X[i];
    }
  }
}
} // namespace

template <>
template <typename T>
void ReplaceNaNOp<HIPContext>::ReplaceNaN(
    const T& value,
    const int64_t size,
    const T* X,
    T* Y) {
  replace_nan_kernel<<<
      CAFFE_GET_BLOCKS(size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(value, size, X, Y);
}
REGISTER_HIP_OPERATOR(ReplaceNaN, ReplaceNaNOp<HIPContext>);
} // namespace caffe2
