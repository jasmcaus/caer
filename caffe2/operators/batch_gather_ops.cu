#include "hip/hip_runtime.h"
#include <fstream>
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/batch_gather_ops.h"
// Shared batch kernel
#include "caffe2/operators/gather_op.cuh"

namespace caffe2 {

template <>
bool BatchGatherOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
      this, OperatorBase::Input<Tensor>(INDICES, CUDA));
}

template <>
template <typename TInd>
bool BatchGatherOp<HIPContext>::DoRunWithType() {
  // BatchGather is a special-case of Gather with Axis = 1, wrap = false.
  return gather_helper::gather_impl_cuda<TInd>(
      this, DATA, INDICES, 0, 1, false, match_outer_);
}

template <typename T_INDEX, typename TData>
__global__ void BatchGatherGradientKernel(
    const TData* grad_data,
    TData* out,
    const T_INDEX* indices,
    const int outer_dims_product,
    const int N,
    const int data_batch_size,
    const int gathered_batch_size,
    const int block_size,
    const int src_indexing_axis_dim,
    const bool wrap_indices) {
  int begin_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int num_items = outer_dims_product * N * block_size;

  for (int s = begin_idx; s < num_items; s += blockDim.x * gridDim.x) {
    const int k = s % block_size;
    const int j = s / block_size % N;
    const int i = s / block_size / N;
    T_INDEX idx = indices[j];
    if (wrap_indices && idx < 0) {
      idx = idx + src_indexing_axis_dim;
    }
    const float* src_offset =
        grad_data + i * gathered_batch_size + j * block_size;
    float* dst_offset = out + i * data_batch_size + idx * block_size;
    atomicAdd(dst_offset + k, src_offset[k]);
  }
}

template <>
bool BatchGatherGradientOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
      this, OperatorBase::Input<Tensor>(INDICES, CUDA));
}

template <>
template <typename TInd>
bool BatchGatherGradientOp<HIPContext>::DoRunWithType() {
  return DispatchHelper<
      TensorTypes2<float, GenericTensorImplementation>,
      TInd>::call(this, OperatorBase::Input<Tensor>(DATA, CUDA));
}

template <>
template <typename TInd, typename TData>
bool BatchGatherGradientOp<HIPContext>::DoRunWithType2() {
  CAFFE_ENFORCE(
      !match_outer_, "match_outer=true is currently only supported for CPU");

  auto& data = Input(DATA);
  auto& indices = Input(INDICES);
  auto& grad = Input(GRAD);

  // ONNX allows negative axis to index from the back, valid range: [-r, r].
  int axis = axis_;
  if (axis < 0) {
    axis = data.dim() + axis;
  }
  // Outer dimensions of input data and gradient should be the same
  // because they are preserved for gathers with axis > 0.
  for (int acheck = 0; acheck < axis; acheck++) {
    CAFFE_ENFORCE_EQ(
        data.size(acheck), grad.size(acheck), "batch sizes should be the same");
  }

  auto* output = Output(0, data.sizes(), at::dtype<float>());
  auto* out_data = output->template mutable_data<float>();
  math::Set<float, HIPContext>(output->numel(), 0, out_data, &context_);

  const auto* grad_data = grad.template data<float>();
  const TInd* idxs = indices.template data<TInd>();

  // Treat all outer dimensions as a unit as they contribute to larger batch.
  const int outer_dims_product = grad.size_to_dim(axis);
  const int block_size = data.size_from_dim(axis + 1);

  const int N = indices.numel();
  const auto data_batch_size = data.size_from_dim(axis);
  const auto gathered_batch_size = N * block_size;
  const int src_indexing_axis_dim = data.dim(axis);

  // Assign each thread index its own 'float' in block_size * N (kernel will
  // loop if there is more data than fits NUM_BLOCKS * NUM_THREADS limit).
  BatchGatherGradientKernel<<<
      std::min(outer_dims_product, CAFFE_MAXIMUM_NUM_BLOCKS),
      std::min(N * block_size, CAFFE_CUDA_NUM_THREADS),
      0,
      context_.hip_stream()>>>(
      grad_data,
      out_data,
      idxs,
      outer_dims_product,
      N,
      data_batch_size,
      gathered_batch_size,
      block_size,
      src_indexing_axis_dim,
      false); // TBD: Add proper index wrapping support to Gather gradients.

  return true;
}

template <>
template <typename TInd>
bool BatchGatherGradientOp<HIPContext>::DoRunWithOtherType2() {
  CAFFE_THROW(
      "BatchGatherGradient is not implemented on tensor of type ",
      Input(DATA).meta().name(),
      "consider adding it as a type in the DispatchHelper list or implementing"
      " a generic version (which won't work for duplicated indices though)");
}

REGISTER_HIP_OPERATOR(BatchGather, BatchGatherOp<HIPContext>);
REGISTER_HIP_OPERATOR(BatchGatherGradient, BatchGatherGradientOp<HIPContext>);

} // namespace caffe2
