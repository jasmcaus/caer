#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/operator_fallback_gpu.h"
#include "caffe2/operators/sparse_lp_regularizer_op.h"

namespace caffe2 {
REGISTER_HIP_OPERATOR(SparseLpRegularizer, GPUFallbackOp);
}
