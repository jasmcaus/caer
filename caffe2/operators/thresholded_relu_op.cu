#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/thresholded_relu_op.h"

namespace caffe2 {
namespace {
template <typename T>
__global__ void ThresholdedReluKernel(const int N, const T* X, T* Y, T alpha_) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = X[i] > alpha_ ? X[i] : 0;
  }
}

template <typename T>
__global__ void
ThresholdedReluGradientKernel(const int N, const T* Y, const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = Y[i] > 0 ? dY[i] : 0;
  }
}
} // namespace

template <>
bool ThresholdedReluOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  CAFFE_ENFORCE_GT(X.numel(), 0);
  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  ThresholdedReluKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.numel(), X.data<float>(), Y->template mutable_data<float>(), alpha_);
  return true;
}

template <>
bool ThresholdedReluGradientOp<float, HIPContext>::RunOnDevice() {
  auto& Y = Input(0);
  auto& dY = Input(1);

  CAFFE_ENFORCE_GT(Y.numel(), 0);
  CAFFE_ENFORCE_EQ(dY.numel(), Y.numel());
  auto* dX = Output(0, Y.sizes(), at::dtype<float>());
  ThresholdedReluGradientKernel<<<
      CAFFE_GET_BLOCKS(Y.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      Y.numel(),
      Y.data<float>(),
      dY.data<float>(),
      dX->template mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(ThresholdedRelu, ThresholdedReluOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    ThresholdedReluGradient,
    ThresholdedReluGradientOp<float, HIPContext>);
} // namespace caffe2
