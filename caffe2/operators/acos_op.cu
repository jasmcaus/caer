#include "hip/hip_runtime.h"
#include "caffe2/operators/acos_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

__global__ void AcosGradientCUDAKernel(
    const int N,
    const float* dY,
    const float* X,
    float* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    dX[i] = -__ldg(dY + i) * rsqrtf(1.0f - __ldg(X + i) * __ldg(X + i));
#else
    dX[i] = -dY[i] * rsqrtf(1.0f - X[i] * X[i]);
#endif
  }
}

} // namespace

template <>
template <typename T>
bool AcosGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& X_dims,
    const std::vector<int>& /* dY_dims */,
    const T* X,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      X_dims.cbegin(), X_dims.cend(), 1, std::multiplies<int>());
  AcosGradientCUDAKernel<<<
      CAFFE_GET_BLOCKS(size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(size, dY, X, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    Acos,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        AcosFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AcosGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        AcosGradientFunctor<HIPContext>>);

} // namespace caffe2
