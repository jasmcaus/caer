#include "hip/hip_runtime.h"
#include <cmath>
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/filler_op.h"
#include "caffe2/operators/operator_fallback_gpu.h"

namespace caffe2 {

namespace {
__global__ void FillRangeKernel(const int n, float* data) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    data[index] = index;
  }
}

template <typename T>
__global__ void FillDiagonalKernel(
    const int num_diagonal_elements,
    const int64_t step_size,
    const T value,
    T* data) {
  CUDA_1D_KERNEL_LOOP(index, num_diagonal_elements) {
    data[index * step_size] = value;
  }
}
}

template <>
bool RangeFillOp<float, HIPContext>::Fill(Tensor* output) {
  int N = output->numel();
  FillRangeKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(N, output->template mutable_data<float>());
  return true;
}

template <>
template <typename T>
bool DiagonalFillOp<HIPContext>::FillWithType(Tensor* output) {
  VerifyOutputShape(output);
  auto* data = output->template mutable_data<T>();
  int size = output->numel();
  // first fill everything with 0
  math::Set<T, HIPContext>(size, T(0), data, &context_);

  T value = OperatorBase::GetSingleArgument<T>("value", 0);
  int64_t step_size = GetStepSize(output);
  int num_diagonal_elements = ceil((float)size / step_size);

  FillDiagonalKernel<<<
      CAFFE_GET_BLOCKS(num_diagonal_elements),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(num_diagonal_elements, step_size, value, data);
  return true;
}

REGISTER_HIP_OPERATOR(UniformFill, UniformFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(UniformIntFill, UniformFillOp<int, HIPContext>);
REGISTER_HIP_OPERATOR(ConstantFill, ConstantFillOp<HIPContext>);
REGISTER_HIP_OPERATOR(DiagonalFill, DiagonalFillOp<HIPContext>);
REGISTER_HIP_OPERATOR(GaussianFill, GaussianFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(XavierFill, XavierFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MSRAFill, MSRAFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(RangeFill, RangeFillOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(LengthsRangeFill, GPUFallbackOp);

} // namespace caffe2
