#include "hip/hip_runtime.h"
#include "caffe2/operators/relu_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

#ifdef __HIP_PLATFORM_HCC__
#include <hip/hip_version.h>
#endif

namespace caffe2 {

namespace {

#ifdef __HIPCC__
using half2 = __half2;
#endif // __HIPCC__

template <typename T>
__global__ void ReluCUDAKernel(const int N, const T* X, T* Y);

#define DELEGATE_RELU_CUDA_KERNEL(T, MaxFunc)                        \
  template <>                                                        \
  __global__ void ReluCUDAKernel<T>(const int N, const T* X, T* Y) { \
    const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x; \
    if (i < N) {                                                     \
      Y[i] = MaxFunc(X[i], T(0));                                    \
    }                                                                \
  }
DELEGATE_RELU_CUDA_KERNEL(float, fmaxf)
#undef DELEGATE_RELU_CUDA_KERNEL

template <>
__global__ void ReluCUDAKernel<half>(const int N, const half* X, half* Y) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
    const half kZero = __float2half(0.0f);
#if __CUDA_ARCH__ >= 530 || HIP_VERSION >= 300
    Y[i] = __hgt(__ldg(X + i), kZero) ? __ldg(X + i) : kZero;
#else
    Y[i] = (__half2float(X[i]) > 0) ? X[i] : kZero;
#endif
  }
}

template <>
__global__ void ReluCUDAKernel<half2>(const int N, const half2* X, half2* Y) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
    const half2 kZero = __float2half2_rn(0.0f);
#if __CUDA_ARCH__ >= 530 || HIP_VERSION >= 300
    Y[i] = __hmul2(__hgt2(__ldg(X + i), kZero), __ldg(X + i));
#else
    const float2 xx = __half22float2(X[i]);
    // There are explicit cast to float here, because it may otherwise cause ambiguity on ROCm and can be triggered
    // sometimes:
    //
    //   error: conditional expression is ambiguous; 'const hip_impl::Scalar_accessor<float, Native_vec_, 0>' can be
    //   converted to 'float' and vice versa
    Y[i] = __floats2half2_rn(xx.x > 0.0f ? static_cast<float>(xx.x) : 0.0f,
                             xx.y > 0.0f ? static_cast<float>(xx.y) : 0.0f);
#endif
  }
}

template <typename T>
__global__ void
ReluGradientCUDAKernel(const int N, const T* dY, const T* Y, T* dX) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
#if __CUDA_ARCH__ >= 350 || HIP_VERSION >= 300
    dX[i] = __ldg(Y + i) > T(0) ? __ldg(dY + i) : T(0);
#else
    dX[i] = Y[i] > T(0) ? dY[i] : T(0);
#endif
  }
}

template <>
__global__ void ReluGradientCUDAKernel<half>(
    const int N,
    const half* dY,
    const half* Y,
    half* dX) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
    const half kZero = __float2half(0.0f);
#if __CUDA_ARCH__ >= 530 || HIP_VERSION >= 300
    dX[i] = __hgt(__ldg(Y + i), kZero) ? __ldg(dY + i) : kZero;
#else
    dX[i] = (__half2float(Y[i]) > 0) ? dY[i] : kZero;
#endif
  }
}

template <>
__global__ void ReluGradientCUDAKernel<half2>(
    const int N,
    const half2* dY,
    const half2* Y,
    half2* dX) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
    const half2 kZero = __float2half2_rn(0.0f);
#if __CUDA_ARCH__ >= 530 || HIP_VERSION >= 300
    dX[i] = __hmul2(__hgt2(__ldg(Y + i), kZero), __ldg(dY + i));
#else
    const float2 dy = __half22float2(dY[i]);
    const float2 yy = __half22float2(Y[i]);
    // There are explicit cast to float here, because it may otherwise cause ambiguity on ROCm and can be triggered
    // sometimes:
    //
    //   error: conditional expression is ambiguous; 'const hip_impl::Scalar_accessor<float, Native_vec_, 1>' can be
    //   converted to 'float' and vice versa

     dX[i] = __floats2half2_rn(yy.x > 0.0f ? static_cast<float>(dy.x) : 0.0f,
                               yy.y > 0.0f ? static_cast<float>(dy.y) : 0.0f);
#endif
  }
}

} // namespace

template <>
template <typename T>
bool ReluFunctor<HIPContext>::
operator()(const int N, const T* X, T* Y, HIPContext* context) const {
  if (N > 0) {
    const int M = math::DivUp(N, CAFFE_CUDA_NUM_THREADS);
    ReluCUDAKernel<T>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(N, X, Y);
  }
  return true;
}

template <>
template <>
bool ReluFunctor<HIPContext>::operator()<at::Half>(
    const int N,
    const at::Half* X,
    at::Half* Y,
    HIPContext* context) const {
  if (N == 0) {
    return true;
  }
  if (N % 2 == 0) {
    const int M = math::DivUp(N / 2, CAFFE_CUDA_NUM_THREADS);
    ReluCUDAKernel<half2>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
            N / 2,
            reinterpret_cast<const half2*>(X),
            reinterpret_cast<half2*>(Y));
  } else {
    const int M = math::DivUp(N, CAFFE_CUDA_NUM_THREADS);
    ReluCUDAKernel<half>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
            N, reinterpret_cast<const half*>(X), reinterpret_cast<half*>(Y));
  }
  return true;
}

template <>
template <typename T>
bool ReluGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& Y_dims,
    const std::vector<int>& /* dY_dims */,
    const T* Y,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int N = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  if (N > 0) {
    const int M = math::DivUp(N, CAFFE_CUDA_NUM_THREADS);
    ReluGradientCUDAKernel<T>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
            N, dY, Y, dX);
  }
  return true;
}

template <>
template <>
bool ReluGradientFunctor<HIPContext>::Forward<at::Half>(
    const std::vector<int>& Y_dims,
    const std::vector<int>& /* dY_dims */,
    const at::Half* Y,
    const at::Half* dY,
    at::Half* dX,
    HIPContext* context) const {
  const int N = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  if (N == 0) {
    return true;
  }
  if (N % 2 == 0) {
    const int M = math::DivUp(N / 2, CAFFE_CUDA_NUM_THREADS);
    ReluGradientCUDAKernel<half2>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
            N / 2,
            reinterpret_cast<const half2*>(dY),
            reinterpret_cast<const half2*>(Y),
            reinterpret_cast<half2*>(dX));
  } else {
    const int M = math::DivUp(N, CAFFE_CUDA_NUM_THREADS);
    ReluGradientCUDAKernel<half>
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
            N,
            reinterpret_cast<const half*>(dY),
            reinterpret_cast<const half*>(Y),
            reinterpret_cast<half*>(dX));
  }
  return true;
}

REGISTER_HIP_OPERATOR(
    Relu,
    UnaryElementwiseOp<
        TensorTypes<float, at::Half>,
        HIPContext,
        ReluFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    ReluGradient,
    BinaryElementwiseOp<
        TensorTypes<float, at::Half>,
        HIPContext,
        ReluGradientFunctor<HIPContext>>);

} // namespace caffe2
