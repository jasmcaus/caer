#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/cosine_embedding_criterion_op.h"

namespace caffe2 {
namespace {


__global__ void CECKernel(
    const int N, const float* S, const int* Y, const float margin,
    float* output) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    output[i] = Y[i] == 1 ? (1. - S[i]) : fmaxf(0.f, S[i] - margin);
  }
}

__global__ void CECGradientKernel(
    const int N, const float* S, const int* Y, const float* dOutput,
    const float margin, float* dS) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dS[i] = dOutput[i] * (Y[i] == 1 ? -1 : static_cast<float>(S[i] >= margin));
  }
}
}  // namespace

template <>
bool CosineEmbeddingCriterionOp<HIPContext>::RunOnDevice() {
  auto& S = Input(0);
  auto& Y = Input(1);
  
  CAFFE_ENFORCE(S.numel() == Y.numel(),
                "The embedding and label should have the same size.");
  auto* output = Output(0, S.sizes(), at::dtype<float>());

  const float* Sdata = S.data<float>();
  const int* Ydata = Y.data<int>();
  float* output_data = output->template mutable_data<float>();

  CECKernel<<<CAFFE_GET_BLOCKS(S.numel()), CAFFE_CUDA_NUM_THREADS,
              0, context_.hip_stream()>>>(
      S.numel(), Sdata, Ydata, margin_, output_data);
  return true;
}

template <>
bool CosineEmbeddingCriterionGradientOp<HIPContext>::RunOnDevice() {
  auto& S = Input(0);
  auto& Y = Input(1);
  auto& dOutput = Input(2);
  

  auto* dS = Output(0, S.sizes(), at::dtype<float>());

  const float* Sdata = S.data<float>();
  const int* Ydata = Y.data<int>();
  const float* dOutput_data = dOutput.data<float>();
  float* dSdata = dS->template mutable_data<float>();
  CECGradientKernel<<<CAFFE_GET_BLOCKS(S.numel()), CAFFE_CUDA_NUM_THREADS,
                      0, context_.hip_stream()>>>(
      S.numel(), Sdata, Ydata, dOutput_data, margin_, dSdata);
  return true;
}

REGISTER_HIP_OPERATOR(
    CosineEmbeddingCriterion,
    CosineEmbeddingCriterionOp<HIPContext>);
REGISTER_HIP_OPERATOR(
    CosineEmbeddingCriterionGradient,
    CosineEmbeddingCriterionGradientOp<HIPContext>);
}  // namespace caffe2
