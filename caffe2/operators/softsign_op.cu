#include "hip/hip_runtime.h"
#include "caffe2/operators/softsign_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

using c10::cuda::compat::abs;

template <typename T>
inline __host__ __device__ T SquareCUDA(const T x) {
  return x * x;
}

template <typename T>
__global__ void SoftsignCUDAKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    Y[i] = __ldg(X + i) / (T(1) + abs(__ldg(X + i)));
#else
    Y[i] = X[i] / (T(1) + abs(X[i]));
#endif
  }
}

template <typename T>
__global__ void
SoftsignGradientCUDAKernel(const int N, const T* dY, const T* X, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    dX[i] = __ldg(dY + i) / SquareCUDA(T(1) + abs(__ldg(X + i)));
#else
    dX[i] = dY[i] / SquareCUDA(T(1) + abs(X[i]));
#endif
  }
}

} // namespace

template <>
template <typename T>
bool SoftsignFunctor<HIPContext>::
operator()(const int N, const T* X, T* Y, HIPContext* context) const {
  SoftsignCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(N, X, Y);
  return true;
}

template <>
template <typename T>
bool SoftsignGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& X_dims,
    const std::vector<int>& /* dY_dims */,
    const T* X,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      X_dims.cbegin(), X_dims.cend(), 1, std::multiplies<int>());
  SoftsignGradientCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(size, dY, X, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    Softsign,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        SoftsignFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    SoftsignGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        SoftsignGradientFunctor<HIPContext>>);

} // namespace caffe2
