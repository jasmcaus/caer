#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/accuracy_op.h"
#include "caffe2/utils/math.h"

#include <cub/block/block_reduce.cuh>

namespace caffe2 {

namespace {
__global__ void AccuracyKernel(
    const int N,
    const int D,
    const int top_k,
    const float* Xdata,
    const int* labelData,
    float* accuracy) {
  typedef hipcub::BlockReduce<int, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int correct = 0;
  for (int row = blockIdx.x; row < N; row += gridDim.x) {
    const int label = labelData[row];
    const float label_pred = Xdata[row * D + label];
    int ngt = 0;
    for (int col = threadIdx.x; col < D; col += blockDim.x) {
      const float pred = Xdata[row * D + col];
      if (pred > label_pred || (pred == label_pred && col <= label)) {
        ++ngt;
      }
    }
    ngt = BlockReduce(temp_storage).Sum(ngt);
    if (ngt <= top_k) {
      ++correct;
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    atomicAdd(accuracy, static_cast<float>(correct));
  }
}

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  *accuracy /= N;
}
}  // namespace

template <>
bool AccuracyOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(PREDICTION);
  auto& label = Input(LABEL);
  
  CAFFE_ENFORCE_EQ(X.dim(), 2);
  int N = X.dim32(0);
  int D = X.dim32(1);
  CAFFE_ENFORCE_EQ(label.dim(), 1);
  CAFFE_ENFORCE_EQ(label.dim32(0), N);
  auto* Y = Output(0, vector<int64_t>(), at::dtype<float>());
  float* Ydata = Y->template mutable_data<float>();
  math::Set<float, HIPContext>(1, 0, Ydata, &context_);
  AccuracyKernel<<<
      std::min(CAFFE_MAXIMUM_NUM_BLOCKS, N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N, D, top_k_, X.data<float>(), label.data<int>(), Ydata);
  // This is going to be executed only in one single kernel. Not very beautiful,
  // but probably we have to do this?
  AccuracyDivideKernel<<<1, 1, 0, context_.hip_stream()>>>(
      N, Ydata);
  return true;
}

REGISTER_HIP_OPERATOR(Accuracy, AccuracyOp<float, HIPContext>);
}  // namespace caffe2
