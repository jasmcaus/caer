#include "hip/hip_runtime.h"
#include "caffe2/operators/pool_op.h"

#include <array>
#include <functional>
#include <limits>
#include <numeric>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void AveragePool1DForwardNCHWCUDAKernel(
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int nc = blockIdx.x;
  const T* X_ptr = X + nc * X_size;
  T* Y_ptr = Y + nc * Y_size;
  for (int y = threadIdx.x; y < Y_size; y += blockDim.x) {
    const int x = y * stride - pad;
    const int l = max(x, 0);
    const int r = min(x + kernel, X_size);
    const T scale = T(1) / static_cast<T>(count_include_pad ? kernel : r - l);
    T sum = 0;
    for (int i = l; i < r; ++i) {
#if __CUDA_ARCH__ >= 350
      sum += __ldg(X_ptr + i);
#else
      sum += X_ptr[i];
#endif
    }
    Y_ptr[y] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool1DForwardNHWCCUDAKernel(
    const int C,
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int n = blockIdx.x / Y_size;
  const int y = blockIdx.x % Y_size;
  const int x = y * stride - pad;
  const int l = max(x, 0);
  const int r = min(x + kernel, X_size);
  const T scale = T(1) / static_cast<T>(count_include_pad ? kernel : r - l);
  const T* X_ptr = X + n * X_size * C;
  T* Y_ptr = Y + n * Y_size * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = l; i < r; ++i) {
#if __CUDA_ARCH__ >= 350
      sum += __ldg(X_ptr + i * C + c);
#else
      sum += X_ptr[i * C + c];
#endif
    }
    Y_ptr[y * C + c] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool2DForwardNCHWCUDAKernel(
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int nc = blockIdx.x / Y_H;
  const int yh = blockIdx.x % Y_H;
  const T* X_ptr = X + nc * X_HxW;
  T* Y_ptr = Y + nc * Y_HxW;
  const int xh = yh * stride_h - pad_t;
  const int t = max(xh, 0);
  const int b = min(xh + kernel_h, X_H);
  for (int yw = threadIdx.x; yw < Y_W; yw += blockDim.x) {
    const int xw = yw * stride_w - pad_l;
    const int l = max(xw, 0);
    const int r = min(xw + kernel_w, X_W);
    const T scale = T(1) /
        static_cast<T>(count_include_pad ? kernel_h * kernel_w
                                         : (b - t) * (r - l));
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
#if __CUDA_ARCH__ >= 350
        sum += __ldg(X_ptr + i * X_W + j);
#else
        sum += X_ptr[i * X_W + j];
#endif
      }
    }
    Y_ptr[yh * Y_W + yw] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool2DForwardNHWCCUDAKernel(
    const int C,
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int n = blockIdx.x / Y_HxW;
  const int y = blockIdx.x % Y_HxW;
  const int yh = y / Y_W;
  const int yw = y % Y_W;
  const int xh = yh * stride_h - pad_t;
  const int xw = yw * stride_w - pad_l;
  const int t = max(xh, 0);
  const int b = min(xh + kernel_h, X_H);
  const int l = max(xw, 0);
  const int r = min(xw + kernel_w, X_W);
  const T scale = T(1) /
      static_cast<T>(count_include_pad ? kernel_h * kernel_w
                                       : (b - t) * (r - l));
  const T* X_ptr = X + n * X_HxW * C;
  T* Y_ptr = Y + n * Y_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
#if __CUDA_ARCH__ >= 350
        sum += __ldg(X_ptr + (i * X_W + j) * C + c);
#else
        sum += X_ptr[(i * X_W + j) * C + c];
#endif
      }
    }
    Y_ptr[y * C + c] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool3DForwardNCHWCUDAKernel(
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int yy = blockIdx.x / Y_H;
  const int nc = yy / Y_D;
  const int yd = yy % Y_D;
  const int yh = blockIdx.x % Y_H;
  const T* X_ptr = X + nc * X_HxW;
  T* Y_ptr = Y + nc * Y_HxW;
  const int xd = yd * stride_d - pad_p;
  const int xh = yh * stride_h - pad_t;
  const int p = max(xd, 0);
  const int a = min(xd + kernel_d, X_D);
  const int t = max(xh, 0);
  const int b = min(xh + kernel_h, X_H);
  for (int yw = threadIdx.x; yw < Y_W; yw += blockDim.x) {
    const int xw = yw * stride_w - pad_l;
    const int l = max(xw, 0);
    const int r = min(xw + kernel_w, X_W);
    const T scale = T(1) /
        static_cast<T>(count_include_pad ? kernel_d * kernel_h * kernel_w
                                         : (a - p) * (b - t) * (r - l));
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
#if __CUDA_ARCH__ >= 350
          sum += __ldg(X_ptr + (i * X_H + j) * X_W + k);
#else
          sum += X_ptr[(i * X_H + j) * X_W + k];
#endif
        }
      }
    }
    Y_ptr[(yd * Y_H + yh) * Y_W + yw] = sum * scale;
  }
}

template <typename T>
__global__ void AveragePool3DForwardNHWCCUDAKernel(
    const int C,
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const bool count_include_pad,
    const T* X,
    T* Y) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int n = blockIdx.x / Y_HxW;
  const int y = blockIdx.x % Y_HxW;
  const int yy = y / Y_W;
  const int yd = yy / Y_H;
  const int yh = yy % Y_H;
  const int yw = y % Y_W;
  const int xd = yd * stride_d - pad_p;
  const int xh = yh * stride_h - pad_t;
  const int xw = yw * stride_w - pad_l;
  const int p = max(xd, 0);
  const int a = min(xd + kernel_d, X_D);
  const int t = max(xh, 0);
  const int b = min(xh + kernel_h, X_H);
  const int l = max(xw, 0);
  const int r = min(xw + kernel_w, X_W);
  const T scale = T(1) /
      static_cast<T>(count_include_pad ? kernel_d * kernel_h * kernel_w
                                       : (a - p) * (b - t) * (r - l));
  const T* X_ptr = X + n * X_HxW * C;
  T* Y_ptr = Y + n * Y_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
#if __CUDA_ARCH__ >= 350
          sum += __ldg(X_ptr + ((i * X_H + j) * X_W + k) * C + c);
#else
          sum += X_ptr[((i * X_H + j) * X_W + k) * C + c];
#endif
        }
      }
    }
    Y_ptr[y * C + c] = sum * scale;
  }
}

template <typename T>
__global__ void GlobalAveragePoolingBackwardNCHWCUDAKernel(
    const int K,
    const int HxW,
    const T scale,
    const T* dY,
    T* dX) {
  const int nc = blockIdx.x / K;
  const int block = blockIdx.x % K;
  const int x = threadIdx.x + block * CAFFE_CUDA_NUM_THREADS;
  if (x < HxW) {
#if __CUDA_ARCH__ >= 350
    dX[nc * HxW + x] = __ldg(dY + nc) * scale;
#else
    dX[nc * HxW + x] = dY[nc] * scale;
#endif
  }
}

template <typename T>
__global__ void GlobalAveragePoolingBackwardNHWCCUDAKernel(
    const int C,
    const int HxW,
    const T scale,
    const T* dY,
    T* dX) {
  const int n = blockIdx.x / HxW;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
#if __CUDA_ARCH__ >= 350
    dX[blockIdx.x * C + c] = __ldg(dY + n * C + c) * scale;
#else
    dX[blockIdx.x * C + c] = dY[n * C + c] * scale;
#endif
  }
}

template <typename T, bool kCountIncludePad>
__global__ void AveragePool1DBackwardNCHWCUDAKernel(
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const T* dY,
    T* dX) {
  const int nc = blockIdx.x;
  const T* dY_ptr = dY + nc * Y_size;
  T* dX_ptr = dX + nc * X_size;
  for (int x = threadIdx.x; x < X_size; x += blockDim.x) {
    const int w = x + pad;
    const int l = w < kernel ? 0 : (w - kernel) / stride + 1;
    const int r = min(w / stride + 1, Y_size);
    T sum = 0;
    for (int i = l; i < r; ++i) {
      if (kCountIncludePad) {
#if __CUDA_ARCH__ >= 350
        sum += __ldg(dY_ptr + i);
#else
        sum += dY_ptr[i];
#endif
      } else {
        const int xx = i * stride - pad;
        const int xl = max(xx, 0);
        const int xr = min(xx + kernel, X_size);
#if __CUDA_ARCH__ >= 350
        sum += __ldg(dY_ptr + i) / static_cast<T>(xr - xl);
#else
        sum += dY_ptr[i] / static_cast<T>(xr - xl);
#endif
      }
    }
    if (kCountIncludePad) {
      dX_ptr[x] = sum / static_cast<T>(kernel);
    } else {
      dX_ptr[x] = sum;
    }
  }
}

template <typename T, bool kCountIncludePad>
__global__ void AveragePool1DBackwardNHWCCUDAKernel(
    const int C,
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const T* dY,
    T* dX) {
  const int n = blockIdx.x / X_size;
  const int x = blockIdx.x % X_size;
  const int w = x + pad;
  const int l = w < kernel ? 0 : (w - kernel) / stride + 1;
  const int r = min(w / stride + 1, Y_size);
  const T scale = T(1) / static_cast<T>(kernel);
  const T* dY_ptr = dY + n * Y_size * C;
  T* dX_ptr = dX + n * X_size * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = l; i < r; ++i) {
      if (kCountIncludePad) {
#if __CUDA_ARCH__ >= 350
        sum += __ldg(dY_ptr + i * C + c);
#else
        sum += dY_ptr[i * C + c];
#endif
      } else {
        const int xx = i * stride - pad;
        const int xl = max(xx, 0);
        const int xr = min(xx + kernel, X_size);
#if __CUDA_ARCH__ >= 350
        sum += __ldg(dY_ptr + i * C + c) / static_cast<T>(xr - xl);
#else
        sum += dY_ptr[i * C + c] / static_cast<T>(xr - xl);
#endif
      }
    }
    if (kCountIncludePad) {
      dX_ptr[x * C + c] = sum * scale;
    } else {
      dX_ptr[x * C + c] = sum;
    }
  }
}

template <typename T, bool kCountIncludePad>
__global__ void AveragePool2DBackwardNCHWCUDAKernel(
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const T* dY,
    T* dX) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int nc = blockIdx.x / X_H;
  const int hh = blockIdx.x % X_H;
  const T* dY_ptr = dY + nc * Y_HxW;
  T* dX_ptr = dX + nc * X_HxW;
  const int h = hh + pad_t;
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  for (int ww = threadIdx.x; ww < X_W; ww += blockDim.x) {
    const int w = ww + pad_l;
    const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int r = min(w / stride_w + 1, Y_W);
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
        if (kCountIncludePad) {
#if __CUDA_ARCH__ >= 350
          sum += __ldg(dY_ptr + i * Y_W + j);
#else
          sum += dY_ptr[i * Y_W + j];
#endif
        } else {
          const int xh = i * stride_h - pad_t;
          const int xw = j * stride_w - pad_l;
          const int xt = max(xh, 0);
          const int xb = min(xh + kernel_h, X_H);
          const int xl = max(xw, 0);
          const int xr = min(xw + kernel_w, X_W);
#if __CUDA_ARCH__ >= 350
          sum += __ldg(dY_ptr + i * Y_W + j) /
              static_cast<T>((xb - xt) * (xr - xl));
#else
          sum += dY_ptr[i * Y_W + j] / static_cast<T>((xb - xt) * (xr - xl));
#endif
        }
      }
    }
    if (kCountIncludePad) {
      dX_ptr[hh * X_W + ww] = sum / static_cast<T>(kernel_h * kernel_w);
    } else {
      dX_ptr[hh * X_W + ww] = sum;
    }
  }
}

template <typename T, bool kCountIncludePad>
__global__ void AveragePool2DBackwardNHWCCUDAKernel(
    const int C,
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const T* dY,
    T* dX) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int n = blockIdx.x / X_HxW;
  const int x = blockIdx.x % X_HxW;
  const int h = x / X_W + pad_t;
  const int w = x % X_W + pad_l;
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
  const int r = min(w / stride_w + 1, Y_W);
  const T scale = T(1) / static_cast<T>(kernel_h * kernel_w);
  const T* dY_ptr = dY + n * Y_HxW * C;
  T* dX_ptr = dX + n * X_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
        if (kCountIncludePad) {
#if __CUDA_ARCH__ >= 350
          sum += __ldg(dY_ptr + (i * Y_W + j) * C + c);
#else
          sum += dY_ptr[(i * Y_W + j) * C + c];
#endif
        } else {
          const int xh = i * stride_h - pad_t;
          const int xw = j * stride_w - pad_l;
          const int xt = max(xh, 0);
          const int xb = min(xh + kernel_h, X_H);
          const int xl = max(xw, 0);
          const int xr = min(xw + kernel_w, X_W);
#if __CUDA_ARCH__ >= 350
          sum += __ldg(dY_ptr + (i * Y_W + j) * C + c) /
              static_cast<T>((xb - xt) * (xr - xl));
#else
          sum += dY_ptr[(i * Y_W + j) * C + c] /
              static_cast<T>((xb - xt) * (xr - xl));
#endif
        }
      }
    }
    if (kCountIncludePad) {
      dX_ptr[x * C + c] = sum * scale;
    } else {
      dX_ptr[x * C + c] = sum;
    }
  }
}

template <typename T, bool kCountIncludePad>
__global__ void AveragePool3DBackwardNCHWCUDAKernel(
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const T* dY,
    T* dX) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int xx = blockIdx.x / X_H;
  const int nc = xx / X_D;
  const int dd = xx % X_D;
  const int hh = blockIdx.x % X_H;
  const T* dY_ptr = dY + nc * Y_HxW;
  T* dX_ptr = dX + nc * X_HxW;
  const int d = dd + pad_p;
  const int h = hh + pad_t;
  const int p = d < kernel_d ? 0 : (d - kernel_d) / stride_d + 1;
  const int a = min(d / stride_d + 1, Y_D);
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  for (int ww = threadIdx.x; ww < X_W; ww += blockDim.x) {
    const int w = ww + pad_l;
    const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int r = min(w / stride_w + 1, Y_W);
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
          if (kCountIncludePad) {
#if __CUDA_ARCH__ >= 350
            sum += __ldg(dY_ptr + (i * Y_H + j) * Y_W + k);
#else
            sum += dY_ptr[(i * Y_H + j) * Y_W + k];
#endif
          } else {
            const int xd = i * stride_d - pad_p;
            const int xh = j * stride_h - pad_t;
            const int xw = k * stride_w - pad_l;
            const int xp = max(xd, 0);
            const int xa = min(xd + kernel_d, X_D);
            const int xt = max(xh, 0);
            const int xb = min(xh + kernel_h, X_H);
            const int xl = max(xw, 0);
            const int xr = min(xw + kernel_w, X_W);
#if __CUDA_ARCH__ >= 350
            sum += __ldg(dY_ptr + (i * Y_H + j) * Y_W + k) /
                static_cast<T>((xa - xp) * (xb - xt) * (xr - xl));
#else
            sum += dY_ptr[(i * Y_H + j) * Y_W + k] /
                static_cast<T>((xa - xp) * (xb - xt) * (xr - xl));
#endif
          }
        }
      }
    }
    if (kCountIncludePad) {
      dX_ptr[(dd * X_H + hh) * X_W + ww] =
          sum / static_cast<T>(kernel_d * kernel_h * kernel_w);
    } else {
      dX_ptr[(dd * X_H + hh) * X_W + ww] = sum;
    }
  }
}

template <typename T, bool kCountIncludePad>
__global__ void AveragePool3DBackwardNHWCCUDAKernel(
    const int C,
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const T* dY,
    T* dX) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int n = blockIdx.x / X_HxW;
  const int x = blockIdx.x % X_HxW;
  const int xx = x / X_W;
  const int d = xx / X_H + pad_p;
  const int h = xx % X_H + pad_t;
  const int w = x % X_W + pad_l;
  const int p = d < kernel_d ? 0 : (d - kernel_d) / stride_d + 1;
  const int a = min(d / stride_d + 1, Y_D);
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
  const int r = min(w / stride_w + 1, Y_W);
  const T scale = T(1) / static_cast<T>(kernel_d * kernel_h * kernel_w);
  const T* dY_ptr = dY + n * Y_HxW * C;
  T* dX_ptr = dX + n * X_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
          if (kCountIncludePad) {
#if __CUDA_ARCH__ >= 350
            sum += __ldg(dY_ptr + ((i * Y_H + j) * Y_W + k) * C + c);
#else
            sum += dY_ptr[((i * Y_H + j) * Y_W + k) * C + c];
#endif
          } else {
            const int xd = i * stride_d - pad_p;
            const int xh = j * stride_h - pad_t;
            const int xw = k * stride_w - pad_l;
            const int xp = max(xd, 0);
            const int xa = min(xd + kernel_d, X_D);
            const int xt = max(xh, 0);
            const int xb = min(xh + kernel_h, X_H);
            const int xl = max(xw, 0);
            const int xr = min(xw + kernel_w, X_W);
#if __CUDA_ARCH__ >= 350
            sum += __ldg(dY_ptr + ((i * Y_H + j) * Y_W + k) * C + c) /
                static_cast<T>((xa - xp) * (xb - xt) * (xr - xl));
#else
            sum += dY_ptr[((i * Y_H + j) * Y_W + k) * C + c] /
                static_cast<T>((xa - xp) * (xb - xt) * (xr - xl));
#endif
          }
        }
      }
    }
    if (kCountIncludePad) {
      dX_ptr[x * C + c] = sum * scale;
    } else {
      dX_ptr[x * C + c] = sum;
    }
  }
}

} // namespace

template <>
template <>
bool AveragePoolFunctor<HIPContext>::
    GlobalPoolingForward<float, StorageOrder::NCHW>(
        const int N,
        const int C,
        const int HxW,
        const float* X,
        float* Y,
        HIPContext* context) const {
  const std::array<int, 2> X_dims = {N * C, HxW};
  const std::array<int, 2> Y_dims = {N * C, 1};
  math::ReduceMean<float, HIPContext>(
      2, X_dims.data(), Y_dims.data(), 1.0f, X, Y, context);
  return true;
}

template <>
template <>
bool AveragePoolFunctor<HIPContext>::
    GlobalPoolingForward<float, StorageOrder::NHWC>(
        const int N,
        const int C,
        const int HxW,
        const float* X,
        float* Y,
        HIPContext* context) const {
  if (ones.numel() != HxW) {
    ones.Resize(HxW);
    math::Set<float, HIPContext>(
        HxW, 1.0f, ones.mutable_data<float>(), context);
  }
  math::GemmStridedBatched<float, HIPContext>(
      CblasTrans,
      CblasNoTrans,
      N,
      C,
      1,
      HxW,
      1.0f / static_cast<float>(HxW),
      X,
      HxW * C,
      ones.data<float>(),
      0,
      0.0f,
      Y,
      C,
      context);
  return true;
}

template <>
template <>
bool AveragePoolFunctor<HIPContext>::Forward<float, StorageOrder::NCHW>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* X,
    float* Y,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  switch (ndim) {
    case 1: {
      const int num_blocks = N * C;
      AveragePool1DForwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              Y_dims[0],
              kernel[0],
              stride[0],
              pads[0],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 2: {
      const int num_blocks = N * C * Y_dims[0];
      AveragePool2DForwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 3: {
      const int num_blocks = N * C * Y_dims[0] * Y_dims[1];
      AveragePool3DForwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              count_include_pad,
              X,
              Y);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
template <>
bool AveragePoolFunctor<HIPContext>::Forward<float, StorageOrder::NHWC>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* X,
    float* Y,
    HIPContext* context) const {
  // Each CUDA block handles one point, one thread per channel.
  const int ndim = X_dims.size();
  const int Y_HxW = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  switch (ndim) {
    case 1: {
      AveragePool1DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              Y_dims[0],
              kernel[0],
              stride[0],
              pads[0],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 2: {
      AveragePool2DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              count_include_pad,
              X,
              Y);
      return true;
    }
    case 3: {
      AveragePool3DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              count_include_pad,
              X,
              Y);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
template <>
bool AveragePoolFunctor<HIPContext>::
    GlobalPoolingBackward<float, StorageOrder::NCHW>(
        const int N,
        const int C,
        const int HxW,
        const float* dY,
        const float* /* X */,
        const float* /* Y */,
        float* dX,
        HIPContext* context) const {
  const float scale = 1.0f / static_cast<float>(HxW);
  const int K = (HxW + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
  GlobalAveragePoolingBackwardNCHWCUDAKernel<float>
      <<<N * C * K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          K, HxW, scale, dY, dX);
  return true;
}

template <>
template <>
bool AveragePoolFunctor<HIPContext>::
    GlobalPoolingBackward<float, StorageOrder::NHWC>(
        const int N,
        const int C,
        const int HxW,
        const float* dY,
        const float* /* X */,
        const float* /* Y */,
        float* dX,
        HIPContext* context) const {
  const float scale = 1.0f / static_cast<float>(HxW);
  GlobalAveragePoolingBackwardNHWCCUDAKernel<float>
      <<<N * HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          C, HxW, scale, dY, dX);
  return true;
}

#define DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1(                       \
    cond, Func, T, num_blocks, threads_per_block, hip_stream, ...)         \
  do {                                                                      \
    if (cond) {                                                             \
      Func<T, true>                                                         \
          <<<num_blocks, threads_per_block, 0, hip_stream>>>(__VA_ARGS__); \
    } else {                                                                \
      Func<T, false>                                                        \
          <<<num_blocks, threads_per_block, 0, hip_stream>>>(__VA_ARGS__); \
    }                                                                       \
  } while (false)

template <>
template <>
bool AveragePoolFunctor<HIPContext>::Backward<float, StorageOrder::NCHW>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* dY,
    const float* /* X */,
    const float* /* Y */,
    float* dX,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  switch (ndim) {
    case 1: {
      const int num_blocks = N * C;
      DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1(
          count_include_pad,
          AveragePool1DBackwardNCHWCUDAKernel,
          float,
          num_blocks,
          CAFFE_CUDA_NUM_THREADS,
          context->hip_stream(),
          X_dims[0],
          Y_dims[0],
          kernel[0],
          stride[0],
          pads[0],
          dY,
          dX);
      return true;
    }
    case 2: {
      const int num_blocks = N * C * X_dims[0];
      DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1(
          count_include_pad,
          AveragePool2DBackwardNCHWCUDAKernel,
          float,
          num_blocks,
          CAFFE_CUDA_NUM_THREADS,
          context->hip_stream(),
          X_dims[0],
          X_dims[1],
          Y_dims[0],
          Y_dims[1],
          kernel[0],
          kernel[1],
          stride[0],
          stride[1],
          pads[0],
          pads[1],
          dY,
          dX);
      return true;
    }
    case 3: {
      const int num_blocks = N * C * X_dims[0] * X_dims[1];
      DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1(
          count_include_pad,
          AveragePool3DBackwardNCHWCUDAKernel,
          float,
          num_blocks,
          CAFFE_CUDA_NUM_THREADS,
          context->hip_stream(),
          X_dims[0],
          X_dims[1],
          X_dims[2],
          Y_dims[0],
          Y_dims[1],
          Y_dims[2],
          kernel[0],
          kernel[1],
          kernel[2],
          stride[0],
          stride[1],
          stride[2],
          pads[0],
          pads[1],
          pads[2],
          dY,
          dX);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
template <>
bool AveragePoolFunctor<HIPContext>::Backward<float, StorageOrder::NHWC>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* dY,
    const float* /* X */,
    const float* /* Y */,
    float* dX,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  const int X_HxW = std::accumulate(
      X_dims.cbegin(), X_dims.cend(), 1, std::multiplies<int>());
  const int num_blocks = N * X_HxW;
  switch (ndim) {
    case 1: {
      DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1(
          count_include_pad,
          AveragePool1DBackwardNHWCCUDAKernel,
          float,
          num_blocks,
          CAFFE_CUDA_NUM_THREADS,
          context->hip_stream(),
          C,
          X_dims[0],
          Y_dims[0],
          kernel[0],
          stride[0],
          pads[0],
          dY,
          dX);
      return true;
    }
    case 2: {
      DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1(
          count_include_pad,
          AveragePool2DBackwardNHWCCUDAKernel,
          float,
          num_blocks,
          CAFFE_CUDA_NUM_THREADS,
          context->hip_stream(),
          C,
          X_dims[0],
          X_dims[1],
          Y_dims[0],
          Y_dims[1],
          kernel[0],
          kernel[1],
          stride[0],
          stride[1],
          pads[0],
          pads[1],
          dY,
          dX);
      return true;
    }
    case 3: {
      DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1(
          count_include_pad,
          AveragePool3DBackwardNHWCCUDAKernel,
          float,
          num_blocks,
          CAFFE_CUDA_NUM_THREADS,
          context->hip_stream(),
          C,
          X_dims[0],
          X_dims[1],
          X_dims[2],
          Y_dims[0],
          Y_dims[1],
          Y_dims[2],
          kernel[0],
          kernel[1],
          kernel[2],
          stride[0],
          stride[1],
          stride[2],
          pads[0],
          pads[1],
          pads[2],
          dY,
          dX);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

#undef DISPATCH_KERNEL_FUNCTION_BY_BOOL_WITH_TYPE_1

namespace {

template <typename T>
__global__ void MaxPool1DForwardNCHWCUDAKernel(
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const T* X,
    T* Y) {
  const int nc = blockIdx.x;
  const T* X_ptr = X + nc * X_size;
  T* Y_ptr = Y + nc * Y_size;
  for (int y = threadIdx.x; y < Y_size; y += blockDim.x) {
    const int x = y * stride;
    const int l = max(x - pad, 0);
    const int r = min(x - pad + kernel, X_size);
    T val = std::numeric_limits<T>::lowest();
    for (int i = l; i < r; ++i) {
#if __CUDA_ARCH__ >= 350
      val = max(val, __ldg(X_ptr + i));
#else
      val = max(val, X_ptr[i]);
#endif
    }
    Y_ptr[y] = val;
  }
}

template <typename T>
__global__ void MaxPool1DForwardNHWCCUDAKernel(
    const int C,
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const T* X,
    T* Y) {
  const int n = blockIdx.x / Y_size;
  const int y = blockIdx.x % Y_size;
  const int x = y * stride;
  const int l = max(x - pad, 0);
  const int r = min(x - pad + kernel, X_size);
  const T* X_ptr = X + n * X_size * C;
  T* Y_ptr = Y + n * Y_size * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T val = std::numeric_limits<T>::lowest();
    for (int i = l; i < r; ++i) {
#if __CUDA_ARCH__ >= 350
      val = max(val, __ldg(X_ptr + i * C + c));
#else
      val = max(val, X_ptr[i * C + c]);
#endif
    }
    Y_ptr[y * C + c] = val;
  }
}

template <typename T>
__global__ void MaxPool2DForwardNCHWCUDAKernel(
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const T* X,
    T* Y) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int nc = blockIdx.x / Y_H;
  const int yh = blockIdx.x % Y_H;
  const T* X_ptr = X + nc * X_HxW;
  T* Y_ptr = Y + nc * Y_HxW;
  const int xh = yh * stride_h;
  const int t = max(xh - pad_t, 0);
  const int b = min(xh - pad_t + kernel_h, X_H);
  for (int yw = threadIdx.x; yw < Y_W; yw += blockDim.x) {
    const int xw = yw * stride_w;
    const int l = max(xw - pad_l, 0);
    const int r = min(xw - pad_l + kernel_w, X_W);
    T val = std::numeric_limits<T>::lowest();
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
#if __CUDA_ARCH__ >= 350
        val = max(val, __ldg(X_ptr + i * X_W + j));
#else
        val = max(val, X_ptr[i * X_W + j]);
#endif
      }
    }
    Y_ptr[yh * Y_W + yw] = val;
  }
}

template <typename T>
__global__ void MaxPool2DForwardNHWCCUDAKernel(
    const int C,
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const T* X,
    T* Y) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int n = blockIdx.x / Y_HxW;
  const int y = blockIdx.x % Y_HxW;
  const int yh = y / Y_W;
  const int yw = y % Y_W;
  const int xh = yh * stride_h;
  const int xw = yw * stride_w;
  const int t = max(xh - pad_t, 0);
  const int b = min(xh - pad_t + kernel_h, X_H);
  const int l = max(xw - pad_l, 0);
  const int r = min(xw - pad_l + kernel_w, X_W);
  const T* X_ptr = X + n * X_HxW * C;
  T* Y_ptr = Y + n * Y_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T val = std::numeric_limits<T>::lowest();
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
#if __CUDA_ARCH__ >= 350
        val = max(val, __ldg(X_ptr + (i * X_W + j) * C + c));
#else
        val = max(val, X_ptr[(i * X_W + j) * C + c]);
#endif
      }
    }
    Y_ptr[y * C + c] = val;
  }
}

template <typename T>
__global__ void MaxPool3DForwardNCHWCUDAKernel(
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const T* X,
    T* Y) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int yy = blockIdx.x / Y_H;
  const int nc = yy / Y_D;
  const int yd = yy % Y_D;
  const int yh = blockIdx.x % Y_H;
  const T* X_ptr = X + nc * X_HxW;
  T* Y_ptr = Y + nc * Y_HxW;
  const int xd = yd * stride_d;
  const int xh = yh * stride_h;
  const int p = max(xd - pad_p, 0);
  const int a = min(xd - pad_p + kernel_d, X_D);
  const int t = max(xh - pad_t, 0);
  const int b = min(xh - pad_t + kernel_h, X_H);
  for (int yw = threadIdx.x; yw < Y_W; yw += blockDim.x) {
    const int xw = yw * stride_w;
    const int l = max(xw - pad_l, 0);
    const int r = min(xw - pad_l + kernel_w, X_W);
    T val = std::numeric_limits<T>::lowest();
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
#if __CUDA_ARCH__ >= 350
          val = max(val, __ldg(X_ptr + (i * X_H + j) * X_W + k));
#else
          val = max(val, X_ptr[(i * X_H + j) * X_W + k]);
#endif
        }
      }
    }
    Y_ptr[(yd * Y_H + yh) * Y_W + yw] = val;
  }
}

template <typename T>
__global__ void MaxPool3DForwardNHWCCUDAKernel(
    const int C,
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const T* X,
    T* Y) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int n = blockIdx.x / Y_HxW;
  const int y = blockIdx.x % Y_HxW;
  const int yy = y / Y_W;
  const int yw = y % Y_W;
  const int yh = yy % Y_H;
  const int yd = yy / Y_H;
  const int xd = yd * stride_d;
  const int xh = yh * stride_h;
  const int xw = yw * stride_w;
  const int p = max(xd - pad_p, 0);
  const int a = min(xd - pad_p + kernel_d, X_D);
  const int t = max(xh - pad_t, 0);
  const int b = min(xh - pad_t + kernel_h, X_H);
  const int l = max(xw - pad_l, 0);
  const int r = min(xw - pad_l + kernel_w, X_W);
  const T* X_ptr = X + n * X_HxW * C;
  T* Y_ptr = Y + n * Y_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T val = std::numeric_limits<T>::lowest();
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
#if __CUDA_ARCH__ >= 350
          val = max(val, __ldg(X_ptr + ((i * X_H + j) * X_W + k) * C + c));
#else
          val = max(val, X_ptr[((i * X_H + j) * X_W + k) * C + c]);
#endif
        }
      }
    }
    Y_ptr[y * C + c] = val;
  }
}

template <typename T>
__global__ void GlobalMaxPoolingBackwardNCHWCUDAKernel(
    const int K,
    const int HxW,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int nc = blockIdx.x / K;
  const int block = blockIdx.x % K;
  const int x = threadIdx.x + block * CAFFE_CUDA_NUM_THREADS;
  if (x < HxW) {
#if __CUDA_ARCH__ >= 350
    dX[nc * HxW + x] =
        (__ldg(X + nc * HxW + x) == __ldg(Y + nc)) ? __ldg(dY + nc) : T(0);
#else
    dX[nc * HxW + x] = (X[nc * HxW + x] == Y[nc]) ? dY[nc] : T(0);
#endif
  }
}

template <typename T>
__global__ void GlobalMaxPoolingBackwardNHWCCUDAKernel(
    const int C,
    const int HxW,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int n = blockIdx.x / HxW;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
#if __CUDA_ARCH__ >= 350
    dX[blockIdx.x * C + c] =
        (__ldg(X + blockIdx.x * C + c) == __ldg(Y + n * C + c))
        ? __ldg(dY + n * C + c)
        : T(0);
#else
    dX[blockIdx.x * C + c] =
        (X[blockIdx.x * C + c] == Y[n * C + c]) ? dY[n * C + c] : T(0);
#endif
  }
}

template <typename T>
__global__ void MaxPool1DBackwardNCHWCUDAKernel(
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int nc = blockIdx.x;
  const T* dY_ptr = dY + nc * Y_size;
  const T* X_ptr = X + nc * X_size;
  const T* Y_ptr = Y + nc * Y_size;
  T* dX_ptr = dX + nc * X_size;
  for (int x = threadIdx.x; x < X_size; x += blockDim.x) {
    const int w = x + pad;
    const int l = w < kernel ? 0 : (w - kernel) / stride + 1;
    const int r = min(w / stride + 1, Y_size);
    T sum = 0;
    for (int i = l; i < r; ++i) {
#if __CUDA_ARCH__ >= 350
      if (__ldg(X_ptr + x) == __ldg(Y_ptr + i)) {
        sum += __ldg(dY_ptr + i);
      }
#else
      if (X_ptr[x] == Y_ptr[i]) {
        sum += dY_ptr[i];
      }
#endif
    }
    dX_ptr[x] = sum;
  }
}

template <typename T>
__global__ void MaxPool1DBackwardNHWCCUDAKernel(
    const int C,
    const int X_size,
    const int Y_size,
    const int kernel,
    const int stride,
    const int pad,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int n = blockIdx.x / X_size;
  const int x = blockIdx.x % X_size;
  const int w = x + pad;
  const int l = w < kernel ? 0 : (w - kernel) / stride + 1;
  const int r = min(w / stride + 1, Y_size);
  const T* dY_ptr = dY + n * Y_size * C;
  const T* X_ptr = X + n * X_size * C;
  const T* Y_ptr = Y + n * Y_size * C;
  T* dX_ptr = dX + n * X_size * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = l; i < r; ++i) {
#if __CUDA_ARCH__ >= 350
      if (__ldg(X_ptr + x * C + c) == __ldg(Y_ptr + i * C + c)) {
        sum += __ldg(dY_ptr + i * C + c);
      }
#else
      if (X_ptr[x * C + c] == Y_ptr[i * C + c]) {
        sum += dY_ptr[i * C + c];
      }
#endif
    }
    dX_ptr[x * C + c] = sum;
  }
}

template <typename T>
__global__ void MaxPool2DBackwardNCHWCUDAKernel(
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int nc = blockIdx.x / X_H;
  const int xh = blockIdx.x % X_H;
  const T* dY_ptr = dY + nc * Y_HxW;
  const T* X_ptr = X + nc * X_HxW;
  const T* Y_ptr = Y + nc * Y_HxW;
  T* dX_ptr = dX + nc * X_HxW;
  const int h = xh + pad_t;
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  for (int xw = threadIdx.x; xw < X_W; xw += blockDim.x) {
    const int w = xw + pad_l;
    const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int r = min(w / stride_w + 1, Y_W);
    const int x = xh * X_W + xw;
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
        const int y = i * Y_W + j;
#if __CUDA_ARCH__ >= 350
        if (__ldg(X_ptr + x) == __ldg(Y_ptr + y)) {
          sum += __ldg(dY_ptr + y);
        }
#else
        if (X_ptr[x] == Y_ptr[y]) {
          sum += dY_ptr[y];
        }
#endif
      }
    }
    dX_ptr[x] = sum;
  }
}

template <typename T>
__global__ void MaxPool2DBackwardNHWCCUDAKernel(
    const int C,
    const int X_H,
    const int X_W,
    const int Y_H,
    const int Y_W,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int X_HxW = X_H * X_W;
  const int Y_HxW = Y_H * Y_W;
  const int n = blockIdx.x / X_HxW;
  const int x = blockIdx.x % X_HxW;
  const int h = x / X_W + pad_t;
  const int w = x % X_W + pad_l;
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
  const int r = min(w / stride_w + 1, Y_W);
  const T* dY_ptr = dY + n * Y_HxW * C;
  const T* X_ptr = X + n * X_HxW * C;
  const T* Y_ptr = Y + n * Y_HxW * C;
  T* dX_ptr = dX + n * X_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = t; i < b; ++i) {
      for (int j = l; j < r; ++j) {
        const int y = i * Y_W + j;
#if __CUDA_ARCH__ >= 350
        if (__ldg(X_ptr + x * C + c) == __ldg(Y_ptr + y * C + c)) {
          sum += __ldg(dY_ptr + y * C + c);
        }
#else
        if (X_ptr[x * C + c] == Y_ptr[y * C + c]) {
          sum += dY_ptr[y * C + c];
        }
#endif
      }
    }
    dX_ptr[x * C + c] = sum;
  }
}

template <typename T>
__global__ void MaxPool3DBackwardNCHWCUDAKernel(
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int xx = blockIdx.x / X_H;
  const int nc = xx / X_D;
  const int xd = xx % X_D;
  const int xh = blockIdx.x % X_H;
  const T* dY_ptr = dY + nc * Y_HxW;
  const T* X_ptr = X + nc * X_HxW;
  const T* Y_ptr = Y + nc * Y_HxW;
  T* dX_ptr = dX + nc * X_HxW;
  const int d = xd + pad_p;
  const int h = xh + pad_t;
  const int p = d < kernel_d ? 0 : (d - kernel_d) / stride_d + 1;
  const int a = min(d / stride_d + 1, Y_D);
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  for (int xw = threadIdx.x; xw < X_W; xw += blockDim.x) {
    const int w = xw + pad_l;
    const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
    const int r = min(w / stride_w + 1, Y_W);
    const int x = (xd * X_H + xh) * X_W + xw;
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
          const int y = (i * Y_H + j) * Y_W + k;
#if __CUDA_ARCH__ >= 350
          if (__ldg(X_ptr + x) == __ldg(Y_ptr + y)) {
            sum += __ldg(dY_ptr + y);
          }
#else
          if (X_ptr[x] == Y_ptr[y]) {
            sum += dY_ptr[y];
          }
#endif
        }
      }
    }
    dX_ptr[x] = sum;
  }
}

template <typename T>
__global__ void MaxPool3DBackwardNHWCCUDAKernel(
    const int C,
    const int X_D,
    const int X_H,
    const int X_W,
    const int Y_D,
    const int Y_H,
    const int Y_W,
    const int kernel_d,
    const int kernel_h,
    const int kernel_w,
    const int stride_d,
    const int stride_h,
    const int stride_w,
    const int pad_p,
    const int pad_t,
    const int pad_l,
    const T* dY,
    const T* X,
    const T* Y,
    T* dX) {
  const int X_HxW = X_D * X_H * X_W;
  const int Y_HxW = Y_D * Y_H * Y_W;
  const int n = blockIdx.x / X_HxW;
  const int x = blockIdx.x % X_HxW;
  const int xx = x / X_W;
  const int d = xx / X_H + pad_p;
  const int h = xx % X_H + pad_t;
  const int w = x % X_W + pad_l;
  const int p = d < kernel_d ? 0 : (d - kernel_d) / stride_d + 1;
  const int a = min(d / stride_d + 1, Y_D);
  const int t = h < kernel_h ? 0 : (h - kernel_h) / stride_h + 1;
  const int b = min(h / stride_h + 1, Y_H);
  const int l = w < kernel_w ? 0 : (w - kernel_w) / stride_w + 1;
  const int r = min(w / stride_w + 1, Y_W);
  const T* dY_ptr = dY + n * Y_HxW * C;
  const T* X_ptr = X + n * X_HxW * C;
  const T* Y_ptr = Y + n * Y_HxW * C;
  T* dX_ptr = dX + n * X_HxW * C;
  for (int c = threadIdx.x; c < C; c += blockDim.x) {
    T sum = 0;
    for (int i = p; i < a; ++i) {
      for (int j = t; j < b; ++j) {
        for (int k = l; k < r; ++k) {
          const int y = (i * Y_H + j) * Y_W + k;
#if __CUDA_ARCH__ >= 350
          if (__ldg(X_ptr + x * C + c) == __ldg(Y_ptr + y * C + c)) {
            sum += __ldg(dY_ptr + y * C + c);
          }
#else
          if (X_ptr[x * C + c] == Y_ptr[y * C + c]) {
            sum += dY_ptr[y * C + c];
          }
#endif
        }
      }
    }
    dX_ptr[x * C + c] = sum;
  }
}

} // namespace

template <>
template <>
bool MaxPoolFunctor<HIPContext>::
    GlobalPoolingForward<float, StorageOrder::NCHW>(
        const int N,
        const int C,
        const int HxW,
        const float* X,
        float* Y,
        HIPContext* context) const {
  const std::array<int, 2> X_dims = {N * C, HxW};
  const std::array<int, 2> Y_dims = {N * C, 1};
  math::ReduceMax<float, HIPContext>(
      2, X_dims.data(), Y_dims.data(), 1.0f, X, Y, context);
  return true;
}

template <>
template <>
bool MaxPoolFunctor<HIPContext>::
    GlobalPoolingForward<float, StorageOrder::NHWC>(
        const int N,
        const int C,
        const int HxW,
        const float* X,
        float* Y,
        HIPContext* context) const {
  const std::array<int, 3> X_dims = {N, HxW, C};
  const std::array<int, 3> Y_dims = {N, 1, C};
  math::ReduceMax<float, HIPContext>(
      3, X_dims.data(), Y_dims.data(), 1.0f, X, Y, context);
  return true;
}

template <>
template <>
bool MaxPoolFunctor<HIPContext>::Forward<float, StorageOrder::NCHW>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* X,
    float* Y,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  switch (ndim) {
    case 1: {
      const int num_blocks = N * C;
      MaxPool1DForwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0], Y_dims[0], kernel[0], stride[0], pads[0], X, Y);
      return true;
    }
    case 2: {
      const int num_blocks = N * C * Y_dims[0];
      MaxPool2DForwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              X,
              Y);
      return true;
    }
    case 3: {
      const int num_blocks = N * C * Y_dims[0] * Y_dims[1];
      MaxPool3DForwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              X,
              Y);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
template <>
bool MaxPoolFunctor<HIPContext>::Forward<float, StorageOrder::NHWC>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* X,
    float* Y,
    HIPContext* context) const {
  // Each CUDA block handles one point, one thread per channel.
  const int ndim = X_dims.size();
  const int Y_HxW = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  switch (ndim) {
    case 1: {
      MaxPool1DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C, X_dims[0], Y_dims[0], kernel[0], stride[0], pads[0], X, Y);
      return true;
    }
    case 2: {
      MaxPool2DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              X,
              Y);
      return true;
    }
    case 3: {
      MaxPool3DForwardNHWCCUDAKernel<float>
          <<<N * Y_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              X,
              Y);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
template <>
bool MaxPoolFunctor<HIPContext>::
    GlobalPoolingBackward<float, StorageOrder::NCHW>(
        const int N,
        const int C,
        const int HxW,
        const float* dY,
        const float* X,
        const float* Y,
        float* dX,
        HIPContext* context) const {
  const int K = (HxW + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
  GlobalMaxPoolingBackwardNCHWCUDAKernel<float>
      <<<N * C * K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          K, HxW, dY, X, Y, dX);
  return true;
}

template <>
template <>
bool MaxPoolFunctor<HIPContext>::
    GlobalPoolingBackward<float, StorageOrder::NHWC>(
        const int N,
        const int C,
        const int HxW,
        const float* dY,
        const float* X,
        const float* Y,
        float* dX,
        HIPContext* context) const {
  GlobalMaxPoolingBackwardNHWCCUDAKernel<float>
      <<<N * HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          C, HxW, dY, X, Y, dX);
  return true;
}

template <>
template <>
bool MaxPoolFunctor<HIPContext>::Backward<float, StorageOrder::NCHW>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* dY,
    const float* X,
    const float* Y,
    float* dX,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  switch (ndim) {
    case 1: {
      const int num_blocks = N * C;
      MaxPool1DBackwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              Y_dims[0],
              kernel[0],
              stride[0],
              pads[0],
              dY,
              X,
              Y,
              dX);
      return true;
    }
    case 2: {
      const int num_blocks = N * C * X_dims[0];
      MaxPool2DBackwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              dY,
              X,
              Y,
              dX);
      return true;
    }
    case 3: {
      const int num_blocks = N * C * X_dims[0] * X_dims[1];
      MaxPool3DBackwardNCHWCUDAKernel<float>
          <<<num_blocks, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              dY,
              X,
              Y,
              dX);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

template <>
template <>
bool MaxPoolFunctor<HIPContext>::Backward<float, StorageOrder::NHWC>(
    const int N,
    const int C,
    const std::vector<int>& X_dims,
    const std::vector<int>& Y_dims,
    const std::vector<int>& kernel,
    const std::vector<int>& /* dilation */,
    const std::vector<int>& stride,
    const std::vector<int>& pads,
    const float* dY,
    const float* X,
    const float* Y,
    float* dX,
    HIPContext* context) const {
  const int ndim = X_dims.size();
  const int X_HxW = std::accumulate(
      X_dims.cbegin(), X_dims.cend(), 1, std::multiplies<int>());
  switch (ndim) {
    case 1: {
      MaxPool1DBackwardNHWCCUDAKernel<float>
          <<<N * X_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              Y_dims[0],
              kernel[0],
              stride[0],
              pads[0],
              dY,
              X,
              Y,
              dX);
      return true;
    }
    case 2: {
      MaxPool2DBackwardNHWCCUDAKernel<float>
          <<<N * X_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              Y_dims[0],
              Y_dims[1],
              kernel[0],
              kernel[1],
              stride[0],
              stride[1],
              pads[0],
              pads[1],
              dY,
              X,
              Y,
              dX);
      return true;
    }
    case 3: {
      MaxPool3DBackwardNHWCCUDAKernel<float>
          <<<N * X_HxW, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
              C,
              X_dims[0],
              X_dims[1],
              X_dims[2],
              Y_dims[0],
              Y_dims[1],
              Y_dims[2],
              kernel[0],
              kernel[1],
              kernel[2],
              stride[0],
              stride[1],
              stride[2],
              pads[0],
              pads[1],
              pads[2],
              dY,
              X,
              Y,
              dX);
      return true;
    }
    default: {
      CAFFE_THROW("Unsupported pooling dim: ", ndim);
      return false;
    }
  }
}

REGISTER_HIP_OPERATOR(
    AveragePool,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePoolGradient,
    PoolGradientOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);

REGISTER_HIP_OPERATOR(
    AveragePool1D,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePool1DGradient,
    PoolGradientOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);

REGISTER_HIP_OPERATOR(
    AveragePool2D,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePool2DGradient,
    PoolGradientOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);

REGISTER_HIP_OPERATOR(
    AveragePool3D,
    PoolOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    AveragePool3DGradient,
    PoolGradientOp<float, HIPContext, AveragePoolFunctor<HIPContext>>);

REGISTER_HIP_OPERATOR(
    MaxPool,
    PoolOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    MaxPoolGradient,
    PoolGradientOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);

REGISTER_HIP_OPERATOR(
    MaxPool1D,
    PoolOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    MaxPool1DGradient,
    PoolGradientOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);

REGISTER_HIP_OPERATOR(
    MaxPool2D,
    PoolOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    MaxPool2DGradient,
    PoolGradientOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);

REGISTER_HIP_OPERATOR(
    MaxPool3D,
    PoolOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    MaxPool3DGradient,
    PoolGradientOp<float, HIPContext, MaxPoolFunctor<HIPContext>>);

} // namespace caffe2
