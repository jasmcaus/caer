#include "hip/hip_runtime.h"
#include "caffe2/operators/erf_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

__global__ void ErfGradientCUDAKernel(
    const int N,
    const float* dY,
    const float* X,
    float* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    dX[i] = 2.0f / sqrtf(PI) * expf(-powf(__ldg(X+i), 2.0f)) * __ldg(dY + i);
#else
    dX[i] = 2.0f / sqrtf(PI) * expf(-powf(X[i], 2.0f)) * dY[i];
#endif
  }
}

} // namespace

template <>
template <typename T>
bool ErfGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& X_dims,
    const std::vector<int>& /* dY_dims */,
    const T* X,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      X_dims.cbegin(), X_dims.cend(), 1, std::multiplies<int>());
  ErfGradientCUDAKernel<<<
      CAFFE_GET_BLOCKS(size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(size, dY, X, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    Erf,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        ErfFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    ErfGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        ErfGradientFunctor<HIPContext>>);

} // namespace caffe2
