#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/resize_op.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

__global__ void NearestNeighborKernel(
    const int size,
    const int num_channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const float height_scale,
    const float width_scale,
    const float* X,
    float* Y) {
  CUDA_1D_KERNEL_LOOP(index, size) {
    int indexTemp = index;
    const int w = indexTemp % output_width;
    indexTemp /= output_width;
    const int h = indexTemp % output_height;
    indexTemp /= output_height;
    const int c = indexTemp % num_channels;
    indexTemp /= num_channels;
    const int n = indexTemp;

    const int in_y = fminf(h / height_scale, input_height - 1);
    const int in_x = fminf(w / width_scale, input_width - 1);
    Y[index] =
        X[((n * num_channels + c) * input_height + in_y) * input_width + in_x];
  }
}

__global__ void NearestNeighborGradientKernel(
    const int size,
    const int num_channels,
    const int input_height,
    const int input_width,
    const int output_height,
    const int output_width,
    const float height_scale,
    const float width_scale,
    const float* dY,
    float* dX) {
  CUDA_1D_KERNEL_LOOP(index, size) {
    int indexTemp = index;
    const int x = indexTemp % input_width;
    indexTemp /= input_width;
    const int y = indexTemp % input_height;
    indexTemp /= input_height;
    const int c = indexTemp % num_channels;
    indexTemp /= num_channels;
    const int n = indexTemp;

    const int out_y = fminf(y / height_scale, output_height - 1);
    const int out_x = fminf(x / width_scale, output_width - 1);
    const int out_index =
        ((n * num_channels + c) * output_height + out_y) * output_width + out_x;
#if __CUDA_ARCH__ >= 350
    atomicAdd(dX + out_index, __ldg(dY + index));
#else
    atomicAdd(dX + out_index, *(dY + index));
#endif
  }
}

} // namespace

template <>
bool ResizeNearestOp<float, HIPContext>::RunOnDevice() {
  const auto& X = Input(0);

  const auto inputDims = X.sizes();
  CAFFE_ENFORCE_EQ(4, inputDims.size());
  const int batch_size = X.dim32(0), num_channels = X.dim32(1),
            input_height = X.dim32(2), input_width = X.dim32(3);
  if (InputSize() == 2) {
    const auto& scales = Input(1);
    CAFFE_ENFORCE_EQ(scales.dim(), 1);
    CAFFE_ENFORCE_EQ(scales.numel(), 2);
    float scales_data[2];
    context_.CopyToCPU<float>(2, scales.data<float>(), scales_data);
    height_scale_ = scales_data[0];
    width_scale_ = scales_data[1];
  }
  int output_width = input_width * width_scale_;
  int output_height = input_height * height_scale_;
  auto* Y = Output(
      0,
      {batch_size, num_channels, output_height, output_width},
      at::dtype<float>());

  const auto size = Y->numel();
  NearestNeighborKernel<<<
      CAFFE_GET_BLOCKS(size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      size,
      num_channels,
      input_height,
      input_width,
      output_height,
      output_width,
      height_scale_,
      width_scale_,
      X.data<float>(),
      Y->template mutable_data<float>());

  return true;
}

template <>
bool ResizeNearestGradientOp<float, HIPContext>::RunOnDevice() {
  const auto& dY = Input(0);
  const auto& X = Input(1);

  const auto inputDims = dY.sizes();
  CAFFE_ENFORCE_EQ(4, inputDims.size());
  const int batch_size = dY.dim32(0), num_channels = dY.dim32(1),
            input_height = dY.dim32(2), input_width = dY.dim32(3);
  int output_height = X.dim32(2);
  int output_width = X.dim32(3);
  if (InputSize() == 3) {
    const auto& scales = Input(2);
    CAFFE_ENFORCE_EQ(scales.dim(), 1);
    CAFFE_ENFORCE_EQ(scales.numel(), 2);
    float scales_data[2];
    context_.CopyToCPU<float>(2, scales.data<float>(), scales_data);
    height_scale_ = scales_data[0];
    width_scale_ = scales_data[1];
  }
  auto* dX = Output(
      0,
      {batch_size, num_channels, output_height, output_width},
      at::dtype<float>());
  math::Set<float, HIPContext>(
      dX->numel(), 0.0f, dX->template mutable_data<float>(), &context_);

  const auto size = dY.numel();
  NearestNeighborGradientKernel<<<
      CAFFE_GET_BLOCKS(size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      size,
      num_channels,
      input_height,
      input_width,
      output_height,
      output_width,
      height_scale_,
      width_scale_,
      dY.data<float>(),
      dX->template mutable_data<float>());

  return true;
}

REGISTER_HIP_OPERATOR(ResizeNearest, ResizeNearestOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    ResizeNearestGradient,
    ResizeNearestGradientOp<float, HIPContext>);
} // namespace caffe2

using ResizeNearestOpFloatCUDA =
    caffe2::ResizeNearestOp<float, caffe2::HIPContext>;

C10_EXPORT_CAFFE2_OP_TO_C10_CUDA(ResizeNearest, ResizeNearestOpFloatCUDA);
