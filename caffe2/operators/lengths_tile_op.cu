#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/lengths_tile_op.h"

namespace caffe2 {

template <typename T>
__global__ void lengthsTileKernel(
    int numElements,
    int rowSize,
    const T* input,
    T* output,
    const int32_t* inputRowOffsets) {
  CUDA_1D_KERNEL_LOOP(i, numElements) {
    auto outputRowIndex = i / rowSize;
    auto inputBlockOffset = inputRowOffsets[outputRowIndex];
    auto indexInRow = i - outputRowIndex * rowSize;
    output[i] = input[inputBlockOffset + indexInRow];
  }
}

template <>
bool LengthsTileOp<HIPContext>::RunOnDevice() {
  auto& data = Input(DATA);
  auto& lengths = Input(LENGTHS);


  CAFFE_ENFORCE_EQ(lengths.dim(), 1, "LENGTHS must be 1-D");
  CAFFE_ENFORCE_GE(data.dim(), 1, "DATA should be at least 1-D");
  CAFFE_ENFORCE_EQ(lengths.numel(), data.dim(0));

  lengths_host_.CopyFrom(lengths); // sync copy
  auto lengths_size = lengths_host_.numel();
  auto* lengths_data = lengths_host_.data<int32_t>();

  int32_t total_length = 0;
  CPUContext cpuContext;
  math::Sum<int32_t, CPUContext>(
      lengths_size, lengths_data, &total_length, &cpuContext);

  auto shape = data.sizes().vec();
  shape[0] = total_length;
  auto* output = Output(0, shape, at::dtype<float>());

  auto numElementsPerRow = data.size_from_dim(1);
  auto numElements = total_length * numElementsPerRow;
  auto numBlocks = CAFFE_GET_BLOCKS(numElements);

  ReinitializeTensor(&rowMappingHost_, {total_length}, at::dtype<int32_t>().device(CPU));
  ReinitializeTensor(&rowMappingDevice_, {total_length}, at::dtype<int32_t>().device(CPU));
  auto* rowOffsets = rowMappingHost_.mutable_data<int32_t>();
  int32_t outputRow = 0;
  for (int64_t i = 0; i < lengths_size; i++) {
    auto length = lengths_data[i];
    for (int32_t j = 0; j < length; j++) {
      rowOffsets[outputRow++] = i * numElementsPerRow;
    }
  }

  context_.CopyFromCPU<int32_t>(
      total_length,
      rowMappingHost_.data<int32_t>(),
      rowMappingDevice_.mutable_data<int32_t>());
  context_.FinishDeviceComputation();

  if (data.template IsType<float>()) {
    lengthsTileKernel<<<
        numBlocks,
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        numElements,
        numElementsPerRow,
        data.data<float>(),
        output->mutable_data<float>(),
        rowMappingDevice_.data<int32_t>());
  } else if (data.template IsType<int>()) {
    lengthsTileKernel<<<
        numBlocks,
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        numElements,
        numElementsPerRow,
        data.data<int>(),
        output->mutable_data<int>(),
        rowMappingDevice_.data<int32_t>());
  } else if (data.template IsType<int64_t>()) {
    lengthsTileKernel<<<
        numBlocks,
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        numElements,
        numElementsPerRow,
        data.data<int64_t>(),
        output->mutable_data<int64_t>(),
        rowMappingDevice_.data<int32_t>());
  } else {
    CAFFE_THROW(
        "LengthsTile operator only supports 32-bit float, int and int64_t"
        " types but input was of type ",
        data.meta().name());
  }
  return true;
}

REGISTER_HIP_OPERATOR(LengthsTile, LengthsTileOp<HIPContext>);

} // namespace caffe2
