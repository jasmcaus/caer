#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/operator_fallback_gpu.h"
#include "caffe2/operators/sparse_normalize_op.h"

namespace caffe2 {
REGISTER_HIP_OPERATOR(
    SparseNormalize,
    GPUFallbackOp);
}
