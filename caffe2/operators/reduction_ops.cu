#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/reduction_ops.h"
#include "caffe2/utils/conversions.h"

#include <hipcub/hipcub.hpp>

namespace caffe2 {

REGISTER_HIP_OPERATOR(SumElements, SumElementsOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SumElementsInt, SumElementsIntOp<int, HIPContext>);
REGISTER_HIP_OPERATOR(SumSqrElements, SumSqrElementsOp<HIPContext>);
REGISTER_HIP_OPERATOR(RowwiseMax, MaxReductionOp<float, HIPContext, true>);
REGISTER_HIP_OPERATOR(ColwiseMax, MaxReductionOp<float, HIPContext, false>);
REGISTER_HIP_OPERATOR(
    RowwiseMaxGradient,
    MaxReductionGradientOp<float, HIPContext, true>)
REGISTER_HIP_OPERATOR(
    ColwiseMaxGradient,
    MaxReductionGradientOp<float, HIPContext, false>)

REGISTER_HIP_OPERATOR(
    SumElementsGradient,
    SumElementsGradientOp<float, HIPContext>);

template <typename T>
__global__ void
SumElementsGradientKernel(bool average, const int N, const T* dY, T* dX) {
  const T value = average ? (*dY) / N : *dY;
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = value;
  }
}

__global__ void rowwise_max_gradient_kernel(
    const int batch_size,
    const int M,
    const int N,
    const float* X,
    const float* Y,
    const float* dY,
    float* dX) {
  const int input_size = M * N;
  CUDA_1D_KERNEL_LOOP(i, batch_size * M * N) {
    const int b_i = i / input_size;
    const int b_n = i / input_size / N;
    const int y_index = b_i * M + b_n;
    if (X[i] == Y[y_index]) {
      dX[i] = dY[y_index];
    } else {
      dX[i] = 0.0;
    }
  }
}

template <>
bool SumSqrElementsOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<float, at::Half>>::call(this, Input(0));
}


__global__ void colwise_max_gradient_kernel(
    const int batch_size,
    const int M,
    const int N,
    const float* X,
    const float* Y,
    const float* dY,
    float* dX) {
  const int input_size = M * N;
  CUDA_1D_KERNEL_LOOP(i, batch_size * M * N) {
    const int b_i = i / input_size;
    const int b_n = i % input_size % N;
    const int y_index = b_i * N + b_n;
    if (X[i] == Y[y_index]) {
      dX[i] = dY[y_index];
    } else {
      dX[i] = 0.0;
    }
  }
}

template <>
bool SumElementsGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& dY = Input(1);
  DCHECK_EQ(dY.numel(), 1);

  auto* dX = Output(0, X.sizes(), at::dtype<float>());
  SumElementsGradientKernel<float>
      <<<CAFFE_GET_BLOCKS(X.numel()),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          average_,
          X.numel(),
          dY.data<float>(),
          dX->template mutable_data<float>());
  return true;
}

template <typename T, class Context, bool ROWWISE>
bool MaxReductionGradientOp<T, Context, ROWWISE>::RunOnDevice() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dY = Input(2);

  auto* dX = Output(0, X.sizes(), at::dtype<T>());

  CAFFE_ENFORCE_EQ(X.dim(), 3);

  const int batch_size = X.dim32(0);
  const int M = X.dim32(1);
  const int N = X.dim32(2);

  const T* Xdata = X.template data<T>();
  const T* Ydata = Y.template data<T>();
  const T* dYdata = dY.template data<T>();
  T* dXdata = dX->template mutable_data<T>();

  const int input_size = M * N;
  if (ROWWISE) {
    rowwise_max_gradient_kernel<<<
        CAFFE_GET_BLOCKS(batch_size * input_size),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        batch_size, M, N, Xdata, Ydata, dYdata, dXdata);
  } else {
    colwise_max_gradient_kernel<<<
        CAFFE_GET_BLOCKS(batch_size * input_size),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        batch_size, M, N, Xdata, Ydata, dYdata, dXdata);
  }
  return true;
}

} // namespace caffe2
