#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/softplus_op.h"

namespace caffe2 {
namespace {
template <typename T>
__global__ void SoftplusKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = log(exp(X[i]) + 1.0f);
  }
}

template <typename T>
__global__ void
SoftplusGradientKernel(const int N, const T* Y, const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float nexpY = exp(-Y[i]);
    dX[i] = dY[i] * (1 - nexpY);
  }
}
} // namespace

template <>
bool SoftplusOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  DCHECK_GT(X.numel(), 0);
  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  SoftplusKernel<float>
      <<<CAFFE_GET_BLOCKS(X.numel()),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          X.numel(), X.data<float>(), Y->template mutable_data<float>());
  return true;
}

template <>
bool SoftplusGradientOp<float, HIPContext>::RunOnDevice() {
  auto& Y = Input(0);
  auto& dY = Input(1);

  DCHECK_GT(Y.numel(), 0);
  DCHECK_EQ(dY.numel(), Y.numel());
  auto* dX = Output(0, Y.sizes(), at::dtype<float>());
  SoftplusGradientKernel<float>
      <<<CAFFE_GET_BLOCKS(Y.numel()),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          Y.numel(),
          Y.data<float>(),
          dY.data<float>(),
          dX->template mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(Softplus, SoftplusOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SoftplusGradient,
    SoftplusGradientOp<float, HIPContext>);
} // namespace caffe2
