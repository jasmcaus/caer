#include "hip/hip_runtime.h"
#include "caffe2/operators/roi_align_op.h"

#include <stdio.h>
#include <cfloat>
#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

template <typename T>
__device__ T bilinear_interpolate(
    const T* bottom_data,
    const int height,
    const int width,
    T y,
    T x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void RoIAlignForward(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois,
    int roi_cols,
    T* top_data,
    bool continuous_coordinate) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    // RoI could have 4 or 5 columns
    const T* offset_bottom_rois = bottom_rois + n * roi_cols;
    int roi_batch_ind = 0;
    if (roi_cols == 5) {
      roi_batch_ind = offset_bottom_rois[0];
      offset_bottom_rois++;
    }

    // Do not using rounding; this implementation detail is critical
    T roi_offset = continuous_coordinate ? T(0.5) : 0;
    T roi_start_w = offset_bottom_rois[0] * spatial_scale - roi_offset;
    T roi_start_h = offset_bottom_rois[1] * spatial_scale - roi_offset;
    T roi_end_w = offset_bottom_rois[2] * spatial_scale - roi_offset;
    T roi_end_h = offset_bottom_rois[3] * spatial_scale - roi_offset;

    T roi_width = roi_end_w - roi_start_w;
    T roi_height = roi_end_h - roi_start_h;
    if (!continuous_coordinate) { // backward compatibility
      // Force malformed ROIs to be 1x1
      roi_width = c10::cuda::compat::max(roi_width, (T)1.);
      roi_height = c10::cuda::compat::max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T x = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        T val = bilinear_interpolate(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}

} // namespace

template <>
C10_EXPORT bool RoIAlignOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Input data to pool
  auto& R = Input(1); // RoIs
                      // RoI pooled data

  if (R.numel() == 0) {
    // Handle empty rois
    Output(0, {0, X.dim32(1), pooled_h_, pooled_w_}, at::dtype<float>());
    return true;
  }

  assert(sampling_ratio_ >= 0);

  auto* Y = Output(
      0, {R.dim32(0), X.dim32(1), pooled_h_, pooled_w_}, at::dtype<float>());
  int output_size = Y->numel();
  RoIAlignForward<float>
      <<<CAFFE_GET_BLOCKS(output_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          spatial_scale_,
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          pooled_h_,
          pooled_w_,
          sampling_ratio_,
          R.data<float>(),
          R.dim32(1),
          Y->mutable_data<float>(),
          aligned_);
  return true;
}

REGISTER_HIP_OPERATOR(RoIAlign, RoIAlignOp<float, HIPContext>);
} // namespace caffe2

using RoIAlignOpFloatCUDA = caffe2::RoIAlignOp<float, caffe2::HIPContext>;

C10_EXPORT_CAFFE2_OP_TO_C10_CUDA(RoIAlign, RoIAlignOpFloatCUDA);
