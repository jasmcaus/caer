#include "hip/hip_runtime.h"
#include "caffe2/operators/elementwise_ops.h"

#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>
#include <cub/device/device_reduce.cuh>

#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/conversions.h"

#ifdef __HIPCC__
#include <hip/hip_version.h>
#if HIP_VERSION < 210
// rocblas doesn't fully support fp16 yet
#define ROCBLAS_FP16 0
#endif
#endif

namespace caffe2 {

REGISTER_HIP_OPERATOR(
    Not,
    UnaryElementwiseOp<BoolTypes, HIPContext, NotFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    Sign,
    UnaryElementwiseOp<NumericTypes, HIPContext, SignFunctor<HIPContext>>);

#define REGISTER_CUDA_COMPARE_OPERATOR(Op)                    \
  REGISTER_HIP_OPERATOR(                                     \
      Op,                                                     \
      BinaryElementwiseOp<                                    \
          TensorTypes<bool, int32_t, int64_t, float, double>, \
          HIPContext,                                        \
          Op##Functor<HIPContext>,                           \
          FixedType<bool>>)

REGISTER_CUDA_COMPARE_OPERATOR(EQ);
REGISTER_CUDA_COMPARE_OPERATOR(NE);
REGISTER_CUDA_COMPARE_OPERATOR(LT);
REGISTER_CUDA_COMPARE_OPERATOR(LE);
REGISTER_CUDA_COMPARE_OPERATOR(GT);
REGISTER_CUDA_COMPARE_OPERATOR(GE);

#undef REGISTER_CUDA_COMPARE_OPERATOR

#define REGISTER_CUDA_LOGICAL_BINARY_OPERATOR(Op) \
  REGISTER_HIP_OPERATOR(                         \
      Op,                                         \
      BinaryElementwiseOp<BoolTypes, HIPContext, Op##Functor<HIPContext>>)

REGISTER_CUDA_LOGICAL_BINARY_OPERATOR(And);
REGISTER_CUDA_LOGICAL_BINARY_OPERATOR(Or);
REGISTER_CUDA_LOGICAL_BINARY_OPERATOR(Xor);

#undef REGISTER_CUDA_LOGICAL_BINARY_OPERATOR

#define REGISTER_CUDA_BITWISE_BINARY_OPERATOR(Op) \
  REGISTER_HIP_OPERATOR(                         \
      Op,                                         \
      BinaryElementwiseOp<                        \
          IntBoolTypes,                           \
          HIPContext,                            \
          Op##Functor<HIPContext>>)

REGISTER_CUDA_BITWISE_BINARY_OPERATOR(BitwiseAnd);
REGISTER_CUDA_BITWISE_BINARY_OPERATOR(BitwiseOr);
REGISTER_CUDA_BITWISE_BINARY_OPERATOR(BitwiseXor);

#undef REGISTER_CUDA_BITWISE_BINARY_OPERATOR

namespace {

template <typename T>
__global__ void
reduce_sum_like_post1(const T* g_idata, T* g_odata, int pre, int N) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  if (n >= N) {
    return;
  }

  float sum = 0.0;
  for (int i = 0; i < pre; ++i) {
    sum += convert::To<T, float>(g_idata[i * N + n]);
  }

  g_odata[n] = convert::To<float, T>(sum);
}

template <typename T>
void device_reduce(
    const T* d_in,
    T* d_out,
    int N,
    Tensor* buffer,
    HIPContext* context) {
  // Determine temporary device storage requirements
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Sum(
      NULL, temp_storage_bytes, d_in, d_out, N, context->hip_stream());

  auto buffer_size = temp_storage_bytes / sizeof(T);
  buffer_size += temp_storage_bytes % sizeof(T) != 0 ? 1 : 0;
  buffer->Resize(buffer_size);
  void* d_temp_storage = static_cast<void*>(buffer->template mutable_data<T>());
  // Run sum-reduction
  hipcub::DeviceReduce::Sum(
      d_temp_storage,
      temp_storage_bytes,
      d_in,
      d_out,
      N,
      context->hip_stream());
}

template <>
void device_reduce<at::Half>(
    const at::Half* in,
    at::Half* out,
    int N,
    Tensor* buffer,
    HIPContext* context) {
#if HIP_VERSION >= 210
  auto buffer_size = 1;

  if (buffer->numel() != buffer_size) {
    buffer->Resize(buffer_size);

    math::Set<at::Half, HIPContext>(
        N,
        convert::To<float, at::Half>(1.),
        buffer->template mutable_data<at::Half>(),
        context);
  }

  CUBLAS_ENFORCE(rocblas_hdot(
      context->cublas_handle(),
      N,
      reinterpret_cast<const rocblas_half*>(in),
      1,
      reinterpret_cast<const rocblas_half*>(buffer->data<at::Half>()),
      0,
      reinterpret_cast<rocblas_half*>(out)));
#elif HIP_VERSION < 210
   CAFFE_THROW("HIP rocblas doesn't fully support fp16 device_reduce yet.");
#else
  auto buffer_size = 1;

  if (buffer->numel() != buffer_size) {
    buffer->Resize(buffer_size);

    math::Set<at::Half, HIPContext>(
        N,
        convert::To<float, at::Half>(1.),
        buffer->template mutable_data<at::Half>(),
        context);
  }

  CUBLAS_ENFORCE(hipblasDotEx(
      context->cublas_handle(),
      N,
      in,
      HIP_R_16F,
      1,
      buffer->data<at::Half>(),
      HIP_R_16F,
      0,
      out,
      HIP_R_16F,
      HIP_R_32F));
#endif
}

template <typename T, int BLOCK_THREADS>
__global__ void
reduce_sum_like(const T* g_idata, T* g_odata, int pre, int N, int post) {
  int n = blockIdx.x;
  float sum = 0.0;
  int limit = pre * post;
  for (int i = threadIdx.x; i < limit; i += blockDim.x) {
    int curPre = i / post;
    int curPost = i % post;

    sum +=
        convert::To<T, float>(g_idata[curPre * N * post + n * post + curPost]);
  }
  // uses a shared memory reduction within block
  typedef hipcub::BlockReduce<float, BLOCK_THREADS> BlockReduceT;
  // Shared memory
  __shared__ typename BlockReduceT::TempStorage temp_storage;
  float aggregate = BlockReduceT(temp_storage).Sum(sum);
  if (threadIdx.x == 0) {
    g_odata[n] = convert::To<float, T>(aggregate);
  }
}
} // namespace

template <>
template <typename T>
bool SumReduceLikeOp<HIPContext>::DoRunWithType() {
  const auto& A = Input(0);
  const auto& B = Input(1);
  auto* C = Output(0);
  auto count = A.size();
  CAFFE_ENFORCE(&B != C, "In-place is not allowed.");
  C->ResizeLike(B);
  const T* Adata = A.template data<T>();
  auto* Cdata = C->template mutable_data<T>();

  if (C->size() == 0) {
    // output is empty, nothing to do, not even launching the CUDA kernel
    return true;
  }

  if (B.size() == 1) {
    device_reduce<T>(Adata, Cdata, count, &sum_buffer_, &context_);
  } else {
    size_t pre, n, post;
    std::tie(pre, n, post) =
        elementwise_ops_utils::ComputeLegacyBroadcastSizes(A, B, axis_);
    // because we check shape(B) \in shape(A) before,
    // post and pre cannot be 1 at same time
    if (post == 1) {
      reduce_sum_like_post1<T>
          <<<CAFFE_GET_BLOCKS(n),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context_.hip_stream()>>>(Adata, Cdata, pre, n);
    } else {
      if (post >= 128) {
        reduce_sum_like<T, 512>
            <<<n, 512, 0, context_.hip_stream()>>>(Adata, Cdata, pre, n, post);
      } else if (post >= 64) {
        reduce_sum_like<T, 128>
            <<<n, 128, 0, context_.hip_stream()>>>(Adata, Cdata, pre, n, post);
      } else if (post >= 32) {
        reduce_sum_like<T, 64>
            <<<n, 64, 0, context_.hip_stream()>>>(Adata, Cdata, pre, n, post);
      } else {
        reduce_sum_like<T, 32>
            <<<n, 32, 0, context_.hip_stream()>>>(Adata, Cdata, pre, n, post);
      }
    }
  }
  return true;
}

template <>
bool SumReduceLikeOp<HIPContext>::RunOnDevice() {
  return DispatchHelper<TensorTypes<float, at::Half>>::call(this, Input(0));
}

REGISTER_HIP_OPERATOR(SumReduceLike, SumReduceLikeOp<HIPContext>);

} // namespace caffe2
