#include "hip/hip_runtime.h"
#include "caffe2/operators/relu_n_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void
ReluNCUDAKernel(const int N, const T threshold, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    Y[i] = __ldg(X + i) > 0
        ? (__ldg(X + i) < threshold ? __ldg(X + i) : threshold)
        : T(0);
#else
    Y[i] = X[i] > 0 ? (X[i] < threshold ? X[i] : threshold) : T(0);
#endif
  }
}

template <typename T>
__global__ void ReluNGradientCUDAKernel(
    const int N,
    const T threshold,
    const T* dY,
    const T* Y,
    T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    dX[i] = (__ldg(Y + i) > 0 && __ldg(Y + i) < threshold) ? dY[i] : T(0);
#else
    dX[i] = (Y[i] > 0 && Y[i] < threshold) ? dY[i] : T(0);
#endif
  }
}

} // namespace

template <>
template <typename T>
bool ReluNFunctor<HIPContext>::
operator()(const int N, const T* X, T* Y, HIPContext* context) const {
  ReluNCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(N, n, X, Y);
  return true;
}

template <>
template <typename T>
bool ReluNGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& Y_dims,
    const std::vector<int>& /* dY_dims */,
    const T* Y,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  ReluNGradientCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(size, n, dY, Y, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    ReluN,
    UnaryElementwiseWithArgsOp<
        TensorTypes<float>,
        HIPContext,
        ReluNFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    ReluNGradient,
    BinaryElementwiseWithArgsOp<
        TensorTypes<float>,
        HIPContext,
        ReluNGradientFunctor<HIPContext>>);

} // namespace caffe2
