#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/pack_segments.h"

namespace caffe2 {

namespace {

template <typename T, typename Data_T>
__global__ void PackSegmentsKernel(
    const Data_T* data_ptr,
    const T* lengths_ptr,
    const T* lengths_cum_sum,
    const T max_length,
    const int64_t num_seq,
    const int64_t cell_size,
    Data_T padding,
    bool* presence_ptr,
    Data_T* out_ptr) {
  CUDA_1D_KERNEL_LOOP(i, num_seq * max_length * cell_size) {
    int seq = (i / cell_size) / max_length;
    int cell = (i / cell_size) % max_length;
    int offset = i % cell_size;
    if (presence_ptr && offset == 0) {
      presence_ptr[i / cell_size] = cell < lengths_ptr[seq];
    }
    if (cell >= lengths_ptr[seq]) {
      out_ptr[i] = padding;
    } else {
      int32_t idx = (lengths_cum_sum[seq] + cell) * cell_size + offset;
      out_ptr[i] = data_ptr[idx];
    }
  }
}

template <typename T, typename Data_T>
__global__ void UnpackSegmentsKernel(
    const Data_T* data_ptr,
    const T* lengths_ptr,
    const T* lengths_cum_sum,
    const T max_length,
    const int64_t num_seq,
    const int64_t cell_size,
    Data_T* out_ptr) {
  CUDA_1D_KERNEL_LOOP(i, num_seq * max_length * cell_size) {
    int seq = (i / cell_size) / max_length;
    int cell = (i / cell_size) % max_length;
    int offset = i % cell_size;
    if (cell < lengths_ptr[seq]) {
      int idx = (lengths_cum_sum[seq] + cell) * cell_size + offset;
      out_ptr[idx] = data_ptr[i];
    }
  }
}

template <typename T>
int64_t int_array_sum(
    const T* dev_array,
    int64_t num_items,
    Tensor& dev_buffer,
    Tensor& dev_sum,
    Tensor& host_sum,
    HIPContext& context) {
  // Retrieve buffer size
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Sum(
      nullptr,
      temp_storage_bytes,
      dev_array,
      dev_sum.mutable_data<int64_t>(),
      num_items,
      context.hip_stream());

  // Allocate temporary storage
  auto buffer_size = (temp_storage_bytes + sizeof(T)) / sizeof(T);
  dev_buffer.Resize(buffer_size);
  void* dev_temp_storage = static_cast<void*>(dev_buffer.mutable_data<T>());

  // Find sumimum
  hipcub::DeviceReduce::Sum(
      dev_temp_storage,
      temp_storage_bytes,
      dev_array,
      dev_sum.mutable_data<int64_t>(),
      num_items,
      context.hip_stream());

  // Copy to host
  host_sum.CopyFrom(dev_sum);
  context.FinishDeviceComputation();
  return *host_sum.data<int64_t>();
}

template <typename T>
T array_max(
    const T* dev_array,
    int64_t num_items,
    Tensor& dev_max_buffer,
    Tensor& dev_max,
    Tensor& host_max,
    HIPContext& context) {
  // Retrieve buffer size
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Max(
      nullptr,
      temp_storage_bytes,
      dev_array,
      dev_max.mutable_data<T>(),
      num_items,
      context.hip_stream());

  // Allocate temporary storage
  auto buffer_size = (temp_storage_bytes + sizeof(T)) / sizeof(T);
  dev_max_buffer.Resize(buffer_size);
  void* dev_temp_storage = static_cast<void*>(dev_max_buffer.mutable_data<T>());

  // Find maximum
  hipcub::DeviceReduce::Max(
      dev_temp_storage,
      temp_storage_bytes,
      dev_array,
      dev_max.mutable_data<T>(),
      num_items,
      context.hip_stream());

  // Copy to host
  host_max.CopyFrom(dev_max);
  context.FinishDeviceComputation();
  return *host_max.data<T>();
}

template <typename T>
void array_prefix_sum_exclusive(
    const T* dev_array,
    const int32_t num_items,
    Tensor& prefix_buffer,
    Tensor& prefix_sum,
    HIPContext& context) {
  // Retrieve buffer size
  size_t temp_storage_bytes = 0;
  prefix_sum.Resize(num_items);
  hipcub::DeviceScan::ExclusiveSum(
      nullptr,
      temp_storage_bytes,
      dev_array,
      prefix_sum.mutable_data<T>(),
      num_items,
      context.hip_stream());

  // Allocate temporary storage
  auto buffer_size = (temp_storage_bytes + sizeof(T)) / sizeof(T);
  prefix_buffer.Resize(buffer_size);
  void* dev_temp_storage = static_cast<void*>(prefix_buffer.mutable_data<T>());

  // Exclusive sum
  hipcub::DeviceScan::ExclusiveSum(
      dev_temp_storage,
      temp_storage_bytes,
      dev_array,
      prefix_sum.mutable_data<T>(),
      num_items,
      context.hip_stream());
}

} // namespace

template <>
template <typename T>
bool PackSegmentsOp<HIPContext>::DoRunWithType() {
  return DispatchHelper<TensorTypes2<char, int32_t, int64_t, float>, T>::call(
      this, Input(DATA));
}

template <>
template <typename T, typename Data_T>
bool PackSegmentsOp<HIPContext>::DoRunWithType2() {
  const auto& data = Input(DATA);
  const auto& lengths = Input(LENGTHS);
  int64_t num_seq = lengths.dim(0);
  const Data_T* data_ptr = data.data<Data_T>();
  const T* lengths_ptr = lengths.data<T>();
  auto* out = Output(0);
  Tensor* presence_mask = nullptr;
  if (return_presence_mask_) {
    presence_mask = Output(1);
  }

  CAFFE_ENFORCE_GE(data.dim(), 1, "DATA should be at least 1-D");
  CAFFE_ENFORCE_EQ(lengths.dim(), 1, "LENGTH should be 1-D");

  // Find the length of the longest sequence.
  dev_max_length_.Resize(1);
  host_max_length_.Resize(1);

  T temp = num_seq > 0 ? array_max<T>(
                             lengths_ptr,
                             num_seq,
                             dev_buffer_,
                             dev_max_length_,
                             host_max_length_,
                             context_)
                       : 0;
  if (max_length_ != -1) {
    CAFFE_ENFORCE_GE(
        max_length_,
        temp,
        "Pre-defined max_length should be greater than the real max_length");
    temp = max_length_;
  }
  const T& max_length = temp;
  // Compute prefix sum over the lengths
  array_prefix_sum_exclusive<T>(
      lengths_ptr, num_seq, dev_buffer_, dev_lengths_prefix_sum_, context_);
  bool* presence_mask_data = nullptr;
  if (return_presence_mask_) {
    std::vector<int64_t> presence_shape{lengths.numel(), max_length};
    presence_mask->Resize(presence_shape);
    presence_mask_data = presence_mask->template mutable_data<bool>();
  }

  // create output tensor
  auto shape = data.sizes().vec(); // Shape of out is batch_size x max_len x ...
  shape[0] = max_length;
  shape.insert(shape.begin(), lengths.numel());
  out->Resize(shape);
  Data_T* out_ptr = static_cast<Data_T*>(out->raw_mutable_data(data.meta()));

  // Return empty out (with the proper shape) if first dim is 0.
  if (!data.dim(0)) {
    return true;
  }

  // Do padding
  Data_T padding = out->IsType<float>() ? padding_ : 0;
  int64_t cell_size = data.numel() / data.dim(0);
  PackSegmentsKernel<<<
      CAFFE_GET_BLOCKS(num_seq * max_length * cell_size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      data_ptr,
      lengths_ptr,
      dev_lengths_prefix_sum_.data<T>(),
      max_length,
      num_seq,
      cell_size,
      padding,
      presence_mask_data,
      out_ptr);

  return true;
}

template <>
template <typename T>
bool UnpackSegmentsOp<HIPContext>::DoRunWithType() {
  return DispatchHelper<TensorTypes2<char, int32_t, int64_t, float>, T>::call(
      this, Input(DATA));
}
template <>
template <typename T, typename Data_T>
bool UnpackSegmentsOp<HIPContext>::DoRunWithType2() {
  const auto& data = Input(DATA);
  const auto& lengths = Input(LENGTHS);
  int64_t num_seq = lengths.dim(0);
  const Data_T* data_ptr = data.data<Data_T>();
  const T* lengths_ptr = lengths.data<T>();
  auto* out = Output(0);

  CAFFE_ENFORCE_GE(data.dim(), 1, "DATA should be at least 1-D");
  CAFFE_ENFORCE_EQ(lengths.dim(), 1, "LENGTH should be 1-D");
  // Compute prefix sum over the lengths
  array_prefix_sum_exclusive<T>(
      lengths_ptr, num_seq, dev_buffer_, dev_lengths_prefix_sum_, context_);

  // compute max of the lengths
  dev_max_length_.Resize(1);
  host_max_length_.Resize(1);
  T temp = num_seq > 0 ? array_max<T>(
                             lengths_ptr,
                             num_seq,
                             dev_buffer_,
                             dev_max_length_,
                             host_max_length_,
                             context_)
                       : 0;
  if (max_length_ != -1) {
    CAFFE_ENFORCE_EQ(
        max_length_,
        data.dim(1),
        "max_length should be equal to the packed segments");

    CAFFE_ENFORCE_GE(
        max_length_,
        temp,
        "Pre-defined max_length should be greater than the real max_length");

    temp = max_length_;
  }
  const T& max_length = temp;
  // compute num of cells: sum of the lengths
  dev_num_cell_.Resize(1);
  host_num_cell_.Resize(1);
  const int64_t num_cell = int_array_sum<T>(
      lengths_ptr,
      num_seq,
      dev_buffer_,
      dev_num_cell_,
      host_num_cell_,
      context_);

  // create output tensor
  auto shape = data.sizes().vec();
  CAFFE_ENFORCE_EQ(
      shape[0], lengths.dim(0), "LENGTH should match DATA in dimension 0");
  shape.erase(shape.begin());
  shape[0] = num_cell;
  out->Resize(shape);
  Data_T* out_ptr = static_cast<Data_T*>(out->raw_mutable_data(data.meta()));

  // Return empty out (with the proper shape) if any of the dimensions is 0.
  if (data.dim(0) == 0 || data.dim(1) == 0) {
    return true;
  }

  // Unpack
  int64_t cell_size = data.numel() / (data.dim(0) * data.dim(1));
  UnpackSegmentsKernel<<<
      CAFFE_GET_BLOCKS(num_seq * max_length * cell_size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      data_ptr,
      lengths_ptr,
      dev_lengths_prefix_sum_.data<T>(),
      max_length,
      num_seq,
      cell_size,
      out_ptr);
  return true;
}

REGISTER_HIP_OPERATOR(UnpackSegments, UnpackSegmentsOp<HIPContext>);
REGISTER_HIP_OPERATOR(PackSegments, PackSegmentsOp<HIPContext>);
} // namespace caffe2
