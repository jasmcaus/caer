#include "hip/hip_runtime.h"
#ifdef _MSC_VER
#define _USE_MATH_DEFINES // For M_PI
#endif // _MSC_VER
#include <cmath>

#include "caffe2/operators/roi_align_rotated_op.h"

#include <stdio.h>
#include <cfloat>
#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

template <typename T>
__device__ T bilinear_interpolate(
    const T* bottom_data,
    const int height,
    const int width,
    T y,
    T x,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    return 0;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  int y_low = (int)y;
  int x_low = (int)x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename T>
__global__ void RoIAlignRotatedForward(
    const int nthreads,
    const T* bottom_data,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois,
    T* top_data,
    bool continuous_coordinate) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not round
    T roi_offset = continuous_coordinate ? T(0.5) : 0;
    T roi_center_w = offset_bottom_rois[1] * spatial_scale - roi_offset;
    T roi_center_h = offset_bottom_rois[2] * spatial_scale - roi_offset;
    T roi_width = offset_bottom_rois[3] * spatial_scale;
    T roi_height = offset_bottom_rois[4] * spatial_scale;
    T theta = offset_bottom_rois[5] * M_PI / 180.0;

    if (!continuous_coordinate) { // backward compatibility
      // Force malformed ROIs to be 1x1
      roi_width = c10::cuda::compat::max(roi_width, (T)1.);
      roi_height = c10::cuda::compat::max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const T* offset_bottom_data =
        bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // roi_start_h and roi_start_w are computed wrt the center of RoI (x, y).
    // Appropriate translation needs to be applied after.
    T roi_start_h = -roi_height / 2.0;
    T roi_start_w = -roi_width / 2.0;
    T cosTheta = cos(theta);
    T sinTheta = sin(theta);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T yy = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T xx = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        // Rotate by theta around the center and translate
        T x = xx * cosTheta + yy * sinTheta + roi_center_w;
        T y = yy * cosTheta - xx * sinTheta + roi_center_h;

        T val = bilinear_interpolate(
            offset_bottom_data, height, width, y, x, index);
        output_val += val;
      }
    }
    output_val /= count;

    top_data[index] = output_val;
  }
}

} // namespace

template <>
C10_EXPORT bool RoIAlignRotatedOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Input data to pool
  auto& R = Input(1); // RoIs

  CAFFE_ENFORCE_EQ(order_, StorageOrder::NCHW, "RoIAlign CUDA impl needs NCHW");

  if (R.numel() == 0) {
    // Handle empty rois
    Output(
        0,
        {0, X.dim32(1), pooled_height_, pooled_width_},
        at::dtype<float>()); // RoI pooled data
    return true;
  }

  CAFFE_ENFORCE_EQ(R.dim(), 2);
  CAFFE_ENFORCE_EQ(R.dim32(1), 6);

  assert(sampling_ratio_ >= 0);

  auto* Y = Output(
      0,
      {R.dim32(0), X.dim32(1), pooled_height_, pooled_width_},
      at::dtype<float>()); // RoI pooled data

  int output_size = Y->numel();
  RoIAlignRotatedForward<float>
      <<<CAFFE_GET_BLOCKS(output_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          spatial_scale_,
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          pooled_height_,
          pooled_width_,
          sampling_ratio_,
          R.data<float>(),
          Y->mutable_data<float>(),
          aligned_);
  return true;
}

REGISTER_HIP_OPERATOR(RoIAlignRotated, RoIAlignRotatedOp<float, HIPContext>);
} // namespace caffe2

using RoIAlignRotatedOpFloatCUDA =
    caffe2::RoIAlignRotatedOp<float, caffe2::HIPContext>;

C10_EXPORT_CAFFE2_OP_TO_C10_CUDA(RoIAlignRotated, RoIAlignRotatedOpFloatCUDA);
