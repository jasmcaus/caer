#include "caffe2/operators/lengths_pad_op.h"

#include "caffe2/core/context_gpu.h"

namespace caffe2 {
REGISTER_HIP_OPERATOR(LengthsPad, LengthsPadOp<HIPContext>);
} // namespace caffe2
