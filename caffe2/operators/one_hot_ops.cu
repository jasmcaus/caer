#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/one_hot_ops.h"

namespace caffe2 {

__global__ void OneHotOpKernel(
    const int64_t batch_size,
    const int64_t index_size,
    const int64_t* indices,
    float* output) {
  CUDA_1D_KERNEL_LOOP(i, batch_size) {
    output[i * index_size + indices[i]] = 1.;
  }
}

template <>
void OneHotOp<HIPContext>::DoOneHotOp(
    int64_t batch_size,
    int64_t index_size,
    const Tensor& indices,
    Tensor* output) {
  float* output_ptr = output->template mutable_data<float>();
  math::Set<float, HIPContext>(output->numel(), 0., output_ptr, &context_);
  OneHotOpKernel<<<
      CAFFE_GET_BLOCKS(batch_size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      batch_size, index_size, indices.data<int64_t>(), output_ptr);
}

REGISTER_HIP_OPERATOR(OneHot, OneHotOp<HIPContext>);
} // namespace
