#include "hip/hip_runtime.h"
#include <cfloat>
#include <cub/block/block_reduce.cuh>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/softmax_op.h"
#include "caffe2/operators/softmax_with_loss_op.h"
#include "caffe2/operators/spatial_softmax_with_loss_op.h"

namespace caffe2 {

namespace {

__global__ void LabelCrossEntropyKernel(
    const int N,
    const int D,
    const float* logPdata,
    const int* labeldata,
    const float* weights,
    float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    CUDA_KERNEL_ASSERT(labeldata[i] >= 0 && labeldata[i] < D);
    float weight = weights ? weights[i] : 1.0;
    Ydata[i] = -logPdata[i * D + labeldata[i]] * weight;
  }
}

__global__ void LabelCrossEntropyGradientKernel(
    const int N,
    const int D,
    const float* Pdata,
    const int* labeldata,
    float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int idx = i * D + labeldata[i];
    dXdata[idx] = Pdata[idx] - 1.;
  }
}

__global__ void LabelCrossEntropyGradientKernelWeighted(
    const int N,
    const int D,
    const float* Pdata,
    const int* labeldata,
    float* dXdata,
    const float* weights) {
  CUDA_1D_KERNEL_LOOP(i, N * D) {
    int row = i / D;
    int d = i % D;
    float val = Pdata[i] - 1.0 * (d == labeldata[row]);
    float weight = weights[row];
    dXdata[i] = val * weight;
  }
}

__global__ void ProbCrossEntropyKernel(
    const int N,
    const int D,
    const float* Pdata,
    const float* labeldata,
    const float* weights,
    float* Ydata) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  for (int i = blockIdx.x; i < N; i += gridDim.x) {
    float weight = weights ? weights[i] : 1.0;
    float sum = 0.0;
    float total_prob = 0.0;
    for (int j = threadIdx.x; j < D; j += blockDim.x) {
      int idx = i * D + j;
      CUDA_KERNEL_ASSERT(labeldata[idx] >= 0);
      total_prob += labeldata[idx];
      sum += -logf(fmaxf(Pdata[idx], FLT_MIN)) * labeldata[idx] * weight;
    }
    float tot = BlockReduce(temp_storage).Sum(sum);
    __syncthreads();
    float total_prob_sum = BlockReduce(temp_storage).Sum(total_prob);
    if (threadIdx.x == 0) {
      Ydata[i] = tot;
      // Sanity check
      CUDA_KERNEL_ASSERT(fabsf(1.0 - total_prob_sum) < 1e-5f);
    }
    __syncthreads();
  }
}

__global__ void ProbCrossEntropyGradientKernel(
    const int N,
    const int D,
    const float* Pdata,
    const float* labeldata,
    float* dXdata,
    const float* weights) {
  if (weights == NULL) {
    CUDA_1D_KERNEL_LOOP(idx, N * D) {
      dXdata[idx] = Pdata[idx] - labeldata[idx];
    }
  } else {
    CUDA_1D_KERNEL_LOOP(idx, N * D) {
      dXdata[idx] = (Pdata[idx] - labeldata[idx]) * weights[idx / D];
    }
  }
}

__global__ void SpatialSoftmaxKernel(
    const int num,
    const int D,
    const int W,
    const int H,
    const float* Xdata,
    float* Pdata) {
  CUDA_1D_KERNEL_LOOP(index, num * W * H) {
    int x = index % W;
    int y = (index / W) % H;
    int i = index / W / H;

    // Subtract max on each cell for numerical reasons
    float max_val = -FLT_MAX;
    for (int c = 0; c < D; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      max_val = fmaxf(max_val, Xdata[idx]);
    }

    // Exponentiate
    float expsum = 0.0f;
    for (int c = 0; c < D; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      float expx = expf(Xdata[idx] - max_val);
      Pdata[idx] = expx;
      expsum += expx;
    }

    // Normalize
    for (int c = 0; c < D; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      Pdata[idx] /= expsum;
    }
  }
}

#define DONTCARE (-1)

__global__ void SpatialCrossEntropyLossKernel(
    const int N,
    const int D,
    const int W,
    const int H,
    const float* Pdata,
    const int* label_data,
    const float* weights,
    float* loss_data,
    float* weight_data) {
  CUDA_1D_KERNEL_LOOP(index, N * W * H) {
    int x = index % W;
    int y = (index / W) % H;
    int i = index / W / H;
    const int label = static_cast<int>(label_data[index]);

    if (label != DONTCARE) {
      CUDA_KERNEL_ASSERT(label >= 0 && label < D);
      float weight = (weights == NULL ? 1.0 : weights[index]);
      loss_data[index] =
          -logf(
              fmaxf(Pdata[i * W * H * D + label * W * H + y * W + x], 1e-20f)) *
          weight;
      weight_data[index] = weight;
    } else {
      loss_data[index] = 0;
      weight_data[index] = 0;
    }
  }
}

__global__ void SpatialSoftmaxLossGradientKernel(
    const int N,
    const int D,
    const int W,
    const int H,
    const int* label_data,
    const float* weights,
    float* dX_data,
    float* weights_) {
  CUDA_1D_KERNEL_LOOP(index, N * W * H) {
    int x = index % W;
    int y = (index / W) % H;
    int i = index / W / H;
    const int label = static_cast<int>(label_data[index]);

    if (label != DONTCARE) {
      int data_idx = i * (H * W * D) + label * (H * W) + y * W + x;
      dX_data[data_idx] -= 1.0;
      if (weights != NULL) {
        float weight = weights[index];
        for (int c = 0; c < D; ++c) {
          int data_idx = i * (H * W * D) + c * (H * W) + y * W + x;
          dX_data[data_idx] *= weight;
        }
        weights_[index] = weight;
      } else {
        weights_[index] = 1.0;
      }
    } else {
      // Ignore-label, so set all gradients for this positions
      // tp zero
      for (int c = 0; c < D; ++c) {
        int data_idx = i * (H * W * D) + c * (H * W) + y * W + x;
        dX_data[data_idx] = 0.0;
      }
      weights_[index] = 0.0;
    }
  }
}

__global__ void SoftmaxNormalizeLogsKernel(
    const int nthreads,
    const int D,
    const float* logits,
    const float* rowmax,
    const float* scales,
    float* out_log) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index / D;
    out_log[index] =
        logits[index] - rowmax[n] - logf(fmaxf(scales[n], FLT_MIN));
  }
}

__global__ void SoftmaxNormalizeKernel(
    const int nthreads,
    const int D,
    const float* probs,
    const float* scales,
    float* out) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index / D;
    out[index] = probs[index] / scales[n];
  }
}

void Softmax(
    const int N,
    const int D,
    const float* logits,
    const float* sum_multiplier,
    float* scales,
    float* rowmax,
    float* probs,
    bool log_softmax,
    HIPContext* context) {
  const int size = N * D;

  math::RowwiseMax<float, HIPContext>(N, D, logits, rowmax, context);
  // Put the intermediate result X - max(X) into Y
  context->CopySameDevice<float>(size, logits, probs);
  // Subtract the scale
  math::Gemm<float, HIPContext>(
      CblasNoTrans,
      CblasNoTrans,
      N,
      D,
      1,
      -1,
      rowmax,
      sum_multiplier,
      1,
      probs,
      context);
  // Exponentiation
  math::Exp<float, HIPContext>(size, probs, probs, context);
  // Sum exponentiated values
  math::Gemv<float, HIPContext>(
      CblasNoTrans, N, D, 1, probs, sum_multiplier, 0, scales, context);
  // Normalize
  if (!log_softmax) {
    SoftmaxNormalizeKernel<<<
        CAFFE_GET_BLOCKS(size),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context->hip_stream()>>>(size, D, probs, scales, probs);
  } else {
    SoftmaxNormalizeLogsKernel<<<
        CAFFE_GET_BLOCKS(size),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context->hip_stream()>>>(size, D, logits, rowmax, scales, probs);
  }
}

} // namespace

template <>
bool SoftmaxWithLossOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Logits
  auto& T = Input(1); // Labels / targets

  const float* weights = (InputSize() > 2 ? Input(2).data<float>() : NULL);
  const auto canonical_axis = X.canonical_axis_index(axis_);
  int N, D;
  N = X.size_to_dim(canonical_axis); // batch size
  D = X.size_from_dim(canonical_axis);

  auto* P =
      Output(0, X.sizes(), at::dtype<float>()); // Probabilities from softmax
  ReinitializeTensor(&total_weight_ptr_, {1}, at::dtype<float>().device(CUDA));
  total_weight_ptr_.Resize(1);

  if (label_prob_mode_) {
    CAFFE_ENFORCE_GE(T.dim(), 2);
    CAFFE_ENFORCE_EQ(T.size_to_dim(canonical_axis), N);
    CAFFE_ENFORCE_EQ(T.size_from_dim(canonical_axis), D);
  } else {
    if (T.dim() == canonical_axis) {
      CAFFE_ENFORCE_EQ(T.numel(), N);
    } else {
      CAFFE_ENFORCE_EQ(T.size_to_dim(canonical_axis), N);
      CAFFE_ENFORCE_EQ(T.size_from_dim(canonical_axis), 1);
    }
  }

  auto* avg_loss =
      Output(1, vector<int64_t>(), at::dtype<float>()); // Average loss
  if (!losses_.defined()) {
    losses_ = caffe2::empty({N}, at::dtype<float>().device(CUDA));
  } else if (losses_.numel() != N) {
    losses_.Resize(N);
  }

  if (!rowmax_.defined()) {
    rowmax_ = caffe2::empty({N}, at::dtype<float>().device(CUDA));
  } else if (rowmax_.numel() != N) {
    rowmax_.Resize(N);
  }

  if (!sum_multiplier_.defined()) {
    sum_multiplier_ = caffe2::empty({D}, at::dtype<float>().device(CUDA));
    math::Set<float, HIPContext>(
        D, 1.f, sum_multiplier_.mutable_data<float>(), &context_);
  } else if (sum_multiplier_.numel() != D) {
    sum_multiplier_.Resize(D);
    math::Set<float, HIPContext>(
        D, 1.f, sum_multiplier_.mutable_data<float>(), &context_);
  }

  Softmax(
      N,
      D,
      X.data<float>(),
      sum_multiplier_.data<float>(),
      losses_.mutable_data<float>(),
      rowmax_.mutable_data<float>(),
      P->template mutable_data<float>(),
      !label_prob_mode_, // logarithmic output
      &context_);
  // Compute label xent loss per example
  if (!label_prob_mode_) {
    LabelCrossEntropyKernel<<<
        CAFFE_GET_BLOCKS(N),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        N,
        D,
        P->data<float>(),
        T.data<int>(),
        weights,
        losses_.mutable_data<float>());
    // Since we had logarithmic output, we need to exponentiate
    // them again.
    math::Exp<float, HIPContext>(
        N * D, P->data<float>(), P->template mutable_data<float>(), &context_);
  } else {
    ProbCrossEntropyKernel<<<
        std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        N,
        D,
        P->data<float>(),
        T.data<float>(),
        weights,
        losses_.mutable_data<float>());
  }

  float total_weight = N;
  if (weights) {
    // Sum weights
    math::Sum<float, HIPContext>(
        N,
        weights,
        total_weight_ptr_.mutable_data<float>(),
        &context_,
        &scratch_);
    CUDA_CHECK(hipMemcpyAsync(
        &total_weight,
        total_weight_ptr_.data<float>(),
        sizeof(float),
        hipMemcpyDeviceToHost,
        context_.hip_stream()));
  }

  // Sum of all losses
  float* avg_loss_data = avg_loss->template mutable_data<float>();
  math::Sum<float, HIPContext>(
      losses_.numel(),
      losses_.data<float>(),
      avg_loss_data,
      &context_,
      &scratch_);
  // Average of input batch size
  if (total_weight > 0) {
    math::Scale<float, float, HIPContext>(
        1, scale_ / total_weight, avg_loss_data, avg_loss_data, &context_);
  }
  if (OutputSize() > 2) {
    OutputTensorAlias(2, losses_);
  }
  return true;
}

template <>
bool SpatialSoftmaxWithLossOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Logits
  auto& T = Input(1); // Labels / targets

  const float* weights = (InputSize() > 2 ? Input(2).data<float>() : NULL);
  int N, D;
  N = X.dim32(0);
  D = X.dim32(1);

  auto* P =
      Output(0, X.sizes(), at::dtype<float>()); // Probabilities from softmax
  ReinitializeTensor(&total_weight_ptr_, {1}, at::dtype<float>().device(CUDA));

  CAFFE_ENFORCE_EQ(X.dim(), 4);
  CAFFE_ENFORCE_EQ(T.dim(), 3);
  CAFFE_ENFORCE_EQ(T.dim32(0), N);

  int H = X.dim32(2);
  int W = X.dim32(3);
  if (!losses_.defined()) {
    losses_ = caffe2::empty({N * W * H}, at::dtype<float>().device(CUDA));
  } else if (losses_.numel() != N * W * H) {
    losses_.Resize(N * W * H);
  }

  if (!weights_.defined()) {
    weights_ = caffe2::empty({N * W * H}, at::dtype<float>().device(CUDA));
  } else if (weights_.numel() != N * W * H) {
    weights_.Resize(N * W * H);
  }

  const float* Xdata = X.data<float>();
  float* Pdata = P->template mutable_data<float>();

  // Softmax for each x,y location
  SpatialSoftmaxKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(N, D, W, H, Xdata, Pdata);

  // Cross entropy
  auto* avg_loss =
      Output(1, vector<int64_t>(), at::dtype<float>()); // Average loss
  float* avg_loss_data = avg_loss->template mutable_data<float>();
  math::Set<float, HIPContext>(1, 0.0f, avg_loss_data, &context_);

  const int* label_data = T.data<int>();
  math::Set<float, HIPContext>(
      1, 0.0f, total_weight_ptr_.mutable_data<float>(), &context_);

  SpatialCrossEntropyLossKernel<<<
      CAFFE_GET_BLOCKS(N * W * H),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N,
      D,
      W,
      H,
      P->data<float>(),
      label_data,
      weights,
      losses_.mutable_data<float>(),
      weights_.mutable_data<float>());

  // Somewhat awkward scalar passing from device to host
  float h_total_weight;
  math::Sum<float, HIPContext>(
      weights_.numel(),
      weights_.data<float>(),
      total_weight_ptr_.mutable_data<float>(),
      &context_,
      &scratch_);
  CUDA_CHECK(hipMemcpyAsync(
      &h_total_weight,
      total_weight_ptr_.data<float>(),
      sizeof(float),
      hipMemcpyDeviceToHost,
      context_.hip_stream()));

  math::Sum<float, HIPContext>(
      losses_.numel(),
      losses_.data<float>(),
      avg_loss_data,
      &context_,
      &scratch_);

  // Final scaling
  if (h_total_weight > 0) {
    math::Scale<float, float, HIPContext>(
        1, scale_ / h_total_weight, avg_loss_data, avg_loss_data, &context_);
  }

  return true;
}

template <>
bool SoftmaxWithLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Logits
  auto& T = Input(1); // Labels / targets
  // Input(2) is weights, if given
  auto& P = Input(InputSize() - 2); // Probabilities from softmax
  auto& d_avg_loss = Input(InputSize() - 1); // Gradient w.r.t. avg loss
  const float* weights = (InputSize() > 4 ? Input(2).data<float>() : NULL);

  Tensor* dX;
  if (only_loss_) {
    // Memory saving trick to share the buffer with the softmax output.
    // Softmax output is thus overwritten.
    dX = OutputTensorAlias(0, P);
    dX->ResizeLike(X);
  } else {
    dX = Output(0, X.sizes(), at::dtype<float>());
  }

  const auto canonical_axis = X.canonical_axis_index(axis_);
  int N, D;
  N = X.size_to_dim(canonical_axis); // batch size
  D = X.size_from_dim(canonical_axis);

  ReinitializeTensor(&total_weight_ptr_, {1}, at::dtype<float>().device(CUDA));

  if (label_prob_mode_) {
    CAFFE_ENFORCE_GE(T.dim(), 2);
    CAFFE_ENFORCE_EQ(T.size_to_dim(canonical_axis), N);
    CAFFE_ENFORCE_EQ(T.size_from_dim(canonical_axis), D);
  } else {
    if (T.dim() == canonical_axis) {
      CAFFE_ENFORCE_EQ(T.numel(), N);
    } else {
      CAFFE_ENFORCE_EQ(T.size_to_dim(canonical_axis), N);
      CAFFE_ENFORCE_EQ(T.size_from_dim(canonical_axis), 1);
    }
  }

  // Subtract 1 from labeled positions
  if (!label_prob_mode_) {
    if (weights == nullptr) {
      // Copy softmax probabilities into dX
      if (!only_loss_) {
        context_.CopySameDevice<float>(
            P.numel(), P.data<float>(), dX->template mutable_data<float>());
      }
      LabelCrossEntropyGradientKernel<<<
          CAFFE_GET_BLOCKS(N),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          N,
          D,
          P.data<float>(),
          T.data<int>(),
          dX->template mutable_data<float>());
    } else {
      // Weighted version gets the Pdata values internally
      LabelCrossEntropyGradientKernelWeighted<<<
          CAFFE_GET_BLOCKS(N * D),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          N,
          D,
          P.data<float>(),
          T.data<int>(),
          dX->template mutable_data<float>(),
          weights);
    }
  } else {
    ProbCrossEntropyGradientKernel<<<
        CAFFE_GET_BLOCKS(N * D),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        N,
        D,
        P.data<float>(),
        T.data<float>(),
        dX->template mutable_data<float>(),
        weights);
  }
  float total_weight = N;
  if (weights) {
    // Sum weights
    math::Sum<float, HIPContext>(
        N,
        weights,
        total_weight_ptr_.mutable_data<float>(),
        &context_,
        &scratch_);
    CUDA_CHECK(hipMemcpyAsync(
        &total_weight,
        total_weight_ptr_.data<float>(),
        sizeof(float),
        hipMemcpyDeviceToHost,
        context_.hip_stream()));
  }

  // Scale by d_avg_loss / N
  if (total_weight > 0) {
    math::Scale<float, float, HIPContext>(
        dX->numel(),
        scale_ / total_weight,
        dX->data<float>(),
        dX->template mutable_data<float>(),
        &context_);
  }
  math::Scale<float, float, HIPContext>(
      dX->numel(),
      d_avg_loss.data<float>(),
      dX->data<float>(),
      dX->template mutable_data<float>(),
      &context_);

  return true;
}

template <>
bool SpatialSoftmaxWithLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Logits
  auto& T = Input(1); // Labels / targets
  // Input(2) is weights, if given
  auto& P = Input(InputSize() - 2); // Probabilities from softmax
  auto& d_avg_loss = Input(InputSize() - 1); // Gradient w.r.t. avg loss
  const float* weights = (InputSize() > 4 ? Input(2).data<float>() : NULL);

  Tensor* dX;
  if (only_loss_) {
    // Memory saving trick to share the buffer with the softmax output.
    // Softmax output is thus overwritten.
    dX = OutputTensorAlias(0, P);
    dX->ResizeLike(X);
  } else {
    dX = Output(0, X.sizes(), at::dtype<float>());
  }

  const auto canonical_axis = X.canonical_axis_index(1);
  int N, D;
  N = X.dim32(0);
  D = X.dim32(1);

  ReinitializeTensor(&total_weight_ptr_, {1}, at::dtype<float>().device(CUDA));
  // Spatial mode, compute softmax for each x, y location
  CAFFE_ENFORCE_EQ(X.dim(), 4);
  CAFFE_ENFORCE_EQ(T.dim(), 3);

  int H = X.dim32(2);
  int W = X.dim32(3);
  dX->ResizeLike(X);
  if (!weights_.defined()) {
    weights_ = caffe2::empty({N * W * H}, at::dtype<float>().device(CUDA));
  } else if (weights_.numel() != N * W * H) {
    weights_.Resize(N * W * H);
  }

  const float* Pdata = P.data<float>();
  float* dX_data = dX->template mutable_data<float>();
  const int* label_data = T.data<int>();
  const float* d_avg_loss_data = d_avg_loss.data<float>();

  // Copy softmax probabilities into dX. All but the neuron
  // corresponding to the correct label has gradient equaling e(x_j)
  // which is the probability under softmax.
  context_.CopySameDevice<float>(P.numel(), Pdata, dX_data);

  math::Set<float, HIPContext>(
      1, 0.0f, total_weight_ptr_.mutable_data<float>(), &context_);

  SpatialSoftmaxLossGradientKernel<<<
      CAFFE_GET_BLOCKS(N * W * H),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N, D, W, H, label_data, weights, dX_data, weights_.mutable_data<float>());

  math::Sum<float, HIPContext>(
      weights_.numel(),
      weights_.data<float>(),
      total_weight_ptr_.mutable_data<float>(),
      &context_,
      &scratch_);

  // Somewhat awkward scalar passing from device to host
  float h_total_weight;
  CUDA_CHECK(hipMemcpyAsync(
      &h_total_weight,
      total_weight_ptr_.data<float>(),
      sizeof(float),
      hipMemcpyDeviceToHost,
      context_.hip_stream()));

  // Final scaling
  if (h_total_weight > 0) {
    math::Scale<float, float, HIPContext>(
        dX->numel(),
        scale_ / h_total_weight,
        dX->data<float>(),
        dX->template mutable_data<float>(),
        &context_);
  }
  math::Scale<float, float, HIPContext>(
      dX->numel(),
      d_avg_loss.data<float>(),
      dX->data<float>(),
      dX->template mutable_data<float>(),
      &context_);

  return true;
}

// Implementation for the CUDA context.
template <>
bool SoftmaxOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  const auto canonical_axis = X.canonical_axis_index(axis_);
  const int N = X.size_to_dim(canonical_axis);
  const int D = X.size_from_dim(canonical_axis);
  auto* P = Output(0, X.sizes(), at::dtype<float>());
  auto* P_data = P->mutable_data<float>();
  if (N == 0 || D == 0) {
    return true;
  }
  if (!sum_multiplier_.defined()) {
    sum_multiplier_ = caffe2::empty({D}, at::dtype<float>().device(CUDA));
    math::Set<float, HIPContext>(
        D, 1.f, sum_multiplier_.mutable_data<float>(), &context_);
  } else if (sum_multiplier_.numel() != D) {
    sum_multiplier_.Resize(D);
    math::Set<float, HIPContext>(
        D, 1.f, sum_multiplier_.mutable_data<float>(), &context_);
  }
  if (!scale_.defined()) {
    scale_ = caffe2::empty({N}, at::dtype<float>().device(CUDA));
  } else if (scale_.numel() != N) {
    scale_.Resize(N);
  }

  if (!rowmax_.defined()) {
    rowmax_ = caffe2::empty({N}, at::dtype<float>().device(CUDA));
  } else if (rowmax_.numel() != N) {
    rowmax_.Resize(N);
  }

  Softmax(
      N,
      D,
      X.data<float>(),
      sum_multiplier_.data<float>(),
      scale_.mutable_data<float>(),
      rowmax_.mutable_data<float>(),
      P_data,
      false,
      &context_);
  return true;
}
#define SOFTMAX_NUM_THREADS 128

// The softmax gradient kernel. This kernel has to be called with the number of
// threads per block being no more than SOFTMAX_NUM_THREADS.
namespace {
__global__ void softmax_gradient_kernel(
    const int dim,
    const float* Y,
    const float* dY,
    float* dX) {
  Y += blockIdx.x * dim;
  dY += blockIdx.x * dim;
  dX += blockIdx.x * dim;
  const int idx = threadIdx.x;
  __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
  float tmp;

  // A two-level reduction to compute the inner products.
  tmp = 0;
  for (int i = idx; i < dim; i += blockDim.x) {
    tmp += dY[i] * Y[i];
  }
  reduction_buffer[idx] = tmp;
  __syncthreads();
  if (idx == 0) {
    tmp = reduction_buffer[0];
    for (int i = 1; i < blockDim.x; ++i)
      tmp += reduction_buffer[i];
    reduction_buffer[0] = tmp;
  }
  __syncthreads();
  // Compute gradient.
  tmp = reduction_buffer[0];
  for (int i = idx; i < dim; i += blockDim.x) {
    dX[i] = Y[i] * (dY[i] - tmp);
  }
}
} // namespace

template <>
bool SoftmaxGradientOp<float, HIPContext>::RunOnDevice() {
  auto& Y = Input(0);
  auto& dY = Input(1);

  const auto canonical_axis = Y.canonical_axis_index(axis_);
  const int N = Y.size_to_dim(canonical_axis);
  const int D = Y.size_from_dim(canonical_axis);
  auto* dX = Output(0, Y.sizes(), at::dtype<float>());
  auto* dX_data = dX->mutable_data<float>();
  if (N == 0 || D == 0) {
    return true;
  }
  softmax_gradient_kernel<<<
      N,
      SOFTMAX_NUM_THREADS,
      0,
      context_.hip_stream()>>>(D, Y.data<float>(), dY.data<float>(), dX_data);
  return true;
}

REGISTER_HIP_OPERATOR(SoftmaxWithLoss, SoftmaxWithLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SoftmaxWithLossGradient,
    SoftmaxWithLossGradientOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SpatialSoftmaxWithLoss,
    SpatialSoftmaxWithLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SpatialSoftmaxWithLossGradient,
    SpatialSoftmaxWithLossGradientOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(Softmax, SoftmaxOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SoftmaxGradient, SoftmaxGradientOp<float, HIPContext>);

} // namespace caffe2
