#include "hip/hip_runtime.h"
#include <assert.h>
#include <cub/block/block_reduce.cuh>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/cross_entropy_op.h"
#include "caffe2/operators/operator_fallback_gpu.h"

namespace caffe2 {

namespace {
__global__ void LabelCrossEntropyKernel(
    const int N, const int D, const float* Xdata, const int* labeldata,
    const float log_threshold, float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    CUDA_KERNEL_ASSERT(labeldata[i] >= 0 && labeldata[i] < D);
    Ydata[i] = -logf(fmaxf(Xdata[i * D + labeldata[i]], log_threshold));
  }
}
__global__ void LabelCrossEntropyGradientKernel(
    const int N, const int D, const float* Xdata, const int* labeldata,
    const float* dYdata, const float log_threshold, float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int idx = i * D + labeldata[i];
    dXdata[idx] = - dYdata[i] / fmaxf(Xdata[idx], log_threshold);
  }
}
}  // namespace

template <>
bool LabelCrossEntropyOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& label = Input(1);

  int N, D;
  if (X.dim() > 1) {
    N = X.dim32(0);
    D = X.size_from_dim(1);
  } else {
    N = 1;
    D = X.dim32(0);
  }
  CAFFE_ENFORCE(
      (label.dim() == 1) || (label.dim() == 2 && label.dim32(1) == 1));
  CAFFE_ENFORCE_EQ(label.dim32(0), N);
  auto* Y = Output(0, vector<int64_t>(size_t(1), N), at::dtype<float>());
  LabelCrossEntropyKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N,
      D,
      X.data<float>(),
      label.data<int>(),
      kLOG_THRESHOLD(),
      Y->template mutable_data<float>());
  return true;
}

template <>
bool LabelCrossEntropyGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto& label = Input(1);
  auto& dY = Input(2);

  int N, D;
  if (X.dim() > 1) {
    N = X.dim32(0);
    D = X.size_from_dim(1);
  } else {
    N = 1;
    D = X.dim32(0);
  }
  CAFFE_ENFORCE(
      (label.dim() == 1) || (label.dim() == 2 && label.dim32(1) == 1));
  CAFFE_ENFORCE_EQ(label.dim32(0), N);
  CAFFE_ENFORCE_EQ(dY.dim(), 1);
  CAFFE_ENFORCE_EQ(dY.dim32(0), N);
  auto* dX = Output(0, X.sizes(), at::dtype<float>());
  math::Set<float, HIPContext>(
      dX->numel(), 0.f, dX->template mutable_data<float>(), &context_);
  LabelCrossEntropyGradientKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N,
      D,
      X.data<float>(),
      label.data<int>(),
      dY.data<float>(),
      kLOG_THRESHOLD(),
      dX->template mutable_data<float>());
  return true;
}

namespace {
__global__ void MakeTwoClassKernel(
    const int N, const float* Xdata, float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Ydata[i * 2] = 1.0 - Xdata[i];
    Ydata[i * 2 + 1] = Xdata[i];
  }
}
__global__ void MakeTwoClassGradientKernel(
    const int N, const float* dYdata, float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dXdata[i] = dYdata[i * 2 + 1] - dYdata[i * 2];
  }
}
}  // namespace

template <>
bool MakeTwoClassOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto shape = X.sizes().vec();
  shape.push_back(2);
  CAFFE_ENFORCE_LT(X.numel(), std::numeric_limits<int>::max() / 2);
  auto* Y = Output(0, shape, at::dtype<float>());
  int N = X.numel();
  MakeTwoClassKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N, X.data<float>(), Y->template mutable_data<float>());
  return true;
}

template <>
bool MakeTwoClassGradientOp<float, HIPContext>::RunOnDevice() {
  auto& dY = Input(0);
  auto shape = dY.sizes().vec();
  CAFFE_ENFORCE_GE(shape.size(), 1);
  CAFFE_ENFORCE_EQ(shape.back(), 2);
  shape.pop_back();
  CAFFE_ENFORCE_LT(dY.numel(), std::numeric_limits<int>::max());
  auto* dX = Output(0, shape, at::dtype<float>());
  int N = dX->numel();
  MakeTwoClassGradientKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      N, dY.data<float>(), dX->template mutable_data<float>());
  return true;
}

namespace {

__device__ float sigmoid_xent_forward(float lgt, float tgt) {
  return lgt * (tgt - (lgt >= 0)) - log(1 + exp(lgt - 2 * lgt * (lgt >= 0)));
}

__device__ float sigmoid_xent_backward(float lgt, float tgt) {
  return tgt - 1. / (1. + exp(-lgt));
}

__device__ float sigmoid_partition(float lgt) {
  // computes log(1 + exp(lgt)) with only exp(x) function when x >= 0
  return lgt * (lgt >= 0) + log(1 + exp(lgt - 2 * lgt * (lgt >= 0)));
}

__device__ float sigmoid_xent_forward_with_log_d_trick(float lgt, float tgt) {
  return (2 * tgt - 1.) * (lgt - sigmoid_partition(lgt));
}

__device__ float sigmoid_xent_backward_with_log_d_trick(float lgt, float tgt) {
  return (2 * tgt - 1.) / (1. + exp(lgt));
}

__device__ float unjoined_sigmoid_xent_forward(float lgt, float tgt) {
  return lgt * tgt + (tgt - 1) * lgt * (lgt >= 0) -
      (1 - tgt) * log(1 + exp(lgt - 2 * lgt * (lgt >= 0)));
}

__device__ float unjoined_sigmoid_xent_backward(float lgt, float tgt) {
  return tgt - (1. - tgt) / (1. + exp(-lgt));
}

__global__ void SigmoidCrossEntropyWithLogitsKernel(
    const int outer_size,
    const int inner_size,
    const bool log_D_trick,
    const bool unjoined_lr_loss,
    const float* logits_ptr,
    const float* targets_ptr,
    float* out_ptr) {
  int i = blockIdx.x;
  int last_idx = (i + 1) * inner_size;
  float value = 0;
  for (int in_idx = i * inner_size + threadIdx.x; in_idx < last_idx;
       in_idx += blockDim.x) {
    if (unjoined_lr_loss) {
      value += unjoined_sigmoid_xent_forward(
          logits_ptr[in_idx], targets_ptr[in_idx]);
    } else {
      value +=
          (log_D_trick
               ? sigmoid_xent_forward_with_log_d_trick(
                     logits_ptr[in_idx], targets_ptr[in_idx])
               : sigmoid_xent_forward(logits_ptr[in_idx], targets_ptr[in_idx]));
    }
  }

  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  float sum = BlockReduce(temp_storage).Sum(value);
  if (threadIdx.x == 0) {
    out_ptr[i] = -sum / inner_size;
  }
}

__global__ void SigmoidCrossEntropyGradientWithLogitsKernel(
    const int outer_size,
    const int inner_size,
    const bool log_D_trick,
    const bool unjoined_lr_loss,
    const float* g_ptr,
    const float* logits_ptr,
    const float* targets_ptr,
    float* out_ptr) {
  CUDA_1D_KERNEL_LOOP(in_idx, outer_size * inner_size) {
    int i = in_idx / inner_size;
    auto g_factor = -g_ptr[i] / inner_size;
    if (unjoined_lr_loss) {
      out_ptr[in_idx] = g_factor *
          unjoined_sigmoid_xent_backward(
                            logits_ptr[in_idx], targets_ptr[in_idx]);
    } else {
      out_ptr[in_idx] = g_factor *
          (log_D_trick ? sigmoid_xent_backward_with_log_d_trick(
                             logits_ptr[in_idx], targets_ptr[in_idx])
                       : sigmoid_xent_backward(
                             logits_ptr[in_idx], targets_ptr[in_idx]));
    }
  }
}
} // namespace

template <>
bool SigmoidCrossEntropyWithLogitsOp<float, HIPContext>::RunOnDevice() {
  auto& logits = Input(0);
  auto& targets = Input(1);
  CAFFE_ENFORCE_EQ(logits.sizes(), targets.sizes());
  const auto inner_size = logits.dim() > 0 ? logits.sizes().back() : 1;
  const auto outer_size = logits.numel() / inner_size;

  std::vector<int64_t> dims;
  if (logits.dim() != 0) {
    dims =
        std::vector<int64_t>(logits.sizes().begin(), logits.sizes().end() - 1);
  }
  auto* out = Output(0, dims, at::dtype<float>());
  auto* out_ptr = out->template mutable_data<float>();

  auto* logits_ptr = logits.data<float>();
  auto* targets_ptr = targets.data<float>();

  if (logits.numel() <= 0) {
    // nothing to do, not even launching kernel
    return true;
  }

  SigmoidCrossEntropyWithLogitsKernel<<<
      outer_size,
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      outer_size,
      inner_size,
      log_D_trick_,
      unjoined_lr_loss_,
      logits_ptr,
      targets_ptr,
      out_ptr);
  return true;
}

template <>
bool SigmoidCrossEntropyWithLogitsGradientOp<float, HIPContext>::
    RunOnDevice() {
  auto& g = Input(0);
  auto& logits = Input(1);
  auto& targets = Input(2);
  CAFFE_ENFORCE_EQ(logits.sizes(), targets.sizes());
  const auto inner_size = logits.dim() > 0 ? logits.sizes().back() : 1;
  const auto outer_size = logits.numel() / inner_size;
  CAFFE_ENFORCE_EQ(g.numel(), outer_size);

  auto* out = Output(0, logits.sizes(), at::dtype<float>());
  auto* out_ptr = out->template mutable_data<float>();

  auto* logits_ptr = logits.data<float>();
  auto* targets_ptr = targets.data<float>();
  auto* g_ptr = g.data<float>();

  SigmoidCrossEntropyGradientWithLogitsKernel<<<
      CAFFE_GET_BLOCKS(outer_size * inner_size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      outer_size,
      inner_size,
      log_D_trick_,
      unjoined_lr_loss_,
      g_ptr,
      logits_ptr,
      targets_ptr,
      out_ptr);
  return true;
}

namespace {

__global__ void WeightedSigmoidCrossEntropyWithLogitsKernel(
    const int outer_size,
    const int inner_size,
    const float* logits_ptr,
    const float* targets_ptr,
    const float* weights_ptr,
    float* out_ptr) {
  int i = blockIdx.x;
  int last_idx = (i + 1) * inner_size;
  float value = 0;
  for (int in_idx = i * inner_size + threadIdx.x; in_idx < last_idx;
       in_idx += blockDim.x) {
    value += sigmoid_xent_forward(logits_ptr[in_idx], targets_ptr[in_idx]) *
        weights_ptr[in_idx];
  }

  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  float sum = BlockReduce(temp_storage).Sum(value);
  if (threadIdx.x == 0) {
    out_ptr[i] = -sum / inner_size;
  }
}

__global__ void WeightedSigmoidCrossEntropyGradientWithLogitsKernel(
    const int outer_size,
    const int inner_size,
    const float* g_ptr,
    const float* logits_ptr,
    const float* targets_ptr,
    const float* weights_ptr,
    float* out_ptr) {
  CUDA_1D_KERNEL_LOOP(in_idx, outer_size * inner_size) {
    int i = in_idx / inner_size;
    auto g_factor = -g_ptr[i] / inner_size;
    out_ptr[in_idx] = g_factor *
        sigmoid_xent_backward(logits_ptr[in_idx], targets_ptr[in_idx]) *
        weights_ptr[in_idx];
  }
}
} // namespace

template <>
bool WeightedSigmoidCrossEntropyWithLogitsOp<float, HIPContext>::
    RunOnDevice() {
  auto& logits = Input(0);
  auto& targets = Input(1);
  auto& weights = Input(2);
  CAFFE_ENFORCE_EQ(logits.sizes(), targets.sizes());
  CAFFE_ENFORCE_EQ(weights.sizes(), targets.sizes());
  const auto inner_size = logits.dim() > 0 ? logits.sizes().back() : 1;
  const auto outer_size = logits.numel() / inner_size;

  std::vector<int64_t> dims;
  if (logits.dim() != 0) {
    dims =
        std::vector<int64_t>(logits.sizes().begin(), logits.sizes().end() - 1);
  }
  auto* out = Output(0, dims, at::dtype<float>());
  auto* out_ptr = out->template mutable_data<float>();

  auto* logits_ptr = logits.data<float>();
  auto* targets_ptr = targets.data<float>();
  auto* weights_ptr = weights.data<float>();

  WeightedSigmoidCrossEntropyWithLogitsKernel<<<
      outer_size,
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      outer_size, inner_size, logits_ptr, targets_ptr, weights_ptr, out_ptr);
  return true;
}

template <>
bool WeightedSigmoidCrossEntropyWithLogitsGradientOp<float, HIPContext>::
    RunOnDevice() {
  auto& g = Input(0);
  auto& logits = Input(1);
  auto& targets = Input(2);
  auto& weights = Input(3);
  CAFFE_ENFORCE_EQ(logits.sizes(), targets.sizes());
  CAFFE_ENFORCE_EQ(weights.sizes(), targets.sizes());
  const auto inner_size = logits.dim() > 0 ? logits.sizes().back() : 1;
  const auto outer_size = logits.numel() / inner_size;
  CAFFE_ENFORCE_EQ(g.numel(), outer_size);

  auto* out = Output(0, logits.sizes(), at::dtype<float>());
  auto* out_ptr = out->template mutable_data<float>();

  auto* logits_ptr = logits.data<float>();
  auto* targets_ptr = targets.data<float>();
  auto* weights_ptr = weights.data<float>();
  auto* g_ptr = g.data<float>();

  WeightedSigmoidCrossEntropyGradientWithLogitsKernel<<<
      CAFFE_GET_BLOCKS(outer_size * inner_size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      outer_size,
      inner_size,
      g_ptr,
      logits_ptr,
      targets_ptr,
      weights_ptr,
      out_ptr);
  return true;
}

REGISTER_HIP_OPERATOR(LabelCrossEntropy,
                       LabelCrossEntropyOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(LabelCrossEntropyGradient,
                       LabelCrossEntropyGradientOp<float, HIPContext>);

REGISTER_HIP_OPERATOR(
    SigmoidCrossEntropyWithLogits,
    SigmoidCrossEntropyWithLogitsOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    SigmoidCrossEntropyWithLogitsGradient,
    SigmoidCrossEntropyWithLogitsGradientOp<float, HIPContext>);

REGISTER_HIP_OPERATOR(
    WeightedSigmoidCrossEntropyWithLogits,
    WeightedSigmoidCrossEntropyWithLogitsOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    WeightedSigmoidCrossEntropyWithLogitsGradient,
    WeightedSigmoidCrossEntropyWithLogitsGradientOp<float, HIPContext>);

REGISTER_HIP_OPERATOR(MakeTwoClass,
                       MakeTwoClassOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MakeTwoClassGradient,
                       MakeTwoClassGradientOp<float, HIPContext>);

//TODO(surya) Add full GPU/CUDA support for the CrossEntropyOp
REGISTER_HIP_OPERATOR(CrossEntropy, GPUFallbackOp);
REGISTER_HIP_OPERATOR(CrossEntropyGradient, GPUFallbackOp);

}  // namespace caffe2
