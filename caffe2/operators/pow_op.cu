#include "hip/hip_runtime.h"
#define HIPCUB_STDERR
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>
#include <cub/device/device_reduce.cuh>
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/pow_op.h"
#include "caffe2/utils/conversions.h"

namespace caffe2 {

// pow, log and other math functions are defined in
// CUDA math library in header file math.h
#define CUDA_POW(x, y) (pow(x, y))

// renaming to PowOpKernel as there exists PowKernel in caffe2/utils/math_gpu.cc
// Kernels with same leads to conflict during hipification for ROCm platform.
template <int b_is_scalar, typename T1, typename T2, typename R>
__global__ void PowOpKernel(const T1* a, const T2* b, T2 e, R* out, int n) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    out[i] = CUDA_POW(a[i], ((b == NULL) ? e : b[b_is_scalar ? 0 : i]));
  }
}
template <typename T1, typename T2, typename R>
__global__ void
PowBroadcastKernel(const T1* a, const T2* b, R* out, int pre, int n) {
  CUDA_1D_KERNEL_LOOP(i, pre * n) {
    out[i] = CUDA_POW(a[i], b[i % n]);
  }
}
template <typename T1, typename T2, typename R>
__global__ void PowBroadcast2Kernel(
    const T1* a,
    const T2* b,
    R* out,
    int pre,
    int n,
    int post) {
  CUDA_1D_KERNEL_LOOP(i, pre * n * post) {
    out[i] = CUDA_POW(a[i], b[(i / post) % n]);
  }
}

struct CudaPowFunctor {
  template <bool b_is_scalar, typename T1, typename T2, typename R>
  inline void
  Run(size_t n, const T1* a, const T2* b, T2 e, R* out, HIPContext* context) {
    PowOpKernel<b_is_scalar, T1, T2, R>
        <<<CAFFE_GET_BLOCKS(n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(a, b, e, out, n);
  }
  template <typename T1, typename T2, typename R>
  void RunWithBroadcast(
      const T1* a,
      const T2* b,
      R* out,
      size_t pre,
      size_t n,
      HIPContext* context) {
    PowBroadcastKernel<T1, T2, R>
        <<<CAFFE_GET_BLOCKS(pre * n),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(a, b, out, pre, n);
  }
  template <typename T1, typename T2, typename R>
  void RunWithBroadcast2(
      const T1* a,
      const T2* b,
      R* out,
      size_t pre,
      size_t n,
      size_t post,
      HIPContext* context) {
    PowBroadcast2Kernel<T1, T2, R>
        <<<CAFFE_GET_BLOCKS(pre * n * post),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(a, b, out, pre, n, post);
  }
};
REGISTER_HIP_OPERATOR(
    Pow,
    PowOp<
        TensorTypes<float> /*NumericTypes*/,
        HIPContext,
        CudaPowFunctor,
        SameTypeAsInput>)

} // namespace caffe2
