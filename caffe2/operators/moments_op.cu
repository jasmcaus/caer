#include "hip/hip_runtime.h"
#include "caffe2/operators/moments_op.h"

#include <array>
#include <functional>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/fixed_divisor.h"

namespace caffe2 {

namespace {

template <typename T, int D>
__global__ void ComputeMomentsGradientCUDAKernel(
    const int X_size,
    const SimpleArray<int, D> Y_strides,
    const SimpleArray<FixedDivisor<int>, D> X_dims,
    const T scale,
    const T* dmean,
    const T* dvariance,
    const T* X,
    const T* mean,
    T* dX) {
  CUDA_1D_KERNEL_LOOP(X_index, X_size) {
    int Y_index = 0;
    int X_index_val = X_index;
#pragma unroll
    for (int i = D - 1; i >= 0; --i) {
      int d;
      X_dims.data[i].DivMod(X_index_val, &X_index_val, &d);
      Y_index += d * Y_strides.data[i];
    }
#if __CUDA_ARCH__ >= 350
    dX[X_index] =
        (__ldg(dmean + Y_index) +
         static_cast<T>(2) * (__ldg(X + X_index) - __ldg(mean + Y_index)) *
             __ldg(dvariance + Y_index)) *
        scale;
#else
    dX[X_index] = (dmean[Y_index] +
                   static_cast<T>(2) * (X[X_index] - mean[Y_index]) *
                       dvariance[Y_index]) *
        scale;
#endif
  }
}

template <typename T, int D>
void ComputeMomentsGradientCUDAImpl(
    const int* Y_dims,
    const int* X_dims,
    const T* dmean,
    const T* dvariance,
    const T* X,
    const T* mean,
    T* dX,
    HIPContext* context) {
  SimpleArray<int, D> Y_strides_array;
  SimpleArray<FixedDivisor<int>, D> X_dims_array;
  int cur_stride = 1;
  for (int i = D - 1; i >= 0; --i) {
    if (X_dims[i] == 0) {
      return;
    }
    Y_strides_array.data[i] = Y_dims[i] == 1 ? 0 : cur_stride;
    X_dims_array.data[i] = FixedDivisor<int>(X_dims[i]);
    cur_stride *= Y_dims[i];
  }
  const int Y_size =
      std::accumulate(Y_dims, Y_dims + D, 1, std::multiplies<int>());
  const int X_size =
      std::accumulate(X_dims, X_dims + D, 1, std::multiplies<int>());
  const T scale = static_cast<T>(Y_size) / static_cast<T>(X_size);
  ComputeMomentsGradientCUDAKernel<T, D>
      <<<CAFFE_GET_BLOCKS(X_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          X_size,
          Y_strides_array,
          X_dims_array,
          scale,
          dmean,
          dvariance,
          X,
          mean,
          dX);
}

} // namespace

template <>
bool MomentsGradientOp<float, HIPContext>::Compute(
    const std::vector<int>& dY_dims,
    const std::vector<int>& dX_dims,
    const float* dmean_data,
    const float* dvariance_data,
    const float* X_data,
    const float* mean_data,
    float* dX_data) {
  const int ndim = dY_dims.size();
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(
      ndim,
      ComputeMomentsGradientCUDAImpl,
      float,
      dY_dims.data(),
      dX_dims.data(),
      dmean_data,
      dvariance_data,
      X_data,
      mean_data,
      dX_data,
      &context_);
  return true;
}

REGISTER_HIP_OPERATOR(Moments, MomentsOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MomentsGradient, MomentsGradientOp<float, HIPContext>);

} // namespace caffe2
