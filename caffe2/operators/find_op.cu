#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/find_op.h"

namespace caffe2 {

template <typename T>
__global__ void FindKernel(
    int num_needles,
    int idx_size,
    const T* idx,
    const T* needles,
    int* out,
    int missing_value) {
  int needle_idx = blockIdx.x; // One cuda block per needle
  T q = needles[needle_idx];
  int res = (-1);
  for (int j = threadIdx.x; j < idx_size; j += CAFFE_CUDA_NUM_THREADS) {
    if (idx[j] == q) {
      res = max(res, j);
    }
  }
  typedef hipcub::BlockReduce<int, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int min_res = BlockReduce(temp_storage).Reduce(res, hipcub::Max());
  if (threadIdx.x == 0) {
    out[needle_idx] = min_res == (-1) ? missing_value : min_res;
  }
}

template <>
template <typename T>
bool FindOp<HIPContext>::DoRunWithType() {
  auto& idx = Input(0);
  auto& needles = Input(1);

  auto* res_indices = Output(0, needles.sizes(), at::dtype<int>());

  const T* idx_data = idx.data<T>();
  const T* needles_data = needles.data<T>();
  int* res_data = res_indices->template mutable_data<int>();

  FindKernel<
      T><<<needles.numel(), CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
      needles.numel(),
      idx.numel(),
      idx_data,
      needles_data,
      res_data,
      missing_value_);
  return true;
}

REGISTER_HIP_OPERATOR(Find, FindOp<HIPContext>)

} // namespace caffe2
