#include "hip/hip_runtime.h"
#ifdef _MSC_VER
#define _USE_MATH_DEFINES // For M_PI
#endif // _MSC_VER
#include <cmath>

#include "caffe2/operators/roi_align_rotated_gradient_op.h"

#include <stdio.h>
#include <cfloat>
#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

template <typename T>
inline __device__ T gpu_atomic_add(const T val, T* address);

template <>
inline __device__ float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height,
    const int width,
    T y,
    T x,
    T& w1,
    T& w2,
    T& w3,
    T& w4,
    int& x_low,
    int& x_high,
    int& y_low,
    int& y_high,
    const int index /* index for debug only*/) {
  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    // empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) {
    y = 0;
  }
  if (x <= 0) {
    x = 0;
  }

  y_low = (int)y;
  x_low = (int)x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T)y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T)x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename T>
__global__ void RoIAlignRotatedBackward(
    const int nthreads,
    const T* top_diff,
    const int num_rois,
    const T spatial_scale,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* bottom_rois,
    bool continuous_coordinate) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not round
    T roi_offset = continuous_coordinate ? T(0.5) : 0;
    T roi_center_w = offset_bottom_rois[1] * spatial_scale - roi_offset;
    T roi_center_h = offset_bottom_rois[2] * spatial_scale - roi_offset;
    T roi_width = offset_bottom_rois[3] * spatial_scale;
    T roi_height = offset_bottom_rois[4] * spatial_scale;
    T theta = offset_bottom_rois[5] * M_PI / 180.0;

    if (!continuous_coordinate) { // backward compatibility
      // Force malformed ROIs to be 1x1
      roi_width = c10::cuda::compat::max(roi_width, (T)1.);
      roi_height = c10::cuda::compat::max(roi_height, (T)1.);
    }
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    T* offset_bottom_diff =
        bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0)
        ? sampling_ratio
        : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w =
        (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // roi_start_h and roi_start_w are computed wrt the center of RoI (x, y).
    // Appropriate translation needs to be applied after.
    T roi_start_h = -roi_height / 2.0;
    T roi_start_w = -roi_width / 2.0;
    T cosTheta = cos(theta);
    T sinTheta = sin(theta);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
    {
      const T yy = roi_start_h + ph * bin_size_h +
          static_cast<T>(iy + .5f) * bin_size_h /
              static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix++) {
        const T xx = roi_start_w + pw * bin_size_w +
            static_cast<T>(ix + .5f) * bin_size_w /
                static_cast<T>(roi_bin_grid_w);

        // Rotate by theta around the center and translate
        T x = xx * cosTheta + yy * sinTheta + roi_center_w;
        T y = yy * cosTheta - xx * sinTheta + roi_center_h;

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(
            height,
            width,
            y,
            x,
            w1,
            w2,
            w3,
            w4,
            x_low,
            x_high,
            y_low,
            y_high,
            index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
          gpu_atomic_add(
              static_cast<T>(g1), offset_bottom_diff + y_low * width + x_low);
          gpu_atomic_add(
              static_cast<T>(g2), offset_bottom_diff + y_low * width + x_high);
          gpu_atomic_add(
              static_cast<T>(g3), offset_bottom_diff + y_high * width + x_low);
          gpu_atomic_add(
              static_cast<T>(g4), offset_bottom_diff + y_high * width + x_high);
        } // if
      } // ix
    } // iy
  } // CUDA_1D_KERNEL_LOOP
} // RoIAlignRotatedBackward

} // namespace

template <>
C10_EXPORT bool RoIAlignRotatedGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0); // Input data to pool
  auto& R = Input(1); // RoIs
  auto& dY = Input(2); // Gradient of net w.r.t. output of "forward" op
                       // (aka "gradOutput")

  auto* dX = Output(
      0, X.sizes(), at::dtype<float>()); // Gradient of net w.r.t. input to
                                         // "forward" op (aka "gradInput")

  // Must zero-out dX before accumulating gradients
  math::Set<float, HIPContext>(
      dX->numel(), 0.f, dX->mutable_data<float>(), &context_);

  if (dY.numel() > 0) { // Handle possibly empty gradient if there were no rois
    RoIAlignRotatedBackward<float>
        <<<CAFFE_GET_BLOCKS(dY.numel()),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            dY.numel(),
            dY.data<float>(),
            R.dim32(0),
            spatial_scale_,
            X.dim32(1),
            X.dim32(2),
            X.dim32(3),
            pooled_height_,
            pooled_width_,
            sampling_ratio_,
            dX->mutable_data<float>(),
            R.data<float>(),
            aligned_);
  }
  return true;
}

REGISTER_HIP_OPERATOR(
    RoIAlignRotatedGradient,
    RoIAlignRotatedGradientOp<float, HIPContext>);
} // namespace caffe2
