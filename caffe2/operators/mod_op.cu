#include "hip/hip_runtime.h"
#include "caffe2/operators/mod_op.h"

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void ModOpSimpleKernel(const int N, const int64_t divisor_,
                            const T* data_ptr, T* output_ptr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    output_ptr[i] = data_ptr[i] % divisor_;
  }
}


template <typename T>
__global__ void ModOpKernel(const int N, const int64_t divisor_,
                            const T* data_ptr, T* output_ptr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    output_ptr[i] = data_ptr[i] % divisor_;
    if (output_ptr[i] && ((output_ptr[i] > 0) != (divisor_ > 0))) {
      output_ptr[i] += divisor_;
    }
  }
}

}  // namespace

template <>
template <typename T>
bool ModOp<HIPContext>::DoRunWithType() {
  auto& data = Input(DATA);
  auto N = data.numel();
  const auto* data_ptr = data.template data<T>();

  auto* output = Output(0, data.sizes(), at::dtype<T>());
  auto* output_ptr = output->template mutable_data<T>();

  if (sign_follow_divisor_) {
    ModOpKernel<<<
        CAFFE_GET_BLOCKS(N),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        N, divisor_, data_ptr, output_ptr);
  } else {
    ModOpSimpleKernel<<<
        CAFFE_GET_BLOCKS(N),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context_.hip_stream()>>>(
        N, divisor_, data_ptr, output_ptr);
  }

  return true;

}

REGISTER_HIP_OPERATOR(Mod, ModOp<HIPContext>);

} // namespace caffe2
