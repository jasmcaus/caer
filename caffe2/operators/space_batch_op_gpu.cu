#include "hip/hip_runtime.h"
#include "caffe2/operators/space_batch_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void SpaceToBatch(
    int N,
    int output_batch,
    int output_depth,
    int output_height,
    int output_width,
    int input_batch,
    int input_depth,
    int input_height,
    int input_width,
    const int pad_l,
    const int pad_t,
    int block_size,
    const float* input,
    float* output) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    // Recall:
    // const auto output_offset =
    //     ((out_b * output_depth + d) * output_height + out_h) * output_width +
    //     out_w;
    const int out_w = i % output_width;
    const int i_2 = i / output_width;
    const int out_h = i_2 % output_height;
    const int i_3 = i_2 / output_height;
    const int d = i_3 % output_depth;
    const int out_b = i_3 / output_depth;

    const int in_b = out_b % input_batch;
    const int offset_w = (out_b / input_batch) % block_size;
    const int offset_h = (out_b / input_batch) / block_size;
    const int in_h = out_h * block_size + offset_h - pad_t;
    const int in_w = out_w * block_size + offset_w - pad_l;

    if (in_h >= 0 && in_w >= 0 && in_h < input_height && in_w < input_width) {
      const auto input_offset =
          ((in_b * input_depth + d) * input_height + in_h) * input_width +
          in_w;
      output[i] = input[input_offset];
    } else {
      output[i] = 0.0;
    }
  }
}

template <>
void spaceToBatch<HIPContext>(
    const Tensor& input,
    int pad_t,
    int pad_l,
    int block_size,
    Tensor* output,
    HIPContext* context) {
  const int output_batch = output->dim32(0);
  const int output_depth = output->dim32(1);
  const int output_height = output->dim32(2);
  const int output_width = output->dim32(3);

  const int input_batch = input.dim32(0);
  const int input_depth = input.dim32(1);
  const int input_height = input.dim32(2);
  const int input_width = input.dim32(3);
  const int N = output->numel();
  SpaceToBatch<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      N,
      output_batch,
      output_depth,
      output_height,
      output_width,
      input_batch,
      input_depth,
      input_height,
      input_width,
      pad_l,
      pad_t,
      block_size,
      input.data<float>(),
      output->template mutable_data<float>());
}


__global__ void BatchToSpace(
    int N,
    int output_batch,
    int output_depth,
    int output_height,
    int output_width,
    int input_batch,
    int input_depth,
    int input_height,
    int input_width,
    const int pad_l,
    const int pad_t,
    int block_size,
    const float* input,
    float* output) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    // Recall:
    // const auto input_offset = ((in_b * input_depth + d) *
    //   input_height + in_h) * input_width + in_w;
    const int in_w = i  % input_width;
    const int i_2 = i / input_width;
    const int in_h = i_2 % input_height;
    const int i_3 = i_2 / input_height;
    const int d = i_3 % input_depth;
    const int in_b = i_3 / input_depth;

    const int out_b = in_b % output_batch;
    const int offset_w = (in_b / output_batch) % block_size;
    const int offset_h = (in_b / output_batch) / block_size;
    const int out_h = in_h * block_size + offset_h - pad_t;
    const int out_w = in_w * block_size + offset_w - pad_l;

    if (out_h >= 0 && out_w >= 0 && out_h < output_height &&
        out_w < output_width) {
      const auto output_offset =
          ((out_b * output_depth + d) * output_height + out_h) *
          output_width +
          out_w;
      output[output_offset] = input[i];
    }
  }
}

template <>
void batchToSpace(
    const Tensor& input,
    int pad_t,
    int pad_l,
    int block_size,
    Tensor* output,
    HIPContext* context) {
  CAFFE_ENFORCE(input.dim() == 4);
  CAFFE_ENFORCE(output->dim() == 4);

  const int output_batch = output->dim32(0);
  const int output_depth = output->dim32(1);
  const int output_height = output->dim32(2);
  const int output_width = output->dim32(3);

  const int input_batch = input.dim32(0);
  const int input_depth = input.dim32(1);
  const int input_height = input.dim32(2);
  const int input_width = input.dim32(3);
  const int N = input.numel();
  BatchToSpace<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      N,
      output_batch,
      output_depth,
      output_height,
      output_width,
      input_batch,
      input_depth,
      input_height,
      input_width,
      pad_l,
      pad_t,
      block_size,
      input.data<float>(),
      output->template mutable_data<float>());
}

REGISTER_HIP_OPERATOR(SpaceToBatch, SpaceToBatchOp<HIPContext>);
REGISTER_HIP_OPERATOR(BatchToSpace, BatchToSpaceOp<HIPContext>);

}
