#include "caffe2/core/context_gpu.h"
#include "caffe2/core/operator.h"

namespace caffe2 {
namespace {

class GetGPUMemoryUsageOp final : public Operator<HIPContext> {
 public:
  template<class... Args> explicit GetGPUMemoryUsageOp(Args&&... args)
      : Operator<HIPContext>(std::forward<Args>(args)...) {}
  ~GetGPUMemoryUsageOp() override {}

  bool RunOnDevice() override {
    CHECK_EQ(InputSize(), 0);
    CHECK_EQ(OutputSize(), 1);
    std::vector<long> total_by_gpu = HIPContext::TotalMemoryByGpu();
    std::vector<long> max_by_gpu = HIPContext::MaxMemoryByGpu();
    CHECK_EQ(total_by_gpu.size(), max_by_gpu.size());


    auto* stats = Output(0, {2, static_cast<int64_t>(total_by_gpu.size())}, at::dtype<long>());
    context_.CopyFromCPU<long>(
        total_by_gpu.size(),
        total_by_gpu.data(),
        stats->template mutable_data<long>());
    context_.CopyFromCPU<long>(
        max_by_gpu.size(),
        max_by_gpu.data(),
        stats->template mutable_data<long>() + total_by_gpu.size());
    return true;
  }
};

OPERATOR_SCHEMA(GetGPUMemoryUsage)
    .NumInputs(0)
    .NumOutputs(1)
    .SetDoc(R"DOC(Fetches GPU memory stats from HIPContext. Result is stored
      in output blob with shape (2, num_gpus). First row contains the total
      current memory usage, and the second row the maximum usage during
      this execution.

      NOTE: --caffe2_gpu_memory_tracking flag must be enabled to use this op.
    )DOC");

REGISTER_HIP_OPERATOR(GetGPUMemoryUsage, GetGPUMemoryUsageOp);
}

} // namespace caffe2
