#include "hip/hip_runtime.h"
// TODO: reduce the apparent redundancy of all the code below.
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/pool_op.h"

namespace caffe2 {
namespace {
struct LpPoolFunctor {
  explicit LpPoolFunctor(const OperatorBase& /* op */) {}
};
} // namespace

namespace {

using c10::cuda::compat::abs;
using c10::cuda::compat::pow;

template <typename T>
__global__ void LpPoolForwardNCHW(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data,
    const T p) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index;
    int pw = n % pooled_width;
    n /= pooled_width;
    int ph = n % pooled_height;
    n /= pooled_height;
    int c = n % channels;
    n /= channels;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    top_data[index] = 0;
    int bottom_offset = (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        top_data[index] +=
            pow(abs(bottom_data[bottom_offset + h * width + w]), p);
      }
    }
    top_data[index] = pow(top_data[index], static_cast<T>(1.0) / p);
  }
}

template <typename T>
__global__ void LpPoolForwardNHWC(
    const int nthreads,
    const T* bottom_data,
    const int num,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* top_data,
    const T p) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int c = index % channels;
    int pw = (index / channels) % pooled_width;
    int ph = (index / channels / pooled_width) % pooled_height;
    int n = index / channels / pooled_width / pooled_height;
    int hstart = ph * stride_h - pad_t;
    int wstart = pw * stride_w - pad_l;
    int hend = min(hstart + kernel_h, height);
    int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T output = 0;
    int bottom_offset = n * height * width * channels + c;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        output += pow(
            abs(bottom_data[bottom_offset + (h * width + w) * channels]), p);
      }
    }
    top_data[index] = pow(output, static_cast<T>(1.0) / p);
  }
}

template <typename T>
__global__ void LpPoolBackwardNCHW(
    const int nthreads,
    const T* const top_diff,
    const T* const top_data,
    const T* const bottom_data,
    const int num,
    const int channels,
    const int height,
    const int width,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff,
    const int p) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_l;
    const int h = (index / width) % height + pad_t;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    const T* const top_data_slice =
        top_data + (n * channels + c) * pooled_height * pooled_width;

    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_t;
        int wstart = pw * stride_w - pad_l;
        int hend = min(hstart + kernel_h, height);
        int wend = min(wstart + kernel_w, width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        gradient += top_diff_slice[ph * pooled_width + pw] *
            bottom_data[index] * pow(abs(bottom_data[index]), p - 2) /
            pow(top_data_slice[ph * pooled_width + pw], p - 1);
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename T>
__global__ void LpPoolBackwardNHWC(
    const int nthreads,
    const T* const top_diff,
    const T* const top_data,
    const T* const bottom_data,
    const int num,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int kernel_h,
    const int kernel_w,
    const int stride_h,
    const int stride_w,
    const int pad_t,
    const int pad_l,
    T* const bottom_diff,
    const T p) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int c = index % channels;
    const int w = index / channels % width + pad_l;
    const int h = (index / channels / width) % height + pad_t;
    const int n = index / channels / width / height;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    T gradient = 0;
    const T* const top_diff_slice =
        top_diff + n * pooled_height * pooled_width * channels + c;
    const T* const top_data_slice =
        top_data + n * pooled_height * pooled_width * channels + c;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_t;
        int wstart = pw * stride_w - pad_l;
        int hend = min(hstart + kernel_h, height);
        int wend = min(wstart + kernel_w, width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        gradient += top_diff_slice[(ph * pooled_width + pw) * channels] *
            bottom_data[index] * pow(abs(bottom_data[index]), p - 2) /
            pow(top_data_slice[(ph * pooled_width + pw) * channels], p - 1);
      }
    }
    bottom_diff[index] = gradient;
  }
}

} // namespace

template <>
bool PoolOp<float, HIPContext, LpPoolFunctor>::RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(1));
  int output_size = Y->numel();
  LpPoolForwardNCHW<float>
      <<<CAFFE_GET_BLOCKS(output_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          Y->dim32(2),
          Y->dim32(3),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          Y->template mutable_data<float>(),
          OperatorBase::GetSingleArgument<float>("p", 2.0));
  return true;
}

template <>
bool PoolOp<float, HIPContext, LpPoolFunctor>::RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto* Y = Output(0);
  ConvPoolOpBase<HIPContext>::SetOutputSize(X, Y, X.dim32(3));
  int output_size = Y->numel();
  LpPoolForwardNHWC<float>
      <<<CAFFE_GET_BLOCKS(output_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          output_size,
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          Y->dim32(1),
          Y->dim32(2),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          Y->template mutable_data<float>(),
          OperatorBase::GetSingleArgument<float>("p", 2.0));
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, LpPoolFunctor>::
    RunOnDeviceWithOrderNCHW() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.dim(), 4);

  auto* dX = Output(0, X.sizes(), at::dtype<float>());
  ConvPoolOpBase<HIPContext>::ComputePads({X.dim32(2), X.dim32(3)});
  LpPoolBackwardNCHW<float>
      <<<CAFFE_GET_BLOCKS(X.numel()),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          X.numel(),
          dY.data<float>(),
          Y.data<float>(),
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          dY.dim32(2),
          dY.dim32(3),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          dX->template mutable_data<float>(),
          OperatorBase::GetSingleArgument<float>("p", 2.0));
  return true;
}

template <>
bool PoolGradientOp<float, HIPContext, LpPoolFunctor>::
    RunOnDeviceWithOrderNHWC() {
  auto& X = Input(0);
  auto& Y = Input(1);
  auto& dY = Input(2);
  CAFFE_ENFORCE_EQ(dY.dim(), 4);

  auto* dX = Output(0, X.sizes(), at::dtype<float>());
  ConvPoolOpBase<HIPContext>::ComputePads({X.dim32(1), X.dim32(2)});
  LpPoolBackwardNHWC<float>
      <<<CAFFE_GET_BLOCKS(X.numel()),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          X.numel(),
          dY.data<float>(),
          Y.data<float>(),
          X.data<float>(),
          X.dim32(0),
          X.dim32(1),
          X.dim32(2),
          X.dim32(3),
          dY.dim32(1),
          dY.dim32(2),
          kernel_h(),
          kernel_w(),
          stride_h(),
          stride_w(),
          pad_t(),
          pad_l(),
          dX->template mutable_data<float>(),
          OperatorBase::GetSingleArgument<float>("p", 2.0));
  return true;
}

REGISTER_HIP_OPERATOR(LpPool, PoolOp<float, HIPContext, LpPoolFunctor>);
REGISTER_HIP_OPERATOR(
    LpPoolGradient,
    PoolGradientOp<float, HIPContext, LpPoolFunctor>);
}
