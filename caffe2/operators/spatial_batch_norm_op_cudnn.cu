#include "caffe2/operators/hip/spatial_batch_norm_op_miopen.hip"

#include <array>
#include <functional>
#include <numeric>
#include <vector>

#include "caffe2/core/context_gpu.h"
#include "caffe2/core/cudnn_wrappers.h"
#include "caffe2/operators/spatial_batch_norm_op_impl.cuh"
#include "caffe2/utils/math.h"

#if CUDNN_VERSION_MIN(5, 0, 0)

namespace caffe2 {

namespace {

void SetTensorDescriptor(
    const hipdnnDataType_t data_type,
    const hipdnnBatchNormMode_t mode,
    const StorageOrder order,
    const std::vector<int>& input_dims,
    hipdnnTensorDescriptor_t data_desc,
    hipdnnTensorDescriptor_t param_desc) {
  const int ndim = input_dims.size();
  const int N = input_dims[0];
  const int C = order == StorageOrder::NCHW ? input_dims[1] : input_dims.back();
  if (ndim == 3) {
    const int H = 1;
    const int W = order == StorageOrder::NCHW ? input_dims[2] : input_dims[1];
    CUDNN_ENFORCE(hipdnnSetTensor4dDescriptor(
        data_desc, GetCudnnTensorFormat(order), data_type, N, C, H, W));
  } else if (ndim == 4) {
    const int H = order == StorageOrder::NCHW ? input_dims[2] : input_dims[1];
    const int W = order == StorageOrder::NCHW ? input_dims[3] : input_dims[2];
    CUDNN_ENFORCE(hipdnnSetTensor4dDescriptor(
        data_desc, GetCudnnTensorFormat(order), data_type, N, C, H, W));
  } else {
    const int H = order == StorageOrder::NCHW ? input_dims[2] : input_dims[1];
    const int W = order == StorageOrder::NCHW ? input_dims[3] : input_dims[2];
    const auto l_iter = order == StorageOrder::NCHW ? input_dims.cbegin() + 4
                                                    : input_dims.cbegin() + 3;
    const auto r_iter =
        order == StorageOrder::NCHW ? input_dims.cend() : input_dims.cend() - 1;
    const int D = std::accumulate(l_iter, r_iter, 1, std::multiplies<int>());
    const std::array<int, 5> dims = {N, C, H, W, D};
    const std::array<int, 5> strides = order == StorageOrder::NCHW
        ? std::array<int, 5>{C * H * W * D, H * W * D, W * D, D, 1}
        : std::array<int, 5>{C * H * W * D, 1, W * D * C, D * C, C};
    CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(
        data_desc, data_type, 5, dims.data(), strides.data()));
  }
  CUDNN_ENFORCE(hipdnnDeriveBNTensorDescriptor(param_desc, data_desc, mode));
}

} // namespace

class CuDNNSpatialBNOp final : public SpatialBNOp<HIPContext> {
 public:
  USE_OPERATOR_FUNCTIONS(HIPContext);

  CuDNNSpatialBNOp(const OperatorDef& operator_def, Workspace* ws)
      : SpatialBNOp<HIPContext>(operator_def, ws),
        cudnn_wrapper_(&context_),
#if CUDNN_VERSION_MIN(7, 0, 0)
        // TODO(T31829456): The new HIPDNN_BATCHNORM_SPATIAL_PERSISTENT mode was
        // introduced in CuDNN 7 for performance optimization, but it results in
        // accuracy losses in convolution models such as ResNeXt-101 and
        // video R(2+1)D. We will fall back to the normal
        // HIPDNN_BATCHNORM_SPATIAL for now
        mode_(HIPDNN_BATCHNORM_SPATIAL) {
#else
        mode_(HIPDNN_BATCHNORM_SPATIAL) {
#endif
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&data_desc_));
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&param_desc_));
    if (epsilon_ < HIPDNN_BN_MIN_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than HIPDNN_BN_MIN_EPSILON. "
                    "Setting it to HIPDNN_BN_MIN_EPSILON instead.";
      epsilon_ = HIPDNN_BN_MIN_EPSILON;
    }
  }

  ~CuDNNSpatialBNOp() {
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(data_desc_));
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(param_desc_));
  }

  bool RunOnDevice() override {
    // CuDNN doesn't support multi-batch SpatialBN and it's NHWC order SpatialBN
    // is much slower, so in such cases fallback to SpatialBNOp<HIPContext>.
    if (num_batches_ > 1 || order_ == StorageOrder::NHWC) {
      return SpatialBNOp<HIPContext>::RunOnDevice();
    }
    return DispatchHelper<TensorTypes<float, at::Half>>::call(this, Input(0));
  }

  template <typename T>
  bool DoRunWithType() {
    typedef typename cudnnTypeWrapper<T>::BNParamType BNParamType;

    const auto& X = Input(INPUT);
    const auto& scale = Input(SCALE);
    const auto& bias = Input(BIAS);
    auto* Y = Output(OUTPUT);
    const int ndim = X.ndim();
    CAFFE_ENFORCE_GE(ndim, 3);
    const int N = X.dim32(0);
    const int C =
        (order_ == StorageOrder::NCHW ? X.dim32(1) : X.dim32(ndim - 1));
    CAFFE_ENFORCE_EQ(scale.size(), C);
    CAFFE_ENFORCE_EQ(bias.size(), C);
    Y->ResizeLike(X);
    const T* X_data = X.data<T>();
    const BNParamType* scale_data = scale.data<BNParamType>();
    const BNParamType* bias_data = bias.data<BNParamType>();
    T* Y_data = Y->mutable_data<T>();

    if (N > 0) {
      const std::vector<int> input_dims(X.sizes().cbegin(), X.sizes().cend());
      if (input_dims != data_dims_) {
        data_dims_ = input_dims;
        SetTensorDescriptor(
            cudnnTypeWrapper<T>::type,
            mode_,
            order_,
            input_dims,
            data_desc_,
            param_desc_);
      }
    }
    if (is_test_) {
      const auto& mean = Input(EST_MEAN);
      const auto& var = Input(EST_VAR);
      CAFFE_ENFORCE_EQ(mean.size(), C);
      CAFFE_ENFORCE_EQ(var.size(), C);
      if (N == 0) {
        return true;
      }
      CUDNN_ENFORCE(hipdnnBatchNormalizationForwardInference(
          cudnn_wrapper_.inline_cudnn_handle(),
          // Note: PERSISTENT not implemented for inference
          HIPDNN_BATCHNORM_SPATIAL,
          cudnnTypeWrapper<T>::kOne(),
          cudnnTypeWrapper<T>::kZero(),
          data_desc_,
          X_data,
          data_desc_,
          Y_data,
          param_desc_,
          scale_data,
          bias_data,
          mean.data<BNParamType>(),
          var.data<BNParamType>(),
          epsilon_));
    } else {
      auto* saved_mean = Output(SAVED_MEAN);
      auto* saved_inv_std = Output(SAVED_INV_STD);
      saved_mean->Resize(C);
      saved_inv_std->Resize(C);
      BNParamType* saved_mean_data = saved_mean->mutable_data<BNParamType>();
      BNParamType* saved_inv_std_data =
          saved_inv_std->mutable_data<BNParamType>();
      auto* running_mean = Output(RUNNING_MEAN);
      auto* running_var = Output(RUNNING_VAR);
      if (running_mean->size() != C) {
        running_mean->Resize(C);
        math::Set<BNParamType, HIPContext>(
            C,
            BNParamType(0),
            running_mean->mutable_data<BNParamType>(),
            &context_);
      }
      if (running_var->size() != C) {
        running_var->Resize(C);
        math::Set<BNParamType, HIPContext>(
            C,
            BNParamType(0),
            running_var->mutable_data<BNParamType>(),
            &context_);
      }
      BNParamType* running_mean_data =
          running_mean->mutable_data<BNParamType>();
      BNParamType* running_var_data = running_var->mutable_data<BNParamType>();
      if (N == 0) {
        math::Set<BNParamType, HIPContext>(
            C, BNParamType(0), saved_mean_data, &context_);
        math::Set<BNParamType, HIPContext>(
            C, BNParamType(0), saved_inv_std_data, &context_);
        return true;
      }
      const double alpha = static_cast<double>(1.0f - momentum_);
      CUDNN_ENFORCE(hipdnnBatchNormalizationForwardTraining(
          cudnn_wrapper_.inline_cudnn_handle(),
          mode_,
          cudnnTypeWrapper<T>::kOne(),
          cudnnTypeWrapper<T>::kZero(),
          data_desc_,
          X_data,
          data_desc_,
          Y_data,
          param_desc_,
          scale_data,
          bias_data,
          alpha,
          running_mean_data,
          running_var_data,
          epsilon_,
          saved_mean_data,
          saved_inv_std_data));
    }
    return true;
  }

 private:
  CuDNNWrapper cudnn_wrapper_;
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t param_desc_;
  hipdnnBatchNormMode_t mode_;

  std::vector<int> data_dims_;
};

class CuDNNSpatialBNGradientOp final : public SpatialBNGradientOp<HIPContext> {
 public:
  USE_OPERATOR_FUNCTIONS(HIPContext);

  CuDNNSpatialBNGradientOp(const OperatorDef& operator_def, Workspace* ws)
      : SpatialBNGradientOp<HIPContext>(operator_def, ws),
        cudnn_wrapper_(&context_),
#if CUDNN_VERSION_MIN(7, 0, 0)
        // TODO(T31829456): The new HIPDNN_BATCHNORM_SPATIAL_PERSISTENT mode was
        // introduced in CuDNN 7 for performance optimization, but it results in
        // accuracy losses in convolution models such as ResNeXt-101 and
        // video R(2+1)D. We will fall back to the normal
        // HIPDNN_BATCHNORM_SPATIAL for now
        mode_(HIPDNN_BATCHNORM_SPATIAL) {
#else
        mode_(HIPDNN_BATCHNORM_SPATIAL) {
#endif
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&data_desc_));
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&param_desc_));
    if (epsilon_ < HIPDNN_BN_MIN_EPSILON) {
      LOG(ERROR) << "Provided epsilon is smaller than HIPDNN_BN_MIN_EPSILON. "
                    "Setting it to HIPDNN_BN_MIN_EPSILON instead.";
      epsilon_ = HIPDNN_BN_MIN_EPSILON;
    }
  }

  ~CuDNNSpatialBNGradientOp() {
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(data_desc_));
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(param_desc_));
  }

  bool RunOnDevice() override {
    // CuDNN doesn't support multi-batch SpatialBN and it's NHWC order SpatialBN
    // is much slower, so in such cases fallback to SpatialBNOp<HIPContext>.
    if (num_batches_ > 1 || order_ == StorageOrder::NHWC) {
      return SpatialBNGradientOp<HIPContext>::RunOnDevice();
    }
    return DispatchHelper<TensorTypes<float, at::Half>>::call(this, Input(0));
  }

  template <typename T>
  bool DoRunWithType() {
    typedef typename cudnnTypeWrapper<T>::BNParamType BNParamType;

    const auto& X = Input(INPUT);
    const auto& scale = Input(SCALE);
    const auto& dY = Input(OUTPUT_GRAD);
    const auto& saved_mean = Input(SAVED_MEAN);
    const auto& saved_rstd = Input(SAVED_INV_STD);
    auto* dX = Output(INPUT_GRAD);
    auto* dscale = Output(SCALE_GRAD);
    auto* dbias = Output(BIAS_GRAD);
    const int ndim = X.ndim();
    CAFFE_ENFORCE_GE(ndim, 3);
    const int N = X.dim32(0);
    const int C =
        (order_ == StorageOrder::NCHW ? X.dim32(1) : X.dim32(ndim - 1));
    CAFFE_ENFORCE_EQ(scale.size(), C);
    CAFFE_ENFORCE_EQ(saved_mean.size(), C);
    CAFFE_ENFORCE_EQ(saved_rstd.size(), C);
    dX->ResizeLike(X);
    dscale->ResizeLike(scale);
    dbias->ResizeLike(scale);
    const T* X_data = X.template data<T>();
    const T* scale_data = scale.template data<T>();
    const T* dY_data = dY.template data<T>();
    const BNParamType* saved_mean_data =
        saved_mean.template data<BNParamType>();
    const BNParamType* saved_rstd_data =
        saved_rstd.template data<BNParamType>();
    T* dX_data = dX->template mutable_data<T>();
    BNParamType* dscale_data = dscale->template mutable_data<BNParamType>();
    BNParamType* dbias_data = dbias->template mutable_data<BNParamType>();
    if (N == 0) {
      math::Set<BNParamType, HIPContext>(
          C, BNParamType(0), dscale_data, &context_);
      math::Set<BNParamType, HIPContext>(
          C, BNParamType(0), dbias_data, &context_);
      return true;
    }

    const std::vector<int> input_dims(X.sizes().cbegin(), X.sizes().cend());
    if (input_dims != data_dims_) {
      data_dims_ = input_dims;
      SetTensorDescriptor(
          cudnnTypeWrapper<T>::type,
          mode_,
          order_,
          input_dims,
          data_desc_,
          param_desc_);
    }
    CUDNN_ENFORCE(hipdnnBatchNormalizationBackward(
        cudnn_wrapper_.inline_cudnn_handle(),
        mode_,
        cudnnTypeWrapper<T>::kOne(),
        cudnnTypeWrapper<T>::kZero(),
        cudnnTypeWrapper<T>::kOne(),
        cudnnTypeWrapper<T>::kZero(),
        data_desc_,
        X_data,
        data_desc_,
        dY_data,
        data_desc_,
        dX_data,
        param_desc_,
        scale_data,
        dscale_data,
        dbias_data,
        epsilon_,
        saved_mean_data,
        saved_rstd_data));

    return true;
  }

 private:
  CuDNNWrapper cudnn_wrapper_;
  hipdnnTensorDescriptor_t data_desc_;
  hipdnnTensorDescriptor_t param_desc_;
  hipdnnBatchNormMode_t mode_;

  // TODO: int -> int64_t
  std::vector<int> data_dims_;
};

REGISTER_CUDNN_OPERATOR(SpatialBN, CuDNNSpatialBNOp);
REGISTER_CUDNN_OPERATOR(SpatialBNGradient, CuDNNSpatialBNGradientOp);

} // namespace caffe2

#endif // CUDNN_VERSION_MIN(5, 0, 0)
