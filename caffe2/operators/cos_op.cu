#include "hip/hip_runtime.h"
#include "caffe2/operators/cos_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void
CosGradientCUDAKernel(const int N, const T* dY, const T* X, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    dX[i] = -__ldg(dY + i) * sin(__ldg(X + i));
#else
    dX[i] = -dY[i] * sin(X[i]);
#endif
  }
}

} // namespace

template <>
template <typename T>
bool CosGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& X_dims,
    const std::vector<int>& /* dY_dims */,
    const T* X,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      X_dims.cbegin(), X_dims.cend(), 1, std::multiplies<int>());
  CosGradientCUDAKernel<<<
      CAFFE_GET_BLOCKS(size),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(size, dY, X, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    Cos,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        CosFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    CosGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        CosGradientFunctor<HIPContext>>);

} // namespace caffe2
