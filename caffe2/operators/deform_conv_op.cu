#include "hip/hip_runtime.h"
/*!
 ******************* BEGIN Caffe Copyright Notice and Disclaimer ****************
 *
 * COPYRIGHT
 *
 * All contributions by the University of California:
 * Copyright (c) 2014-2017 The Regents of the University of California (Regents)
 * All rights reserved.
 *
 * All other contributions:
 * Copyright (c) 2014-2017, the respective contributors
 * All rights reserved.
 *
 * Caffe uses a shared copyright model: each contributor holds copyright over
 * their contributions to Caffe. The project versioning records all such
 * contribution and copyright details. If a contributor wants to further mark
 * their specific copyright on a particular contribution, they should indicate
 * their copyright solely in the commit message of the change when it is
 * committed.
 *
 * LICENSE
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
 * ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * CONTRIBUTION AGREEMENT
 *
 * By contributing to the BVLC/caffe repository through pull-request, comment,
 * or otherwise, the contributor releases their content to the
 * license and copyright terms herein.
 *
 ***************** END Caffe Copyright Notice and Disclaimer ********************
 *
 * Copyright (c) 2017 Microsoft
 * Licensed under The Apache-2.0 License [see LICENSE for details]
 * \file deformable_im2col.cuh
 * \brief Function definitions of converting an image to
 * column matrix based on kernel, padding, dilation, and offset.
 * These functions are mainly used in deformable convolution operators.
 * \ref: https://arxiv.org/abs/1703.06211
 * \author Yuwen Xiong, Haozhi Qi, Jifeng Dai
 */

#include <cub/block/block_reduce.cuh>
#include <vector>
#include "caffe2/core/common.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/deform_conv_op.h"
#include "caffe2/operators/deform_conv_op_impl.h"

namespace caffe2 {

typedef int64_t index_t;
typedef std::vector<int64_t> TShape;

template <typename DType>
__device__ DType deformable_im2col_bilinear(
    const DType* bottom_data,
    const int data_width,
    const int height,
    const int width,
    DType h,
    DType w) {
  int h_low = floor(h);
  int w_low = floor(w);
  int h_high;
  int w_high;
  if (h_low >= height - 1) {
    h_high = h_low = height - 1;
    h = (DType)h_low;
  } else {
    h_high = h_low + 1;
  }

  if (w_low >= width - 1) {
    w_high = w_low = width - 1;
    w = (DType)w_low;
  } else {
    w_high = w_low + 1;
  }

  DType lh = h - h_low;
  DType lw = w - w_low;
  DType hh = 1 - lh, hw = 1 - lw;

  DType v1 = bottom_data[h_low * data_width + w_low];
  DType v2 = bottom_data[h_low * data_width + w_high];
  DType v3 = bottom_data[h_high * data_width + w_low];
  DType v4 = bottom_data[h_high * data_width + w_high];
  DType w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  DType val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename DType>
__device__ DType get_gradient_weight(
    DType argmax_h,
    DType argmax_w,
    const int h,
    const int w,
    const int height,
    const int width) {
  if (argmax_h < 0 || argmax_h > height || argmax_w < 0 || argmax_w > width) {
    // empty
    return 0;
  }

  argmax_h = max(argmax_h, (DType)0.0f);
  argmax_w = max(argmax_w, (DType)0.0f);

  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (DType)argmax_h_low;
  } else {
    argmax_h_high = argmax_h_low + 1;
  }
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (DType)argmax_w_low;
  } else {
    argmax_w_high = argmax_w_low + 1;
  }
  DType weight = 0;
  if (h == argmax_h_low) {
    if (w == argmax_w_low) {
      weight = (h + 1 - argmax_h) * (w + 1 - argmax_w);
    } else if (w == argmax_w_high) {
      weight = (h + 1 - argmax_h) * (argmax_w + 1 - w);
    }
  } else if (h == argmax_h_high) {
    if (w == argmax_w_low) {
      weight = (argmax_h + 1 - h) * (w + 1 - argmax_w);
    } else if (w == argmax_w_high) {
      weight = (argmax_h + 1 - h) * (argmax_w + 1 - w);
    }
  }
  return weight;
}

template <typename DType>
__device__ DType get_coordinate_weight(
    DType argmax_h,
    DType argmax_w,
    const int height,
    const int width,
    const DType* im_data,
    const int data_width,
    const int bp_dir) {
  if (argmax_h < 0 || argmax_h > height || argmax_w < 0 || argmax_w > width) {
    // empty
    return 0;
  }

  if (argmax_h < 0)
    argmax_h = 0;
  if (argmax_w < 0)
    argmax_w = 0;

  int argmax_h_low = (int)argmax_h;
  int argmax_w_low = (int)argmax_w;
  int argmax_h_high;
  int argmax_w_high;
  if (argmax_h_low >= height - 1) {
    argmax_h_high = argmax_h_low = height - 1;
    argmax_h = (DType)argmax_h_low;
  } else {
    argmax_h_high = argmax_h_low + 1;
  }
  if (argmax_w_low >= width - 1) {
    argmax_w_high = argmax_w_low = width - 1;
    argmax_w = (DType)argmax_w_low;
  } else {
    argmax_w_high = argmax_w_low + 1;
  }
  DType weight = 0;

  if (bp_dir == 0) {
    weight += -1 * (argmax_w_low + 1 - argmax_w) *
        im_data[argmax_h_low * data_width + argmax_w_low];
    weight += -1 * (argmax_w - argmax_w_low) *
        im_data[argmax_h_low * data_width + argmax_w_high];
    weight += (argmax_w_low + 1 - argmax_w) *
        im_data[argmax_h_high * data_width + argmax_w_low];
    weight += (argmax_w - argmax_w_low) *
        im_data[argmax_h_high * data_width + argmax_w_high];
  } else if (bp_dir == 1) {
    weight += -1 * (argmax_h_low + 1 - argmax_h) *
        im_data[argmax_h_low * data_width + argmax_w_low];
    weight += (argmax_h_low + 1 - argmax_h) *
        im_data[argmax_h_low * data_width + argmax_w_high];
    weight += -1 * (argmax_h - argmax_h_low) *
        im_data[argmax_h_high * data_width + argmax_w_low];
    weight += (argmax_h - argmax_h_low) *
        im_data[argmax_h_high * data_width + argmax_w_high];
  }

  return weight;
}

/*!
 * \brief deformable_im2col gpu kernel.
 * DO NOT call this directly. Use wrapper function im2col() instead;
 */
template <typename DType>
__global__ void deformable_im2col_gpu_kernel(
    const int n,
    const DType* data_im,
    const DType* data_offset,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int height_col,
    const int width_col,
    DType* data_col) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    // index index of output matrix
    const int w_col = index % width_col;
    const int h_col = (index / width_col) % height_col;
    const int c_im = (index / width_col) / height_col;
    const int c_col = c_im * kernel_h * kernel_w;

    // compute deformable group index
    const int deformable_group_index = c_im / channel_per_deformable_group;

    const int h_in = h_col * stride_h - pad_h;
    const int w_in = w_col * stride_w - pad_w;
    DType* data_col_ptr =
        data_col + (c_col * height_col + h_col) * width_col + w_col;
    const DType* data_im_ptr = data_im + (c_im * height + h_in) * width + w_in;
    const DType* data_offset_ptr = data_offset +
        deformable_group_index * 2 * kernel_h * kernel_w * height_col *
            width_col;

    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        const int data_offset_h_ptr =
            ((2 * (i * kernel_w + j)) * height_col + h_col) * width_col + w_col;
        const int data_offset_w_ptr =
            ((2 * (i * kernel_w + j) + 1) * height_col + h_col) * width_col +
            w_col;
        const DType offset_h = data_offset_ptr[data_offset_h_ptr];
        const DType offset_w = data_offset_ptr[data_offset_w_ptr];
        DType val = static_cast<DType>(0);
        const DType h_im = h_in + i * dilation_h + offset_h;
        const DType w_im = w_in + j * dilation_w + offset_w;
        if (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) {
          const DType map_h = i * dilation_h + offset_h;
          const DType map_w = j * dilation_w + offset_w;
          const int cur_height = height - h_in;
          const int cur_width = width - w_in;
          val = deformable_im2col_bilinear(
              data_im_ptr, width, cur_height, cur_width, map_h, map_w);
        }
        *data_col_ptr = val;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

/*!\brief
 * cpu function of deformable_im2col algorithm
 * \param s device stream
 * \param data_im pointer of an image (C, H, W, ...) in the image batch
 * \param data_offset pointer of offset (C, H, W, ...) in the offset batch
 * \param im_shape input image shape in dimensions (N, C, H, W,)
 * \param col_shape column buffer shape (#channels, output_im_height,
 * output_im_width, ...) \param kernel_shape kernel filter shape \param pad pad
 * shape \param stride stride shape \param dilation dilation shape \param
 * deformable_group #offset group that deformable convolution use \param
 * data_col column buffer pointer
 */
template <typename DType, typename Context>
void DeformConvOpBase<DType, Context>::DeformableIm2col(
    const DType* data_im,
    const DType* data_offset,
    at::IntArrayRef im_shape,
    at::IntArrayRef col_shape,
    DType* data_col) {
  CHECK_LT(2, CAFFE_CUDA_NUM_THREADS);
  CAFFE_ENFORCE_EQ(pad_t(), pad_b());
  CAFFE_ENFORCE_EQ(pad_l(), pad_r());
  const int pad_h = pad_t();
  const int pad_w = pad_l();
  index_t channel_per_deformable_group = im_shape[1] / deformable_group_;
  index_t num_kernels = im_shape[1] * size_from_dim_(1, col_shape);
  deformable_im2col_gpu_kernel<DType>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          num_kernels,
          data_im,
          data_offset,
          im_shape[2],
          im_shape[3],
          kernel_h(),
          kernel_w(),
          pad_h,
          pad_w,
          stride_h(),
          stride_w(),
          dilation_h(),
          dilation_w(),
          channel_per_deformable_group,
          col_shape[1],
          col_shape[2],
          data_col);
}

/*!
 * \brief deformable_col2im gpu kernel.
 * \brief DO NOT call this directly. Use wrapper function deformable_col2im()
 * instead;
 */
template <typename DType>
__global__ void deformable_col2im_gpu_kernel(
    const int n,
    const DType* data_col,
    const DType* data_offset,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int height_col,
    const int width_col,
    DType* grad_im) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int j = (index / width_col / height_col) % kernel_w;
    const int i = (index / width_col / height_col / kernel_w) % kernel_h;
    const int c = index / width_col / height_col / kernel_w / kernel_h;
    // compute the start and end of the output

    const int deformable_group_index = c / channel_per_deformable_group;

    int w_out = index % width_col;
    int h_out = (index / width_col) % height_col;
    int w_in = w_out * stride_w - pad_w;
    int h_in = h_out * stride_h - pad_h;

    const DType* data_offset_ptr = data_offset +
        deformable_group_index * 2 * kernel_h * kernel_w * height_col *
            width_col;
    const int data_offset_h_ptr =
        ((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out;
    const int data_offset_w_ptr =
        ((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col + w_out;
    const DType offset_h = data_offset_ptr[data_offset_h_ptr];
    const DType offset_w = data_offset_ptr[data_offset_w_ptr];
    const DType cur_inv_h_data = h_in + i * dilation_h + offset_h;
    const DType cur_inv_w_data = w_in + j * dilation_w + offset_w;

    const DType cur_top_grad = data_col[index];
    const int cur_h = (int)cur_inv_h_data;
    const int cur_w = (int)cur_inv_w_data;
    for (int dy = -2; dy <= 2; dy++) {
      for (int dx = -2; dx <= 2; dx++) {
        if (cur_h + dy >= 0 && cur_h + dy < height && cur_w + dx >= 0 &&
            cur_w + dx < width &&
            c10::cuda::compat::abs(cur_inv_h_data - (cur_h + dy)) < 1 &&
            c10::cuda::compat::abs(cur_inv_w_data - (cur_w + dx)) < 1) {
          int cur_bottom_grad_pos =
              (c * height + cur_h + dy) * width + cur_w + dx;
          DType weight = get_gradient_weight(
              cur_inv_h_data,
              cur_inv_w_data,
              cur_h + dy,
              cur_w + dx,
              height,
              width);
          atomicAdd(grad_im + cur_bottom_grad_pos, weight * cur_top_grad);
        }
      }
    }
  }
}

/*!\brief
 * gpu function of deformable_col2im algorithm
 * \param s device stream
 * \param data_col start pointer of the column buffer to be filled
 * \param data_offset pointer of offset (C, H, W, ...) in the offset batch
 * \param im_shape input image shape in dimensions (N, C, H, W,)
 * \param col_shape column buffer shape
 * \param kernel_shape kernel filter shape
 * \param pad pad shape
 * \param stride stride shape
 * \param dilation dilation shape
 * \param deformable_group #offset group that deformable convolution use
 * \param grad_im pointer of a image (C, H, W,...) in the image batch
 */
template <typename DType, typename Context>
void DeformConvOpBase<DType, Context>::DeformableCol2im(
    const DType* data_col,
    const DType* data_offset,
    at::IntArrayRef im_shape,
    at::IntArrayRef col_shape,
    DType* grad_im) {
  CAFFE_ENFORCE_EQ(pad_t(), pad_b());
  CAFFE_ENFORCE_EQ(pad_l(), pad_r());
  const int pad_h = pad_t();
  const int pad_w = pad_l();
  index_t im_size = size_from_dim_(1, im_shape);
  index_t channel_per_deformable_group = im_shape[1] / deformable_group_;
  index_t num_kernels = size_from_dim_(0, col_shape);
  // num_axes should be smaller than block size
  CHECK_LT(2, CAFFE_CUDA_NUM_THREADS);
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  deformable_col2im_gpu_kernel<DType>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          num_kernels,
          data_col,
          data_offset,
          im_shape[1],
          im_shape[2],
          im_shape[3],
          kernel_h(),
          kernel_w(),
          pad_h,
          pad_w,
          stride_h(),
          stride_w(),
          dilation_h(),
          dilation_w(),
          channel_per_deformable_group,
          col_shape[1],
          col_shape[2],
          grad_im);
}

/*!
 * \brief deformable_col2im_coord gpu kernel.
 * \brief DO NOT call this directly. Use wrapper function
 * deformable_col2im_coord() instead;
 */
template <typename DType>
__global__ void deformable_col2im_coord_gpu_kernel(
    const int n,
    const DType* data_col,
    const DType* data_im,
    const DType* data_offset,
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int pad_h,
    const int pad_w,
    const int stride_h,
    const int stride_w,
    const int dilation_h,
    const int dilation_w,
    const int channel_per_deformable_group,
    const int height_col,
    const int width_col,
    DType* grad_offset) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    DType val = 0;
    int w = index % width_col;
    int h = (index / width_col) % height_col;
    int c = index / width_col / height_col;
    // compute the start and end of the output

    const int deformable_group_index = c / (2 * kernel_h * kernel_w);
    const int col_step = kernel_h * kernel_w;
    int cnt = 0;
    const DType* data_col_ptr = data_col +
        deformable_group_index * channel_per_deformable_group * width_col *
            height_col;
    const DType* data_im_ptr = data_im +
        deformable_group_index * channel_per_deformable_group / kernel_h /
            kernel_w * height * width;
    const DType* data_offset_ptr = data_offset +
        deformable_group_index * 2 * kernel_h * kernel_w * height_col *
            width_col;

    const int offset_c = c - deformable_group_index * 2 * kernel_h * kernel_w;

    for (int col_c = (offset_c / 2); col_c < channel_per_deformable_group;
         col_c += col_step) {
      const int col_pos = ((col_c * height_col) + h) * width_col + w;
      const int bp_dir = offset_c % 2;

      int j = (col_pos / width_col / height_col) % kernel_w;
      int i = (col_pos / width_col / height_col / kernel_w) % kernel_h;
      int w_out = col_pos % width_col;
      int h_out = (col_pos / width_col) % height_col;
      int w_in = w_out * stride_w - pad_w;
      int h_in = h_out * stride_h - pad_h;
      const int data_offset_h_ptr =
          (((2 * (i * kernel_w + j)) * height_col + h_out) * width_col + w_out);
      const int data_offset_w_ptr =
          (((2 * (i * kernel_w + j) + 1) * height_col + h_out) * width_col +
           w_out);
      const DType offset_h = data_offset_ptr[data_offset_h_ptr];
      const DType offset_w = data_offset_ptr[data_offset_w_ptr];
      DType inv_h = h_in + i * dilation_h + offset_h;
      DType inv_w = w_in + j * dilation_w + offset_w;
      if (inv_h < 0 || inv_w < 0 || inv_h >= height || inv_w >= width) {
        inv_h = inv_w = -1;
      }
      const DType weight = get_coordinate_weight(
          inv_h,
          inv_w,
          height,
          width,
          data_im_ptr + cnt * height * width,
          width,
          bp_dir);
      val += weight * data_col_ptr[col_pos];
      cnt += 1;
    }

    grad_offset[index] = val;
  }
}

/*!\brief
 * gpu function of deformable_col2im_coord algorithm
 * \param s device stream
 * \param data_col start pointer of the column buffer to be filled
 * \param data_im pointer of an image (C, H, W, ...) in the image batch
 * \param data_offset pointer of offset (C, H, W, ...) in the offset batch
 * \param im_shape input image shape in dimensions (N, C, H, W,)
 * \param col_shape column buffer shape
 * \param kernel_shape kernel filter shape
 * \param pad pad shape
 * \param stride stride shape
 * \param dilation dilation shape
 * \param deformable_group #offset group that deformable convolution use
 * \param grad_offset pointer of the offset (C, H, W,...) in the offset batch
 */
template <typename DType, typename Context>
void DeformConvOpBase<DType, Context>::DeformableCol2imCoord(
    const DType* data_col,
    const DType* data_im,
    const DType* data_offset,
    at::IntArrayRef im_shape,
    at::IntArrayRef col_shape,
    DType* grad_offset) {
  CAFFE_ENFORCE_EQ(pad_t(), pad_b());
  CAFFE_ENFORCE_EQ(pad_l(), pad_r());
  const int pad_h = pad_t();
  const int pad_w = pad_l();
  index_t num_kernels = col_shape[1] * col_shape[2] * 2 * kernel_h() *
      kernel_w() * deformable_group_;
  index_t channel_per_deformable_group = col_shape[0] / deformable_group_;
  // num_axes should be smaller than block size
  CHECK_LT(2, CAFFE_CUDA_NUM_THREADS);
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  deformable_col2im_coord_gpu_kernel<DType>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          num_kernels,
          data_col,
          data_im,
          data_offset,
          im_shape[1],
          im_shape[2],
          im_shape[3],
          kernel_h(),
          kernel_w(),
          pad_h,
          pad_w,
          stride_h(),
          stride_w(),
          dilation_h(),
          dilation_w(),
          channel_per_deformable_group,
          col_shape[1],
          col_shape[2],
          grad_offset);
}

REGISTER_HIP_OPERATOR(DeformConv, DeformConvOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    DeformConvGradient,
    DeformConvGradientOp<float, HIPContext>);

} // namespace caffe2
