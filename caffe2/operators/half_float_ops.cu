#include "hip/hip_runtime.h"
#include "caffe2/operators/half_float_ops.h"

#include "caffe2/core/context_gpu.h"

#ifdef CAFFE_HAS_CUDA_FP16

namespace caffe2 {
namespace {
__global__ void FloatToHalfKernel(const int N, const float* X, half* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __float2half(X[i]);
  }
}

__global__ void HalfToFloatKernel(const int N, const half* X, float* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = __half2float(X[i]);
  }
}
}

template <>
bool FloatToHalfOp<HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  auto* Y = Output(0, X.sizes(), at::dtype<at::Half>());
  FloatToHalfKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.numel(),
      X.data<float>(),
      reinterpret_cast<half*>(Y->template mutable_data<at::Half>()));
  return true;
}

template <>
bool HalfToFloatOp<HIPContext>::RunOnDevice() {
  auto& X = Input(0);

  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  HalfToFloatKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.numel(),
      reinterpret_cast<const half*>(X.data<at::Half>()),
      Y->template mutable_data<float>());
  return true;
}

template <>
bool Float16UniformFillOp<HIPContext>::RunOnDevice() {
  auto* output = Output(0, shape_, at::dtype<at::Half>());
  at::Half* out = output->template mutable_data<at::Half>();

  auto leading_dim_sz = output->size(0);
  CAFFE_ENFORCE_GT(leading_dim_sz, 0,
      "The input shape should have the first dimension greater than 0");
  int rowsz = output->numel() / output->size(0);

  ReinitializeTensor(
    &temp_data_buffer_, {rowsz}, at::dtype<float>().device(CUDA));
  float* temp_data = temp_data_buffer_.template mutable_data<float>();

  for (uint64_t i = 0; i < leading_dim_sz; i++) {
    math::RandUniform<float, HIPContext>(
        rowsz, min_, max_, temp_data, &context_);

    FloatToHalfKernel<<<
      CAFFE_GET_BLOCKS(rowsz),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      rowsz,
      temp_data,
      reinterpret_cast<half*>(out + i * rowsz));
  }

  return true;
}

REGISTER_HIP_OPERATOR(FloatToHalf, FloatToHalfOp<HIPContext>);
REGISTER_HIP_OPERATOR(HalfToFloat, HalfToFloatOp<HIPContext>);
REGISTER_HIP_OPERATOR(Float16UniformFill, Float16UniformFillOp<HIPContext>);
} // namespace caffe2

#endif // CAFFE_HAS_CUDA_FP16
