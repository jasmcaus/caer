#include "hip/hip_runtime.h"
#include "caffe2/core/hip/common_miopen.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/core/cudnn_wrappers.h"
#include "caffe2/operators/conv_op.h"
#include "caffe2/operators/conv_op_cache_cudnn.h"
#include "caffe2/operators/conv_pool_op_base.h"

// Adopted from caffe2 depthwise conv at
// pytorch/caffe2/caffe2/operators/depthwise_3x3_conv_op_cudnn.cu

namespace caffe2 {

struct DepthwiseArgs {
  // Input layer dimensions
  int batch{0};
  int in_rows{0};
  int in_cols{0};
  int in_length{0};
  int in_depth{0};

  // filter size
  int filter_rows{0};
  int filter_cols{0};
  int filter_length{0};

  // strides and pads
  int stride{0};
  int temporal_stride{0};
  int pad_rows{0};
  int pad_cols{0};
  int pad_length{0};

  // Output layer dimensions
  int out_rows{0};
  int out_cols{0};
  int out_length{0};
  int out_depth{0};
};

template <typename T>
__global__ void DepthwiseConv3dGPUKernelNCHW(
    const DepthwiseArgs args,
    const T* input,
    const T* filter,
    T* output,
    int num_outputs) {
  const int in_rows = args.in_rows;
  const int in_cols = args.in_cols;
  const int in_length = args.in_length;
  const int in_depth = args.in_depth;
  const int filter_rows = args.filter_rows;
  const int filter_cols = args.filter_cols;
  const int filter_length = args.filter_length;
  const int stride = args.stride;
  const int temporal_stride = args.temporal_stride;
  const int pad_rows = args.pad_rows;
  const int pad_cols = args.pad_cols;
  const int pad_length = args.pad_length;
  const int out_rows = args.out_rows;
  const int out_cols = args.out_cols;
  const int out_length = args.out_length;
  const int out_depth = args.out_depth;

  CUDA_1D_KERNEL_LOOP(thread_id, num_outputs) {
    const int OW = thread_id % out_cols;
    const int OH = (thread_id / out_cols) % out_rows;
    const int OL = (thread_id / out_cols / out_rows) % out_length;
    const int OC = (thread_id / out_cols / out_rows / out_length) % out_depth;
    const int OB = thread_id / out_cols / out_rows / out_length / out_depth;
    const int in_d = OC;

    const int input_offset_temp =
        (OB * in_depth + OC) * (in_length * in_rows * in_cols);
    const int input_row_start = OH * stride - pad_rows;
    const int input_col_start = OW * stride - pad_cols;
    const int input_length_start = OL * temporal_stride - pad_length;
    const int input_row_end = input_row_start + filter_rows;
    const int input_col_end = input_col_start + filter_cols;
    const int input_length_end = input_length_start + filter_length;
    const float* filter_start =
        filter + in_d * filter_rows * filter_cols * filter_length;

    T sum = 0;
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_length_start >= 0 && input_row_end < in_rows &&
        input_col_end < in_cols && input_length_end < in_length) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int f_l = 0; f_l < filter_length; ++f_l) {
        const int in_l = input_length_start + f_l;
#pragma unroll
        for (int f_r = 0; f_r < filter_rows; ++f_r) {
          const int in_r = input_row_start + f_r;
          const float* filter_offset = filter_start +
              filter_cols * filter_rows * f_l + filter_cols * f_r;
#pragma unroll
          for (int f_c = 0; f_c < filter_cols; ++f_c) {
            const int in_c = input_col_start + f_c;

            const int input_offset = (input_offset_temp) +
                (in_l * in_cols * in_rows) + (in_r * in_cols) + in_c;
#if __CUDA_ARCH__ >= 350
            sum += __ldg(input + input_offset) * __ldg(filter_offset + f_c);
#else
            sum += input[input_offset] * filter_offset[f_c];
#endif
          }
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int f_l = 0; f_l < filter_length; ++f_l) {
        const int in_l = input_length_start + f_l;
#pragma unroll
        for (int f_r = 0; f_r < filter_rows; ++f_r) {
          const int in_r = input_row_start + f_r;
          const float* filter_offset = filter_start +
              filter_cols * filter_rows * f_l + filter_cols * f_r;
#pragma unroll
          for (int f_c = 0; f_c < filter_cols; ++f_c) {
            const int in_c = input_col_start + f_c;
            if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols &&
                in_l >= 0 && in_l < in_length) {
              const int input_offset = (input_offset_temp) +
                  (in_l * in_cols * in_rows) + (in_r * in_cols) + in_c;
#if __CUDA_ARCH__ >= 350
              sum += __ldg(input + input_offset) * __ldg(filter_offset + f_c);
#else
              sum += input[input_offset] * filter_offset[f_c];
#endif
            }
          }
        }
      }
    }

    output[thread_id] = sum;
  }
}

// A Cuda kernel to compute the depthwise convolution backprop w.r.t. filter.
template <typename T>
__global__ void DepthwiseConv3dBackpropFilterGPUKernelNCHW(
    const DepthwiseArgs args,
    const T* out_backprop,
    const T* input,
    T* filter_backprop,
    int num_out_backprop) {
  const int in_rows = args.in_rows;
  const int in_cols = args.in_cols;
  const int in_length = args.in_length;
  const int in_depth = args.in_depth;
  const int filter_rows = args.filter_rows;
  const int filter_cols = args.filter_cols;
  const int filter_length = args.filter_length;
  const int stride = args.stride;
  const int temporal_stride = args.temporal_stride;
  const int pad_rows = args.pad_rows;
  const int pad_cols = args.pad_cols;
  const int pad_length = args.pad_length;
  const int out_rows = args.out_rows;
  const int out_cols = args.out_cols;
  const int out_length = args.out_length;
  const int out_depth = args.out_depth;

  CUDA_1D_KERNEL_LOOP(thread_id, num_out_backprop) {
    // Compute the indexes of this thread in the output.
    const int OW = thread_id % out_cols;
    const int OH = (thread_id / out_cols) % out_rows;
    const int OL = (thread_id / out_cols / out_rows) % out_length;
    const int OC = (thread_id / out_cols / out_rows / out_length) % out_depth;
    const int OB = thread_id / out_cols / out_rows / out_length / out_depth;

    // Compute the input depth and the index of depth multiplier.
    const int in_d = OC;

    // Decide if all input is valid, if yes, we can skip the boundary checks
    // for each input.
    const int in_r_start = OH * stride - pad_rows;
    const int in_c_start = OW * stride - pad_cols;
    const int in_l_start = OL * temporal_stride - pad_length;
    const int in_r_end = in_r_start + filter_rows;
    const int in_c_end = in_c_start + filter_cols;
    const int in_l_end = in_l_start + filter_length;

    const int out_backprop_offset =
        (OB * out_depth * out_length * out_rows * out_cols) +
        (OC * out_length * out_rows * out_cols) + (OL * out_rows * out_cols) +
        (OH * out_cols) + (OW);

#if __CUDA_ARCH__ >= 350
    const T out_bp = __ldg(out_backprop + out_backprop_offset);
#else
    const T out_bp = out_backprop[out_backprop_offset];
#endif
    if (in_r_start >= 0 && in_c_start >= 0 && in_r_end < in_rows &&
        in_c_end < in_cols && in_l_start >= 0 && in_l_end < in_length) {
#pragma unroll
      for (int f_l = 0; f_l < filter_length; ++f_l) {
        const int in_l = in_l_start + f_l;
#pragma unroll
        for (int f_r = 0; f_r < filter_rows; ++f_r) {
          const int in_r = in_r_start + f_r;
          // Avoid repeated computation.
          const int input_offset_temp =
              (OB * in_depth * in_length * in_rows * in_cols) +
              (OC * in_length * in_rows * in_cols) +
              (in_l * in_rows * in_cols) + (in_r * in_cols);

#pragma unroll
          for (int f_c = 0; f_c < filter_cols; ++f_c) {
            const int in_c = in_c_start + f_c;
            const int input_offset = input_offset_temp + in_c;
#if __CUDA_ARCH__ >= 350
            T partial_sum = __ldg(input + input_offset) * out_bp;
#else
            T partial_sum = input[input_offset] * out_bp;
#endif
            T* addr = filter_backprop +
                (in_d * filter_rows * filter_cols * filter_length) +
                (f_l * filter_rows * filter_cols) + (f_c + filter_cols * f_r);
            atomicAdd(addr, partial_sum);
          }
        }
      }
    } else {
#pragma unroll
      for (int f_l = 0; f_l < filter_length; ++f_l) {
        const int in_l = in_l_start + f_l;
#pragma unroll
        for (int f_r = 0; f_r < filter_rows; ++f_r) {
          const int in_r = in_r_start + f_r;
          // Avoid repeated computation.
          const int input_offset_temp =
              (OB * in_depth * in_length * in_rows * in_cols) +
              (OC * in_length * in_rows * in_cols) +
              (in_l * in_rows * in_cols) + (in_r * in_cols);
#pragma unroll
          for (int f_c = 0; f_c < filter_cols; ++f_c) {
            const int in_c = in_c_start + f_c;

            if (in_r >= 0 && in_r < in_rows && in_c >= 0 && in_c < in_cols &&
                in_l >= 0 && in_l < in_length) {
              const int input_offset = input_offset_temp + in_c;
#if __CUDA_ARCH__ >= 350
              T partial_sum = __ldg(input + input_offset) * out_bp;
#else
              T partial_sum = input[input_offset] * out_bp;
#endif
              T* addr = filter_backprop +
                  (in_d * filter_rows * filter_cols * filter_length) +
                  (f_l * filter_rows * filter_cols) + (f_c + filter_cols * f_r);
              atomicAdd(addr, partial_sum);
            }
          }
        }
      }
    }
  }
}

template <typename T>
__global__ void DepthwiseConv3dBackpropInputGPUKernelNCHW(
    const DepthwiseArgs args,
    const T* out_backprop,
    const T* filter,
    T* in_backprop,
    int num_in_backprop) {
  const int in_rows = args.in_rows;
  const int in_cols = args.in_cols;
  const int in_length = args.in_length;
  const int in_depth = args.in_depth;
  const int filter_rows = args.filter_rows;
  const int filter_cols = args.filter_cols;
  const int filter_length = args.filter_length;
  const int stride = args.stride;
  const int temporal_stride = args.temporal_stride;
  const int pad_rows = args.pad_rows;
  const int pad_cols = args.pad_cols;
  const int pad_length = args.pad_length;
  const int out_rows = args.out_rows;
  const int out_cols = args.out_cols;
  const int out_length = args.out_length;
  const int out_depth = args.out_depth;

  CUDA_1D_KERNEL_LOOP(thread_id, num_in_backprop) {
    const int IW = thread_id % in_cols;
    const int IH = (thread_id / in_cols) % in_rows;
    const int IL = (thread_id / in_cols / in_rows) % in_length;
    const int IC = (thread_id / in_cols / in_rows / in_length) % in_depth;
    const int IB = thread_id / in_cols / in_rows / in_length / in_depth;

    T sum = 0;

    const int out_r_start =
        max(0, (IH - filter_rows + pad_rows + stride) / stride);
    const int out_r_end = min(out_rows - 1, (IH + pad_rows) / stride);
    const int out_c_start =
        max(0, (IW - filter_cols + pad_cols + stride) / stride);
    const int out_c_end = min(out_cols - 1, (IW + pad_cols) / stride);
    const int out_l_start = max(
        0,
        (IL - filter_length + pad_length + temporal_stride) / temporal_stride);
    const int out_l_end =
        min(out_length - 1, (IL + pad_length) / temporal_stride);

#pragma unroll
    for (int out_l = out_l_start; out_l <= out_l_end; ++out_l) {
      const int f_l = IL + pad_length - out_l * temporal_stride;
      for (int out_r = out_r_start; out_r <= out_r_end; ++out_r) {
        const int f_r = IH + pad_rows - out_r * stride;
        for (int out_c = out_c_start; out_c <= out_c_end; ++out_c) {
          const int f_c = IW + pad_cols - out_c * stride;
          const int filter_offset =
              IC * filter_rows * filter_cols * filter_length +
              f_l * filter_cols * filter_rows + f_r * filter_cols + f_c;
          const int out_backprop_offset =
              (IB * out_depth * out_length * out_rows * out_cols) +
              (IC * out_length * out_rows * out_cols) +
              (out_l * out_rows * out_cols) + (out_r * out_cols) + (out_c);

#if __CUDA_ARCH__ >= 350
          sum += __ldg(out_backprop + out_backprop_offset) *
              __ldg(filter + filter_offset);
#else
          sum += out_backprop[out_backprop_offset] * filter[filter_offset];
#endif
        }
      }
    }
    const int in_backprop_offset =
        (IB * in_rows * in_cols * in_length * in_depth) +
        (IC * in_rows * in_cols * in_length) + (IL * in_rows * in_cols) +
        (IH * in_cols) + (IW);
    in_backprop[in_backprop_offset] = sum;
  }
}

class ChannelwiseConv3dOp final : public ConvPoolOpBase<HIPContext> {
 public:
  USE_CONV_POOL_BASE_FUNCTIONS(HIPContext);
  ChannelwiseConv3dOp(const OperatorDef& operator_def, Workspace* ws)
      : ConvPoolOpBase<HIPContext>(operator_def, ws),
        cudnn_wrapper_(&context_) {
    OPERATOR_NEEDS_FEATURE(
        this->order_ == StorageOrder::NCHW,
        "ChannelwiseConv3dOp only supports NCHW order");
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&bias_desc_));
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&top_desc_for_bias_));
  }

  ~ChannelwiseConv3dOp() {
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(bias_desc_));
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(top_desc_for_bias_));
  }

  bool RunOnDeviceWithOrderNCHW() override {
    const Tensor& X = Input(0);
    auto& filter = Input(1);
    const int C = X.dim32(1);
    CAFFE_ENFORCE_EQ(X.ndim(), filter.ndim());
    const int M = filter.dim32(0); // number of output filters

    // enforce input/output filters are the same
    CAFFE_ENFORCE_EQ(M, X.dim32(1));
    CAFFE_ENFORCE_EQ(C, X.dim32(1));

    // check group parameters
    CAFFE_ENFORCE_EQ(C, this->group_);
    CAFFE_ENFORCE_GT(this->group_, 1);

    auto sizes = ConvPoolOpBase<HIPContext>::GetOutputSize(X, filter.dim32(0));
    Tensor* Y = Output(0, sizes, at::dtype<float>());

    DepthwiseArgs args;
    args.batch = X.dim32(0);
    args.in_length = X.dim32(2);
    args.in_rows = X.dim32(3);
    args.in_cols = X.dim32(4);
    args.in_depth = X.dim32(1);

    CAFFE_ENFORCE_EQ(kernel_.size(), 3);
    args.filter_cols = kernel_[2];
    args.filter_rows = kernel_[1];
    args.filter_length = kernel_[0];

    CAFFE_ENFORCE_EQ(stride_.size(), 3);
    args.stride = stride_[1];
    CAFFE_ENFORCE_EQ(stride_[1], stride_[2]);
    args.temporal_stride = stride_[0];

    CAFFE_ENFORCE_EQ(pads_.size(), 6);
    args.pad_length = pads_[0];
    args.pad_rows = pads_[1];
    args.pad_cols = pads_[2];

    CAFFE_ENFORCE_EQ(Y->dim32(0), X.dim32(0));
    args.out_rows = Y->dim32(3);
    args.out_cols = Y->dim32(4);
    args.out_length = Y->dim32(2);
    args.out_depth = Y->dim32(1);

    DepthwiseConv3dGPUKernelNCHW<float>
        <<<CAFFE_GET_BLOCKS(Y->size()),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            args,
            X.data<float>(),
            filter.data<float>(),
            Y->mutable_data<float>(),
            Y->size());

    if (InputSize() == 3) {
      std::vector<int> bias_dims(X.ndim(), 1);
      bias_dims[1] = M;
      std::vector<int> strides = {M, 1, 1, 1, 1};
      CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(
          bias_desc_,
          cudnnTypeWrapper<float>::type,
          X.ndim(),
          bias_dims.data(),
          strides.data()));

      vector<int> dims = {
          Y->dim32(0), M, Y->dim32(2), Y->dim32(3), Y->dim32(4)};
      strides = {M * Y->dim32(2) * Y->dim32(3) * Y->dim32(4),
                 Y->dim32(2) * Y->dim32(3) * Y->dim32(4),
                 Y->dim32(3) * Y->dim32(4),
                 Y->dim32(4),
                 1};
      CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(
          top_desc_for_bias_,
          cudnnTypeWrapper<float>::type,
          X.ndim(),
          dims.data(),
          strides.data()));

      auto& bias = Input(2);
      CAFFE_ENFORCE_EQ(bias.ndim(), 1);
      CAFFE_ENFORCE_EQ(bias.dim32(0), M);
      CUDNN_ENFORCE(hipdnnAddTensor(
          cudnn_wrapper_.inline_cudnn_handle(),
          cudnnTypeWrapper<float>::kOne(),
          bias_desc_,
          bias.data<float>(),
          cudnnTypeWrapper<float>::kOne(),
          top_desc_for_bias_,
          Y->mutable_data<float>()));
    }

    return true;
  }

 private:
  CuDNNWrapper cudnn_wrapper_;
  hipdnnTensorDescriptor_t bias_desc_;
  hipdnnTensorDescriptor_t top_desc_for_bias_;
};

class ChannelwiseConv3dGradientOp final : public ConvPoolOpBase<HIPContext> {
 public:
  USE_CONV_POOL_BASE_FUNCTIONS(HIPContext);
  ChannelwiseConv3dGradientOp(const OperatorDef& operator_def, Workspace* ws)
      : ConvPoolOpBase<HIPContext>(operator_def, ws),
        cudnn_wrapper_(&context_),
        no_bias_(OperatorBase::GetSingleArgument<int>("no_bias", 0)) {
    CAFFE_ENFORCE(
        !(no_bias_ && OutputSize() == 3),
        "If bias is not present, you should not have 3 grad output.");
    OPERATOR_NEEDS_FEATURE(
        this->order_ == StorageOrder::NCHW,
        "ChannelwiseConv3dGradientOp only supports NCHW order");
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&bias_desc_));
    CUDNN_ENFORCE(hipdnnCreateTensorDescriptor(&top_desc_for_bias_));
  }

  ~ChannelwiseConv3dGradientOp() {
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(bias_desc_));
    CUDNN_ENFORCE(hipdnnDestroyTensorDescriptor(top_desc_for_bias_));
  }

  bool RunOnDeviceWithOrderNCHW() override {
    auto& X = Input(INPUT);
    auto& filter = Input(FILTER);
    auto& dY = Input(OUTPUT_GRAD);
    auto* dfilter = Output(FILTER_GRAD);
    const int C = X.dim32(1);

    const vector<int> input_dims = this->GetDims(X);
    ConvPoolOpBase<HIPContext>::ComputePads(input_dims);
    CAFFE_ENFORCE_EQ(X.ndim(), filter.ndim());
    const int M = filter.dim32(0);
    CAFFE_ENFORCE(filter.dim32(1) * group_ == C);
    CAFFE_ENFORCE(M % group_ == 0);
    dfilter->ResizeLike(filter);

    DepthwiseArgs args;
    args.batch = X.dim32(0);
    args.in_rows = X.dim32(3);
    args.in_cols = X.dim32(4);
    args.in_length = X.dim32(2);
    args.in_depth = X.dim32(1);

    args.filter_cols = kernel_[2];
    args.filter_rows = kernel_[1];
    args.filter_length = kernel_[0];

    args.stride = stride_[1];
    CAFFE_ENFORCE_EQ(stride_[1], stride_[2]);
    args.temporal_stride = stride_[0];

    args.pad_length = pads_[0];
    args.pad_rows = pads_[1];
    args.pad_cols = pads_[2];

    args.out_rows = dY.dim32(3);
    args.out_cols = dY.dim32(4);
    args.out_length = dY.dim32(2);
    args.out_depth = dY.dim32(1);

    CAFFE_ENFORCE(OutputSize() == 3 || (no_bias_ && (OutputSize() == 2)));
    auto* dX = Output(no_bias_ ? BIAS_OR_INPUT_GRAD : INPUT_GRAD);
    dX->ResizeLike(X);
    math::Set<float, HIPContext>(
        dfilter->size(), 0, dfilter->mutable_data<float>(), &context_);

    DepthwiseConv3dBackpropFilterGPUKernelNCHW<float>
        <<<CAFFE_GET_BLOCKS(dY.size()),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            args,
            dY.data<float>(),
            X.data<float>(),
            dfilter->mutable_data<float>(),
            dY.size());
    DepthwiseConv3dBackpropInputGPUKernelNCHW<float>
        <<<CAFFE_GET_BLOCKS(dX->size()),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            args,
            dY.data<float>(),
            filter.data<float>(),
            dX->mutable_data<float>(),
            dX->size());

    if (!no_bias_) {
      std::vector<int> bias_dims(X.ndim(), 1);
      bias_dims[1] = M;
      std::vector<int> strides = {M, 1, 1, 1, 1};
      CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(
          bias_desc_,
          cudnnTypeWrapper<float>::type,
          X.ndim(),
          bias_dims.data(),
          strides.data()));

      std::vector<int> dims = {
          dY.dim32(0), M, dY.dim32(2), dY.dim32(3), dY.dim32(4)};
      strides = {M * dY.dim32(2) * dY.dim32(3) * dY.dim32(4),
                 dY.dim32(2) * dY.dim32(3) * dY.dim32(4),
                 dY.dim32(3) * dY.dim32(4),
                 dY.dim32(4),
                 1};
      CUDNN_ENFORCE(hipdnnSetTensorNdDescriptor(
          top_desc_for_bias_,
          cudnnTypeWrapper<float>::type,
          X.ndim(),
          dims.data(),
          strides.data()));

      auto* dbias = Output(BIAS_OR_INPUT_GRAD);
      dbias->Resize(M);
      CUDNN_ENFORCE(hipdnnConvolutionBackwardBias(
          cudnn_wrapper_.inline_cudnn_handle(),
          cudnnTypeWrapper<float>::kOne(),
          top_desc_for_bias_,
          dY.data<float>(),
          cudnnTypeWrapper<float>::kZero(),
          bias_desc_,
          dbias->mutable_data<float>()));
    }
    return true;
  }

 private:
  CuDNNWrapper cudnn_wrapper_;
  hipdnnTensorDescriptor_t bias_desc_;
  hipdnnTensorDescriptor_t top_desc_for_bias_;

  bool no_bias_;

  INPUT_TAGS(INPUT, FILTER, OUTPUT_GRAD);
  OUTPUT_TAGS(FILTER_GRAD, BIAS_OR_INPUT_GRAD, INPUT_GRAD);
};

REGISTER_CUDA_OPERATOR_WITH_ENGINE(Conv, CHANNELWISE_3D, ChannelwiseConv3dOp);
REGISTER_CUDA_OPERATOR_WITH_ENGINE(
    ConvGradient,
    CHANNELWISE_3D,
    ChannelwiseConv3dGradientOp);

} // namespace caffe2
