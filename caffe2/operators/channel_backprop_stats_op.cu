#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/channel_backprop_stats_op.h"

namespace caffe2 {

namespace {

// based on "Optimizing Parallel Reduction in CUDA" by Mark Harris

// note - volatile keyword is needed to allow doing a warp reduction without
// synchronization on recent architectures
template <unsigned int blockSize>
__device__ void warpReduce(volatile float* sdata, unsigned int tid) {
  // note - the if statements are "free" as they are resolved at compile time
  if (blockSize >= 64)
    sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32)
    sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16)
    sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8)
    sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4)
    sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2)
    sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void ChannelBackpropStatsBlockKernel(
    int N,
    int C,
    int valsPerChannel,
    const float* X,
    const float* dY,
    const float* mean,
    const float* invStddev,
    float* dBiasBlocks,
    float* dScaleBlocks) {
  __shared__ float dBiasData[blockSize];
  __shared__ float dScaleData[blockSize];

  auto tid = threadIdx.x;
  auto numBlocksPerChannel = (valsPerChannel + blockSize - 1) / blockSize;
  auto localBlockIndex = blockIdx.x % numBlocksPerChannel;
  auto inputIndex = (blockIdx.x / numBlocksPerChannel) * valsPerChannel +
      localBlockIndex * blockSize + tid;
  auto n = blockIdx.x / numBlocksPerChannel / C;
  auto c = (blockIdx.x / numBlocksPerChannel) % C;

  dBiasData[tid] = 0;
  dScaleData[tid] = 0;

  if (localBlockIndex * blockSize + tid < valsPerChannel) {
    dBiasData[tid] += dY[inputIndex];
    dScaleData[tid] +=
        (X[inputIndex] - mean[c]) * invStddev[c] * dY[inputIndex];
  }

  __syncthreads();
  if (blockSize >= 512) {
    if (tid < 256) {
      dBiasData[tid] += dBiasData[tid + 256];
      dScaleData[tid] += dScaleData[tid + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (tid < 128) {
      dBiasData[tid] += dBiasData[tid + 128];
      dScaleData[tid] += dScaleData[tid + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (tid < 64) {
      dBiasData[tid] += dBiasData[tid + 64];
      dScaleData[tid] += dScaleData[tid + 64];
    }
    __syncthreads();
  }

  if (tid < 32) {
    warpReduce<blockSize>(dBiasData, tid);
    warpReduce<blockSize>(dScaleData, tid);
  }

  // output block data sorted by C to simplify second reduction
  if (tid == 0) {
    auto outputIndex = (c * N + n) * numBlocksPerChannel + localBlockIndex;
    dBiasBlocks[outputIndex] = dBiasData[0];
    dScaleBlocks[outputIndex] = dScaleData[0];
  }
}

template <unsigned int blockSize>
__global__ void ChannelBackpropStatsFinalSumsKernel(
    int N,
    int C,
    int numSumsPerChannel,
    const float* dBiasScratch,
    const float* dScaleScratch,
    float* dBias,
    float* dScale) {
  __shared__ float dBiasData[blockSize];
  __shared__ float dScaleData[blockSize];

  auto tid = threadIdx.x;
  auto inputIndex = blockIdx.x * N * numSumsPerChannel + tid;
  dBiasData[tid] = 0;
  dScaleData[tid] = 0;
  for (auto i = inputIndex; i < (blockIdx.x + 1) * N * numSumsPerChannel;
       i += blockSize) {
    dBiasData[tid] += dBiasScratch[i];
    dScaleData[tid] += dScaleScratch[i];
  }
  __syncthreads();
  if (blockSize >= 512) {
    if (tid < 256) {
      dBiasData[tid] += dBiasData[tid + 256];
      dScaleData[tid] += dScaleData[tid + 256];
    }
    __syncthreads();
  }
  if (blockSize >= 256) {
    if (tid < 128) {
      dBiasData[tid] += dBiasData[tid + 128];
      dScaleData[tid] += dScaleData[tid + 128];
    }
    __syncthreads();
  }
  if (blockSize >= 128) {
    if (tid < 64) {
      dBiasData[tid] += dBiasData[tid + 64];
      dScaleData[tid] += dScaleData[tid + 64];
    }
    __syncthreads();
  }
  if (tid < 32) {
    warpReduce<blockSize>(dBiasData, tid);
    warpReduce<blockSize>(dScaleData, tid);
  }

  if (tid == 0) {
    dBias[blockIdx.x] = dBiasData[0];
    dScale[blockIdx.x] = dScaleData[0];
  }
}
} // namespace

template <>
bool ChannelBackpropStatsOp<HIPContext>::RunOnDevice() {
  const auto& X = Input(INPUT);
  const auto& dY = Input(OUTPUT_GRAD);
  const auto& mean = Input(SAVED_MEAN);
  const auto& invStddev = Input(SAVED_INV_STDDEV);
  CAFFE_ENFORCE(X.dim() >= 3 && X.dim() <= 5);
  const int N = X.dim32(0);
  const int C = X.dim32(1);
  const int H = X.dim32(2);
  const int W = X.dim() > 3 ? X.dim32(3) : 1;
  const int D = X.dim() > 4 ? X.dim32(4) : 1;

  const auto Xarr = X.data<float>();
  const auto dYarr = dY.data<float>();
  const auto meanArr = mean.data<float>();
  const auto invStddevArr = invStddev.data<float>();

  auto dBias = Output(BIAS_GRAD, {C}, at::dtype<float>());
  auto dScale = Output(SCALE_GRAD, {C}, at::dtype<float>());

  const auto valsPerChannel = H * W * D;

  const auto numBlocksPerChannel = CAFFE_GET_BLOCKS(valsPerChannel);
  const auto numBlocksTotal = numBlocksPerChannel * N * C;

  ReinitializeTensor(
      &dBiasScratch_, {numBlocksTotal}, at::dtype<float>().device(CUDA));
  ReinitializeTensor(
      &dScaleScratch_, {numBlocksTotal}, at::dtype<float>().device(CUDA));

  ChannelBackpropStatsBlockKernel<CAFFE_CUDA_NUM_THREADS>
      <<<numBlocksTotal, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          N,
          C,
          valsPerChannel,
          Xarr,
          dYarr,
          meanArr,
          invStddevArr,
          dBiasScratch_.mutable_data<float>(),
          dScaleScratch_.mutable_data<float>());

  ChannelBackpropStatsFinalSumsKernel<CAFFE_CUDA_NUM_THREADS>
      <<<C, CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
          N,
          C,
          numBlocksPerChannel,
          dBiasScratch_.data<float>(),
          dScaleScratch_.data<float>(),
          dBias->template mutable_data<float>(),
          dScale->template mutable_data<float>());

  return true;
}

REGISTER_HIP_OPERATOR(
    ChannelBackpropStats,
    ChannelBackpropStatsOp<HIPContext>);

} // namespace caffe2
