#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/clip_op.h"

namespace caffe2 {
namespace {

template <typename T>
__device__ T cuda_min(T x, T y);
template <typename T>
__device__ T cuda_max(T x, T y);
template <>
__device__ float cuda_min(float x, float y) { return fminf(x, y); }
template <>
__device__ float cuda_max(float x, float y) { return fmaxf(x, y); }

// Disabled since we don't use it right now.
/*
template <>
__device__ double cuda_min(double x, double y) { return fmin(x, y); }
template <>
__device__ double cuda_max(double x, double y) { return fmax(x, y); }
*/


template <typename T>
__global__ void ClipKernel(const int N, const T minval, const T maxval,
                           const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = cuda_min<T>(cuda_max<T>(X[i], minval), maxval);
  }
}

template <typename T>
__global__ void ClipGradientKernel(const int N,  const T minval,
                                   const T maxval, const T* Y,
                                   const T* dY, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    dX[i] = dY[i] * (Y[i] > minval && Y[i] < maxval);
  }
}
}  // namespace

template <>
bool ClipOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  
  CAFFE_ENFORCE_GE(X.numel(), 0);
  auto* Y = Output(0, X.sizes(), at::dtype<float>());
  ClipKernel<<<
      CAFFE_GET_BLOCKS(X.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X.numel(), min_, max_, X.data<float>(), Y->template mutable_data<float>());
  return true;
}

template <>
bool ClipGradientOp<float, HIPContext>::RunOnDevice() {
  auto& Y = Input(0);
  auto& dY = Input(1);
  
  CAFFE_ENFORCE_GE(Y.numel(), 0);
  CAFFE_ENFORCE_EQ(dY.numel(), Y.numel());
  auto* dX = Output(0, Y.sizes(), at::dtype<float>());
  ClipGradientKernel<<<
      CAFFE_GET_BLOCKS(Y.numel()),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      Y.numel(),
      min_,
      max_,
      Y.data<float>(),
      dY.data<float>(),
      dX->template mutable_data<float>());
  return true;
}

REGISTER_HIP_OPERATOR(Clip, ClipOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(ClipGradient, ClipGradientOp<float, HIPContext>);
}  // namespace caffe2
