#include "hip/hip_runtime.h"
// YellowFin: An automatic tuner for momentum SGD
// (https://arxiv.org/abs/1706.03471)

#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/sgd/yellowfin_op.h"

namespace caffe2 {

__global__ void GetLrMuKernel(
    const float* g_norm2_max_deb,
    const float* g_norm2_min_deb,
    const float* distance_deb,
    const float* variance,
    float* mu,
    float* lr) {
  const float curv_ratio = sqrtf(*g_norm2_max_deb / *g_norm2_min_deb);
  const float mu_limit = (curv_ratio - 1.0f) / (curv_ratio + 1.0f);
  const float pre_p = *distance_deb * *g_norm2_min_deb;
  const float p = (pre_p * pre_p) / (2.0f * *variance);
  const float w3 = (-sqrtf(p * p + 4.0f / 27.0f * p * p * p) - p) / 2.0f;
  const float w3_sign = w3 > 0.0f ? 1.0f : -1.0f;
  const float w = w3_sign * powf(fabsf(w3), 1.0f / 3.0f);
  const float y = w - p / 3.0f / w;
  const float root = y + 1.0f;
  *mu = fmaxf(root * root, mu_limit * mu_limit);
  *lr = powf(1.0f - sqrtf(*mu), 2) / *g_norm2_min_deb;
}

template <>
void YellowFinOp<float, HIPContext>::GetLrMu() {
  // Finding root of cubic formula for YF's Single Step
  GetLrMuKernel<<<1, 1, 0, context_.hip_stream()>>>(
      g_norm2_max_deb_, g_norm2_min_deb_, distance_deb_, variance_, mu_, lr_);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  MovingAverage(1, mu_, mu_avg_, mu_avg_out_, mu_deb_);
  MovingAverage(1, lr_, lr_avg_, lr_avg_out_, lr_deb_);
}

__global__ void MomentumSgdKernel(
    const int N,
    const float* mu_ptr,
    const float* lr_ptr,
    const float* param,
    const float* grad,
    const float* moment,
    float* param_out,
    float* moment_out,
    bool nesterov) {
  const float mu = *mu_ptr;
  const float lr = *lr_ptr;
  if (!nesterov) {
    CUDA_1D_KERNEL_LOOP(i, N) {
      moment_out[i] = mu * moment[i] + lr * grad[i];
      param_out[i] = param[i] - moment_out[i];
    }
  } else {
    CUDA_1D_KERNEL_LOOP(i, N) {
      const float moment_i = moment[i];
      moment_out[i] = mu * moment_i + lr * grad[i];
      param_out[i] = param[i] - (1 + mu) * moment_out[i] + mu * moment_i;
    }
  }
}

template <>
void YellowFinOp<float, HIPContext>::MomentumSgdUpdate() {
  MomentumSgdKernel<<<
      CAFFE_GET_BLOCKS(D_),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      D_,
      mu_avg_out_,
      lr_avg_out_,
      param_,
      grad_,
      moment_,
      param_out_,
      moment_out_,
      nesterov_);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

REGISTER_HIP_OPERATOR(YellowFin, YellowFinOp<float, HIPContext>);

} // namespace caffe2
