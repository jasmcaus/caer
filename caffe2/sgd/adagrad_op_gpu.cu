#include "hip/hip_runtime.h"
#include <algorithm>

#include <cub/block/block_reduce.cuh>
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/sgd/adagrad_op.h"

namespace caffe2 {

__global__ void AdagradUpdate(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    float decay,
    const float* lr,
    float weight_decay = 0.f) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i] + weight_decay * w[i];
    float hi = nh[i] = decay * h[i] + gi * gi;
    nw[i] = w[i] + lr[0] * gi / (sqrtf(hi) + epsilon);
  }
}

template <>
void adagrad_update<HIPContext>(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    float decay,
    const float* lr,
    HIPContext* context,
    float weight_decay) {
  AdagradUpdate<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      N, w, g, h, nw, nh, epsilon, decay, lr, weight_decay);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename SIndex, typename THalf>
__global__ void SparseAdagradKernel(
    const size_t N,
    const size_t grad_slice_sz,
    const float epsilon,
    THalf* param,
    THalf* param_mom,
    const SIndex* indices,
    const float* grad,
    const float* lr,
    float weight_decay = 0.f) {
  const float LR = lr[0];
  CUDA_1D_KERNEL_LOOP(i, N) {
    const size_t gradIdx = i;
    const SIndex index = indices[i / grad_slice_sz];
    const size_t paramIdx = index * grad_slice_sz + (i % grad_slice_sz);

    float gi = grad[gradIdx] + weight_decay * param[paramIdx];
    float mom_new = gi * gi + param_mom[paramIdx];
    param_mom[paramIdx] = mom_new;
    float param_new = LR * gi / (sqrtf(mom_new) + epsilon) + param[paramIdx];
    param[paramIdx] = param_new;
  }
}

/**
 * Calculate RowwiseSparseAdagrad
 * M: gradients.dims[0]
 * N: gradients.size_from_dim(1)
 * grad: pointer to the gradients
 * param: pointer to weights
 * param_mom: pointer to the momentum
 * indices: keys
 */
template <typename SIndex>
__global__ void RowWiseSparseAdagradKernel(
    const int M,
    const int N,
    const float epsilon,
    float* param,
    float* param_mom,
    const SIndex* indices,
    const float* grad,
    const float* lr,
    float weight_decay = 0.f) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ BlockReduce::TempStorage temp_storage;
  int valid = min(N, CAFFE_CUDA_NUM_THREADS);
  // in case gridDim is smaller than M
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
    const SIndex index = indices[i];
    float sum_squares = 0.0;
    __shared__ float row_sum_squares_avg;

    // in case N is bigger than block size which is 512 by default
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const float x_ij = grad[i * N + j] + weight_decay * param[index * N + j];
      sum_squares += x_ij * x_ij;
    }
    float reduce_result = BlockReduce(temp_storage).Sum(sum_squares, valid);
    if (threadIdx.x == 0) {
      row_sum_squares_avg = reduce_result / (float)N;
      param_mom[index] += row_sum_squares_avg;
    }
    __syncthreads();
    // update param
    float step = lr[0] / (sqrtf(param_mom[index]) + epsilon);
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const float x_ij = grad[i * N + j] + weight_decay * param[index * N + j];
      param[index * N + j] = param[index * N + j] + x_ij * step;
    }
  }
}

template <typename T, class Context>
class CUDASparseAdagradOp final : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDASparseAdagradOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)),
        weight_decay_(
            this->template GetSingleArgument<float>("weight_decay", 0.f)) {
    VLOG(1) << "gradient optimization operator in use: "
            << "CUDASparseAdagradOp"
            << " weight_decay_=" << weight_decay_;
    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(PARAM).size(), Input(MOMENT_1).size());
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename THalf>
  bool DoRunWithType2() {
    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const auto* gradIn = Input(GRAD).template data<T>();
    const auto* paramIn = Input(PARAM).template data<THalf>();
    const auto* momentIn = Input(MOMENT_1).template data<THalf>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<THalf>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>();

    auto N = Input(GRAD).size();
    auto grad_slice_sz = Input(GRAD).size_from_dim(Input(INDICES).ndim());
    if (N == 0) {
      // empty grad, nothing to do here, not even launching the kernel
      return true;
    }
    SparseAdagradKernel<IndexType, THalf>
        <<<CAFFE_GET_BLOCKS(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            N,
            grad_slice_sz,
            epsilon_,
            Output(OUTPUT_PARAM)->template mutable_data<THalf>(),
            Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>(),
            Input(INDICES).template data<IndexType>(),
            Input(GRAD).template data<float>(),
            Input(LR).template data<float>(),
            weight_decay_);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    return true;
  }

 protected:
  T epsilon_;
  T weight_decay_;
  INPUT_TAGS(PARAM, MOMENT_1, INDICES, GRAD, LR);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1);
};

template <>
template <typename SIndex>
bool RowWiseSparseAdagradOp<HIPContext>::DoRunWithType() {
  auto N = Input(GRAD).size();
  if (N == 0) {
    // empty grad, nothing to do here, not even launching the kernel
    return true;
  }
  // size of the 1st dimension of the input gradient
  auto GRAD_M = Input(GRAD).dim32(0);
  auto GRAD_N = N / GRAD_M;

  // Cases with GRAND_N < 128 can have more swarms if number of threads is lower
  int num_threads = CAFFE_CUDA_NUM_THREADS;
  if (GRAD_N < num_threads) {
    num_threads = GRAD_N;
  }

  // each thread block will handle multiple rows of the input and output
  RowWiseSparseAdagradKernel<<<
      std::min(GRAD_M, CAFFE_MAXIMUM_NUM_BLOCKS),
      num_threads,
      0,
      context_.hip_stream()>>>(
      GRAD_M,
      GRAD_N,
      epsilon_,
      Output(OUTPUT_PARAM)->template mutable_data<float>(),
      Output(OUTPUT_MOMENT_1)->template mutable_data<float>(),
      Input(INDICES).template data<SIndex>(),
      Input(GRAD).template data<float>(),
      Input(LR).template data<float>(),
      weight_decay_);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return true;
}

REGISTER_HIP_OPERATOR(Adagrad, AdagradOp<HIPContext>);
REGISTER_HIP_OPERATOR(SparseAdagrad, CUDASparseAdagradOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagrad,
    RowWiseSparseAdagradOp<HIPContext>);
} // namespace caffe2
