#include "hip/hip_runtime.h"
#include "caffe2/sgd/momentum_sgd_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

inline int CaffeGetBlocksSGD(const int N) {
  return std::max(
      (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS,
      // Use at least 1 block, since CUDA does not allow empty block
      1);
}
template <bool nesterov>
__global__ void MomentumSGDKernel(
    const int N,
    const float* g,
    const float* m,
    float* ng,
    float* nm,
    const float* lr,
    const float momentum,
    float* param);

template <>
__global__ void MomentumSGDKernel<true>(
    const int N,
    const float* g,
    const float* m,
    float* ng,
    float* nm,
    const float* lr,
    const float momentum,
    float* param) {
  const float LR = lr[0];
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float mi = m[i];
    const float mi_new = momentum * mi + LR * g[i];
    nm[i] = mi_new;
    ng[i] = fmaf(momentum, mi_new - mi, mi_new);
    if (param != nullptr) {
      param[i] -= ng[i];
    }
  }
}

template <>
__global__ void MomentumSGDKernel<false>(
    const int N,
    const float* g,
    const float* m,
    float* ng,
    float* nm,
    const float* lr,
    const float momentum,
    float* param) {
  const float LR = lr[0];
  CUDA_1D_KERNEL_LOOP(i, N) {
    const float adjusted_gradient = LR * g[i] + momentum * m[i];
    nm[i] = adjusted_gradient;
    ng[i] = adjusted_gradient;
    if (param != nullptr) {
      param[i] -= adjusted_gradient;
    }
  }
}

template <>
void momentum_sgd_update<HIPContext>(
    const int N,
    const float* g,
    const float* m,
    float* ng,
    float* nm,
    const float* lr,
    const float momentum,
    const bool nesterov,
    float* param,
    HIPContext* context) {
  if (nesterov) {
    MomentumSGDKernel<true>
        <<<CaffeGetBlocksSGD(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(N, g, m, ng, nm, lr, momentum, param);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    MomentumSGDKernel<false>
        <<<CaffeGetBlocksSGD(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(N, g, m, ng, nm, lr, momentum, param);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}


template <typename SIndex>
__global__ void SparseMomentumSGDKernel(
    const size_t N,
    const size_t sz,
    const float momentum,
    const bool nesterov,
    float *param,
    float *param_mom,
    const SIndex *indices,
    const float *gradIn,
    float *gradOut,
    const float *lr)
{
  const float LR = lr[0];
  CUDA_1D_KERNEL_LOOP(i, N)
  {
    const size_t gradIdx = i;
    const SIndex index = indices[i / sz];
    const size_t paramIdx = index * sz + (i % sz);

    if (!nesterov)
    {
      const float adjusted_gradient = LR * gradIn[gradIdx] +
          momentum * param_mom[paramIdx];
      gradOut[gradIdx] = adjusted_gradient;
      param_mom[paramIdx] = adjusted_gradient;
      param[paramIdx] -= adjusted_gradient;
    } else {
      const float mom_old = param_mom[paramIdx];
      const float mom_new = LR * gradIn[gradIdx] + momentum * mom_old;
      param_mom[paramIdx] = mom_new;
      const float adjusted_gradient = (1 + momentum) * mom_new -
          momentum * mom_old;
      gradOut[gradIdx] = adjusted_gradient;
      param[paramIdx] -= adjusted_gradient;
    }
  }
}


// Specialization of DoRunWithType for CUDA
template <>
template <typename SIndex>
bool SparseMomentumSGDUpdateOp<float, HIPContext>::DoRunWithType() {
  auto N = Input(GRAD).size();
  auto grad_slice_sz = Input(GRAD).size_from_dim(Input(INDICES).ndim());

  SparseMomentumSGDKernel<SIndex><<<
    CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0,
    context_.hip_stream()>>>(
        N, grad_slice_sz,
        momentum_, nesterov_,
        Output(OUTPUT_PARAM)->template mutable_data<float>(),
        Output(OUTPUT_MOMENTUM)->template mutable_data<float>(),
        Input(INDICES).template data<SIndex>(),
        Input(GRAD).template data<float>(),
        Output(OUTPUT_GRAD)->template mutable_data<float>(),
        Input(LR).template data<float>());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return true;
}

REGISTER_HIP_OPERATOR(MomentumSGD, MomentumSGDOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(MomentumSGDUpdate, MomentumSGDUpdateOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SparseMomentumSGDUpdate, SparseMomentumSGDUpdateOp<float, HIPContext>);

}
