#include "hip/hip_runtime.h"
#include "caffe2/sgd/rmsprop_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void RmsPropUpdate(
    int N,
    const float* g,
    const float* ms,
    const float* mom,
    float* ng,
    float* nms,
    float* nmom,
    float decay,
    float momentum,
    float epsilon,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    // Update new mean square estimate
    nms[i] = ms[i] + (1.0f - decay) * (g[i] * g[i] - ms[i]);
    // Update momentum estimate
    nmom[i] =
        mom[i] * momentum + lr[0] * g[i] / sqrtf(epsilon + nms[i]);
    // New gradient is the momentum
    ng[i] = nmom[i];
  }
}

template <>
void rmsprop_update<HIPContext>(
    int N,
    const float* g,
    const float* ms,
    const float* mom,
    float* ng,
    float* nms,
    float* nmom,
    float decay,
    float momentum,
    float epsilon,
    const float* lr,
    HIPContext* context) {
  RmsPropUpdate<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
      N, g, ms, mom, ng, nms, nmom, decay, momentum, epsilon, lr);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}


REGISTER_HIP_OPERATOR(RmsProp, RmsPropOp<float, HIPContext>);

}
