#include "hip/hip_runtime.h"
// Implements the math functions for GPU.

#include "caffe2/utils/math.h"

#include <cstring>
#include <limits>
#include <numeric>
#include <vector>

#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

#include <thrust/device_vector.h>
#include <thrust/functional.h>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/conversions.h"

#include "caffe2/utils/fixed_divisor.h"
// TODO: Move this to fixed_divisor.h
#ifdef __HIP_PLATFORM_HCC__
#define FIXED_DIVISOR int32_t
#define FIXED_DIVISOR_DIV(d, n) (n / d)
#define FIXED_DIVISOR_MOD(d, n) (n % d)
#define FIXED_DIVISOR_DIV_MOD(d, n, q, r) \
  do {                                    \
    const auto n_copy = n;                \
    *q = n_copy / d;                      \
    *r = n_copy % d;                      \
  } while (0)
#else // __HIP_PLATFORM_HCC__
#define FIXED_DIVISOR FixedDivisor<int32_t>
#define FIXED_DIVISOR_DIV(d, n) (d.Div(n))
#define FIXED_DIVISOR_MOD(d, n) (d.Mod(n))
#define FIXED_DIVISOR_DIV_MOD(d, n, q, r) (d.DivMod(n, q, r))
#endif // __HIP_PLATFORM_HCC__

#ifdef __HIP_PLATFORM_HCC__
#include <hip/hip_version.h>
using CUBLAS_HALF_TYPE = rocblas_half;
#else // __HIP_PLATFORM_HCC
using CUBLAS_HALF_TYPE = __half;
#endif // __HIP_PLATFORM_HCC

#include "caffe2/utils/math/utils.h"

#if THRUST_VERSION >= 100800
#define THRUST_SUPPORTS_PER_THREAD
#endif // THRUST_VERSION >= 100800

namespace caffe2 {
namespace math {

namespace {

#define DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Func, expr)          \
  template <typename T>                                                 \
  struct Func##Functor {                                                \
    inline __host__ __device__ T                                        \
    operator()(const T& lhs, const T& rhs) const {                      \
      return lhs expr rhs;                                              \
    }                                                                   \
  };                                                                    \
  template <>                                                           \
  struct Func##Functor<at::Half> {                                      \
    inline __host__ __device__ at::Half operator()(                     \
        const at::Half& lhs,                                            \
        const at::Half& rhs) const {                                    \
      return convert::To<float, at::Half>(convert::To<at::Half, float>( \
          lhs) expr convert::To<at::Half, float>(rhs));                 \
    }                                                                   \
  };
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Add, +)
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Sub, -)
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Mul, *)
DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR(Div, /)
#undef DELEGATE_SIMPLE_HOST_DEVICE_BINARY_FUNCTOR

template <typename TIn, typename TOut, class BinaryOperator>
__global__ void SimpleBinaryOpCUDAKernel(
    const int N,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    C[i] = op(A[i], B[i]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator, bool broadcast_1st>
__global__ void RowwiseBinaryOpCUDAKenel(
    const int size,
    const FIXED_DIVISOR cols,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(C_index, size) {
    const int j = FIXED_DIVISOR_MOD(cols, C_index);
    const int A_index = broadcast_1st ? j : C_index;
    const int B_index = broadcast_1st ? C_index : j;
    C[C_index] = op(A[A_index], B[B_index]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator, bool broadcast_1st>
__global__ void ColwiseBinaryOpCUDAKenel(
    const int size,
    const FIXED_DIVISOR cols,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(C_index, size) {
    const int i = FIXED_DIVISOR_DIV(cols, C_index);
    const int A_index = broadcast_1st ? i : C_index;
    const int B_index = broadcast_1st ? C_index : i;
    C[C_index] = op(A[A_index], B[B_index]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator, int D>
__global__ void BroadcastBinaryOpCUDAKernel(
    const int size,
    const SimpleArray<int, D> A_strides,
    const SimpleArray<int, D> B_strides,
    const SimpleArray<FIXED_DIVISOR, D> C_dims,
    const BinaryOperator op,
    const TIn* A,
    const TIn* B,
    TOut* C) {
  CUDA_1D_KERNEL_LOOP(C_index, size) {
    int A_index = 0;
    int B_index = 0;
    int C_index_val = C_index;
#pragma unroll
    for (int i = D - 1; i >= 0; --i) {
      int d;
      FIXED_DIVISOR_DIV_MOD(C_dims.data[i], C_index_val, &C_index_val, &d);
      A_index += d * A_strides.data[i];
      B_index += d * B_strides.data[i];
    }
    C[C_index] = op(A[A_index], B[B_index]);
  }
}

template <typename TIn, typename TOut, class BinaryOperator>
CAFFE2_CUDA_EXPORT void BinaryOpWith2DBroadcasting(
    const int rows,
    const int cols,
    const bool rowwise_broadcast,
    const bool broadcast_1st,
    const BinaryOperator& op,
    const TIn* A,
    const TIn* B,
    TOut* C,
    HIPContext* context) {
  if (rows == 0 || cols == 0) {
    return;
  }
  const int size = rows * cols;
  const FIXED_DIVISOR cols_div(cols);
  if (rowwise_broadcast) {
    if (broadcast_1st) {
      RowwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, true>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    } else {
      RowwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, false>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    }
  } else {
    if (broadcast_1st) {
      ColwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, true>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    } else {
      ColwiseBinaryOpCUDAKenel<TIn, TOut, BinaryOperator, false>
          <<<CAFFE_GET_BLOCKS(size),
             CAFFE_CUDA_NUM_THREADS,
             0,
             context->hip_stream()>>>(size, cols_div, op, A, B, C);
    }
  }
}

template <typename TIn, typename TOut, class BinaryOperator, int D>
CAFFE2_CUDA_EXPORT void BroadcastBinaryOpImpl(
    const int* A_dims,
    const int* B_dims,
    const int* C_dims,
    const BinaryOperator& op,
    const TIn* A,
    const TIn* B,
    TOut* C,
    HIPContext* context) {
  SimpleArray<int, D> A_strides_array;
  SimpleArray<int, D> B_strides_array;
  SimpleArray<FIXED_DIVISOR, D> C_dims_array;
  int A_stride = 1;
  int B_stride = 1;
  for (int i = D - 1; i >= 0; --i) {
    if (C_dims[i] == 0) {
      return;
    }
    A_strides_array.data[i] = A_dims[i] == 1 ? 0 : A_stride;
    B_strides_array.data[i] = B_dims[i] == 1 ? 0 : B_stride;
    A_stride *= A_dims[i];
    B_stride *= B_dims[i];
    C_dims_array.data[i] = FIXED_DIVISOR(C_dims[i]);
  }
  const int size =
      std::accumulate(C_dims, C_dims + D, 1, std::multiplies<int>());
  BroadcastBinaryOpCUDAKernel<TIn, TOut, BinaryOperator, D>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          size, A_strides_array, B_strides_array, C_dims_array, op, A, B, C);
}

template <typename TIn, typename TOut, class BinaryOperator>
CAFFE2_CUDA_EXPORT void BroadcastBinaryOp(
    const int A_ndim,
    const int* A_dims,
    const int B_ndim,
    const int* B_dims,
    const BinaryOperator& op,
    const TIn* A,
    const TIn* B,
    TOut* C,
    HIPContext* context) {
  const int ndim = std::max(A_ndim, B_ndim);
  std::vector<int> A_dims_array(ndim);
  std::vector<int> B_dims_array(ndim);
  std::vector<int> C_dims_array(ndim);
  utils::ComputeBroadcastBinaryOpDims(
      A_ndim,
      A_dims,
      B_ndim,
      B_dims,
      A_dims_array.data(),
      B_dims_array.data(),
      C_dims_array.data());
  if (A_dims_array == B_dims_array) {
    const int size = std::accumulate(
        C_dims_array.cbegin(), C_dims_array.cend(), 1, std::multiplies<int>());
    SimpleBinaryOpCUDAKernel<TIn, TOut, BinaryOperator>
        <<<CAFFE_GET_BLOCKS(size),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(size, op, A, B, C);
    return;
  }
  int rows;
  int cols;
  bool broadcast_1st;
  if (utils::IsRowwiseBroadcastBinaryOp(
          ndim,
          A_dims_array.data(),
          B_dims_array.data(),
          &rows,
          &cols,
          &broadcast_1st)) {
    BinaryOpWith2DBroadcasting<TIn, TOut, BinaryOperator>(
        rows, cols, true, broadcast_1st, op, A, B, C, context);
    return;
  }
  if (utils::IsColwiseBroadcastBinaryOp(
          ndim,
          A_dims_array.data(),
          B_dims_array.data(),
          &rows,
          &cols,
          &broadcast_1st)) {
    BinaryOpWith2DBroadcasting<TIn, TOut, BinaryOperator>(
        rows, cols, false, broadcast_1st, op, A, B, C, context);
    return;
  }
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_3(
      ndim,
      BroadcastBinaryOpImpl,
      TIn,
      TOut,
      BinaryOperator,
      A_dims_array.data(),
      B_dims_array.data(),
      C_dims_array.data(),
      op,
      A,
      B,
      C,
      context);
}

} // namespace

#define DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(TIn, TOut, Func, Op)   \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Rowwise##Func<TIn, HIPContext, true>(          \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    RowwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, true>                    \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }                                                                       \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Rowwise##Func<TIn, HIPContext, false>(         \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    RowwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, false>                   \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }                                                                       \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Colwise##Func<TIn, HIPContext, true>(          \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    ColwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, true>                    \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }                                                                       \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Colwise##Func<TIn, HIPContext, false>(         \
      const int rows,                                                     \
      const int cols,                                                     \
      const TIn* A,                                                       \
      const TIn* B,                                                       \
      TOut* C,                                                            \
      HIPContext* context) {                                             \
    if (rows == 0 || cols == 0) {                                         \
      return;                                                             \
    }                                                                     \
    const int size = rows * cols;                                         \
    const FIXED_DIVISOR cols_div(cols);                                   \
    ColwiseBinaryOpCUDAKenel<TIn, TOut, Op<TIn>, false>                   \
        <<<CAFFE_GET_BLOCKS(size),                                        \
           CAFFE_CUDA_NUM_THREADS,                                        \
           0,                                                             \
           context->hip_stream()>>>(size, cols_div, Op<TIn>(), A, B, C); \
  }

#define DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(Func, Op)                \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(std::int32_t, bool, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(std::int64_t, bool, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(float, bool, Func, Op)        \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(double, bool, Func, Op)       \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op)

DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(EQ, thrust::equal_to)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(NE, thrust::not_equal_to)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(LT, thrust::less)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(LE, thrust::less_equal)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(GT, thrust::greater)
DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION(GE, thrust::greater_equal)

#undef DEFINE_2D_BROADCAST_CUDA_COMPARE_FUNCTION

#define DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Func, Op)             \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int32_t, std::int32_t, Func, Op)                            \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int64_t, std::int64_t, Func, Op)                            \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(float, float, Func, Op)   \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(double, double, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(at::Half, at::Half, Func, Op)

DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Add, AddFunctor)
DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Sub, SubFunctor)
DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Mul, MulFunctor)
DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION(Div, DivFunctor)

#undef DEFINE_2D_BROADCAST_CUDA_BINARY_FUNCTION

DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, And, thrust::logical_and)
DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Or, thrust::logical_or)
DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Xor, thrust::bit_xor)

#define DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op) \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                      \
      std::int32_t, std::int32_t, Func, Op)                        \
  DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION(                      \
      std::int64_t, std::int64_t, Func, Op)

DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseAnd, thrust::bit_and)
DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseOr, thrust::bit_or)
DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseXor, thrust::bit_xor)

#undef DEFINE_2D_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION

#undef DELEGATE_2D_BROADCAST_CUDA_BINARY_FUNCTION

#define DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(TIn, TOut, Func, Op)  \
  template <>                                                         \
  CAFFE2_CUDA_EXPORT void Func<TIn, HIPContext>(                     \
      const int A_ndim,                                               \
      const int* A_dims,                                              \
      const int B_ndim,                                               \
      const int* B_dims,                                              \
      const TIn* A,                                                   \
      const TIn* B,                                                   \
      TOut* C,                                                        \
      HIPContext* context) {                                         \
    BroadcastBinaryOp<TIn, TOut, Op<TIn>>(                            \
        A_ndim, A_dims, B_ndim, B_dims, Op<TIn>(), A, B, C, context); \
  }

#define DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(Func, Op)                \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(std::int32_t, bool, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(std::int64_t, bool, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(float, bool, Func, Op)        \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(double, bool, Func, Op)       \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op)

DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(EQ, thrust::equal_to)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(NE, thrust::not_equal_to)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(LT, thrust::less)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(LE, thrust::less_equal)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(GT, thrust::greater)
DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION(GE, thrust::greater_equal)

#undef DEFINE_BROADCAST_CUDA_COMPARE_FUNCTION

#define DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Func, Op)             \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int32_t, std::int32_t, Func, Op)                         \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(                          \
      std::int64_t, std::int64_t, Func, Op)                         \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(float, float, Func, Op)   \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(double, double, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(at::Half, at::Half, Func, Op)

DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Add, AddFunctor)
DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Sub, SubFunctor)
DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Mul, MulFunctor)
DEFINE_BROADCAST_CUDA_BINARY_FUNCTION(Div, DivFunctor)

#undef DEFINE_BROADCAST_CUDA_BINARY_FUNCTION

DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, And, thrust::logical_and)
DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Or, thrust::logical_or)
DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Xor, thrust::bit_xor)

#define DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(bool, bool, Func, Op) \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(                      \
      std::int32_t, std::int32_t, Func, Op)                     \
  DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION(std::int64_t, std::int64_t, Func, Op)

DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseAnd, thrust::bit_and)
DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseOr, thrust::bit_or)
DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION(BitwiseXor, thrust::bit_xor)

#undef DEFINE_BROADCAST_CUDA_BITWISE_BINARY_FUNCTION

#undef DELEGATE_BROADCAST_CUDA_BINARY_FUNCTION

#define DELEGATE_REDUCTION_FUNCTION(T, Funcname, func)                   \
  template <>                                                            \
  CAFFE2_CUDA_EXPORT void Funcname<T, HIPContext>(                      \
      const int N,                                                       \
      const T* src,                                                      \
      T* dst,                                                            \
      Tensor* scratch_ptr,                                               \
      HIPContext* context) {                                            \
    size_t memRequired = 0;                                              \
    hipcub::DeviceReduce::func(                                             \
        nullptr, memRequired, src, dst, N, context->hip_stream());      \
    auto buffer_size =                                                   \
        static_cast<int64_t>((memRequired + sizeof(T) - 1) / sizeof(T)); \
    scratch_ptr->Resize(std::vector<int64_t>{buffer_size});              \
    hipcub::DeviceReduce::func(                                             \
        static_cast<void*>(scratch_ptr->mutable_data<T>()),              \
        memRequired,                                                     \
        src,                                                             \
        dst,                                                             \
        N,                                                               \
        context->hip_stream());                                         \
  }

DELEGATE_REDUCTION_FUNCTION(float, ReduceMin, Min)
DELEGATE_REDUCTION_FUNCTION(float, ReduceMax, Max)
DELEGATE_REDUCTION_FUNCTION(int32_t, ReduceMax, Max)
DELEGATE_REDUCTION_FUNCTION(int64_t, ReduceMax, Max)

#undef DELEGATE_REDUCTION_FUNCTION

// Caffe2 gemm provides a simpler interface to the gemm functions, with the
// limitation that the data has to be contiguous in memory.
template <>
CAFFE2_CUDA_EXPORT void Gemm<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      N));
}

template <>
CAFFE2_CUDA_EXPORT void Gemm<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const at::Half* B,
    const float beta,
    at::Half* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  if (math_type == TensorProto_DataType_FLOAT) {
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
#ifdef __HIP_PLATFORM_HCC__
    // rocblas doesn't support cublasSgemmEx type API yet.
    // It has more general rocblas_gemm_ex API which is more close to
    // hipblasGemmEx rocblas_gemm_ex does D = alpha*op( A )*op( B ) + beta*C,
    // whereas cublasgemmEx does C = alpha*op( A )*op( B ) + beta*C
    ROCBLAS_ENFORCE(rocblas_gemm_ex(
        context->rocblashandle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B,
        rocblas_datatype_f16_r,
        ldb,
        A,
        rocblas_datatype_f16_r,
        lda,
        &beta,
        C,
        rocblas_datatype_f16_r,
        N,
        C, // D
        rocblas_datatype_f16_r, // D type
        N, // ldd
        rocblas_datatype_f32_r, // compute type
        rocblas_gemm_algo_standard, // rocblas_gemm_algo
        0,   // solution index, reserved for future use
        0)); // flags, reserved for future use
#else
    CUBLAS_ENFORCE(cublasSgemmEx(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B,
        HIP_R_16F,
        ldb,
        A,
        HIP_R_16F,
        lda,
        &beta,
        C,
        HIP_R_16F,
        N));
#endif // __HIP_PLATFORM_HCC__
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    // convert alpha, beta from float -> __half
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    // call hipblasHgemm
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemm(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(&alpha_fp16),
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(B),
        ldb,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(A),
        lda,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(&beta_fp16),
        reinterpret_cast<CUBLAS_HALF_TYPE*>(C),
        N));
  } else {
    // fail
    CAFFE_THROW("Unsupported math type");
  }
}

template <>
CAFFE2_CUDA_EXPORT void BiasCHW<float, HIPContext>(
    const float* bias,
    const float* bias_multiplier,
    const int bias_channels,
    const int image_size,
    float* image,
    HIPContext* context) {
  Gemm<float, HIPContext>(
      CblasNoTrans,
      CblasNoTrans,
      bias_channels,
      image_size,
      1,
      1,
      bias,
      bias_multiplier,
      1,
      image,
      context);
}

template <>
CAFFE2_CUDA_EXPORT void GemmBatched<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float** A,
    const float** B,
    const float beta,
    float** C,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if __CUDACC_VER_MAJOR__ < 8 || defined(__HIP_PLATFORM_HCC__)
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<float, HIPContext>(
        trans_A,
        trans_B,
        M,
        N,
        K,
        alpha,
        A[i],
        B[i],
        beta,
        C[i],
        context,
        math_type);
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  thrust::device_vector<const float*> A_device(A, A + batch_size);
  thrust::device_vector<const float*> B_device(B, B + batch_size);
  thrust::device_vector<float*> C_device(C, C + batch_size);
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemmBatched(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B_device.data().get(),
      ldb,
      A_device.data().get(),
      lda,
      &beta,
      C_device.data().get(),
      ldc,
      batch_size));
#endif
}

template <>
CAFFE2_CUDA_EXPORT void GemmStridedBatched<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const int A_stride,
    const float* B,
    const int B_stride,
    const float beta,
    float* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if __CUDACC_VER_MAJOR__ < 8 && !defined(__HIP_PLATFORM_HCC__)
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<float, HIPContext>(
        trans_A, trans_B, M, N, K, alpha, A, B, beta, C, context, math_type);
    A += A_stride;
    B += B_stride;
    C += C_stride;
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemmStridedBatched(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      B_stride,
      A,
      lda,
      A_stride,
      &beta,
      C,
      ldc,
      C_stride,
      batch_size));
#endif
}

template <>
CAFFE2_CUDA_EXPORT void GemmBatched<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half** A,
    const at::Half** B,
    const float beta,
    at::Half** C,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if __CUDACC_VER_MAJOR__ < 9
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<at::Half, HIPContext>(
        trans_A,
        trans_B,
        M,
        N,
        K,
        alpha,
        A[i],
        B[i],
        beta,
        C[i],
        context,
        math_type);
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  if (math_type == TensorProto_DataType_FLOAT) {
    thrust::device_vector<const void*> A_device(A, A + batch_size);
    thrust::device_vector<const void*> B_device(B, B + batch_size);
    thrust::device_vector<void*> C_device(C, C + batch_size);
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasGemmBatchedEx(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B_device.data().get(),
        HIP_R_16F,
        ldb,
        A_device.data().get(),
        HIP_R_16F,
        lda,
        &beta,
        C_device.data().get(),
        HIP_R_16F,
        ldc,
        batch_size,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    // Convert alpha, beta from float -> __half
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    thrust::host_vector<const __half*> A_array(batch_size);
    thrust::host_vector<const __half*> B_array(batch_size);
    thrust::host_vector<__half*> C_array(batch_size);
    for (int i = 0; i < batch_size; ++i) {
      A_array[i] = reinterpret_cast<const __half*>(A[i]);
      B_array[i] = reinterpret_cast<const __half*>(B[i]);
      C_array[i] = reinterpret_cast<__half*>(C[i]);
    }
    thrust::device_vector<const __half*> A_device(
        A_array.cbegin(), A_array.cend());
    thrust::device_vector<const __half*> B_device(
        B_array.cbegin(), B_array.cend());
    thrust::device_vector<__half*> C_device(C_array.cbegin(), C_array.cend());
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemmBatched(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha_fp16,
        B_device.data().get(),
        ldb,
        A_device.data().get(),
        lda,
        &beta_fp16,
        C_device.data().get(),
        ldc,
        batch_size));
  } else {
    CAFFE_THROW("Unsupported math type");
  }
#endif
}

template <>
CAFFE2_CUDA_EXPORT void GemmStridedBatched<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const int A_stride,
    const at::Half* B,
    const int B_stride,
    const float beta,
    at::Half* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
#if __CUDACC_VER_MAJOR__ < 8 && !defined(__HIP_PLATFORM_HCC__)
  // loop over matrices in the batch
  for (int i = 0; i < batch_size; ++i) {
    Gemm<at::Half, HIPContext>(
        trans_A, trans_B, M, N, K, alpha, A, B, beta, C, context, math_type);
    A += A_stride;
    B += B_stride;
    C += C_stride;
  }
#else
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const int ldc = N;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  if (math_type == TensorProto_DataType_FLOAT) {
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
#ifdef __HIP_PLATFORM_HCC__
    // D[i*stride_d] = alpha*op(A[i*stride_a])*op(B[i*stride_b]) +
    // beta*C[i*stride_c], for i in [0,batch_count-1]
    ROCBLAS_ENFORCE(rocblas_gemm_strided_batched_ex(
        context->rocblashandle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B,
        rocblas_datatype_f16_r,
        ldb,
        B_stride,
        A,
        rocblas_datatype_f16_r,
        lda,
        A_stride,
        &beta,
        C,
        rocblas_datatype_f16_r,
        ldc,
        C_stride,
        C, // D
        rocblas_datatype_f16_r, // D type
        ldc, // ldd
        C_stride, // D stride
        batch_size,
        rocblas_datatype_f32_r, // compute type
        rocblas_gemm_algo_standard, // rocblas_gemm_algo
        0,   // solution index, reserved for future use
        0)); // flags, reserved for future use
#else
    CUBLAS_ENFORCE(hipblasGemmStridedBatchedEx(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        &alpha,
        B,
        HIP_R_16F,
        ldb,
        B_stride,
        A,
        HIP_R_16F,
        lda,
        A_stride,
        &beta,
        C,
        HIP_R_16F,
        ldc,
        C_stride,
        batch_size,
        HIP_R_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP));
#endif // __HIP_PLATFORM_HCC__
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    // Convert alpha, beta from float -> __half
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemmStridedBatched(
        context->cublas_handle(),
        cu_trans_B,
        cu_trans_A,
        N,
        M,
        K,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(&alpha_fp16),
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(B),
        ldb,
        B_stride,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(A),
        lda,
        A_stride,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(&beta_fp16),
        reinterpret_cast<CUBLAS_HALF_TYPE*>(C),
        ldc,
        C_stride,
        batch_size));
  } else {
    CAFFE_THROW("Unsupported math type");
  }
#endif
}

template <>
CAFFE2_CUDA_EXPORT void Gemv<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const int M,
    const int N,
    const float alpha,
    const float* A,
    const float* x,
    const float beta,
    float* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemv(
      context->cublas_handle(),
      cu_trans_A,
      N,
      M,
      &alpha,
      A,
      N,
      x,
      1,
      &beta,
      y,
      1));
}

template <>
CAFFE2_CUDA_EXPORT void Gemv<at::Half, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const int M,
    const int N,
    const float alpha,
    const at::Half* A,
    const at::Half* x,
    const float beta,
    at::Half* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  // sort out what we need to call cublasSgemmEx / hipblasHgemm
  const int m = (cu_trans_A == HIPBLAS_OP_N) ? N : M;
  const int k = (cu_trans_A == HIPBLAS_OP_N) ? M : N;
  const int lda = (cu_trans_A == HIPBLAS_OP_N) ? m : k;
  const int ldc = m;

  if (math_type == TensorProto_DataType_FLOAT) {
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
#ifdef __HIP_PLATFORM_HCC__
    // rocblas doesn't support cublasSgemmEx type API yet.
    // It has more general rocblas_gemm_ex API which is more close to
    // hipblasGemmEx rocblas_gemm_ex does D = alpha*op( A )*op( B ) + beta*C,
    // whereas cublasgemmEx does C = alpha*op( A )*op( B ) + beta*C
    ROCBLAS_ENFORCE(rocblas_gemm_ex(
        context->rocblashandle(),
        cu_trans_A,
        rocblas_operation_none,
        m,
        1,
        k,
        &alpha,
        A,
        rocblas_datatype_f16_r,
        lda,
        x,
        rocblas_datatype_f16_r,
        k,
        &beta,
        y,
        rocblas_datatype_f16_r,
        ldc,
        y, // D
        rocblas_datatype_f16_r, // D type
        ldc, // ldd
        rocblas_datatype_f32_r, // compute type
        rocblas_gemm_algo_standard, // rocblas_gemm_algo
        0,   // solution index, reserved for future use
        0)); // flags, reserved for future use
#else
    CUBLAS_ENFORCE(cublasSgemmEx(
        context->cublas_handle(),
        cu_trans_A,
        HIPBLAS_OP_N,
        m,
        1,
        k,
        &alpha,
        A,
        HIP_R_16F,
        lda,
        x,
        HIP_R_16F,
        k,
        &beta,
        y,
        HIP_R_16F,
        ldc));
#endif // __HIP_PLATFORM_HCC__
  } else if (math_type == TensorProto_DataType_FLOAT16) {
    const __half alpha_fp16 = at::Half(alpha);
    const __half beta_fp16 = at::Half(beta);
    CUBLAS_ENFORCE(hipblasSetPointerMode(
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
    CUBLAS_ENFORCE(hipblasHgemm(
        context->cublas_handle(),
        cu_trans_A,
        HIPBLAS_OP_N,
        m,
        1,
        k,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(&alpha_fp16),
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(A),
        lda,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(x),
        k,
        reinterpret_cast<const CUBLAS_HALF_TYPE*>(&beta_fp16),
        reinterpret_cast<CUBLAS_HALF_TYPE*>(y),
        ldc));
  } else {
    // fail
    CAFFE_THROW("Unsupported math type");
  }
}

#ifndef __HIP_PLATFORM_HCC__

// No change, but required. Defer to default CUDA engine
template <>
CAFFE2_CUDA_EXPORT void Gemm<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const float* B,
    const float beta,
    float* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  return Gemm<float, HIPContext>(
      trans_A, trans_B, M, N, K, alpha, A, B, beta, C, context, math_type);
}

template <>
CAFFE2_CUDA_EXPORT void Gemm<at::Half, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const at::Half* B,
    const float beta,
    at::Half* C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const int lda = (trans_A == CblasNoTrans) ? K : M;
  const int ldb = (trans_B == CblasNoTrans) ? N : K;
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

  // enable TensorCore for this call on this handle
  if (TensorCoreAvailable()) {
    CUBLAS_ENFORCE(
        hipblasSetMathMode(context->cublas_handle(), HIPBLAS_TENSOR_OP_MATH));
  }

  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasGemmEx(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      HIP_R_16F,
      ldb,
      A,
      HIP_R_16F,
      lda,
      &beta,
      C,
      HIP_R_16F,
      N,
      HIP_R_32F,
      CUBLAS_GEMM_DFALT_TENSOR_OP));

  // Now disable TensorCore math for subsequent calls to this handle
  if (TensorCoreAvailable()) {
    CUBLAS_ENFORCE(
        hipblasSetMathMode(context->cublas_handle(), HIPBLAS_DEFAULT_MATH));
  }
}

template <>
CAFFE2_CUDA_EXPORT void GemmBatched<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float** A,
    const float** B,
    const float beta,
    float** C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  GemmBatched<float, HIPContext, DefaultEngine>(
      trans_A,
      trans_B,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      B,
      beta,
      C,
      context,
      math_type);
}

template <>
CAFFE2_CUDA_EXPORT void GemmBatched<at::Half, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half** A,
    const at::Half** B,
    const float beta,
    at::Half** C,
    HIPContext* context,
    TensorProto::DataType math_type) {
  GemmBatched<at::Half, HIPContext, DefaultEngine>(
      trans_A,
      trans_B,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      B,
      beta,
      C,
      context,
      math_type);
}

template <>
CAFFE2_CUDA_EXPORT void
GemmStridedBatched<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const int A_stride,
    const float* B,
    const int B_stride,
    const float beta,
    float* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
  GemmStridedBatched<float, HIPContext, DefaultEngine>(
      trans_A,
      trans_B,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      A_stride,
      B,
      B_stride,
      beta,
      C,
      C_stride,
      context,
      math_type);
}

template <>
CAFFE2_CUDA_EXPORT void
GemmStridedBatched<at::Half, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int batch_size,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const at::Half* A,
    const int A_stride,
    const at::Half* B,
    const int B_stride,
    const float beta,
    at::Half* C,
    const int C_stride,
    HIPContext* context,
    TensorProto::DataType math_type) {
  GemmStridedBatched<at::Half, HIPContext, DefaultEngine>(
      trans_A,
      trans_B,
      batch_size,
      M,
      N,
      K,
      alpha,
      A,
      A_stride,
      B,
      B_stride,
      beta,
      C,
      C_stride,
      context,
      math_type);
}

template <>
CAFFE2_CUDA_EXPORT void Gemv<float, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const int M,
    const int N,
    const float alpha,
    const float* A,
    const float* x,
    const float beta,
    float* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
  Gemv<float, HIPContext, DefaultEngine>(
      trans_A, M, N, alpha, A, x, beta, y, context, math_type);
}

template <>
CAFFE2_CUDA_EXPORT void Gemv<at::Half, HIPContext, TensorCoreEngine>(
    const CBLAS_TRANSPOSE trans_A,
    const int M,
    const int N,
    const float alpha,
    const at::Half* A,
    const at::Half* x,
    const float beta,
    at::Half* y,
    HIPContext* context,
    TensorProto::DataType math_type) {
  Gemv<at::Half, HIPContext, DefaultEngine>(
      trans_A, M, N, alpha, A, x, beta, y, context, math_type);
}

#endif

template <>
CAFFE2_CUDA_EXPORT void GemmEx<float, HIPContext>(
    const CBLAS_TRANSPOSE trans_A,
    const CBLAS_TRANSPOSE trans_B,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float* A,
    const int lda,
    const float* B,
    const int ldb,
    const float beta,
    float* C,
    const int ldc,
    HIPContext* context) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  const hipblasOperation_t cu_trans_A =
      (trans_A == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  const hipblasOperation_t cu_trans_B =
      (trans_B == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(
      hipblasSetPointerMode(context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cu_trans_B,
      cu_trans_A,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      ldc));
}

// Batched Add variants
namespace {

template <typename T>
__global__ void AddStripedBatchKernel(
    const int N,
    const T* first,
    T* Y,
    const int stripe,
    const int batch) {
  for (int j = 0; j < batch; j++) {
    const T* x = first + j * stripe;
    CUDA_1D_KERNEL_LOOP(i, N) {
      float tmpY = convert::To<T, float>(Y[i]);
      tmpY += convert::To<T, float>(x[i]);
      Y[i] = convert::To<float, T>(tmpY);
    }
  }
}
} // namespace

#define CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(T)              \
  template <>                                                     \
  CAFFE2_CUDA_EXPORT void AddStripedBatch<T, HIPContext>(        \
      const int N,                                                \
      const T* first,                                             \
      T* Y,                                                       \
      const int stripe,                                           \
      const int batch,                                            \
      HIPContext* context) {                                     \
    AddStripedBatchKernel<T>                                      \
        <<<CAFFE_GET_BLOCKS(N),                                   \
           CAFFE_CUDA_NUM_THREADS,                                \
           0,                                                     \
           context->hip_stream()>>>(N, first, Y, stripe, batch); \
  }

CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(float);
CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH(at::Half);
#undef CAFFE2_SPECIALIZED_CUDA_ADD_STRIPED_BATCH

namespace {
template <typename T>
__global__ void
UniformShift(const size_t N, const float min, const float max, T* x) {
  float scale = max - min;
  CUDA_1D_KERNEL_LOOP(i, N) {
    x[i] = convert::To<float, T>(convert::To<T, float>(x[i]) * scale + min);
  }
}

__global__ void
UniformIntFit(const size_t N, const int min, const int max, unsigned int* x) {
  int* x_int = reinterpret_cast<int*>(x);
  int range = (max - min + 1);
  CUDA_1D_KERNEL_LOOP(i, N) {
    x_int[i] = min + static_cast<int>(x[i] % range);
  }
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void RandUniform<float, HIPContext>(
    const size_t n,
    const float min,
    const float max,
    float* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerateUniform(context->curand_generator(), r, n));
  UniformShift<float>
      <<<CAFFE_GET_BLOCKS(n),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(n, min, max, r);
}

template <>
CAFFE2_CUDA_EXPORT void RandUniform<double, HIPContext>(
    const size_t n,
    const double min,
    const double max,
    double* r,
    HIPContext* context) {
  CURAND_ENFORCE(
      hiprandGenerateUniformDouble(context->curand_generator(), r, n));
  UniformShift<double>
      <<<CAFFE_GET_BLOCKS(n),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(n, min, max, r);
}

template <>
CAFFE2_CUDA_EXPORT void RandUniform<int, HIPContext>(
    const size_t n,
    const int min,
    const int max,
    int* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerate(
      context->curand_generator(), reinterpret_cast<unsigned int*>(r), n));
  UniformIntFit<<<
      CAFFE_GET_BLOCKS(n),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(
      n, min, max, reinterpret_cast<unsigned int*>(r));
}

template <typename T>
size_t HandleOddLengthRandGaussian(
    const size_t n,
    const T mean,
    const T std,
    T* r,
    HIPContext* context) {
  if (n % 2 == 1) {
    std::default_random_engine generator;
    std::normal_distribution<T> distribution(mean, std);
    const T random_value = distribution(generator);
    Set<T, HIPContext>(1, random_value, r + (n - 1), context);
    return n - 1;
  }
  return n;
}

template <>
CAFFE2_CUDA_EXPORT void RandGaussian<float, HIPContext>(
    const size_t n,
    const float mean,
    const float std,
    float* r,
    HIPContext* context) {
  // If n is odd, we add a random Gaussian value at the end manually
  // and generate n-1 random values using hiprandGenerateNormal.
  // hiprandGenerateNormal requires n to be even.
  const size_t even_n =
      HandleOddLengthRandGaussian<float>(n, mean, std, r, context);
  CURAND_ENFORCE(
      hiprandGenerateNormal(context->curand_generator(), r, even_n, mean, std));
}

template <>
CAFFE2_CUDA_EXPORT void RandGaussian<double, HIPContext>(
    const size_t n,
    const double mean,
    const double std,
    double* r,
    HIPContext* context) {
  const size_t even_n =
      HandleOddLengthRandGaussian<double>(n, mean, std, r, context);
  CURAND_ENFORCE(hiprandGenerateNormalDouble(
      context->curand_generator(), r, even_n, mean, std));
}

template <>
CAFFE2_CUDA_EXPORT void Dot<float, HIPContext>(
    const int n,
    const float* a,
    const float* b,
    float* y,
    HIPContext* context) {
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasSdot(context->cublas_handle(), n, a, 1, b, 1, y));
}

template <>
CAFFE2_CUDA_EXPORT void Dot<at::Half, HIPContext>(
    const int n,
    const at::Half* a,
    const at::Half* b,
    at::Half* y,
    HIPContext* context) {
#if defined __HIP_PLATFORM_HCC__ && HIP_VERSION < 210
  CAFFE_THROW("HIP currently does not support FP16 completely yet.");
#elif defined __HIP_PLATFORM_HCC__ && HIP_VERSION >= 210
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(rocblas_hdot(
      context->cublas_handle(),
      n,
      reinterpret_cast<const rocblas_half*>(a),
      1,
      reinterpret_cast<const rocblas_half*>(b),
      1,
      reinterpret_cast<rocblas_half*>(y)));
#else
  // execute with 32-bit math
  CUBLAS_ENFORCE(hipblasSetPointerMode(
      context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));
  CUBLAS_ENFORCE(hipblasDotEx(
      context->cublas_handle(),
      n,
      a,
      HIP_R_16F,
      1,
      b,
      HIP_R_16F,
      1,
      y,
      HIP_R_16F,
      HIP_R_32F));
#endif
}

// A previous version of caffe2 used Thrust but it turns out that thrust
// reduction has an implicit scratch space allocation and deallocation, which
// may interfere with NCCL and create a deadlock. Hence we are using a custom
// reduction here.
#define SUM_KERNEL_NTHREADS 128
template <typename T>
__global__ void SumKernel(const int N, const T* X, T* Y, bool square) {
  const int idx = threadIdx.x;
  __shared__ float reduction_buffer[SUM_KERNEL_NTHREADS];

  reduction_buffer[idx] = 0;

  // A multilevel reduction.
  // N -> 128
  if (!square) {
    for (int i = idx; i < N; i += SUM_KERNEL_NTHREADS) {
      reduction_buffer[idx] += convert::To<T, float>(X[i]);
    }
  } else {
    for (int i = idx; i < N; i += SUM_KERNEL_NTHREADS) {
      float Xi = convert::To<T, float>(X[i]);
      reduction_buffer[idx] += Xi * Xi;
    }
  }
  __syncthreads();
  // 128 -> 32
  if (idx < 32) {
    reduction_buffer[idx] += reduction_buffer[idx + 32] +
        reduction_buffer[idx + 64] + reduction_buffer[idx + 96];
  }
  __syncthreads();
  // 32 -> 1
  if (idx == 0) {
    float tmp = 0;
    for (int i = 0; i < 32; ++i) {
      tmp += reduction_buffer[i];
    }
    *Y = convert::To<float, T>(tmp);
  }
}

// According to the benchmarks script
// caffe2/caffe2/experiments/python/device_reduce_sum_bench.py,
// device reduce is slower for N <= 10000.
#define DEVICE_REDUCE_SIZE_THRESHOLD 10000

namespace {

template <typename T>
__global__ void SumConvertKernel(float* sum, T* dest) {
  *dest = convert::To<float, T>(*sum);
}

template <typename T, typename IterT>
CAFFE2_CUDA_EXPORT void SumGenericIter(
    const int N,
    IterT it,
    T*& dest,
    HIPContext* context,
    Tensor* scratch_ptr) {
  size_t memRequired = 0;
  hipcub::DeviceReduce::Sum(
      nullptr, memRequired, it, dest, N, context->hip_stream());
  auto buffer_size =
      static_cast<int64_t>((memRequired + sizeof(T) - 1) / sizeof(T));
  if (!dest) {
    // allocate one more T at the end of scratch for dest
    scratch_ptr->Resize(std::vector<int64_t>{buffer_size + 1});
    dest = scratch_ptr->template mutable_data<T>() + buffer_size;
  } else {
    scratch_ptr->Resize(std::vector<int64_t>{buffer_size});
  }
  hipcub::DeviceReduce::Sum(
      static_cast<void*>(scratch_ptr->template mutable_data<T>()),
      memRequired,
      it,
      dest,
      N,
      context->hip_stream());
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void Sum<float, HIPContext>(
    const int N,
    const float* x,
    float* y,
    HIPContext* context,
    Tensor* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SumGenericIter<float>(N, x, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, false);
  }
}

template <>
CAFFE2_CUDA_EXPORT void Sum<int32_t, HIPContext>(
    const int N,
    const int32_t* x,
    int32_t* y,
    HIPContext* context,
    Tensor* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SumGenericIter<int32_t>(N, x, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, false);
  }
}

namespace {
template <typename T>
struct FloatTransform {
  inline __host__ __device__ float operator()(const T v) const {
    return convert::To<T, float>(v);
  }
};
} // namespace

#define CAFFE2_MATH_SUM_FUNC(T)                                           \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Sum<T, HIPContext>(                            \
      const int N,                                                        \
      const T* x,                                                         \
      T* y,                                                               \
      HIPContext* context,                                               \
      Tensor* scratch_ptr) {                                              \
    if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {                \
      FloatTransform<T> transform;                                        \
      hipcub::TransformInputIterator<float, FloatTransform<T>, const T*> it( \
          x, transform);                                                  \
      float* sum = nullptr;                                               \
      SumGenericIter<float>(N, it, sum, context, scratch_ptr);            \
      SumConvertKernel<<<1, 1, 0, context->hip_stream()>>>(sum, y);      \
    } else {                                                              \
      SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(   \
          N, x, y, false);                                                \
    }                                                                     \
  }

CAFFE2_MATH_SUM_FUNC(at::Half)
#undef CAFFE2_MATH_SUM_FUNC

namespace {
template <typename T>
struct SqrTransform {
  inline __host__ __device__ T operator()(const T v) const {
    return v * v;
  }
};
} //  namespace

template <>
CAFFE2_CUDA_EXPORT void SumSqr<float, HIPContext>(
    const int N,
    const float* x,
    float* y,
    HIPContext* context,
    Tensor* scratch_ptr) {
  if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {
    SqrTransform<float> transform;
    hipcub::TransformInputIterator<float, SqrTransform<float>, const float*> it(
        x, transform);
    SumGenericIter<float>(N, it, y, context, scratch_ptr);
  } else {
    SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(
        N, x, y, true);
  }
}

#define CAFFE2_MATH_SUMSQR_FUNC(T)                                      \
  template <>                                                           \
  CAFFE2_CUDA_EXPORT void SumSqr<T, HIPContext>(                       \
      const int N,                                                      \
      const T* x,                                                       \
      T* y,                                                             \
      HIPContext* context,                                             \
      Tensor* scratch_ptr) {                                            \
    if (scratch_ptr && N > DEVICE_REDUCE_SIZE_THRESHOLD) {              \
      FloatTransform<T> float_transform;                                \
      hipcub::TransformInputIterator<float, FloatTransform<T>, const T*>   \
          float_it(x, float_transform);                                 \
      SqrTransform<float> sqr_transform;                                \
      hipcub::TransformInputIterator<                                      \
          float,                                                        \
          SqrTransform<float>,                                          \
          decltype(float_it)>                                           \
          it(float_it, sqr_transform);                                  \
      float* sum = nullptr;                                             \
      SumGenericIter<float>(N, it, sum, context, scratch_ptr);          \
      SumConvertKernel<<<1, 1, 0, context->hip_stream()>>>(sum, y);    \
    } else {                                                            \
      SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>( \
          N, x, y, true);                                               \
    }                                                                   \
  }

CAFFE2_MATH_SUMSQR_FUNC(at::Half)
#undef CAFFE2_MATH_SUMSQR_FUNC
#undef DEVICE_REDUCE_SIZE_THRESHOLD

namespace {
template <typename T>
__global__ void
SelectKernel(const int N, const int D, const T* x, const int* idx, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = x[i * D + idx[i]];
  }
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void Select<float, HIPContext>(
    const int N,
    const int D,
    const float* x,
    const int* idx,
    float* y,
    HIPContext* context) {
  SelectKernel<float>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(N, D, x, idx, y);
}

template <>
CAFFE2_CUDA_EXPORT void Select<at::Half, HIPContext>(
    const int N,
    const int D,
    const at::Half* x,
    const int* idx,
    at::Half* y,
    HIPContext* context) {
  SelectKernel<at::Half>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(N, D, x, idx, y);
}

namespace {

template <typename T>
__global__ void Im2ColNCHWCUDAKernel(
    const int n,
    const int input_h,
    const int input_w,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_h,
    const int output_w,
    const T* img_data,
    T* col_data) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int w_out = index % output_w;
    const int h_index = index / output_w;
    const int h_out = h_index % output_h;
    const int channel_in = h_index / output_h;
    const int channel_out = channel_in * kernel_h * kernel_w;
    const int h_in = h_out * stride_h - pad_t;
    const int w_in = w_out * stride_w - pad_l;
    const int output_size = output_h * output_w;
    T* col_data_ptr =
        col_data + (channel_out * output_h + h_out) * output_w + w_out;
    const T* img_data_ptr =
        img_data + (channel_in * input_h + h_in) * input_w + w_in;
    int dh = 0;
    for (int i = 0; i < kernel_h; ++i) {
      int dw = 0;
      for (int j = 0; j < kernel_w; ++j) {
        const int h = h_in + dh;
        const int w = w_in + dw;
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? __ldg(img_data_ptr + dh * input_w + dw)
            : 0;
#else
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? img_data_ptr[dh * input_w + dw]
            : 0;
#endif
        col_data_ptr += output_size;
        dw += dilation_w;
      }
      dh += dilation_h;
    }
  }
}

template <typename T>
__global__ void Im2ColNHWCCUDAKernel(
    const int n,
    const int input_h,
    const int input_w,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_w,
    const int channels,
    const T* img_data,
    T* col_data) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    const int channel_in = index % channels;
    const int w_out = index / channels % output_w;
    const int h_out = index / channels / output_w;
    const int h_in = h_out * stride_h - pad_t;
    const int w_in = w_out * stride_w - pad_l;
    T* col_data_ptr = col_data +
        (h_out * output_w + w_out) * channels * kernel_h * kernel_w +
        channel_in;
    int dh = 0;
    for (int i = 0; i < kernel_h; ++i) {
      int dw = 0;
      for (int j = 0; j < kernel_w; ++j) {
        const int h = h_in + dh;
        const int w = w_in + dw;
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? __ldg(img_data + (h * input_w + w) * channels + channel_in)
            : 0;
#else
        *col_data_ptr = utils::IsAGeZeroAndALtB(h, input_h) &&
                utils::IsAGeZeroAndALtB(w, input_w)
            ? img_data[(h * input_w + w) * channels + channel_in]
            : 0;
#endif
        col_data_ptr += channels;
        dw += dilation_w;
      }
      dh += dilation_h;
    }
  }
}

template <typename T>
__global__ void Col2ImNCHWCUDAKernel(
    const int n,
    const int input_h,
    const int input_w,
    const int patch_h,
    const int patch_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_h,
    const int output_w,
    const T* col_data,
    T* img_data) {
  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    const int w = index % input_w + pad_l;
    const int h = index / input_w % input_h + pad_t;
    const int c = index / (input_h * input_w);

    // compute the start and end of the output
    const int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    const int w_col_end = min(w / stride_w + 1, output_w);
    const int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    const int h_col_end = min(h / stride_h + 1, output_h);

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = (h - h_col * stride_h);
        int w_k = (w - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          const int col_data_index =
              (((c * patch_h + h_k) * patch_w + w_k) * output_h + h_col) *
                  output_w +
              w_col;
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
          val += __ldg(col_data + col_data_index);
#else
          val += col_data[col_data_index];
#endif
        }
      }
    }
    img_data[index] = val;
  }
}

template <typename T>
__global__ void Col2ImNHWCCUDAKernel(
    const int n,
    const int input_w,
    const int channels,
    const int patch_h,
    const int patch_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int stride_h,
    const int stride_w,
    const int output_h,
    const int output_w,
    const T* col_data,
    T* img_data) {
  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    const int c = index % channels;
    const int w = index / channels % input_w + pad_l;
    const int h = index / channels / input_w + pad_t;
    // compute the start and end of the output
    const int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    const int w_col_end = min(w / stride_w + 1, output_w);
    const int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    const int h_col_end = min(h / stride_h + 1, output_h);
    const int channels_col = patch_h * patch_w * channels;

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = h - h_col * stride_h;
        int w_k = w - w_col * stride_w;
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          const int c_col = (h_k * patch_w + w_k) * channels + c;
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
          val += __ldg(
              col_data + (h_col * output_w + w_col) * channels_col + c_col);
#else
          val += col_data[(h_col * output_w + w_col) * channels_col + c_col];
#endif
        }
      }
    }
    img_data[index] = val;
  }
}

template <typename T, int N, bool kCol2Im>
__global__ void Im2ColNdNCHWCUDAKernel(
    const int outer_size,
    const int inner_size,
    const int kernel_size,
    SimpleArray<int, N + 1> img_shape,
    SimpleArray<int, N + 1> col_shape,
    SimpleArray<int, N> kernel_shape,
    SimpleArray<int, N> stride,
    SimpleArray<int, N> dilation,
    SimpleArray<int, N> pad,
    const T* X_data,
    T* Y_data) {
  int d_offset[N];
  int d_iter[N];
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    int offset_i = i;
#pragma unroll
    for (int d_i = N - 1; d_i >= 0; --d_i) {
      d_offset[d_i] = offset_i % kernel_shape.data[d_i];
      offset_i /= kernel_shape.data[d_i];
    }
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      int offset_j = j;
#pragma unroll
      for (int d_i = N - 1; d_i >= 0; --d_i) {
        d_iter[d_i] = offset_j % col_shape.data[d_i + 1];
        offset_j /= col_shape.data[d_i + 1];
      }
      const int col_index = i * inner_size + j;
      int img_index = i / kernel_size;
      bool is_padding = false;
#pragma unroll
      for (int d_i = 0; d_i < N; ++d_i) {
        const int d_img = d_iter[d_i] * stride.data[d_i] - pad.data[d_i] +
            d_offset[d_i] * dilation.data[d_i];
        is_padding |= !utils::IsAGeZeroAndALtB(d_img, img_shape.data[d_i + 1]);
        img_index = img_index * img_shape.data[d_i + 1] + d_img;
      }
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
      if (!kCol2Im) {
        Y_data[col_index] = is_padding ? 0 : __ldg(X_data + img_index);
      } else if (!is_padding) {
        atomicAdd(Y_data + img_index, __ldg(X_data + col_index));
      }
#else
      if (!kCol2Im) {
        Y_data[col_index] = is_padding ? 0 : X_data[img_index];
      } else if (!is_padding) {
        atomicAdd(Y_data + img_index, X_data[col_index]);
      }
#endif
    }
  }
}

template <typename T, int N>
CAFFE2_CUDA_EXPORT void Im2ColNdNCHWCUDAImpl(
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* img_data,
    float* col_data,
    HIPContext* context) {
  const int outer_size = col_shape[0];
  const int inner_size = col_size / outer_size;
  const int kernel_size = std::accumulate(
      kernel_shape, kernel_shape + N, 1, std::multiplies<int>());
  SimpleArray<int, N + 1> img_shape_array;
  SimpleArray<int, N + 1> col_shape_array;
  SimpleArray<int, N> kernel_shape_array;
  SimpleArray<int, N> stride_array;
  SimpleArray<int, N> dilation_array;
  SimpleArray<int, N> pad_array;
  std::memcpy(img_shape_array.data, img_shape, (N + 1) * sizeof(int));
  std::memcpy(col_shape_array.data, col_shape, (N + 1) * sizeof(int));
  std::memcpy(kernel_shape_array.data, kernel_shape, N * sizeof(int));
  std::memcpy(stride_array.data, stride, N * sizeof(int));
  std::memcpy(dilation_array.data, dilation, N * sizeof(int));
  std::memcpy(pad_array.data, pad, N * sizeof(int));
  Im2ColNdNCHWCUDAKernel<T, N, false>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size,
          inner_size,
          kernel_size,
          img_shape_array,
          col_shape_array,
          kernel_shape_array,
          stride_array,
          dilation_array,
          pad_array,
          img_data,
          col_data);
}

template <typename T, int N>
CAFFE2_CUDA_EXPORT void Col2ImNdNCHWCUDAImpl(
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* col_data,
    float* img_data,
    HIPContext* context) {
  const int outer_size = col_shape[0];
  const int inner_size = col_size / outer_size;
  const int kernel_size = std::accumulate(
      kernel_shape, kernel_shape + N, 1, std::multiplies<int>());
  SimpleArray<int, N + 1> img_shape_array;
  SimpleArray<int, N + 1> col_shape_array;
  SimpleArray<int, N> kernel_shape_array;
  SimpleArray<int, N> stride_array;
  SimpleArray<int, N> dilation_array;
  SimpleArray<int, N> pad_array;
  std::memcpy(img_shape_array.data, img_shape, (N + 1) * sizeof(int));
  std::memcpy(col_shape_array.data, col_shape, (N + 1) * sizeof(int));
  std::memcpy(kernel_shape_array.data, kernel_shape, N * sizeof(int));
  std::memcpy(stride_array.data, stride, N * sizeof(int));
  std::memcpy(dilation_array.data, dilation, N * sizeof(int));
  std::memcpy(pad_array.data, pad, N * sizeof(int));
  Set<T, HIPContext>(img_size, 0, img_data, context);
  Im2ColNdNCHWCUDAKernel<T, N, true>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size,
          inner_size,
          kernel_size,
          img_shape_array,
          col_shape_array,
          kernel_shape_array,
          stride_array,
          dilation_array,
          pad_array,
          col_data,
          img_data);
}

} // namespace

template <>
CAFFE2_CUDA_EXPORT void Im2Col<float, HIPContext, StorageOrder::NCHW>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int /* groups */) {
  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = channels * output_h * output_w;
  Im2ColNCHWCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          height,
          width,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_h,
          output_w,
          img_data,
          col_data);
}

template <>
CAFFE2_CUDA_EXPORT void Im2Col<float, HIPContext, StorageOrder::NHWC>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int groups) {
  CAFFE_ENFORCE_EQ(groups, 1, "groups must be 1 for GPU NHWC Im2Col");

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = output_h * output_w * channels;
  Im2ColNHWCCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          height,
          width,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_w,
          channels,
          img_data,
          col_data);
}

template <>
CAFFE2_CUDA_EXPORT void Col2Im<float, HIPContext, StorageOrder::NCHW>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    const int /* groups */) {
  // In NCHW, the number of groups doesn't affect Col2Im.
  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = channels * height * width;
  Col2ImNCHWCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          height,
          width,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_h,
          output_w,
          col_data,
          img_data);
}

template <>
CAFFE2_CUDA_EXPORT void Col2Im<float, HIPContext, StorageOrder::NHWC>(
    const int channels,
    const int height,
    const int width,
    const int kernel_h,
    const int kernel_w,
    const int dilation_h,
    const int dilation_w,
    const int pad_t,
    const int pad_l,
    const int pad_b,
    const int pad_r,
    const int stride_h,
    const int stride_w,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    const int groups) {
  CAFFE_ENFORCE_EQ(groups, 1, "groups must be 1 for GPU NHWC Col2Im");

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;
  const int output_h = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  const int output_w = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  const int num_kernels = height * width * channels;
  Col2ImNHWCCUDAKernel<float>
      <<<CAFFE_GET_BLOCKS(num_kernels),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          num_kernels,
          width,
          channels,
          kernel_h,
          kernel_w,
          dilation_h,
          dilation_w,
          pad_t,
          pad_l,
          stride_h,
          stride_w,
          output_h,
          output_w,
          col_data,
          img_data);
}

template <>
CAFFE2_CUDA_EXPORT void Im2ColNd<float, HIPContext, StorageOrder::NCHW>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int /* groups */) {
  // In NCHW, the number of groups doesn't affect Im2Col.
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(
      N,
      Im2ColNdNCHWCUDAImpl,
      float,
      img_size,
      col_size,
      img_shape,
      col_shape,
      kernel_shape,
      stride,
      dilation,
      pad,
      img_data,
      col_data,
      context);
}

template <>
CAFFE2_CUDA_EXPORT void Im2ColNd<float, HIPContext, StorageOrder::NHWC>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* img_data,
    float* col_data,
    HIPContext* context,
    const int groups) {
  CAFFE_NOT_IMPLEMENTED;
}

template <>
CAFFE2_CUDA_EXPORT void Col2ImNd<float, HIPContext, StorageOrder::NCHW>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    int /* groups */) {
  // In NCHW, the number of groups doesn't affect Col2Im.
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(
      N,
      Col2ImNdNCHWCUDAImpl,
      float,
      img_size,
      col_size,
      img_shape,
      col_shape,
      kernel_shape,
      stride,
      dilation,
      pad,
      col_data,
      img_data,
      context);
}

template <>
CAFFE2_CUDA_EXPORT void Col2ImNd<float, HIPContext, StorageOrder::NHWC>(
    const int N,
    const int img_size,
    const int col_size,
    const int* img_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const float* col_data,
    float* img_data,
    HIPContext* context,
    int groups) {
  CAFFE_NOT_IMPLEMENTED;
}

template <>
CAFFE2_CUDA_EXPORT void CopyMatrix<HIPContext>(
    const size_t itemsize,
    const int M,
    const int N,
    const void* A,
    const int lda,
    void* B,
    const int ldb,
    HIPContext* context,
    TypeMeta::Copy copy) {
  CAFFE_ENFORCE(!copy, "Copy constructor is not supported in CUDA context");
  hipMemcpy2DAsync(
      B,
      ldb * itemsize,
      A,
      lda * itemsize,
      N * itemsize,
      M,
      hipMemcpyDeviceToDevice,
      context->hip_stream());
}

#define CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(T) \
  template <>                                  \
  void CopyMatrix<T, HIPContext>(             \
      const int M,                             \
      const int N,                             \
      const T* A,                              \
      const int lda,                           \
      T* B,                                    \
      const int ldb,                           \
      HIPContext* context) {                  \
    if (M == 0 || N == 0) {                    \
      return;                                  \
    }                                          \
    hipMemcpy2DAsync(                         \
        B,                                     \
        sizeof(T) * ldb,                       \
        A,                                     \
        sizeof(T) * lda,                       \
        sizeof(T) * N,                         \
        M,                                     \
        hipMemcpyDeviceToDevice,              \
        context->hip_stream());               \
  }
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(float)
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(double)
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(int)
CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX(int64_t)
#undef CAFFE2_SPECIALIZED_CUDA_COPY_MATRIX

template <>
CAFFE2_CUDA_EXPORT void CopyVector<float, HIPContext>(
    const int N,
    const float* src,
    float* dst,
    HIPContext* context) {
  if (src != dst && N > 0) {
    hipMemcpyAsync(
        dst,
        src,
        sizeof(float) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
}

template <>
CAFFE2_CUDA_EXPORT void CopyVector<int, HIPContext>(
    const int N,
    const int* src,
    int* dst,
    HIPContext* context) {
  if (src != dst && N > 0) {
    hipMemcpyAsync(
        dst,
        src,
        sizeof(int) * N,
        hipMemcpyDeviceToDevice,
        context->hip_stream());
  }
}

namespace {

template <typename T>
using BlockReduce = hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS>;

template <typename T, class Reducer>
__global__ void RowwiseReduceKernel(
    const int rows,
    const int cols,
    const Reducer reducer,
    const T init,
    const T alpha,
    const T* X,
    T* Y) {
  __shared__ typename BlockReduce<T>::TempStorage temp_storage;
  for (int i = blockIdx.x; i < rows; i += gridDim.x) {
    T val = init;
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
      val = reducer(X[i * cols + j], val);
    }
    val = BlockReduce<T>(temp_storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      Y[i] = val * alpha;
    }
    __syncthreads();
  }
}

template <typename T, class Reducer>
__global__ void ColwiseReduceKernel(
    const int rows,
    const int cols,
    const Reducer reducer,
    const T init,
    const T alpha,
    const T* X,
    T* Y) {
  __shared__ typename BlockReduce<T>::TempStorage temp_storage;
  for (int i = blockIdx.x; i < cols; i += gridDim.x) {
    T val = init;
    for (int j = threadIdx.x; j < rows; j += blockDim.x) {
      val = reducer(X[j * cols + i], val);
    }
    val = BlockReduce<T>(temp_storage).Reduce(val, reducer);
    if (threadIdx.x == 0) {
      Y[i] = val * alpha;
    }
    __syncthreads();
  }
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_ROWWISE_MAX(T)                            \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void RowwiseMax<T, HIPContext>(                     \
      const int N, const int D, const T* x, T* y, HIPContext* context) { \
    RowwiseReduceKernel<<<                                                \
        std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),                            \
        CAFFE_CUDA_NUM_THREADS,                                           \
        0,                                                                \
        context->hip_stream()>>>(                                        \
        N, D, hipcub::Max(), std::numeric_limits<T>::lowest(), T(1), x, y);  \
  }
CAFFE2_SPECIALIZED_CUDA_ROWWISE_MAX(float)
#undef CAFFE2_SPECIALIZED_CUDA_ROWWISE_MAX

#define CAFFE2_SPECIALIZED_CUDA_COLWISE_MAX(T)                            \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void ColwiseMax<T, HIPContext>(                     \
      const int N, const int D, const T* x, T* y, HIPContext* context) { \
    ColwiseReduceKernel<<<                                                \
        std::min(D, CAFFE_MAXIMUM_NUM_BLOCKS),                            \
        CAFFE_CUDA_NUM_THREADS,                                           \
        0,                                                                \
        context->hip_stream()>>>(                                        \
        N, D, hipcub::Max(), std::numeric_limits<T>::lowest(), T(1), x, y);  \
  }
CAFFE2_SPECIALIZED_CUDA_COLWISE_MAX(float)
#undef CAFFE2_SPECIALIZED_CUDA_COLWISE_MAX

namespace {
__global__ void
maximum_kernel(const int N, const float alpha, const float* x, float* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = fmaxf(x[i], alpha);
  }
}
} // namespace

template <>
CAFFE2_CUDA_EXPORT void Maximum(
    const int N,
    const float alpha,
    const float* x,
    float* y,
    HIPContext* context) {
  maximum_kernel<<<
      std::min(N, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, alpha, x, y);
}

namespace {

template <typename T, int D>
__global__ void BroadcastCUDAKernel(
    const int Y_size,
    const SimpleArray<int, D> X_strides,
    const SimpleArray<FIXED_DIVISOR, D> Y_dims,
    const T alpha,
    const T* X,
    T* Y) {
  CUDA_1D_KERNEL_LOOP(Y_index, Y_size) {
    int X_index = 0;
    int Y_index_val = Y_index;
#pragma unroll
    for (int i = D - 1; i >= 0; --i) {
      int d;
      FIXED_DIVISOR_DIV_MOD(Y_dims.data[i], Y_index_val, &Y_index_val, &d);
      X_index += d * X_strides.data[i];
    }
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
    Y[Y_index] = __ldg(X + X_index) * alpha;
#else
    Y[Y_index] = X[X_index] * alpha;
#endif
  }
}

template <typename T, int D>
CAFFE2_CUDA_EXPORT void BroadcastCUDAImpl(
    const int X_ndim,
    const int* X_dims,
    const int* Y_dims,
    const T alpha,
    const T* X,
    T* Y,
    HIPContext* context) {
  SimpleArray<int, D> X_strides_array;
  SimpleArray<FIXED_DIVISOR, D> Y_dims_array;
  const int d = D - X_ndim;
  std::fill(X_strides_array.data, X_strides_array.data + d, 0);
  int cur_stride = 1;
  for (int i = D - 1; i >= d; --i) {
    CAFFE_ENFORCE(X_dims[i - d] == 1 || X_dims[i - d] == Y_dims[i]);
    X_strides_array.data[i] = X_dims[i - d] == 1 ? 0 : cur_stride;
    cur_stride *= X_dims[i - d];
  }
  for (int i = 0; i < D; ++i) {
    if (Y_dims[i] == 0) {
      return;
    }
    Y_dims_array.data[i] = FIXED_DIVISOR(Y_dims[i]);
  }
  const int Y_size =
      std::accumulate(Y_dims, Y_dims + D, 1, std::multiplies<int>());
  BroadcastCUDAKernel<T, D>
      <<<CAFFE_GET_BLOCKS(Y_size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          Y_size, X_strides_array, Y_dims_array, alpha, X, Y);
}

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_BROADCAST(T)         \
  template <>                                        \
  CAFFE2_CUDA_EXPORT void Broadcast<T, HIPContext>( \
      const int X_ndim,                              \
      const int* X_dims,                             \
      const int Y_ndim,                              \
      const int* Y_dims,                             \
      const T alpha,                                 \
      const T* X,                                    \
      T* Y,                                          \
      HIPContext* context) {                        \
    CAFFE_ENFORCE_LE(X_ndim, Y_ndim);                \
    DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_1(          \
        Y_ndim,                                      \
        BroadcastCUDAImpl,                           \
        T,                                           \
        X_ndim,                                      \
        X_dims,                                      \
        Y_dims,                                      \
        alpha,                                       \
        X,                                           \
        Y,                                           \
        context);                                    \
  }
CAFFE2_SPECIALIZED_CUDA_BROADCAST(std::int32_t)
CAFFE2_SPECIALIZED_CUDA_BROADCAST(std::int64_t)
CAFFE2_SPECIALIZED_CUDA_BROADCAST(float)
CAFFE2_SPECIALIZED_CUDA_BROADCAST(double)
#undef CAFFE2_SPECIALIZED_CUDA_BROADCAST

namespace {

template <typename T>
__global__ void
InvStdCUDAKernel(const int N, const T epsilon, const T* var, T* inv_std);

#define DELEGATE_INV_STD_KERNEL_FUNCTION(T, Func)               \
  template <>                                                   \
  __global__ void InvStdCUDAKernel<T>(                          \
      const int N, const T epsilon, const T* var, T* inv_std) { \
    CUDA_1D_KERNEL_LOOP(i, N) {                                 \
      inv_std[i] = Func(var[i] + epsilon);                      \
    }                                                           \
  }
DELEGATE_INV_STD_KERNEL_FUNCTION(float, rsqrtf)
#undef DELEGATE_INV_STD_KERNEL_FUNCTION

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_INV_STD(T)                      \
  template <>                                                   \
  CAFFE2_CUDA_EXPORT void InvStd<T, HIPContext>(               \
      const int N,                                              \
      const T epsilon,                                          \
      const T* var,                                             \
      T* inv_std,                                               \
      HIPContext* context) {                                   \
    InvStdCUDAKernel<T>                                         \
        <<<CAFFE_GET_BLOCKS(N),                                 \
           CAFFE_CUDA_NUM_THREADS,                              \
           0,                                                   \
           context->hip_stream()>>>(N, epsilon, var, inv_std); \
  }
CAFFE2_SPECIALIZED_CUDA_INV_STD(float)
#undef CAFFE2_SPECIALIZED_CUDA_INV_STD

} // namespace math
} // namespace caffe2
