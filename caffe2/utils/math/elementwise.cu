#include "hip/hip_runtime.h"
#include "caffe2/utils/math/elementwise.h"

#include <type_traits>

#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/conversions.h"
#include "caffe2/utils/math/half_utils.h"
#include "caffe2/utils/math/utils.h"

namespace caffe2 {
namespace math {

namespace {

template <typename T>
__global__ void SinCosCUDAKernel(const int N, const T* X, T* S, T* C) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
    c10::cuda::compat::sincos(__ldg(X + i), S + i, C + i);
#else
    c10::cuda::compat::sincos(X[i], S + i, C + i);
#endif
  }
}

#ifdef __HIP_PLATFORM_HCC__

template <typename TAlpha, typename TData>
__global__ void AxpyCUDAKernel(
    const std::int64_t N,
    const TAlpha alpha,
    const TData* X,
    TData* Y) {
  const int64_t index = static_cast<int64_t>(blockIdx.x) *
          static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +
      static_cast<int64_t>(threadIdx.x);
  if (index < N) {
    Y[index] += static_cast<TData>(alpha) * __ldg(X + index);
  }
}

template <typename TAlpha, typename TData>
__global__ void AxpyCUDAKernel(
    const std::int64_t N,
    const TAlpha* alpha,
    const TData* X,
    TData* Y) {
  __shared__ TData a;
  if (threadIdx.x == 0) {
    a = static_cast<TData>(__ldg(alpha));
  }
  __syncthreads();
  const int64_t index = static_cast<int64_t>(blockIdx.x) *
          static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +
      static_cast<int64_t>(threadIdx.x);
  if (index < N) {
    Y[index] += a * __ldg(X + index);
  }
}

#define DELEGATE_HALF_AXPY_CUDA_KERNEL(TAlpha, FMAFunc)                \
  template <>                                                          \
  __global__ void AxpyCUDAKernel<TAlpha, at::Half>(                    \
      const std::int64_t N,                                            \
      const TAlpha alpha,                                              \
      const at::Half* X,                                               \
      at::Half* Y) {                                                   \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *           \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +             \
        static_cast<int64_t>(threadIdx.x);                             \
    if (index < N) {                                                   \
      Y[index] = convert::To<TAlpha, at::Half>(FMAFunc(                \
          alpha,                                                       \
          convert::To<at::Half, TAlpha>(X[index]),                     \
          convert::To<at::Half, TAlpha>(Y[index])));                   \
    }                                                                  \
  }                                                                    \
  template <>                                                          \
  __global__ void AxpyCUDAKernel<TAlpha, at::Half>(                    \
      const std::int64_t N,                                            \
      const TAlpha* alpha,                                             \
      const at::Half* X,                                               \
      at::Half* Y) {                                                   \
    __shared__ TAlpha a;                                               \
    if (threadIdx.x == 0) {                                            \
      a = __ldg(alpha);                                                \
    }                                                                  \
    __syncthreads();                                                   \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *           \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +             \
        static_cast<int64_t>(threadIdx.x);                             \
    if (index < N) {                                                   \
      Y[index] = convert::To<TAlpha, at::Half>(FMAFunc(                \
          a,                                                           \
          convert::To<at::Half, TAlpha>(X[index]),                     \
          convert::To<at::Half, TAlpha>(Y[index])));                   \
    }                                                                  \
  }
DELEGATE_HALF_AXPY_CUDA_KERNEL(float, fmaf)
#undef DELEGATE_HALF_AXPY_CUDA_KERNEL

#endif // __HIP_PLATFORM_HCC__

template <typename TAlpha, typename TData>
__global__ void AxpbyCUDAKernel(
    const std::int64_t N,
    const TAlpha alpha,
    const TData* X,
    const TAlpha beta,
    TData* Y);

template <typename TAlpha, typename TData>
__global__ void AxpbyCUDAKernel(
    const std::int64_t N,
    const TAlpha* alpha,
    const TData* X,
    const TAlpha* beta,
    TData* Y);

#define DELEGATE_AXPBY_CUDA_KERNEL(TAlpha, TData, FMAFunc)             \
  template <>                                                          \
  __global__ void AxpbyCUDAKernel<TAlpha, TData>(                      \
      const std::int64_t N,                                            \
      const TAlpha alpha,                                              \
      const TData* X,                                                  \
      const TAlpha beta,                                               \
      TData* Y) {                                                      \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *           \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +             \
        static_cast<int64_t>(threadIdx.x);                             \
    if (index < N) {                                                   \
      Y[index] = FMAFunc(                                              \
          static_cast<TData>(alpha),                                   \
          X[index],                                                    \
          static_cast<TData>(beta) * Y[index]);                        \
    }                                                                  \
  }                                                                    \
  template <>                                                          \
  __global__ void AxpbyCUDAKernel<TAlpha, TData>(                      \
      const std::int64_t N,                                            \
      const TAlpha* alpha,                                             \
      const TData* X,                                                  \
      const TAlpha* beta,                                              \
      TData* Y) {                                                      \
    __shared__ TData a;                                                \
    __shared__ TData b;                                                \
    if (threadIdx.x == 0) {                                            \
      a = static_cast<TData>(*alpha);                                  \
      b = static_cast<TData>(*beta);                                   \
    }                                                                  \
    __syncthreads();                                                   \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *           \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +             \
        static_cast<int64_t>(threadIdx.x);                             \
    if (index < N) {                                                   \
      Y[index] = FMAFunc(a, X[index], b * Y[index]);                   \
    }                                                                  \
  }
DELEGATE_AXPBY_CUDA_KERNEL(float, float, fmaf)
DELEGATE_AXPBY_CUDA_KERNEL(float, double, fma)
#undef DELEGATE_AXPBY_CUDA_KERNEL

#define DELEGATE_HALF_AXPBY_CUDA_KERNEL(TAlpha, FMAFunc)               \
  template <>                                                          \
  __global__ void AxpbyCUDAKernel<TAlpha, at::Half>(                   \
      const std::int64_t N,                                            \
      const TAlpha alpha,                                              \
      const at::Half* X,                                               \
      const TAlpha beta,                                               \
      at::Half* Y) {                                                   \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *           \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +             \
        static_cast<int64_t>(threadIdx.x);                             \
    if (index < N) {                                                   \
      Y[index] = convert::To<TAlpha, at::Half>(FMAFunc(                \
          alpha,                                                       \
          convert::To<at::Half, TAlpha>(X[index]),                     \
          beta * convert::To<at::Half, TAlpha>(Y[index])));            \
    }                                                                  \
  }                                                                    \
  template <>                                                          \
  __global__ void AxpbyCUDAKernel<TAlpha, at::Half>(                   \
      const std::int64_t N,                                            \
      const TAlpha* alpha,                                             \
      const at::Half* X,                                               \
      const TAlpha* beta,                                              \
      at::Half* Y) {                                                   \
    __shared__ TAlpha a;                                               \
    __shared__ TAlpha b;                                               \
    if (threadIdx.x == 0) {                                            \
      a = *alpha;                                                      \
      b = *beta;                                                       \
    }                                                                  \
    __syncthreads();                                                   \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *           \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +             \
        static_cast<int64_t>(threadIdx.x);                             \
    if (index < N) {                                                   \
      Y[index] = convert::To<TAlpha, at::Half>(FMAFunc(                \
          a,                                                           \
          convert::To<at::Half, TAlpha>(X[index]),                     \
          b * convert::To<at::Half, TAlpha>(Y[index])));               \
    }                                                                  \
  }
DELEGATE_HALF_AXPBY_CUDA_KERNEL(float, fmaf)
#undef DELEGATE_HALF_AXPBY_CUDA_KERNEL

template <typename TAlpha, typename TData>
__global__ void ScaleCUDAKernel(
    const std::int64_t N,
    const TAlpha alpha,
    const TData* X,
    TData* Y);

template <typename TAlpha, typename TData>
__global__ void ScaleCUDAKernel(
    const std::int64_t N,
    const TAlpha* alpha,
    const TData* X,
    TData* Y);

#define CAFFE2_SPECIALIZED_SCALE_CUDA_KERNEL(TAlpha, TData)                  \
  template <>                                                                \
  __global__ void ScaleCUDAKernel<TAlpha, TData>(                            \
      const std::int64_t N, const TAlpha alpha, const TData* X, TData* Y) {  \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *                 \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +                   \
        static_cast<int64_t>(threadIdx.x);                                   \
    if (index < N) {                                                         \
      Y[index] = static_cast<TData>(alpha) * X[index];                       \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  __global__ void ScaleCUDAKernel<TAlpha, TData>(                            \
      const std::int64_t N, const TAlpha* alpha, const TData* X, TData* Y) { \
    __shared__ TData a;                                                      \
    if (threadIdx.x == 0) {                                                  \
      a = static_cast<TData>(*alpha);                                        \
    }                                                                        \
    __syncthreads();                                                         \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *                 \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +                   \
        static_cast<int64_t>(threadIdx.x);                                   \
    if (index < N) {                                                         \
      Y[index] = a * X[index];                                               \
    }                                                                        \
  }
CAFFE2_SPECIALIZED_SCALE_CUDA_KERNEL(float, float)
CAFFE2_SPECIALIZED_SCALE_CUDA_KERNEL(double, double)
CAFFE2_SPECIALIZED_SCALE_CUDA_KERNEL(float, double)
CAFFE2_SPECIALIZED_SCALE_CUDA_KERNEL(std::int32_t, std::int32_t)
CAFFE2_SPECIALIZED_SCALE_CUDA_KERNEL(std::int64_t, std::int64_t)
#undef CAFFE2_SPECIALIZED_SCALE_CUDA_KERNEL

#define CAFFE2_SPECIALIZED_HALF_SCALE_CUDA_KERNEL(TAlpha)              \
  template <>                                                          \
  __global__ void ScaleCUDAKernel<TAlpha, at::Half>(                   \
      const std::int64_t N,                                            \
      const TAlpha alpha,                                              \
      const at::Half* X,                                               \
      at::Half* Y) {                                                   \
    const int64_t index = static_cast<int64_t>(blockIdx.x) *           \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +             \
        static_cast<int64_t>(threadIdx.x);                             \
    if (index < N) {                                                   \
      Y[index] = convert::To<TAlpha, at::Half>(                        \
          alpha * convert::To<at::Half, TAlpha>(X[index]));            \
    }                                                                  \
  }                                                                    \
  template <>                                                          \
  __global__ void ScaleCUDAKernel<TAlpha, at::Half>(                   \
      const std::int64_t N,                                            \
      const TAlpha* alpha,                                             \
      const at::Half* X,                                               \
      at::Half* Y) {                                                   \
    __shared__ TAlpha a;                                               \
    if (threadIdx.x == 0) {                                            \
      a = *alpha;                                                      \
    }                                                                  \
    __syncthreads();                                                   \
    const int64_t index = static_cast<int64_t>(blockIdx.x) * \
            static_cast<int64_t>(CAFFE_CUDA_NUM_THREADS) +        \
        static_cast<int64_t>(threadIdx.x);                        \
    if (index < N) {                                                   \
      Y[index] = convert::To<TAlpha, at::Half>(                        \
          a * convert::To<at::Half, TAlpha>(X[index]));                \
    }                                                                  \
  }
CAFFE2_SPECIALIZED_HALF_SCALE_CUDA_KERNEL(float)
#undef CAFFE2_SPECIALIZED_HALF_SCALE_CUDA_KERNEL

} // namespace

#define CAFFE2_SPECIALIZED_CUDA_SET(T)                                    \
  template <>                                                             \
  CAFFE2_CUDA_EXPORT void Set<T, HIPContext>(                            \
      const std::int64_t N, const T alpha, T* Y, HIPContext* context) {  \
    if (N == 0) {                                                         \
      return;                                                             \
    }                                                                     \
    if (alpha == T(0)) {                                                  \
      hipMemsetAsync(Y, 0, sizeof(T) * N, context->hip_stream());       \
    } else {                                                              \
      thrust::fill(                                                       \
          thrust::cuda::par.on(context->hip_stream()), Y, Y + N, alpha); \
    }                                                                     \
  }
CAFFE2_SPECIALIZED_CUDA_SET(bool)
CAFFE2_SPECIALIZED_CUDA_SET(char)
CAFFE2_SPECIALIZED_CUDA_SET(std::int8_t)
CAFFE2_SPECIALIZED_CUDA_SET(std::int16_t)
CAFFE2_SPECIALIZED_CUDA_SET(std::int32_t)
CAFFE2_SPECIALIZED_CUDA_SET(std::int64_t)
CAFFE2_SPECIALIZED_CUDA_SET(std::uint8_t)
CAFFE2_SPECIALIZED_CUDA_SET(std::uint16_t)
CAFFE2_SPECIALIZED_CUDA_SET(float)
CAFFE2_SPECIALIZED_CUDA_SET(double)
CAFFE2_SPECIALIZED_CUDA_SET(at::Half)
CAFFE2_SPECIALIZED_CUDA_SET(at::BFloat16)
#undef CAFFE2_SPECIALIZED_CUDA_SET

#define DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(T, Func, DeviceFunc) \
  template <>                                                    \
  CAFFE2_CUDA_EXPORT void Func<T, HIPContext>(                  \
      const int N, const T* X, T* Y, HIPContext* context) {     \
    if (N > 0) {                                                 \
      thrust::transform(                                         \
          thrust::cuda::par.on(context->hip_stream()),          \
          X,                                                     \
          X + N,                                                 \
          Y,                                                     \
          [] __device__(const T x) { return DeviceFunc(x); });   \
    }                                                            \
  }
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Exp, expf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Log, logf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Log1p, log1pf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sin, sinf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Asin, asinf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cos, cosf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Acos, acosf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Tan, tanf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Atan, atanf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sinh, sinhf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cosh, coshf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Tanh, tanhf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Abs, fabsf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Inv, utils::Inv<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Inv, utils::Inv<double>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqr, utils::Square<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqrt, sqrtf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Rsqrt, rsqrtf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int32_t,
    Cube,
    utils::Cube<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int64_t,
    Cube,
    utils::Cube<std::int64_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cube, utils::Cube<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Cube, utils::Cube<double>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cbrt, cbrtf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Erf, erff)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Erf, erf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, CdfNorm, normcdff)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, CdfNorm, normcdf)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(bool, Not, utils::Not<bool>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int32_t,
    Neg,
    utils::Negate<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int64_t,
    Neg,
    utils::Negate<std::int64_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Neg, utils::Negate<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Neg, utils::Negate<double>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int32_t,
    Sign,
    utils::Sign<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(
    std::int64_t,
    Sign,
    utils::Sign<std::int64_t>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sign, utils::Sign<float>)
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Sign, utils::Sign<double>)
#undef DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION

#define DELEGATE_CUDA_POWX(T, DeviceFunc)                               \
  template <>                                                           \
  CAFFE2_CUDA_EXPORT void Powx<T, HIPContext>(                         \
      const int N, const T* A, const T b, T* Y, HIPContext* context) { \
    thrust::transform(                                                  \
        thrust::cuda::par.on(context->hip_stream()),                   \
        A,                                                              \
        A + N,                                                          \
        Y,                                                              \
        [b] __device__(const T x) { return DeviceFunc(x, b); });        \
  }
DELEGATE_CUDA_POWX(float, powf)
#undef DELEGATE_CUDA_POWX

#define CAFFE2_SPECIALIZED_CUDA_SINCOS(T)                             \
  template <>                                                         \
  CAFFE2_CUDA_EXPORT void SinCos<T, HIPContext>(                     \
      const int N, const T* X, T* S, T* C, HIPContext* context) {    \
    if (N > 0) {                                                      \
      const int K = DivUp(N, CAFFE_CUDA_NUM_THREADS);                 \
      SinCosCUDAKernel<T>                                             \
          <<<K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>( \
              N, X, S, C);                                            \
    }                                                                 \
  }
CAFFE2_SPECIALIZED_CUDA_SINCOS(float)
CAFFE2_SPECIALIZED_CUDA_SINCOS(double)
#undef CAFFE2_SPECIALIZED_CUDA_SINCOS

#define DELEGATE_CUDA_SCALE(T, CuBLASFunc)                                   \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<T, T, HIPContext>(                          \
      const std::int64_t N,                                                  \
      const T alpha,                                                         \
      const T* X,                                                            \
      T* Y,                                                                  \
      HIPContext* context) {                                                \
    if (N == 0) {                                                            \
      return;                                                                \
    }                                                                        \
    if (Y == X) {                                                            \
      CUBLAS_ENFORCE(hipblasSetPointerMode(                                   \
          context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));              \
      CUBLAS_ENFORCE(CuBLASFunc(context->cublas_handle(), N, &alpha, Y, 1)); \
    } else {                                                                 \
      const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
      ScaleCUDAKernel<T, T>                                                  \
          <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
              N, alpha, X, Y);                                               \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<T, T, HIPContext>(                          \
      const std::int64_t N,                                                  \
      const T* alpha,                                                        \
      const T* X,                                                            \
      T* Y,                                                                  \
      HIPContext* context) {                                                \
    if (N == 0) {                                                            \
      return;                                                                \
    }                                                                        \
    if (Y == X) {                                                            \
      CUBLAS_ENFORCE(hipblasSetPointerMode(                                   \
          context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));            \
      CUBLAS_ENFORCE(CuBLASFunc(context->cublas_handle(), N, alpha, Y, 1));  \
    } else {                                                                 \
      const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
      ScaleCUDAKernel<T, T>                                                  \
          <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
              N, alpha, X, Y);                                               \
    }                                                                        \
  }
DELEGATE_CUDA_SCALE(float, hipblasSscal)
DELEGATE_CUDA_SCALE(double, hipblasDscal)
#undef DELEGATE_CUDA_SCALE

#ifndef __HIP_PLATFORM_HCC__

#define DELEGATE_CUDA_SCALE_EX(                                              \
    TAlpha, TData, kAlphaType, kDataType, kExecutionType)                    \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>(                 \
      const std::int64_t N,                                                  \
      const TAlpha alpha,                                                    \
      const TData* X,                                                        \
      TData* Y,                                                              \
      HIPContext* context) {                                                \
    if (N == 0) {                                                            \
      return;                                                                \
    }                                                                        \
    if (Y == X) {                                                            \
      CUBLAS_ENFORCE(hipblasSetPointerMode(                                   \
          context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));              \
      CUBLAS_ENFORCE(hipblasScalEx(                                           \
          context->cublas_handle(),                                          \
          N,                                                                 \
          &alpha,                                                            \
          kAlphaType,                                                        \
          Y,                                                                 \
          kDataType,                                                         \
          1,                                                                 \
          kExecutionType));                                                  \
    } else {                                                                 \
      const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
      ScaleCUDAKernel<TAlpha, TData>                                         \
          <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
              N, alpha, X, Y);                                               \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>(                 \
      const std::int64_t N,                                                  \
      const TAlpha* alpha,                                                   \
      const TData* X,                                                        \
      TData* Y,                                                              \
      HIPContext* context) {                                                \
    if (N == 0) {                                                            \
      return;                                                                \
    }                                                                        \
    if (Y == X) {                                                            \
      CUBLAS_ENFORCE(hipblasSetPointerMode(                                   \
          context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));            \
      CUBLAS_ENFORCE(hipblasScalEx(                                           \
          context->cublas_handle(),                                          \
          N,                                                                 \
          alpha,                                                             \
          kAlphaType,                                                        \
          Y,                                                                 \
          kDataType,                                                         \
          1,                                                                 \
          kExecutionType));                                                  \
    } else {                                                                 \
      const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
      ScaleCUDAKernel<TAlpha, TData>                                         \
          <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
              N, alpha, X, Y);                                               \
    }                                                                        \
  }
DELEGATE_CUDA_SCALE_EX(float, double, HIP_R_32F, HIP_R_64F, HIP_R_64F)
DELEGATE_CUDA_SCALE_EX(float, at::Half, HIP_R_32F, HIP_R_16F, HIP_R_32F)
#undef DELEGATE_CUDA_SCALE_EX

#endif // __HIP_PLATFORM_HCC__

#define CAFFE2_SPECIALIZED_CUDA_SCALE(TAlpha, TData)                         \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>(                 \
      const std::int64_t N,                                                  \
      const TAlpha alpha,                                                    \
      const TData* X,                                                        \
      TData* Y,                                                              \
      HIPContext* context) {                                                \
    if (N > 0) {                                                             \
      const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
      ScaleCUDAKernel<TAlpha, TData>                                         \
          <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
              N, alpha, X, Y);                                               \
    }                                                                        \
  }                                                                          \
  template <>                                                                \
  CAFFE2_CUDA_EXPORT void Scale<TAlpha, TData, HIPContext>(                 \
      const std::int64_t N,                                                  \
      const TAlpha* alpha,                                                   \
      const TData* X,                                                        \
      TData* Y,                                                              \
      HIPContext* context) {                                                \
    if (N > 0) {                                                             \
      const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
      ScaleCUDAKernel<TAlpha, TData>                                         \
          <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
              N, *alpha, X, Y);                                              \
    }                                                                        \
  }
CAFFE2_SPECIALIZED_CUDA_SCALE(std::int32_t, std::int32_t)
CAFFE2_SPECIALIZED_CUDA_SCALE(std::int64_t, std::int64_t)

#ifdef __HIP_PLATFORM_HCC__
CAFFE2_SPECIALIZED_CUDA_SCALE(float, double)
CAFFE2_SPECIALIZED_CUDA_SCALE(float, at::Half)
#endif // __HIP_PLATFORM_HCC__
#undef CAFFE2_SPECIALIZED_CUDA_SCALE

#define DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(T, Func, DeviceFunc)        \
  template <>                                                            \
  CAFFE2_CUDA_EXPORT void Func<T, HIPContext>(                          \
      const int N, const T* A, const T* B, T* C, HIPContext* context) { \
    if (N > 0) {                                                         \
      thrust::transform(                                                 \
          thrust::cuda::par.on(context->hip_stream()),                  \
          A,                                                             \
          A + N,                                                         \
          B,                                                             \
          C,                                                             \
          DeviceFunc);                                                   \
    }                                                                    \
  }
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Add,
    thrust::plus<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Add,
    thrust::plus<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Add, thrust::plus<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Add, thrust::plus<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Add, utils::HalfAddFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Sub,
    thrust::minus<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Sub,
    thrust::minus<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Sub, thrust::minus<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Sub, thrust::minus<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Sub, utils::HalfSubFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Mul,
    thrust::multiplies<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Mul,
    thrust::multiplies<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Mul, thrust::multiplies<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Mul, thrust::multiplies<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Mul, utils::HalfMulFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Div,
    thrust::divides<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Div,
    thrust::divides<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Div, thrust::divides<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Div, thrust::divides<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Div, utils::HalfDivFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Min, thrust::minimum<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Min, thrust::minimum<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Max, thrust::maximum<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Max, thrust::maximum<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, And, thrust::logical_and<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, Or, thrust::logical_or<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, Xor, thrust::bit_xor<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, BitwiseAnd, thrust::bit_and<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    BitwiseAnd,
    thrust::bit_and<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    BitwiseAnd,
    thrust::bit_and<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, BitwiseOr, thrust::bit_or<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    BitwiseOr,
    thrust::bit_or<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    BitwiseOr,
    thrust::bit_or<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, BitwiseXor, thrust::bit_xor<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    BitwiseXor,
    thrust::bit_xor<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    BitwiseXor,
    thrust::bit_xor<std::int64_t>())
#undef DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION

#define DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(T, Func, DeviceComp)          \
  template <>                                                               \
  CAFFE2_CUDA_EXPORT void Func<T, HIPContext>(                             \
      const int N, const T* A, const T* B, bool* C, HIPContext* context) { \
    if (N > 0) {                                                            \
      thrust::transform(                                                    \
          thrust::cuda::par.on(context->hip_stream()),                     \
          A,                                                                \
          A + N,                                                            \
          B,                                                                \
          C,                                                                \
          DeviceComp);                                                      \
    }                                                                       \
  }
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, EQ, thrust::equal_to<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    EQ,
    thrust::equal_to<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    EQ,
    thrust::equal_to<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, EQ, thrust::equal_to<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, EQ, thrust::equal_to<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, NE, thrust::not_equal_to<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    NE,
    thrust::not_equal_to<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    NE,
    thrust::not_equal_to<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, NE, thrust::not_equal_to<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    double,
    NE,
    thrust::not_equal_to<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, LT, thrust::less<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    LT,
    thrust::less<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    LT,
    thrust::less<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, LT, thrust::less<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, LT, thrust::less<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, LE, thrust::less_equal<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    LE,
    thrust::less_equal<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    LE,
    thrust::less_equal<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, LE, thrust::less_equal<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, LE, thrust::less_equal<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, GT, thrust::greater<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    GT,
    thrust::greater<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    GT,
    thrust::greater<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, GT, thrust::greater<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, GT, thrust::greater<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, GE, thrust::greater_equal<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    GE,
    thrust::greater_equal<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    GE,
    thrust::greater_equal<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, GE, thrust::greater_equal<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    double,
    GE,
    thrust::greater_equal<double>())
#undef DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION

#define DELEGATE_CUDA_AXPY(T, CuBLASFunc)                             \
  template <>                                                         \
  CAFFE2_CUDA_EXPORT void Axpy<T, T, HIPContext>(                    \
      const std::int64_t N,                                           \
      const T alpha,                                                  \
      const T* X,                                                     \
      T* Y,                                                           \
      HIPContext* context) {                                         \
    CUBLAS_ENFORCE(hipblasSetPointerMode(                              \
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));         \
    CUBLAS_ENFORCE(                                                   \
        CuBLASFunc(context->cublas_handle(), N, &alpha, X, 1, Y, 1)); \
  }                                                                   \
  template <>                                                         \
  CAFFE2_CUDA_EXPORT void Axpy<T, T, HIPContext>(                    \
      const std::int64_t N,                                           \
      const T* alpha,                                                 \
      const T* X,                                                     \
      T* Y,                                                           \
      HIPContext* context) {                                         \
    CUBLAS_ENFORCE(hipblasSetPointerMode(                              \
        context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE));       \
    CUBLAS_ENFORCE(                                                   \
        hipblasSaxpy(context->cublas_handle(), N, alpha, X, 1, Y, 1)); \
  }
DELEGATE_CUDA_AXPY(float, hipblasSaxpy)
#undef DELEGATE_CUDA_AXPY

#ifndef __HIP_PLATFORM_HCC__

#define DELEGATE_CUDA_AXPY_EX(                                  \
    TAlpha, TData, kAlphaType, kDataType, kExecutionType)       \
  template <>                                                   \
  CAFFE2_CUDA_EXPORT void Axpy<TAlpha, TData, HIPContext>(     \
      const std::int64_t N,                                     \
      const TAlpha alpha,                                       \
      const TData* X,                                           \
      TData* Y,                                                 \
      HIPContext* context) {                                   \
    CUBLAS_ENFORCE(hipblasSetPointerMode(                        \
        context->cublas_handle(), HIPBLAS_POINTER_MODE_HOST));   \
    CUBLAS_ENFORCE(hipblasAxpyEx(                                \
        context->cublas_handle(),                               \
        N,                                                      \
        &alpha,                                                 \
        kAlphaType,                                             \
        X,                                                      \
        kDataType,                                              \
        1,                                                      \
        Y,                                                      \
        kDataType,                                              \
        1,                                                      \
        kExecutionType));                                       \
  }                                                             \
  template <>                                                   \
  CAFFE2_CUDA_EXPORT void Axpy<TAlpha, TData, HIPContext>(     \
      const std::int64_t N,                                     \
      const TAlpha* alpha,                                      \
      const TData* X,                                           \
      TData* Y,                                                 \
      HIPContext* context) {                                   \
    CUBLAS_ENFORCE(hipblasSetPointerMode(                        \
        context->cublas_handle(), HIPBLAS_POINTER_MODE_DEVICE)); \
    CUBLAS_ENFORCE(hipblasAxpyEx(                                \
        context->cublas_handle(),                               \
        N,                                                      \
        alpha,                                                  \
        kAlphaType,                                             \
        X,                                                      \
        kDataType,                                              \
        1,                                                      \
        Y,                                                      \
        kDataType,                                              \
        1,                                                      \
        kExecutionType));                                       \
  }
DELEGATE_CUDA_AXPY_EX(float, double, HIP_R_32F, HIP_R_64F, HIP_R_64F)
DELEGATE_CUDA_AXPY_EX(float, at::Half, HIP_R_32F, HIP_R_16F, HIP_R_32F)
#undef DELEGATE_CUDA_AXPY_EX

#else // __HIP_PLATFORM_HCC__

#define CAFFE2_SPECIALIZED_CUDA_AXPY(TAlpha, TData)                        \
  template <>                                                              \
  CAFFE2_CUDA_EXPORT void Axpy<TAlpha, TData, HIPContext>(                \
      const std::int64_t N,                                                \
      const TAlpha alpha,                                                  \
      const TData* X,                                                      \
      TData* Y,                                                            \
      HIPContext* context) {                                              \
    const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
    AxpyCUDAKernel<TAlpha, TData>                                          \
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
            N, alpha, X, Y);                                               \
  }                                                                        \
  template <>                                                              \
  CAFFE2_CUDA_EXPORT void Axpy<TAlpha, TData, HIPContext>(                \
      const std::int64_t N,                                                \
      const TAlpha* alpha,                                                 \
      const TData* X,                                                      \
      TData* Y,                                                            \
      HIPContext* context) {                                              \
    const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
    AxpyCUDAKernel<TAlpha, TData>                                          \
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
            N, alpha, X, Y);                                               \
  }
CAFFE2_SPECIALIZED_CUDA_AXPY(float, double)
CAFFE2_SPECIALIZED_CUDA_AXPY(float, at::Half)
#undef CAFFE2_SPECIALIZED_CUDA_AXPY

#endif // __HIP_PLATFORM_HCC__

#define CAFFE2_SPECIALIZED_CUDA_AXPBY(TAlpha, TData)                       \
  template <>                                                              \
  CAFFE2_CUDA_EXPORT void Axpby<TAlpha, TData, HIPContext>(               \
      const std::int64_t N,                                                \
      const TAlpha alpha,                                                  \
      const TData* X,                                                      \
      const TAlpha beta,                                                   \
      TData* Y,                                                            \
      HIPContext* context) {                                              \
    const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
    AxpbyCUDAKernel<TAlpha, TData>                                         \
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
            N, alpha, X, beta, Y);                                         \
  }                                                                        \
  template <>                                                              \
  CAFFE2_CUDA_EXPORT void Axpby<TAlpha, TData, HIPContext>(               \
      const std::int64_t N,                                                \
      const TAlpha* alpha,                                                 \
      const TData* X,                                                      \
      const TAlpha* beta,                                                  \
      TData* Y,                                                            \
      HIPContext* context) {                                              \
    const std::int64_t M = DivUp<std::int64_t>(N, CAFFE_CUDA_NUM_THREADS); \
    AxpbyCUDAKernel<TAlpha, TData>                                         \
        <<<M, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(        \
            N, alpha, X, beta, Y);                                         \
  }
CAFFE2_SPECIALIZED_CUDA_AXPBY(float, float)
CAFFE2_SPECIALIZED_CUDA_AXPBY(float, double)
CAFFE2_SPECIALIZED_CUDA_AXPBY(float, at::Half)
#undef CAFFE2_SPECIALIZED_CUDA_AXPBY

} // namespace math
} // namespace caffe2
