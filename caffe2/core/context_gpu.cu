#include "hip/hip_runtime.h"
#include <algorithm>
#include <atomic>
#include <cstdlib>
#include <string>
#include <unordered_map>

#include <ATen/Context.h>
#include <c10/cuda/CUDAFunctions.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include "cub/util_allocator.cuh"

// Needed to be included first to check the CAFFE2_USE_CUDNN macros.
#include "caffe2/core/macros.h"

#include "caffe2/core/blob_stats.h"
#ifdef CAFFE2_USE_CUDNN
#include "caffe2/core/hip/common_miopen.h"
#endif // CAFFE2_USE_CUDNN
#include "caffe2/core/context_gpu.h"
#include "caffe2/core/init.h"
#include "caffe2/core/logging.h"
#include "caffe2/core/tensor.h"
#include "caffe2/utils/string_utils.h"

C10_DEFINE_string(
    caffe2_cuda_memory_pool,
    "",
    "Sets the memory pool used by caffe2. Possible values are "
    "none, cnmem, thc and cub.");

// For description of CUB caching allocator configuration, see
// https://nvlabs.github.io/cub/structcub_1_1_caching_device_allocator.html
C10_DEFINE_int(
    caffe2_cub_bin_growth,
    8,
    "If using cub as the memory allocator, sets the growth of bins "
    "used by the cub pool.");
C10_DEFINE_int(
    caffe2_cub_min_bin,
    3,
    "If using cub as the memory allocator, sets the min number of "
    "bins.");
C10_DEFINE_int(
    caffe2_cub_max_bin,
    10,
    "If using cub as the memory allocator, sets the max number of "
    "bins.");
C10_DEFINE_int(
    caffe2_cub_max_managed_mb,
    10 * 1024,
    "If using cub as the memory allocators, sets the maximum amount "
    "of memory managed in gigabytes");

C10_DEFINE_bool(
    caffe2_cub_print_allocation_events,
    false,
    "If true CachingDeviceAllocator will print allocation and deallocation "
    "events to stdout.");

C10_DEFINE_bool(
    caffe2_gpu_memory_tracking,
    false,
    "If set, logs changes in GPU memory allocations");
C10_DEFINE_int(
    caffe2_gpu_memory_report_interval_mb,
    128,
    "The threshold in MB on how frequently to report memory changes");

namespace at {

REGISTER_CONTEXT(DeviceType::CUDA, caffe2::HIPContext);
} // namespace at

namespace caffe2 {

// Generic implementation - CUDA will handle the right function to call for us
void HIPContext::CopyBytesAsync(
    size_t nbytes,
    const void* src,
    Device src_device,
    void* dst,
    Device dst_device) {
  // TODO: verify that the CUDA handles copy from device to device correctly
  // even without SetDevice()
  // TODO: verify whether source or dest device should be a priority in picking
  // the stream
  // NB: right now the cross-device copy logic is invoked only in the contexts
  // when surrounding code explicitly manages data dependencies and sets up
  // events, so it's fine.  In order to make it a standalone function proper
  // synchronization between stream is required
  int gpu_id = 0;
  if (dst_device.type() == DeviceType::CUDA) {
    gpu_id = dst_device.index();
  } else if (src_device.type() == DeviceType::CUDA) {
    gpu_id = src_device.index();
  } else {
    LOG(FATAL) << "shouldn't be called with non-cuda device";
  }
  CUDA_ENFORCE(hipMemcpyAsync(
      dst,
      src,
      nbytes,
      hipMemcpyDefault,
      HIPContext::getCudaObjects().GetStream(gpu_id)));
}

void HIPContext::CopyBytesSync(
    size_t nbytes,
    const void* src,
    Device src_device,
    void* dst,
    Device dst_device) {
  // This emulates Caffe2 original behavior where sync copy doesn't change the
  // device. It's probably better for clarity to switch to the target device
  // explicitly here, but in the worst case CUDA would sync for us.
  // TODO: change it to CUDAGuard
  HIPContext context(-1); // take current device
  CUDA_ENFORCE(hipMemcpyAsync(
      dst, src, nbytes, hipMemcpyDefault, context.hip_stream()));
  // destructor of context synchronizes
}

// For the CPU context, we also allow a (probably expensive) function
// to copy the data from a cuda context. Inside the function, we create
// a temporary HIPContext object to carry out the copy. From the caller's
// side, these functions are synchronous with respect to the host, similar
// to a normal CPUContext::CopyBytes<CPUContext, CPUContext> call.
template <>
inline void CPUContext::CopyBytes<HIPContext, CPUContext>(
    size_t nbytes,
    const void* src,
    void* dst) {
  HIPContext context(GetGPUIDForPointer(src));
  context.CopyBytes<HIPContext, CPUContext>(nbytes, src, dst);
}
template <>
inline void CPUContext::CopyBytes<CPUContext, HIPContext>(
    size_t nbytes,
    const void* src,
    void* dst) {
  HIPContext context(GetGPUIDForPointer(dst));
  context.CopyBytes<CPUContext, HIPContext>(nbytes, src, dst);
}

} // namespace caffe2

namespace caffe2 {

ThreadLocalCUDAObjects& HIPContext::getCudaObjects() {
  static thread_local ThreadLocalCUDAObjects cuda_objects_;
  return cuda_objects_;
}

// TODO(jiayq): these variables shouldn't be currently accessed during static
// initialization. We should consider moving them to a Mayer's singleton to
// be totally safe against SIOF.

// Static global variables for setting up the memory pool.
CudaMemoryPoolType g_cuda_memory_pool_type;

std::unique_ptr<hipcub::CachingDeviceAllocator> g_cub_allocator;

// an unordered map that holds the map from the cuda memory pointer to the
// device id that it is allocated from. This is used in the cuda memory pool
// cases, where we need the device id to carry out the deletion.
// Note(jiayq): an alternate approach is to use cudaGetPointerAttributes, but
// that is usually quite slow. We might want to benchmark the speed difference
// though.
// Note(jiayq): another alternate approach is to augment the Tensor class that
// would allow one to record the device id. However, this does not address any
// non-tensor allocation and deallocation.
// Ideally, a memory pool should already have the device id information, as
// long as we are using UVA (as of CUDA 5 and later) so the addresses are
// unique.
static std::unordered_map<void*, uint8_t> g_cuda_device_affiliation;

// Data structures for optional memory tracking. Access to these structures
// is guarded by the HIPContext::mutex.
static std::unordered_map<void*, long> g_size_map;
static std::vector<long> g_total_by_gpu_map(C10_COMPILE_TIME_MAX_GPUS, 0);
static std::vector<long> g_max_by_gpu_map(C10_COMPILE_TIME_MAX_GPUS, 0);

static long g_total_mem = 0;
static long g_last_rep = 0;

CudaMemoryPoolType GetCudaMemoryPoolType() {
  return g_cuda_memory_pool_type;
}

///////////////////////////////////////////////////////////////////////////////
// A wrapper to allow us to lazily initialize all cuda environments that Caffe
// uses. This gets done the first time a caffe2::HIPContext::New() gets called
// which is probably the decisive indication that this caffe2 run is going to
// use GPUs. We avoid cuda initialization with core/init.h functionalities so
// that we have minimal resource impact in case we will need to run multiple
// caffe2 instances on a GPU machine.
///////////////////////////////////////////////////////////////////////////////

static void Caffe2InitializeCuda() {
  // If the current run does not have any cuda devices, do nothing.
  if (!HasCudaGPU()) {
    VLOG(1) << "No cuda gpu present. Skipping.";
    return;
  }
  C10_LOG_API_USAGE_ONCE("caffe2.init.cuda");
  // Check if the number of GPUs matches the expected compile-time max number
  // of GPUs.
  CAFFE_ENFORCE_LE(
      NumCudaDevices(),
      C10_COMPILE_TIME_MAX_GPUS,
      "Number of CUDA devices on the machine is larger than the compiled "
      "max number of gpus expected (",
      C10_COMPILE_TIME_MAX_GPUS,
      "). Increase that and recompile.");

  for (DeviceIndex i = 0; i < NumCudaDevices(); ++i) {
    CUDAGuard g(i);
    // Enable peer access.
    const int peer_group = i / CAFFE2_CUDA_MAX_PEER_SIZE;
    const int peer_start = peer_group * CAFFE2_CUDA_MAX_PEER_SIZE;
    const int peer_end = std::min(
        NumCudaDevices(), (peer_group + 1) * CAFFE2_CUDA_MAX_PEER_SIZE);
    VLOG(1) << "Enabling peer access within group #" << peer_group
            << ", from gpuid " << peer_start << " to " << peer_end - 1
            << ", for gpuid " << i << ".";

    for (int j = peer_start; j < peer_end; ++j) {
      if (i == j) continue;
      int can_access;
      CUDA_ENFORCE(hipDeviceCanAccessPeer(&can_access, i, j));
      if (can_access) {
        VLOG(1) << "Enabling peer access from " << i << " to " << j;
        // Note: just for future reference, the 0 here is not a gpu id, it is
        // a reserved flag for hipDeviceEnablePeerAccess that should always be
        // zero currently.
        // It is ok if peer access is already enabled...
        hipError_t err = hipDeviceEnablePeerAccess(j, 0);
        if ((err != hipErrorPeerAccessAlreadyEnabled) &&
            (err != hipSuccess)) {
          CAFFE_THROW(hipGetErrorString(err));
        }
        hipGetLastError(); // reset cuda error code
      }
    }
  }

#ifdef CAFFE2_USE_CUDNN
  // Check the versions of cuDNN that were compiled and linked with are compatible
  CheckCuDNNVersions();
#endif // CAFFE2_USE_CUDNN
}

static void SetUpCub() {
  VLOG(1) << "Setting up cub memory pool.";
  // Sets up the cub memory pool
  try {
    g_cub_allocator.reset(new hipcub::CachingDeviceAllocator(
        FLAGS_caffe2_cub_bin_growth,
        FLAGS_caffe2_cub_min_bin,
        FLAGS_caffe2_cub_max_bin,
        size_t(FLAGS_caffe2_cub_max_managed_mb) * 1024L * 1024L,
        false,
        FLAGS_caffe2_cub_print_allocation_events));
  } catch (...) {
    CAFFE_THROW("Some error happened at cub initialization.");
  }
  VLOG(1) << "Done setting up cub memory pool.";
}

static void Caffe2SetCUDAMemoryPool() {
  if (FLAGS_caffe2_cuda_memory_pool == "" ||
      FLAGS_caffe2_cuda_memory_pool == "none") {
    g_cuda_memory_pool_type = CudaMemoryPoolType::NONE;
  } else if (FLAGS_caffe2_cuda_memory_pool == "cnmem") {
    CAFFE_THROW("CNMEM is no longer used by Caffe2. Use cub instead. "
                "This error message may go away in the future.");
  } else if (FLAGS_caffe2_cuda_memory_pool == "cub") {
    // Sets up cub.
    g_cuda_memory_pool_type = CudaMemoryPoolType::CUB;
    SetUpCub();
  } else if (FLAGS_caffe2_cuda_memory_pool == "thc") {
    g_cuda_memory_pool_type = CudaMemoryPoolType::THC;
    // Initialize caching allocator
    at::globalContext().lazyInitCUDA();
  } else {
    CAFFE_THROW(
        "Unrecognized cuda memory pool type: ", FLAGS_caffe2_cuda_memory_pool);
  }
}

/**
 * An allocator that does the CPU memory allocation with pinned memory.
 *
 * This is needed because if we want to do any asynchronous cuda memcpy,
 * the underlying CPU memory also needs to be allocated into pinned memory
 * space. As a result, whenever Caffe2 is built with GPU and there is
 * GPU present during runtime, at global initialization time we will set
 * the CPU memory allocator to allocate pinned memory.
 *
 * NB: This behavior is probably too aggressive. We should consider asking users
 * to do on-demand memory pinning (like exposed in PyTorch APIs) instead.
 */
struct CAFFE2_CUDA_API PinnedCPUAllocator final : public at::Allocator {
  PinnedCPUAllocator() {
    baseAllocator_ = GetDefaultCPUAllocator();
  }
  ~PinnedCPUAllocator() override {}
  at::DataPtr allocate(size_t nbytes) const override {
    if (nbytes == 0) {
      // replicate c10::alloc_cpu behavior - return nullptr
      return {nullptr, nullptr, &Delete, at::Device(CPU)};
    }
    void* data;
    at::DataPtr data_ptr;
    std::lock_guard<std::mutex> lock(HIPContext::mutex());
    if (IsNUMAEnabled()) {
      at::DeleterFnPtr expected_deleter = baseAllocator_->raw_deleter();
      data_ptr = baseAllocator_->allocate(nbytes);
      data = data_ptr.get();
      CAFFE_ENFORCE(data);
      CUDA_ENFORCE(hipHostRegister(data, nbytes, hipHostRegisterDefault));
      CAFFE_ENFORCE(
          data_ptr.compare_exchange_deleter(expected_deleter, &Delete),
          "Failed to swap deleter (already swapped?)");
    } else {
      CUDA_ENFORCE(hipHostMalloc(&data, nbytes));
      profiledCPUMemoryReporter().New(data, nbytes);
      data_ptr = {data, data, &Delete, at::Device(CPU)};
    }
    memset(data, 0, nbytes);
    return data_ptr;
  }

  at::DeleterFnPtr raw_deleter() const override {
    return &Delete;
  }

 private:
  static void Delete(void* data) {
    if (!data) {
      return;
    }
    // Caffe2 uses a lazy way to figure out if one is actually going to use GPUs
    // or not. If a HIPContext::New() call is made, inside the HIPContext
    // function we will switch the cpu side allocator to a PinnedCPUAllocator.
    // But, if one calls CPUContext::New() before any cuda allocations,
    // PinnedCPUAllocator can still delete the corresponding memory.
    std::lock_guard<std::mutex> lock(HIPContext::mutex());
    if (IsNUMAEnabled()) {
      CUDA_ENFORCE(hipHostUnregister(data));
      GetDefaultCPUAllocator()->raw_deleter()(data);
    } else {
      hipError_t err = hipHostFree(data);
      profiledCPUMemoryReporter().Delete(data);
      if (err == hipErrorInvalidValue) {
        free(data);
        // Calling hipGetLastError will reset the cuda error.
        hipError_t _err = hipGetLastError();
      } else {
        // For all other errors, still do a cuda check.
        CUDA_ENFORCE(err);
      }
    }
  }

  at::Allocator* baseAllocator_;
};

static PinnedCPUAllocator g_pinned_cpu_alloc;

// An initialization function that sets the CPU side to use pinned cpu
// allocator.
void Caffe2UsePinnedCPUAllocator() {
#if C10_ASAN_ENABLED
  // Note(jiayq): for more details, see
  //     https://github.com/google/sanitizers/issues/629
  LOG(WARNING) << "There are known issues between address sanitizer and "
                  "hipHostMalloc. As a result, caffe2 will not enable pinned "
                  "memory allocation in asan mode. If you are expecting any "
                  "behavior that depends on asan, be advised that it is not "
                  "turned on.";
#else
  if (!HasCudaGPU()) {
    VLOG(1) << "No GPU present. I won't use pinned allocator then.";
    return;
  }
  VLOG(1) << "Caffe2 gpu: setting CPUAllocator to PinnedCPUAllocator.";

  // If CUDA is enabled, using CPU allocators other than PinnedCPUAllocator
  // will cause memory corruptions. Therefore, we need to set the priority
  // to highest to avoid being overwritten.
  SetCPUAllocator(
      &g_pinned_cpu_alloc,
      std::numeric_limits<uint8_t>::max() /* priority */);
#endif
}

// Caffe2CudaInitializerHelper is a minimal struct whose sole purpose is to
// detect the first hint that this Caffe2 run is going to use GPU: either
// HIPContext is initialized or HIPContext::New is called. It then runs
// all the related cuda initialization functions.
namespace {
struct Caffe2CudaInitializerHelper {
  Caffe2CudaInitializerHelper() {
    // We cannot use bool because nvcc changes bool to __nv_bool which does
    // not have a std::atomic instantiation.
    static std::atomic<char> first_call(1);
    if (first_call.fetch_and((char)0)) {
      Caffe2InitializeCuda();
      Caffe2SetCUDAMemoryPool();
      Caffe2UsePinnedCPUAllocator();
    }
  }
};
} // namespace

/**
 * A utility function to rectify the gpu id. If the context specifies the
 * gpu id to be -1, it means that we will just use the current gpu id when
 * the function is being called.
 */
static inline DeviceIndex RectifyGPUID(DeviceIndex gpu_id) {
  return gpu_id == -1 ? CaffeCudaGetDevice() : gpu_id;
}

HIPContext::HIPContext(DeviceIndex gpu_id)
    : gpu_id_(RectifyGPUID(gpu_id)), random_seed_(RandomNumberSeed()) {
  static Caffe2CudaInitializerHelper g_cuda_initializer_;
}

HIPContext::HIPContext(const DeviceOption& option)
    : gpu_id_(
          option.has_device_id() ? RectifyGPUID(option.device_id())
                                   : CaffeCudaGetDevice()),
      random_seed_(
          option.has_random_seed() ? option.random_seed()
                                   : RandomNumberSeed()) {
  static Caffe2CudaInitializerHelper g_cuda_initializer_;
  DCHECK_EQ(option.device_type(), PROTO_CUDA);
}

HIPContext::~HIPContext() {
  try {
    if (curand_generator_) {
      CURAND_CHECK(hiprandDestroyGenerator(curand_generator_));
    }
    // HIPContext is used in 2 cases now:
    // - long-lived instance inside OperatorBase in which case what happens in
    //   destructor doesn't really matter
    // - short-lived on-the-fly instances that are utilized as CUDAGuard - in
    //   this case there's only one stream id (passed to SwitchToDevice) and
    //   it's preferrable to synchronize in the destructor
    FinishDeviceComputation();
  } catch (const std::exception& e)  {
    LOG(ERROR) << "Encountered following in " << __FUNCTION__ << ": " << e.what();
  }
}

// shared mutex to lock out alloc / free during NCCL launches
std::mutex& HIPContext::mutex() {
  static std::mutex m;
  return m;
}

std::vector<long> HIPContext::TotalMemoryByGpu() {
  std::lock_guard<std::mutex> lock(HIPContext::mutex());
  CAFFE_ENFORCE(
      FLAGS_caffe2_gpu_memory_tracking,
      "Pass --caffe2_gpu_memory_tracking to enable memory stats");
  return g_total_by_gpu_map;
}

std::vector<long> HIPContext::MaxMemoryByGpu() {
  std::lock_guard<std::mutex> lock(HIPContext::mutex());
  CAFFE_ENFORCE(
      FLAGS_caffe2_gpu_memory_tracking,
      "Pass --caffe2_gpu_memory_tracking to enable memory stats");
  return g_max_by_gpu_map;
}

namespace {
void TrackMemoryAlloc(size_t nbytes) {
  int this_gpu = CaffeCudaGetDevice();
  g_total_by_gpu_map[this_gpu] += nbytes;
  g_max_by_gpu_map[this_gpu] =
      std::max(g_max_by_gpu_map[this_gpu], g_total_by_gpu_map[this_gpu]);
  g_total_mem += nbytes;
  if (g_total_mem - g_last_rep >
      FLAGS_caffe2_gpu_memory_report_interval_mb * 1024 * 1024) {
    for (int gpu = 0; gpu < g_total_by_gpu_map.size(); gpu++) {
      long t = g_total_by_gpu_map[gpu];
      long max_t = g_max_by_gpu_map[gpu];
      if (max_t > 0) {
        if (max_t != t) {
          VLOG(1) << "GPU " << gpu << ": " << t / 1024 / 1024 << " MB"
                  << " (max: " << max_t / 1024 / 1024 << " MB)";
        } else {
          VLOG(1) << "GPU " << gpu << ": " << t / 1024 / 1024 << " MB";
        }
      }
    }
    VLOG(1) << "Total: " << g_total_mem / 1024 / 1024 << " MB";
    g_last_rep = g_total_mem;
  }
}
}

struct DefaultCUDAAllocator final : public at::Allocator {
  DefaultCUDAAllocator() {}
  ~DefaultCUDAAllocator() override {}
  at::DataPtr allocate(size_t nbytes) const override {
    // Lock the mutex
    std::lock_guard<std::mutex> lock(HIPContext::mutex());
    // A one-time caffe2 cuda initializer.
    static Caffe2CudaInitializerHelper g_cuda_initializer_;
    void* ptr = nullptr;

    if (FLAGS_caffe2_gpu_memory_tracking) {
      TrackMemoryAlloc(nbytes);
    }
    switch (g_cuda_memory_pool_type) {
      case CudaMemoryPoolType::NONE:
        if (nbytes != 0) {
          CUDA_ENFORCE(hipMalloc(&ptr, nbytes));
        }
        if (FLAGS_caffe2_gpu_memory_tracking) {
          g_size_map[ptr] = nbytes;
          g_cuda_device_affiliation[ptr] = CaffeCudaGetDevice();
        }
        return {ptr, ptr, &Delete, at::Device(CUDA, CaffeCudaGetDevice())};
      case CudaMemoryPoolType::CUB:
        if (nbytes != 0) {
          CUDA_ENFORCE(g_cub_allocator->DeviceAllocate(&ptr, nbytes));
        }
        g_cuda_device_affiliation[ptr] = CaffeCudaGetDevice();
        VLOG(2) << "CUB allocating pointer " << ptr << " on device "
                << CaffeCudaGetDevice();
        if (FLAGS_caffe2_gpu_memory_tracking) {
          g_size_map[ptr] = nbytes;
        }
        return {ptr, ptr, &Delete, at::Device(CUDA, CaffeCudaGetDevice())};
      case CudaMemoryPoolType::THC:
        {
          // The reason we have this stream guard here is to preserve
          // the historical behavior of the 'thc' allocator in Caffe2,
          // which is to put all allocations on the same (default)
          // stream.  This behavior is morally wrong (since passing
          // allocations between streams allows for the possibility
          // of you handing out some memory that an old stream
          // is still working on), but it doesn't seem to cause issues
          // in Caffe2 today.  Our hypothesis for why this is the case
          // is that Caffe2 doesn't really do very many allocations
          // on the fly; instead they allocate once and then reuse
          // the allocations for the whole program.  In this case,
          // the hazard is avoided.
          //
          // We intend to remove this stream guard, but the benefit
          // to putting all allocations on the same stream is it
          // reduces per-stream fragmentation, and this helps
          // some models that are currently running with the thc
          // allocator fit in memory.  We will need to find some
          // way of resolving this problem.
          cuda::CUDAStreamGuard g(
            Stream(
              Stream::DEFAULT,
              Device(kCUDA, CaffeCudaGetDevice())
            ));
          ptr = cuda::CUDACachingAllocator::raw_alloc(nbytes);
        }
        if (FLAGS_caffe2_gpu_memory_tracking) {
          g_size_map[ptr] = nbytes;
          g_cuda_device_affiliation[ptr] = CaffeCudaGetDevice();
        }
        return {ptr, ptr, &Delete, at::Device(CUDA, CaffeCudaGetDevice())};
    }
    return {nullptr, nullptr, &Delete, at::Device(CUDA, CaffeCudaGetDevice())};
  }

  at::DeleterFnPtr raw_deleter() const override {
    return &Delete;
  }

 private:
  static void Delete(void* ptr) {
    // lock the mutex
    std::lock_guard<std::mutex> lock(HIPContext::mutex());
    if (FLAGS_caffe2_gpu_memory_tracking) {
      auto sz_it = g_size_map.find(ptr);
      DCHECK(sz_it != g_size_map.end());
      auto aff_it = g_cuda_device_affiliation.find(ptr);
      DCHECK(aff_it != g_cuda_device_affiliation.end());
      g_total_mem -= sz_it->second;
      g_total_by_gpu_map[aff_it->second] -= sz_it->second;
      g_size_map.erase(sz_it);
    }

    switch (g_cuda_memory_pool_type) {
      case CudaMemoryPoolType::NONE: {
        // If memory pool is not set up, use simple hipFree.
        hipError_t error = hipFree(ptr);
        // For some reason, in Python runtime we sometimes delete a data pointer
        // after the cuda runtime exits - this is odd but is probably caused by
        // a static workspace that pycaffe2 uses, and the destruction got
        // entangled in some race condition. Anyway, since cuda runtime is
        // exiting anyway, we will not need to worry about memory leak, so we
        // basically ignore it. This is definitely not ideal but works for now.
        if (error != hipSuccess && error != hipErrorDeinitialized) {
          LOG(FATAL) << "Error at: " << __FILE__ << ":" << __LINE__ << ": "
                     << hipGetErrorString(error);
        }

        if (FLAGS_caffe2_gpu_memory_tracking) {
          g_cuda_device_affiliation.erase(g_cuda_device_affiliation.find(ptr));
        }

        break;
      }
      case CudaMemoryPoolType::CUB: {
        auto it = g_cuda_device_affiliation.find(ptr);
        DCHECK(it != g_cuda_device_affiliation.end());
        VLOG(2) << "CUB freeing pointer " << ptr << " on device " << it->second;
        CUDA_ENFORCE(g_cub_allocator->DeviceFree(it->second, ptr));
        g_cuda_device_affiliation.erase(it);
        break;
      }
      case CudaMemoryPoolType::THC: {
        cuda::CUDACachingAllocator::raw_delete(ptr);
        if (FLAGS_caffe2_gpu_memory_tracking) {
          g_cuda_device_affiliation.erase(g_cuda_device_affiliation.find(ptr));
        }
        break;
      }
    }
  }
};

static DefaultCUDAAllocator g_cuda_alloc;
REGISTER_ALLOCATOR(CUDA, &g_cuda_alloc);

} // namespace caffe2

namespace at {
REGISTER_COPY_BYTES_FUNCTION(
    DeviceType::CUDA,
    DeviceType::CUDA,
    caffe2::HIPContext::CopyBytesSync,
    caffe2::HIPContext::CopyBytesAsync);

REGISTER_COPY_BYTES_FUNCTION(
    DeviceType::CUDA,
    DeviceType::CPU,
    caffe2::HIPContext::CopyBytesSync,
    caffe2::HIPContext::CopyBytesAsync);

REGISTER_COPY_BYTES_FUNCTION(
    DeviceType::CPU,
    DeviceType::CUDA,
    caffe2::HIPContext::CopyBytesSync,
    caffe2::HIPContext::CopyBytesAsync);
} // namespace at
