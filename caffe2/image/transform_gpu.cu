#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/image/transform_gpu.h"
#include "caffe2/utils/conversions.h"

/**
 *
 * Copyright (c) 2016, NVIDIA CORPORATION, All rights reserved
 * Distributed under 2-clause BSD license; see accompanying LICENSE file
 *
 **/

namespace caffe2 {

namespace {

// input in (int8, NHWC), output in (fp32, NCHW)
template <typename In, typename Out>
__global__ void transform_kernel(
    const int N,
    const int C,
    const int H,
    const int W,
    const float* mean,
    const float* std,
    const In* in,
    Out* out) {
  const int n = blockIdx.x;

  const int nStride = C*H*W;

  // pointers to data for this image
  const In* input_ptr = &in[n*nStride];
  Out* output_ptr = &out[n*nStride];

  // either read or write uncoalesced - try reading
  for (int c=0; c < C; ++c) {
    for (int h=threadIdx.y; h < H; h += blockDim.y) {
      for (int w=threadIdx.x; w < W; w += blockDim.x) {
        int in_idx = c + C*w + C*W*h;  // HWC
        int out_idx = c*H*W + h*W + w;  // CHW

        output_ptr[out_idx] = convert::To<float,Out>(
          (convert::To<In,float>(input_ptr[in_idx])-mean[c]) * std[c]);
      }
    }
  }
}

}

template <typename T_IN, typename T_OUT, class Context>

bool TransformOnGPU(
    Tensor& X,
    Tensor* Y,
    Tensor& mean,
    Tensor& std,
    Context* context) {
  const int N = X.dim32(0), C = X.dim32(3), H = X.dim32(1), W = X.dim32(2);
  auto* input_data = X.template data<T_IN>();
  auto* output_data = Y->template mutable_data<T_OUT>();

  transform_kernel<
    T_IN, T_OUT><<<N, dim3(16, 16), 0, context->hip_stream()>>>(
      N, C, H, W, mean.template data<float>(), std.template data<float>(),
      input_data, output_data);
  return true;
};

template bool TransformOnGPU<uint8_t, float, HIPContext>(
    Tensor& X,
    Tensor* Y,
    Tensor& mean,
    Tensor& std,
    HIPContext* context);

template bool TransformOnGPU<uint8_t, at::Half, HIPContext>(
    Tensor& X,
    Tensor* Y,
    Tensor& mean,
    Tensor& std,
    HIPContext* context);

}  // namespace caffe2
