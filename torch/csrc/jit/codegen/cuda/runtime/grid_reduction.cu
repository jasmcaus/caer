#include "hip/hip_runtime.h"
// Inter-block reduction.
//
// Function gridReduce performs point-wise reductions of scalars across thread
// blocks. Thread blocks are disjointly partitioned into groups of thread
// blocks, "reduction segments," that are collectively defined by boolean
// template parameters, X_BLOCK, Y_BLOCK and Z_BLOCK. Each of X/Y/Z_BLOCK
// determines whether thread blocks along the dimension should be grouped into
// the same reduction segment. Cross-block reducitons are independently done
// within each segment and generates distinctive results per segment. For
// instance, if all of X/Y/Z_BLOCK are true, reductions will be done across all
// thread blocks since there will be just a single segment consisting of all
// thread blocks. If none of them are true, each thread block will become a
// segment by itself, so no reduction will be performed.
//
// The input scalars to reduce within each segment are a certain subset of
// thread-private scalars provided as part of the gridReduce function
// parameters. Boolean template parameters, X_THREAD, Y_THREAD and Z_THREAD,
// determine which subset of the scalars should be used for inter-block
// reductions. Specifically, all the input scalars of threads along each
// dimension will be used when X/Y/Z_THREAD are true. Otherwise, only the value
// held at offset 0 of each dimension will be used. Thus, for example, if all of
// X/Y/Z_THREAD are true, the scalars of all threads in each block will
// participate in inter-block reductions. If all of them are false, only one
// scalar of the thread at threadIdx.x == threadIdx.y == threadIdx.z == 0 will
// be used. In the code below, we call the subset of threads a "reduction
// block."
//
// Inter-block reductions perform point-wise reductions of scalars of reduction
// blocks within each reduction segment. More specifically, let rb be a
// reduction block and rs be a reduction segment. Let IN(thread_idx, block_idx)
// denote the input scalar of thread at thread_idx and block_idx. The result of
// each reduction segment, OUT(thread_idx, block_idx_out), is defined only for
// each thread_idx in thread block block_idx_out in the segment as follows:
//
//   OUT(thread_idx, block_idx_out) =
//     Reduction of IN(thread_idx, block_idx) for
//       all block_idx in a reduction segment
//
// OUT is not given for all threads that are not in block_idx_out and the
// reduction block.
//
// See also the function comment of gridReduce.

namespace reduction {

// Utility functions
template <typename _dim3>
__device__ __forceinline__ size_t size(const _dim3& d) {
  return (size_t)d.x * (size_t)d.y * (size_t)d.z;
}

#define isize(d) d.x* d.y* d.z

template <typename _dim3pos, typename _dim3dim>
__device__ __forceinline__ size_t
offset(const _dim3pos& pos, const _dim3dim& dim) {
  return (size_t)pos.x + (size_t)pos.y * (size_t)dim.x +
      (size_t)pos.z * (size_t)dim.x * (size_t)dim.y;
}

#define ioffset(pos, dim) pos.x + pos.y* dim.x + pos.z* dim.x* dim.y

// Returns dim3 of each reduction segment.
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, typename _dim3>
__device__ dim3 dimension_of_reduction_segment(const _dim3& grid_dim) {
  return dim3{X_BLOCK ? grid_dim.x : 1,
              Y_BLOCK ? grid_dim.y : 1,
              Z_BLOCK ? grid_dim.z : 1};
}

// Returns the number of blocks in each reduction segment.
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, typename _dim3>
__device__ size_t size_of_reduction_segment(const _dim3& grid_dim) {
  return size(
      dimension_of_reduction_segment<X_BLOCK, Y_BLOCK, Z_BLOCK>(grid_dim));
}

// Returns the total number of reduction segments.
template <bool X_BLOCK, bool Y_BLOCK, bool Z_BLOCK, typename _dim3>
__device__ size_t number_of_reduction_segments(const _dim3& grid_dim) {
  return (X_BLOCK ? 1 : grid_dim.x) * (Y_BLOCK ? 1 : grid_dim.y) *
      (Z_BLOCK ? 1 : grid_dim.z);
}

// Returns the 1-D index of the segment of thread block of block_idx.
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    typename _dim3bi,
    typename _dim3gd>
__device__ size_t
index_of_reduction_segment(const _dim3bi& block_idx, const _dim3gd& grid_dim) {
  size_t seg_idx = 0;
  if (!Z_BLOCK)
    seg_idx += block_idx.z;
  if (!Y_BLOCK)
    seg_idx = seg_idx * grid_dim.y + block_idx.y;
  if (!X_BLOCK)
    seg_idx = seg_idx * grid_dim.x + block_idx.x;
  return seg_idx;
}

// Returns the offset of thread block in its reduction segment.
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    typename _dim3bi,
    typename _dim3gd>
__device__ size_t
offset_in_reduction_segment(const _dim3bi& block_idx, const _dim3gd& grid_dim) {
  size_t offset = 0;
  if (Z_BLOCK)
    offset = offset * grid_dim.z + block_idx.z;
  if (Y_BLOCK)
    offset = offset * grid_dim.y + block_idx.y;
  if (X_BLOCK)
    offset = offset * grid_dim.x + block_idx.x;
  return offset;
}

// Returns dim3 of each reduction block.
template <bool X_THREAD, bool Y_THREAD, bool Z_THREAD, typename _dim3>
__device__ dim3 dimension_of_reduction_block(const _dim3& block_dim) {
  return dim3{X_THREAD ? block_dim.x : 1,
              Y_THREAD ? block_dim.y : 1,
              Z_THREAD ? block_dim.z : 1};
}

// Returns the number of threads of each reduction block.
template <bool X_THREAD, bool Y_THREAD, bool Z_THREAD, typename _dim3>
__device__ int size_of_reduction_block(const _dim3& block_dim) {
  auto tmp_dim =
      dimension_of_reduction_block<X_THREAD, Y_THREAD, Z_THREAD>(block_dim);
  return isize(tmp_dim);
}

// Returns the linear offset of a thread in a reduction block.
template <
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    typename _dim3ti,
    typename _dim3bd>
__device__ int offset_in_reduction_block(
    const _dim3ti& thread_idx,
    const _dim3bd& block_dim) {
  int offset = 0;
  if (Z_THREAD)
    offset += thread_idx.z;
  if (Y_THREAD)
    offset = offset * block_dim.y + thread_idx.y;
  if (X_THREAD)
    offset = offset * block_dim.x + thread_idx.x;
  return offset;
}

// Reduces all the reduction blocks in each reduction segment.
//
// This is only used by one thread block per reduction segment. The input
// reduction blocks of the segment are stored in an intermediate buffer pointed
// by parameter in. Template parameters X/Y/Z_THREAD denote how the reduction
// block is formed.
//
// The size of a reduction block is by definition smaller or equal to the size
// of a thread block. We use the remaining threads to parallelize reductions
// across reduction blocks. For example, when X/Y/Z_THREAD = {true, false,
// false}, we use blockDim.y*blockDim.z threads for each output value. This is
// done first by loading the input values in parallel and then by reducing
// across threads of dimensions whose XYZ_THREAD are false.
//
// Note that what is done here after the loading from global memory is similar
// to what the existing blockReduce function does. The main difference is that
// the logical block to reduce is a 2D domain where the leading dimension is the
// size of a reduction block and the second dimension is the remaining factor in
// each thread block. For example, when X/Y/Z_THREAD = {false, true, false}, the
// threads are arranged as (blockDim.y, blockDim.x*blockDim.z). We do not reduce
// along the first dimension but only the second dimension. So, it is possible
// to reuse the existing blockReduce with dim3{blockDim.y,
// blockDim.x*blockDim.z} instead of blockDim and with X_THREAD and Y_THREAD
// being false and true, respectively. Also, it still need to shuffle the final
// output values to their actual corresponding threads. In the case of when
// X/Y/Z_THREAD = {false, true, false}, after the intra-block reduction, the
// final results will still be held by the first blockDim.y threads, which need
// to be transferred to threads at threadIdx.x == 0 and threadIdx.z == 0.
template <
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    typename T,
    typename Func>
__device__ void gridReduceLastBlock(
    T& out,
    const T* in,
    const size_t in_size,
    Func reduction_op,
    T* shared_buf,
    bool read_write_pred,
    T init_val) {
  const int tid = ioffset(threadIdx, blockDim);
  const int block_size = isize(blockDim);
  const int rblock_size =
      size_of_reduction_block<X_THREAD, Y_THREAD, Z_THREAD>(blockDim);

  T inp = init_val;
  if (tid < in_size) {
    inp = in[tid];
  }
  for (size_t i = tid + block_size; i < in_size; i += block_size) {
    reduction_op(inp, in[i]);
  }

  const auto should_write = (X_THREAD || threadIdx.x == 0) &&
      (Y_THREAD || threadIdx.y == 0) && (Z_THREAD || threadIdx.z == 0);

  auto rem_size = block_size / rblock_size;

  if (rem_size > 1) {
    const int rblock_offset = tid % rblock_size;
    const int rblock_idx = tid / rblock_size;
    blockReduce<false, true, false>(
        inp,
        inp,
        reduction_op,
        dim3{(unsigned)rblock_offset, (unsigned)rblock_idx, 0},
        dim3{(unsigned)rblock_size, (unsigned)rem_size},
        shared_buf,
        true,
        init_val);
    __syncthreads();
    if (tid < rblock_size) {
      shared_buf[tid] = inp;
    }
    __syncthreads();
    if (should_write) {
      inp = shared_buf[offset_in_reduction_block<X_THREAD, Y_THREAD, Z_THREAD>(
          threadIdx, blockDim)];
    }
  }

  if (should_write && read_write_pred) {
    out = inp;
  }
}

// Reduces per-thread values across thread blocks.
//
// Function parameters:
// - out: Per-thread output location
// - inp_val: Per-thread input value
// - reduction_op: Scalar reduction function
// - work_buf: Temporary buffer for cross-block reductions
// - sync_flags: A vector of integers for synchronizations
// - shared_buf: Shared memory buffer for intra-block reduction
//
// Return true when the thread block has the valid result.
//
// Template parameters:
// - X/Y/Z_BLOCK: When true, reduces across thread blocks along the X/Y/Z
//   dimensions
// - X/Y/Z_THREAD: When true, all threads along the X/Y/Z dimensions participate
//   in the cross-block reduction. Otherwise, only threads at offset 0 do.
// - T: Scalar data type of input/output data
// - Func: Type of scalara reduction function
//
// Template parameters X/Y/Z_BLOCK define a group of thread blocks that are
// reduced together. We call it a reduction segment. Some examples are:
//
// Case 1: X/Y/Z_BLOCK == true/true/true -> There is only one segment, which
// includes all thread blocks. It is effecively the same as the grid.
//
// Case 2: X/Y/Z_BLOCK == false/false/false -> Each thread block comprises an
// individual segment by itself.
//
// Case 3: X/Y/Z_BLOCK == true/false/false -> Each segment contains thread
// blocks that have the same blockDim.x. There will be blockDim.y*blockDim.z
// such segments.
//
// X/Y/Z_THREAD defines a sub region of a thread block that should be reduced
// with the sub regions of other thread blocks. We call it a reduction block.
// E.g.,
//
// Case 1: X/Y/Z_THREAD == false/false/false -> Only thread 0 participates in
// the cross-block reductions. The reduction block is 1x1x1 with thread 0.
//
// Case 2: X/Y/Z_THREAD == true/true/true-> All threads in a thread block
// participate in the cross-block reductions. The reduction block in this case
// is equivalent to the thread block.
//
// After the function completes, only one thread block per reduction segment
// gets valid reduction results. There is no guarantee which particular block
// gets the final results.
//
template <
    bool X_BLOCK,
    bool Y_BLOCK,
    bool Z_BLOCK,
    bool X_THREAD,
    bool Y_THREAD,
    bool Z_THREAD,
    typename T,
    typename Func>
__device__ bool gridReduce(
    T& out,
    T inp_val,
    Func reduction_op,
    volatile T* work_buf,
    Tensor<int64_t, 1> sync_flags,
    T* shared_buf,
    bool read_write_pred,
    T init_val) {
  // Number of values to reduce in the grid dimensions
  const auto seg_size =
      size_of_reduction_segment<X_BLOCK, Y_BLOCK, Z_BLOCK>(gridDim);

  // Index of the reduction we're performing out of the seg_size
  const auto seg_idx =
      index_of_reduction_segment<X_BLOCK, Y_BLOCK, Z_BLOCK>(blockIdx, gridDim);

  // Number of threads we can use in final reduction, Seems to assume all
  // threads in the block participate
  const auto rblock_size =
      size_of_reduction_block<X_THREAD, Y_THREAD, Z_THREAD>(blockDim);

  // advance to the offset for this segment
  // index of reduction * size of the reduction * size of threads
  work_buf += seg_idx * seg_size * rblock_size;

  if ((X_THREAD || threadIdx.x == 0) && (Y_THREAD || threadIdx.y == 0) &&
      (Z_THREAD || threadIdx.z == 0)) {
    auto rblock_offset = offset_in_reduction_segment<X_BLOCK, Y_BLOCK, Z_BLOCK>(
        blockIdx, gridDim);
    auto thread_offset =
        offset_in_reduction_block<X_THREAD, Y_THREAD, Z_THREAD>(
            threadIdx, blockDim);
    auto work_buf_offset = rblock_size * rblock_offset + thread_offset;
    if (read_write_pred) {
      work_buf[work_buf_offset] = inp_val;
    } else {
      work_buf[work_buf_offset] = init_val;
    }
  }
  __syncthreads();

  __shared__ bool last_block;
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    __threadfence();
    // printf("%ld\n", sync_flags[seg_idx]);
    auto old = (int64_t)atomicAdd((unsigned long long*)&sync_flags[seg_idx], 1);
    last_block = old + 1 == seg_size;
    // printf("Last_block = %d + 1 == %d\n", (int)old, (int)seg_size);
  }
  __syncthreads();

  if (last_block) {
    // printf("Last block %d %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
    // final reduction
    gridReduceLastBlock<X_THREAD, Y_THREAD, Z_THREAD>(
        out,
        (T*)work_buf,
        seg_size * rblock_size,
        reduction_op,
        shared_buf,
        read_write_pred,
        init_val);
    return true;
  } else {
    // printf("Not last block %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
    return false;
  }
}

} // namespace reduction
