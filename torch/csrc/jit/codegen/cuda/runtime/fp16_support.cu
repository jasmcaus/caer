
#include <hip/hip_runtime.h>
#define __HALF_TO_US(var) *(reinterpret_cast<unsigned short*>(&(var)))
#define __HALF_TO_CUS(var) *(reinterpret_cast<const unsigned short*>(&(var)))

struct __align__(2) __half {
  __host__ __device__ __half() {}

 protected:
  unsigned short __x;
};

__device__ __half __float2half(const float f) {
  __half val;
  asm("{  cvt.rn.f16.f32 %0, %1;}\n" : "=h"(__HALF_TO_US(val)) : "f"(f));
  return val;
}

__device__ float __half2float(const __half h) {
  float val;
  asm("{  cvt.f32.f16 %0, %1;}\n" : "=f"(val) : "h"(__HALF_TO_CUS(h)));
  return val;
}
