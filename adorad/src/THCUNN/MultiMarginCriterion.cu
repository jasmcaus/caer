#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THCUNN/common.h>
#include <TH/THHalf.h>
#include <THC/THCNumerics.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>

#define MULTIMARGIN_THREADS 128

template <int P, typename Dtype, typename Acctype>
__global__ void cunn_MultiMarginCriterion_updateOutput_kernel(Dtype *output, Dtype *input, THCIndex_t *target, Dtype *weights, int nframe, int dim, bool sizeAverage, Dtype margin)
{
  __shared__ Acctype buffer[MULTIMARGIN_THREADS];
  int k = blockIdx.x;
  Dtype *input_k = input + k*dim;
  Dtype *output_k = output + k;
  int target_k = ((int)target[k]);
  Dtype input_target_k = input_k[target_k];

  int i_start = threadIdx.x;
  int i_end = dim;
  int i_step = blockDim.x;

  buffer[threadIdx.x] = 0;
  for (int i = i_start; i < i_end; i += i_step)
  {
    Dtype z = margin - input_target_k + input_k[i];
    if (i == target_k)
      continue;

    if (z > 0) {
      Dtype h = (P==1) ? z : z*z;
      if(weights)
        h *= weights[target_k];
      buffer[threadIdx.x] += h;
    }
  }
  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    Acctype sum = 0;
    for (int i=0; i < blockDim.x; i++)
      sum += buffer[i];

    *output_k = ScalarConvert<Acctype, Dtype>::to(sum/dim);
    if(sizeAverage)
      *output_k /= nframe;
  }
}

template <int P, typename Dtype, typename Acctype>
__global__ void cunn_MultiMarginCriterion_updateGradInput_kernel(Dtype *gradInput,
                                                                 Dtype *gradOutput,
                                                                 Dtype *input,
                                                                 THCIndex_t *target,
                                                                 Dtype *weights,
                                                                 int nframe,
                                                                 int dim,
                                                                 bool sizeAverage,
                                                                 Dtype margin,
                                                                 int reduce)
{
  __shared__ Acctype buffer[MULTIMARGIN_THREADS];
  int k = blockIdx.x;
  Dtype *input_k = input + k*dim;
  Dtype *gradInput_k = gradInput + k*dim;
  int target_k = ((int)target[k]);
  Dtype input_target_k = input_k[target_k];

  Dtype *gradOutput_k = gradOutput;
  if (!reduce) {
    gradOutput_k += k;
  }

  Acctype g = (sizeAverage && reduce ? 1./((Acctype)(nframe*dim)) : 1./((Acctype)dim));

  int i_start = threadIdx.x;
  int i_end = dim;
  int i_step = blockDim.x;

  buffer[threadIdx.x] = 0;
  for (int i=i_start; i<i_end; i+=i_step)
  {
    Dtype z = margin - input_target_k + input_k[i];
    if (i == target_k)
      continue;

    if (z > 0)
    {
      Dtype h = ScalarConvert<Acctype, Dtype>::to((P == 1) ? g : 2*g*z);
      if(weights)
        h *= weights[target_k];
      buffer[threadIdx.x] -= h;
      gradInput_k[i] = h;
    }
    else
      gradInput_k[i] = ScalarConvert<int, Dtype>::to(0);
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    Acctype gradInput_target_k = 0;
    for (int i=0; i<blockDim.x; i++)
      gradInput_target_k += buffer[i];
    gradInput_k[target_k] = ScalarConvert<Acctype, Dtype>::to(gradInput_target_k);
  }

  for (int i=i_start; i<i_end; i+= i_step)
  {
    gradInput_k[i] *= * gradOutput_k;
  }
}

#include <THCUNN/generic/MultiMarginCriterion.cu>
#include <THC/THCGenerateFloatTypes.h>

#undef MULTIMARGIN_THREADS
