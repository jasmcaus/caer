
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/GatedLinearUnit.cu"
#else

void THNN_(GatedLinear_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int dim)
{
  THCUNN_assertSameGPU(state, 2, input, output);

  dim = at::maybe_wrap_dim(dim, input);
  // size output to half of input
  const int64_t nIn = THCTensor_(sizeLegacyNoScalars)(state, input, dim);
  THArgCheck(nIn % 2 == 0, 2, "Halving dimension must be even. Dim %d is size %ld",
      dim, nIn);
  const int64_t inputSize = THCTensor_(size)(state, input, dim) / 2;
  std::vector<int64_t> newSizes = THTensor_sizesLegacyNoScalars(input);
  newSizes[dim] = inputSize;
  THCTensor_(resize)(state, output, newSizes, {});

  // halve tensor
  THCTensor *firstHalf = THCTensor_(newNarrow)(state, input, dim, 0, inputSize);
  THCTensor *secondHalf = THCTensor_(newNarrow)(state, input, dim, inputSize, inputSize);

  // x = x1:cmul( sigmoid(x2) )
  THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, output, secondHalf, firstHalf, gatedLinearCSigMul_functor<scalar_t, accreal>());

  THCTensor_(free)(state, firstHalf);
  THCTensor_(free)(state, secondHalf);
}

void THNN_(GatedLinear_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int dim)
{
  THCUNN_assertSameGPU(state, 2, gradOutput, gradInput);
  dim = at::maybe_wrap_dim(dim, input);
  const int64_t nIn = THCTensor_(size)(state, input, dim);
  THArgCheck(nIn % 2 == 0, 2, "Halving dimension must be even. Dim %d is size %ld",
      dim, nIn);

  THCTensor_(resizeAs)(state, gradInput, input);
  const int64_t inputSize = THCTensor_(size)(state, input, dim) / 2;
  THCTensor *firstHalf = THCTensor_(newNarrow)(state, input, dim, 0, inputSize);
  THCTensor *gradInputfirstHalf = THCTensor_(newNarrow)(state, gradInput, dim, 0, inputSize);
  const int64_t stride_i = THCTensor_(stride)(state, input, dim) * inputSize;
  const int64_t stride_gI = THCTensor_(stride)(state, gradInput, dim) * inputSize;
  THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, gradInputfirstHalf, gradOutput, firstHalf, gatedLinearDerivative<scalar_t,accreal>(stride_i, stride_gI));
  THCTensor_(free)(state, firstHalf);
  THCTensor_(free)(state, gradInputfirstHalf);
}

#endif
