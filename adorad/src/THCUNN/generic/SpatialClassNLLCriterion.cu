
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialClassNLLCriterion.cu"
#else

void THNN_(SpatialClassNLLCriterion_shapeCheck)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *weights)
{
  TORCH_CHECK(target->dim() == 3, 1,
           "only batches of spatial targets supported (3D tensors)" \
           " but got targets of size: : ", target->sizes());
  TORCH_CHECK(input->dim() == 4, 2,
           "only batches of spatial inputs supported (4D tensors), "      \
           "but got input of size: ", input->sizes());
  if (THCTensor_(size)(state, input, 0) != THCIndexTensor_(size)(state, target, 0) ||
      THCTensor_(size)(state, input, 2) != THCIndexTensor_(size)(state, target, 1) ||
      THCTensor_(size)(state, input, 3) != THCIndexTensor_(size)(state, target, 2)) {
    THCDescBuff input_size = THCTensor_(sizeDesc)(state, input);
    THCDescBuff target_size = THCIndexTensor_(sizeDesc)(state, target);
    THError("input and target batch or spatial sizes don't match: target %s, input %s",
            target_size.str, input_size.str);
  }

  if (weights && THCTensor_(nElement)(state, weights) != THCTensor_(size)(state, input, 1)) {
    THError("weight tensor should be defined either for all or no classes");
  }
}

static void THNN_(SpatialClassNLLCriterion_gradOutput_no_reduce_shapeCheck)(
           THCState *state,
           THCTensor *gradOutput,
           THCIndexTensor *target)
{
  TORCH_CHECK(THCTensor_(nDimensionLegacyNoScalars)(state, gradOutput) == 3, 2,
    "gradOutput must have same dimension as target (3) but got dimension: ", gradOutput->sizes());
  if (THCTensor_(size)(state, gradOutput, 0) != THCIndexTensor_(size)(state, target, 0) ||
      THCTensor_(size)(state, gradOutput, 1) != THCIndexTensor_(size)(state, target, 1) ||
      THCTensor_(size)(state, gradOutput, 2) != THCIndexTensor_(size)(state, target, 2)) {
    THCDescBuff gradOutput_size = THCTensor_(sizeDesc)(state, gradOutput);
    THCDescBuff target_size = THCIndexTensor_(sizeDesc)(state, target);
    THError("gradOutput sizes don't match target sizes: target %s, gradOutput %s",
            target_size.str, gradOutput_size.str);
  }
}

void THNN_(SpatialClassNLLCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           int64_t reduction,
           THCTensor *weights,
           THCTensor *total_weight,
           int64_t ignore_index)
{
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  at::globalContext().alertNotDeterministic("SpatialClassNLLCriterion_updateOutput");
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);
  THCTensor_(resize0d)(state, output);
  THCTensor_(resize0d)(state, total_weight);

  if (weights)
    THCUNN_assertSameGPU(state, 5, input, target, weights, output, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, output, total_weight);

  if (reduction == at::Reduction::None) {
    int64_t batch_size = THCTensor_(size)(state, input, 0);
    int64_t H = THCTensor_(size)(state, input, 2);
    int64_t W = THCTensor_(size)(state, input, 3);
    int64_t count = batch_size * H * W;

    THCTensor_(resize3d)(state, output, batch_size, H, W);

    if (count == 0) {
      // This guards from unnecessary operations and launching CUDA kernel with 0 blocks.
      return;
    }
    if (weights) {
      weights = THCTensor_(newContiguous)(state, weights);
    }

    SpatialClassNLLCriterion_updateOutput_no_reduce_kernel<scalar_t>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, c10::cuda::getCurrentCUDAStream()>>>(
        count,
        toDeviceTensor<scalar_t, 4>(state, input),
        toDeviceTensor<THCIndex_t, 3>(state, target),
        toDeviceTensor<scalar_t, 3>(state, output),
        weights ? THCTensor_(data)(state, weights) : NULL,
        ignore_index);

    if (weights) {
      THCTensor_(free)(state, weights);
    }
    return;
  }

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  scalar_t *input_data = THCTensor_(data)(state, input);
  scalar_t *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  THCIndex_t  *target_data = THCIndexTensor_(data)(state, target);
  scalar_t *output_data = THCTensor_(data)(state, output);
  scalar_t *total_weight_data = THCTensor_(data)(state, total_weight);
  THCTensor_(fill)(state, output, ScalarConvert<int, scalar_t>::to(0));
  THCTensor_(fill)(state, total_weight, ScalarConvert<int, scalar_t>::to(0));

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  if (batch_size != 0) { // This guards from unnecessary operations and launching CUDA kernel with 0 blocks.
    THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
    int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
    blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
    int total_blocks = blocks_per_sample * batch_size;

    cunn_SpatialClassNLLCriterion_updateOutput_kernel<scalar_t, accreal>
      <<<total_blocks, CUDA_NUM_THREADS, 0, c10::cuda::getCurrentCUDAStream()>>>(
        output_data,
        total_weight_data,
        input_data,
        target_data,
        weights_data,
        reduction == at::Reduction::Mean,
        THCTensor_(size)(state, input, 0),
        THCTensor_(size)(state, input, 1),
        THCTensor_(size)(state, input, 2) * THCTensor_(size)(state, input, 3),
        blocks_per_sample,
        ignore_index
    );
    THCudaCheck(cudaGetLastError());
  }
  if (reduction == at::Reduction::Mean) {
    cunn_SpatialClassNLLCriterion_sizeAverage_kernel<<<1, 1, 0, c10::cuda::getCurrentCUDAStream()>>>(
      output_data, total_weight_data, THCTensor_(nElement)(state, input)
    );
    THCudaCheck(cudaGetLastError());
  }

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

void THNN_(SpatialClassNLLCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction,
           THCTensor *weights,
           THCTensor *total_weight,
           int64_t ignore_index)
{
  THNN_(SpatialClassNLLCriterion_shapeCheck)(state, input, target, weights);
  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);
  THArgCheck(THCTensor_(isContiguous)(state, gradInput), 4,
             "gradInput must be contiguous");

  if (weights)
    THCUNN_assertSameGPU(state, 5, weights, input, target, gradInput, total_weight);
  else
    THCUNN_assertSameGPU(state, 4, input, target, gradInput, total_weight);

  if (reduction == at::Reduction::None) {
    THNN_(SpatialClassNLLCriterion_gradOutput_no_reduce_shapeCheck)(
        state,
        gradOutput,
        target);

    int64_t batch_size = THCTensor_(size)(state, input, 0);
    int64_t H = THCTensor_(size)(state, input, 2);
    int64_t W = THCTensor_(size)(state, input, 3);
    int64_t count = batch_size * H * W;

    if (count == 0) {
      // This guards from unnecessary operations and launching CUDA kernel with 0 blocks.
      return;
    }
    if (weights) {
      weights = THCTensor_(newContiguous)(state, weights);
    }

    SpatialClassNLLCriterion_updateGradInput_no_reduce_kernel<scalar_t>
      <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, c10::cuda::getCurrentCUDAStream()>>>(
        count,
        toDeviceTensor<THCIndex_t, 3>(state, target),
        toDeviceTensor<scalar_t, 3>(state, gradOutput),
        toDeviceTensor<scalar_t, 4>(state, gradInput),
        weights ? THCTensor_(data)(state, weights) : NULL,
        ignore_index);

    if (weights) {
      THCTensor_(free)(state, weights);
    }
    return;
  }

  input = THCTensor_(newContiguous)(state, input);
  weights = weights ? THCTensor_(newContiguous)(state, weights) : NULL;
  target = THCIndexTensor_(newContiguous)(state, target);

  scalar_t *gradOutput_data = THCTensor_(data)(state, gradOutput);
  scalar_t *weights_data = weights ? THCTensor_(data)(state, weights) : NULL;
  scalar_t *gradInput_data = THCTensor_(data)(state, gradInput);
  THCIndex_t *target_data = THCIndexTensor_(data)(state, target);
  scalar_t *total_weight_data = THCTensor_(data)(state, total_weight);

  THCIndex_t batch_size = THCIndexTensor_(size)(state, target, 0);
  if (batch_size != 0) { // This guards from unnecessary operations and launching CUDA kernel with 0 blocks.
    THCIndex_t map_nelem = THCIndexTensor_(nElement)(state, target) / batch_size;
    int blocks_per_sample = GET_BLOCKS(map_nelem) / 128;
    blocks_per_sample = (blocks_per_sample == 0) ? 1 : blocks_per_sample;
    int total_blocks = blocks_per_sample * batch_size;

    cunn_SpatialClassNLLCriterion_updateGradInput_kernel
      <<<total_blocks, CUDA_NUM_THREADS, 0, c10::cuda::getCurrentCUDAStream()>>>(
        gradInput_data,
        gradOutput_data,
        target_data,
        weights_data,
        total_weight_data,
        reduction == at::Reduction::Mean,
        THCTensor_(size)(state, input, 0),
        THCTensor_(size)(state, input, 1),
        THCTensor_(size)(state, input, 2) *THCTensor_(size)(state, input, 3),
        blocks_per_sample,
        ignore_index
    );
    THCudaCheck(cudaGetLastError());
  }

  if (weights)
    THCTensor_(free)(state, weights);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, input);
}

#endif
