
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/MultiMarginCriterion.cu"
#else

// TODO: improve error messages
void THNN_(MultiMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           int64_t reduction,
           int p,
           THCTensor *weights,
           accreal margin_)
{
  scalar_t margin = ScalarConvert<accreal, scalar_t>::to(margin_);
  THCUNN_assertSameGPU(state, 2, input, target);
  input = THCTensor_(newContiguous)(state, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);
  if (THTensor_nDimensionLegacyNoScalars(input) == 1)
  {
    int nframe = 1;
    THArgCheck(!target->is_empty() && (THTensor_nDimensionLegacyNoScalars(target) == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe), 3,
               "inconsistent target size");
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);
    if (reduction == at::Reduction::None) {
      THCTensor_(resizeAs)(state, output, target);
    } else {
      THCTensor_(resize0d)(state, output);
    }
    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(input, 0),
        reduction == at::Reduction::Mean,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(input, 0),
        reduction == at::Reduction::Mean,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->dim() == 2)
  {
    int nframe = input->size(0);
    THArgCheck(!target->is_empty() && (THTensor_nDimensionLegacyNoScalars(target) == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe), 3,
               "inconsistent target size");
    dim3 blocks(input->size(0));
    dim3 threads(MULTIMARGIN_THREADS);

    if (reduction == at::Reduction::None)
    {
      THCTensor_(resizeAs)(state, output, target);
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size(1),
          false,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size(1),
          false,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
    }
    else
    {
      THCTensor_(resize0d)(state, output);
      THCTensor *output_ = THCTensor_(newWithSize1d)(state, input->size(0));  // tmp output buffer
      if (p == 1)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          nframe, input->size(1),
          reduction == at::Reduction::Mean,
          margin
        );
      }
      else if (p == 2)
      {
        cunn_MultiMarginCriterion_updateOutput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output_),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          weights ? THCTensor_(data)(state, weights) : NULL,
          input->size(0), input->size(1),
          reduction == at::Reduction::Mean,
          margin
        );
      }
      THCudaCheck(cudaGetLastError());
      auto t = THTensor_wrap(output_);
      auto r = THTensor_wrap(output);
      at::native::sum_out(r, t, at::IntArrayRef(std::vector<int64_t>{}), false, r.scalar_type());
      THCTensor_(free)(state, output_);
    }
  }
  else
  {
    AT_ERROR("non-empty vector or matrix expected, got sizes: ", input->sizes());
  }

  THCTensor_(free)(state, input);
  if(weights)
    THCTensor_(free)(state, weights);
}

void THNN_(MultiMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           int64_t reduction,
           int p,
           THCTensor *weights,
           accreal margin_)
{
  scalar_t margin = ScalarConvert<accreal, scalar_t>::to(margin_);
  THCUNN_assertSameGPU(state, 3, input, gradInput, target);
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);

  if (THTensor_nDimensionLegacyNoScalars(input) == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(gradInput, 0),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, THTensor_sizeLegacyNoScalars(gradInput, 0),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->dim() == 2)
  {
    int nframe = gradInput->size(0);
    THArgCheck(!target->is_empty() && (THTensor_nDimensionLegacyNoScalars(target) == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe), 3,
               "inconsistent target size");
    dim3 blocks(gradInput->size(0));
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size(1),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, scalar_t, accreal> <<<blocks,threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        nframe, gradInput->size(1),
        reduction == at::Reduction::Mean,
        margin,
        reduction != at::Reduction::None
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else
  {
    AT_ERROR("non-empty vector or matrix expected, got ", input->sizes());
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
  if(weights)
    THCTensor_(free)(state, weights);
}

#endif
