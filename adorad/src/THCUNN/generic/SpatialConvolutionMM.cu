
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/SpatialConvolutionMM.cu"
#else

#include <ATen/div_rtn.h>
#include <ATen/cuda/CUDABlas.h>

static inline void THNN_(SpatialConvolutionMM_shapeCheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *gradOutput,
                         THCTensor *weight, THCTensor *bias,
                         int kH, int kW, int dH, int dW, int padH, int padW,
                         int weight_nullable) {
  THArgCheck(kW > 0 && kH > 0, 9,
             "kernel size should be greater than zero, but got kH: %d kW: %d", kH, kW);
  THArgCheck(dW > 0 && dH > 0, 11,
             "stride should be greater than zero, but got dH: %d dW: %d", dH, dW);

  if (weight != NULL) {
    THCUNN_argCheck(state, !weight->is_empty() && (weight->dim() == 2 || weight->dim() == 4), 5, weight,
                    "non-empty 2D or 4D weight tensor expected, but got: %s");
    if (bias != NULL) {
      THCUNN_check_dim_size(state, bias, 1, 0, weight->size(0));
    }
  } else if (!weight_nullable) {
    THError("weight tensor is expected to be non-nullable");
  }

  int ndim = input->dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  // Allow for empty batch size but not other dimensions
  bool valid_empty = false;
  if (ndim == 3) {
    valid_empty = input->size(0) == 0 && input->size(1) != 0 && input->size(2) != 0;
  } else if (ndim == 4) {
    valid_empty = input->size(0) == 0 && input->size(1) != 0 && input->size(2) != 0 && input->size(3) != 0;
  }


  THCUNN_argCheck(state, (!input->is_empty() || valid_empty) && (ndim == 3 || ndim == 4), 2, input,
                  "non-empty 3D or 4D input tensor expected but got: %s");

  int64_t inputHeight  = input->size(dimh);
  int64_t inputWidth   = input->size(dimw);

  int64_t exactInputHeight = inputHeight + 2 * padH;
  int64_t exactInputWidth = inputWidth + 2 * padW;

  if (exactInputHeight < kH || exactInputWidth < kW) {
    THError("Calculated padded input size per channel: (%ld x %ld). "
      "Kernel size: (%d x %d). Kernel size can't be greater than actual input size",
      exactInputHeight, exactInputWidth, kH, kW);
  }

  int64_t outputHeight = div_rtn<int64_t>(exactInputHeight - kH, dH) + 1;
  int64_t outputWidth  = div_rtn<int64_t>(exactInputWidth - kW, dW) + 1;

  if (outputWidth < 1 || outputHeight < 1) {
    THError("Given input size per channel: (%ld x %ld). "
      "Calculated output size per channel: (%ld x %ld). Output size is too small",
      inputHeight, inputWidth, outputHeight, outputWidth);
  }

  if (weight != NULL) {
    int64_t nInputPlane = weight->size(1);
    if (weight->dim() == 2) {
      nInputPlane /= (kH * kW);
    }
    THCUNN_check_dim_size(state, input, ndim, dimf, nInputPlane);
  }

  if (gradOutput != NULL) {
    if (weight != NULL) {
      int64_t nOutputPlane = weight->size(0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    } else if (bias != NULL) {
      int64_t nOutputPlane = bias->dim() == 0 ? 1 : bias->size(0);
      THCUNN_check_dim_size(state, gradOutput, ndim, dimf, nOutputPlane);
    }
    THCUNN_check_dim_size(state, gradOutput, ndim, dimh, outputHeight);
    THCUNN_check_dim_size(state, gradOutput, ndim, dimw, outputWidth);
  }
}

static THCTensor* THNN_(newViewWeightMM2d)(THCState *state, THCTensor *weight) {
  weight = THCTensor_(newContiguous)(state, weight);
  if (weight->dim() == 4) {
    int64_t s1 = weight->size(0);
    int64_t s2 = weight->size(1) * weight->size(2) * weight->size(3);
    THCTensor *old_weight = weight;
    weight = THTensor_wrap(weight).view({s1, s2}).unsafeReleaseTensorImpl();
    THCTensor_(free)(state, old_weight);
  }
  return weight;
}

void THNN_(SpatialConvolutionMM_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH) {
  THCUNN_assertSameGPU(state, 5, input, output, weight, columns, ones);
  if (bias) {
    THCUNN_assertSameGPU(state, 2, weight, bias);
  }
  weight = THNN_(newViewWeightMM2d)(state, weight);
  THNN_(SpatialConvolutionMM_shapeCheck)
       (state, input, NULL, weight, bias, kH, kW, dH, dW, padH, padW, 0);
  THArgCheck(!bias || THCTensor_(isContiguous)(state, bias), 5,
             "bias tensor has to be contiguous");

  int ndim = input->dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  int64_t nInputPlane = input->size(dimf);
  int64_t inputHeight  = input->size(dimh);
  int64_t inputWidth   = input->size(dimw);
  int64_t nOutputPlane = weight->size(0);
  int64_t outputHeight = (inputHeight + 2*padH - kH) / dH + 1;
  int64_t outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;


  input = THCTensor_(newContiguous)(state, input);
  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
  }

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize4d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (bias) {
    if (ones->dim() != 2 || ones->size(0)*ones->size(1) < outputHeight*outputWidth) {
      // Resize plane and fill with ones...
      THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
      THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
    }
  }

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // Do Bias first:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m_ = nOutputPlane;
    int64_t n_ = outputHeight * outputWidth;
    int64_t k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #elif defined(THC_REAL_IS_BFLOAT16)
      THCudaBlas_Bgemm(
      #endif
          state,
          't', 'n',
          n_, m_, k_,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, ones), k_,
          THCTensor_(data)(state, bias), k_,
          ScalarConvert<int, scalar_t>::to(0),
          THCTensor_(data)(state, output_n), n_
      );
    } else {
      THCTensor_(zero)(state, output_n);
    }

    if (kW != 1 || kH != 1) {
      // Extract columns:
      at::native::im2col<scalar_t>(
        c10::cuda::getCurrentCUDAStream(),
        THCTensor_(data)(state, input_n),
        nInputPlane, inputHeight, inputWidth,
        outputHeight, outputWidth,
        kH, kW, padH, padW, dH, dW,
        1, 1,
        columns->data<scalar_t>()
      );
    }

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = nOutputPlane;
    int64_t n = columns->size(1);
    int64_t k = nInputPlane*kH*kW;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    auto gemm_in_ptr = (kW != 1 || kH != 1) ?
        THCTensor_(data)(state, columns) : THCTensor_(data)(state, input_n);
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #elif defined(THC_REAL_IS_BFLOAT16)
    THCudaBlas_Bgemm(
    #endif
        state,
        'n', 'n',
        n, m, k,
        ScalarConvert<int, scalar_t>::to(1),
        gemm_in_ptr, n,
        THCTensor_(data)(state, weight), k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, output_n), n
    );
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, weight);
}

void THNN_(SpatialConvolutionMM_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *gradColumns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH) {
  THCUNN_assertSameGPU(state, 5, input, gradOutput, weight,
                       gradColumns, gradInput);
  weight = THNN_(newViewWeightMM2d)(state, weight);

  THNN_(SpatialConvolutionMM_shapeCheck)
       (state, input, gradOutput, weight, NULL, kH, kW, dH, dW, padH, padW, 0);

  // Params
  int nInputPlane = weight->dim() == 2 ? weight->size(1)/(kW*kH) : weight->size(1);
  int nOutputPlane = weight->size(0);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2));
  }

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  int64_t outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Resize output
  THCTensor_(resize4d)(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *gradInput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    int64_t m = nInputPlane*kW*kH;
    int64_t n = gradColumns->size(1);
    int64_t k = nOutputPlane;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #elif defined(THC_REAL_IS_BFLOAT16)
    THCudaBlas_Bgemm(
    #endif
        state,
        'n', 't',
        n, m, k,
        ScalarConvert<int, scalar_t>::to(1),
        THCTensor_(data)(state, gradOutput_n), n,
        THCTensor_(data)(state, weight), m,
        ScalarConvert<int, scalar_t>::to(0),
        THCTensor_(data)(state, gradColumns), n
    );

    // Unpack columns back into input:
    at::native::col2im<scalar_t, accreal>(
      c10::cuda::getCurrentCUDAStream(),
      THCTensor_(data)(state, gradColumns),
      nInputPlane, inputHeight, inputWidth, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, gradInput_n)
    );
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);
  THCTensor_(free)(state, weight);

  // Resize output
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize3d)(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

void THNN_(SpatialConvolutionMM_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           accreal scale_) {
  scalar_t scale = ScalarConvert<accreal, scalar_t>::to(scale_);
  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight, gradBias, columns, ones);
  if (gradWeight) {
    THArgCheck(THCTensor_(isContiguous)(state, gradWeight), 4, "gradWeight needs to be contiguous");
    gradWeight = THNN_(newViewWeightMM2d)(state, gradWeight);
  }
  if (gradBias) {
    THArgCheck(THCTensor_(isContiguous)(state, gradBias), 5, "gradBias needs to be contiguous");
    THArgCheck(THCTensor_(isContiguous)(state, ones), 7, "ones needs to be contiguous");
  }

  THNN_(SpatialConvolutionMM_shapeCheck)
       (state, input, gradOutput, gradWeight, gradBias, kH, kW, dH, dW, padH, padW, 1);

  // Params
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  int is_batch = 1;
  if (input->dim() == 3) {
    // Force batch
    is_batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size(0), input->size(1), input->size(2));
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size(0), gradOutput->size(1), gradOutput->size(2));
  }

  int64_t nInputPlane = input->size(1);
  int64_t nOutputPlane = gradOutput->size(1);

  int64_t inputWidth   = input->size(3);
  int64_t inputHeight  = input->size(2);
  int64_t outputWidth  = (inputWidth + 2*padW - kW) / dW + 1;
  int64_t outputHeight = (inputHeight + 2*padH - kH) / dH + 1;

  // Batch size + input planes
  int64_t batchSize = input->size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones->dim() != 2 || ones->size(0)*ones->size(1) < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, scalar_t>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Do Weight:
    if (gradWeight) {
      // Matrix mulitply per output:
      THCTensor_(select)(state, input_n, input, 0, elt);

      if (kW != 1 || kH != 1) {
        // Extract columns:
        at::native::im2col<scalar_t>(
          c10::cuda::getCurrentCUDAStream(),
          THCTensor_(data)(state, input_n),
          nInputPlane, inputHeight, inputWidth,
          outputHeight, outputWidth,
          kH, kW, padH, padW, dH, dW,
          1, 1,
          columns->data<scalar_t>()
        );
      }

      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m = nOutputPlane;
      int64_t n = nInputPlane*kW*kH;
      int64_t k = columns->size(1);

      // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
      auto gemm_in_ptr = (kW != 1 || kH != 1) ?
          THCTensor_(data)(state, columns) : THCTensor_(data)(state, input_n);
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #elif defined(THC_REAL_IS_BFLOAT16)
      THCudaBlas_Bgemm(
      #endif
          state,
          't', 'n',
          n, m, k,
          scale,
          gemm_in_ptr, k,
          THCTensor_(data)(state, gradOutput_n), k,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradWeight), n
      );
    }

    // Do Bias:
    if (gradBias) {
      // M,N,K are dims of matrix A and B
      // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
      int64_t m_ = nOutputPlane;
      int64_t k_ = outputHeight * outputWidth;

      // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      at::cuda::blas::gemv<scalar_t>(
          't',
          k_, m_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), 1,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_BFLOAT16)
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_BFLOAT16)
      THCudaBlas_Bgemm(
      #endif
          state,
          't', 'n',
          m_, 1, k_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), k_,
          ScalarConvert<int, scalar_t>::to(1),
          THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);
  if (gradWeight)
    THCTensor_(free)(state, gradWeight);

  // Resize
  if (is_batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif
