
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/MultiLabelMarginCriterion.cu"
#else

static inline void THNN_(MultiLabelMarginCriterion_shapeCheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *target) {
  if (input->dim() <= 1) {
    int dim = input->dim() == 0 ? 1 : input->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    TORCH_CHECK(!target->is_empty() && (target->dim() <= 1) && (target_size == dim),
                "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
  } else if (input->dim() == 2) {
    int nframe = input->size(0);
    int dim = input->size(1);
    TORCH_CHECK(!target->is_empty() && (target->dim() == 2)
                && (target->size(0) == nframe) && (target->size(1) == dim),
                "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
  } else {
    TORCH_CHECK(false, "non-empty vector or matrix expected, got size: ", input->sizes());
  }
}

// TODO: improve error messages
void THNN_(MultiLabelMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           THCTensor *istarget,
           int64_t reduction)
{
  THNN_(MultiLabelMarginCriterion_shapeCheck)(state, input, target);
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, istarget, target);

  if(input->dim() <= 1)
  {
    int dim = input->dim() == 0 ? 1 : input->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    THCTensor_(resize0d)(state, output);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        reduction == at::Reduction::Mean
        );
    THCudaCheck(cudaGetLastError());
  }
  else if(input->dim() == 2)
  {
    int nframe = input->size(0);
    int dim = input->size(1);
    dim3 blocks(input->size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    if (reduction != at::Reduction::None)
    {
      THCTensor *output_tmp = THCTensor_(newWithSize1d)(state, input->size(0));
      THCTensor_(resize0d)(state, output);

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
        <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output_tmp),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          reduction == at::Reduction::Mean
          );
      THCudaCheck(cudaGetLastError());
      auto t = THTensor_wrap(output_tmp);
      auto r = THTensor_wrap(output);
      at::native::sum_out(r, t, at::IntArrayRef(std::vector<int64_t>{}), false, r.scalar_type());
      THCTensor_(free)(state, output_tmp);
    }
    else
    {
      THCTensor_(resize1d)(state, output, input->size(0));

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
        <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          false
          );
      THCudaCheck(cudaGetLastError());
    }
  }
  else {
    TORCH_INTERNAL_ASSERT(false, "non-empty vector or matrix expected (shouldn't get here)");
  }

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
}

void THNN_(MultiLabelMarginCriterion_updateGradInput)(
            THCState *state,
            THCTensor *input,
            THCIndexTensor *target,
            THCTensor *gradOutput,
            THCTensor *gradInput,
            THCTensor *istarget,
            int64_t reduction)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  if(gradInput->dim() <= 1)
  {
    int dim = gradInput->dim() == 0 ? 1 : gradInput->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    THArgCheck(!target->is_empty() && (target->dim() <= 1) && (target_size == dim), 3,
               "inconsistent target size");
    TORCH_CHECK(target->sizes() == istarget->sizes(), "inconsistent isTarget size");
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        reduction == at::Reduction::Mean,
        reduction != at::Reduction::None);

  }
  else if(gradInput->dim() == 2)
  {
    int nframe = gradInput->size(0);
    int dim = gradInput->size(1);
    THArgCheck(!target->is_empty() && (target->dim() == 2) && (target->size(0) == nframe)
               && (target->size(1) == dim), 3, "inconsistent target size");
    THArgCheck(!istarget->is_empty() && (istarget->dim() == 2) && (istarget->size(0) == nframe)
               && (istarget->size(1) == dim), 3, "inconsistent isTarget size");
    dim3 blocks(gradInput->size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        gradInput->size(0), gradInput->size(1),
        reduction == at::Reduction::Mean,
        reduction != at::Reduction::None);
  }
  else {
    AT_ERROR("non-empty vector or matrix expected, got size: ", gradInput->sizes());
  }

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
  THCTensor_(free)(state, gradOutput);
}

#endif
