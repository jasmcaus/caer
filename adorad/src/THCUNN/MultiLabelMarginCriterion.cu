#include "hip/hip_runtime.h"
#include <THCUNN/THCUNN.h>
#include <THC/THCTensor.hpp>
#include <THCUNN/common.h>
#include <THC/THCReduceApplyUtils.cuh>
#include <TH/THHalf.h>
#include <THC/THCNumerics.cuh>
#include <c10/macros/Macros.h>

#include <thrust/functional.h>

#define MULTILABELMARGIN_THREADS 1024

template <typename Dtype, typename Acctype>
#if defined(__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_1(MULTILABELMARGIN_THREADS)
#endif
__global__ void cunn_MultiLabelMarginCriterion_updateOutput_kernel(Dtype *output,
                                                                   Dtype *input,
                                                                   THCIndex_t *target,
                                                                   Dtype *istarget,
                                                                   int nframe,
                                                                   int dim,
                                                                   int sizeaverage)
{
  // Temporary sums (for mapreduce)
  __shared__ Acctype sums[MULTILABELMARGIN_THREADS];

  // vectors:
  int k = blockIdx.x;
  Dtype *input_k = input + k*dim;
  THCIndex_t *target_k = target + k*dim;
  Dtype *output_k = output + k;
  Dtype *istarget_k = istarget + k*dim;

  // zero istarget
  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    istarget_k[d] = ScalarConvert<int, Dtype>::to(0);
  }
  __syncthreads();

  // mark targets in istarget
  if (threadIdx.x == 0) {
    for (int dt = 0; dt < dim; dt++) {
      int target_idx = target_k[dt];
      if (target_idx < 0) break;
      istarget_k[target_idx] = ScalarConvert<int, Dtype>::to(1);
    }
  }
  __syncthreads();

  // iterate over targets
  Acctype sum = 0;
  for (int dt = 0; dt < dim; dt++) {
    // next target:
    int target_idx = target_k[dt];
    if (target_idx < 0) break;

    // current value for target
    Dtype input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    for (int d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!ScalarConvert<Dtype, int>::to(istarget_k[d])) {
        Dtype z = 1 - input_target_k + input_k[d];
        if (z > 0)
          sum += z;
      }
    }
  }

  // reduce
  Acctype totalSum = reduceBlock(sums, blockDim.x, sum, thrust::plus<Acctype>(), (Acctype)0);
  if (threadIdx.x == 0) {
    if (sizeaverage) {
      *output_k = ScalarConvert<Acctype, Dtype>::to((totalSum / dim) / nframe);
    } else {
      *output_k = ScalarConvert<Acctype, Dtype>::to(totalSum / dim);
    }
  }
}

template <typename Dtype, typename Acctype>
#if defined(__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_1(MULTILABELMARGIN_THREADS)
#endif
__global__ void cunn_MultiLabelMarginCriterion_updateGradInput_kernel(Dtype *gradInput,
                                                                      Dtype *gradOutput,
                                                                      Dtype *input,
                                                                      THCIndex_t *target,
                                                                      Dtype *istarget,
                                                                      int nframe,
                                                                      int dim,
                                                                      int sizeaverage,
                                                                      int reduce)
{
  // Temporary sums (for mapreduce)
  __shared__ Acctype sums[MULTILABELMARGIN_THREADS];

  // vectors:
  int k = blockIdx.x;
  Dtype *input_k = input + k*dim;
  Dtype *gradInput_k = gradInput + k*dim;
  THCIndex_t *target_k = target + k*dim;
  Dtype *istarget_k = istarget + k*dim;

  Dtype *gradOutput_k = gradOutput;
  if (!reduce) {
    gradOutput_k += k;
  }

  // gain:
  Dtype g = ScalarConvert<Acctype, Dtype>::to( sizeaverage && reduce ? 1./((Acctype)(nframe*dim)) : 1./((Acctype)dim) );

  // zero gradients:
  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    gradInput_k[d] = ScalarConvert<int, Dtype>::to(0);
  }
  __syncthreads();

  // iterate over targets
  for (int dt = 0; dt < dim; dt++) {
    // next target:
    int target_idx = (int)target_k[dt];
    if (target_idx < 0) break;

    // current value for target
    Dtype input_target_k = input_k[target_idx];

    // compare to all inputs (multithreaded):
    Acctype sum = 0;
    for (int d = threadIdx.x; d < dim; d += blockDim.x) {
      // contribute to loss only if not a target
      if (!ScalarConvert<Dtype, int>::to(istarget_k[d])) {
        Dtype z = 1 - input_target_k + input_k[d];
        if (z > 0) {
          sum -= g;
          gradInput_k[d] += g;
        }
      }
    }
    __syncthreads();

    // reduce sum
    Acctype totalSum = reduceBlock(sums, blockDim.x, sum, thrust::plus<Acctype>(), (Acctype)0);
    if (threadIdx.x == 0) {
      gradInput_k[target_idx] += ScalarConvert<Acctype, Dtype>::to(totalSum);
    }
  }

  for (int d = threadIdx.x; d < dim; d += blockDim.x) {
    gradInput_k[d] *= *gradOutput_k;
  }
}

#include <THCUNN/generic/MultiLabelMarginCriterion.cu>
#include <THC/THCGenerateFloatTypes.h>

#include <THCUNN/generic/MultiLabelMarginCriterion.cu>
#include <THC/THCGenerateBFloat16Type.h>

#undef MULTILABELMARGIN_THREADS
