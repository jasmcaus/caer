#include "hip/hip_runtime.h"
#include <algorithm>
#include <utility>

#include <THCUNN/THCUNN.h>
#include <TH/THHalf.h>
#include <THC/THCNumerics.cuh>
#include <THC/THCApply.cuh>
#include <THCUNN/common.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

// copied from cutorch/lib/THC/THCTensorRandom.cu
#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256
#define NUM_BLOCKS(n) \
  (std::min((int)THCCeilDiv(n, (ptrdiff_t)BLOCK_SIZE), MAX_NUM_BLOCKS))

template<typename T>
inline T __device__ curand_uniform_type(hiprandStatePhilox4_32_10_t *state);

template <>
inline THHalf __device__ curand_uniform_type<THHalf>(hiprandStatePhilox4_32_10_t *state) {
  auto rand = hiprand_uniform4(state);
  return ScalarConvert<float, THHalf>::to(rand.x);
}

template <>
inline float __device__ curand_uniform_type<float>(hiprandStatePhilox4_32_10_t *state) {
  auto rand = hiprand_uniform4(state);
  return rand.x;
}

template <>
inline double __device__ curand_uniform_type<double>(hiprandStatePhilox4_32_10_t *state) {
  auto rand = hiprand_uniform2_double(state);
  return rand.x;
}

template <typename T>
__global__ void rreluUpdateOutputTrain(int n, at::PhiloxCudaState philox_args,
  T *input, T* noise, T *output, double a, double b)
{
  auto seeds = at::cuda::philox::unpack(philox_args);
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  CUDA_KERNEL_LOOP(i, n)
  {
    if (input[i] <= 0)
    {
      T r = curand_uniform_type<T>(&state);
      r = ScalarConvert<double, T>::to(r * (b-a) + a);
      output[i] = input[i] * r;
      noise[i] = r;
    }
    else
    {
      output[i] = input[i];
      noise[i] = ScalarConvert<int, T>::to(1);
    }
  }
}

template <typename T>
struct RReLUUpdateOutputEval_functor
{
  const T negSlope_;

  RReLUUpdateOutputEval_functor(T negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(T *out, T *in)
  {
    const T x = *in;
    const T r = x <= 0 ? negSlope_ : ScalarConvert<int, T>::to(1);
    *out = x * r;
  }
};

template <typename T>
struct RReLUUpdateOutputEvalIP_functor
{
  const T negSlope_;

  RReLUUpdateOutputEvalIP_functor(T negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(T *x)
  {
    if (*x <= 0)
    {
      *x = *x * negSlope_;
    }
  }
};

template <typename T>
struct RReLUupdateGradInputEval_functor
{
  const T negSlope_;

  RReLUupdateGradInputEval_functor(T negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(T *gradIn, T *gradOut, T *in)
  {
    *gradIn = (*in) <= 0 ? (*gradOut) * negSlope_ : (*gradOut);
  }
};

template <typename T>
struct RReLUupdateGradInputEvalIP_functor
{
  const T negSlope_;

  RReLUupdateGradInputEvalIP_functor(T negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(T *gradOut, T *in)
  {
    if (*in <= 0)
    {
      *gradOut = (*gradOut) * negSlope_;
    }
  }
};

#include <THCUNN/generic/RReLU.cu>
#include <THC/THCGenerateFloatTypes.h>
