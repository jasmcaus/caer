#include "hip/hip_runtime.h"
#include <ATen/native/TensorIterator.h>
#include <ATen/native/quantized/affine_quantizer.h>
#include <math.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/Loops.cuh>

namespace at {
namespace native {
namespace {

void quantize_tensor_per_tensor_affine_cuda(
    Tensor rtensor,
    Tensor qtensor,
    double scale,
    int64_t zero_point) {
  AT_DISPATCH_QINT_TYPES(
      qtensor.scalar_type(), "quantize_tensor_per_tensor_affine_cuda", [&]() {
        constexpr int64_t qmin = std::numeric_limits<underlying_t>::min();
        constexpr int64_t qmax = std::numeric_limits<underlying_t>::max();

        auto iter = TensorIteratorConfig()
          .check_all_same_dtype(false)
          .add_output(qtensor)
          .add_input(rtensor)
          .add_input(qtensor)
          .build();

        gpu_kernel(iter,
          [=] GPU_LAMBDA (float raw_val, scalar_t quantized_val) -> scalar_t {
            int64_t qvalue = static_cast<int64_t>(nearbyint(raw_val / scale + zero_point));
            qvalue = std::max<int64_t>(qvalue, qmin);
            qvalue = std::min<int64_t>(qvalue, qmax);
            quantized_val.val_ = qvalue;
            return quantized_val;
        });
      });
}

void dequantize_tensor_per_tensor_affine_cuda(
    Tensor qtensor,
    Tensor rtensor,
    double scale,
    int64_t zero_point) {
  AT_DISPATCH_QINT_TYPES(
      qtensor.scalar_type(), "dequantize_tensor_per_tensor_affine_cuda", [&]() {
        auto iter = TensorIteratorConfig()
          .check_all_same_dtype(false)
          .add_output(rtensor)
          .add_input(qtensor)
          .build();
        gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t value) -> float {
          return (static_cast<float>(value.val_) - zero_point) * scale;
        });
      });
}

} // anonymous namespace

REGISTER_DISPATCH(
    quantize_tensor_per_tensor_affine_stub,
    &quantize_tensor_per_tensor_affine_cuda);
REGISTER_DISPATCH(
    dequantize_tensor_per_tensor_affine_stub,
    &dequantize_tensor_per_tensor_affine_cuda);

} // namespace native
} // namespace at
