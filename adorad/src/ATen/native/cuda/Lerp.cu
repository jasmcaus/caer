#include "hip/hip_runtime.h"
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>

namespace {

inline void lerp_cuda(at::Tensor& ret, const at::Tensor& self, const at::Tensor& end, const at::Tensor& weights) {
  TORCH_CHECK(self.dtype() == end.dtype(), "expected dtype ", self.dtype(), " for `end` but got dtype ", end.dtype());
  TORCH_CHECK(self.dtype() == weights.dtype(), "expected dtype ", self.dtype(), " for `weights` but got dtype ", weights.dtype());
  at::TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(self)
      .add_input(end)
      .add_input(weights)
      .build();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "lerp_cuda", [&]{
    at::native::gpu_kernel(iter,
      [] GPU_LAMBDA (
          scalar_t self_val,
          scalar_t end_val,
          scalar_t weight_val) -> scalar_t {
          return (weight_val < 0.5) ?
              self_val + weight_val * (end_val - self_val) : end_val - (end_val - self_val) * (1 - weight_val);
        });
      });
}

template <typename scalar_t>
void lerp_scalar_cuda(at::Tensor& ret, const at::Tensor& self, const at::Tensor& end, scalar_t weight_val) {
  TORCH_CHECK(self.dtype() == end.dtype(), "expected dtype ", self.dtype(), " for `end` but got dtype ", end.dtype());
  at::TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(self)
      .add_input(end)
      .build();
  at::native::gpu_kernel(iter,
    [=] GPU_LAMBDA (scalar_t self_val, scalar_t end_val) {
      return (weight_val < 0.5) ? self_val + weight_val * (end_val - self_val) : end_val - (end_val - self_val) * (1 - weight_val);
    });
}
} // namespace

namespace at {
namespace native {

Tensor& lerp_cuda_tensor_out(Tensor& result, const Tensor& self,
                            const Tensor& end, const Tensor& weight) {
  Tensor b_self, b_end, b_weight;
  TORCH_CHECK(weight.dim() <= std::max(self.dim(), end.dim()),
           "weight should be of dimension max(self.dim(), end.dim()) or lesser");
  std::tie(b_self, b_end, b_weight) = expand_outplace(self, end, weight, "lerp_out_cuda");
  lerp_cuda(result, b_self, b_end, b_weight);
  return result;
}

Tensor& lerp_cuda_scalar_out(Tensor& result, const Tensor& self,
                            const Tensor& end, Scalar weight) {
  Tensor b_self, b_end;
  std::tie(b_self, b_end) = expand_outplace(self, end, "lerp_out_cuda");
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "lerp_out_cuda", [&]{
    lerp_scalar_cuda<scalar_t>(result, b_self, b_end, weight.to<scalar_t>());
  });
  return result;
}

Tensor& lerp_cuda_tensor_(Tensor& self, const Tensor& end, const Tensor& weight) {
  Tensor b_self, b_end, b_weight;
  std::tie(b_self, b_end, b_weight) = expand_outplace(self, end, weight, "lerp__cuda");
  TORCH_CHECK(b_self.sizes() == self.sizes(),
           "output with shape ", self.sizes(),
           " doesn't match the broadcast shape ", b_self.sizes());
  TORCH_CHECK(weight.dim() <= std::max(self.dim(), end.dim()),
           "weight should be of dimension max(self.dim(), end.dim()) or lesser");
  lerp_cuda(self, b_self, b_end, b_weight);
  return self;
}

Tensor& lerp_cuda_scalar_(Tensor& self, const Tensor& end, Scalar weight) {
  Tensor b_self, b_end;
  std::tie(b_self, b_end) = expand_outplace(self, end, "lerp__cuda");
  TORCH_CHECK(b_self.sizes() == self.sizes(),
           "output with shape ", self.sizes(),
           " doesn't match the broadcast shape ", b_self.sizes());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "lerp__cuda", [&]{
    lerp_scalar_cuda<scalar_t>(self, b_self, b_end, weight.to<scalar_t>());
  });
  return self;
}

Tensor lerp_cuda_tensor(const Tensor& self, const Tensor& end, const Tensor& weight) {
  Tensor b_self, b_end, b_weight;
  TORCH_CHECK(weight.dim() <= std::max(self.dim(), end.dim()),
           "weight should be of dimension max(self.dim(), end.dim()) or lesser");
  std::tie(b_self, b_end, b_weight) = expand_outplace(self, end, weight, "lerp_cuda");
  Tensor result = at::empty_like(b_self, b_self.suggest_memory_format());
  lerp_cuda(result, b_self, b_end, b_weight);
  return result;
}

Tensor lerp_cuda_scalar(const Tensor& self, const Tensor& end, Scalar weight) {
  Tensor b_self, b_end;
  std::tie(b_self, b_end) = expand_outplace(self, end, "lerp_cuda");
  Tensor result = at::empty_like(b_self, b_self.suggest_memory_format());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "lerp_cuda", [&]{
    lerp_scalar_cuda<scalar_t>(result, b_self, b_end, weight.to<scalar_t>());
  });
  return result;
}

} // namespace native
} // namespace at
