#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <THC/THCNumerics.cuh>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCGeneral.h>
#include <cub/device/device_scan.cuh>


namespace at { namespace native {

template <typename integer>
constexpr inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

template<typename scalar_t, typename idx_t, typename BinaryOperation>
__device__ void binary_op_update(const scalar_t lhs, scalar_t& rhs, const idx_t lhs_idx, idx_t& rhs_idx, BinaryOperation binary_op) {
  if(!THCNumerics<scalar_t>::isnan(rhs) && (THCNumerics<scalar_t>::isnan(lhs) || !binary_op(rhs, lhs))) {
    rhs = lhs;
    rhs_idx = lhs_idx;
  }
}
/* Perform an inclusive scan along the innermost dimension of a tensor.
 *
 * - num_rows is the size of the flattened outer dimensions;
 * - row_size is the size of the innermost dimension;
 *
 * The outer dimensions of the tensor are considered as a single dimension, i.e. the tensor is
 * considered as having 'num_rows' rows of size 'row_size'.
 * Each thread block processes one or more sets of contiguous rows (processing multiple rows
 * per thread block is quicker than processing a single row, especially for short rows).
 */
template<typename scalar_t, int num_threads_x, int num_threads_y, class BinaryFunction>
__global__ void tensor_kernel_scan_innermost_dim_with_indices(const scalar_t *self_, scalar_t *values_, int64_t *indices_,
                                                int num_rows, int row_size,
                                                scalar_t init, BinaryFunction binary_op) {
  __shared__ scalar_t vbuf[num_threads_y][2 * num_threads_x];
  __shared__ int64_t ibuf[num_threads_y][2 * num_threads_x];
  scalar_t* row_buf = vbuf[threadIdx.y];
  int64_t* row_idx_buf = ibuf[threadIdx.y];

  for (int block_row = blockIdx.x * blockDim.y;
       block_row < num_rows;
       block_row += blockDim.y * gridDim.x) {
    int row = block_row + threadIdx.y;
    const scalar_t *row_self = self_ + row * row_size;
    scalar_t *row_values = values_ + row * row_size;
    int64_t *row_indices = indices_ + row * row_size;
    scalar_t block_total = init;
    int64_t block_idx_final = 0;
    // Perform scan on one block at a time, keeping track of the total value of
    // all blocks processed so far.
    for (int block_col = 0; block_col < row_size; block_col += 2 * num_threads_x) {
      // Load data into shared memory (two values per thread).
      int col1 = block_col + threadIdx.x;
      int col2 = block_col + num_threads_x + threadIdx.x;
      if (row < num_rows) {
        if (col1 < row_size) {
          row_buf[threadIdx.x] = row_self[col1];
          row_idx_buf[threadIdx.x] = col1;
        } else {
          row_buf[threadIdx.x] = init;
          // No need to set the index here as the value in init will never be selected
        }

        if (col2 < row_size) {
          row_buf[num_threads_x + threadIdx.x] = row_self[col2];
          row_idx_buf[num_threads_x + threadIdx.x] = col2;
        } else {
          row_buf[num_threads_x + threadIdx.x] = init;
          // No need to set the index here as the value in init will never be selected
        }

        // Add the total value of all previous blocks to the first value of this block.
        if (threadIdx.x == 0) {
          binary_op_update(block_total, row_buf[0], block_idx_final, row_idx_buf[0], binary_op);
        }
      }
      __syncthreads();

      // Parallel reduction (up-sweep).
      for (int s = num_threads_x, d = 1; s >= 1; s >>= 1, d <<= 1) {
        if (row < num_rows && threadIdx.x < s) {
          int offset = (2 * threadIdx.x + 1) * d - 1;
          binary_op_update(row_buf[offset], row_buf[offset + d], row_idx_buf[offset], row_idx_buf[offset + d], binary_op);
        }
        __syncthreads();
      }

      // Down-sweep.
      for (int s = 2, d = num_threads_x / 2; d >= 1; s <<= 1, d >>= 1) {
        if (row < num_rows && threadIdx.x < s - 1) {
          int offset = 2 * (threadIdx.x + 1) * d - 1;
          binary_op_update(row_buf[offset], row_buf[offset + d], row_idx_buf[offset], row_idx_buf[offset + d], binary_op);
        }
        __syncthreads();
      }

      // Write back to output.
      if (row < num_rows) {
        if (col1 < row_size){
          row_values[col1] = row_buf[threadIdx.x];
          row_indices[col1] = row_idx_buf[threadIdx.x];
        }
        if (col2 < row_size) {
          row_values[col2] = row_buf[num_threads_x + threadIdx.x];
          row_indices[col2] = row_idx_buf[num_threads_x + threadIdx.x];
        }
      }
      block_total = row_buf[2 * num_threads_x - 1];
      block_idx_final = row_idx_buf[2 * num_threads_x - 1];
      __syncthreads();
    }
  }
}

/* Perform an inclusive scan along an outer dimension of a tensor.
 *
 * - num_orows is the size of the flattened outer dimensions;
 * - num_irows is the size of the flattened inner dimensions;
 * - row_size is the size of the dimension along which to compute the variance;
 *
 * The dimensions to the outside and inside of the specified dimension are considered as flattened.
 * Thread blocks with the same blockIdx.y process an "outer row" (i.e. an element of the flattened
 * outer dimensions, which contains several "inner rows").
 * Each thread processes a single inner row at a time.
 */
template<typename scalar_t, class BinaryFunction>
__global__ void tensor_kernel_scan_outer_dim_with_indices(scalar_t *self_, scalar_t *values_, int64_t *indices_,
                  const uint32_t num_orows, const uint32_t num_irows, const uint32_t row_size, scalar_t init, BinaryFunction binary_op) {
  for (uint32_t orow = blockIdx.x; orow < num_orows; orow += gridDim.x) {
    for (uint32_t irow = blockIdx.y * blockDim.x + threadIdx.x; irow < num_irows; irow += gridDim.y * blockDim.x) {
      scalar_t *self = self_ + orow * row_size * num_irows + irow;
      scalar_t *values = values_ + orow * row_size * num_irows + irow;
      int64_t *indices = indices_ + orow * row_size * num_irows + irow;
      scalar_t out = init;
      int64_t out_idx = 0;

      for (auto col = decltype(row_size){0}; col < row_size; ++col) {
        if(THCNumerics<scalar_t>::isnan(*self) || (!THCNumerics<scalar_t>::isnan(out) && binary_op(*self, out))) {
          out = *self;
          out_idx = col;
        }
        *values = out;
        *indices = out_idx;
        self += num_irows;
        values += num_irows;
        indices += num_irows;
      }
    }
  }
}

void check_fits_in_unsigned(int64_t val, const char* name) {
  constexpr auto umax = std::numeric_limits<uint32_t>::max();
  TORCH_CHECK(
      val >= 0 && val <= umax, name, " must fit in a 32-bit uint32_t value");
}


template<typename scalar_t, class BinaryFunction>
__host__ void scan_outer_dim_with_indices(const Tensor& self, Tensor& values, Tensor& indices,
                                       int dim, scalar_t init, BinaryFunction binary_op) {
  int64_t row_size = self.size(dim);
  auto sizes = self.sizes();

  // Treat all outer dimensions (i.e. dim_ < dim) as one.
  const int64_t num_orows = prod_intlist(sizes.begin(), sizes.begin() + dim);

  // Treat all inner dimensions (i.e. dim > dimension) as one.
  const int64_t num_irows = prod_intlist(sizes.begin() + dim + 1, sizes.end());
  //for performance reasons, cuda kernels use uint32_t for loops over irows, orows and row,
  //make sure that input is not bigger than supported by uint32_t
  check_fits_in_unsigned(num_irows, "num_irows");
  check_fits_in_unsigned(num_orows, "num_orows");
  check_fits_in_unsigned(row_size, "row_size");


  dim3 threads(std::min(512, int(num_irows)));
  int64_t maxGridDim = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
  dim3 grid(std::min(maxGridDim, num_orows), std::min(maxGridDim, ceil_div(num_irows, int64_t{threads.x})));
  tensor_kernel_scan_outer_dim_with_indices<scalar_t><<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
    self.data_ptr<scalar_t>(), values.data_ptr<scalar_t>(), indices.data_ptr<int64_t>(),
    num_orows, num_irows, row_size, init, binary_op);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename scalar_t, class BinaryFunction>
__host__ void scan_innermost_dim_with_indices(const Tensor& self, Tensor& values, Tensor& indices, scalar_t init, BinaryFunction binary_op) {
  int ndim = self.dim();
  // Treat all outer dimensions as a single dimension.
  int row_size = self.size(ndim - 1);
  int num_rows = self.numel() / row_size;

  dim3 threads(16, 32);
  dim3 grid(std::min(at::cuda::getCurrentDeviceProperties()->maxGridSize[0], ceil_div(num_rows, int(threads.y))));

  tensor_kernel_scan_innermost_dim_with_indices<scalar_t, 16, 32><<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
    self.data_ptr<scalar_t>(), values.data_ptr<scalar_t>(), indices.data_ptr<int64_t>(),
    num_rows, row_size, init, binary_op);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename scalar_t, typename BinaryFunction>
void scan_dim_with_indices(const Tensor& self, Tensor& values, Tensor& indices, //int64_t dim) {
     int64_t dim, scalar_t init, BinaryFunction binary_op) {
  int ndim = self.dim();
  Tensor self_ = self.contiguous();
  Tensor values_ = values.contiguous();
  Tensor indices_ = indices.contiguous();
  bool copy_values = !values.is_contiguous();
  bool copy_indices = !indices.is_contiguous();
   if (dim == ndim - 1) {
     scan_innermost_dim_with_indices<scalar_t>(self_, values_, indices_, init, binary_op);
   } else {
     scan_outer_dim_with_indices<scalar_t>(self_, values_, indices_, dim, init, binary_op);
   }
   if (copy_values){
     values.copy_(values_);
   }
   if (copy_indices){
     indices.copy_(indices_);
   }
}

void cummax_helper_cuda(const Tensor& self, Tensor& values, Tensor& indices, int64_t dim) {
  TensorArg output_arg{ values, "output", 1 };
  TensorArg indices_arg{ indices, "indices", 2 };
  TensorArg input_arg{ self, "input", 3 };
  checkAllSameGPU("cummax", {output_arg, indices_arg, input_arg});
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Bool, at::ScalarType::Half,
    self.scalar_type(), "cummax_cuda", [&]() {
    scalar_t init = self.is_floating_point() ? (-1*std::numeric_limits<scalar_t>::infinity()) : std::numeric_limits<scalar_t>::lowest();
    scan_dim_with_indices<scalar_t>(self, values, indices, dim, init, std::greater_equal<scalar_t>());
  });
}

void cummin_helper_cuda(const Tensor& self, Tensor& values, Tensor& indices, int64_t dim) {
  TensorArg output_arg{ values, "output", 1 };
  TensorArg indices_arg{ indices, "indices", 2 };
  TensorArg input_arg{ self, "input", 3 };
  checkAllSameGPU("cummin", {output_arg, indices_arg, input_arg});
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Bool, at::ScalarType::Half,
    self.scalar_type(), "cummin_cuda", [&]() {
    scalar_t init = self.is_floating_point() ? std::numeric_limits<scalar_t>::infinity() : std::numeric_limits<scalar_t>::max();
    scan_dim_with_indices<scalar_t>(self, values, indices, dim, init, std::less_equal<scalar_t>());
  });
}

// TODO: The implementation of `tensor_kernel_scan_outer_dim` and
// `tensor_kernel_scan_innermost_dim` is similar to
// `tensor_kernel_scan_outer_dim_with_indices`
// `tensor_kernel_scan_outer_dim_with_indices` and should be refactored to
// remove the duplication.

/* Perform an inclusive scan along an outer dimension of a tensor.
 *
 * - num_orows is the size of the flattened outer dimensions;
 * - num_irows is the size of the flattened inner dimensions;
 * - row_size is the size of the dimension along which to scan;
 *
 * The dimensions to the outside and inside of the specified dimension are considered as flattened.
 * Thread blocks with the same blockIdx.y process an "outer row" (i.e. an element of the flattened
 * outer dimensions, which contains several "inner rows").
 * Each thread processes a single inner row at a time.
 */
template<typename scalar_t, class BinaryOp>
__global__ void tensor_kernel_scan_outer_dim(scalar_t *tgt_, scalar_t *src_,
                                              const uint32_t num_orows, const uint32_t num_irows, const uint32_t row_size,
                                              const scalar_t init, BinaryOp binary_op)
{
  for (uint32_t orow = blockIdx.x; orow < num_orows; orow += gridDim.x) {
    for (uint32_t irow = blockIdx.y * blockDim.x + threadIdx.x; irow < num_irows; irow += gridDim.y * blockDim.x) {
      scalar_t *src = src_ + orow * row_size * num_irows + irow;
      scalar_t *tgt = tgt_ + orow * row_size * num_irows + irow;
      scalar_t acc = init;

      for (uint32_t col = 0; col < row_size; ++col) {
        acc = binary_op(acc, *src);
        *tgt = acc;

        src += num_irows;
        tgt += num_irows;
      }
    }
  }
}

/* Perform an inclusive scan along the innermost dimension of a tensor.
 *
 * - num_rows is the size of the flattened outer dimensions;
 * - row_size is the size of the innermost dimension;
 *
 * The outer dimensions of the tensor are considered as a single dimension, i.e. the tensor is
 * considered as having 'num_rows' rows of size 'row_size'.
 * Each thread block processes one or more sets of contiguous rows (processing multiple rows
 * per thread block is quicker than processing a single row, especially for short rows).
 */
template<typename T, int num_threads_x, int num_threads_y, class BinaryFunction>
__device__ void tensor_kernel_scan_innermost_dim_impl(T* row_buf, T *tgt_, T *src_,
                                      const uint32_t num_rows, const uint32_t row_size,
                                      T init, BinaryFunction binary_op){
  for (uint32_t block_row = blockIdx.x * blockDim.y;
       block_row < num_rows;
       block_row += blockDim.y * gridDim.x) {
    uint32_t row = block_row + threadIdx.y;
    T block_total = init;

    T *row_src = src_ + row * row_size;
    T *row_tgt = tgt_ + row * row_size;

    // Perform scan on one block at a time, keeping track of the total value of
    // all blocks processed so far.
    for (uint32_t block_col = 0; block_col < row_size; block_col += 2 * num_threads_x) {
      // Load data into shared memory (two values per thread).
      uint32_t col1 = block_col + threadIdx.x;
      uint32_t col2 = block_col + num_threads_x + threadIdx.x;
      if (row < num_rows) {
        if (col1 < row_size) {
          row_buf[threadIdx.x] = row_src[col1];
        } else {
          row_buf[threadIdx.x] = init;
        }

        if (col2 < row_size) {
          row_buf[num_threads_x + threadIdx.x] = row_src[col2];
        } else {
          row_buf[num_threads_x + threadIdx.x] = init;
        }

        // Add the total value of all previous blocks to the first value of this block.
        if (threadIdx.x == 0) {
          row_buf[0] = binary_op(row_buf[0], block_total);
        }
      }
      __syncthreads();

      // Parallel reduction (up-sweep).
      for (uint32_t s = num_threads_x, d = 1; s >= 1; s >>= 1, d <<= 1) {
        if (row < num_rows && threadIdx.x < s) {
          uint32_t offset = (2 * threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = binary_op(row_buf[offset], row_buf[offset + d]);
        }
        __syncthreads();
      }

      // Down-sweep.
      for (uint32_t s = 2, d = num_threads_x / 2; d >= 1; s <<= 1, d >>= 1) {
        if (row < num_rows && threadIdx.x < s - 1) {
          uint32_t offset = 2 * (threadIdx.x + 1) * d - 1;
          row_buf[offset + d] = binary_op(row_buf[offset], row_buf[offset + d]);
        }
        __syncthreads();
      }

      // Write back to output.
      if (row < num_rows) {
        if (col1 < row_size) row_tgt[col1] = row_buf[threadIdx.x];
        if (col2 < row_size) row_tgt[col2] = row_buf[num_threads_x + threadIdx.x];
      }
      block_total = row_buf[2 * num_threads_x - 1];
      __syncthreads();
    }
  }
}

template <
    typename T,
    int num_threads_x,
    int num_threads_y,
    class BinaryFunction>
__global__ typename std::enable_if<!c10::is_complex<T>::value, void>::type
tensor_kernel_scan_innermost_dim(
    T* tgt_,
    T* src_,
    const uint32_t num_rows,
    const uint32_t row_size,
    T init,
    BinaryFunction binary_op) {
  __shared__ T sbuf[num_threads_y][2 * num_threads_x];
  T* row_buf = sbuf[threadIdx.y];

  tensor_kernel_scan_innermost_dim_impl<T, num_threads_x, num_threads_y>(
      row_buf, tgt_, src_, num_rows, row_size, init, binary_op);
}

template <
    typename T,
    int num_threads_x,
    int num_threads_y,
    class BinaryFunction>
__global__ typename std::enable_if<c10::is_complex<T>::value, void>::type
tensor_kernel_scan_innermost_dim(
    T* tgt_,
    T* src_,
    const uint32_t num_rows,
    const uint32_t row_size,
    T init,
    BinaryFunction binary_op) {
  // As we cannot directly initialize shared array for complex types
  // Reference:
  //  `error: initializer not allowed for __shared__ variable`
  // We instead get the base scalar type and allocate twice number of
  // elements required of base type and reinterpret them as complex.
  using base_t = typename scalar_value_type<T>::type;
  __shared__ base_t sbuf[num_threads_y][4 * num_threads_x];

  T* row_buf = reinterpret_cast<T*>(sbuf[threadIdx.y]);

  tensor_kernel_scan_innermost_dim_impl<T, num_threads_x, num_threads_y>(
      row_buf, tgt_, src_, num_rows, row_size, init, binary_op);
}


template<typename scalar_t, class BinaryFunction>
__host__ void scan_outer_dim(const Tensor& self, Tensor& result,
                                       int dim, scalar_t init, BinaryFunction binary_op) {
  const int64_t row_size = self.size(dim);
  auto sizes = self.sizes();

  // Treat all outer dimensions (i.e. dim_ < dim) as one.
  const int64_t num_orows = prod_intlist(sizes.begin(), sizes.begin() + dim);

  // Treat all inner dimensions (i.e. dim > dimension) as one.
  const int64_t num_irows = prod_intlist(sizes.begin() + dim + 1, sizes.end());

  dim3 threads(std::min(512, int(num_irows)));
  int64_t maxGridDim = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
  dim3 grid(std::min(maxGridDim, num_orows), std::min(maxGridDim, ceil_div(num_irows, int64_t{threads.x})));

  check_fits_in_unsigned(num_irows, "num_irows");
  check_fits_in_unsigned(num_orows, "num_orows");
  check_fits_in_unsigned(row_size, "row_size");

  tensor_kernel_scan_outer_dim<scalar_t><<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
    result.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(),
    num_orows, num_irows, row_size, init, binary_op);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename scalar_t, class BinaryFunction>
void scan_innermost_dim(const Tensor& self, Tensor& result, scalar_t init, BinaryFunction binary_op) {
  int64_t ndim = self.dim();
  // Treat all outer dimensions as a single dimension.
  int64_t row_size = self.size(ndim - 1);
  int64_t num_rows = self.numel() / row_size;

  dim3 threads(16, 32);
  int64_t maxGridDim = at::cuda::getCurrentDeviceProperties()->maxGridSize[0];
  dim3 grid(std::min(maxGridDim, ceil_div(num_rows, int64_t{threads.y})));

  check_fits_in_unsigned(num_rows, "Number of rows (self.numel()/self.size(self.dim()-1))");
  check_fits_in_unsigned(row_size, "row_size");

  tensor_kernel_scan_innermost_dim<scalar_t, 16, 32><<<grid, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
    result.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(),
    num_rows, row_size, init, binary_op);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename scalar_t, class func_t>
__global__ void transform_vals(scalar_t * a, scalar_t * b, scalar_t * out, func_t binary_op){
   *out = binary_op(*a, *b);
}

template<typename scalar_t, typename BinaryFunction>
void scan_cub(const Tensor& self, Tensor& result, scalar_t init, BinaryFunction binary_op) {
  int64_t size = self.numel();
  // non synchronizing cub call
  // even though cub is supposed to support tensors with int_max elements, in reality it doesn't,
  // so split at int_max/2
  constexpr int max_cub_size = std::numeric_limits<int>::max() / 2 + 1; // 2**30
  for (int64_t i = 0; i < size; i += max_cub_size) {
    int size_cub = std::min<int64_t>(size - i, max_cub_size);
    Tensor first_elem; // need to save it for all iterations other than first
    if (i > 0) {
      // need to temporarily transform first element of the range we are
      // operating on; self might be multi-d, but we need to index a single
      // element
      auto self_view = at::_unsafe_view(self, -1);
      first_elem = self_view[i].clone();
      transform_vals<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
          self.data_ptr<scalar_t>() + i,
          result.data_ptr<scalar_t>() + i - 1,
          self.data_ptr<scalar_t>() + i,
          binary_op);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
    size_t temp_storage_bytes = 0;
    AT_CUDA_CHECK(hipcub::DeviceScan::InclusiveScan(
        nullptr,
        temp_storage_bytes,
        self.data_ptr<scalar_t>() + i,
        result.data_ptr<scalar_t>() + i,
        binary_op,
        size_cub,
        at::cuda::getCurrentCUDAStream()));
    auto temp_storage = at::native::empty_cuda(
        {static_cast<int64_t>(temp_storage_bytes)},
        kByte, self.options().layout_opt(), self.options().device_opt(),
        self.options().pinned_memory_opt());
    AT_CUDA_CHECK(hipcub::DeviceScan::InclusiveScan(
        temp_storage.data_ptr(),
        temp_storage_bytes,
        self.data_ptr<scalar_t>() + i,
        result.data_ptr<scalar_t>() + i,
        binary_op,
        size_cub,
        at::cuda::getCurrentCUDAStream()));
    if (i > 0) {
      if (self.data_ptr<scalar_t>() != result.data_ptr<scalar_t>()) {
        // restore modified first element only if it's not an inplace operation
        auto self_view = at::_unsafe_view(self, -1);
        self_view[i].copy_(first_elem, /*non_blocking=*/true);
      }
    }
  }
}

template<typename scalar_t, typename BinaryFunction>
void scan_dim(const Tensor& self, Tensor& result,
     int64_t dim, scalar_t init, BinaryFunction binary_op) {
  int ndim = self.dim();
  Tensor self_ = self.contiguous();
  bool copy_result = !result.is_contiguous();
  Tensor result_ = result.contiguous();

  if (self.numel() == self.size(dim)) {
    scan_cub<scalar_t>(self_, result_, init, binary_op);
  } else if (dim == ndim - 1) {
    scan_innermost_dim<scalar_t>(self_, result_, init, binary_op);
  } else {
    scan_outer_dim<scalar_t>(self_, result_, dim, init, binary_op);
  }
  if (copy_result) {
    result.copy_(result_);
  }
}

Tensor& _logcumsumexp_out_cuda(Tensor& result, const Tensor& self, int64_t dim) {
  result.resize_(self.sizes());
  if (self.dim() == 0) {
    result.fill_(self);
    return result;
  }
  if (self.numel() == 0) {
    result.zero_();
    return result;
  }
  auto wrap_dim = maybe_wrap_dim(dim, self.dim());

  TensorArg output_arg{ result, "output", 1 };
  TensorArg input_arg{ self, "input", 2 };
  checkAllSameGPU("logcumsumexp", {output_arg, input_arg});

  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half,
    self.scalar_type(), "logcumsumexp_cuda", [&]() {
    scalar_t init = -std::numeric_limits<scalar_t>::infinity();
    auto log_add_exp = [] C10_HOST_DEVICE (const scalar_t x, const scalar_t y) -> scalar_t {
      return ::log1p(std::exp(std::min(x, y) - std::max(x, y))) +
          std::max(x, y);
    };
    scan_dim<scalar_t>(self, result, wrap_dim, init, log_add_exp);
  });

  return result;
}

Tensor _logcumsumexp_cuda(const Tensor& self, int64_t dim) {
  Tensor result = at::empty_like(self, MemoryFormat::Contiguous);
  return _logcumsumexp_out_cuda(result, self, dim);
}

Tensor& _cumsum_out_cuda(Tensor& result, const Tensor& self, int64_t dim) {
  TensorArg output_arg{result, "output", 1};
  TensorArg input_arg{self, "input", 2};
  checkAllSameGPU("cumsum", {output_arg, input_arg});
  checkSameType("cumsum", output_arg, input_arg);

  result.resize_(self.sizes());
  if (self.dim() == 0) {
    result.fill_(self);
    return result;
  }
  if (self.numel() == 0) {
    result.zero_();
    return result;
  }
  auto wrap_dim = maybe_wrap_dim(dim, self.dim());

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(
      at::ScalarType::Half, self.scalar_type(), "cumsum_cuda", [&]() {
        scalar_t init = 0;
        scan_dim<scalar_t>(
            self,
            result,
            wrap_dim,
            init,
            std::plus<scalar_t>());
      });

  return result;
}

Tensor _cumsum_cuda(const Tensor& self, int64_t dim) {
  Tensor result = at::empty_like(self, MemoryFormat::Contiguous);
  return _cumsum_out_cuda(result, self, dim);
}

Tensor& _cumprod_out_cuda(Tensor& result, const Tensor& self, int64_t dim) {
  TensorArg output_arg{result, "output", 1};
  TensorArg input_arg{self, "input", 2};
  checkAllSameGPU("cumprod", {output_arg, input_arg});
  checkSameType("cumprod", output_arg, input_arg);

  result.resize_(self.sizes());
  if (self.dim() == 0) {
    result.fill_(self);
    return result;
  }
  if (self.numel() == 0) {
    result.zero_();
    return result;
  }
  auto wrap_dim = maybe_wrap_dim(dim, self.dim());

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(
      at::ScalarType::Half, self.scalar_type(), "cumprod_cuda", [&]() {
        scalar_t init = 1;
        scan_dim<scalar_t>(
            self,
            result,
            wrap_dim,
            init,
            std::multiplies<scalar_t>());
      });

  return result;
}

Tensor _cumprod_cuda(const Tensor& self, int64_t dim) {
  Tensor result = at::empty_like(self, MemoryFormat::Contiguous);
  return _cumprod_out_cuda(result, self, dim);
}

}} // namespace at::native
