#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/TensorUtils.h>
#include <ATen/NativeFunctions.h>

#include <ATen/AccumulateType.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>
#include <THC/THCAtomics.cuh>

#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>

#include <ATen/native/cuda/EmbeddingBackwardKernel.cuh>

#include <c10/macros/Macros.h>

namespace at {
namespace native {

namespace {

constexpr int MODE_SUM = 0;
constexpr int MODE_MEAN = 1;
constexpr int MODE_MAX = 2;

// This kernel assumes that all input tensors except `weight` and
// per_sample_weights are contiguous.
template <typename scalar_t, typename index_t>
__global__ void EmbeddingBag_updateOutputKernel(
    index_t *input, index_t *offsets, scalar_t *weight, scalar_t *output,
    index_t *offset2bag, int64_t numIndices, int64_t numBags,
    int64_t featureSize, int64_t weight_stride0, int64_t weight_stride1,
    int mode, index_t *bag_size, index_t *max_indices,
    scalar_t* per_sample_weights, int64_t per_sample_weights_stride) {

  // the strategy here is that each bag x feature is handled by a single thread

  using accscalar_t = acc_type<scalar_t, true>;
  int64_t chunksPerBag = THCCeilDiv(featureSize, (int64_t)blockDim.x);
  int64_t numChunks = numBags * chunksPerBag;
  int64_t chunkOffset = blockIdx.x * blockDim.y + threadIdx.y;
  int64_t chunkStride = gridDim.x * blockDim.y;

  for (int64_t chunk = chunkOffset; chunk < numChunks; chunk += chunkStride) {
    int64_t featureDim = (chunk % chunksPerBag) * blockDim.x + threadIdx.x;
    if (featureDim < featureSize) {
      int64_t bag = chunk / chunksPerBag;
      scalar_t *weightFeat = weight + featureDim * weight_stride1;
      int64_t begin = bag == 0 ? 0 : offsets[bag]; // forces first offset to be 0 instead of asserting on it
      int64_t end = (bag < numBags - 1) ? (offsets[bag + 1]) : numIndices;
      CUDA_KERNEL_ASSERT(end >= begin);

      accscalar_t weightFeatSum = 0;
      scalar_t weightFeatMax;

      int64_t bag_size_ = 0;
      int64_t maxWord = -1;
      for (int64_t emb = begin; emb < end; emb++) {
        const int64_t weightRow = input[emb] * weight_stride0;
        scalar_t weightValue = weightFeat[weightRow];

        if (mode == MODE_MAX) {
          if (emb == begin || weightValue > weightFeatMax) {
            weightFeatMax = weightValue;
            maxWord = input[emb];
          }
        } else {
          if (per_sample_weights) {
            accscalar_t scaleWeightBy = static_cast<accscalar_t>(
                per_sample_weights[emb * per_sample_weights_stride]);
            weightFeatSum += scaleWeightBy * static_cast<accscalar_t>(weightValue);
          } else {
            weightFeatSum += static_cast<accscalar_t>(weightValue);
          }
        }

        bag_size_++;
        if (featureDim == 0) {
          offset2bag[emb] = bag;
        }
      }
      if (mode == MODE_MEAN) {
        if (end == begin) {
          bag_size[bag] = 0;
        } else {
          weightFeatSum = weightFeatSum / static_cast<accscalar_t>(bag_size_);
          bag_size[bag] = bag_size_;
        }
      }

      if (mode == MODE_MEAN || mode == MODE_SUM) {
        output[bag * featureSize + featureDim] = static_cast<scalar_t>(weightFeatSum);
      }
      else if (mode == MODE_MAX) {
        if (end == begin) {
          // If bag is empty, set output to 0.
          weightFeatMax = 0;
        }
        max_indices[bag * featureSize + featureDim] = maxWord;
        output[bag * featureSize + featureDim] = weightFeatMax;
      }
    }
  }
}



Tensor embedding_bag_backward_cuda_sum_avg(
                                   const Tensor &grad,
                                   const Tensor &indices,
                                   const Tensor &offset2bag,
                                   const Tensor &bag_size,
                                   int64_t num_weights,
                                   bool scale_grad_by_freq, int64_t mode,
                                   const Tensor& per_sample_weights) {

  auto grad_weight = at::zeros({num_weights, grad.size(1)}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  ptrdiff_t numel = indices.numel();

  if (numel == 0) {
    // all empty bags
    return at::zeros({num_weights, grad.size(1)}, grad.options());
  }

  int64_t stride = grad_weight.stride(0);

  auto sorted_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto orig_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor count;

  AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_bag_backward_cuda_sum_avg", [&] () {
    using device_ptr = thrust::device_ptr<index_t>;

    // Sort the inputs into sorted with the corresponding indices; we
    // don't need a stable or multidimensional sort, so just use Thrust
    // directly
    {
      sorted_indices.copy_(indices);

      auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
      auto policy = thrust::cuda::par(allocator).on(stream);

      // Fill sortedOrigIndices with sequential indices
      auto count_iter = thrust::counting_iterator<index_t>(0);
      auto orig_data = device_ptr(orig_indices.data_ptr<index_t>());
      thrust::copy(policy, count_iter, count_iter + numel, orig_data);

      // Sort; a stable sort is not required
      auto sorted_data = device_ptr(sorted_indices.data_ptr<index_t>());
      thrust::sort_by_key(policy, sorted_data, sorted_data + numel, orig_data,
                          ThrustLTOp<index_t>());
    }

    if (scale_grad_by_freq) {
      count = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

      auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
      auto policy = thrust::cuda::par(allocator).on(stream);

      // Compute an increasing sequence per unique item in sortedIndices:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 1 2 3 1 2 1 1 2
      auto sorted_data = device_ptr(sorted_indices.data_ptr<index_t>());
      auto count_data = device_ptr(count.data_ptr<index_t>());
      thrust::inclusive_scan_by_key(policy, sorted_data, sorted_data + numel,
                                    thrust::make_constant_iterator(1),
                                    count_data);

      // Take the maximum of each count per unique key in reverse:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 3 3 3 2 2 1 2 2
      thrust::inclusive_scan_by_key(
          policy, thrust::make_reverse_iterator(sorted_data + numel),
          thrust::make_reverse_iterator(sorted_data),
          thrust::make_reverse_iterator(count_data + numel),
          thrust::make_reverse_iterator(count_data + numel),
          thrust::equal_to<index_t>(), thrust::maximum<index_t>());
    }
  });
  return embedding_backward_cuda_kernel(grad, orig_indices, sorted_indices,
      count, num_weights, /* padding_idx= */ -1, scale_grad_by_freq,
      mode == MODE_MEAN, offset2bag, bag_size, per_sample_weights);
}

template <typename scalar_t, typename index_t>
__global__ void EmbeddingBag_accGradParametersKernel_max(
    index_t *max_indices, scalar_t *gradOutput,
    scalar_t *gradWeight, int64_t stride, int64_t numBags) {

  using accscalar_t = acc_type<scalar_t, true>;

  int64_t chunksPerBag = THCCeilDiv(stride, (int64_t)blockDim.x);
  int64_t numChunks = numBags * chunksPerBag;
  int64_t chunkOffset = blockIdx.x * blockDim.y + threadIdx.y;
  int64_t chunkStride = gridDim.x * blockDim.y;

  for (int64_t chunk = chunkOffset; chunk < numChunks; chunk += chunkStride) {
    int64_t featureDim = (chunk % chunksPerBag) * blockDim.x + threadIdx.x;
    if (featureDim < stride) {
      int64_t bag = chunk / chunksPerBag;

      index_t word_idx = max_indices[bag * stride + featureDim];
      if (word_idx >= 0) {
        // If bag is empty, we have max_indices[idx] set to -1 in forward.
        gpuAtomicAdd(&(gradWeight[word_idx * stride + featureDim]),
                gradOutput[bag * stride + featureDim]);
      }
    }
  }
}

Tensor embedding_bag_backward_cuda_max(const Tensor &grad,
                                   const Tensor &max_indices,
                                   int64_t num_weights) {

  auto grad_weight = at::zeros({num_weights, grad.size(1)}, grad.options());

  int64_t stride = grad_weight.stride(0);

  int64_t numBags = grad.size(0);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

#ifdef __HIP_PLATFORM_HCC__
  dim3 block = dim3(64, 4);
#else
  dim3 block = dim3(32, 8);
#endif
  int grid = 1024;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad.scalar_type(), "embedding_bag_backward_cuda_max", [&] {
        AT_DISPATCH_INDEX_TYPES(max_indices.scalar_type(), "embedding_bag_backward_cuda_max", [&] () {
          EmbeddingBag_accGradParametersKernel_max<
              scalar_t, index_t><<<grid, block, 0, stream>>>(
              max_indices.data_ptr<index_t>(), grad.data_ptr<scalar_t>(),
              grad_weight.data_ptr<scalar_t>(), stride, numBags);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  });

  return grad_weight;
}
}

// Assumes all input tensors are contiguous.
// See NOTE [ embedding_bag Native Functions ] in native_functions.yaml for details
std::tuple<Tensor, Tensor, Tensor, Tensor>
_embedding_bag_forward_only_cuda(const Tensor &weight, const Tensor &indices,
                   const Tensor &offsets, const bool scale_grad_by_freq,
                   const int64_t mode, bool sparse,
                   const Tensor& per_sample_weights,
                   bool include_last_offset) {
  return _embedding_bag_cuda(
      weight,
      indices,
      offsets,
      scale_grad_by_freq,
      mode,
      sparse,
      per_sample_weights,
      include_last_offset);
}

// Assumes all input tensors are contiguous.
// See NOTE [ embedding_bag Native Functions ] in native_functions.yaml for details
std::tuple<Tensor, Tensor, Tensor, Tensor>
_embedding_bag_cuda(const Tensor &weight, const Tensor &indices,
                   const Tensor &offsets, const bool scale_grad_by_freq,
                   const int64_t mode, bool sparse,
                   const Tensor& per_sample_weights,
                   bool include_last_offset) {
  auto indices_arg = TensorArg(indices, "indices", 1);
  checkScalarTypes("embedding_bag_cuda", indices_arg, {kLong, kInt});
  auto offsets_arg = TensorArg(offsets, "offsets", 1);
  checkScalarTypes("embedding_bag_cuda", offsets_arg, {kLong, kInt});
  checkSameType("embedding_bag_cuda", indices_arg, offsets_arg);
  auto weight_arg = TensorArg(weight, "weight", 1);
  checkSameGPU("embedding_bag_cuda", weight_arg, indices_arg);
  checkSameGPU("embedding_bag_cuda", weight_arg, offsets_arg);

  int64_t numIndices = indices.size(0);
  int64_t numBags = offsets.size(0);
  if (include_last_offset) {
    // Check https://github.com/pytorch/pytorch/issues/29019
    // We plan to add one more element in offsets, which is equal to the size of
    // indices. Currently for cuda devices, we still use the legacy
    // implementation even this flag is enabled.
    TORCH_CHECK(
        numBags >= 1, "include_last_offset: numBags should be at least 1");
    numBags -= 1;
  }
  int64_t featureSize = weight.size(1);

  auto bag_size = at::empty(offsets.sizes(), indices.options());
  auto offset2bag =
      at::empty({indices.size(0)}, indices.options()); // offset2bag = [0 0 0 0 0]

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto output = at::empty({numBags, featureSize}, weight.options());

  Tensor max_indices;

  if (mode == MODE_MAX) {
    max_indices = at::empty({numBags, featureSize}, indices.options());
  } else {
    // No need to allocate if we aren't doing a backwards pass
    max_indices = at::empty({0}, indices.options());
  }

#ifdef __HIP_PLATFORM_HCC__
  dim3 block = dim3(64, 4);
#else
  dim3 block = dim3(32, 8);
#endif
  int grid = 1024;
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, weight.scalar_type(), "embedding_bag_cuda", [&] {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "embedding_bag_cuda", [&] {
      AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_bag_cuda", [&] () {
        EmbeddingBag_updateOutputKernel<scalar_t, index_t><<<grid, block, 0, stream>>>(
            indices.data_ptr<index_t>(), offsets.data_ptr<index_t>(),
            weight.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
            offset2bag.data_ptr<index_t>(), numIndices, numBags, featureSize,
            weight.stride(0), weight.stride(1), mode, bag_size.data_ptr<index_t>(),
            mode == MODE_MAX ? max_indices.data_ptr<index_t>() : NULL,
            per_sample_weights.defined() ? per_sample_weights.data_ptr<scalar_t>() : NULL,
            per_sample_weights.defined() ? per_sample_weights.stride(0) : 0);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
    });
  });

  return std::tuple<Tensor, Tensor, Tensor, Tensor>(output, offset2bag, bag_size, max_indices);
}

Tensor _embedding_bag_dense_backward_cuda(const Tensor &grad_, const Tensor &indices,
                                   const Tensor &offsets,
                                   const Tensor &offset2bag,
                                   const Tensor &bag_size_,
                                   const Tensor &max_indices,
                                   int64_t num_weights,
                                   bool scale_grad_by_freq, int64_t mode,
                                   const Tensor& per_sample_weights) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("_embedding_bag_dense_backward_cuda");

  // indices, offsets and offset2bag are assumed having correct dtypes and
  // contiguous here due to the checks in _embedding_bag_backward in
  // EmbeddingBag.cpp.
  // Also see NOTE [ embedding_bag Native Functions ] in native_functions.yaml
  // for more details.

  Tensor grad = grad_.contiguous();
  auto indices_arg = TensorArg(indices, "indices", 1);
  auto offsets_arg = TensorArg(offsets, "offsets", 1);
  auto grad_arg = TensorArg(grad, "grad", 1);
  checkSameGPU("embedding_bag_cuda", grad_arg, offsets_arg);
  checkSameGPU("embedding_bag_cuda", grad_arg, indices_arg);


  switch (mode) {
    case MODE_SUM:
    case MODE_MEAN:
      if (mode == MODE_MEAN)
        AT_ASSERT(!per_sample_weights.defined());
      return embedding_bag_backward_cuda_sum_avg(grad, indices, offset2bag,
              bag_size_, num_weights, scale_grad_by_freq, mode, per_sample_weights);

    case MODE_MAX:
      AT_ASSERT(!per_sample_weights.defined());
      return embedding_bag_backward_cuda_max(grad, max_indices, num_weights);

    default:
      AT_ERROR(
          "Unknown mode for embedding_bag_backward_cuda ", mode);
  }
}

template <typename scalar_t>
__inline__ __device__
static scalar_t warpReduceSum(scalar_t val) {
  for (int offset = C10_WARP_SIZE/2; offset > 0; offset /= 2)
    val += WARP_SHFL_DOWN(val, offset);
  return val;
}

template <typename scalar_t, typename index_t>
__global__ static void _embedding_bag_per_sample_weights_backward_kernel(
    const scalar_t* grad, int64_t grad_stride0, int64_t grad_stride1,
    const scalar_t* weight, int64_t weight_stride0, int64_t weight_stride1,
    const index_t* indices,  // contiguous
    const index_t* offset2bag,  // contiguous
    int64_t num_samples,
    int64_t embedding_features,
    scalar_t* output) {
  using accscalar_t = acc_type<scalar_t, true>;
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int warp = idx / C10_WARP_SIZE;
  const int thread_in_warp = idx % C10_WARP_SIZE;
  const int num_warps = blockDim.x * gridDim.x / C10_WARP_SIZE;

  // Each warp is responsible for the accumulation of one sample.
  // This involves doing one dot product between grad[bag_idx] and weight[embedding_idx].
  for (int sample_idx = warp; sample_idx < num_samples; sample_idx += num_warps) {
    accscalar_t result = 0.;
    const int bag_idx = (int)offset2bag[sample_idx];
    const int embedding_idx = (int)indices[sample_idx];
    for (int feature_idx = thread_in_warp; feature_idx < embedding_features;
        feature_idx += C10_WARP_SIZE) {
      result +=
          grad[grad_stride0 * bag_idx + grad_stride1 * feature_idx] *
          weight[weight_stride0 * embedding_idx + weight_stride1 * feature_idx];
    }
    result = warpReduceSum<accscalar_t>(result);
    if (thread_in_warp == 0) {
      output[sample_idx] = result;
    }
  }
}

Tensor _embedding_bag_per_sample_weights_backward_cuda(
    const Tensor& grad,
    const Tensor& weight,  // NB: embedding table, not per_sample_weights
    const Tensor& indices,
    const Tensor& offsets,
    const Tensor& offset2bag,
    int64_t mode) {
  TORCH_CHECK(
      mode == MODE_SUM,
      "embedding_bag_backward: per_sample_weights only supported for mode='sum'");

  AT_ASSERT(grad.dim() == 2);
  auto embedding_features = grad.size(1);

  AT_ASSERT(indices.dim() == 1);
  auto num_samples = indices.size(0);

  AT_ASSERT(weight.dim() == 2);
  AT_ASSERT(weight.size(1) == embedding_features);

  const int threads_per_block = 1024;
  const int warps_per_block = threads_per_block / C10_WARP_SIZE;

  dim3 block(threads_per_block);
  dim3 grid((num_samples + warps_per_block - 1) / warps_per_block);

  auto output = at::empty({num_samples}, grad.options());

  // Early return when there is no samples in the batch. This saves unnecesary kernel
  // launch, but also prevents hipGetLastError() to complain about invalid launch args
  if (num_samples == 0) {
    return output;
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad.scalar_type(), "_embedding_bag_per_sample_weights_backward_cuda", [&]() {
      AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "_embedding_bag_per_sample_weights_backward_cuda", [&]() {
        _embedding_bag_per_sample_weights_backward_kernel<scalar_t, index_t>
          <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            grad.data_ptr<scalar_t>(), grad.stride(0), grad.stride(1),
            weight.data_ptr<scalar_t>(), weight.stride(0), weight.stride(1),
            indices.data_ptr<index_t>(),
            offset2bag.data_ptr<index_t>(),
            num_samples,
            embedding_features,
            output.data_ptr<scalar_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
    }
  );
  return output;
}

}
}
