#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

namespace at { namespace native {

void random_from_to_kernel(TensorIterator& iter, uint64_t range, int64_t base, c10::optional<Generator> gen_) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::random_from_to_kernel(iter, range, base, gen);
}

void random_full_64_bits_range_kernel(TensorIterator& iter, c10::optional<Generator> gen_) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::random_full_64_bits_range_kernel(iter, gen);
}

void random_kernel(TensorIterator& iter, c10::optional<Generator> gen_) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::random_kernel(iter, gen);
}

REGISTER_DISPATCH(random_from_to_stub, &random_from_to_kernel);
REGISTER_DISPATCH(random_stub, &random_kernel);
REGISTER_DISPATCH(random_full_64_bits_range_stub, &random_full_64_bits_range_kernel);

}} // namespace at::native
