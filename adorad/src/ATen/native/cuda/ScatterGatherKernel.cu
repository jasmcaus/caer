#include "hip/hip_runtime.h"
#include <ATen/native/TensorAdvancedIndexing.h>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/MemoryOverlap.h>

#include <ATen/native/ScatterGatherChecks.h>
#include <ATen/native/ReduceOpsUtils.h>
#include <ATen/native/TensorIterator.h>

#include <ATen/native/cuda/Loops.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>

namespace at { namespace native {

// Implement as functors since lambdas don't get optimized.
class ReduceMultiply {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t * self_data, const scalar_t * src_data) const {
    gpuAtomicMul(self_data, *src_data);
  }
};
static ReduceMultiply reduce_multiply;

class ReduceAdd {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t * self_data, const scalar_t * src_data) const {
    gpuAtomicAdd(self_data, *src_data);
  }
};
static ReduceAdd reduce_add;

class TensorAssign {
public:
  template <typename scalar_t>
  constexpr C10_DEVICE void operator() (scalar_t * self_data, const scalar_t * src_data) const {
    *self_data = *src_data;
  }
};
static TensorAssign tensor_assign;

// The kernels are implemented on an opaque,
// self-aligned type of the correct size,
// to avoid redundant kernels for different types
// of the same size.
template <int N> struct alignas(N) OpaqueType { char data[N]; };

// essentialy rewritten related to legacy::launch_kernel parts
template <int nt, int vt, typename func_t>
C10_LAUNCH_BOUNDS_2(nt, vt)
__global__ void _scatter_gather_elementwise_kernel(int N, func_t f) {
  constexpr int nv = nt * vt;
  int idx = nv * blockIdx.x + threadIdx.x;

  #pragma unroll
  for (int i = 0; i < vt; ++i) {
    if (idx < N) {
      f(idx);
      idx += nt;
    }
  }
}

template <int nt, int vt, typename func_t>
static void _launch_scatter_gather_kernel(int64_t N, const func_t& f) {
  TORCH_INTERNAL_ASSERT(N >= 0 && N <= std::numeric_limits<int32_t>::max());
  if (N == 0) {
    return;
  }

  const dim3 block(nt);
  const dim3 grid((N + block.x * vt - 1) / (block.x * vt));
  const auto stream = at::cuda::getCurrentCUDAStream();
  _scatter_gather_elementwise_kernel<nt, vt, func_t><<<grid, block, 0, stream>>>(N, f);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}


template <bool is_scatter_like, typename scalar_t>
struct _cuda_scatter_gather_internal_kernel {
  template <typename func_t>
  void operator() (
    TensorIterator& iter,
    int64_t index_size,
    int64_t index_stride,
    const func_t& f
  ) {
    if (iter.numel() == 0) {
      return;
    }

    if (!iter.can_use_32bit_indexing()) {
      for (auto& sub_iter : iter.with_32bit_indexing()) {
        _cuda_scatter_gather_internal_kernel<is_scatter_like, scalar_t>()(
          sub_iter, index_size, index_stride, f
        );
      }
      return;
    }

    char* self_ptr = (char*)iter.data_ptr(0);
    char* src_ptr = (char*)iter.data_ptr(1);
    char* index_ptr = (char*)iter.data_ptr(2);

    auto offset_calc = make_offset_calculator<3>(iter);
    auto loop = [=]C10_DEVICE(int i) {
      auto offsets = offset_calc.get(i);

      int64_t idx_dim = *(int64_t*)(index_ptr + offsets[2]);
      CUDA_KERNEL_ASSERT(idx_dim >= 0 && idx_dim < index_size
        && "index out of bounds");

      char* self_data = self_ptr + offsets[0];
      char* src_data = src_ptr + offsets[1];

      f(
        (scalar_t*)self_data + (is_scatter_like ? idx_dim * index_stride : 0),
        (scalar_t*)src_data + (is_scatter_like ? 0 : idx_dim * index_stride)
      );

    };

    _launch_scatter_gather_kernel<num_threads, thread_work_size>(iter.numel(), loop);
  }
}; // struct _cuda_scatter_fill_internal_kernel

template <bool is_scatter_like = true, bool cast_to_opaque = true>
struct cuda_scatter_gather_base_kernel {
  template <typename func_t>
  void operator()(
    Tensor& self, int64_t dim,
    const Tensor& index, const Tensor& src,
    const std::string& method_name,
    const func_t& f
  ) {
    // no-op if index is empty
    if (index.numel() == 0) {
      return;
    }
    at::assert_no_internal_overlap(self);

    dim = maybe_wrap_dim(dim, self.dim());

    scatter_gather_dtype_check(method_name, self, index, src);
    if (is_scatter_like) {
      scatter_shape_check(self, dim, index, src);
    }
    else {
      gather_shape_check(self, dim, index, src);
    }

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());
    auto self_strides = ensure_nonempty_vec(self.strides().vec());
    auto src_strides = ensure_nonempty_vec(src.strides().vec());

    // restride self and src such that
    // self.shape = src.shape = index.shape
    //
    // restride stride[dim] such that
    // if (is_scatter_like) self.stride[dim] = 0
    // else src.stride[dim] = 0
    auto self_restrided = is_scatter_like ?
        restride_dim(self, dim, index_sizes)
      : self.as_strided(index_sizes, self_strides);
    auto src_restrided = is_scatter_like ?
        src.as_strided(index_sizes, src_strides)
      : restride_dim(src, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_input(src_restrided)
      .add_input(index)
      .build();

    auto self_dim_stride = ensure_nonempty_stride(self, dim);
    auto self_dim_size = ensure_nonempty_size(self, dim);

    auto src_dim_stride = ensure_nonempty_stride(src, dim);
    auto src_dim_size = ensure_nonempty_size(src, dim);

    auto index_size = is_scatter_like ? self_dim_size : src_dim_size;
    auto index_stride = is_scatter_like ? self_dim_stride : src_dim_stride;


    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_gather_base_kernel_func", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        _cuda_scatter_gather_internal_kernel<is_scatter_like, dtype>()(
          iter, index_size, index_stride, f
        );
      }
    );
  }

  void operator()(
    Tensor& self, int64_t dim,
    const Tensor& index, const Tensor& src,
    const std::string& method_name,
    const ReduceMultiply& f
  ) {
    // no-op if index is empty
    if (index.numel() == 0) {
      return;
    }
    at::assert_no_internal_overlap(self);

    dim = maybe_wrap_dim(dim, self.dim());

    scatter_gather_dtype_check(method_name, self, index, src);
    if (is_scatter_like) {
      scatter_shape_check(self, dim, index, src);
    }
    else {
      gather_shape_check(self, dim, index, src);
    }

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());
    auto self_strides = ensure_nonempty_vec(self.strides().vec());
    auto src_strides = ensure_nonempty_vec(src.strides().vec());

    // restride self and src such that
    // self.shape = src.shape = index.shape
    //
    // restride stride[dim] such that
    // if (is_scatter_like) self.stride[dim] = 0
    // else src.stride[dim] = 0
    auto self_restrided = is_scatter_like ?
        restride_dim(self, dim, index_sizes)
      : self.as_strided(index_sizes, self_strides);
    auto src_restrided = is_scatter_like ?
        src.as_strided(index_sizes, src_strides)
      : restride_dim(src, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_input(src_restrided)
      .add_input(index)
      .build();

    auto self_dim_stride = ensure_nonempty_stride(self, dim);
    auto self_dim_size = ensure_nonempty_size(self, dim);

    auto src_dim_stride = ensure_nonempty_stride(src, dim);
    auto src_dim_size = ensure_nonempty_size(src, dim);

    auto index_size = is_scatter_like ? self_dim_size : src_dim_size;
    auto index_stride = is_scatter_like ? self_dim_stride : src_dim_stride;


    AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_gather_base_kernel_reduce_multiply", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        _cuda_scatter_gather_internal_kernel<is_scatter_like, dtype>()(
          iter, index_size, index_stride, f
        );
      }
    );
  }
}; // struct cuda_scatter_gather_base_kernel

template <typename scalar_t>
struct _cuda_scatter_fill_internal_kernel {
  template <typename func_t>
  void operator()(
    TensorIterator& iter,
    scalar_t src_val,
    int64_t index_size,
    int64_t index_stride,
    const func_t& f
  ) {
    if (iter.numel() == 0) {
      return;
    }

    if (!iter.can_use_32bit_indexing()) {
      for (auto& sub_iter : iter.with_32bit_indexing()) {
        _cuda_scatter_fill_internal_kernel<scalar_t>()(
          sub_iter, src_val, index_size, index_stride, f
        );
      }
      return;
    }

    char* self_ptr = (char*)iter.data_ptr(0);
    char* index_ptr = (char*)iter.data_ptr(1);

    auto offset_calc = make_offset_calculator<2>(iter);
    auto loop = [=]C10_DEVICE(int i) {
      auto offsets = offset_calc.get(i);

      int64_t idx_dim = *(int64_t*)(index_ptr + offsets[1]);
      CUDA_KERNEL_ASSERT(idx_dim >= 0 && idx_dim < index_size
        && "index out of bounds"
      );

      char* self_data = self_ptr + offsets[0];

      f(
        (scalar_t*)self_data + idx_dim * index_stride,
        (scalar_t*)&src_val
      );

    };

    _launch_scatter_gather_kernel<num_threads, thread_work_size>(iter.numel(), loop);
  }
}; // struct _cuda_scatter_fill_internal_kernel

template <bool cast_to_opaque = true>
struct cuda_scatter_fill_base_kernel {
  template <typename func_t>
  void operator()(
    Tensor& self, int64_t dim,
    const Tensor& index, Scalar src,
    const std::string& method_name,
    const func_t& f
  ) {
    // no-op if index is empty
    if (index.numel() == 0) {
      return;
    }
    at::assert_no_internal_overlap(self);

    dim = maybe_wrap_dim(dim, self.dim());

    scatter_gather_dtype_check(method_name, self, index);
    scatter_shape_check(self, dim, index);

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());

    // restride self such that
    // self.shape = index.shape and
    // self.stride[dim] = 0
    auto self_restrided = restride_dim(self, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_input(index)
      .build();

    auto index_size = ensure_nonempty_size(self, dim);
    auto index_stride = ensure_nonempty_stride(self, dim);

    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_fill_base_kernel_func", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        auto src_scalar_val = src.to<scalar_t>();
        auto src_val = *(dtype*)&src_scalar_val;

        _cuda_scatter_fill_internal_kernel<dtype>()(
          iter, src_val, index_size, index_stride, f
        );
      }
    );
  }

  void operator()(
    Tensor& self, int64_t dim,
    const Tensor& index, Scalar src,
    const std::string& method_name,
    const ReduceMultiply& f
  ) {
    // no-op if index is empty
    if (index.numel() == 0) {
      return;
    }
    at::assert_no_internal_overlap(self);

    dim = maybe_wrap_dim(dim, self.dim());

    scatter_gather_dtype_check(method_name, self, index);
    scatter_shape_check(self, dim, index);

    auto index_sizes = ensure_nonempty_vec(index.sizes().vec());

    // restride self such that
    // self.shape = index.shape and
    // self.stride[dim] = 0
    auto self_restrided = restride_dim(self, dim, index_sizes);

    auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self_restrided)
      .add_input(index)
      .build();

    auto index_size = ensure_nonempty_size(self, dim);
    auto index_stride = ensure_nonempty_stride(self, dim);

    AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      iter.dtype(),
      "cuda_scatter_fill_base_kernel_reduce_multiply", [&] {
        using dtype = typename std::conditional<cast_to_opaque,
          OpaqueType<sizeof(scalar_t)>, scalar_t>::type;

        auto src_scalar_val = src.to<scalar_t>();
        auto src_val = *(dtype*)&src_scalar_val;

        _cuda_scatter_fill_internal_kernel<dtype>()(
          iter, src_val, index_size, index_stride, f
        );
      }
    );
  }
}; // struct cuda_scatter_fill_base_kernel

void gather_cuda_kernel(Tensor& result, const Tensor& self, int64_t dim, const Tensor& index) {
  cuda_scatter_gather_base_kernel</*is_scatter_like=*/false>()(
    result, dim, index, self,
    "gather_out_cuda", tensor_assign);
}

void scatter_cuda_kernel(Tensor& self, int64_t dim, const Tensor& index, const Tensor& src) {
  cuda_scatter_gather_base_kernel<>()(
    self, dim, index, src,
    "scatter_cuda_", tensor_assign);
}

void scatter_fill_cuda_kernel(Tensor& self, int64_t dim, const Tensor& index, Scalar src) {
  cuda_scatter_fill_base_kernel<>()(
    self, dim, index, src,
    "scatter_fill_cuda_", tensor_assign);
}

void scatter_add_cuda_kernel(Tensor& self, int64_t dim, const Tensor& index, const Tensor& src) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("scatter_add_cuda_kernel");
  cuda_scatter_gather_base_kernel</*is_scatter_like=*/true, /*cast_to_opaque=*/false>()(
    self, dim, index, src,
    "scatter_add_cuda_", reduce_add);
}

void scatter_reduce_cuda_kernel(Tensor& self, const int64_t dim, const Tensor& index,
                               const Tensor& src, const SCATTER_GATHER_OP& reduce) {
  switch (reduce) {
  case SCATTER_GATHER_OP::REDUCE_ADD :
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
                                       "scatter_reduce_cuda_add_", reduce_add);
    break;
  case SCATTER_GATHER_OP::REDUCE_MULTIPLY :
    cuda_scatter_gather_base_kernel<true, false>()(self, dim, index, src,
                                       "scatter_reduce_cuda_multiply_", reduce_multiply);
    break;
  }
}

void scatter_scalar_reduce_cuda_kernel(Tensor& self, const int64_t dim, const Tensor& index,
                               Scalar& value, const SCATTER_GATHER_OP& reduce) {
  switch (reduce) {
  case SCATTER_GATHER_OP::REDUCE_ADD :
    cuda_scatter_fill_base_kernel<false>()(self, dim, index, value,
                                      "scatter_fill_cuda_add_", reduce_add);
    break;
  case SCATTER_GATHER_OP::REDUCE_MULTIPLY :
    cuda_scatter_fill_base_kernel<false>()(self, dim, index, value,
                                      "scatter_fill_cuda_multiply_", reduce_multiply);
    break;
  }
}


REGISTER_DISPATCH(gather_stub, &gather_cuda_kernel);
REGISTER_DISPATCH(scatter_stub, &scatter_cuda_kernel);
REGISTER_DISPATCH(scatter_fill_stub, &scatter_fill_cuda_kernel);
REGISTER_DISPATCH(scatter_add_stub, &scatter_add_cuda_kernel);
REGISTER_DISPATCH(scatter_reduce_stub, &scatter_reduce_cuda_kernel);
REGISTER_DISPATCH(scatter_scalar_reduce_stub, &scatter_scalar_reduce_cuda_kernel);

}} // namespace at::native
