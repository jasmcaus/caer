#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <ATen/NamedTensorUtils.h>
#include <ATen/native/Pool.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCNumerics.cuh>
#include <c10/macros/Macros.h>


namespace at {
namespace native {
namespace {

__device__ inline int min(int a, int b) {
  return a <= b ? a : b;
}

template <typename scalar_t>
__global__ static void max_pool3d_with_indices_single_out_frame(
  scalar_t* inputData,
  PackedTensorAccessor64<scalar_t, 4> output,
  PackedTensorAccessor64<int64_t, 4> indices,
  int itime, int iheight, int iwidth,
  int kT, int kH, int kW,
  int dT, int dH, int dW,
  int pT, int pH, int pW,
  int dilationT, int dilationH, int dilationW,
  int offsetZ)
{
  int oColumn = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow    = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame  = (blockIdx.z + offsetZ) % output.size(1); // output frame/time
  int slice   = (blockIdx.z + offsetZ) / output.size(1); // output slice/feature

  if (oRow < output.size(2) && oColumn < output.size(3))
  {
    int tStart = oFrame  * dT - pT;
    int hStart = oRow    * dH - pH;
    int wStart = oColumn * dW - pW;
    int tEnd = min(tStart + (kT - 1) * dilationT + 1, itime);
    int hEnd = min(hStart + (kH - 1) * dilationH + 1, iheight);
    int wEnd = min(wStart + (kW - 1) * dilationW + 1, iwidth);

    while(tStart < 0)
      tStart += dilationT;
    while(hStart < 0)
      hStart += dilationH;
    while(wStart < 0)
      wStart += dilationW;

    int maxIndex =  tStart * iheight * iwidth + hStart * iwidth + wStart;
    inputData += slice * itime * iheight * iwidth;

    scalar_t max = at::numeric_limits<scalar_t>::lower_bound(); // -Infinity

    for (int t = tStart; t < tEnd; t += dilationT)
    {
      for (int h = hStart; h < hEnd; h += dilationH)
      {
        for (int w = wStart; w < wEnd; w += dilationW)
        {
          int index = t * iheight * iwidth + h * iwidth + w;
          scalar_t val = inputData[index];

          if ((max < val) || THCNumerics<scalar_t>::isnan(val))
          {
            max = val;
            maxIndex = index;
          }
        }
      }
    }

    output[slice][oFrame][oRow][oColumn] = max;
    indices[slice][oFrame][oRow][oColumn] = maxIndex;
  }
}

template <typename scalar_t>
void max_pool3d_with_indices_out_frame(
  scalar_t* input_data,
  const Tensor& output,
  const Tensor& indices,
  int totalZ,
  int itime, int iheight, int iwidth,
  int otime, int oheight, int owidth,
  int kT, int kH, int kW,
  int dT, int dH, int dW,
  int pT, int pH, int pW,
  int dilationT, int dilationH, int dilationW)
{
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(cuda::ATenCeilDiv(owidth, static_cast<int>(block.x)),
              cuda::ATenCeilDiv(oheight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    max_pool3d_with_indices_single_out_frame
      <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
         input_data,
         output.packed_accessor64<scalar_t, 4>(),
         indices.packed_accessor64<int64_t, 4>(),
         itime, iheight, iwidth,
         kT, kH, kW,
         dT, dH, dW,
         pT, pH, pW,
         dilationT, dilationH, dilationW,
         offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    totalZ -= 65535;
    offsetZ += 65535;
  }
}

#undef UPDATE_OUTPUT_KERNEL_WIDTH

template <typename scalar_t>
__global__ static void max_pool3d_with_indices_backward_single_out_frame(
  scalar_t *gradInputData,
  PackedTensorAccessor64<scalar_t, 4> gradOutput,
  PackedTensorAccessor64<int64_t, 4> indices,
  int itime, int iheight, int iwidth,
  int dT, int dH, int dW,
  int pT, int pH, int pW,
  int dilationT, int dilationH, int dilationW,
  int offsetZ)
{
  int oColumn = blockIdx.x * blockDim.x + threadIdx.x;
  int oRow    = blockIdx.y * blockDim.y + threadIdx.y;
  int oFrame  = (blockIdx.z + offsetZ) % gradOutput.size(1); // output frame/time
  int slice   = (blockIdx.z + offsetZ) / gradOutput.size(1); // output slice/feature

  if (oRow < gradOutput.size(2) && oColumn < gradOutput.size(3))
  {
    int maxIndex = indices[slice][oFrame][oRow][oColumn];
    if (maxIndex != -1) {
      gpuAtomicAdd(&gradInputData[slice * itime * iheight * iwidth + maxIndex],
                gradOutput[slice][oFrame][oRow][oColumn]);
    }
  }
}

template <typename scalar_t>
void max_pool3d_with_indices_backward_out_frame(
  scalar_t *gradInputData,
  const Tensor& gradOutput,
  const Tensor& indices,
  int64_t totalZ,
  int itime, int iheight, int iwidth,
  int oheight, int owidth,
  int dT, int dH, int dW,
  int pT, int pH, int pW,
  int dilationT, int dilationH, int dilationW)
{
  int offsetZ = 0;
  dim3 block(32, 8);

  while (totalZ > 0) {
    dim3 grid(cuda::ATenCeilDiv(owidth, static_cast<int>(block.x)),
              cuda::ATenCeilDiv(oheight, static_cast<int>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);

    max_pool3d_with_indices_backward_single_out_frame
      <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
        gradInputData,
        gradOutput.packed_accessor64<scalar_t, 4>(),
        indices.packed_accessor64<int64_t, 4>(),
        itime, iheight, iwidth,
        dT, dH, dW,
        pT, pH, pW,
        dilationT, dilationH, dilationW,
        offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    totalZ -= 65535;
    offsetZ += 65535;
  }
}

void max_pool3d_with_indices_out_cuda_template(
           Tensor& output,
           Tensor& indices,
           const Tensor& input,
           IntArrayRef kernel_size,
           IntArrayRef stride,
           IntArrayRef padding,
           IntArrayRef dilation,
           bool ceil_mode)
{
  TensorArg output_arg{ output, "output", 1 };
  TensorArg indices_arg{ indices, "indices", 2 };
  TensorArg input_arg{ input, "input", 3 };

  checkAllSameGPU("max_pool3d_with_indices_out_cuda",
                  {output_arg, indices_arg, input_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK(kernel_size.size() == 1 || kernel_size.size() == 3,
    "max_pool3d: kernel_size must either be a single int, or a tuple of three ints")
  const int kT = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kH = kernel_size.size() == 1 ? kT : safe_downcast<int, int64_t>(kernel_size[1]);
  const int kW = kernel_size.size() == 1 ? kT : safe_downcast<int, int64_t>(kernel_size[2]);

  TORCH_CHECK(stride.size() == 0 || stride.size() == 1 || stride.size() == 3,
    "max_pool3d: stride must either be omitted, a single int, or a tuple of three ints")
  const int dT = stride.empty() ? kT : safe_downcast<int, int64_t>(stride[0]);
  const int dH = stride.empty() ? kH :
                 stride.size() == 1 ? dT : safe_downcast<int, int64_t>(stride[1]);
  const int dW = stride.empty() ? kW :
                 stride.size() == 1 ? dT : safe_downcast<int, int64_t>(stride[2]);

  TORCH_CHECK(padding.size() == 1 || padding.size() == 3,
    "max_pool3d: padding must be either be a single int, or a tuple of three ints");
  const int pT = safe_downcast<int, int64_t>(padding[0]);
  const int pH = padding.size() == 1 ? pT : safe_downcast<int, int64_t>(padding[1]);
  const int pW = padding.size() == 1 ? pT : safe_downcast<int, int64_t>(padding[2]);

  TORCH_CHECK(dilation.size() == 1 || dilation.size() == 3,
    "max_pool3d: dilation must be either a single int, or a tuple of three ints");
  const int dilationT = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationH = dilation.size() == 1 ? dilationT : safe_downcast<int, int64_t>(dilation[1]);
  const int dilationW = dilation.size() == 1 ? dilationT : safe_downcast<int, int64_t>(dilation[2]);

  TORCH_CHECK((input.ndimension() == 4 || input.ndimension() == 5),
    "non-empty 4D or 5D (batch mode) tensor expected for input");

  const int64_t nbatch = input.ndimension() == 5 ? input.size(-5) : 1;
  const int64_t nslices = input.size(-4);
  const int64_t itime = input.size(-3);
  const int64_t iheight = input.size(-2);
  const int64_t iwidth = input.size(-1);

  const int64_t otime = pooling_output_shape<int64_t>(itime, kT, pT, dT, dilationT, ceil_mode);
  const int64_t oheight = pooling_output_shape<int64_t>(iheight, kH, pH, dH, dilationH, ceil_mode);
  const int64_t owidth = pooling_output_shape<int64_t>(iwidth, kW, pW, dW, dilationW, ceil_mode);

  pool3d_shape_check(
    input,
    nslices,
    kT, kH, kW,
    dT, dH, dW,
    pT, pH, pW,
    dilationT, dilationH, dilationW,
    itime, iheight, iwidth,
    otime, oheight, owidth);

  if (input.ndimension() == 4) {
    output.resize_({ nslices, otime, oheight, owidth});
    indices.resize_({nslices, otime, oheight, owidth});
  }
  else {
    output.resize_({nbatch, nslices, otime, oheight, owidth});
    indices.resize_({nbatch, nslices, otime, oheight, owidth});
  }

  Tensor work_input = input.contiguous();
  Tensor work_output = output;
  Tensor work_indices = indices;
  if (input.ndimension() == 5) {
    // Collapse batch and feature dimensions.
    work_input = work_input.reshape({nbatch * nslices, itime, iheight, iwidth});
    work_output = work_output.reshape({nbatch * nslices, otime, oheight, owidth});
    work_indices = work_indices.reshape({nbatch * nslices, otime, oheight, owidth});
  }

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
    input.scalar_type(),
    "max_pool3d_with_indices_out_frame",
    [&]{
      scalar_t *input_data = work_input.data_ptr<scalar_t>();
      int64_t totalZ = otime * nslices * nbatch;

      max_pool3d_with_indices_out_frame(
        input_data, work_output, work_indices,
        totalZ,
        itime, iheight, iwidth,
        otime, oheight, owidth,
        kT, kH, kW,
        dT, dH, dW,
        pT, pH, pW,
        dilationT, dilationH, dilationW);
    }
  );
}

void max_pool3d_with_indices_backward_out_cuda_template(
           Tensor& gradInput,
           const Tensor& gradOutput,
           const Tensor& input,
           const Tensor& indices,
           IntArrayRef kernel_size,
           IntArrayRef stride,
           IntArrayRef padding,
           IntArrayRef dilation,
           bool ceil_mode)
{
  TensorArg gradInput_arg{ gradInput, "gradInput", 1 };
  TensorArg gradOutput_arg{ gradOutput, "gradOutput", 2 };
  TensorArg input_arg{ input, "input", 3 };
  TensorArg indices_arg{ indices, "indices", 4 };

  checkAllSameGPU("max_pool3d_with_indices_backward_out_cuda",
                  {gradInput_arg, gradOutput_arg, input_arg, indices_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK(kernel_size.size() == 1 || kernel_size.size() == 3,
    "max_pool3d: kernel_size must either be a single int, or a tuple of three ints")
  const int kT = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kH = kernel_size.size() == 1 ? kT : safe_downcast<int, int64_t>(kernel_size[1]);
  const int kW = kernel_size.size() == 1 ? kT : safe_downcast<int, int64_t>(kernel_size[2]);

  TORCH_CHECK(stride.size() == 0 || stride.size() == 1 || stride.size() == 3,
    "max_pool3d: stride must either be omitted, a single int, or a tuple of three ints")
  const int dT = stride.empty() ? kT : safe_downcast<int, int64_t>(stride[0]);
  const int dH = stride.empty() ? kH :
                 stride.size() == 1 ? dT : safe_downcast<int, int64_t>(stride[1]);
  const int dW = stride.empty() ? kW :
                 stride.size() == 1 ? dT : safe_downcast<int, int64_t>(stride[2]);

  TORCH_CHECK(padding.size() == 1 || padding.size() == 3,
    "max_pool3d: padding must be either be a single int, or a tuple of three ints");
  const int pT = safe_downcast<int, int64_t>(padding[0]);
  const int pH = padding.size() == 1 ? pT : safe_downcast<int, int64_t>(padding[1]);
  const int pW = padding.size() == 1 ? pT : safe_downcast<int, int64_t>(padding[2]);

  TORCH_CHECK(dilation.size() == 1 || dilation.size() == 3,
    "max_pool3d: dilation must be either a single int, or a tuple of three ints");
  const int dilationT = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationH = dilation.size() == 1 ? dilationT : safe_downcast<int, int64_t>(dilation[1]);
  const int dilationW = dilation.size() == 1 ? dilationT : safe_downcast<int, int64_t>(dilation[2]);

  TORCH_CHECK((input.ndimension() == 4 || input.ndimension() == 5),
    "non-empty 4D or 5D (batch mode) tensor expected for input");

  TORCH_CHECK((gradOutput.ndimension() == 4 || gradOutput.ndimension() == 5),
    "non-empty 4D or 5D (batch mode) tensor expected for gradOutput");

  // Resize and initialize result tensor.
  gradInput.resize_as_(input);
  gradInput.zero_();

  const int64_t nbatch = input.ndimension() == 5 ? input.size(-5) : 1;
  const int64_t nslices = input.size(-4);

  const int64_t otime = gradOutput.size(-3);
  const int64_t oheight = gradOutput.size(-2);
  const int64_t owidth = gradOutput.size(-1);

  const int64_t itime = gradInput.size(-3);
  const int64_t iheight = gradInput.size(-2);
  const int64_t iwidth = gradInput.size(-1);

  max_pool3d_backward_shape_check(
    input,
    gradOutput,
    indices,
    nslices,
    kT, kH, kW,
    dT, dH, dW,
    pT, pH, pW,
    dilationT, dilationH, dilationW,
    itime, iheight, iwidth,
    otime, oheight, owidth);

  Tensor work_grad_input = gradInput;
  Tensor work_grad_output = gradOutput.contiguous();
  Tensor work_indices = indices.contiguous();

  if (input.ndimension() == 5) {
      // Collapse batch and feature dimensions.
      work_grad_input = work_grad_input.reshape({nbatch * nslices, itime, iheight, iwidth});
      work_grad_output = work_grad_output.reshape({nbatch * nslices, otime, oheight, owidth});
      work_indices = work_indices.reshape({nbatch * nslices, otime, oheight, owidth});
  }

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, input.scalar_type(),
    "max_pool3d_with_indices_backward_out_frame",
    [&] {
      const int64_t totalZ = otime * nslices * nbatch;
      scalar_t *grad_input_data = work_grad_input.data_ptr<scalar_t>();

      max_pool3d_with_indices_backward_out_frame(
        grad_input_data, work_grad_output, work_indices,
        totalZ,
        itime, iheight, iwidth,
        oheight, owidth,
        dT, dH, dW,
        pT, pH, pW,
        dilationT, dilationH, dilationW);
    }
  );
}

} // namespace

std::tuple<Tensor&, Tensor&> max_pool3d_with_indices_out_cuda(
  Tensor& output,
  Tensor& indices,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode)
{
  max_pool3d_with_indices_out_cuda_template(
    output,
    indices,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return std::tuple<Tensor&, Tensor&>(output, indices);
}

std::tuple<Tensor, Tensor> max_pool3d_with_indices_cuda(
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode)
{
  NoNamesGuard guard;

  Tensor output = at::empty({0}, input.options());
  Tensor indices = at::empty({0}, input.options().dtype(kLong));
  max_pool3d_with_indices_out_cuda_template(
    output,
    indices,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);

  guard.reset();
  namedinference::propagate_names(output, input);
  namedinference::propagate_names(indices, input);

  return std::tuple<Tensor, Tensor>(output, indices);
}

Tensor& max_pool3d_with_indices_backward_out_cuda(
  Tensor& gradInput,
  const Tensor& gradOutput,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode,
  const Tensor& indices)
{
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("max_pool3d_with_indices_backward_out_cuda");
  max_pool3d_with_indices_backward_out_cuda_template(
    gradInput,
    gradOutput,
    input,
    indices,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return gradInput;
}

Tensor max_pool3d_with_indices_backward_cuda(
  const Tensor& gradOutput,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode,
  const Tensor& indices)
{
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("max_pool3d_with_indices_backward_cuda");
  auto gradInput = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  max_pool3d_with_indices_backward_out_cuda_template(
    gradInput,
    gradOutput,
    input,
    indices,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return gradInput;
}

} // at::native
} // at
