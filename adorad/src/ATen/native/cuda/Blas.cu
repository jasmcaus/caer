#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDABlas.h>

namespace at { namespace native {

Tensor &addmv_impl_cuda(Tensor& result, const Tensor &self, const Tensor &mat, const Tensor &vec, Scalar beta_, Scalar alpha_) {
  auto r_stride = result.stride(0);
  auto vec_stride = vec.stride(0);

  // Check for contiguity of `vec` and update `vec_stride` accordingly
  const auto vec_contiguous = vec_stride == 0 ? vec.contiguous() : vec;
  vec_stride = vec_contiguous.stride(0);

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, mat.scalar_type(), "addmv_impl_cuda", [&] {
    auto beta = beta_.to<scalar_t>();
    auto alpha = alpha_.to<scalar_t>();
    if (mat.stride(0) == 1 && mat.stride(1) >= std::max<int64_t>(1, mat.size(0))) {
      at::cuda::blas::gemv<scalar_t>('n',
        mat.size(0), mat.size(1), alpha, mat.data_ptr<scalar_t>(), mat.stride(1), vec_contiguous.data_ptr<scalar_t>(),
        vec_stride, beta, result.data_ptr<scalar_t>(), r_stride);
    }
    else if (mat.stride(1) == 1 && mat.stride(0) >= std::max<int64_t>(1, mat.size(1))) {
      at::cuda::blas::gemv<scalar_t>('t',
        mat.size(1), mat.size(0), alpha, mat.data_ptr<scalar_t>(), mat.stride(0),
        vec_contiguous.data_ptr<scalar_t>(), vec_stride, beta, result.data_ptr<scalar_t>(), r_stride);
    }
    else {
      Tensor cmat = mat.contiguous();
      at::cuda::blas::gemv<scalar_t>('t',
          mat.size(1), mat.size(0), alpha, cmat.data_ptr<scalar_t>(), cmat.stride(0),
          vec_contiguous.data_ptr<scalar_t>(), vec_stride, beta, result.data_ptr<scalar_t>(), r_stride);
    }
  });
  return result;
}

}} // namespace at::native
