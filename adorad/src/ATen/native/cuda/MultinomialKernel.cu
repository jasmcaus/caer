#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/LegacyTHFunctionsCUDA.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/LaunchUtils.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>

#include <THC/THCReduceApplyUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCNumerics.cuh>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

namespace at { namespace native {

namespace {

#define MAX_NUM_BLOCKS 200

// Normalizes the L1 norm of every row to 1; used by multinomial
template <typename scalar_t>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void renormRowsL1(scalar_t* dist, long rows, long cols) {
  extern __shared__  unsigned char my_smem[];
  scalar_t *smem = reinterpret_cast<scalar_t *>(my_smem);
  scalar_t zero = static_cast<scalar_t>(0);
  scalar_t val;
  for (int64_t row = blockIdx.x; row < rows; row += gridDim.x) {
    scalar_t sum = static_cast<scalar_t>(0);
    for (int64_t col = threadIdx.x; col < cols; col += blockDim.x) {
      val = dist[row * cols + col];
      CUDA_KERNEL_ASSERT(!THCNumerics<scalar_t>::lt(val, zero)); // ! < 0 for NaN handling
      sum = sum + val;
    }

    sum = reduceBlock(smem, blockDim.x, sum, ReduceAdd<scalar_t>(), zero);
    if (threadIdx.x == 0) {
      CUDA_KERNEL_ASSERT(!THCNumerics<scalar_t>::lt(val, zero)); // ! < 0 for NaN handling
      smem[0] = sum;
    }
    __syncthreads();

    sum = smem[0];
    if (sum > zero) {
      for (int64_t col = threadIdx.x; col < cols; col += blockDim.x) {
        dist[row * cols + col] = dist[row * cols + col] / sum;
      }
    }
  }
}

void renormRows(Tensor& t) {
  TORCH_CHECK(t.dim() == 2);
  int64_t rows = t.size(0);
  int64_t cols = t.size(1);

  auto props = at::cuda::getCurrentDeviceProperties();
  CUDA_KERNEL_ASSERT(props != NULL);
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(t.scalar_type(), "renormRows_cuda", [&] {
    renormRowsL1<scalar_t>
        <<<grid, block, block.x * sizeof(scalar_t),
        at::cuda::getCurrentCUDAStream()>>>(t.data_ptr<scalar_t>(),
            rows, cols);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

template <typename scalar_t>
__device__ int binarySearchForMultinomial(scalar_t* cumdist,
                                          scalar_t* dist,
                                          int size,
                                          scalar_t val) {
  int start = 0;
  int end = size;
  // cumdist[size - 1] = 0 => all zero prob dist
  CUDA_KERNEL_ASSERT(cumdist[size - 1] > static_cast<scalar_t>(0));

  while (end - start > 0) {
    int mid = start + (end - start) / 2;

    scalar_t midVal = cumdist[mid];
    if (midVal < val) {
      start = mid + 1;
    } else {
      end = mid;
    }
  }

  if (start == size) {
    // No probability mass or precision problems; just return the
    // first non-zero element by setting start to size-1 here,
    // the code below will move it to the last non-zero probability
    // this actually can happen when the random number is 1
    // (github pytorch issue #4858).
    start = size - 1;
  }

  while(start >= 1 && dist[start] == 0) start--;

  return start;
}

template <typename scalar_t>
__global__ void
sampleMultinomialWithReplacement(PhiloxCudaState philox_args,
                                 int totalSamples,
                                 int64_t* dest,
                                 int64_t distributions,
                                 int categories,
                                 scalar_t* normDistPrefixSum,
                                 scalar_t* normDist) {
  // At the moment, each warp computes one sample value in the binary
  // search due to divergence. It seems possible to compute multiple
  // values and limit divergence though later on.

  auto seeds = at::cuda::philox::unpack(philox_args);

  // global index formula for 2D grid of 1D blocks
  int idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  // The block determines the distribution for which we generate a point
  for (int64_t curDist = blockIdx.y;
       curDist < distributions;
       curDist += gridDim.y) {
    for (int sample = blockIdx.x*blockDim.x + threadIdx.x;
         sample < totalSamples; sample += blockDim.x*gridDim.x) {

      //we are losing 3 out of 4 generated numbers but it's ok
      //this kernel is not very efficient anyway
      auto rand = hiprand_uniform4(&state);
      scalar_t r = static_cast<scalar_t>(rand.x);

      // Find the bucket that a uniform sample lies in
      int choice = binarySearchForMultinomial<scalar_t>(
          normDistPrefixSum + curDist * categories,
          normDist + curDist * categories,
          categories,
          r);

      dest[curDist * totalSamples + sample] = choice;

    }
  }
}

template <typename scalar_t, typename accscalar_t>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void
sampleMultinomialOnce(int64_t* dest,
                      int64_t distributions,
                      int categories,
                      scalar_t* sampled,
                      scalar_t* dist,
                      int stride_dist,        // dist->stride(0)
                      int stride_categories   // dist->stride(1)
) {
  extern __shared__  unsigned char my_smem[];
  __shared__ bool found;

  // Shared Memory hold blockdim.x T for holding the cumulative sum,
  // blockDim.x AccT for normalizing the probabilities,
  scalar_t *smem = reinterpret_cast<scalar_t *>(my_smem);
  accscalar_t *asmem = reinterpret_cast<accscalar_t *>(&my_smem[blockDim.x * sizeof(scalar_t)]);

  accscalar_t accZero = static_cast<accscalar_t>(0);
  scalar_t zero = static_cast<scalar_t>(0);

  for (int64_t curDist = blockIdx.x;
       curDist < distributions; curDist += gridDim.x) {
    // Each block handles one distribution
    // First pass, find the total sum of the distribution
    accscalar_t sum = accZero;
    scalar_t val;
    for (int cat = threadIdx.x; cat < categories; cat += blockDim.x) {
      val = dist[curDist * stride_dist + cat * stride_categories];
      CUDA_KERNEL_ASSERT(val >= zero);
      CUDA_KERNEL_ASSERT(!THCNumerics<scalar_t>::isinf(val));
      CUDA_KERNEL_ASSERT(!THCNumerics<scalar_t>::isnan(val));
      sum = sum + static_cast<accscalar_t>(val);
    }

    // threadIdx.x == 0 has the sum value from this
    sum = reduceBlock(asmem, blockDim.x, sum, ReduceAdd<accscalar_t>(), accZero);

    // Broadcast sum and sample value
    if (threadIdx.x == 0) {
      // Make sure the sum of our distribution didn't overflow
      CUDA_KERNEL_ASSERT(!THCNumerics<accscalar_t>::isinf(sum));
      CUDA_KERNEL_ASSERT(sum > accZero);

      asmem[0] = sum;
      smem[0] = sampled[curDist];
    }
    __syncthreads();

    sum = asmem[0];
    scalar_t sample = smem[0];
    __syncthreads();

    if (sum == accZero) {
      // Choose the first element
      if (threadIdx.x == 0) {
        dest[curDist] = 0;
      }

      continue;
    }

    int chunks = (categories + (int)blockDim.x - 1) / blockDim.x;
    scalar_t prevHighProb = zero;
    found = false;

    for (int chunk = 0; chunk < chunks && !found; ++chunk) {
      // All threads in bounds load a value
      int cat = chunk * blockDim.x + threadIdx.x;

      accscalar_t a_dist_val = cat < categories ?
                               static_cast<accscalar_t>(dist[curDist * stride_dist + cat * stride_categories]) / sum :
                               accZero;
      scalar_t dist_val = static_cast<scalar_t>(a_dist_val);

      smem[threadIdx.x] = dist_val;
      __syncthreads();

      // Perform an inclusive prefix sum of the shared memory contents
      for (int offset = 1; offset < blockDim.x; offset *= 2) {
        scalar_t val = zero;

        if (threadIdx.x >= offset) {
          val = smem[threadIdx.x - offset] + smem[threadIdx.x];
        }

        __syncthreads();
        if (threadIdx.x >= offset) {
          smem[threadIdx.x] = val;
        }
        __syncthreads();
      }

      // Each thread will check to see if the sample falls in its
      // bucket
      scalar_t curBucket = smem[threadIdx.x] + prevHighProb;
      scalar_t prevBucket =
          threadIdx.x == 0 ? prevHighProb :
          smem[threadIdx.x - 1] + prevHighProb;
      bool inBucket =
          (cat < categories) &&
          (!(sample >= curBucket) &&
           (sample >= prevBucket) &&
           (dist_val > zero));

      if (inBucket) {
        // We're done; we have the sample
        // Torch indices are 1-based
        dest[curDist] = cat;
        found = true;
      }

      // Store the previous scan's high value for future use
      prevHighProb = prevHighProb + smem[blockDim.x - 1];

      __syncthreads();
    }

    if (threadIdx.x == 0 && !found) {
      // This should address a rare bug where we don't select a valid index. This likely occurs when
      // due to floating point arithmetic rounding errors, our cumulative sum does not add up to 1, but
      // and our uniform sample is greater than this value. In this case we likely have unitialized memory
      // in dest[curDist]. So basically we will loop through the distribution and pick the largest index
      // where the distribution is non-zero. This is obviously terribly inefficient, but due to the
      // rarity in which this occurs, this should not be an issue.
      for (int cat = categories - 1; cat >= 0; --cat) {
        if (dist[curDist * stride_dist + cat * stride_categories] > zero) {
          dest[curDist] = cat;
          break;
        }
      }
    }
  }
}

void multinomial_kernel_impl(Tensor& result, const Tensor& self, const int64_t n_sample, const bool with_replacement, c10::optional<Generator> generator) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(generator, cuda::detail::getDefaultCUDAGenerator());

  int inputSize = self.dim();
  int64_t numDist =
      inputSize == 1 ? 1 : self.size(0);
  int numCategories =
      inputSize == 1 ? self.size(0) : self.size(1);

  // Restructure data for 2d
  auto self_v = inputSize == 1 ? self.view({numDist, numCategories}) : self;

  result.resize_({numDist, n_sample});

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self_v.scalar_type(), "multinomial_kernel_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto props = at::cuda::getCurrentDeviceProperties();
    CUDA_KERNEL_ASSERT(props != NULL);
    int numSM = props->multiProcessorCount;
    int maxThreads = props->maxThreadsPerBlock;
    int maxShared = props->sharedMemPerBlock;
    int requiredShared = (numCategories < maxThreads ? numCategories : maxThreads)
                         * (sizeof(scalar_t) + sizeof(accscalar_t));

    if (n_sample == 1 && maxShared >= requiredShared) {
      // Optimized allocation-free implementation
      // To exploit greater parallelism for the sampling, generate the
      // Uniform random samples in a separate kernel launch, into
      // temporarily allocated memory. The device RNG is thread-limited
      Tensor sampled = native::empty_cuda({numDist, n_sample}, optTypeMetaToScalarType(self_v.options().dtype_opt()),
                                          self_v.options().layout_opt(), self_v.options().device_opt(),
                                          self_v.options().pinned_memory_opt());
      at::native::uniform_(sampled, 0.0, 1.0, generator);

      dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
      dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

      sampleMultinomialOnce<scalar_t, accscalar_t>
          <<<grid, block,
          requiredShared,
          at::cuda::getCurrentCUDAStream()>>>(
              result.data_ptr<int64_t>(),
                  numDist,
                  numCategories,
                  sampled.data_ptr<scalar_t>(),
                  self_v.data_ptr<scalar_t>(),
                  self_v.stride(0),
                  self_v.stride(1)
          );
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      // Generic, slow implementation with memory allocations

      // For sampling without replacement, we modify the distribution
      // for subsequent samples in this space
      Tensor origDist = native::empty_like(self_v, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      origDist.copy_(self_v);

      Tensor normDist = native::empty_like(self_v, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

      Tensor prefixSum = native::empty_like(self_v, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

      // Renorm along rows
      normDist.copy_(origDist);
      renormRows(normDist);

      // Prefix sum along rows
      at::_cumsum_out(prefixSum, normDist, 1);

      PhiloxCudaState rng_engine_inputs;

      if (with_replacement) {
        // Binary search is warp divergent (so effectively we're running
        // with just a single thread), but for better utilization,
        // we need each block to have at least 4 warps.
        dim3 block(128);

        // Each block will generate a sample from one
        // distribution concurrently.
        int grid_y=std::min<int>(numDist, at::cuda::getCurrentDeviceProperties()->maxGridSize[1]);
        dim3 grid((n_sample-1)/block.x+1, grid_y);
        {
          // See Note [Acquire lock when using random generators]
          std::lock_guard<std::mutex> lock(gen->mutex_);

          // each thread generates a single sample for (numdist/numblocks.y) distributions, however, since we have to use
          // hiprand_uniform4 (See Note [Register spilling in hiprand call for CUDA < 10]),
          // offset is 4 times that.
          auto offset = ((numDist-1)/grid.y+1)*4;
          rng_engine_inputs = gen->philox_cuda_state(offset);
        }
        // Sample with replacement

        sampleMultinomialWithReplacement
            <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
                rng_engine_inputs,
                n_sample,
                result.data_ptr<int64_t>(),
                numDist, numCategories,
                prefixSum.data_ptr<scalar_t>(),
                normDist.data_ptr<scalar_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    }
  });

  if (inputSize == 1) {
    result.resize_({n_sample});
  }
}
}

REGISTER_DISPATCH(multinomial_stub, &multinomial_kernel_impl);

}}
