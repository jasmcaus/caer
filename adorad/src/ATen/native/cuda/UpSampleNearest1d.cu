#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>

namespace at {
namespace native {
namespace {

#define MAX_THREADS 512

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest1d_out_frame(
    const scalar_t* input,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_w,
    size_t dst_dim_w,
    scalar_t* output,
    float scale_factor) {
  int dst_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_w)
    return;

  int c = (dst_idx / dst_dim_w) % dim_c;

  int dst_x = dst_idx % dst_dim_w;
  int src_x = nearest_neighbor_compute_source_index(scale_factor, dst_x, src_dim_w);

  int src_idx = c * src_dim_w + src_x;
  int src_stride = dim_c * src_dim_w;
  int dst_stride = dim_c * dst_dim_w;

  for (int b = 0; b < dim_b; b++) {
    output[dst_idx] = input[src_idx];
    src_idx += src_stride;
    dst_idx += dst_stride;
  }
}

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
// Backward operation
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest1d_backward_out_frame(
    const scalar_t* grad_o,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_w,
    size_t dst_dim_w,
    scalar_t* grad_i,
    float scale_factor) {

  int dst_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_w)
    return;

  int c = (dst_idx / (dst_dim_w)) % dim_c;

  int dst_x = dst_idx % dst_dim_w;
  int src_x = nearest_neighbor_bw_compute_source_index(scale_factor, dst_x, src_dim_w);
  int src_x_up = nearest_neighbor_bw_compute_source_index(scale_factor, dst_x+1, src_dim_w+1);

  for (int b = 0; b < dim_b; b++) {
    accscalar_t grad = 0;
    int src_idx = b * dim_c * src_dim_w + c * src_dim_w + src_x;
    for (int x = src_x; x < src_x_up; x++) {
      grad += grad_o[src_idx++];
    }
    grad_i[dst_idx] = grad;
    dst_idx += dim_c * dst_dim_w;
  }
}

static void upsample_nearest1d_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    IntArrayRef output_size,
    c10::optional<double> scales) {
  TensorArg input_arg{input_, "input_", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_nearest1d_out_cuda", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 1,
      "It is expected output_size equals to 1, but got size ",
      output_size.size());

  int output_width = output_size[0];

  int nbatch = input_.size(0);
  int channels = input_.size(1);
  int input_width = input_.size(2);

  upsample_1d_shape_check(
      input_, Tensor(), nbatch, channels, input_width, output_width);

  AT_ASSERT(input_width > 0 && output_width > 0);

  Tensor input = input_.contiguous();
  output.resize_({input.size(0), input.size(1), output_width});

  if (input.numel() == 0) {
    return;
  }

  // upsample_1d_shape_check makes sure `nbatch != 0`
  unsigned int n = output.numel() / nbatch;
  dim3 bdim{std::min<unsigned int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
  dim3 gdim{cuda::ATenCeilDiv(n, bdim.x)};
  // safe check for int32 indexing; implicitly restrict launch config for kernel
  TORCH_CHECK(output.numel() <= std::numeric_limits<int32_t>::max());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::Byte, input.scalar_type(), "upsample_nearest1d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.data_ptr<scalar_t>();
        auto odata = output.data_ptr<scalar_t>();

        const float scale_factor = compute_scales_value<float>(scales, input_width, output_width);

        upsample_nearest1d_out_frame<scalar_t><<<gdim, bdim, 0, stream>>>(
            idata, nbatch, channels, input_width, output_width, odata, scale_factor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_nearest1d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    c10::optional<double> scales) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      "upsample_nearest1d_backward_out_cuda_template",
      {grad_output_arg, grad_input_arg});

  TORCH_CHECK(
      output_size.size() == 1,
      "It is expected output_size equals to 1, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 3,
      "It is expected input_size equals to 3, but got size ",
      input_size.size());

  int output_width = output_size[0];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_width = input_size[2];

  upsample_1d_shape_check(
      Tensor(), grad_output_, nbatch, channels, input_width, output_width);

  Tensor grad_output = grad_output_.contiguous();
  grad_input.resize_({nbatch, channels, input_width});

  if (grad_input.numel() == 0) {
    return;
  }

  // upsample_1d_shape_check makes sure `nbatch != 0`
  unsigned int n = grad_input.numel() / nbatch;
  dim3 bdim{std::min<unsigned int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
  dim3 gdim{cuda::ATenCeilDiv(n, bdim.x)};
  // safe check for int32 indexing; implicitly restrict launch config for kernel
  TORCH_CHECK(grad_input.numel() <= std::numeric_limits<int32_t>::max());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::Byte, grad_output.scalar_type(), "upsample_nearest1d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.data_ptr<scalar_t>();
        auto odata = grad_output.data_ptr<scalar_t>();

        const float scale_factor = compute_scales_value_backwards<float>(scales, output_width, input_width);

        upsample_nearest1d_backward_out_frame<scalar_t, accscalar_t>
            <<<gdim, bdim, 0, stream>>>(
                odata, nbatch, channels, output_width, input_width, idata, scale_factor);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

TORCH_IMPL_FUNC(upsample_nearest1d_out_cuda) (
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    c10::optional<double> scales) {
  upsample_nearest1d_out_cuda_template(output, input, output_size, scales);
}

TORCH_IMPL_FUNC(upsample_nearest1d_backward_out_cuda) (
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    c10::optional<double> scales) {
  upsample_nearest1d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, scales);
}

using at::native::upsample::compute_output_size;
using at::native::upsample_cuda::get_scale_value;

Tensor upsample_nearest1d_cuda(
    const Tensor& input,
    c10::optional<IntArrayRef> output_size,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto osize = compute_output_size(input.sizes(), output_size, scale_factors);
  auto scale_w = get_scale_value(scale_factors, 0);
  upsample_nearest1d_out_cuda_template(output, input, osize, scale_w);
  return output;
}

Tensor upsample_nearest1d_backward_cuda(
    const Tensor& grad_output,
    c10::optional<IntArrayRef> output_size,
    IntArrayRef input_size,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto osize = compute_output_size(input_size, output_size, scale_factors);
  auto scale_w = get_scale_value(scale_factors, 0);
  auto grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_nearest1d_backward_out_cuda_template(
      grad_input, grad_output, osize, input_size, scale_w);
  return grad_input;
}

} // namespace native
} // namespace at
