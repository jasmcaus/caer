#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/execution_policy.h>

#include <tuple>
#include <iterator>
#include <thrust/adjacent_difference.h>
#include <thrust/unique.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>

namespace at {
namespace native{

namespace {


template <
  typename policy_t, typename scalar_t,
  typename equal_t, typename not_equal_t
>
std::tuple<Tensor, Tensor, int64_t> compute_unique(
  const policy_t &policy,
  scalar_t *data,
  int64_t num_inp,
  const Tensor &sorted_indices,
  const bool return_inverse,
  const bool return_counts,
  TensorOptions options,
  equal_t equal,
  not_equal_t not_equal
) {

  // inverse indices
  Tensor inverse_indices;
  if (!return_inverse || num_inp == 0) {
    inverse_indices = at::empty({0}, options);
  } else {
    TORCH_CHECK(sorted_indices.defined(),
      "return_inverse is set to true, but sorted_indices is undefined. Send a bug report!");
    const int64_t *sorted_indices_ptr = sorted_indices.data_ptr<int64_t>();
    Tensor inv_loc = at::empty({num_inp}, options);
    inverse_indices = at::empty({num_inp}, options);
    int64_t* inv_loc_ptr = inv_loc.data_ptr<int64_t>();
    int64_t* inverse_indices_ptr = inverse_indices.data_ptr<int64_t>();
    thrust::adjacent_difference(policy, data, data + num_inp, inv_loc_ptr, not_equal);
    inv_loc[0] = 0;
    thrust::inclusive_scan(policy, inv_loc_ptr, inv_loc_ptr + num_inp, inv_loc_ptr);
    thrust::scatter(policy, inv_loc_ptr, inv_loc_ptr + num_inp, sorted_indices_ptr, inverse_indices_ptr);
  }

  // unique and count
  Tensor counts = at::empty({0}, options);
  int64_t num_out;
  if (!return_counts) {
    num_out = thrust::unique(policy, data, data + num_inp, equal) - data;
  } else {
    Tensor range = at::arange(0, num_inp + 1, options);
    int64_t *range_ptr = range.data_ptr<int64_t>();
    num_out = thrust::unique_by_key(policy, data, data + num_inp, range_ptr, equal).first - data;
    range[num_out] = num_inp;
    counts.resize_(num_out);
    int64_t* counts_ptr = counts.data_ptr<int64_t>();
    thrust::adjacent_difference(policy, range_ptr + 1, range_ptr + num_out + 1, counts_ptr);
  }

  AT_CUDA_CHECK(hipGetLastError());
  return std::tuple<Tensor, Tensor, int64_t>(inverse_indices, counts, num_out);
}

template <typename scalar_t>
std::tuple<Tensor, Tensor, Tensor> unique_cuda_template(
  const Tensor& self,
  const bool consecutive,
  const bool return_inverse,
  const bool return_counts
) {

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto options = self.options().dtype(kLong);
  Tensor output = self.clone(at::MemoryFormat::Contiguous).reshape(-1);
  int64_t num_inp = output.numel();
  scalar_t* output_data = output.data_ptr<scalar_t>();

  Tensor sorted_indices;
  if (!return_inverse) {
    if (!consecutive) {
      thrust::sort(policy, output_data, output_data + num_inp);
    }
  } else {
    sorted_indices = at::arange(0, num_inp, options);
    if (!consecutive) {
      int64_t *sorted_indices_ptr = sorted_indices.data_ptr<int64_t>();
      thrust::sort_by_key(policy, output_data, output_data + num_inp, sorted_indices_ptr);
    }
  }

  Tensor inverse_indices, counts;
  int64_t num_out;
  std::tie(inverse_indices, counts, num_out) = compute_unique(
    policy, output_data, num_inp, sorted_indices,
    return_inverse, return_counts, options,
    thrust::equal_to<scalar_t>(),
    thrust::not_equal_to<scalar_t>()
  );
  output.resize_(num_out);

  if (return_inverse) {
      inverse_indices.resize_(self.sizes());
  }

  return std::tuple<Tensor, Tensor, Tensor>(output, inverse_indices, counts);
}

template <typename scalar_t>
std::tuple<Tensor, Tensor, Tensor> unique_dim_cuda_template(
  const Tensor& self,
  const int64_t dim,
  const bool consecutive,
  const bool return_inverse,
  const bool return_counts
) {

  /**
    * The idea for implementing this is basically the same as unique.
    * For unique_dim, we are taking the unique with respect to a index
    * tensor, but during the processes, we override the compare and equal
    * operator by checking the data underlying it instead. After the
    * algorithm, we would use index_select to map the resulting indicies
    * to the result on the actual data.
    */

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto sizes = self.sizes().vec();
  // check how many zero dimensions exist
  auto num_zero_dims = std::count(sizes.begin(), sizes.end(), 0);

  // tensor is not well formed as it has 0 sized dimensions
  if (self.size(dim) == 0){
    TORCH_CHECK(
        num_zero_dims == 1,
        "Number of zero sized dimensions is more than one, so unique cannot be applied ")
    Tensor output = at::empty({0}, self.options());
    Tensor inverse_indices =
        at::empty({0}, self.options().dtype(kLong));
    Tensor counts = at::empty({0}, self.options().dtype(kLong));

    return std::make_tuple(output, inverse_indices, counts);
  }

  TORCH_CHECK(num_zero_dims == 0,
    "There are 0 sized dimensions, and they aren't selected, so unique cannot be applied");

  int64_t num_inp = self.size(dim);
  auto options = self.options().dtype(kLong);
  Tensor input_flat = self.transpose(dim, 0).contiguous().view({num_inp, -1});
  int64_t n = input_flat.size(1);
  scalar_t *input_flat_ptr = input_flat.data_ptr<scalar_t>();

  Tensor indices = at::arange(0, num_inp, options);
  int64_t *indices_data = indices.data_ptr<int64_t>();
  if (!consecutive) {
    thrust::sort(policy, indices_data, indices_data + num_inp,
      [=] __device__ (int64_t a, int64_t b) -> bool {
        for (int64_t i = 0; i < n; ++i) {
          scalar_t lhs = input_flat_ptr[i + a * n];
          scalar_t rhs = input_flat_ptr[i + b * n];
          if (lhs < rhs) {
            return true;
          } else if (lhs > rhs) {
            return false;
          }
        }
        return false;
      }
    );
  }

  Tensor inverse_indices, counts;
  int64_t num_out;
  std::tie(inverse_indices, counts, num_out) = compute_unique(
    policy, indices_data, num_inp, indices,
    return_inverse, return_counts, options,
    [=] __device__ (int64_t a, int64_t b) -> bool {
      for (int64_t i = 0; i < n; ++i) {
        scalar_t lhs = input_flat_ptr[i + a * n];
        scalar_t rhs = input_flat_ptr[i + b * n];
        if (lhs != rhs) {
          return false;
        }
      }
      return true;
    },
    [=] __device__ (int64_t a, int64_t b) -> int64_t {
      for (int64_t i = 0; i < n; ++i) {
        scalar_t lhs = input_flat_ptr[i + a * n];
        scalar_t rhs = input_flat_ptr[i + b * n];
        if (lhs != rhs) {
          return 1;
        }
      }
      return 0;
    }
  );
  indices.resize_(num_out);

  return std::tuple<Tensor, Tensor, Tensor>(self.index_select(dim, indices), inverse_indices, counts);
}

} // namespace


std::tuple<Tensor, Tensor>
_unique_cuda(const Tensor& self, const bool sorted, const bool return_inverse) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique", [&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    Tensor output, inverse;
    std::tie(output, inverse, std::ignore) = unique_cuda_template<scalar_t>(self, false, return_inverse, false);
    return std::make_tuple(output, inverse);
  });
}

std::tuple<Tensor, Tensor, Tensor>
_unique2_cuda(const Tensor& self, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique", [&] {
    // The current CUDA implementation of unique always sort due to the
    // lack of hashtable implementation in thrust
    return unique_cuda_template<scalar_t>(self, false, return_inverse, return_counts);
  });
}

std::tuple<Tensor, Tensor, Tensor>
unique_dim_cuda(const Tensor& self, const int64_t dim, const bool sorted, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique_dim", [&] {
    return unique_dim_cuda_template<scalar_t>(self, dim, false, return_inverse, return_counts);
  });
}

std::tuple<Tensor, Tensor, Tensor>
unique_dim_consecutive_cuda(const Tensor& self, const int64_t dim, const bool return_inverse, const bool return_counts) {
  return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique_dim", [&] {
    return unique_dim_cuda_template<scalar_t>(self, dim, true, return_inverse, return_counts);
  });
}

std::tuple<Tensor, Tensor, Tensor>
unique_consecutive_cuda(const Tensor& self, const bool return_inverse, const bool return_counts, c10::optional<int64_t> dim) {
  if (!dim.has_value()) {
    return AT_DISPATCH_ALL_TYPES_AND2(kBool, kHalf, self.scalar_type(), "unique", [&] {
      // The current CUDA implementation of unique always sort due to the
      // lack of hashtable implementation in thrust
      return unique_cuda_template<scalar_t>(self, true, return_inverse, return_counts);
    });
  }
  return unique_dim_consecutive_cuda(self, dim.value(), return_inverse, return_counts);
}

}  // namespace native
}  // namespace at
