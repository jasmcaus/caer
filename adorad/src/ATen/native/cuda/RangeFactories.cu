#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/detail/FunctionTraits.h>
#include <cmath>
#include <limits>

#define GPU_LAMBDA __device__ __host__

namespace {

constexpr int num_threads = C10_WARP_SIZE * 2;
constexpr int thread_work_size = 1;
constexpr int block_work_size = thread_work_size * num_threads;

template<typename index_t, typename func_t>
C10_LAUNCH_BOUNDS_1(num_threads)
__global__ void elementwise_kernel_with_index(index_t N, func_t f, typename function_traits<func_t>::result_type *data) {
  #pragma unroll
  for (int i = 0; i < thread_work_size; i++) {
    index_t idx = block_work_size * blockIdx.x + num_threads * i + threadIdx.x;
    if (idx < N) {
      data[idx] = f(idx);
    }
  }
}

template<typename func_t>
void gpu_kernel_with_index(at::Tensor &output, func_t f) {
  int64_t N = output.numel();
  if (N == 0) {
    return;
  }
  int64_t grid = (N + block_work_size - 1) / block_work_size;
  auto stream = at::cuda::getCurrentCUDAStream();
  using scalar_t = typename function_traits<func_t>::result_type;
  if (N <= std::numeric_limits<int>::max()) {
    elementwise_kernel_with_index<int><<<grid, num_threads, 0, stream>>>(N, f, output.data_ptr<scalar_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  } else {
    elementwise_kernel_with_index<int64_t><<<grid, num_threads, 0, stream>>>(N, f, output.data_ptr<scalar_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

}  // namespace

namespace at {
namespace native {

Tensor& linspace_cuda_out(Tensor& result, Scalar start, Scalar end, c10::optional<int64_t> optional_steps) {
  const auto steps = optional_steps.value_or(100);
  TORCH_CHECK(steps >= 0, "number of steps must be non-negative");

  if (!optional_steps.has_value()) {
    TORCH_WARN_ONCE(
      "Not providing a value for linspace's steps is deprecated and will "
      "throw a runtime error in a future release. This warning will appear "
      "only once per process.");
  }

  if (result.numel() != steps) {
    result.resize_({steps});
  }
  bool is_contiguous = result.is_contiguous();
  Tensor r = !is_contiguous ? at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

  if (steps == 0) {
    // skip
  } else if (steps == 1) {
    r.fill_(start);
  } else if (isIntegralType(r.scalar_type(), 0)) {
    AT_DISPATCH_INTEGRAL_TYPES(r.scalar_type(), "linspace_cuda", [&]() {
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      // Cast `end` and `start` to `float`, since range can be larger than scalar_t for integral types
      float step = (static_cast<float>(scalar_end) - static_cast<float>(scalar_start)) / (steps - 1);
      const int64_t halfway = steps / 2;
      gpu_kernel_with_index(r, [scalar_start, scalar_end, steps, step, halfway]GPU_LAMBDA(int64_t ind) -> scalar_t {
        if (ind < halfway) {
          return scalar_start + (step * ind);
        }

        return scalar_end - step * (steps - ind - 1);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, r.scalar_type(), "linspace_cuda", [&]() {
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      scalar_t step = (scalar_end - scalar_start) / static_cast<scalar_t>(steps - 1);
      const int64_t halfway = steps / 2;
      gpu_kernel_with_index(r, [scalar_start, scalar_end, steps, step, halfway]GPU_LAMBDA(int64_t ind) -> scalar_t {
        if (ind < halfway) {
          return scalar_start + (step * ind);
        }

        return scalar_end - step * (steps - ind - 1);
      });
    });
  }

  if (!is_contiguous) {
    result.copy_(r);
  }

  return result;
}

Tensor& logspace_cuda_out(Tensor& result, Scalar start, Scalar end, c10::optional<int64_t> optional_steps, double base) {
  const auto steps = optional_steps.value_or(100);
  TORCH_CHECK(steps >= 0, "number of steps must be non-negative");

  if (!optional_steps.has_value()) {
    TORCH_WARN_ONCE(
      "Not providing a value for logspace's steps is deprecated and will "
      "throw a runtime error in a future release. This warning will appear "
      "only once per process.");
  }

  if (result.numel() != steps) {
    result.resize_({steps});
  }
  bool is_contiguous = result.is_contiguous();
  Tensor r = !is_contiguous ? at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

  if (steps == 0) {
    // skip
  } else if (steps == 1) {
    r.fill_(std::pow(base, start.to<double>()));
  } else if (isIntegralType(r.scalar_type(), 0)) {
    AT_DISPATCH_INTEGRAL_TYPES(r.scalar_type(), "logspace_cuda", [&]() {
      float scalar_base = static_cast<float>(base); // Use float to avoid promotion to double
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      float step = static_cast<float>(scalar_end - scalar_start) / (steps - 1);
      const int64_t halfway = steps / 2;
      gpu_kernel_with_index(r, [scalar_start, scalar_end, scalar_base, steps, step, halfway]GPU_LAMBDA(int64_t ind) -> scalar_t {
        if (ind < halfway) {
          return std::pow(scalar_base, scalar_start + step * ind);
        }
        return std::pow(scalar_base, scalar_end - step * (steps - ind - 1));
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, r.scalar_type(), "logspace_cuda", [&]() {
      scalar_t scalar_base = static_cast<scalar_t>(base);
      scalar_t scalar_start = start.to<scalar_t>();
      scalar_t scalar_end = end.to<scalar_t>();
      scalar_t step = (scalar_end - scalar_start) / static_cast<scalar_t>(steps - 1);
      const int64_t halfway = steps / 2;
      gpu_kernel_with_index(r, [scalar_start, scalar_end, scalar_base, steps, step, halfway]GPU_LAMBDA(int64_t ind) -> scalar_t {
        if (ind < halfway) {
          return std::pow(scalar_base, scalar_start + step * ind);
        }
        return std::pow(scalar_base, scalar_end - step * (steps - ind - 1));
      });
    });
  }

  if (!is_contiguous) {
    result.copy_(r);
  }

  return result;
}

Tensor& range_cuda_out(Tensor& result, Scalar start, Scalar end, Scalar step) {
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, result.scalar_type(), "range_cuda", [&]() {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto xstart = start.to<accscalar_t>();
    auto xend = end.to<accscalar_t>();
    auto xstep = step.to<accscalar_t>();

    TORCH_CHECK(xstep > 0 || xstep < 0, "step must be nonzero");
    TORCH_CHECK(std::isfinite(static_cast<double>(xstart)) &&
             std::isfinite(static_cast<double>(xend)),
             "unsupported range: ", xstart, " -> ", xend);
    TORCH_CHECK(((xstep > 0) && (xend >= xstart)) || ((xstep < 0) && (xend <= xstart)),
             "upper bound and larger bound inconsistent with step sign");
    int64_t size = static_cast<int64_t>(((xend - xstart) / xstep) + 1);

    if (result.numel() != size) {
      result.resize_({size});
    }
    bool is_contiguous = result.is_contiguous();
    Tensor r = !is_contiguous ?  at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

    gpu_kernel_with_index(r, [xstart, xstep]GPU_LAMBDA(int64_t ind) -> scalar_t {
        accscalar_t inc = xstep * static_cast<accscalar_t>(ind);
        accscalar_t val = xstart + inc;
        return static_cast<scalar_t>(val);
    });

    if(!is_contiguous) {
      result.copy_(r);
    }

  });

  return result;
}

Tensor& arange_cuda_out(Tensor& result, Scalar start, Scalar end, Scalar step) {
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, result.scalar_type(), "arange_cuda", [&]() {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto xstart = start.to<accscalar_t>();
    auto xend = end.to<accscalar_t>();
    auto xstep = step.to<accscalar_t>();

    // we use double precision for (start - end) / step
    // to compute size_d for consistency across devices.
    // The problem with using accscalar_t is that accscalar_t might be float32 on gpu for a float32 scalar_t,
    // but double on cpu for the same,
    // and the effective output size starts differing on CPU vs GPU because of precision issues, which
    // we dont want.
    // the corner-case we do want to take into account is int64_t, which has higher precision than double
    double size_d;
    if (std::is_same<scalar_t, int64_t>::value) {
      size_d = std::ceil(static_cast<double>(end.to<accscalar_t>() - start.to<accscalar_t>())
                          / step.to<accscalar_t>());
    } else {
      size_d = std::ceil(static_cast<double>(end.to<double>() - start.to<double>())
                          / step.to<double>());
    }

    TORCH_CHECK(xstep > 0 || xstep < 0, "step must be nonzero");
    TORCH_CHECK(std::isfinite(static_cast<double>(xstart)) &&
              std::isfinite(static_cast<double>(xend)),
              "unsupported range: ", xstart, " -> ", xend);
    TORCH_CHECK(((xstep > 0) && (xend >= xstart)) || ((xstep < 0) && (xend <= xstart)),
              "upper bound and larger bound inconsistent with step sign");

    TORCH_CHECK(size_d >= 0 && size_d <= static_cast<double>(std::numeric_limits<int64_t>::max()),
              "invalid size, possible overflow?");
    int64_t size = static_cast<int64_t>(size_d);
    int64_t numel = result.numel();

    if (numel != size) {
      if(numel > 0){
        TORCH_WARN("The number of elements in the out tensor of shape ", result.sizes(),
                    " is ", numel, " which does not match the computed number of elements ", size,
                    ". Note that this may occur as a result of rounding error. "
                    "The out tensor will be resized to a tensor of shape (", size, ",).");
      }
      result.resize_({size});
    }
    bool is_contiguous = result.is_contiguous();
    Tensor r = !is_contiguous ? at::empty_like(result, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : result;

    gpu_kernel_with_index(r, [xstart, xstep]GPU_LAMBDA(int64_t ind) -> scalar_t {
        accscalar_t inc = xstep * static_cast<accscalar_t>(ind);
        accscalar_t val = xstart + inc;
        return static_cast<scalar_t>(val);
    });

    if(!is_contiguous) {
      result.copy_(r);
    }
  });

  return result;
}

}} // namespace at::native
