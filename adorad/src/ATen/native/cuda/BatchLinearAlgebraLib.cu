#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/PinnedMemoryAllocator.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDASolver.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/LinearAlgebraUtils.h>
#include <ATen/native/cuda/MiscUtils.h>
#include <ATen/native/cuda/BatchLinearAlgebraLib.h>

#ifdef USE_CUSOLVER

namespace at {
namespace native {

inline static Tensor column_major_identity_matrix_like(const Tensor& self) {
  auto size = self.sizes();
  auto size_slice = IntArrayRef(size.data(), size.size()-1);
  return at::ones(size_slice, self.options()).diag_embed().transpose(-2, -1);
}

template <typename scalar_t>
inline static void _apply_single_inverse_helper(scalar_t* self_ptr, scalar_t* self_inv_ptr, int* ipiv_ptr, int* info_ptr, int n) {
  // self_inv_ptr should already be an identity matrix

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  at::cuda::solver::getrf<scalar_t>(handle, n, n, self_ptr, n, ipiv_ptr, info_ptr);
  at::cuda::solver::getrs<scalar_t>(handle, n, n, self_ptr, n, ipiv_ptr, self_inv_ptr, n, info_ptr + 1);
}

template <typename scalar_t>
static void apply_batched_inverse_lib(Tensor& self, Tensor& self_inv, Tensor& infos) {
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");
  const int n = cuda_int_cast(self.size(-2), "self.size(-2)");

  auto self_data = self.data_ptr<scalar_t>();
  auto self_mat_stride = matrixStride(self);
  auto self_inv_data = self_inv.data_ptr<scalar_t>();
  auto self_inv_mat_stride = matrixStride(self_inv);

  auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

  if (use_loop_launch(batch_size, n)) {
    int* p_infos = infos.data_ptr<int>();
    auto main_stream = at::cuda::getCurrentCUDAStream();

    at::cuda::CUDAEvent main_event;
    main_event.record(main_stream);

    for (int64_t i = 0; i < batch_size; i++) {
      auto stream = at::cuda::getStreamFromPool();
      at::cuda::CUDAStreamGuard guard(stream);

      main_event.block(stream);

      auto dataPtr = allocator.allocate(sizeof(int) * n);
      int* pivot = reinterpret_cast<int*>(dataPtr.get());
      _apply_single_inverse_helper<scalar_t>(
        &self_data[i * self_mat_stride], &self_inv_data[i * self_inv_mat_stride], pivot, p_infos + i * 2, n);

      at::cuda::CUDAEvent finished;
      finished.record(stream);
      finished.block(main_stream);
    }
  } else {
    // cublas batched kernels require input be "device array of device pointers"
    Tensor self_array = at::arange(
      reinterpret_cast<long>(self_data),
      reinterpret_cast<long>(&self_data[(batch_size-1) * self_mat_stride]) + 1,
      static_cast<long>(self_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));
    Tensor self_inv_array = at::arange(
      reinterpret_cast<long>(self_inv_data),
      reinterpret_cast<long>(&self_inv_data[(batch_size-1) * self_inv_mat_stride]) + 1,
      static_cast<long>(self_inv_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));

    auto dataPtr = allocator.allocate(sizeof(int)*batch_size*n);
    int* ipiv_array = reinterpret_cast<int*>(dataPtr.get());

    Tensor _info1 = at::zeros({batch_size}, self.options().dtype(at::kInt));
    Tensor _info2 = at::zeros({batch_size}, self.options().dtype(at::kInt));

    at::cuda::blas::getrfBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), n,
      ipiv_array, _info1.data_ptr<int>(), batch_size);

    at::cuda::blas::getriBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), n,
      ipiv_array, _info2.data_ptr<int>(), batch_size, reinterpret_cast<scalar_t**>(self_inv_array.data_ptr()));

    infos = at::stack({_info1, _info2}, 1);
  }
}

template <typename scalar_t>
static void apply_single_inverse_lib(const Tensor& self, Tensor& self_inv, Tensor& info) {
  int n = cuda_int_cast(self.size(-2), "self.size(-2)");

  Tensor ipiv = at::empty({n}, self.options().dtype(at::kInt));

  _apply_single_inverse_helper<scalar_t>(
    self.data_ptr<scalar_t>(), self_inv.data_ptr<scalar_t>(), ipiv.data_ptr<int>(), info.data_ptr<int>(), n);
}

Tensor _inverse_helper_cuda_lib(const Tensor& self) {
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  Tensor self_inv_working_copy = column_major_identity_matrix_like(self_working_copy);
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");

  if (self.dim() > 2 && batch_size > 1) {
    Tensor infos = at::zeros({batchCount(self) * 2}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        self_working_copy, self_inv_working_copy, infos);
    });
    batchCheckErrors(infos, "inverse_cuda", false, 2);
  } else {
    Tensor info = at::zeros({2}, self.options().dtype(at::kInt));
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(self_working_copy, self_inv_working_copy, info);
    });
    batchCheckErrors(info, "inverse_cuda", false, 2);
  }

  return self_inv_working_copy;
}

}} // namespace at::native

#endif  // USE_CUSOLVER
