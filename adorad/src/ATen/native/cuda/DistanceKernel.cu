#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/Exceptions.h>
#include <THC/THCTensorMathReduce.cuh>
#include <math.h>

#include <ATen/native/Distance.h>

#include <c10/macros/Macros.h>

namespace at { namespace native {

namespace {

static const int forward_threads = 256;

template <typename scalar_t>
static __forceinline__ __device__ scalar_t device_sqrt(scalar_t val);

template <>
__forceinline__ __device__ float device_sqrt(float val) {
  return ::sqrtf(val);
}

template <>
__forceinline__ __device__ double device_sqrt(double val) {
  return ::sqrt(val);
}

template <typename scalar_t>
struct dists {

  static __forceinline__ __device__ scalar_t sign(scalar_t val) {
    return (0 < val) - (val < 0);
  }

  // Zero norm
  struct zero {
    static __forceinline__ __device__ void inc(scalar_t& agg, const scalar_t diff, const scalar_t p) { agg += diff != 0.0; }
    static __forceinline__ __device__ scalar_t finish(const scalar_t agg, const scalar_t p) { return agg; }
    static __forceinline__ __device__ void agg(scalar_t& update, const scalar_t other) { update += other; }
  };

  // One norm
  struct one {
    static __forceinline__ __device__ void inc(scalar_t& agg, const scalar_t diff, const scalar_t p) { agg += diff; }
    static __forceinline__ __device__ scalar_t finish(const scalar_t agg, const scalar_t p) { return agg; }
    static __forceinline__ __device__ void agg(scalar_t& update, const scalar_t other) { update += other; }
    static __forceinline__ __device__ scalar_t backward(const scalar_t diff, const scalar_t grad, const scalar_t dist, const scalar_t p) { return grad * sign(diff); }
  };

  // Special case backward when p is less than two
  struct lt_two {
    static __forceinline__ __device__ scalar_t backward(const scalar_t diff, const scalar_t grad, const scalar_t dist, const scalar_t p) {
      return (dist == 0.0 || (diff == 0.0 && p < 1)) ? 0 : (sign(diff) * std::pow(std::abs(diff), p - 1) * grad / std::pow(dist, p - 1));
    }
  };

  // Two norm
  struct two {
    static __forceinline__ __device__ void inc(scalar_t& agg, const scalar_t diff, const scalar_t p) { agg += diff * diff; }
    static __forceinline__ __device__ scalar_t finish(const scalar_t agg, const scalar_t p) { return device_sqrt<scalar_t>(agg); }
    static __forceinline__ __device__ void agg(scalar_t& update, const scalar_t other) { update += other; }
    static __forceinline__ __device__ scalar_t backward(const scalar_t diff, const scalar_t grad, const scalar_t dist, const scalar_t p) { return dist == 0.0 ? 0 : grad * diff / dist; }
  };

  // General p norm
  struct p {
    static __forceinline__ __device__ void inc(scalar_t& agg, const scalar_t diff, const scalar_t p) { agg += std::pow(diff, p); }
    static __forceinline__ __device__ scalar_t finish(const scalar_t agg, const scalar_t p) { return std::pow(agg, static_cast<scalar_t>(1) / p); }
    static __forceinline__ __device__ void agg(scalar_t& update, const scalar_t other) { update += other; }
    static __forceinline__ __device__ scalar_t backward(const scalar_t diff, const scalar_t grad, const scalar_t dist, const scalar_t p) { return dist == 0.0 ? 0 : diff * std::pow(std::abs(diff), p - 2) * grad / std::pow(dist, p - 1); }
  };

  // Inf norm
  struct inf {
    static __forceinline__ __device__ void inc(scalar_t& agg, const scalar_t diff, const scalar_t p) { if (diff > agg) { agg = diff; } }
    static __forceinline__ __device__ scalar_t finish(const scalar_t agg, const scalar_t p) { return agg; }
    static __forceinline__ __device__ void agg(scalar_t& update, const scalar_t other) { if (other > update) { update = other; } }
    static __forceinline__ __device__ scalar_t backward(const scalar_t diff, const scalar_t grad, const scalar_t dist, const scalar_t p) { return grad * sign(diff) * (std::abs(diff) == dist); }
  };

};

template <typename scalar_t, typename F>
__device__ static inline scalar_t reduce_agg(scalar_t agg) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    F::agg(agg, WARP_SHFL_DOWN(agg, offset));
  }

  __shared__ scalar_t shared[forward_threads];
  int lane = threadIdx.x % warpSize;
  int warp_id = threadIdx.x / warpSize;
  if (lane == 0) {
    shared[warp_id] = agg;
  }

  __syncthreads();
  agg = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0;
  if (warp_id == 0) {
    for (int offset = blockDim.x / warpSize / 2; offset > 0; offset /= 2) {
      F::agg(agg, WARP_SHFL_DOWN(agg, offset));
    }
  }
  return agg;
}

template <typename scalar_t, typename F>
__global__ static void pdist_kernel_cuda_impl(scalar_t * result, const scalar_t * self, const int64_t n, const int64_t m, const scalar_t p,
                                              const double n2, const double n2_squared_minus_1) {
  const int64_t k = blockIdx.x;
  const int stride = blockDim.x;

  // The -1 accounts for floating point truncation issues
  int64_t i = static_cast<int64_t>((n2 - device_sqrt<double>(n2_squared_minus_1 - 2 * k)));
  int64_t j = k - n * i + i * (i + 1) / 2 + i + 1;

  const scalar_t * const start = self + i * m;
  const scalar_t * const end = start + m;
  const scalar_t * a = start + threadIdx.x;
  const scalar_t * b = self + j * m + threadIdx.x;
  scalar_t agg = 0.0;
  for (; a < end; a += stride, b += stride) {
    F::inc(agg, std::abs(*a - *b), p);
  }

  agg = reduce_agg<scalar_t, F>(agg);
  if (threadIdx.x == 0) {
    result[k] = F::finish(agg, p);
  }
}

template <typename scalar_t, typename F>
__global__ static void cdist_backward_kernel_cuda_impl(scalar_t * buffer, const scalar_t * grad, const scalar_t * x1, const scalar_t * x2, const scalar_t * dist, int64_t gs,
                                                       const scalar_t p, const int64_t r1, const int64_t r2, const int64_t m, const int64_t count, const int64_t r_size, const int64_t l1_size, const int64_t l2_size) {
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int init = blockIdx.x * blockDim.x + threadIdx.x;
  if (y >= count || init >= m) {
    return;
  }
  const int l = y / r_size;
  const int k = y % r_size;
  const int stride = blockDim.x * gridDim.x;
  const int l_size = r_size * m;

  int64_t i = k / r2;
  int64_t j = k % r2;

  const scalar_t grad_k = grad[y];
  const scalar_t dist_k = dist[y];

  const scalar_t * const start = x1 + l * l1_size + i * m;
  const scalar_t * const end = start + m;
  const scalar_t * self_i = start + init;
  const scalar_t * self_j = x2 + l * l2_size + j * m + init;

  scalar_t * buff_i = buffer + l * l_size + (r1 * j + i) * m + init;

  for (; self_i < end; self_i += stride, self_j += stride, buff_i += stride) {
    const scalar_t res = F::backward(*self_i - *self_j, grad_k, dist_k, p);
    *buff_i = res;
  }
}

template <typename scalar_t, typename F>
__global__ static void pdist_backward_kernel_cuda_impl(scalar_t * buffer, const scalar_t * grad, const scalar_t * self, const scalar_t * dist, int64_t gs, const int64_t n, const int64_t m, const int64_t combs, const scalar_t p,
                                                       const double n2, const double n2_squared_minus_1) {
  const int64_t k = blockIdx.x * blockDim.x + threadIdx.x;
  const int init = blockIdx.y * blockDim.y + threadIdx.y;
  const int stride = blockDim.y * gridDim.y;

  if (k >= combs) {
    return;
  }

  // The -1 accounts for floating point truncation issues
  int64_t i = static_cast<int64_t>((n2 - device_sqrt<double>(n2_squared_minus_1 - 2 * k)));
  int64_t j = k - n * i + i * (i + 1) / 2 + i + 1;
  int64_t ib = j - i - 1;
  int64_t jb = n - 2 - i;

  const scalar_t grad_k = grad[k * gs];
  const scalar_t dist_k = dist[k];

  const scalar_t * const start = self + i * m;
  const scalar_t * const end = start + m;
  const scalar_t * self_i = start + init;
  const scalar_t * self_j = self + j * m + init;
  scalar_t * buff_i = buffer + (ib * n + i) * m + init;
  scalar_t * buff_j = buffer + (jb * n + j) * m + init;
  for (; self_i < end; self_i += stride, self_j += stride, buff_i += stride, buff_j += stride) {
    const scalar_t res = F::backward(*self_i - *self_j, grad_k, dist_k, p);
    *buff_i = res;
    *buff_j = -res;
  }
}

template <typename scalar_t, typename F>
__global__ static void cdist_kernel_cuda_impl(scalar_t * result, const scalar_t * x1, const scalar_t * x2,
    const scalar_t p, const int64_t r1, const int64_t r2, const int64_t m, const int64_t r_size, const int64_t l1_size, const int64_t l2_size) {
  const int64_t l = blockIdx.x / r_size;
  const int64_t k = blockIdx.x % r_size;
  const int64_t i = k / r2;
  const int64_t j = k % r2;
  const int stride = blockDim.x;

  const scalar_t * const start = x1 + l * l1_size + i * m;
  const scalar_t * const end = start + m;
  const scalar_t * a = start + threadIdx.x;
  const scalar_t * b = x2 + l * l2_size + j * m + threadIdx.x;

  scalar_t agg = 0.0;
  for (; a < end; a += stride, b += stride) {
    F::inc(agg, std::abs(*a - *b), p);
  }
  agg = reduce_agg<scalar_t, F>(agg);
  if (threadIdx.x == 0) {
    result[blockIdx.x] = F::finish(agg, p);
  }
}

void cdist_kernel_impl(Tensor& result, const Tensor& x1, const Tensor& x2, double p) {
  const int64_t r1 = x1.size(-2);
  const int64_t r2 = x2.size(-2);
  const int64_t m = x1.size(-1);
  const int64_t r_size = r1 * r2;
  const int64_t l1_size = r1 * m;
  const int64_t l2_size = r2 * m;
  const dim3 grid(result.numel());
  const dim3 block(forward_threads);

  AT_DISPATCH_FLOATING_TYPES(x1.scalar_type(), "cdist_cuda", [&] {
    if (p == 0.0) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::zero><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), p, r1, r2, m, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p == 1.0) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::one><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), p, r1, r2, m, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p == 2.0) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::two><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), p, r1, r2, m, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (std::isinf(p)) {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), p, r1, r2, m, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      cdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::p><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), p, r1, r2, m, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  });
}

void pdist_forward_kernel_impl(Tensor& result, const Tensor& self, double p) {
  const dim3 grid(result.numel());
  const dim3 block(forward_threads);
  int64_t n = self.size(0);
  int64_t m = self.size(1);
  // https://github.com/pytorch/pytorch/issues/15511 demonstrated we need to do
  // some math in fp64 -- this is just minimizing the amount of fp64 math we do on the device.
  const double n2 = n - .5;
  const double n2_squared_minus_1 = n2 * n2 - 1;

  AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "pdist_cuda", [&] {
    if (p == 0.0) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::zero><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), n, m, p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p == 1.0) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::one><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), n, m, p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p == 2.0) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::two><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), n, m, p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (std::isinf(p)) {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), n, m, p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      pdist_kernel_cuda_impl<scalar_t, dists<scalar_t>::p><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(result.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), n, m, p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  });
}

void pdist_backward_kernel_impl(Tensor& result, const Tensor& grad, const Tensor& self, const double p, const Tensor& dist) {
  if (p == 0.0 || grad.numel() == 0 || self.numel() == 0) {
    result.fill_(0);
    return;
  }

  const int64_t n = result.size(0);
  int64_t m = self.size(1);
  const int block_x = 16;
  // NB: be careful with changing block_y; as it's currently written, grid_y is limited to be 2^16.
  // block_y of 64 gives us max pdist dim1 of 2**24
  const int block_y = 64;
  const int grid_x = (dist.numel() + block_x - 1) / block_x;
  const int grid_y = (m + block_y * 8 - 1) / (block_y * 8);
  const dim3 grid(grid_x, grid_y);
  const dim3 block(block_x, block_y);
  // https://github.com/pytorch/pytorch/issues/15511 demonstrated we need to do
  // some math in fp64 -- this is just minimizing the amount of fp64 math we do on the device.
  const double n2 = n - .5;
  const double n2_squared_minus_1 = n2 * n2 - 1;

  Tensor buffer = at::empty({n - 1, result.size(0), result.size(1)}, result.options());
  AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "pdist_cuda_backward", [&] {
    if (p == 1.0) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::one><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(), grad.stride(0), n, m, dist.numel(), p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p < 2.0) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::lt_two><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(), grad.stride(0), n, m, dist.numel(), p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p == 2.0) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::two><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(), grad.stride(0), n, m, dist.numel(), p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (std::isinf(p)) {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(), grad.stride(0), n, m, dist.numel(), p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      pdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::p><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), self.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(), grad.stride(0), n, m, dist.numel(), p, n2, n2_squared_minus_1);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  });

  at::sum_out(result, buffer, 0);
}

void cdist_backward_kernel_impl(Tensor& result, const Tensor& grad, const Tensor& x1, const Tensor& x2, const double p, const Tensor& dist) {
  if (p == 0.0 || grad.numel() == 0 || x1.numel() == 0 || x2.numel() == 0) {
    result.fill_(0);
    return;
  }

  const int64_t r1 = x1.size(-2);
  const int64_t r2 = x2.size(-2);
  const int64_t m = x1.size(-1);
  int64_t batch = x1.dim() > 2 ? x1.size(0) : 1;
  const int block_x = 64;
  const int block_y = 16;
  const int grid_x = (m + block_x * 8 - 1) / (block_x * 8);
  const int grid_y = (dist.numel() + block_y - 1) / block_y;

  const dim3 grid(grid_x, grid_y);
  const dim3 block(block_x, block_y);

  const int64_t count = dist.numel();
  const int64_t r_size = r1 * r2;
  const int64_t l1_size = r1 * m;
  const int64_t l2_size = r2 * m;
  //current implementation supports only gradient that can be collapsed to 1D. However, to avoid checking this assumption,
  //we call grad.contiguous() before backward, so stride is guaranteed to be 1
  const int64_t gs = 1;

  Tensor buffer = (x1.dim() > 2) ? at::empty({batch, r2, r1, m}, result.options()) : at::empty({r2, r1, m}, result.options());
  AT_DISPATCH_FLOATING_TYPES(result.scalar_type(), "cdist_cuda_backward", [&] {
    if (p == 1.0) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::one><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(),
      grad.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(),
      gs, p, r1, r2, m, count, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p < 2.0) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::lt_two><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(),
      grad.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(),
      gs, p, r1, r2, m, count, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (p == 2.0) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::two><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(),
      grad.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(),
      gs, p, r1, r2, m, count, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else if (std::isinf(p)) {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::inf><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(),
      grad.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(),
      gs, p, r1, r2, m, count, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      cdist_backward_kernel_cuda_impl<scalar_t, dists<scalar_t>::p><<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(buffer.data_ptr<scalar_t>(),
      grad.data_ptr<scalar_t>(), x1.data_ptr<scalar_t>(), x2.data_ptr<scalar_t>(), dist.data_ptr<scalar_t>(),
      gs, p, r1, r2, m, count, r_size, l1_size, l2_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  });

  if (x1.dim() > 2) {
    at::sum_out(result, buffer, 1);
  } else {
    at::sum_out(result, buffer, 0);
  }

}


} // anonymous namespace

REGISTER_DISPATCH(pdist_forward_stub, &pdist_forward_kernel_impl);
REGISTER_DISPATCH(pdist_backward_stub, &pdist_backward_kernel_impl);
REGISTER_DISPATCH(cdist_stub, &cdist_kernel_impl);
REGISTER_DISPATCH(cdist_backward_stub, &cdist_backward_kernel_impl);

}} // at::native
