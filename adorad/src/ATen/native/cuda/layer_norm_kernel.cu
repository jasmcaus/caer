#include "hip/hip_runtime.h"
#include <ATen/native/layer_norm.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/block_reduce.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <c10/cuda/CUDAMathCompat.h>

namespace at {
namespace native {

namespace {

constexpr int kCUDANumThreads = 256;
constexpr int kColwiseReduceTileSize = 32;

template <typename T>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N,
    T eps,
    const T* X,
    T* mean,
    T* rstd) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC m_shared[C10_WARP_SIZE];
  __shared__ T_ACC v_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    sum1 += static_cast<T_ACC>(X[index]);
    sum2 += static_cast<T_ACC>(X[index]) * static_cast<T_ACC>(X[index]);
  }
  sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, m_shared);
  sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, v_shared);
  if (threadIdx.x == 0) {
    const T_ACC scale = T_ACC(1) / static_cast<T_ACC>(N);
    sum1 *= scale;
    sum2 = c10::cuda::compat::max(sum2 * scale - sum1 * sum1, T_ACC(0));
    mean[i] = sum1;
    rstd[i] = c10::cuda::compat::rsqrt(sum2 + static_cast<T_ACC>(eps));
  }
}

template <typename T>
__global__ void LayerNormForwardCUDAKernel(
    int64_t N,
    const T* X,
    const T* mean,
    const T* rstd,
    const T* gamma,
    const T* beta,
    T* Y) {
  using T_ACC = acc_type<T, true>;
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
            static_cast<T_ACC>(rstd[i]) * gamma_v +
        beta_v;
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    acc_type<T, true>* ds,
    acc_type<T, true>* db) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC ds_shared[C10_WARP_SIZE];
  __shared__ T_ACC db_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    sum1 +=
        static_cast<T_ACC>(dY[index]) * static_cast<T_ACC>(X[index]) * gamma_v;
    sum2 += static_cast<T_ACC>(dY[index]) * gamma_v;
  }
  sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, ds_shared);
  sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, db_shared);
  if (threadIdx.x == 0) {
    ds[i] = sum1;
    db[i] = sum2;
  }
}

template <typename T>
__global__ void ComputeGradientFusedParamsCUDAKernel(
    int64_t M,
    int64_t N,
    const T* mean,
    const T* rstd,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    acc_type<T, true>* c1,
    acc_type<T, true>* c2) {
  using T_ACC = acc_type<T, true>;
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < M) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(N);
    const T_ACC a = (db[index] * static_cast<T_ACC>(mean[index]) - ds[index]) *
        static_cast<T_ACC>(rstd[index]) * static_cast<T_ACC>(rstd[index]) *
        static_cast<T_ACC>(rstd[index]) * s;
    c1[index] = a;
    c2[index] =
        -(a * static_cast<T_ACC>(mean[index]) +
          db[index] * static_cast<T_ACC>(rstd[index]) * s);
  }
}

template <typename T>
__global__ void LayerNormBackwardCUDAKenrel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    const T* a,
    const acc_type<T, true>* b,
    const acc_type<T, true>* c,
    T* dX) {
  using T_ACC = acc_type<T, true>;
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    dX[index] =
        static_cast<T_ACC>(a[i]) * static_cast<T_ACC>(dY[index]) * gamma_v +
        b[i] * static_cast<T_ACC>(X[index]) + c[i];
  }
}

template <typename T>
__global__ void GammaBetaBackwardSimpleCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T* mean,
    const T* rstd,
    T* dg,
    T* db) {
  using T_ACC = acc_type<T, true>;
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N) {
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dg == nullptr ? T_ACC(0)
                            : static_cast<T_ACC>(dY[index]) *
              (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
              static_cast<T_ACC>(rstd[i]);
      sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index]);
    }
    if (dg != nullptr) {
      dg[j] = sum1;
    }
    if (db != nullptr) {
      db[j] = sum2;
    }
  }
}

template <typename T>
__global__ void GammaBetaBackwardCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T* mean,
    const T* rstd,
    T* dg,
    T* db) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC g_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  __shared__ T_ACC b_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (j < N) {
    for (int64_t i = threadIdx.y; i < M; i += blockDim.y * 2) {
      const int64_t i1 = i;
      const int64_t i2 = i + blockDim.y;
      const int64_t index1 = i1 * N + j;
      const int64_t index2 = i2 * N + j;
      dg_sum1 += dg == nullptr ? T_ACC(0)
                               : static_cast<T_ACC>(dY[index1]) *
              (static_cast<T_ACC>(X[index1]) - static_cast<T_ACC>(mean[i1])) *
              static_cast<T_ACC>(rstd[i1]);
      db_sum1 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index1]);
      if (i2 < M) {
        dg_sum2 += dg == nullptr ? T_ACC(0)
                                 : static_cast<T_ACC>(dY[index2]) *
                (static_cast<T_ACC>(X[index2]) - static_cast<T_ACC>(mean[i2])) *
                static_cast<T_ACC>(rstd[i2]);
        db_sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index2]);
      }
    }
  }
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = cuda_utils::WarpReduceSum(sum1);
  sum2 = cuda_utils::WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = cuda_utils::WarpReduceSum(sum1);
  sum2 = cuda_utils::WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
}

template <typename T>
void LayerNormKernelImplInternal(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    T eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  DCHECK_EQ(X.numel(), M * N);
  DCHECK(!gamma.defined() || gamma.numel() == N);
  DCHECK(!beta.defined() || beta.numel() == N);
  const T* X_data = X.data_ptr<T>();
  const T* gamma_data = gamma.defined() ? gamma.data_ptr<T>() : nullptr;
  const T* beta_data = beta.defined() ? beta.data_ptr<T>() : nullptr;
  T* Y_data = Y->data_ptr<T>();
  T* mean_data = mean->data_ptr<T>();
  T* rstd_data = rstd->data_ptr<T>();
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  RowwiseMomentsCUDAKernel<T>
      <<<M, cuda_utils::kCUDABlockReduceNumThreads, 0, hip_stream>>>(
          N, eps, X_data, mean_data, rstd_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  LayerNormForwardCUDAKernel<T><<<M, kCUDANumThreads, 0, hip_stream>>>(
      N, X_data, mean_data, rstd_data, gamma_data, beta_data, Y_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void LayerNormKernelImpl(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    double eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
      X.scalar_type(), "LayerNormKernelImpl", [&]() {
        LayerNormKernelImplInternal<scalar_t>(
            X, gamma, beta, M, N, static_cast<scalar_t>(eps), Y, mean, rstd);
      });
}

template <typename T>
void LayerNormBackwardKernelImplInternal(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t M,
    int64_t N,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  using T_ACC = acc_type<T, true>;
  DCHECK_EQ(dY.numel(), M * N);
  DCHECK_EQ(X.numel(), M * N);
  DCHECK_EQ(mean.numel(), M);
  DCHECK_EQ(rstd.numel(), M);
  DCHECK(!gamma.defined() || gamma.numel() == N);
  const T* dY_data = dY.template data_ptr<T>();
  const T* X_data = X.template data_ptr<T>();
  const T* mean_data = mean.template data_ptr<T>();
  const T* rstd_data = rstd.template data_ptr<T>();
  const T* gamma_data =
      gamma.defined() ? gamma.template data_ptr<T>() : nullptr;
  T* dX_data = dX->defined() ? dX->template data_ptr<T>() : nullptr;
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  if (dX_data != nullptr) {
    const auto kAccType = (X.scalar_type() == kHalf || X.scalar_type() == kBFloat16) ? kFloat : X.scalar_type();
    Tensor ds = at::empty({M}, X.options().dtype(kAccType));
    Tensor db = at::empty({M}, X.options().dtype(kAccType));
    Tensor scale = at::empty({M}, X.options().dtype(kAccType));
    Tensor bias = at::empty({M}, X.options().dtype(kAccType));
    T_ACC* ds_data = ds.template data_ptr<T_ACC>();
    T_ACC* db_data = db.template data_ptr<T_ACC>();
    T_ACC* scale_data = scale.template data_ptr<T_ACC>();
    T_ACC* bias_data = bias.template data_ptr<T_ACC>();
    ComputeInternalGradientsCUDAKernel<T>
        <<<M, cuda_utils::kCUDABlockReduceNumThreads, 0, hip_stream>>>(
            N, dY_data, X_data, gamma_data, ds_data, db_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    const int64_t B = (M + kCUDANumThreads - 1) / kCUDANumThreads;
    ComputeGradientFusedParamsCUDAKernel<T>
        <<<B, kCUDANumThreads, 0, hip_stream>>>(
            M,
            N,
            mean_data,
            rstd_data,
            ds_data,
            db_data,
            scale_data,
            bias_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    LayerNormBackwardCUDAKenrel<T><<<M, kCUDANumThreads, 0, hip_stream>>>(
        N,
        dY_data,
        X_data,
        gamma_data,
        rstd_data,
        scale_data,
        bias_data,
        dX_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
  if (dgamma->defined() || dbeta->defined()) {
    T* dgamma_data =
        dgamma->defined() ? dgamma->template data_ptr<T>() : nullptr;
    T* dbeta_data = dbeta->defined() ? dbeta->template data_ptr<T>() : nullptr;
    if (M < 512) {
      // For small batch size, do colwise reduce directly.
      const int64_t B = (N + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardSimpleCUDAKernel<T>
          <<<B, kCUDANumThreads, 0, hip_stream>>>(
              M,
              N,
              dY_data,
              X_data,
              mean_data,
              rstd_data,
              dgamma_data,
              dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      const int64_t B =
          (N + kColwiseReduceTileSize - 1) / kColwiseReduceTileSize;
      constexpr int kThreadX = kColwiseReduceTileSize;
      constexpr int kThreadY = kColwiseReduceTileSize / 2;
      GammaBetaBackwardCUDAKernel<T>
          <<<B, dim3(kThreadX, kThreadY), 0, hip_stream>>>(
              M,
              N,
              dY_data,
              X_data,
              mean_data,
              rstd_data,
              dgamma_data,
              dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
}

void LayerNormBackwardKernelImpl(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t M,
    int64_t N,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
      X.scalar_type(), "LayerNormBackwardKernelImpl", [&]() {
        LayerNormBackwardKernelImplInternal<scalar_t>(
            dY, X, mean, rstd, gamma, M, N, dX, dgamma, dbeta);
      });
}

} // namespace

std::tuple<Tensor, Tensor, Tensor> layer_norm_cuda(
    const Tensor& X,
    const Tensor& gamma /* optional */,
    const Tensor& beta /* optional */,
    int64_t M,
    int64_t N,
    double eps) {
  Tensor Y = at::native::empty_like(X, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor mean = at::empty({M}, X.options());
  Tensor rstd = at::empty({M}, X.options());
  if (M > 0) {
    LayerNormKernelImpl(X, gamma, beta, M, N, eps, &Y, &mean, &rstd);
  }
  return std::make_tuple(std::move(Y), std::move(mean), std::move(rstd));
}

std::tuple<Tensor, Tensor, Tensor> layer_norm_backward_cuda(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t M,
    int64_t N,
    std::array<bool, 3> grad_input_mask) {
  Tensor dX;
  Tensor dgamma;
  Tensor dbeta;
  if (grad_input_mask[0]) {
    dX = at::native::empty_like(X, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (grad_input_mask[1]) {
    dgamma = M > 0 ? at::native::empty_like(gamma, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : at::native::zeros_like(gamma, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (grad_input_mask[2]) {
    dbeta = M > 0 ? at::native::empty_like(gamma, LEGACY_CONTIGUOUS_MEMORY_FORMAT) : at::native::zeros_like(gamma, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (M > 0) {
    LayerNormBackwardKernelImpl(
        dY, X, mean, rstd, gamma, M, N, &dX, &dgamma, &dbeta);
  }
  return std::make_tuple(std::move(dX), std::move(dgamma), std::move(dbeta));
}


REGISTER_DISPATCH(LayerNormKernel, &LayerNormKernelImpl);
REGISTER_DISPATCH(LayerNormBackwardKernel, &LayerNormBackwardKernelImpl);

} // namespace native
} // namespace at
