#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/core/Array.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <c10/cuda/CUDAMathCompat.h>

namespace at {
namespace native {

// -----------------------------------
// prelu forward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_kernel_share_weights(
  const Tensor& input,
  Tensor& result,
  const scalar_t* weight_data)
{
  auto iter = TensorIterator::unary_op(result, input);

  at::native::gpu_kernel(iter,
    [weight_data] GPU_LAMBDA (scalar_t input_val) {
        return (input_val > 0) ? input_val : *weight_data * input_val;
    });
}

template <typename scalar_t>
__global__ void prelu_cuda_kernel_multi_weights(
  scalar_t* result_data,
  const scalar_t* input_data,
  const scalar_t* weight_data,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;

  // multiply values at each channel with weight[channel_index]
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  result_data[linearId] = (input_data_val > 0) ? input_data_val : weight_data[channel] * input_data_val;
}

Tensor prelu_cuda(const Tensor& self, const Tensor& weight_) {
  TORCH_CHECK(self.is_cuda());
  TORCH_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto weight = weight_.contiguous();

  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(weight.is_contiguous());

  int64_t weight_num = weight.numel();
  Tensor result = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto strides = input.strides();

  // case1: shared weight for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_share_weights<scalar_t>(
        input,
        result,
        weight.data_ptr<scalar_t>());
    });
  }
  else { // case2: multiple weights, one for each channel
    int64_t input_ndim = input.dim();
    TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    TORCH_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_cuda", [&] {
      prelu_cuda_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        result.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }
  return result;
}

// -----------------------------------
// prelu backward
// -----------------------------------
template <typename scalar_t>
void prelu_cuda_backward_kernel_share_weights(
  const Tensor& input,
  const Tensor& grad_out,
  Tensor& input_grad,
  Tensor& weight_grad_collector,
  const scalar_t* weight_data) {
  at::TensorIterator iter = TensorIteratorConfig()
      .add_output(input_grad)
      .add_output(weight_grad_collector)
      .add_input(input)
      .add_input(grad_out)
      .build();

  // N.B. `std::tuple` does not support `::operator=` on device code.
  gpu_kernel_multiple_outputs(iter, [=] GPU_LAMBDA (scalar_t input, scalar_t grad_out) -> thrust::tuple<scalar_t, scalar_t> {
    scalar_t input_grad = input > 0 ? grad_out : (*weight_data) * grad_out;
    scalar_t weight_grad_collector = input > 0 ? scalar_t(0) : input * grad_out;
    return {input_grad, weight_grad_collector};
  });
}

template <typename scalar_t>
__global__ void prelu_cuda_backward_kernel_multi_weights(
  const scalar_t* input_data,
  const scalar_t* weight_data,
  const scalar_t* grad_out_data,
  scalar_t* input_grad_data,
  scalar_t* weight_grad_collector,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  scalar_t grad_out_data_val = grad_out_data[linearId];
  input_grad_data[linearId] = (input_data_val > 0) ? grad_out_data_val : weight_data[channel] * grad_out_data_val;
  weight_grad_collector[linearId] = (input_data_val > 0) ? scalar_t(0) : input_data_val * grad_out_data_val;
}

std::tuple<Tensor, Tensor> prelu_backward_cuda(const Tensor& grad_out_, const Tensor& self, const Tensor& weight_) {
  TORCH_CHECK(grad_out_.is_cuda());
  TORCH_CHECK(self.is_cuda());
  TORCH_CHECK(weight_.is_cuda());

  auto input = self.contiguous();
  auto grad_out = grad_out_.contiguous();
  auto weight = weight_.contiguous();

  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(weight.is_contiguous());
  TORCH_CHECK(grad_out.is_contiguous());

  int64_t weight_num = weight.numel();
  auto strides = input.strides();
  auto dims = input.dim();
  Tensor input_grad = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor weight_grad = at::empty_like(weight, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor weight_grad_collector = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  // case1: shared parameter for all channels
  if (weight_num == 1) {
    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_share_weights<scalar_t>(
        input,
        grad_out,
        input_grad,
        weight_grad_collector,
        weight.data_ptr<scalar_t>());
    });
    weight_grad.fill_(weight_grad_collector.sum());
  }
  else { // case2: multiple parameters, one for each channel
    int64_t input_ndim = input.dim();
    TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

    int64_t channel_size = 1; // channel_size default to 1
    int64_t input_stride0 = 1, input_stride1 = 1;

    if (input_ndim > 1) {
      channel_size = input.size(1); // channel is the 2nd dim of input
      input_stride0 = strides[0];
      input_stride1 = strides[1];
    }
    TORCH_CHECK(channel_size == weight_num,
      "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
      " and channel size = ", channel_size, ".");

    // config to run cuda kernel
    int64_t input_numel = input.numel();
    const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
    dim3 grid;
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
    TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu_backward_cuda: input too large or too many dimensions");

    AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_backward_cuda", [&] {
      prelu_cuda_backward_kernel_multi_weights<scalar_t>
      <<<grid, block, 0, stream>>>(
        input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        grad_out.data_ptr<scalar_t>(),
        input_grad.data_ptr<scalar_t>(),
        weight_grad_collector.data_ptr<scalar_t>(),
        input_stride0,
        input_stride1,
        input_numel);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
    // update weight_grad
    std::vector<int64_t> reduce_dims;
    reduce_dims.push_back(0);
    if (dims > 2) {
      for(int64_t i = 2; i < dims; i++) reduce_dims.push_back(i);
    }
    weight_grad = weight_grad_collector.sum(reduce_dims);
  }
  return std::tuple<Tensor, Tensor>{input_grad, weight_grad};
}

// -----------------------------------
// hardshrink
// -----------------------------------
void hardshrink_kernel(TensorIterator& iter, Scalar value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a >= -lambd && a <= lambd) ? scalar_t(0) : a;
    });
  });
}

void softshrink_kernel(TensorIterator& iter, Scalar value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > lambd ? a - lambd : (a < -lambd ? a + lambd : scalar_t(0));
    });
  });
}

void shrink_backward_kernel(TensorIterator& iter, Scalar value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "shrink_backward_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t grad_val, scalar_t self_val) -> scalar_t {
      return (self_val >= -lambd && self_val <= lambd) ? scalar_t(0) : grad_val;
    });
  });
}

void hardtanh_backward_kernel(TensorIterator& iter, Scalar min, Scalar max) {
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, iter.dtype(), "hardtanh_backward_cuda", [&]() {
    auto min_val = min.to<scalar_t>();
    auto max_val = max.to<scalar_t>();
    gpu_kernel(iter, [min_val, max_val]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return (b <= min_val) || (b >= max_val) ? scalar_t(0) : a;
    });
  });
}

void softplus_kernel(TensorIterator& iter, Scalar beta_, Scalar threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a * beta) > threshold ? a : static_cast<scalar_t>(::log1p(std::exp(a * beta))) / beta;
    });
  });
}

void softplus_backward_kernel(TensorIterator& iter, Scalar beta_, Scalar threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_backward_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      scalar_t z = std::exp(b * beta);
      return (b * beta) > threshold ? a : a * (z - scalar_t(1.)) / z;
    });
  });
}

template <typename scalar_t>
void threshold_kernel_impl(TensorIterator& iter, scalar_t threshold, scalar_t value) {
  gpu_kernel_with_scalars(iter, [=]GPU_LAMBDA(scalar_t x, scalar_t other) -> scalar_t {
    return x <= threshold ? value : other;
  });
}

static void threshold_kernel(TensorIterator& iter, Scalar threshold, Scalar value) {
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "threshold_cuda", [&] {
    threshold_kernel_impl<scalar_t>(iter, threshold.to<scalar_t>(), value.to<scalar_t>());
  });
}

void elu_kernel(TensorIterator& iter, Scalar alpha, Scalar scale, Scalar input_scale) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a * poscoef : (static_cast<scalar_t>(std::exp(a * negiptcoef)) - scalar_t(1.)) * negcoef;
    });
  });
}

void elu_backward_kernel(TensorIterator& iter, Scalar alpha, Scalar scale, Scalar input_scale) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_backward_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return b <= scalar_t(0) ? a * negiptcoef * (b + negcoef) : a * poscoef;
    });
  });
}

namespace {

void GeluCUDAKernelImpl(TensorIterator& it) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, it.dtype(), "GeluCUDAKernelImpl", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    gpu_kernel(it, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
      return static_cast<T_ACC>(x) *
          c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
    });
  });
}

void GeluBackwardCUDAKernelImpl(TensorIterator& it) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
      it.dtype(), "GeluBackwardCUDAKernelImpl", [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        gpu_kernel(it, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
          constexpr T_ACC kBeta = M_2_SQRTPI * M_SQRT1_2 * T_ACC(0.5);
          const T_ACC cdf = c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
          const T_ACC pdf =
              c10::cuda::compat::exp(
                  T_ACC(-0.5) * static_cast<T_ACC>(x) * static_cast<T_ACC>(x)) *
              kBeta;
          return static_cast<T_ACC>(dy) * (cdf + static_cast<T_ACC>(x) * pdf);
        });
      });
}

void leaky_relu_kernel(TensorIterator& iter, Scalar negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a : a * negval;
    });
  });
}

void leaky_relu_backward_kernel(TensorIterator& iter, Scalar negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_backward_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a > scalar_t(0) ? b : b * negval;
    });
  });
}

void hardswish_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return x * std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardswish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_half(0.5f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_half]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        if (self_val < neg_three) {
          return zero;
        } else if (self_val <= three) {
          return grad_val * ((self_val / three) + one_half);
        } else {
          return grad_val;
        }
    });
  });
}

void hardsigmoid_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardsigmoid_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_sixth]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        return (self_val >= neg_three && self_val <= three)
          ? grad_val * one_sixth
          : zero;
    });
  });
}

void silu_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC x_acc = static_cast<T_ACC>(x);
              return x_acc / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
            });
      });
}

void silu_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_backward_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC dy_acc = static_cast<T_ACC>(dy);
              const T_ACC x_acc = static_cast<T_ACC>(x);
              const T_ACC s_acc =
                  T_ACC(1) / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
              return dy_acc * s_acc * (T_ACC(1) + x_acc * (T_ACC(1) - s_acc));
            });
      });
}

} // namespace

Tensor gelu_cuda(const Tensor& self) {
  Tensor Y = at::native::empty_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto it = TensorIterator::unary_op(Y, self);
  GeluCUDAKernelImpl(it);
  return Y;
}

Tensor gelu_backward_cuda(const Tensor& grad, const Tensor& self) {
  Tensor dX = at::native::empty_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto it = TensorIterator::binary_op(dX, grad, self);
  GeluBackwardCUDAKernelImpl(it);
  return dX;
}

// computes `result = self <= threshold ? value : other`
// other is `self` in threshold() and `grad` in threshold_backward()
static Tensor threshold_out_cuda(
    optional<Tensor> opt_result,
    const Tensor& self,
    Scalar threshold,
    Scalar value,
    const Tensor& other) {
  Tensor result = opt_result.value_or(Tensor());
  auto iter = TensorIteratorConfig()
    .set_check_mem_overlap(false)  // threshold is idempotent, so overlap is okay
    .add_output(result)
    .add_input(self)
    .add_input(other)
    .allow_cpu_scalars(true)
    .promote_inputs_to_common_dtype(true)
    .cast_common_dtype_to_outputs(true)
    .enforce_safe_casting_to_output(true)
    .build();
  threshold_kernel(iter, threshold, value);
  return iter.output();
}

Tensor threshold_cuda(const Tensor& self, Scalar threshold, Scalar value) {
  return threshold_out_cuda(nullopt, self, threshold, value, self);
}

Tensor& threshold__cuda(Tensor& self, Scalar threshold, Scalar value) {
  threshold_out_cuda(make_optional(self), self, threshold, value, self);
  return self;
}

Tensor& threshold_out_cuda(Tensor& result, const Tensor& self, Scalar threshold, Scalar value) {
  threshold_out_cuda(make_optional(result), self, threshold, value, self);
  return result;
}

Tensor threshold_backward_cuda(const Tensor& grad, const Tensor& self, Scalar threshold) {
  return threshold_out_cuda(nullopt, self, threshold, 0, grad);
}

REGISTER_DISPATCH(hardtanh_backward_stub, &hardtanh_backward_kernel);
REGISTER_DISPATCH(hardshrink_stub, &hardshrink_kernel);
REGISTER_DISPATCH(softshrink_stub, &softshrink_kernel);
REGISTER_DISPATCH(shrink_backward_stub, &shrink_backward_kernel);
REGISTER_DISPATCH(elu_stub, &elu_kernel);
REGISTER_DISPATCH(elu_backward_stub, &elu_backward_kernel);
REGISTER_DISPATCH(leaky_relu_stub, &leaky_relu_kernel);
REGISTER_DISPATCH(leaky_relu_backward_stub, &leaky_relu_backward_kernel);
REGISTER_DISPATCH(hardswish_stub, &hardswish_kernel);
REGISTER_DISPATCH(hardswish_backward_stub, &hardswish_backward_kernel);
REGISTER_DISPATCH(hardsigmoid_stub, &hardsigmoid_kernel);
REGISTER_DISPATCH(hardsigmoid_backward_stub, &hardsigmoid_backward_kernel);
REGISTER_DISPATCH(softplus_stub, &softplus_kernel);
REGISTER_DISPATCH(softplus_backward_stub, &softplus_backward_kernel);
REGISTER_DISPATCH(silu_stub, &silu_kernel);
REGISTER_DISPATCH(silu_backward_stub, &silu_backward_kernel);

} // namespace native
} // namespace at
