#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>

#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <ATen/native/cuda/im2col.cuh>

namespace at {
namespace native {
namespace {

static inline void slow_conv_transpose2d_shape_check(
    const Tensor& input,
    const Tensor& grad_output,
    const Tensor& weight,
    const Tensor& bias,
    int kernel_height,
    int kernel_width,
    int stride_height,
    int stride_width,
    int pad_height,
    int pad_width,
    int output_padding_height,
    int output_padding_width,
    int dilation_height,
    int dilation_width,
    bool weight_nullable) {
  TORCH_CHECK(
      kernel_width > 0 && kernel_height > 0,
      "kernel size should be greater than zero, but got kernel_height: ",
      kernel_height,
      " kernel_width: ",
      kernel_width);
  TORCH_CHECK(
      stride_width > 0 && stride_height > 0,
      "stride should be greater than zero, but got stride_height: ",
      stride_height,
      " stride_width: ",
      stride_width);
  TORCH_CHECK(
      dilation_width > 0 && dilation_height > 0,
      "dilation should be greater than zero, but got dilation_height: ",
      dilation_height,
      ", dilation_width: ",
      dilation_width);
  TORCH_CHECK(
      (output_padding_width < stride_width ||
       output_padding_width < dilation_width) &&
          (output_padding_height < stride_height ||
           output_padding_height < dilation_height),
      "output padding must be smaller than either stride or dilation, ",
      "but got output_padding_height: ",
      output_padding_height,
      " output_padding_width: ",
      output_padding_width,
      " stride_height: ",
      stride_height,
      " stride_width: ",
      stride_width,
      " dilation_height: ",
      dilation_height,
      " dilation_width: ",
      dilation_width);

  if (weight.defined()) {
    TORCH_CHECK(
        weight.numel() != 0 && (weight.dim() == 2 || weight.dim() == 4),
        "non-empty 2D or 4D weight tensor expected, but got: ",
        weight.sizes());
    if (bias.defined()) {
      check_dim_size(bias, 1, 0, weight.size(1));
    }
  } else if (!weight_nullable) {
    AT_ERROR("weight tensor is expected to be non-nullable");
  }

  int ndim = input.dim();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  TORCH_CHECK(
      input.numel() != 0 && (ndim == 3 || ndim == 4),
      "non-empty 3D or 4D input tensor expected but got a tensor with size ",
      input.sizes());

  int64_t input_height = input.size(dimh);
  int64_t input_width = input.size(dimw);
  int64_t output_height = (input_height - 1) * stride_height - 2 * pad_height +
      (dilation_height * (kernel_height - 1) + 1) + output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * pad_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  if (output_width < 1 || output_height < 1) {
    AT_ERROR(
        "Given input size per channel: (",
        input_height,
        " x ",
        input_width,
        "). Calculated output spatial size per channel: (",
        output_height,
        " x ",
        output_width,
        "). Output size is too small");
  }

  if (weight.defined()) {
    int64_t n_input_plane = weight.size(0);
    check_dim_size(input, ndim, dimf, n_input_plane);
  }

  if (grad_output.defined()) {
    if (weight.defined()) {
      int64_t n_output_plane = weight.size(1);
      check_dim_size(grad_output, ndim, dimf, n_output_plane);
    } else if (bias.defined()) {
      int64_t n_output_plane = bias.size(0);
      check_dim_size(grad_output, ndim, dimf, n_output_plane);
    }
    check_dim_size(grad_output, ndim, dimh, output_height);
    check_dim_size(grad_output, ndim, dimw, output_width);
  }
}

void slow_conv_transpose2d_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    const Tensor& weight_,
    IntArrayRef kernel_size,
    const Tensor& bias_,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    Tensor& columns_,
    Tensor& ones_) {
  TORCH_CHECK(
      kernel_size.size() == 2,
      "It is expected kernel_size equals to 2, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 2,
      "It is expected dilation equals to 2, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 2,
      "It is expected padding equals to 2, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 2,
      "It is expected stride equals to 2, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 2,
      "It is expected stride equals to 2, but got size ",
      output_padding.size());

  TensorArg input_arg{input_, "input", 1}, output_arg{output, "output", 2},
      weight_arg{weight_, "weight", 3}, bias_arg{bias_, "bias", 4},
      columns_arg{columns_, "columns", 5}, ones_arg{ones_, "ones", 6};

  checkAllSameGPU(
      "slow_conv_transpose2d_out_cuda",
      {input_arg, output_arg, weight_arg, bias_arg, columns_arg, ones_arg});

  int n_input_plane = weight_.size(0);
  int n_output_plane = weight_.size(1);

  Tensor columns = columns_;
  Tensor ones = ones_;

  int64_t kernel_height = kernel_size[0];
  int64_t kernel_width = kernel_size[1];
  int64_t dilation_height = dilation[0];
  int64_t dilation_width = dilation[1];
  int64_t pad_height = padding[0];
  int64_t pad_width = padding[1];
  int64_t stride_height = stride[0];
  int64_t stride_width = stride[1];
  int64_t output_padding_height = output_padding[0];
  int64_t output_padding_width = output_padding[1];

  slow_conv_transpose2d_shape_check(
      input_,
      Tensor(),
      weight_,
      bias_,
      kernel_height,
      kernel_width,
      stride_height,
      stride_width,
      pad_height,
      pad_width,
      output_padding_height,
      output_padding_width,
      dilation_height,
      dilation_width,
      false);

  Tensor input = input_.contiguous();
  Tensor weight = weight_.contiguous();

  Tensor bias = Tensor();

  if (bias_.defined()) {
    bias = bias_.contiguous();
    TORCH_CHECK(ones.is_contiguous(), "ones needs to be contiguous");
  }

  bool is_batch = false;
  if (input.dim() == 3) {
    // Force batch
    is_batch = true;
    input.resize_({1, input.size(0), input.size(1), input.size(2)});
  }

  int64_t input_height = input.size(2);
  int64_t input_width = input.size(3);
  int64_t output_height = (input_height - 1) * stride_height - 2 * pad_height +
      (dilation_height * (kernel_height - 1) + 1) + output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * pad_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Resize output
  output.resize_({batch_size, n_output_plane, output_height, output_width});

  // Resize temporary columns
  columns.resize_({n_output_plane * kernel_width * kernel_height,
                   input_height * input_width});

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets
  // increased, and always contains ones.
  if (ones.dim() != 2 ||
      ones.size(0) * ones.size(1) < output_height * output_width) {
    // Resize plane and fill with ones...
    ones.resize_({output_height, output_width});
    ones.fill_(1);
  }

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "slow_conv_transpose2d_out_cuda", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        // Helpers
        Tensor input_n;
        Tensor output_n;

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix mulitply per output:
          input_n = input.select(0, elt);
          output_n = output.select(0, elt);

          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m = weight.size(1) * weight.size(2) * weight.size(3);
          int64_t n = columns.size(1);
          int64_t k = weight.size(0);

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          at::cuda::blas::gemm<scalar_t>(
              'n',
              't',
              n,
              m,
              k,
              1,
              input_n.data_ptr<scalar_t>(),
              n,
              weight.data_ptr<scalar_t>(),
              m,
              0,
              columns.data_ptr<scalar_t>(),
              n);

          // Unpack columns back into input:
          col2im<scalar_t, accscalar_t>(
              at::cuda::getCurrentCUDAStream(),
              columns.data_ptr<scalar_t>(),
              n_output_plane,
              output_height,
              output_width,
              input_height,
              input_width,
              kernel_height,
              kernel_width,
              pad_height,
              pad_width,
              stride_height,
              stride_width,
              dilation_height,
              dilation_width,
              output_n.data_ptr<scalar_t>());

          // Do Bias after:
          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m_ = n_output_plane;
          int64_t n_ = output_height * output_width;
          int64_t k_ = 1;

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          if (bias.defined()) {
            at::cuda::blas::gemm<scalar_t>(
                't',
                'n',
                n_,
                m_,
                k_,
                1,
                ones.data_ptr<scalar_t>(),
                k_,
                bias.data_ptr<scalar_t>(),
                k_,
                1,
                output_n.data_ptr<scalar_t>(),
                n_);
          }
        }

        // Resize output
        if (is_batch) {
          output.resize_({n_output_plane, output_height, output_width});
          input.resize_({n_input_plane, input_height, input_width});
        }
      }); // end of dispatch
}

static void slow_conv_transpose2d_backward_out_cuda_template(
    const Tensor& input_,
    const Tensor& grad_output_,
    Tensor& grad_input,
    const Tensor& weight_,
    const Tensor& grad_columns_,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  TORCH_CHECK(
      kernel_size.size() == 2,
      "It is expected kernel_size equals to 2, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 2,
      "It is expected dilation equals to 2, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 2,
      "It is expected padding equals to 2, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 2,
      "It is expected stride equals to 2, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 2,
      "It is expected stride equals to 2, but got size ",
      output_padding.size());

  TensorArg input_arg{input_, "input", 1},
      grad_output_arg{grad_output_, "grad_output", 2},
      weight_arg{weight_, "weight", 3},
      grad_columns_arg{grad_columns_, "grad_columns", 4},
      grad_input_arg{grad_input, "grad_input", 5};

  checkAllSameGPU(
      "slow_conv_transpose2d_backward_out_cuda",
      {input_arg,
       grad_output_arg,
       weight_arg,
       grad_columns_arg,
       grad_input_arg});

  int n_input_plane = weight_.size(0);
  int n_output_plane = weight_.size(1);

  int64_t kernel_height = kernel_size[0];
  int64_t kernel_width = kernel_size[1];
  int64_t dilation_height = dilation[0];
  int64_t dilation_width = dilation[1];
  int64_t pad_height = padding[0];
  int64_t pad_width = padding[1];
  int64_t stride_height = stride[0];
  int64_t stride_width = stride[1];
  int64_t output_padding_height = output_padding[0];
  int64_t output_padding_width = output_padding[1];

  Tensor grad_columns = grad_columns_;

  slow_conv_transpose2d_shape_check(
      input_,
      grad_output_,
      weight_,
      Tensor(),
      kernel_height,
      kernel_width,
      stride_height,
      stride_width,
      pad_height,
      pad_width,
      output_padding_height,
      output_padding_width,
      dilation_height,
      dilation_width,
      false);

  Tensor input = input_.contiguous();
  Tensor grad_output = grad_output_.contiguous();
  Tensor weight = weight_.contiguous();

  bool is_batch = false;
  if (input.dim() == 3) {
    // Force batch
    is_batch = true;
    input.resize_({1, input.size(0), input.size(1), input.size(2)});
    grad_output.resize_(
        {1, grad_output.size(0), grad_output.size(1), grad_output.size(2)});
  }

  int64_t input_width = input.size(3);
  int64_t input_height = input.size(2);
  int64_t output_height = (input_height - 1) * stride_height - 2 * pad_height +
      (dilation_height * (kernel_height - 1) + 1) + output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * pad_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Resize output
  grad_input.resize_({batch_size, n_input_plane, input_height, input_width});

  // Resize temporary columns
  grad_columns.resize_({n_output_plane * kernel_width * kernel_height,
                        input_height * input_width});

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      grad_output.scalar_type(), "slow_conv_transpose2d_backward_out_cuda", [&] {
        // Helpers
        Tensor grad_input_n = Tensor();
        Tensor grad_output_n = Tensor();

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix mulitply per sample:
          grad_input_n = grad_input.select(0, elt);
          grad_output_n = grad_output.select(0, elt);

          if (kernel_height != 1 || kernel_width != 1) {
            im2col<scalar_t>(
                at::cuda::getCurrentCUDAStream(),
                grad_output_n.data_ptr<scalar_t>(),
                n_output_plane,
                output_height,
                output_width,
                input_height,
                input_width,
                kernel_height,
                kernel_width,
                pad_height,
                pad_width,
                stride_height,
                stride_width,
                dilation_height,
                dilation_width,
                grad_columns.data_ptr<scalar_t>());
          }

          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m = weight.size(0);
          int64_t n = grad_columns.size(1);
          int64_t k = weight.size(1) * weight.size(2) * weight.size(3);

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          auto gemm_in_ptr = (kernel_height != 1 || kernel_width != 1) ?
              grad_columns.data_ptr<scalar_t>() : grad_output_n.data_ptr<scalar_t>();
          at::cuda::blas::gemm<scalar_t>(
              'n',
              'n',
              n,
              m,
              k,
              1,
              gemm_in_ptr,
              n,
              weight.data_ptr<scalar_t>(),
              k,
              0,
              grad_input_n.data_ptr<scalar_t>(),
              n);
        }

        // Resize output
        if (is_batch) {
          grad_output.resize_({n_output_plane, output_height, output_width});
          input.resize_({n_input_plane, input_height, input_width});
          grad_input.resize_({n_input_plane, input_height, input_width});
        }
      }); // end of dispatch
}

void slow_conv_transpose2d_acc_grad_parameters_cuda_template(
    const Tensor& input_,
    const Tensor& grad_output_,
    Tensor& grad_weight,
    Tensor& grad_bias,
    const Tensor& columns_,
    const Tensor& ones_,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    int scale_) {
  TORCH_CHECK(
      kernel_size.size() == 2,
      "It is expected kernel_size equals to 2, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 2,
      "It is expected dilation equals to 2, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 2,
      "It is expected padding equals to 2, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 2,
      "It is expected stride equals to 2, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 2,
      "It is expected stride equals to 2, but got size ",
      output_padding.size());

  TensorArg input_arg{input_, "input", 1},
      grad_output_arg{grad_output_, "grad_output", 2},
      grad_weight_arg{grad_weight, "grad_weight", 3},
      grad_bias_arg{grad_bias, "grad_bias", 4},
      columns_arg{columns_, "columns", 5}, ones_arg{ones_, "ones", 6};

  checkAllSameGPU(
      "slow_conv_transpose2d_acc_grad_parameters_cuda",
      {input_arg,
       grad_output_arg,
       grad_weight_arg,
       grad_bias_arg,
       columns_arg,
       ones_arg});

  int64_t kernel_height = kernel_size[0];
  int64_t kernel_width = kernel_size[1];
  int64_t dilation_height = dilation[0];
  int64_t dilation_width = dilation[1];
  int64_t pad_height = padding[0];
  int64_t pad_width = padding[1];
  int64_t stride_height = stride[0];
  int64_t stride_width = stride[1];
  int64_t output_padding_height = output_padding[0];
  int64_t output_padding_width = output_padding[1];

  Tensor columns = columns_;
  Tensor ones = ones_;

  slow_conv_transpose2d_shape_check(
      input_,
      grad_output_,
      grad_weight,
      grad_bias,
      kernel_height,
      kernel_width,
      stride_height,
      stride_width,
      pad_height,
      pad_width,
      output_padding_height,
      output_padding_width,
      dilation_height,
      dilation_width,
      true);

  Tensor input = input_.contiguous();
  Tensor grad_output = grad_output_.contiguous();

  int64_t n_output_plane;
  if (grad_weight.defined()) {
    n_output_plane = grad_weight.size(1);
  } else if (grad_bias.defined()) {
    n_output_plane = grad_bias.size(0);
  } else {
    return;
  }

  if (grad_weight.defined()) {
    TORCH_CHECK(
        grad_weight.is_contiguous(), "grad_weight needs to be contiguous");
  }

  TORCH_CHECK(columns.is_contiguous(), "columns needs to be contiguous");

  if (grad_bias.defined()) {
    TORCH_CHECK(grad_bias.is_contiguous(), "grad_bias needs to be contiguous");
    TORCH_CHECK(ones.is_contiguous(), "ones needs to be contiguous");
  }

  bool is_batch = false;
  if (input.dim() == 3) {
    // Force batch
    is_batch = true;
    input.resize_({1, input.size(0), input.size(1), input.size(2)});
    grad_output.resize_(
        {1, grad_output.size(0), grad_output.size(1), grad_output.size(2)});
  }

  int64_t input_width = input.size(3);
  int64_t input_height = input.size(2);
  int64_t output_height = (input_height - 1) * stride_height - 2 * pad_height +
      (dilation_height * (kernel_height - 1) + 1) + output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * pad_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones.dim() != 2 ||
      ones.size(0) * ones.size(1) < output_height * output_width) {
    // Resize plane and fill with ones...
    ones.resize_({output_height, output_width});
    ones.fill_(1); // or static_cast<scalar_t>(1)
  }

  // Resize temporary columns
  columns.resize_({n_output_plane * kernel_width * kernel_height,
                   input_height * input_width});

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "slow_conv_transpose2d_acc_grad_parameters_cuda", [&] {
        // Helpers
        Tensor input_n = Tensor();
        Tensor grad_output_n = Tensor();

        scalar_t scale = static_cast<scalar_t>(scale_);

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix mulitply per output:
          grad_output_n = grad_output.select(0, elt);

          // Do Weight:
          if (grad_weight.defined()) {
            // Matrix mulitply per output:
            input_n = input.select(0, elt);

            if (kernel_height != 1 || kernel_width != 1) {
              // Extract columns:
              im2col<scalar_t>(
                  at::cuda::getCurrentCUDAStream(),
                  grad_output_n.data_ptr<scalar_t>(),
                  n_output_plane,
                  output_height,
                  output_width,
                  input_height,
                  input_width,
                  kernel_height,
                  kernel_width,
                  pad_height,
                  pad_width,
                  stride_height,
                  stride_width,
                  dilation_height,
                  dilation_width,
                  columns.data_ptr<scalar_t>());
            }

            // M,N,K are dims of matrix A and B
            // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
            int64_t n = columns.size(0); // n_output_plane * kh * kw
            int64_t m = input_n.size(0); // n_input_plane
            int64_t k = columns.size(1); // input_height * input_width

            // Do GEMM (note: this is a bit confusing because gemm assumes
            // column-major matrices)
            auto gemm_in_ptr = (kernel_height != 1 || kernel_width != 1) ?
                columns.data_ptr<scalar_t>() : grad_output_n.data_ptr<scalar_t>();
            at::cuda::blas::gemm<scalar_t>(
                't',
                'n',
                n,
                m,
                k,
                scale,
                gemm_in_ptr,
                k,
                input_n.data_ptr<scalar_t>(),
                k,
                1,
                grad_weight.data_ptr<scalar_t>(),
                n);
          }

          // Do Bias:
          if (grad_bias.defined()) {
            // M,N,K are dims of matrix A and B
            // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
            int64_t m_ = n_output_plane;
            int64_t k_ = output_height * output_width;

            // Do GEMV (note: this is a bit confusing because gemv assumes
            // column-major matrices)
            at::cuda::blas::gemv<scalar_t>(
                't',
                k_,
                m_,
                scale,
                grad_output_n.data_ptr<scalar_t>(),
                k_,
                ones.data_ptr<scalar_t>(),
                1,
                1,
                grad_bias.data_ptr<scalar_t>(),
                1);
          }
        }

        // Resize
        if (is_batch) {
          grad_output.resize_({n_output_plane, output_height, output_width});
          input.resize_({input.size(1), input_height, input_width});
        }
      }); // end of dispatch
}
} // namespace

Tensor& slow_conv_transpose2d_out_cuda(
    Tensor& output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    const Tensor& bias,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  Tensor columns = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor ones = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  slow_conv_transpose2d_out_cuda_template(
      output,
      input,
      weight,
      kernel_size,
      bias,
      stride,
      padding,
      output_padding,
      dilation,
      columns,
      ones);

  return output;
}

Tensor slow_conv_transpose2d_cuda(
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    const Tensor& bias,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor columns = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor ones = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  slow_conv_transpose2d_out_cuda_template(
      output,
      input,
      weight,
      kernel_size,
      bias,
      stride,
      padding,
      output_padding,
      dilation,
      columns,
      ones);

  return output;
}

std::tuple<Tensor&, Tensor&, Tensor&> slow_conv_transpose2d_backward_out_cuda(
    Tensor& grad_input,
    Tensor& grad_weight,
    Tensor& grad_bias,
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    const Tensor& columns,
    const Tensor& ones) {
  if (grad_input.defined()) {
    slow_conv_transpose2d_backward_out_cuda_template(
        input,
        grad_output,
        grad_input,
        weight,
        columns,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation);
  }

  if (grad_weight.defined()) {
    grad_weight.resize_(weight.sizes());
    grad_weight.zero_();
  }

  if (grad_bias.defined()) {
    grad_bias.resize_({weight.size(1)});
    grad_bias.zero_();
  }

  if (grad_weight.defined() || grad_bias.defined()) {
    slow_conv_transpose2d_acc_grad_parameters_cuda_template(
        input,
        grad_output,
        grad_weight,
        grad_bias,
        columns,
        ones,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation,
        1);
  }

  return std::tuple<Tensor&, Tensor&, Tensor&>(
      grad_input, grad_weight, grad_bias);
}

std::tuple<Tensor, Tensor, Tensor> slow_conv_transpose2d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    const Tensor& columns,
    const Tensor& ones,
    std::array<bool, 3> output_mask) {
  Tensor grad_input;
  Tensor grad_weight;
  Tensor grad_bias;

  if (output_mask[0]) {
    grad_input = at::empty({0}, grad_output.options());
  } else {
    grad_input = Tensor();
  }

  if (output_mask[1]) {
    grad_weight = at::empty({0}, grad_output.options());
  } else {
    grad_weight = Tensor();
  }

  if (output_mask[2]) {
    grad_bias = at::empty({0}, grad_output.options());
  } else {
    grad_bias = Tensor();
  }

  if (grad_input.defined()) {
    slow_conv_transpose2d_backward_out_cuda_template(
        input,
        grad_output,
        grad_input,
        weight,
        columns,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation);
  }

  if (grad_weight.defined()) {
    grad_weight.resize_(weight.sizes());
    grad_weight.zero_();
  }

  if (grad_bias.defined()) {
    grad_bias.resize_({weight.size(1)});
    grad_bias.zero_();
  }

  if (grad_weight.defined() || grad_bias.defined()) {
    slow_conv_transpose2d_acc_grad_parameters_cuda_template(
        input,
        grad_output,
        grad_weight,
        grad_bias,
        columns,
        ones,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation,
        1);
  }

  return std::tuple<Tensor, Tensor, Tensor>(grad_input, grad_weight, grad_bias);
}

} // namespace native
} // namespace at
