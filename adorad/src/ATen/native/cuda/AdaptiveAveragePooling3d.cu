#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCGeneral.h>
#include <THC/THCNumerics.cuh>
#include <THC/THCAtomics.cuh>  // for gpuAtomicAdd
#include <c10/util/Exception.h>

#include <algorithm>
#include <cfloat>
#include <cmath>

namespace at {
namespace native {

namespace {

__device__ inline int start_index(int a, int b, int c) {
  return (int)std::floor((float)(a * c) / b);
}

__device__ inline int end_index(int a, int b, int c) {
  return (int)std::ceil((float)((a + 1) * c) / b);
}

// 5d tensor B x D x T x H x W
// All kernels view batch dim B and dim D as collapsed.

/*
 * Description:
 *    this function adaptively average pools an input 5D tensor along dimensions
 * 2, 3, and 4 5D input, 5D output
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 */
template <typename scalar_t>
__global__ void adaptiveaveragepool(
    scalar_t *input, scalar_t *output,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t istrideD,
    int64_t istrideT, int64_t istrideH, int64_t istrideW,
    int64_t offsetZ) {
  // iterates on output pixels
  int ot, oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH = osizeH;
  int ostepH = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW = osizeW;
  int ostepW = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  ot = o_plane % osizeT; // output frame/time
  int d = o_plane / osizeT; // slice/feature

  // input frame/time range is fixed.
  int istartT = start_index(ot, osizeT, isizeT);
  int iendT = end_index(ot, osizeT, isizeT);
  int kT = iendT - istartT;

  // input offset by slice/feature and earliest relevant frame/time
  scalar_t *input_dt = input + d*istrideD + istartT*istrideT;
  // output offset by slice/feature and frame/time
  scalar_t *output_dt = output + o_plane*osizeH*osizeW;

  // For all output pixels...
  for (oh = ostartH; oh < oendH; oh += ostepH) {
    int istartH = start_index(oh, osizeH, isizeH);
    int iendH = end_index(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for (ow = ostartW; ow < oendW; ow += ostepW) {
      int istartW = start_index(ow, osizeW, isizeW);
      int iendW = end_index(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the average pooling from corresponding input pixels
      scalar_t *ptr_input = input_dt + istartH*istrideH + istartW*istrideW;
      scalar_t *ptr_output = output_dt + oh*osizeW + ow;
      scalar_t sum = ScalarConvert<int, scalar_t>::to(0);

      int it, ih, iw;
      for (it = 0; it < kT; ++it) {
        for (ih = 0; ih < kH; ++ih) {
          for (iw = 0; iw < kW; ++iw) {
            scalar_t val = ptr_input[ih*istrideH + iw*istrideW];
            sum += val;
          }
        }
        ptr_input += istrideT; // next input frame
      }
      // Update output
      *ptr_output = sum / kT / kH / kW;
    }
  }
}

template <typename scalar_t>
void adaptiveaveragepool_loop(
    scalar_t *input_data, scalar_t *output_data,
    int64_t totalZ,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t istrideD, int64_t istrideT, int64_t istrideH, int64_t istrideW) {
  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = std::max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    adaptiveaveragepool<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        input_data, output_data,
        isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW,
        istrideD,
        istrideT, istrideH, istrideW,
        offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    totalZ -= 65535;
    offsetZ += 65535;
  }
}

/*
 * Description:
 *    This function computes the gradInput from gradOutput.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 */
template <typename scalar_t>
__global__ void adaptiveaveragegradinput(
    scalar_t *gradInput, scalar_t *gradOutput,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t offsetZ)
{
  // iterators on input pixels
  int it, ih, iw;

  // compute offsets based on thread/block ID
  int istartH = blockIdx.y * blockDim.y + threadIdx.y;
  int iendH = isizeH;
  int istepH = gridDim.y * blockDim.y;
  int istartW = threadIdx.x;
  int iendW = isizeW;
  int istepW = blockDim.x;

  // select input plane
  int64_t i_plane = blockIdx.x + offsetZ;
  it = i_plane % isizeT; // output frame/time
  int d = i_plane / isizeT; // slice/feature

  // output frame/time range is fixed.
  int ostartT = start_index(it, isizeT, osizeT);
  int oendT = end_index(it, isizeT, osizeT);

  // gradInput offset by slice/feature and frame/time.
  scalar_t *gradInput_dt = gradInput + i_plane*isizeH*isizeW;
  // gradOutput offset by slice/feature and earliest relevant frame/time
  scalar_t *gradOutput_dt = gradOutput + (d*osizeT + ostartT)*osizeH*osizeW;

  // For all input pixels...
  for (ih = istartH; ih < iendH; ih += istepH) {
    int ostartH = start_index(ih, isizeH, osizeH);
    int oendH = end_index(ih, isizeH, osizeH);

    for (iw = istartW; iw < iendW; iw += istepW) {
      int ostartW = start_index(iw, isizeW, osizeW);
      int oendW = end_index(iw, isizeW, osizeW);

      // Compute the gradients from corresponding output pixels
      scalar_t *ptr_gradInput = gradInput_dt + ih*isizeW + iw;
      scalar_t *ptr_gradOutput = gradOutput_dt;

      // for all relevant output pixels
      int ot, oh, ow;
      for (ot = ostartT; ot < oendT; ++ot) {
        int kT = end_index(ot, osizeT, isizeT) - start_index(ot, osizeT, isizeT);
        for (oh = ostartH; oh < oendH; ++oh) {
          int kH = end_index(oh, osizeH, isizeH) - start_index(oh, osizeH, isizeH);
          for (ow = ostartW; ow < oendW; ++ow) {
            int kW = end_index(ow, osizeW, isizeW) - start_index(ow, osizeW, isizeW);
            scalar_t grad_delta = ptr_gradOutput[oh*isizeW + ow] / kW / kH / kT;
            *ptr_gradInput += grad_delta;
          }
        }
        ptr_gradOutput += osizeH*osizeW; // next output frame
      }
    }
  }
}

template <typename scalar_t>
void adaptiveaveragegradinput_loop(
    scalar_t *gradInput_data, scalar_t *gradOutput_data,
    int64_t totalZ,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW) {
  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = std::max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    adaptiveaveragegradinput<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        gradInput_data, gradOutput_data,
        isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW,
        offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    totalZ -= 65535;
    offsetZ += 65535;
  }
}

/*
 * Description:
 *    This function computes the gradInput from gradOutput.
 *
 *    gridDim.y blocks work together on a single 2D output plane specified by
 *    (blockIdx.x + offsetZ).
 *
 *    (uses atomic add)
 *
 */
template <typename scalar_t>
__global__ void atomicadaptiveaveragegradinput(
    scalar_t *gradInput, scalar_t *gradOutput,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW,
    int64_t offsetZ)
{
  // iterators on output pixels
  int ot, oh, ow;

  // compute offsets based on thread/block ID
  int ostartH = blockIdx.y * blockDim.y + threadIdx.y;
  int oendH = osizeH;
  int ostepH = gridDim.y * blockDim.y;
  int ostartW = threadIdx.x;
  int oendW = osizeW;
  int ostepW = blockDim.x;

  // select output plane
  int64_t o_plane = blockIdx.x + offsetZ;
  ot = o_plane % osizeT; // output frame/time
  int d = o_plane / osizeT; // output slice/feature

  // input frame/time range is fixed.
  int istartT = start_index(ot, osizeT, isizeT);
  int iendT = end_index(ot, osizeT, isizeT);
  int kT = iendT - istartT;

  // gradInput offset by slice/feature and earliest relevant frame/time
  scalar_t *gradInput_nt = gradInput + (d*isizeT + istartT)*isizeH*isizeW;
  // gradOutput offset by slice/feature and frame/time
  scalar_t *gradOutput_nt = gradOutput + o_plane*osizeH*osizeW;

  // For all output pixels...
  for (oh = ostartH; oh < oendH; oh += ostepH) {
    int istartH = start_index(oh, osizeH, isizeH);
    int iendH = end_index(oh, osizeH, isizeH);
    int kH = iendH - istartH;

    for (ow = ostartW; ow < oendW; ow += ostepW) {
      int istartW = start_index(ow, osizeW, isizeW);
      int iendW = end_index(ow, osizeW, isizeW);
      int kW = iendW - istartW;

      // Compute the gradients from corresponding input pixels
      scalar_t *ptr_gradInput = gradInput_nt + istartH*isizeW + istartW;
      scalar_t *ptr_gradOutput = gradOutput_nt + oh*osizeW + ow;
      scalar_t grad_delta = *ptr_gradOutput / kT / kH / kW;

      int it, ih, iw;
      for (it = 0; it < kT; ++it) {
        for (ih = 0; ih < kH; ++ih) {
          for (iw = 0; iw < kW; ++iw) {
            gpuAtomicAdd(&(ptr_gradInput[ih*isizeW + iw]), grad_delta);
          }
        }
        ptr_gradInput += isizeH*isizeW; // next input frame
      }
    }
  }
}

template <typename scalar_t>
void atomicadaptiveaveragegradinput_loop(
    scalar_t* gradInput_data, scalar_t* gradOutput_data,
    int64_t totalZ,
    int isizeT, int isizeH, int isizeW,
    int osizeT, int osizeH, int osizeW) {
  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  int blocksH = std::max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    atomicadaptiveaveragegradinput<<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        gradInput_data, gradOutput_data,
        isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW,
        offsetZ);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    totalZ -= 65535;
    offsetZ += 65535;
  }
}

// 5D tensor B x D x T x H x w

void adaptive_avg_pool3d_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    IntArrayRef& output_size) {
  TensorArg output_arg{output, "output", 1};
  TensorArg input_arg{input_, "input_", 2};

  checkAllSameGPU("adaptive_avg_pool3d_cuda", {output_arg, input_arg});

  for (int64_t i = 0; i < input_.ndimension(); i++) {
    TORCH_CHECK(
        input_.size(i) > 0,
        "adaptive_avg_pool3d_cuda(): expected input to have non-empty spatial dimensions, "
        "but input has sizes ", input_.sizes(),
        " with dimension ", i, " being empty");
  }

  TORCH_CHECK(
      (input_.ndimension() == 4 || input_.ndimension() == 5),
      "non-empty 4D or 5D (batch mode) tensor expected for input");

  // the jit sometimes passes output_size.size() == 1
  TORCH_CHECK(
      output_size.size() == 1 || output_size.size() == 3,
      "adaptive_avg_pool3d: internal error: output_size.size() must be 1 or 3");

  int64_t osizeT = output_size[0];
  int64_t osizeH = output_size[1];
  int64_t osizeW = output_size[2];

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t istrideD, istrideT, istrideH, istrideW;
  int64_t totalZ;

  const Tensor& input = input_.ndimension() == 4 ? input_ : input_.contiguous();

  if (input.ndimension() == 4) {
    sizeD = input.size(0);
    isizeT = input.size(1);
    isizeH = input.size(2);
    isizeW = input.size(3);

    istrideD = input.stride(0);
    istrideT = input.stride(1);
    istrideH = input.stride(2);
    istrideW = input.stride(3);

    output.resize_({sizeD, osizeT, osizeH, osizeW});

    totalZ = sizeD * osizeT;
  } else {
    int64_t sizeB = input.size(0);
    sizeD = input.size(1);
    isizeT = input.size(2);
    isizeH = input.size(3);
    isizeW = input.size(4);

    istrideD = input.stride(1);
    istrideT = input.stride(2);
    istrideH = input.stride(3);
    istrideW = input.stride(4);

    output.resize_({sizeB, sizeD, osizeT, osizeH, osizeW});

    totalZ = sizeB * sizeD * osizeT;
  }

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "adaptive_avg_pool3d_cuda", [&] {
        scalar_t* input_data = input.data_ptr<scalar_t>();
        scalar_t* output_data = output.data_ptr<scalar_t>();

        adaptiveaveragepool_loop(
            input_data, output_data,
            totalZ,
            isizeT, isizeH, isizeW,
            osizeT, osizeH, osizeW,
            istrideD, istrideT, istrideH, istrideW);
      });
}

void adaptive_avg_pool3d_backward_out_cuda_template(
    Tensor& gradInput,
    const Tensor& gradOutput_,
    const Tensor& input) {
  TensorArg grad_input_arg{gradInput, "gradInput", 1};
  TensorArg grad_output_arg{gradOutput_, "gradOutput_", 2};
  TensorArg input_arg{input, "input", 3};

  checkAllSameGPU(
      "adaptive_avg_pool3d_out_cuda",
      {grad_input_arg, grad_output_arg, input_arg});

  const Tensor gradOutput = gradOutput_.contiguous();

  gradInput.resize_as_(input);
  gradInput.zero_();

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t osizeT, osizeH, osizeW;
  int64_t totalZ;

  if (input.ndimension() == 4) {
    sizeD = input.size(0);
    isizeT = input.size(1);
    isizeH = input.size(2);
    isizeW = input.size(3);

    osizeT = gradOutput.size(1);
    osizeH = gradOutput.size(2);
    osizeW = gradOutput.size(3);
  } else {
    sizeD = input.size(1);
    isizeT = input.size(2);
    isizeH = input.size(3);
    isizeW = input.size(4);

    osizeT = gradOutput.size(2);
    osizeH = gradOutput.size(3);
    osizeW = gradOutput.size(4);
  }

  bool atomic = (isizeW%osizeW != 0) || (isizeH%osizeH != 0) || (isizeT%osizeT != 0);

  if (input.ndimension() == 4) {
    totalZ = atomic ? sizeD * osizeT : sizeD * isizeT;
  } else {
    int sizeB = input.size(0);
    totalZ = atomic ? sizeB * sizeD * osizeT : sizeB * sizeD * isizeT;
  }

  if (atomic) {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
        input.scalar_type(), "adaptive_avg_pool3d_backward_cuda", [&] {
          scalar_t* gradInput_data = gradInput.data_ptr<scalar_t>();
          scalar_t* gradOutput_data = gradOutput.data_ptr<scalar_t>();

          atomicadaptiveaveragegradinput_loop(
              gradInput_data, gradOutput_data,
              totalZ,
              isizeT, isizeH, isizeW,
              osizeT, osizeH, osizeW);
        });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
        input.scalar_type(), "adaptive_avg_pool3d_backward_cuda", [&] {
          scalar_t* gradInput_data = gradInput.data_ptr<scalar_t>();
          scalar_t* gradOutput_data = gradOutput.data_ptr<scalar_t>();

          adaptiveaveragegradinput_loop(
              gradInput_data, gradOutput_data,
              totalZ,
              isizeT, isizeH, isizeW,
              osizeT, osizeH, osizeW);
        });
  }
}

} // namespace

Tensor& adaptive_avg_pool3d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size) {
  adaptive_avg_pool3d_out_cuda_template(output, input, output_size);
  return output;
}

Tensor adaptive_avg_pool3d_cuda(
    const Tensor& input,
    IntArrayRef output_size) {
  auto output = at::empty({0}, input.options());
  adaptive_avg_pool3d_out_cuda_template(output, input, output_size);
  return output;
}

Tensor& adaptive_avg_pool3d_backward_out_cuda(
    Tensor& gradInput,
    const Tensor& gradOutput_,
    const Tensor& input) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("adaptive_avg_pool3d_backward_out_cuda");
  adaptive_avg_pool3d_backward_out_cuda_template(gradInput, gradOutput_, input);
  return gradInput;
}

Tensor adaptive_avg_pool3d_backward_cuda(
    const Tensor& gradOutput_,
    const Tensor& input) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("adaptive_avg_pool3d_backward_cuda");
  auto gradInput = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  adaptive_avg_pool3d_backward_out_cuda_template(gradInput, gradOutput_, input);
  return gradInput;
}

} // namespace native
} // namespace at
