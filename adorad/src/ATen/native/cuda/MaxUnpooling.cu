#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/util/Exception.h>

namespace at {
namespace native {

using namespace at::cuda::detail;

template <typename T>
__host__ __device__ __forceinline__ T ceilDiv(T a, T b) {
  return (a + b - 1) / b;
}

template <typename T>
__global__ void max_unpooling2d_forward_kernel(
    const int64_t numInputElements,
    const T* input,
    const int64_t* indices,
    const int64_t numChannels,
    const int64_t inputHeight,
    const int64_t inputWidth,
    const int64_t outputHeight,
    const int64_t outputWidth,
    T* output) {
  CUDA_KERNEL_LOOP(linearIndex, numInputElements) {
    int c = (linearIndex / inputWidth / inputHeight) % numChannels;
    int n = linearIndex / inputWidth / inputHeight / numChannels;
    output += (n * numChannels + c) * outputHeight * outputWidth;
    int maxind = indices[linearIndex];
    output[maxind] = input[linearIndex];
  }
}

template <typename T>
__global__ void max_unpooling3d_forward_kernel(
    PackedTensorAccessor64<T, 4> input,
    PackedTensorAccessor64<int64_t, 4> indices,
    T* output,
    const int64_t oT,
    const int64_t oH,
    const int64_t oW,
    const int64_t offsetZ) {
  int64_t iColumn = blockIdx.x * blockDim.x + threadIdx.x;
  int64_t iRow = blockIdx.y * blockDim.y + threadIdx.y;
  int64_t iFrame = (blockIdx.z + offsetZ) % input.size(1); // input frame/time
  int64_t slice = (blockIdx.z + offsetZ) / input.size(1); // input slice/feature
  if (iRow < input.size(2) && iColumn < input.size(3)) {
    T val = input[slice][iFrame][iRow][iColumn];
    int64_t index = indices[slice][iFrame][iRow][iColumn];
    output[slice * oT * oH * oW + index] = val;
  }
}

template <typename T>
__global__ void max_unpooling2d_backward_kernel(
    const int64_t numInputElements,
    const T* input,
    const int64_t* indices,
    const int64_t numChannels,
    const int64_t inputHeight,
    const int64_t inputWidth,
    const int64_t outputHeight,
    const int64_t outputWidth,
    T* output) {
  CUDA_KERNEL_LOOP(linearIndex, numInputElements) {
    int c = (linearIndex / inputWidth / inputHeight) % numChannels;
    int n = linearIndex / inputWidth / inputHeight / numChannels;
    input += (n * numChannels + c) * outputHeight * outputWidth;
    int maxind = indices[linearIndex];
    output[linearIndex] = input[maxind];
  }
}

template <typename T>
__global__ void max_unpooling3d_backward_kernel(
    T* gradOutputData,
    int64_t oT,
    int64_t oH,
    int64_t oW,
    PackedTensorAccessor64<int64_t, 4> indices,
    PackedTensorAccessor64<T, 4> gradInput,
    int offsetZ) {
  int iColumn = blockIdx.x * blockDim.x + threadIdx.x;
  int iRow = blockIdx.y * blockDim.y + threadIdx.y;
  int iFrame = (blockIdx.z + offsetZ) % gradInput.size(1); // output frame/time
  int slice =
      (blockIdx.z + offsetZ) / gradInput.size(1); // output slice/feature

  if (iRow < gradInput.size(2) && iColumn < gradInput.size(3)) {
    int64_t index = indices[slice][iFrame][iRow][iColumn];
    T grad_val = gradOutputData[slice * oT * oH * oW + index];
    gradInput[slice][iFrame][iRow][iColumn] = grad_val;
  }
}

Tensor& max_unpooling2d_forward_out_cuda(
    Tensor& output,
    const Tensor& self_,
    const Tensor& indices_,
    IntArrayRef output_size) {
  TORCH_CHECK(output.is_contiguous(), "output must be contiguous");
  TORCH_CHECK(
      indices_.scalar_type() == at::ScalarType::Long,
      "elements in indices should be type int64");
  auto oheight = output_size[0];
  auto owidth = output_size[1];

  TensorArg output_arg{output, "output", 1}, self_arg{self_, "self_", 2},
      indices_arg{indices_, "indices_", 3};
  checkAllSameGPU(
      "max_unpooling2d_forward_out_cuda", {output_arg, self_arg, indices_arg});

  TORCH_CHECK(self_.numel() > 0, "Input must be non-empty tensor");

  TORCH_CHECK(
      (self_.ndimension() == 3 || self_.ndimension() == 4),
      "Input to max_unpooling2d should be a 3d or 4d Tensor",
      self_.sizes());
  TORCH_CHECK(
      self_.sizes() == indices_.sizes(),
      "Shape of input must match shape of indices");
  TORCH_CHECK(
      output_size.size() == 2,
      "There should be exactly two elements (width, height) in output_size");

  int64_t dimw = 2;
  int64_t dimh = 1;
  int64_t numBatch = 1;

  int64_t numChannels;
  int64_t inputHeight;
  int64_t inputWidth;

  auto self = self_.contiguous();
  auto indices = indices_.contiguous();

  if (self.ndimension() == 4) {
    numBatch = self.size(0);
    dimw++;
    dimh++;
  }
  numChannels = self.size(dimh - 1);
  inputHeight = self.size(dimh);
  inputWidth = self.size(dimw);

  output.resize_({numBatch, numChannels, oheight, owidth});

  output.zero_();

  auto count = self.numel();
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half,
      self.scalar_type(), "max_unpooling2d_forward_kernel", ([&] {
        max_unpooling2d_forward_kernel<<<
            GET_BLOCKS(count),
            CUDA_NUM_THREADS,
            0,
            at::cuda::getCurrentCUDAStream()>>>(
            self.numel(),
            self.data_ptr<scalar_t>(),
            indices.data_ptr<int64_t>(),
            numChannels,
            inputHeight,
            inputWidth,
            oheight,
            owidth,
            output.data_ptr<scalar_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
  if (self.ndimension() == 3) {
    output.resize_({numChannels, oheight, owidth});
  }
  return output;
}

Tensor max_unpooling2d_forward_cuda(
    const Tensor& self,
    const Tensor& indices,
    IntArrayRef output_size) {
  auto output = at::empty({0}, self.options());
  max_unpooling2d_forward_out_cuda(output, self, indices, output_size);
  return output;
}

static void max_unpooling3d_shape_check(
    const Tensor& input,
    const Tensor& gradOutput,
    const Tensor& indices,
    IntArrayRef output_size,
    IntArrayRef stride,
    IntArrayRef padding) {
  int64_t oT = output_size[0];
  int64_t oH = output_size[1];
  int64_t oW = output_size[2];
  TORCH_CHECK(
      indices.scalar_type() == at::ScalarType::Long,
      "elements in indices should be type int64");
  TORCH_CHECK(
      (input.ndimension() == 4 || input.ndimension() == 5),
      "Input to max_unpooling3d should be a 4d or 5d Tensor",
      input.sizes());
  TORCH_CHECK(
      output_size.size() == 3,
      "There should be exactly three elements (depth, height, width) in output_size");
  TORCH_CHECK(
      stride.size() == 3,
      "There should be exactly three elements (depth, height, width) in stride");
  TORCH_CHECK(
      padding.size() == 3,
      "There should be exactly three elements (depth, height, width) in padding");
  TORCH_CHECK(
      input.sizes() == indices.sizes(),
      "Shape of indices should match shape of input");

  TORCH_CHECK(input.numel() > 0, "Input must be non-empty");

  TORCH_CHECK(
      stride[0] > 0 && stride[1] > 0 && stride[2] > 0,
      "strides should be greater than zero, but got stride: ",
      stride);

  int dimw = 3;
  int dimh = 2;
  int dimt = 1;
  int dimn = 0;

  if (input.ndimension() == 5) {
    dimw++;
    dimh++;
    dimt++;
    dimn++;
  }

  int nslices = input.size(dimn);

  if (gradOutput.defined()) {
    if (oT != gradOutput.size(dimt) || oH != gradOutput.size(dimh) ||
        oW != gradOutput.size(dimw)) {
      AT_ERROR(
          "Inconsistent gradOutput size. oT= ",
          oT,
          ", oH= ",
          oH,
          ", oW= ",
          oW,
          ". gradOutput: ",
          gradOutput.size(dimt),
          "x",
          gradOutput.size(dimh),
          "x",
          gradOutput.size(dimw));
    }
    TORCH_CHECK(
        gradOutput.ndimension() == input.ndimension() &&
            gradOutput.size(dimn) == nslices,
        "gradOutput and input Tensors should have same number of dimensions and also the same number of channels/slices");
  }
}

Tensor& max_unpooling3d_forward_out_cuda(
    Tensor& output,
    const Tensor& self_,
    const Tensor& indices_,
    IntArrayRef output_size,
    IntArrayRef stride,
    IntArrayRef padding) {
  TORCH_CHECK(output.is_contiguous(), "output must be contiguous");
  max_unpooling3d_shape_check(
      self_, Tensor(), indices_, output_size, stride, padding);

  int64_t oT = output_size[0];
  int64_t oH = output_size[1];
  int64_t oW = output_size[2];

  TensorArg output_arg{output, "output", 1}, self_arg{self_, "self_", 2},
      indices_arg{indices_, "indices_", 3};
  checkAllSameGPU(
      "max_unpooling3d_forward_out_cuda", {output_arg, self_arg, indices_arg});

  auto self = self_.contiguous();
  auto indices = indices_.contiguous();

  int64_t batchSize;
  int64_t inputSlices;
  int64_t inputTime;
  int64_t inputHeight;
  int64_t inputWidth;

  if (self.ndimension() == 4) {
    batchSize = 1;
    inputSlices = self.size(0);
    inputTime = self.size(1);
    inputHeight = self.size(2);
    inputWidth = self.size(3);
    output.resize_({inputSlices, oT, oH, oW});
  } else {
    batchSize = self.size(0);
    inputSlices = self.size(1);
    inputTime = self.size(2);
    inputHeight = self.size(3);
    inputWidth = self.size(4);
    output.resize_({batchSize, inputSlices, oT, oH, oW});
  }

  output.zero_();

  // Collapse batch and feature dimensions if needed
  if (self.ndimension() == 5) {
    self = self.reshape({self.size(0) * self.size(1),
                         self.size(2),
                         self.size(3),
                         self.size(4)});
    indices = indices.reshape({indices.size(0) * indices.size(1),
                               indices.size(2),
                               indices.size(3),
                               indices.size(4)});
  }

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;
  dim3 block(32, 8);

  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half,
      self.scalar_type(), "max_unpooling3d_forward_kernel", ([&] {
        while (totalZ > 0) {
          dim3 grid(
              ceilDiv(inputWidth, static_cast<int64_t>(block.x)),
              ceilDiv(inputHeight, static_cast<int64_t>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);
          max_unpooling3d_forward_kernel<<<
              grid,
              block,
              0,
              at::cuda::getCurrentCUDAStream()>>>(
              self.packed_accessor64<scalar_t, 4>(),
              indices.packed_accessor64<int64_t, 4>(),
              output.data_ptr<scalar_t>(),
              oT,
              oH,
              oW,
              offsetZ);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          totalZ -= 65535;
          offsetZ += 65535;
        }
      }));
  return output;
}

Tensor max_unpooling3d_forward_cuda(
    const Tensor& self,
    const Tensor& indices,
    IntArrayRef output_size,
    IntArrayRef stride,
    IntArrayRef padding) {
  auto output = at::empty({0}, self.options());
  max_unpooling3d_forward_out_cuda(
      output, self, indices, output_size, stride, padding);
  return output;
}

at::Tensor& max_unpooling2d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output_,
    const Tensor& self_,
    const Tensor& indices_,
    IntArrayRef output_size) {
  int64_t oheight = output_size[0];
  int64_t owidth = output_size[1];
  TORCH_CHECK(grad_input.is_contiguous(), "grad_input must be contiguous");
  TORCH_CHECK(
      indices_.scalar_type() == at::ScalarType::Long,
      "elements in indices should be type int64");
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2},
      self_arg{self_, "self_", 3}, indices_arg{indices_, "indices_", 4};
  checkAllSameGPU(
      "max_unpooling2d_backward_out_cuda",
      {grad_input_arg, grad_output_arg, self_arg, indices_arg});

  TORCH_CHECK(
      (self_.ndimension() == 3 || self_.ndimension() == 4),
      "Input to max_unpooling2d should be a 3d or 4d Tensor, instead got: ",
      self_);

  TORCH_CHECK(
      self_.sizes() == indices_.sizes(),
      "Input should have same shape as indices");

  TORCH_CHECK(output_size.size() == 2, "output_size must have two elements");

  int64_t nInputCols, nInputRows, nInputPlane;

  int dimw = 2;
  int dimh = 1;

  auto self = self_.contiguous();
  auto indices = indices_.contiguous();
  auto grad_output = grad_output_.contiguous();

  if (self.ndimension() == 3) {
    nInputPlane = self.size(0);
  } else {
    ++dimw;
    ++dimh;
    nInputPlane = self.size(1);
  }

  nInputCols = self.size(dimw);
  nInputRows = self.size(dimh);

  if (oheight != grad_output.size(dimh) || owidth != grad_output.size(dimw)) {
    AT_ERROR(
        "Inconsistent gradOutput size. output height: ",
        oheight,
        ", output width= ",
        owidth,
        ", gradOutput: ",
        grad_output.size(dimh),
        "x",
        grad_output.size(dimw));
  }

  grad_input.resize_as_(self);
  grad_input.zero_();

  int64_t count = self.numel();

  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half,
      self.scalar_type(), "max_unpooling2d_backward_kernel", ([&] {
        max_unpooling2d_backward_kernel<<<
            GET_BLOCKS(count),
            CUDA_NUM_THREADS,
            0,
            at::cuda::getCurrentCUDAStream()>>>(
            count,
            grad_output.data_ptr<scalar_t>(),
            indices.data_ptr<int64_t>(),
            nInputPlane,
            nInputRows,
            nInputCols,
            oheight,
            owidth,
            grad_input.data_ptr<scalar_t>());
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }));
  return grad_input;
}
at::Tensor max_unpooling2d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& self,
    const Tensor& indices,
    IntArrayRef output_size) {
  auto grad_input = at::empty_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  max_unpooling2d_backward_out_cuda(
      grad_input, grad_output, self, indices, output_size);
  return grad_input;
}

at::Tensor& max_unpooling3d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output_,
    const Tensor& self_,
    const Tensor& indices_,
    IntArrayRef output_size,
    IntArrayRef stride,
    IntArrayRef padding) {
  TORCH_CHECK(grad_input.is_contiguous(), "grad_input must be contiguous");
  int64_t oT = output_size[0];
  int64_t oH = output_size[1];
  int64_t oW = output_size[2];

  max_unpooling3d_shape_check(
      self_, grad_output_, indices_, output_size, stride, padding);

  int batchSize = 0;
  int inputSlices = 0;
  int inputTime = 0;
  int64_t inputHeight = 0;
  int64_t inputWidth = 0;

  TensorArg self_arg{self_, "self_", 1}, indices_arg{indices_, "indices_", 2},
      grad_output_arg{grad_output_, "grad_output_", 3},
      grad_input_arg{grad_input, "grad_input", 4};
  checkAllSameGPU(
      "max_unpooling3d_backward_out_cuda",
      {self_arg, indices_arg, grad_output_arg, grad_input_arg});

  auto self = self_.contiguous();
  auto indices = indices_.contiguous();
  auto grad_output = grad_output_.contiguous();

  if (self.ndimension() == 4) {
    batchSize = 1;
    inputSlices = self.size(0);
    inputTime = self.size(1);
    inputHeight = self.size(2);
    inputWidth = self.size(3);
  } else {
    batchSize = self.size(0);
    inputSlices = self.size(1);
    inputTime = self.size(2);
    inputHeight = self.size(3);
    inputWidth = self.size(4);
  }

  grad_input.resize_as_(self);
  grad_input.zero_();

  // Collapse batch and feature dimensions if needed
  auto grad_input_reshaped = grad_input;
  if (grad_input.ndimension() == 5) {
    grad_input_reshaped =
        grad_input.reshape({grad_input.size(0) * grad_input.size(1),
                            grad_input.size(2),
                            grad_input.size(3),
                            grad_input.size(4)});

    indices = indices.reshape({indices.size(0) * indices.size(1),
                               indices.size(2),
                               indices.size(3),
                               indices.size(4)});
  }

  int totalZ = inputTime * inputSlices * batchSize;
  int offsetZ = 0;

  dim3 block(32, 8);

  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half,
      self.scalar_type(), "max_unpooling3d_backward_kernel", ([&] {
        while (totalZ > 0) {
          dim3 grid(
              ceilDiv(inputWidth, static_cast<int64_t>(block.x)),
              ceilDiv(inputHeight, static_cast<int64_t>(block.y)),
              totalZ > 65535 ? 65535 : totalZ);
          max_unpooling3d_backward_kernel<<<
              grid,
              block,
              0,
              at::cuda::getCurrentCUDAStream()>>>(
              grad_output.data_ptr<scalar_t>(),
              oT,
              oH,
              oW,
              indices.packed_accessor64<int64_t, 4>(),
              grad_input_reshaped.packed_accessor64<scalar_t, 4>(),
              offsetZ);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
          totalZ -= 65535;
          offsetZ += 65535;
        }
      }));
  return grad_input;
}

at::Tensor max_unpooling3d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& self,
    const Tensor& indices,
    IntArrayRef output_size,
    IntArrayRef stride,
    IntArrayRef padding) {
  auto grad_input = at::empty_like(self, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  max_unpooling3d_backward_out_cuda(
      grad_input, grad_output, self, indices, output_size, stride, padding);
  return grad_input;
}

} // namespace native
} // namespace at
