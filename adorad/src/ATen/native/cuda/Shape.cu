#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/MemoryOverlap.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/TypeProperties.h>
#include <ATen/Dispatch.h>
#include <c10/core/MemoryFormat.h>
#include <c10/util/Optional.h>

#include <THC/THC.h>

namespace at {
namespace native {

#ifdef __HIP_PLATFORM_HCC__
constexpr int CAT_ARRAY_BATCH_SIZE = 1024;
#else
constexpr int CAT_ARRAY_BATCH_SIZE = 128;
#endif
constexpr int CAT_ARRAY_MAX_INPUT_DIMS = 4;

namespace {

inline bool getCatGrid(ptrdiff_t nTensors, dim3& grid) {
  const int numSM = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

  //X dim of grid for cat array cooperates on a single tensor in the cat.
  //Given half of the GPU, full utilization will always occur.
  grid = dim3( 2LL * numSM, (long long) nTensors );

  return true;
}

// Similar to any other IndexToOffset calculation for copying along a given
// dimension.
template <typename IndexType, int Dims>
struct CatArrIndexToOffset {
  static inline __device__ IndexType compute(
      const IndexType tensorSize[Dims],
      const IndexType tensorStride[Dims],
      const IndexType dimSize,
      const unsigned int concatDim,
      IndexType linearIndex) {
    // linearIndex is not really linear index, but instead the offset in
    // input tensor. If the input tensor is contiguous, then this offset
    // is the linear index, but if the input tensor is channels last, then
    // it is the linear index of the permuted contiguous tensor
    IndexType offset = 0;

#pragma unroll
    for (int i = Dims - 1; i >= 1; --i) {
      IndexType curDimSize = i == concatDim ? dimSize : tensorSize[i];
      IndexType nextDimIndex = linearIndex / curDimSize;
      IndexType curDimIndex = linearIndex - curDimSize * nextDimIndex;
      IndexType curDimOffset = curDimIndex * tensorStride[i];
      offset += curDimOffset;
      linearIndex = nextDimIndex;
    }

    return offset + linearIndex * tensorStride[0];
  }
};

template<typename IndexType, unsigned int MaxDims>
struct TensorSizeStride {
  IndexType tensorSize[MaxDims];
  IndexType tensorStride[MaxDims];
};

/**
  * Kernel used to concatenated grimDim.y tensors into an output tensor. Uses a
  * grid-stride loop based off of the blockIdx.x, threadIdx.x for each input to
  * copy each element from each input tensor into the output.
  *
  * output: base pointer to the storage associated with the output tensor
  * inputs: GPU-allocated array of input metadata for each input to concatenate
  *         in the kernel
  * os: the size/stride vectors for the output tensor
  * concatDim: dimension along which we are concatenating
  * dimStride: the stride of the output tensor at the concatDim
  *
  * The most important assumption made is that the input tensors are contiguous.
  */


// Use pinned memory and and pass the struct by pointer on ROCm
template <typename T, typename IndexType>
struct CatArrInputTensor {
  T* input;
  IndexType offset;
  IndexType dimSize;
  IndexType nElements;
};

template <typename T, typename IndexType, int Dims>
C10_LAUNCH_BOUNDS_1(512)
__global__ void HIP_CatArrayBatchedCopy(
    T* output,
    CatArrInputTensor<T, IndexType>* inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs[blockIdx.y].nElements;

    if(tid >= nElements) return;

    T* data = inputs[blockIdx.y].input;
    IndexType offset = inputs[blockIdx.y].offset;
    IndexType dimSize = inputs[blockIdx.y].dimSize;
    IndexType dataOffset = offset * dimStride;

    IndexType stride = gridDim.x * blockDim.x;

    while( tid < nElements){
    IndexType elementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                  os.tensorSize, os.tensorStride, dimSize, concatDim, tid);
    output[dataOffset + elementOffset] = data[tid];

    tid += stride;
    }
}

// pass meta data directly through kernel argument instead of pin memory
// In contiguous case, we will not need stride_size, setting it as 1 as placeholder
// to pass compile.
template <typename T, typename IndexType, int n, int stride_size>
struct CatArrInputTensorMetadata {
  T* input[n];
  IndexType offset[n];
  IndexType dimSize[n];
  IndexType nElements[n];
  bool isContiguous[n];
  TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> tensorStride[stride_size];
};

template <typename T, typename IndexType, int Dims, int batch_size, int stride_size>
__global__ void CatArrayBatchedCopy(
    T* output,
    CatArrInputTensorMetadata<T, IndexType, batch_size, stride_size> inputs,
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> os,
    const int concatDim,
    IndexType dimStride) {

    IndexType tid = blockIdx.x * blockDim.x + threadIdx.x;
    IndexType nElements = inputs.nElements[blockIdx.y];
    TensorSizeStride<IndexType, CAT_ARRAY_MAX_INPUT_DIMS> ins = stride_size > 1 ? inputs.tensorStride[blockIdx.y] : inputs.tensorStride[0];
    bool isContig = inputs.isContiguous[blockIdx.y];

    if(tid >= nElements) return;

    T* data = inputs.input[blockIdx.y];
    IndexType offset = inputs.offset[blockIdx.y];
    IndexType dimSize = inputs.dimSize[blockIdx.y];
    IndexType dataOffset = offset * dimStride;

    IndexType stride = gridDim.x * blockDim.x;

    while( tid < nElements){
      IndexType elementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                    os.tensorSize, os.tensorStride, dimSize, concatDim, tid);
      if (isContig) {
        output[dataOffset + elementOffset] = data[tid];
      } else {
        IndexType inElementOffset = CatArrIndexToOffset<IndexType, Dims>::compute(
                    ins.tensorSize, ins.tensorStride, dimSize, concatDim, tid);
        output[dataOffset + elementOffset] = data[inElementOffset];
      }
    tid += stride;
    }
}

void check_shape_except_dim(const Tensor &first, const Tensor &second,
                            int dimension, int index)
{
  int first_dims = first.dim();
  int second_dims = second.dim();
  TORCH_CHECK(first_dims == second_dims,
      "Tensors must have same number of dimensions: got ", first_dims,
      " and ", second_dims);
  for (int dim = 0; dim < first_dims; dim++) {
    if (dim == dimension) {
      continue;
    }
    int64_t first_dim_size = at::native::size(first, dim);
    int64_t second_dim_size = at::native::size(second, dim);
    TORCH_CHECK(first_dim_size == second_dim_size,
        "Sizes of tensors must match except in dimension ", dim, ". Got ",
        static_cast<long long>(first_dim_size), " and ",
        static_cast<long long>(second_dim_size), " (The offending index is ",
        index, ")");
  }
}

template <typename scalar_t>
void hip_parallel_cat(Tensor &out, const TensorList &inputs, int64_t dimension,
                  int nDims, c10::MemoryFormat memory_format) {
  // First, let's set up our kernel parameters. We start with a raw pointer to
  // the storage for the output Tensor.
  scalar_t *data = out.data_ptr<scalar_t>();

  // Kernel Parameter
  long tensorMetadataSize =
    sizeof(CatArrInputTensor<scalar_t, unsigned int>) * CAT_ARRAY_BATCH_SIZE;
  auto d_inputs_storage = at::empty(
    {tensorMetadataSize}, out.options().dtype(at::kByte));
  auto d_inputs = static_cast<CatArrInputTensor<scalar_t, unsigned int> *>(
    d_inputs_storage.data_ptr());

  TensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> outputParam;

  // Next, let's initialize the size, stride arrays for the output Tensor.
  if (memory_format == c10::MemoryFormat::Contiguous) {
    for (int i = 0; i < nDims; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i);
      outputParam.tensorStride[i] = out.stride(i);
    }
  } else if (memory_format == c10::MemoryFormat::ChannelsLast || memory_format == c10::MemoryFormat::ChannelsLast3d) {
    // permute the semantics of dims from NCHW to NHWC so that the input
    // tensor is now contiguous
    outputParam.tensorSize[0] = at::native::size(out, 0);
    outputParam.tensorStride[0] = out.stride(0);
    for (int i = 1; i < nDims - 1; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i + 1);
      outputParam.tensorStride[i] = out.stride(i + 1);
    }
    outputParam.tensorSize[nDims - 1] = at::native::size(out, 1);
    outputParam.tensorStride[nDims - 1] = out.stride(1);
  } else {
    TORCH_CHECK(false, "unsupported memory format");
  }

  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

  // Now we loop
  int batchCounter = 0;
  int64_t offset = 0;
  for (int i = 0; i < inputs.size() ; i += CAT_ARRAY_BATCH_SIZE) {
    // Re-allocate stackInputs every iteration to avoid read-after-write hazard
    {
      auto stackInputs_storage = at::empty({tensorMetadataSize},
          out.options().dtype(at::kByte).device(at::kCPU).pinned_memory(true));
      auto stackInputs =
        static_cast<CatArrInputTensor<scalar_t, unsigned int> *>(
          stackInputs_storage.data_ptr());
      for (batchCounter = 0;
           batchCounter < CAT_ARRAY_BATCH_SIZE &&
             (i+batchCounter) < inputs.size();
           ++batchCounter) {
        int64_t dimSize = 0;
        // There is a legacy case where a 1-D empty tensor can be concat with
        // high-dimensional tensor
        if (inputs[i+batchCounter].numel() > 0) {
          dimSize = at::native::size(inputs[i+batchCounter], dimension);
        }

        stackInputs[batchCounter].input =
          inputs[i+batchCounter].data_ptr<scalar_t>();
        stackInputs[batchCounter].offset = offset;
        stackInputs[batchCounter].dimSize = dimSize;
        stackInputs[batchCounter].nElements = inputs[i+batchCounter].numel();

        // update offset
        offset += dimSize;
      }
      at::native::copy_(d_inputs_storage, stackInputs_storage,
                        /* non_blocking= */ true);
    }

    // Next, let's consider how we set our kernel launch parameters.
    // We borrow from THCApply, which the kernel's internal indexing
    // is based on.
    dim3 applyBlock = dim3(32*16);

    //Get grid where x dim fills half gpu and y dim is number of tensors.
    //This will have cating two tensors fill the entire grid, but prevent
    //many threads from needlessly load meta data if their sizes is small.
    dim3 catGrid;
    getCatGrid(batchCounter, catGrid);

    if (memory_format != c10::MemoryFormat::Contiguous) {
      switch (dimension) {
      case 0:
        break;
      case 1:
        dimension = nDims - dimension;
        break;
      default:
        dimension--;
      }
    }
    // Template Declarations for dim = 1, 2, 3, 4
#define HANDLE_CASE(DIMS) \
    HIP_CatArrayBatchedCopy<scalar_t, unsigned int, DIMS><<<\
        catGrid, applyBlock, 0, stream.stream()>>>(\
            data, d_inputs, outputParam, dimension, outputParam.tensorStride[dimension]); \
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    switch (nDims) {
      case 1:
        HANDLE_CASE(1);
        break;
      case 2:
        HANDLE_CASE(2);
        break;
      case 3:
        HANDLE_CASE(3);
        break;
      case 4:
        HANDLE_CASE(4);
        break;
    }
#undef HANDLE_CASE
  }
}

template <typename scalar_t, int batch_size, int stride_size>
void parallel_cat(Tensor &out, const TensorList &inputs, int64_t dimension,
                  int nDims, c10::MemoryFormat memory_format) {
  // First, let's set up our kernel parameters. We start with a raw pointer to
  // the storage for the output Tensor.
  scalar_t *data = out.data_ptr<scalar_t>();
  CatArrInputTensorMetadata<scalar_t, unsigned int, batch_size, stride_size> catMetaData;
  TensorSizeStride<unsigned int, CAT_ARRAY_MAX_INPUT_DIMS> outputParam;

  // Next, let's initialize the size, stride arrays for the output Tensor.
  if (memory_format == c10::MemoryFormat::Contiguous) {
    for (int i = 0; i < nDims; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i);
      outputParam.tensorStride[i] = out.stride(i);
    }
  } else if (memory_format == c10::MemoryFormat::ChannelsLast || memory_format == c10::MemoryFormat::ChannelsLast3d) {
    // permute the semantics of dims from NCHW to NHWC so that the input
    // tensor is now contiguous
    outputParam.tensorSize[0] = at::native::size(out, 0);
    outputParam.tensorStride[0] = out.stride(0);
    for (int i = 1; i < nDims - 1; ++i) {
      outputParam.tensorSize[i] = at::native::size(out, i + 1);
      outputParam.tensorStride[i] = out.stride(i + 1);
    }
    outputParam.tensorSize[nDims - 1] = at::native::size(out, 1);
    outputParam.tensorStride[nDims - 1] = out.stride(1);
  } else {
    TORCH_CHECK(false, "unsupported memory format");
  }

  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

  // Now we loop
  int batchCounter = 0;
  int64_t offset = 0;
  for (int i = 0; i < inputs.size() ; i += batch_size) {
    for (batchCounter = 0;
          batchCounter < batch_size &&
            (i+batchCounter) < inputs.size();
          ++batchCounter) {
      int64_t dimSize = 0;
      // There is a legacy case where a 1-D empty tensor can be concat with
      // high-dimensional tensor
      if (inputs[i+batchCounter].numel() > 0) {
        dimSize = at::native::size(inputs[i+batchCounter], dimension);
      }
      catMetaData.input[batchCounter] = inputs[i+batchCounter].data_ptr<scalar_t>();
      catMetaData.offset[batchCounter] = offset;
      catMetaData.dimSize[batchCounter] = dimSize;
      catMetaData.nElements[batchCounter] = inputs[i+batchCounter].numel();
      if (stride_size > 1) {
        auto strides = inputs[i+batchCounter].strides();
        auto sizes = inputs[i+batchCounter].sizes();
        for(int j = 0; j < nDims; j++){
          catMetaData.tensorStride[batchCounter].tensorSize[j] = sizes[j];
          catMetaData.tensorStride[batchCounter].tensorStride[j] = strides[j];
        }
        catMetaData.isContiguous[batchCounter] = false;
      } else {
        catMetaData.isContiguous[batchCounter] = true;
      }
      // update offset
      offset += dimSize;
    }
    // Next, let's consider how we set our kernel launch parameters.
    // We borrow from THCApply, which the kernel's internal indexing
    // is based on.
    dim3 applyBlock = dim3(32*16);

    //Get grid where x dim fills half gpu and y dim is number of tensors.
    //This will have cating two tensors fill the entire grid, but prevent
    //many threads from needlessly load meta data if their sizes is small.
    dim3 catGrid;
    getCatGrid(batchCounter, catGrid);

    if (memory_format != c10::MemoryFormat::Contiguous) {
      switch (dimension) {
      case 0:
        break;
      case 1:
        dimension = nDims - dimension;
        break;
      default:
        dimension--;
      }
    }
    // Template Declarations for dim = 1, 2, 3, 4
#define HANDLE_CASE(DIMS) \
    CatArrayBatchedCopy<scalar_t, unsigned int, DIMS, batch_size, stride_size><<<\
        catGrid, applyBlock, 0, stream.stream()>>>(\
            data, catMetaData, outputParam, dimension, outputParam.tensorStride[dimension]); \
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    switch (nDims) {
      case 1:
        HANDLE_CASE(1);
        break;
      case 2:
        HANDLE_CASE(2);
        break;
      case 3:
        HANDLE_CASE(3);
        break;
      case 4:
        HANDLE_CASE(4);
        break;
    }
#undef HANDLE_CASE
  }
}
} // namespace

Tensor cat_cuda(TensorList inputs, int64_t dimension) {
  ScalarType high_type = result_type(inputs);
  Tensor out = at::empty({0}, inputs.front().options().dtype(high_type));
  cat_out_cuda(out, inputs, dimension);
  return out;
}

inline c10::MemoryFormat compute_output_memory_format(const TensorList &inputs) {
  c10::optional<c10::MemoryFormat> format = c10::nullopt;
  for (auto &t : inputs) {
    auto f = t.suggest_memory_format();
    if (!format.has_value()) {
      format = f;
      continue;
    }
    if (format.value() == f) {
      continue;
    }
    bool contiguous = (format.value() == c10::MemoryFormat::Contiguous || f == c10::MemoryFormat::Contiguous || format.value() != f);
    if (contiguous) {
      return c10::MemoryFormat::Contiguous;
    }
  }
  return format.value();
}

Tensor& cat_out_cuda(Tensor& out, TensorList inputs, int64_t dimension) {

  // previously, size [0] tensors were the only possible empty tensors; thus, it
  // wasn't possible to cat empty tensors unless all the other tensors were
  // 1-dimensional, so we allowed these tensors to be "skipped".  We maintain
  // this behavior for backwards compatibility, but only for this specific size
  // (i.e. other empty sizes are not skipped).
  // FIXME: warn if this is the case
  auto should_skip = [](const Tensor &t) {
    return t.dim() == 1 && at::native::size(t, 0) == 0;
  };

  const Tensor *notSkippedTensor = NULL;  // non-owning reference
  int nDims = 0;

  // Check for type promotion
  TORCH_CHECK(canCast(result_type(inputs), out.scalar_type()), "input types ",
                      " can't be cast to the desired output type ",
                      out.scalar_type());

  // Inputs cannot alias the output tensor
  for (int i = 0; i < inputs.size(); i++) {
    auto lap = at::get_overlap_status(out, inputs[i]);
    TORCH_CHECK(lap != at::MemOverlapStatus::PARTIAL &&
                lap != at::MemOverlapStatus::FULL,
                "unsupported operation: the input tensors cannot refer to any "
                "of the output memory locations. Found overlap in input "
                "tensor ", i);
  }
  at::assert_no_internal_overlap(out);

  for (int i = 0; i < inputs.size(); i++) {
    if (should_skip(inputs[i])) {
      continue;
    }
    nDims = inputs[i].dim();
    notSkippedTensor = &inputs[i];
  }

  // If all inputs are empty tensors, return an empty tensor
  if (notSkippedTensor == NULL) {
    return out;
  }

  TORCH_CHECK(inputs.size() > 0, "invalid number of inputs ", inputs.size());
  TORCH_CHECK(dimension >= 0, "invalid dimension ", dimension);

  for (const Tensor& t: inputs) {
    TORCH_CHECK(t.device() == notSkippedTensor->device(),
                "All input tensors must be on the same device. Received ",
                t.device(), " and ", notSkippedTensor->device());
  }

  c10::MemoryFormat memory_format = compute_output_memory_format(inputs);

  std::vector<int64_t> size(notSkippedTensor->sizes().vec());

  // Compute size of the result in the cat dimension
  int64_t cat_dim_size = 0;
  for (int i = 0; i < inputs.size(); i++) {
    const Tensor &tensor = inputs[i];
    if (should_skip(tensor)) {
      continue;
    }
    check_shape_except_dim(*notSkippedTensor, tensor, dimension, i);
    cat_dim_size += at::native::size(tensor, dimension);
  }

  // Compute the size of the result
  size[dimension] = cat_dim_size;
  out.resize_(size, memory_format);
  if (out.numel() == 0) {
    return out;
  }

  // We parallelize the copy if all 6 conditions pass:
  //
  // 1. There is more than one input tensor
  // 2. The out tensor is 32-bit indexable
  // 3. The number of dimensions is <= 4
  // 4. All input tensors are contiguous (output tensor may be non-contig)
  // 5. All input tensors can use 32-bit indexing

  const bool all32BitIndexable = std::all_of(inputs.begin(), inputs.end(),
    [] (const Tensor& t) {
      return at::cuda::detail::canUse32BitIndexMath(t);
    });
  const bool allContiguous = std::all_of(inputs.begin(), inputs.end(),
    [=](const Tensor& t) {
      return !t.defined() || t.is_contiguous(memory_format);
    });
  ScalarType firstType = inputs[0].scalar_type();
  bool allSameType = std::all_of(inputs.begin(), inputs.end(),
    [firstType](const Tensor& t) {
      return t.scalar_type() == firstType;
    });
  allSameType = allSameType && (out.scalar_type() == firstType);

#ifdef __HIP_PLATFORM_HCC__
  if (inputs.size() > 1 &&
      out.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(out) &&
      allContiguous &&
      all32BitIndexable &&
      allSameType) {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
          out.scalar_type(), "cat_cuda", [&]() {
        hip_parallel_cat<scalar_t>(out, inputs, dimension, nDims, memory_format);
      });
#else
  // We support the contiguous inputs and non-contiguous input (<=4 dims) in different ways
  // For contiguous input, we don't need to pass stride meta data to cuda kernel through constant
  // memory. Therefore, we could pass more inputs to cuda threads.
  // For non-contiguous, we reduce the number of inputs passed to cuda kernel due to the limitation
  // of constant memory.
  if (inputs.size() > 1 &&
      out.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(out) &&
      allContiguous &&
      all32BitIndexable &&
      allSameType) {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
          out.scalar_type(), "cat_cuda", [&]() {
        parallel_cat<scalar_t, CAT_ARRAY_BATCH_SIZE, 1>(out, inputs, dimension, nDims, memory_format);
      });
  } else if (inputs.size() > 1 &&
      out.dim() <= CAT_ARRAY_MAX_INPUT_DIMS &&
      at::cuda::detail::canUse32BitIndexMath(out) &&
      nDims <= CAT_ARRAY_MAX_INPUT_DIMS &&
      all32BitIndexable &&
      allSameType) {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
          out.scalar_type(), "cat_cuda", [&]() {
        parallel_cat<scalar_t, CAT_ARRAY_BATCH_SIZE/2, CAT_ARRAY_BATCH_SIZE/2>(out, inputs, dimension, nDims, memory_format);
      });
#endif
  } else {
    int64_t offset = 0;
    for (int j = 0; j < inputs.size(); j++)
    {
      if (should_skip(inputs[j])) continue;
      int64_t dimSize = at::native::size(inputs[j], dimension);
      Tensor nt = at::narrow(out, dimension, offset, dimSize);
      copy_(nt, inputs[j]);
      offset += dimSize;
    }
  }

  return out;
}

} // namespace native
} // namespace at
