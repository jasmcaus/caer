#include "hip/hip_runtime.h"
#include <ATen/native/TensorAdvancedIndexing.h>

#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/core/Array.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/ExpandUtils.h>
#include <ATen/MemoryOverlap.h>
#include <THC/THCTensorInfo.cuh>

namespace at { namespace native {

static constexpr int launch_bound2 = 4;

static constexpr int launch_size_nd = 128;

template <int Dims, typename T, typename IndexType>
__device__ __forceinline__ IndexType indexToOffset(
    const cuda::detail::TensorInfo<T, IndexType>& info,
    int64_t index,
    IndexType size) {
  IndexType linearIndex = static_cast<IndexType>(index);
  CUDA_KERNEL_ASSERT(linearIndex < size && linearIndex >= -size);
  if (linearIndex < 0) {
    linearIndex += size;
  }
  return cuda::detail::IndexToOffset<T, IndexType, Dims>::get(linearIndex, info);
}

template<typename IndexType, typename T>
void dispatchTakePutImpl(const Tensor& input, Tensor& output, const Tensor& index) {
  auto inputInfo = cuda::detail::getTensorInfo<T, IndexType>(input);
  inputInfo.collapseDims();
  auto numel = input.numel();
  if (inputInfo.isContiguous()) {
    cuda::CUDA_tensor_apply2<T, int64_t>(
        output,
        index,
        [inputInfo, numel] __device__ (
            T & out, const int64_t& idx) {
            auto offset = indexToOffset<-2, T, IndexType>(inputInfo, idx, numel);
            out = inputInfo.data[offset];
        });
  } else {
    cuda::CUDA_tensor_apply2<T, int64_t>(
        output,
        index,
        [inputInfo, numel] __device__ (
            T & out, const int64_t& idx) {
            auto offset = indexToOffset<-1, T, IndexType>(inputInfo, idx, numel);
            out = inputInfo.data[offset];
        });
  }
}

template<typename T>
void dispatchTakePut(const Tensor& input, Tensor& output, const Tensor& index) {
  if (cuda::detail::canUse32BitIndexMath(input)) {
    dispatchTakePutImpl<int32_t, T>(input, output, index);
  } else {
    dispatchTakePutImpl<int64_t, T>(input, output, index);
  }
}

template<int nt, int vt, typename func_t>
C10_LAUNCH_BOUNDS_2(nt, launch_bound2)
__global__ void index_elementwise_kernel(int N, func_t f) {
  int tid = threadIdx.x;
  int nv = nt * vt;
  int idx = nv * blockIdx.x + tid;
  #pragma unroll
  for (int i = 0; i < vt; i++) {
    if (idx < N) {
      f(idx);
      idx += nt;
    }
  }
}

template<int nt, int vt, typename func_t>
static void launch_kernel(int64_t N, const func_t& f) {
  TORCH_INTERNAL_ASSERT(N >= 0 && N <= std::numeric_limits<int32_t>::max());
  if (N == 0) {
    return;
  }
  dim3 block(nt);
  dim3 grid((N + block.x * vt - 1) / (block.x * vt));
  auto stream = at::cuda::getCurrentCUDAStream();
  index_elementwise_kernel<nt, vt, func_t><<<grid, block, 0, stream>>>(N, f);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename func_t>
void gpu_index_kernel(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride, const func_t& f) {
  int num_indices = index_size.size();
  AT_ASSERT(num_indices == index_stride.size());
  AT_ASSERT(num_indices == iter.ntensors() - 2);

  if (iter.numel() == 0) {
    return;
  }

  if (!iter.can_use_32bit_indexing()) {
    for (auto& sub_iter : iter.with_32bit_indexing()) {
      gpu_index_kernel(sub_iter, index_size, index_stride, f);
    }
    return;
  }

  auto sizes = at::detail::Array<int64_t, 25>(0);
  auto strides = at::detail::Array<int64_t, 25>(0);
  auto index_ptrs = at::detail::Array<char*, 25>(nullptr);
  for (int i = 0; i < num_indices; i++) {
    sizes[i] = index_size[i];
    strides[i] = index_stride[i];
    index_ptrs[i] = (char*)iter.data_ptr(i + 2);
  }

  char* out_ptr = (char*)iter.data_ptr(0);
  char* in_ptr = (char*)iter.data_ptr(1);

  auto offset_calc = make_offset_calculator<3>(iter);
  launch_kernel<launch_size_nd, launch_bound2>(iter.numel(), [=]__device__(int idx) {
    auto offsets = offset_calc.get(idx);
    char* out_data = out_ptr + offsets[0];
    char* in_data = in_ptr + offsets[1];

    int64_t offset = 0;
    #pragma unroll
    for (int i = 0; i < num_indices; i++) {
      int64_t index = *(int64_t*)(index_ptrs[i] + offsets[2]);
      CUDA_KERNEL_ASSERT(index >= -sizes[i] && index < sizes[i] && "index out of bounds");
      if (index < 0) {
        index += sizes[i];
      }
      offset += index * strides[i];
    }

    f(out_data, in_data, offset);
  });
}

// The kernels are templated on an opaque, self-aligned type of the correct
// size to avoid redundant kernels for different types of the same size.
template <int N> struct alignas(N) OpaqueType { char data[N]; };


template <typename scalar_t>
void index_kernel_impl(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride) {
  gpu_index_kernel(iter, index_size, index_stride, []C10_DEVICE(char* out_data, char* in_data, int64_t offset) {
    *(scalar_t*)out_data = *(scalar_t*)(in_data + offset);
  });
}

template <typename scalar_t>
void index_put_kernel_impl(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride) {
  gpu_index_kernel(iter, index_size, index_stride, []C10_DEVICE(char* out_data, char* in_data, int64_t offset) {
    *(scalar_t*)(out_data + offset) = *(scalar_t*)in_data;
  });
}

static void index_kernel(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16, iter.dtype(), "index_cuda", [&] {
    using dtype = OpaqueType<sizeof(scalar_t)>;
    index_kernel_impl<dtype>(iter, index_size, index_stride);
  });
}


static void index_put_kernel(TensorIterator& iter, IntArrayRef index_size, IntArrayRef index_stride, bool accumulate) {
  TORCH_CHECK(!accumulate, "index_put does not support accumulate=true");
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16, iter.dtype(), "index_put", [&] {
    using dtype = OpaqueType<sizeof(scalar_t)>;
    index_put_kernel_impl<dtype>(iter, index_size, index_stride);
  });
}

static Tensor & masked_select_out_cuda_impl(Tensor & result, const Tensor & self, const Tensor & mask) {
  NoNamesGuard guard;

  TORCH_CHECK(mask.scalar_type() == ScalarType::Byte || mask.scalar_type() == ScalarType::Bool,
              "masked_select: expected BoolTensor or ByteTensor for mask");
  TORCH_CHECK(self.scalar_type() == result.scalar_type(),
              "masked_select(): self and result must have the same scalar type");

  Tensor _mask = (mask.dim() == 0) ? mask.unsqueeze(0) : mask;
  Tensor _self = (self.dim() == 0) ? self.unsqueeze(0) : self;
  std::tie(_mask, _self) = expand_outplace(_mask, _self);
  at::native::index_out(result, _self, _mask);

  return result;
}

Tensor masked_select_cuda(const Tensor & self, const Tensor & mask) {
  namedinference::compute_broadcast_outnames(self, mask);
  Tensor result = at::empty({0}, self.options());
  return masked_select_out_cuda_impl(result, self, mask);
}

Tensor & masked_select_out_cuda(Tensor & result, const Tensor & self, const Tensor & mask) {
  namedinference::compute_broadcast_outnames(self, mask);
  return masked_select_out_cuda_impl(result, self, mask);
}

void take_out_cuda_template(Tensor& output, const Tensor& input, const Tensor& index) {
  TORCH_CHECK(output.device().type() == at::kCUDA, "device type of output (", output.device().type(), ") is not GPU");
  TORCH_CHECK(input.device().type() == at::kCUDA, "device type of input (", input.device().type(), ") is not GPU");
  TORCH_CHECK(index.device().type() == at::kCUDA, "device type of index (", index.device().type(), ") is not GPU");

  TORCH_CHECK(output.layout() == Layout::Strided, "take() only supports strided layout, got layout: ", output.layout(), " on output tensor");
  TORCH_CHECK(input.layout() == Layout::Strided, "take() only supports strided layout, got layout: ", input.layout(), " on input tensor");
  TORCH_CHECK(index.layout() == Layout::Strided, "take() only supports strided layout, got layout: ", index.layout(), " on index tensor");

  TORCH_CHECK(output.scalar_type() == input.scalar_type(),
          "output and input scalar type must match. but got different types: ", output.scalar_type(), " and ", input.scalar_type());
  TORCH_CHECK(index.scalar_type() == kLong, "index must be an int64 tensor");

  TensorArg output_arg{ output, "output", 1 };
  TensorArg input_arg{ input, "input", 2 };
  TensorArg index_arg{ index, "index", 3 };
  checkAllSameGPU("take", {output_arg, input_arg, index_arg});

  TORCH_CHECK(input.dim() < MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);
  TORCH_CHECK(output.dim() < MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);
  TORCH_CHECK(index.dim() < MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);

  TORCH_CHECK(!(input.numel() == 0 && index.numel() != 0), "tried to take from an empty tensor");

  at::assert_no_internal_overlap(output);
  at::assert_no_partial_overlap(output, index);
  at::assert_no_overlap(output, input);

  output.resize_(index.sizes());

  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Bool, at::ScalarType::Half, input.scalar_type(), "take_cuda", [&] {
    dispatchTakePut<scalar_t>(input, output, index);
  });
}

Tensor take_cuda(const Tensor& self, const Tensor& index) {
    auto out = at::empty(index.sizes(), self.options());
    take_out_cuda_template(out, self, index);
    return out;
}

Tensor& take_out_cuda(Tensor& out, const Tensor& self, const Tensor& index) {
    take_out_cuda_template(out, self, index);
    return out;
}

REGISTER_DISPATCH(index_stub, &index_kernel);
REGISTER_DISPATCH(index_put_stub, &index_put_kernel);

}} // namespace at::native
