#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/TensorUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/WrapDimUtils.h>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>
#include <c10/macros/Macros.h>

#include <ATen/AccumulateType.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <type_traits>

#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/native/cuda/PersistentSoftmax.cuh>

namespace at {
namespace native {

namespace {

constexpr int ALIGN_BYTES = 16;

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxForwardEpilogue {
  __device__ __forceinline__ LogSoftMaxForwardEpilogue(AccumT max_input, AccumT sum)
    : max_input(max_input),  logsum(std::log(sum)) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(input - max_input - logsum);
}

  const AccumT max_input;
  const AccumT logsum;
};

template<typename T, typename AccumT, typename OutT>
struct LogSoftMaxBackwardEpilogue {
  __device__ __forceinline__ LogSoftMaxBackwardEpilogue(AccumT sum)
    : sum(sum) {}

  __device__ __forceinline__ T operator()(OutT gradOutput, OutT output) const {
    return static_cast<T>(gradOutput - std::exp(static_cast<AccumT>(output)) * sum);
  }

  const AccumT sum;
};

template<typename T, typename AccumT, typename OutT>
struct SoftMaxForwardEpilogue {
  __device__ __forceinline__ SoftMaxForwardEpilogue(AccumT max_input, AccumT sum)
    : max_input(max_input)
    , sum(sum) {}

  __device__ __forceinline__ OutT operator()(T input) const {
    return static_cast<OutT>(std::exp(input - max_input) / sum);
  }

  const AccumT max_input;
  const AccumT sum;
};

template<typename T, typename AccumT, typename OutT>
struct SoftMaxBackwardEpilogue {
  __device__ __forceinline__ SoftMaxBackwardEpilogue(AccumT sum)
    : sum(sum) {}

  // XXX: gradOutput that we get here is really gradOutput * output
  // Look for cmul in SoftMax_updateGradInput
  __device__ __forceinline__ T operator()(OutT gradOutput, OutT output) const {
    return static_cast<T>(gradOutput - output * sum);
  }

  const AccumT sum;
};




////////////////////////////////////////////////////////////////////////////////
// Spatial kernel (fast with large inner_size and small dim_size)
////////////////////////////////////////////////////////////////////////////////
// Let's assume that our input has been flattened to have only three dimension:
//     outer x dim x inner
// The spatial algorithm tries to parallelize along all of them.
// Within a 2d block threadIdx.y parallelizes over dim slices, and threads that
// share it will speed up reductions over dim (along axis x).
// The 2d grid is used to parallelize inner dimension over y axis and outer over x.
inline dim3 SpatialSoftMax_getGridSize(
    dim3 block, uint32_t max_active_blocks,
    uint64_t outer_size, uint64_t dim_size, uint64_t inner_size) {
  // First, tile as many blocks as we can over the y axis
  uint32_t inner_blocks = (inner_size + block.y - 1) / block.y;
  if (inner_blocks > max_active_blocks)
    inner_blocks = max_active_blocks;
  // Fill the x axis with as many blocks as we can fit (a little more is ok too)
  uint32_t outer_blocks = (max_active_blocks + inner_blocks - 1) / inner_blocks;
  if (outer_blocks > outer_size)
    outer_blocks = outer_size;
  return dim3(outer_blocks, inner_blocks);
}

const int max_threads = 1024;

inline dim3 SpatialSoftMax_getBlockSize(
  uint64_t outer_size, uint64_t dim_size, uint64_t inner_size) {
  uint32_t inner_threads = inner_size;
  inner_threads = std::min(inner_threads, static_cast<uint32_t>(max_threads));
  uint32_t dim_threads = 1;
  if (inner_threads <= 64 && dim_size >= 64) {
    while (inner_threads * dim_threads <= max_threads && dim_threads <= dim_size)
      dim_threads *= 2;
    dim_threads /= 2;
  }
  return dim3(dim_threads, inner_threads);
}


template<typename accscalar_t, typename Kernel>
void SpatialSoftMax_getLaunchSizes(
    Kernel k,
    uint64_t outer_size, uint64_t dim_size, uint64_t inner_size,
    dim3& grid, dim3& block, uint32_t& smem_size) {
  block = SpatialSoftMax_getBlockSize(outer_size, dim_size, inner_size);
  uint32_t block_threads = block.x * block.y;
  smem_size = block.x == 1 ? 0 : block_threads * sizeof(accscalar_t);
  int max_active_blocks;
#if defined(__HIP_PLATFORM_HCC__) && HIP_VERSION < 305
  // HIP function signature is not compatible yet.
  uint32_t max_blocks;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_blocks,
                                                k, block_threads, smem_size);
  max_active_blocks = max_blocks;
#else
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                                                k, block_threads, smem_size);
#endif
  max_active_blocks *= at::cuda::getCurrentDeviceProperties()->multiProcessorCount;
  grid = SpatialSoftMax_getGridSize(block, max_active_blocks, outer_size, dim_size, inner_size);
}

inline dim3 SoftMax_getBlockSize(int ILP, uint64_t dim_size) {
  uint64_t block_size = 1;
  uint64_t max_block_size = std::min(dim_size / ILP, static_cast<uint64_t>(max_threads));

  // In the vectorized case we want to trade off allowing more of the buffers to be accessed
  // in a vectorized way against wanting a larger block size to get better utilisation.
  // In general with ILP you can have (ILP-1)/ILP of the buffer accessed vectorised, at the risk
  // of having a very small block size. We choose to keep >= 1/2 of the buffer vectorised while
  // allowing a larger block size.
  if (ILP > 1) {
    max_block_size /= 2;
  }

  while (block_size < (max_block_size)) block_size *= 2;
  // Launch at least a single warp - the kernel assumes that.
  block_size = std::max(block_size, static_cast<uint64_t>(C10_WARP_SIZE));
  return dim3(block_size);
}

template<typename T>
struct Add {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a + b;
  }
};

template<typename T>
struct Max {
  __device__ __forceinline__ T operator()(T a, T b) const {
    return a < b ? b : a;
  }
};

// Note that it's not a complete block-wide reduction.
// Only threads that share threadIdx.y reduce values.
template<typename T, template<typename> class ReduceOp>
__forceinline__ __device__
T spatialBlockReduceX(T *shared, T val) {
  ReduceOp<T> r;
  shared += threadIdx.y * blockDim.x;

  __syncthreads();

  shared[threadIdx.x] = val;

  // NOTE: loop starts with __syncthreads()
  int offset = blockDim.x / 2;
  while (offset > 0) {
    __syncthreads();
    if (threadIdx.x < offset)
      shared[threadIdx.x] = r(shared[threadIdx.x], shared[threadIdx.x + offset]);
    offset /= 2;
  }

  __syncthreads();

  return shared[0];
}

template <typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void cunn_SpatialSoftMaxForward(
    outscalar_t *output, scalar_t *input,
    uint32_t outer_size, uint32_t dim_size, uint32_t inner_size)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  const uint32_t outer_stride = inner_size * dim_size;
  const uint32_t dim_stride = inner_size;

  for (uint32_t outer_index = blockIdx.x; outer_index < outer_size; outer_index += gridDim.x) {
    const uint32_t outer_offset = outer_index * outer_stride;
    for (uint32_t inner_index = blockIdx.y * blockDim.y + threadIdx.y; inner_index < inner_size; inner_index += blockDim.y * gridDim.y) {
      const uint32_t data_offset = outer_offset + inner_index;
      ////////////////////////////////////////////////////////////
      // These two blocks are really equivalent, but specializing on
      // blockDim.x == 1 makes the kernel faster when it's unused.
      // I didn't want to thread an extra template parameter, and nvcc
      // seems to be smart enough to hoist the if outside of the loops.
      ////////////////////////////////////////////////////////////

      if (blockDim.x > 1) {
        accscalar_t max_input = at::numeric_limits<accscalar_t>::lowest();
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          const accscalar_t value = static_cast<accscalar_t>(input[data_offset + d * dim_stride]);
          max_input = Max<accscalar_t>()(max_input, value);
        }
        max_input = spatialBlockReduceX<accscalar_t, Max>(sdata,max_input);

        accscalar_t sum = 0;
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += std::exp(static_cast<accscalar_t>(input[data_offset + d * dim_stride])
                 - max_input);
        sum = spatialBlockReduceX<accscalar_t, Add>(sdata, sum);

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_input, sum);
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          output[data_offset + d * dim_stride] = epilogue(input[data_offset + d * dim_stride]);
      } else {
        accscalar_t max_input = at::numeric_limits<accscalar_t>::lowest();
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          const accscalar_t value = static_cast<accscalar_t>(input[data_offset + d * dim_stride]);
          max_input = Max<accscalar_t>()(max_input, value);
        }
        accscalar_t sum = 0;
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += std::exp(static_cast<accscalar_t>(input[data_offset + d * dim_stride])
                 - max_input);
        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_input, sum);
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          output[data_offset + d * dim_stride] = epilogue(input[data_offset + d * dim_stride]);
      }
    }
  }
}



template <typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void cunn_SpatialSoftMaxBackward(
    scalar_t *gradInput, outscalar_t *output, outscalar_t *gradOutput,
    uint32_t outer_size, uint32_t dim_size, uint32_t inner_size)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  const uint32_t outer_stride = inner_size * dim_size;
  const uint32_t dim_stride = inner_size;

  for (uint32_t outer_index = blockIdx.x; outer_index < outer_size; outer_index += gridDim.x) {
    const uint32_t outer_offset = outer_index * outer_stride;
    for (uint32_t inner_index = blockIdx.y * blockDim.y + threadIdx.y; inner_index < inner_size; inner_index += blockDim.y * gridDim.y) {
      const uint32_t data_offset = outer_offset + inner_index;
      // See the comment in forward kernel
      if (blockDim.x > 1) {
        accscalar_t sum = 0;
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x)
          sum += gradOutput[data_offset + d * dim_stride];
        sum = spatialBlockReduceX<accscalar_t, Add>(sdata, sum);

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum);
        for (uint32_t d = threadIdx.x; d < dim_size; d += blockDim.x) {
          gradInput[data_offset + d * dim_stride] =
            epilogue(gradOutput[data_offset + d * dim_stride],
                    output[data_offset + d * dim_stride]);
        }
      } else {
        accscalar_t sum = 0;
        for (uint32_t d = 0; d < dim_size; d++)
          sum += gradOutput[data_offset + d * dim_stride];

        Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum);
        for (uint32_t d = 0; d < dim_size; d++) {
          gradInput[data_offset + d * dim_stride] =
            epilogue(gradOutput[data_offset + d * dim_stride],
                    output[data_offset + d * dim_stride]);
        }
      }
    }
  }
}


////////////////////////////////////////////////////////////////////////////////
// Regular kernel (fast when dim_size is large; requires inner_size == 1)
////////////////////////////////////////////////////////////////////////////////


template <typename T, typename AccumT>
struct MaxFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const {
    return ::max(max, (AccumT)v);
  }
};

template<typename T, typename AccumT>
struct AddFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + v;
  }
};

template<typename T, typename AccumT>
struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(AccumT v)
    : max_k(v) {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const {
    return sum + std::exp(v - max_k);
  }

  const AccumT max_k;
};

template <template<typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT
blockReduce(AccumT* smem, AccumT val,
            const Reduction<AccumT>& r,
            AccumT defaultVal)
{
  // To avoid RaW races from chaining blockReduce calls together, we need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  AccumT warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  uint32_t mask = (((uint64_t)1) << (blockDim.x / C10_WARP_SIZE)) - 1;
  if (threadIdx.x < C10_WARP_SIZE) {
    int lane = threadIdx.x % C10_WARP_SIZE;
    if (lane < blockDim.x / C10_WARP_SIZE) {
#pragma unroll
      for (int i = 0; i < C10_WARP_SIZE; ++i) {
        warpVal = r(warpVal, smem[lane * C10_WARP_SIZE + i]);
      }
#ifndef __HIP_PLATFORM_HCC__
      __syncwarp(mask);
#endif
      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;

  if (threadIdx.x == 0) {
    for (int i = 0; i < blockDim.x / C10_WARP_SIZE; ++i) {
      blockVal = r(blockVal, smem[i]);
    }
    smem[0] = blockVal;
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <template<typename, typename> class Reduction, int ILP, typename T, typename AccumT>
__device__ __forceinline__ AccumT
ilpReduce(int shift,
          T* data,
          int size,
          const Reduction<T, AccumT>& r,
          AccumT defaultVal)
{
  using LoadT = at::native::memory::aligned_vector<T, ILP>;
  AccumT threadVal = defaultVal;
  int offset = threadIdx.x;

  // shift and do 1
  if(shift > 0){
    data -= shift;
    size += shift;
    if(threadIdx.x >= shift){
      threadVal = r(threadVal, data[offset]);
    }
    size -= blockDim.x;
    data += blockDim.x;
  }
  int last = size % (ILP * blockDim.x);

  T v[ILP];
  LoadT* value = reinterpret_cast<LoadT*>(&v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *value = reinterpret_cast<LoadT*>(data)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      threadVal = r(threadVal, v[j]);
    }
  }

  offset = size - last + threadIdx.x;
  // Epilogue
  for (; offset < size; offset += blockDim.x)
    threadVal = r(threadVal, data[offset]);

  return threadVal;
}

/**
 * This will apply the Epilogue with vectorized reads & writes when input & output have the same shift
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteFpropResultsVectorized(
             int size,
             const int shift,
             scalar_t *input,
             outscalar_t *output,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  int offset = threadIdx.x;

  // if unaligned, do one value / thread and move on, guaranteeing aligned reads/writes later
  if (shift > 0) {
    input -= shift;
    output -= shift;
    size += shift;

    if (threadIdx.x >= shift) {
      output[offset] = epilogue(input[offset]);
    }
    size -= blockDim.x;
    input += blockDim.x;
    output += blockDim.x;
  }

  const int last = size % (ILP * blockDim.x);

  scalar_t in_v[ILP];
  LoadT* in_value = reinterpret_cast<LoadT*>(&in_v);

  outscalar_t out_v[ILP];
  StoreT* out_value = reinterpret_cast<StoreT*>(&out_v);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *in_value = reinterpret_cast<LoadT*>(input)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      out_v[j] = epilogue(in_v[j]);
    }

    reinterpret_cast<StoreT*>(output)[offset] = *out_value;
  }

  offset = size - last + threadIdx.x;
  // handle the tail
  for (; offset < size; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteBpropResultsVectorized(
             int size,
             const int shift,
             scalar_t *gradInput,
             outscalar_t *output,
             outscalar_t *gradOutput,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  using gradInputT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using outputT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  int offset = threadIdx.x;

  // if unaligned, do one value / thread and move on, guaranteeing aligned reads/writes later
  if (shift > 0) {
    gradInput -= shift;
    output -= shift;
    gradOutput -= shift;
    size += shift;

    if (threadIdx.x >= shift) {
      gradInput[offset] = epilogue(gradOutput[offset], output[offset]);
    }
    size -= blockDim.x;
    gradInput += blockDim.x;
    output += blockDim.x;
    gradOutput += blockDim.x;
  }

  const int last = size % (ILP * blockDim.x);

  scalar_t dX[ILP];
  gradInputT *dX_v = reinterpret_cast<gradInputT*>(&dX);

  outscalar_t Y[ILP];
  outputT *Y_v = reinterpret_cast<outputT*>(&Y);

  outscalar_t dY[ILP];
  outputT *dY_v = reinterpret_cast<outputT*>(&dY);

  for (; offset * ILP < (size - last); offset += blockDim.x) {
    *Y_v = reinterpret_cast<outputT*>(output)[offset];
    *dY_v = reinterpret_cast<outputT*>(gradOutput)[offset];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      dX[j] = epilogue(dY[j], Y[j]);
    }

    reinterpret_cast<gradInputT*>(gradInput)[offset] = *dX_v;
  }

  offset = size - last + threadIdx.x;
  for (; offset < size; offset += blockDim.x) {
    gradInput[offset] = epilogue(gradOutput[offset], output[offset]);
  }
}

/**
 * This will apply the Epilogue with non-vectrorized reads & writes for the general case
 */
template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteFpropResults(
             int classes,
             scalar_t *input,
             outscalar_t *output,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {
  int offset = threadIdx.x;

  int last = classes % (ILP * blockDim.x);

  // Main bulk of loop with ILP
  for (; offset < classes - last; offset += blockDim.x * ILP) {
    scalar_t tmp[ILP];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * blockDim.x];
    }
    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      output[offset + j * blockDim.x] = epilogue(tmp[j]);
    }
  }

  // Remainder - no ILP
  for (; offset < classes; offset += blockDim.x) {
    output[offset] = epilogue(input[offset]);
  }
}

template <int ILP, typename scalar_t, typename accum_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__device__ __forceinline__ void
WriteBpropResults(
             int classes,
             scalar_t *gradInput,
             outscalar_t *output,
             outscalar_t *gradOutput,
             Epilogue<scalar_t, accum_t, outscalar_t> epilogue) {

  int offset = threadIdx.x;

  int last = classes % (ILP * blockDim.x);

  for (; offset < classes - last; offset += blockDim.x * ILP) {
    outscalar_t tmpOutput[ILP];
    outscalar_t tmpGradOutput[ILP];

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmpOutput[j] = output[offset + j * blockDim.x];
      tmpGradOutput[j] = gradOutput[offset + j * blockDim.x];
    }

    #pragma unroll
    for (int j = 0; j < ILP; ++j) {
      gradInput[offset + j * blockDim.x] = epilogue(tmpGradOutput[j], tmpOutput[j]);
    }
  }

  // Remainder - no ILP
  for (; offset < classes; offset += blockDim.x) {
    gradInput[offset] = epilogue(gradOutput[offset], output[offset]);
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template <typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxForward(outscalar_t *output, scalar_t *input, int classes)
{
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);

  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * classes;
  output += blockIdx.x * classes;

  const int shift = ((uint64_t)input) % ALIGN_BYTES / sizeof(scalar_t);
  const int output_shift = ((uint64_t)output) % ALIGN_BYTES / sizeof(outscalar_t);

  // find the max
  accscalar_t threadMax = ilpReduce<MaxFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, MaxFloat<scalar_t, accscalar_t>(), -at::numeric_limits<accscalar_t>::max());
  accscalar_t max_k = blockReduce<Max, accscalar_t>(
      sdata, threadMax, Max<accscalar_t>(), -at::numeric_limits<accscalar_t>::max());

  // reduce all values
  accscalar_t threadExp = ilpReduce<SumExpFloat, ILP, scalar_t, accscalar_t>(
      shift, input, classes, SumExpFloat<scalar_t, accscalar_t>(max_k), static_cast<accscalar_t>(0));
  accscalar_t sumAll = blockReduce<Add, accscalar_t>(
      sdata, threadExp, Add<accscalar_t>(), static_cast<accscalar_t>(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(max_k, sumAll);

  if (shift == output_shift) {
    WriteFpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, shift, input, output, epilogue);
  } else {
    WriteFpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, input, output, epilogue);
  }
}

template <int ILP, typename scalar_t, typename accscalar_t, typename outscalar_t, template<typename, typename, typename> class Epilogue>
__global__ void
cunn_SoftMaxBackward(scalar_t *gradInput, outscalar_t *output, outscalar_t *gradOutput, int classes)
{
  using LoadT = at::native::memory::aligned_vector<scalar_t, ILP>;
  using StoreT = at::native::memory::aligned_vector<outscalar_t, ILP>;

  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);
  gradInput += blockIdx.x * classes;
  output += blockIdx.x * classes;
  gradOutput += blockIdx.x * classes;

  const int shift = ((uint64_t)gradInput) % ALIGN_BYTES / sizeof(scalar_t);
  const int output_shift = ((uint64_t)output) % ALIGN_BYTES / sizeof(outscalar_t);
  const int grad_output_shift = ((uint64_t)gradOutput) % ALIGN_BYTES / sizeof(outscalar_t);

  accscalar_t threadSum = ilpReduce<AddFloat, ILP, outscalar_t, accscalar_t>(
      shift, gradOutput, classes, AddFloat<outscalar_t, accscalar_t>(), accscalar_t(0));
  accscalar_t sum_k = blockReduce<Add, accscalar_t>(
        sdata, threadSum, Add<accscalar_t>(), accscalar_t(0));

  Epilogue<scalar_t, accscalar_t, outscalar_t> epilogue(sum_k);

  if (shift == output_shift && shift == grad_output_shift) {
    WriteBpropResultsVectorized<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, shift, gradInput, output, gradOutput, epilogue);
  } else {
    WriteBpropResults<ILP, scalar_t, accscalar_t, outscalar_t, Epilogue>(classes, gradInput, output, gradOutput, epilogue);
  }
}

template<template<typename, typename, typename> class Epilogue, bool is_log_softmax>
Tensor host_softmax(const Tensor & input_, const int64_t dim_, const bool half_to_float){
  if (half_to_float) {
    TORCH_CHECK(input_.scalar_type() == ScalarType::Half, "conversion is supported for Half type only");
  }
  auto input = input_.contiguous();
  Tensor output = half_to_float ? at::empty_like(input, input.options().dtype(ScalarType::Float), LEGACY_CONTIGUOUS_MEMORY_FORMAT) : at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  static_assert(std::is_same<acc_type<at::Half, true>, float>::value, "accscalar_t for half should be float");
  if (input.dim() == 0) input = input.view(1);
  int64_t dim = maybe_wrap_dim(dim_, input.dim());
  TORCH_CHECK(dim >=0 && dim < input.dim(), "dim must be non-negative and less than input dimensions");
  int64_t outer_size = 1;
  int64_t dim_size = input.size(dim);

  if (input.numel() > 0) {
    int64_t inner_size = 1;
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    for (int64_t i = 0; i < dim; ++i)
      outer_size *= input.size(i);
    for (int64_t i = dim + 1; i < input.dim(); ++i)
      inner_size *= input.size(i);
    // This kernel spawns a block per each element in the batch.
    // XXX: it assumes that inner_size == 1

    if (inner_size == 1) {
      dim3 grid(outer_size);
      AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "host_softmax", [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        if (!half_to_float) {
          if (dim_size <= 1024 && dim_size*sizeof(scalar_t) <= 4096) {
            dispatch_softmax_forward<scalar_t, scalar_t, accscalar_t, is_log_softmax>(
                output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), dim_size, dim_size, outer_size);
          } else {
            constexpr int ILP = sizeof(float4) / sizeof(scalar_t);
            dim3 block = SoftMax_getBlockSize(ILP, dim_size);
            cunn_SoftMaxForward<ILP, scalar_t, accscalar_t, scalar_t, Epilogue>
              <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
                output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), dim_size);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }
        } else {
          if (dim_size <= 1024 && dim_size*sizeof(scalar_t) <= 4096) {
            dispatch_softmax_forward<scalar_t, accscalar_t, accscalar_t, is_log_softmax>(
                output.data_ptr<accscalar_t>(), input.data_ptr<scalar_t>(), dim_size, dim_size, outer_size);
          } else {
            constexpr int ILP = sizeof(float4) / sizeof(accscalar_t);
            dim3 block = SoftMax_getBlockSize(ILP, dim_size);
            cunn_SoftMaxForward<ILP, scalar_t, accscalar_t, accscalar_t, Epilogue>
              <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
                output.data_ptr<accscalar_t>(), input.data_ptr<scalar_t>(), dim_size);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
          }
        }
      });
    // This kernel runs in a 2D grid, where each application along y dimension has a fixed
    // outer_size, and runs in parallel over inner_size. Dimension x is parallel over outer_size.
    // Reductions over dim are done in a single-threaded manner.
    } else {
      uint32_t smem_size;
      dim3 grid, block;
      AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "host_softmax", [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        if (!half_to_float) {
            SpatialSoftMax_getLaunchSizes<accscalar_t>(
                &cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, scalar_t, Epilogue>,
                outer_size, dim_size, inner_size,
                grid, block, smem_size);
            cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, scalar_t, Epilogue>
              <<<grid, block, smem_size, stream>>>(
              output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), outer_size, dim_size, inner_size);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        } else {
            SpatialSoftMax_getLaunchSizes<accscalar_t>(
                &cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, accscalar_t, Epilogue>,
                outer_size, dim_size, inner_size,
                grid, block, smem_size);
            cunn_SpatialSoftMaxForward<scalar_t, accscalar_t, accscalar_t, Epilogue>
              <<<grid, block, smem_size, stream>>>(
              output.data_ptr<accscalar_t>(), input.data_ptr<scalar_t>(), outer_size, dim_size, inner_size);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }
      });
    }
  }
  return output;
}

template<template<typename, typename, typename> class Epilogue, bool is_log_softmax>
Tensor host_softmax_backward(const Tensor &grad_, const Tensor &output_, int64_t dim_, bool half_to_float){
  int64_t dim = maybe_wrap_dim(dim_, grad_.dim());
  Tensor gI = half_to_float ? at::empty_like(grad_, grad_.options().dtype(ScalarType::Half), LEGACY_CONTIGUOUS_MEMORY_FORMAT) : at::empty_like(grad_, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  if (grad_.numel() == 0) {
    return gI;
  }
  auto grad = grad_.contiguous();
  static_assert(std::is_same<acc_type<at::Half, true>, float>::value, "accscalar_t for half should be float");
  if (grad.dim() == 0) grad = grad.view(1);
  TORCH_CHECK(dim >=0 && dim < grad.dim(), "dim must be non-negative and less than input dimensions");
  auto output = output_.contiguous();
  if (output.dim() == 0) output = output.view(1);
  int64_t outer_size = 1;
  int64_t dim_size = output.size(dim);
  int64_t inner_size = 1;
  for (int64_t i = 0; i < dim; ++i)
    outer_size *= output.size(i);
  for (int64_t i = dim + 1; i < output.dim(); ++i)
    inner_size *= output.size(i);
// See descriptions of kernels above.
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  if (inner_size == 1) {
    dim3 grid(outer_size);
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, gI.scalar_type(), "host_softmax_backward", [&] {
    using accscalar_t = acc_type<scalar_t, true>;
    if (!half_to_float) {
      if (dim_size <= 1024 && dim_size*sizeof(scalar_t) <= 4096) {
        dispatch_softmax_backward<scalar_t, scalar_t, accscalar_t, is_log_softmax>(
            gI.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), dim_size, dim_size, outer_size);
      } else {
        constexpr int ILP = sizeof(float4) / sizeof(scalar_t);
        dim3 block = SoftMax_getBlockSize(ILP, dim_size);
        cunn_SoftMaxBackward<ILP, scalar_t, accscalar_t, scalar_t, Epilogue>
         <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
            gI.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(), dim_size
        );
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    } else {
      if (dim_size <= 1024 && dim_size*sizeof(scalar_t) <= 4096) {
        dispatch_softmax_backward<accscalar_t, scalar_t, accscalar_t, is_log_softmax>(
            gI.data_ptr<scalar_t>(), grad.data_ptr<accscalar_t>(), output.data_ptr<accscalar_t>(), dim_size, dim_size, outer_size);
      } else {
        constexpr int ILP = sizeof(float4) / sizeof(accscalar_t);
        dim3 block = SoftMax_getBlockSize(ILP, dim_size);
        cunn_SoftMaxBackward<ILP, scalar_t, accscalar_t, accscalar_t, Epilogue>
         <<<grid, block, block.x * sizeof(accscalar_t), stream>>>(
            gI.data_ptr<scalar_t>(), output.data_ptr<accscalar_t>(), grad.data_ptr<accscalar_t>(), dim_size
        );
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    }
    });
  } else {
    uint32_t smem_size;
    dim3 grid, block;
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, gI.scalar_type(), "host_softmax_backward", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      if (!half_to_float) {
          SpatialSoftMax_getLaunchSizes<accscalar_t>(
              &cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, scalar_t, Epilogue>,
              outer_size, dim_size, inner_size,
              grid, block, smem_size);

          cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, scalar_t, Epilogue>
            <<<grid, block, smem_size, stream>>>(
              gI.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), grad.data_ptr<scalar_t>(),
              outer_size, dim_size, inner_size
          );
          C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
          SpatialSoftMax_getLaunchSizes<accscalar_t>(
              &cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, accscalar_t, Epilogue>,
              outer_size, dim_size, inner_size,
              grid, block, smem_size);

          cunn_SpatialSoftMaxBackward<scalar_t, accscalar_t, accscalar_t, Epilogue>
            <<<grid, block, smem_size, stream>>>(
              gI.data_ptr<scalar_t>(), output.data_ptr<accscalar_t>(), grad.data_ptr<accscalar_t>(),
              outer_size, dim_size, inner_size
          );
          C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }

  return gI;
}
}

Tensor log_softmax_cuda(const Tensor &input, const int64_t dim, const bool half_to_float){
  return host_softmax<LogSoftMaxForwardEpilogue,true>(input, dim, half_to_float);
}

Tensor log_softmax_backward_cuda(const Tensor &grad, const Tensor &output, int64_t dim, const Tensor &input){
  bool half_to_float = grad.scalar_type() != input.scalar_type();
  if (half_to_float) {
     TORCH_CHECK((grad.scalar_type() == ScalarType::Float && input.scalar_type() == ScalarType::Half),
                 "expected input and grad types to match, or input to be at::Half and grad to be at::Float");
  }
  return host_softmax_backward<LogSoftMaxBackwardEpilogue,true>(grad, output, dim, half_to_float);
}

Tensor softmax_cuda(const Tensor &input, const int64_t dim, const bool half_to_float){
  return host_softmax<SoftMaxForwardEpilogue,false>(input, dim, half_to_float);
}

Tensor softmax_backward_cuda(const Tensor &grad, const Tensor &output, int64_t dim, const Tensor &input){
  bool half_to_float = grad.scalar_type() != input.scalar_type();
  if (half_to_float) {
     TORCH_CHECK((grad.scalar_type() == ScalarType::Float && input.scalar_type() == ScalarType::Half),
                 "expected input and grad types to match, or input to be at::Half and grad to be at::Float");
  }
  Tensor tmp = grad * output;
  return host_softmax_backward<SoftMaxBackwardEpilogue,false>(tmp, output, dim, half_to_float);
}

}
}
