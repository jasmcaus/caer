#include "hip/hip_runtime.h"
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <THC/THCNumerics.cuh>
#include <ATen/native/ReduceOps.h>
#include<ATen/native/ReduceAllOps.h>
#include <ATen/native/ReduceOpsUtils.h>
#include <ATen/native/TensorCompare.h>


namespace at { namespace native {

template <typename acc_t>
struct MaxNanFunctor {
  __device__ __forceinline__ acc_t operator()(acc_t a, acc_t b) const {
      return (THCNumerics<acc_t>::isnan(a) || a > b) ? a : b;
  }
};

template <typename scalar_t, typename acc_t=scalar_t>
void max_values_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(
    iter, func_wrapper<acc_t> (MaxNanFunctor<acc_t>()),
    at::numeric_limits<acc_t>::lower_bound());
}

template <typename acc_t>
struct MinNanFunctor {
  __device__ __forceinline__ acc_t operator()(acc_t a, acc_t b) const {
      return (THCNumerics<acc_t>::isnan(a) || a < b) ? a : b;
  }
};

template <typename scalar_t, typename acc_t=scalar_t>
void min_values_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(
    iter, func_wrapper<acc_t> (MinNanFunctor<acc_t>()),
    at::numeric_limits<acc_t>::upper_bound());
}

void max_values_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.dtype(), "max_values_cuda", [&]() {
    max_values_kernel_cuda_impl<scalar_t>(iter);
  });
}

void min_values_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.dtype(), "min_values_cuda", [&]() {
    min_values_kernel_cuda_impl<scalar_t>(iter);
  });
}

template <typename scalar_t, typename acc_t=scalar_t>
void argmax_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, int64_t>(
    iter,
    ArgMaxOps<acc_t>{},
    thrust::pair<acc_t, int64_t>(at::numeric_limits<acc_t>::lower_bound(), 0));
};

template <typename scalar_t, typename acc_t=scalar_t>
void argmin_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, int64_t>(
    iter,
    ArgMinOps<acc_t>{},
    thrust::pair<acc_t, int64_t>(at::numeric_limits<acc_t>::upper_bound(), 0));
};

void argmax_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    // Instead of implementing is_nan and warp_shfl_down
    // we can convert halves to float and do all the operations in float
    argmax_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(1), "argmax_cuda", [&]() {
      argmax_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

void argmin_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    // Instead of implementing is_nan and warp_shfl_down
    // we can convert halves to float and do all the operations in float
    argmin_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(1), "argmin_cuda", [&]() {
      argmin_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

static void min_kernel_impl(Tensor& result, Tensor& indice, const Tensor& self, int64_t dim, bool keepdim) {
  at::TensorIterator iter = make_reduction("min", result, indice, self, dim, keepdim, self.scalar_type(), kLong);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.dtype(2), "min_cuda", [&]() {
    gpu_reduce_kernel<scalar_t, scalar_t>(
      iter,
      MinOps<scalar_t>{},
      thrust::pair<scalar_t, int64_t>(at::numeric_limits<scalar_t>::upper_bound(), 0));
  });
}

static void max_kernel_impl(Tensor& result, Tensor& indice, const Tensor& self, int64_t dim, bool keepdim) {
  at::TensorIterator iter = make_reduction("max", result, indice, self, dim, keepdim, self.scalar_type(), kLong);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.dtype(2), "max_cuda", [&]() {
    gpu_reduce_kernel<scalar_t, scalar_t>(
      iter,
      MaxOps<scalar_t>{},
      thrust::pair<scalar_t, int64_t>(at::numeric_limits<scalar_t>::lower_bound(), 0));
  });
}

static void _aminmax_kernel_impl(
    Tensor& min_result,
    Tensor& max_result,
    const Tensor& self,
    int64_t dim,
    bool keepdim) {
  at::TensorIterator iter = make_reduction("_aminmax", min_result, 
    max_result, self, dim, keepdim, self.scalar_type());
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, self.scalar_type(), "_aminmax_cuda", [&]() {
    gpu_reduce_kernel<scalar_t, scalar_t>(
      iter,
      MinMaxOps<scalar_t, scalar_t, int32_t>{},
      thrust::pair<scalar_t, scalar_t>(
        at::numeric_limits<scalar_t>::upper_bound(), 
        at::numeric_limits<scalar_t>::lower_bound()
      )
    );
  });
}

static void min_all_kernel_impl(Tensor& result, const Tensor& input) {
  auto dtype = input.scalar_type();
  auto iter = make_reduction("min_all", result, input, std::vector<int64_t>{}, false, dtype);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, dtype, "min_all_cuda", [&] {
    min_values_kernel_cuda_impl<scalar_t>(iter);
  });
}

static void max_all_kernel_impl(Tensor& result, const Tensor& input) {
  auto dtype = input.scalar_type();
  auto iter = make_reduction("min_all", result, input, std::vector<int64_t>{}, false, dtype);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, dtype, "max_all_cuda", [&] {
    max_values_kernel_cuda_impl<scalar_t>(iter);
  });
}

template <typename scalar_t>
void _min_max_values_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(
    iter, MinMaxOps<scalar_t, scalar_t, int32_t>{}, thrust::pair<scalar_t, scalar_t>(
      at::numeric_limits<scalar_t>::upper_bound(),
      at::numeric_limits<scalar_t>::lower_bound()
  ));
}

void _aminmax_all_kernel_impl(Tensor& min_result, Tensor& max_result, const Tensor& input) {
  auto dtype = input.scalar_type();
  auto iter = make_reduction("_aminmax_all", min_result, max_result, input,
                             std::vector<int64_t>{}, false, dtype);
  TORCH_CHECK(iter.numel() > 0, "min_max on a tensor with no elements is not defined.");
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, dtype, "_aminmax_all_cuda", [&] {
    _min_max_values_kernel_cuda_impl<scalar_t>(iter);
  });
}

REGISTER_DISPATCH(max_values_stub, &max_values_kernel_cuda);
REGISTER_DISPATCH(min_values_stub, &min_values_kernel_cuda);
REGISTER_DISPATCH(argmax_stub, &argmax_kernel_cuda);
REGISTER_DISPATCH(argmin_stub, &argmin_kernel_cuda);
REGISTER_DISPATCH(min_stub, &min_kernel_impl);
REGISTER_DISPATCH(max_stub, &max_kernel_impl);
REGISTER_DISPATCH(_aminmax_stub, &_aminmax_kernel_impl);
REGISTER_DISPATCH(min_all_stub, &min_all_kernel_impl);
REGISTER_DISPATCH(max_all_stub, &max_all_kernel_impl);
REGISTER_DISPATCH(_aminmax_all_stub, &_aminmax_all_kernel_impl);

}} // namespace at::native
