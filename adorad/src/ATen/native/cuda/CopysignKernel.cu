#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

#if defined(__HIPCC__)
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <c10/cuda/CUDAMathCompat.h>
#elif defined(__HIPCC__)
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <c10/hip/HIPMathCompat.h>
#endif

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void copysign_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(kBFloat16, kHalf, iter.common_dtype(), "copysign_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return c10::cuda::compat::copysign(a, b);
    });
  });
}

REGISTER_DISPATCH(copysign_stub, &copysign_kernel_cuda);

}} // namespace at::native
