#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>
#include <THC/THCAtomics.cuh>

namespace at {
namespace native {
namespace {

template <typename scalar_t, typename accscalar_t>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void upsample_linear1d_out_frame(
    const int n,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor64<scalar_t, 3> idata,
    PackedTensorAccessor64<scalar_t, 3> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int width1 = idata.size(2);
  const int width2 = odata.size(2);

  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][w1];
          odata[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val =
            w0lambda * idata[n][c][w1] + w1lambda * idata[n][c][w1 + w1p];
        odata[n][c][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(1024)
#endif
__global__ void upsample_linear1d_out_frame_backward(
    const int n,
    const accscalar_t rwidth,
    const bool align_corners,
    PackedTensorAccessor64<scalar_t, 3> idata,
    const PackedTensorAccessor64<scalar_t, 3> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int width1 = idata.size(2);
  const int width2 = odata.size(2);

  if (index < n) {
    const int w2 = index % width2;
    // special case: just copy
    if (width1 == width2) {
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = odata[n][c][w1];
          idata[n][c][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const scalar_t d2val = odata[n][c][w2];
        gpuAtomicAdd(&idata[n][c][w1], static_cast<scalar_t>(w0lambda * d2val));
        gpuAtomicAdd(
            &idata[n][c][w1 + w1p], static_cast<scalar_t>(w1lambda * d2val));
      }
    }
  }
}

static void upsample_linear1d_out_cuda_template(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_linear1d_out_cuda", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 1,
      "It is expected output_size equals to 1, but got size ",
      output_size.size());

  int output_width = output_size[0];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_width = input.size(2);

  upsample_1d_shape_check(
      input, Tensor(), nbatch, channels, input_width, output_width);

  output.resize_({input.size(0), input.size(1), output_width});
  output.zero_();

  AT_ASSERT(input_width > 0 && output_width > 0);

  const int num_kernels = output_width;
  const int num_threads =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_linear1d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<scalar_t, 3>();
        auto odata = output.packed_accessor64<scalar_t, 3>();

        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
          input_width, output_width, align_corners, scales);

        upsample_linear1d_out_frame<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(num_kernels, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_linear1d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales) {
  TensorArg grad_output_arg{grad_output_, "grad_output_", 1},
      grad_input_arg{grad_input, "grad_input", 2};
  checkAllSameGPU(
      "upsample_linear1d_backward_out_cuda", {grad_output_arg, grad_input_arg});

  TORCH_CHECK(
      output_size.size() == 1,
      "It is expected output_size equals to 1, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 3,
      "It is expected input_size equals to 3, but got size ",
      input_size.size());

  int output_width = output_size[0];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_width = input_size[2];

  upsample_1d_shape_check(
      Tensor(), grad_output_, nbatch, channels, input_width, output_width);

  Tensor grad_output = grad_output_.contiguous();

  grad_input.resize_({nbatch, channels, input_width});
  grad_input.zero_();

  const int num_kernels = output_width;
  const int num_threads =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "upsample_linear1d_out_frame_backward", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.packed_accessor64<scalar_t, 3>();
        auto odata = grad_output.packed_accessor64<scalar_t, 3>();

        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales);

        upsample_linear1d_out_frame_backward<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(num_kernels, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

Tensor& upsample_linear1d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales) {
  upsample_linear1d_out_cuda_template(
      output, input, output_size, align_corners, scales);
  return output;
}

Tensor upsample_linear1d_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_linear1d_out_cuda_template(
      output, input, output_size, align_corners, scales);
  return output;
}

Tensor& upsample_linear1d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_linear1d_backward_out_cuda");
  upsample_linear1d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales);
  return grad_input;
}

Tensor upsample_linear1d_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_linear1d_backward_cuda");
  Tensor grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_linear1d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales);
  return grad_input;
}

using at::native::upsample::compute_output_size;
using at::native::upsample_cuda::get_scale_value;

Tensor upsample_linear1d_cuda(
    const Tensor& input,
    c10::optional<IntArrayRef> output_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto osize = compute_output_size(input.sizes(), output_size, scale_factors);
  auto scale_w = get_scale_value(scale_factors, 0);
  upsample_linear1d_out_cuda_template(output, input, osize, align_corners, scale_w);
  return output;
}

Tensor upsample_linear1d_backward_cuda(
    const Tensor& grad_output,
    c10::optional<IntArrayRef> output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_linear1d_backward_cuda");
  auto osize = compute_output_size(input_size, output_size, scale_factors);
  auto scale_w = get_scale_value(scale_factors, 0);
  auto grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_linear1d_backward_out_cuda_template(
      grad_input, grad_output, osize, input_size, align_corners, scale_w);
  return grad_input;
}

} // namespace native
} // namespace at
