#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Exception.h>
#include <c10/macros/Macros.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/unique.h>

#include <ATen/native/cuda/EmbeddingBackwardKernel.cuh>


namespace at { namespace native {

namespace {

#ifdef __HIP_PLATFORM_HCC__
static const int BLOCKDIMY = 16;
#else
static const int BLOCKDIMY = 32;
#endif

template
  <typename scalar_t,
   typename accscalar_t,
   typename index_t>
__global__ void embedding_backward_feature_kernel
  (index_t* indices,
   const scalar_t* __restrict__ grad,
   scalar_t* __restrict__ grad_weight,
   int n, // OK to pass as int, we don't expect 2 billion+ samples in one shot
   int64_t stride,
   int padding_idx)
{
  extern __shared__ char buf[];
  accscalar_t* smem = (accscalar_t*)buf;
  accscalar_t* my_s = smem + C10_WARP_SIZE*threadIdx.y;
  int* indices_batch = (int*)(buf + sizeof(accscalar_t)*C10_WARP_SIZE*blockDim.y);

  const int s = (int)stride; // OK to make int, we don't expect 2 billion+ embedding row size

  const int f = threadIdx.x + blockIdx.x*blockDim.x; // feature_dim

  for(int batch_start = 0; batch_start < n; batch_start += blockDim.x*blockDim.y)
  {
    // Entire block cooperates to load a batch of 1024 indices to process
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    if(batch_start + tid < n)
      indices_batch[tid] = (int)indices[batch_start + tid];

    int batch_end = batch_start + blockDim.x*blockDim.y < n ?
                    batch_start + blockDim.x*blockDim.y : n;

    // Loop over the batch of <= 1024 loaded indices in chunks of blockDim.y = 32
    for(int chunk_start = batch_start; chunk_start < batch_end; chunk_start += blockDim.y)
    {
      // This does double duty:  it makes sure indices_batch is ready, and it makes sure match-group
      // leaders are done with their accumulates before other warps start loading again.
      __syncthreads();

      int n_this_chunk = (batch_end - chunk_start) < blockDim.y ?
                         (batch_end - chunk_start) : blockDim.y;

      int src_row = chunk_start + threadIdx.y;
      int dst_row = indices_batch[src_row - batch_start]; // This warp's target row in grad_weight

      // All warps load their smem segments with incoming grad data
      if(src_row < n && f < s && dst_row != padding_idx)
        my_s[threadIdx.x] = static_cast<accscalar_t>(grad[src_row*stride + f]);

      __syncthreads();

      // To ensure determinism, we can't just have each warp add its grad data to its dst_row.
      // We need to check if any other warps pulled grad data targeting dst_row.
      // If so, we elect the first warp in each matching group as the leader.
      // Each leader warp serializes the accumulates targeting dst_row in shared memory,
      // then finishes by adding the accumulated buffer to dst_row in grad_weight.
      if(dst_row != padding_idx && src_row < n) // Per-warp exit condition, safe with ballot_sync
      {
        int match_found_this_thread =
          (dst_row == indices_batch[chunk_start - batch_start + threadIdx.x]);
        if(threadIdx.x >= n_this_chunk)
          match_found_this_thread = 0;
#ifdef __HIP_PLATFORM_HCC__
        unsigned long long int matchmask = WARP_BALLOT(match_found_this_thread);
        int first_remaining_peer = __ffsll(matchmask) - 1;
#else
        unsigned int matchmask = WARP_BALLOT(match_found_this_thread);
        int first_remaining_peer = __ffs(matchmask) - 1;
#endif

        if(threadIdx.y == first_remaining_peer) // Nominate lowest-indexed warp as the leader
        {
          matchmask ^= (1 << first_remaining_peer);
          while(matchmask)
          {
#ifdef __HIP_PLATFORM_HCC__
            first_remaining_peer = __ffsll(matchmask) - 1;
#else
            first_remaining_peer = __ffs(matchmask) - 1;
#endif
            my_s[threadIdx.x] += smem[threadIdx.x + C10_WARP_SIZE*first_remaining_peer];
            matchmask ^= (1 << first_remaining_peer);
          }
          if(f < s)
            grad_weight[dst_row*stride + f] += static_cast<scalar_t>(my_s[threadIdx.x]);
        }
      }
    }
  }
}


template <typename scalar_t, typename index_t>
__global__ void embedding_backward_kernel(
  index_t* input, index_t* indices, scalar_t* grad_output, scalar_t* grad_weight,
  index_t* count, int64_t numel, int64_t stride, int padding_idx) {

  using accscalar_t = acc_type<scalar_t, true>;
  int idx = blockIdx.x * 4 + threadIdx.y;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values proceessed by each thread (grain size)
  const int SZ = 4;

  if (idx < numel
      && (idx == 0 || input[idx] != input[idx - 1])
      && input[idx] != padding_idx) {
    do {
      const int start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
      const int weight_row = ((int) input[idx]) * stride;
      const int grad_row = ((int) indices[idx]) * stride;
      const accscalar_t scale = count ? (accscalar_t)1.0 / count[idx] : 1.0;

      accscalar_t gradient[SZ];
      accscalar_t weight[SZ];

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int feature_dim = start_feature + ii * C10_WARP_SIZE;
        if (feature_dim < stride) {
          gradient[ii] = static_cast<accscalar_t>(grad_output[grad_row + feature_dim]);
          weight[ii] = static_cast<accscalar_t>(grad_weight[weight_row + feature_dim]);
        }
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        weight[ii] += gradient[ii] * scale;
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int feature_dim = start_feature + ii * C10_WARP_SIZE;
        if (feature_dim < stride) {
            grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(weight[ii]);
        }
      }

      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}

/* Calculate norms of the rows of weight_ptr given by idx_ptr and capture them in norms */
template <typename scalar_t, typename accscalar_t, typename index_t>
__global__ void renorm_kernel(
    scalar_t* weights, index_t* indices, accscalar_t max_norm,
    accscalar_t norm_type, int64_t dim,
    int64_t weights_stride0, int64_t weights_stride1) {

  // Some casting hacks since dynamic shared memory and templates don't work together:
  extern __shared__ unsigned char smem[];
  auto sdata = reinterpret_cast<accscalar_t*>(smem);

  int tid = threadIdx.x;
  int base_index = indices[blockIdx.x] * weights_stride0;

  accscalar_t v = 0;
  for (int i = tid; i < dim; i += blockDim.x) {
    auto x = static_cast<accscalar_t>(weights[base_index + i * weights_stride1]);
    if (norm_type == 1) {
      v += std::abs(x);
    } else if (norm_type == 2) {
      v += x * x;
    } else {
      v += std::pow(x, norm_type);
    }
  }

  using Op = ReduceAdd<accscalar_t>;
  v = reduceBlock<accscalar_t>(sdata, blockDim.x, v, Op(), 0);

  if (tid == 0) {
    sdata[0] = std::pow(v, static_cast<accscalar_t>(1.0 / norm_type));
  }
  __syncthreads();

  // now we renormalize the blocks that need it
  if (sdata[0] > max_norm) {
    auto factor = static_cast<scalar_t>(max_norm / (sdata[0] + 1e-7));
    for (int i = tid; i < dim; i += blockDim.x) {
      weights[base_index + i * weights_stride1] *= factor;
    }
  }
}

} // anonymous namespace

Tensor embedding_dense_backward_cuda(const Tensor & grad_, const Tensor & indices,
                               int64_t num_weights, int64_t padding_idx,
                               bool scale_grad_by_freq) {
  auto grad_arg = TensorArg(grad_, "grad", 1);
  auto indices_arg = TensorArg(indices, "indices", 1);
  checkScalarTypes("embedding_backward", indices_arg, {kLong, kInt});
  checkSameGPU("embedding_backward", grad_arg, indices_arg);

  auto num_indices = indices.numel();
  auto grad = grad_.contiguous().view({num_indices, grad_.size(-1)});
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (num_indices <= 768 && !scale_grad_by_freq) {
    auto indices_contig = indices.contiguous();
    auto grad_weight = at::zeros({num_weights, grad_.size(-1)}, grad_.options());
    int64_t stride = grad_weight.stride(0);
    dim3 grid(THCCeilDiv(stride, (int64_t)C10_WARP_SIZE));
    dim3 block(C10_WARP_SIZE, BLOCKDIMY);

    AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16,
      grad.scalar_type(),
       "embedding_backward",
       [&]
       {
         AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "embedding_backward", [&] {
           using accscalar_t = acc_type<scalar_t, true>;
           AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_dense_backward_cuda", [&] () {
            embedding_backward_feature_kernel<scalar_t, accscalar_t, index_t>
              <<<grid,
                  block,
                  sizeof(accscalar_t)*C10_WARP_SIZE*BLOCKDIMY + sizeof(int)*C10_WARP_SIZE*BLOCKDIMY,
                  stream>>>
              (indices_contig.data_ptr<index_t>(),
                grad.data_ptr<scalar_t>(),
                grad_weight.data_ptr<scalar_t>(),
                static_cast<int>(num_indices),
                static_cast<int64_t>(stride),
                static_cast<int>(padding_idx));
            C10_CUDA_KERNEL_LAUNCH_CHECK();
           });
         });
       });
    return grad_weight;
  }

  auto sorted_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto orig_indices = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor count;
  AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_dense_backward_cuda", [&] () {
    using device_ptr = thrust::device_ptr<index_t>;

    // Sort the inputs into sorted with the corresponding indices; we
    // don't need a stable or multidimensional sort, so just use Thrust
    // directly
    {
        sorted_indices.copy_(indices);

        auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
        auto policy = thrust::cuda::par(allocator).on(stream);

        // Fill sortedOrigIndices with sequential indices
        auto count_iter = thrust::counting_iterator<index_t>(0);
        auto orig_data = device_ptr(orig_indices.data_ptr<index_t>());
        thrust::copy(policy, count_iter, count_iter + num_indices, orig_data);

        // Sort; a stable sort is not required
        auto sorted_data = device_ptr(sorted_indices.data_ptr<index_t>());
        thrust::sort_by_key(policy, sorted_data, sorted_data + num_indices, orig_data,
                            ThrustLTOp<index_t>());
    }

    if (scale_grad_by_freq) {
      count = at::empty_like(indices, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

      auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
      auto policy = thrust::cuda::par(allocator).on(stream);

      // Compute an increasing sequence per unique item in sortedIndices:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 1 2 3 1 2 1 1 2
      auto sorted_data = device_ptr(sorted_indices.data_ptr<index_t>());
      auto count_data = device_ptr(count.data_ptr<index_t>());
      thrust::inclusive_scan_by_key(
        policy,
        sorted_data,
        sorted_data + num_indices,
        thrust::make_constant_iterator(1),
        count_data
      );

      // Take the maximum of each count per unique key in reverse:
      // sorted: 2 5 5 5 7 7 8 9 9
      //  count: 1 3 3 3 2 2 1 2 2
      thrust::inclusive_scan_by_key(
        policy,
        thrust::make_reverse_iterator(sorted_data + num_indices),
        thrust::make_reverse_iterator(sorted_data),
        thrust::make_reverse_iterator(count_data + num_indices),
        thrust::make_reverse_iterator(count_data + num_indices),
        thrust::equal_to<index_t>(),
        thrust::maximum<index_t>()
      );
    }
  });

  return embedding_backward_cuda_kernel(grad, orig_indices,
      sorted_indices, count, num_weights, padding_idx);
}

Tensor & embedding_renorm_cuda_(Tensor & self, const Tensor & indices,
                                double max_norm, double norm_type) {
  auto self_arg = TensorArg(self, "self", 1);
  auto indices_arg = TensorArg(indices, "indices", 1);
  checkDim("embedding_renorm_", self_arg, 2);
  checkSameGPU("embedding_renorm", self_arg, indices_arg);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  AT_DISPATCH_INDEX_TYPES(indices.scalar_type(), "embedding_renorm_cuda_", [&] () {
    using device_ptr = thrust::device_ptr<index_t>;

    auto num_indices = indices.numel();
    auto indices_contig = std::get<0>(indices.sort()).contiguous();
    auto indices_data = device_ptr(indices_contig.data_ptr<index_t>());

    auto unique_indices = at::empty(indices.numel(), indices.options());
    auto unique_data = device_ptr(unique_indices.data_ptr<index_t>());
    auto end = thrust::unique_copy(policy, indices_data, indices_data + num_indices, unique_data);
    auto num_unique_indices = static_cast<int>(end - unique_data);

    dim3 grid(num_unique_indices);
    dim3 block(128);
    int dim = self.stride(0);

    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "embedding_backward", [&] {
      AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "embedding_backward", [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        renorm_kernel<<<grid, block, 128 * sizeof(accscalar_t), stream>>>(
          self.data_ptr<scalar_t>(),
          unique_indices.data_ptr<index_t>(),
          static_cast<accscalar_t>(max_norm),
          static_cast<accscalar_t>(norm_type),
          dim, self.stride(0), self.stride(1));
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
    });
  });
  return self;
}


}}  // namespace at::native
