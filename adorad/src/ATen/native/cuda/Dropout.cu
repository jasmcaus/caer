#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <c10/macros/Macros.h>
#include <hiprand/hiprand_kernel.h>

#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>

#include <THC/THCGeneral.h>

namespace at{
namespace native{

namespace {

// philox generates 128 bits of randomness at a time. Kernel uses this explicitly by putting suitably transformed result into float4
// for all members of float4 to be consumed UNROLL has to be 4. Don't change!
// Note: VEC <= 4 (and in most real-world cases will be 4), so same logic applies.
const int UNROLL = 4;

template <
          typename scalar_t,
          typename accscalar_t,
          typename IndexType,
          int ADims,
          int VEC>
#if __CUDA_ARCH__ >= 350
C10_LAUNCH_BOUNDS_2(256, 4)
#elif defined (__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void
fused_dropout_kernel_vec(at::cuda::detail::TensorInfo<scalar_t, IndexType> a,
                         at::cuda::detail::TensorInfo<scalar_t, IndexType> b,
                         at::cuda::detail::TensorInfo<uint8_t, IndexType> c,
                         IndexType totalElements, accscalar_t p,
                         PhiloxCudaState philox_args) {
  // make sure we don't break assumption that we can't have > 4 elements / thread
  static_assert(VEC <= 4, "Value of VEC must be in [2, 4]");

  using LoadT = memory::aligned_vector<scalar_t, VEC>;
  using MaskLoadT = memory::aligned_vector<uint8_t, VEC>;

  auto seeds = at::cuda::philox::unpack(philox_args);
  IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  accscalar_t pinv = accscalar_t(1)/p;

  // Note: Vectorized loads means we'll stride each thread by an additional VEC factor, as we'll load VEC elements at a time
  for (IndexType linearIndex = idx * VEC;
      linearIndex < totalElements;
      linearIndex += gridDim.x * blockDim.x * VEC) {
    // local storage
    scalar_t src[VEC];
    // We'll use this to actually cause vectorized loads later
    LoadT *value = reinterpret_cast<LoadT*>(&src);

    //hiprand_uniform_double was pure evil anyway, not doing what it promises, and there's nothing for halfs, so generate float for everything
    // Note: need a new set of random values per 4 elements -- we'll handle VEC elements in this thread, so need ceil(VEC / 4)
    // sets of rand.
    float4 rand = hiprand_uniform4(&state);

    rand.x = rand.x < p;
    rand.y = rand.y < p;
    rand.z = rand.z < p;
    rand.w = rand.w < p;

    // Note: We explicitly check for is_contiguous() before launching the vectorized kernel
    // and replace IndexToOffset call with linearIndex to allow vectorization of NHWC (or other)
    // ordering.
    // Single vectorized load
    *value = *reinterpret_cast<LoadT*>(&a.data[linearIndex]);

    scalar_t r[VEC];
    uint8_t mask[VEC];

    // Perform the actual computation
    #pragma unroll
    for (int ii = 0; ii < VEC; ii++) {
      r[ii] = src[ii]*(&rand.x)[ii]*pinv;
      mask[ii] = (uint8_t)(&rand.x)[ii];
    }
    // Vectorized writes for both mask & result
    *(reinterpret_cast<LoadT*>(&b.data[linearIndex])) = *reinterpret_cast<LoadT*>(&r[0]);
    *(reinterpret_cast<MaskLoadT*>(&c.data[linearIndex])) = *reinterpret_cast<MaskLoadT*>(&mask[0]);

    __syncthreads();
  }
}

template <
          typename scalar_t,
          typename accscalar_t,
          typename IndexType,
          int ADims,
          int BDims=ADims>
#if __CUDA_ARCH__ >= 350
C10_LAUNCH_BOUNDS_2(256, 4)
#elif defined (__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_2(256, 4)
#endif
__global__ void
fused_dropout_kernel(cuda::detail::TensorInfo<scalar_t, IndexType> a,
                     cuda::detail::TensorInfo<scalar_t, IndexType> b,
                     cuda::detail::TensorInfo<uint8_t, IndexType> c,
                     IndexType totalElements, accscalar_t p,
                     PhiloxCudaState philox_args) {
  auto seeds = at::cuda::philox::unpack(philox_args);
  IndexType idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(std::get<0>(seeds),
              idx,
              std::get<1>(seeds),
              &state);

  accscalar_t pinv = accscalar_t(1)/p;

  IndexType rounded_size = ((totalElements - 1)/(blockDim.x * gridDim.x * UNROLL)+1) *
        blockDim.x * gridDim.x * UNROLL;
  for (IndexType linearIndex = idx;
       linearIndex < rounded_size;
       linearIndex += gridDim.x * blockDim.x*UNROLL) {
//hiprand_uniform_double was pure evil anyway, not doing what it promises, and there's nothing for halfs, so generate float for everything
       float4 rand = hiprand_uniform4(&state);
       scalar_t src[UNROLL];
       rand.x = rand.x < p;
       rand.y = rand.y < p;
       rand.z = rand.z < p;
       rand.w = rand.w < p;
       for (int ii = 0; ii < UNROLL; ii++) {
           IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < totalElements) {
    // Convert `linearIndex` into an offset of `a`
               const IndexType aOffset =
                   cuda::detail::IndexToOffset<scalar_t, IndexType, ADims>::get(li, a);
               src[ii] = a.data[aOffset];
           }
       }
       for (int ii = 0; ii < UNROLL; ii++) {
           IndexType li = linearIndex + blockDim.x * gridDim.x * ii;
           if (li < totalElements) {
    // Convert `linearIndex` into an offset of `b`
               const IndexType bOffset =
                   cuda::detail::IndexToOffset<scalar_t, IndexType, BDims>::get(li, b);
               b.data[bOffset] = src[ii]*(&rand.x)[ii]*pinv;
               c.data[bOffset] = (uint8_t)(&rand.x)[ii];
           }
       }
       __syncthreads();
  }
}

template<typename scalar_t, typename accscalar_t>
void masked_scale_kernel(at::Tensor& ret, const at::Tensor src, const at::Tensor mask, accscalar_t scale){
   auto iter = at::TensorIteratorConfig()
     .check_all_same_dtype(false)
     .add_output(ret)
     .add_input(src)
     .add_input(mask)
     .build();

   at::native::gpu_kernel(
       iter,
       [=]GPU_LAMBDA(const scalar_t src_val, const uint8_t mask_val) -> scalar_t {
          return (float)mask_val * src_val * scale;
       });
}

template <typename scalar_t>
int get_vector_size(at::Tensor self, at::Tensor ret, at::Tensor mask) {
  int vec_size = 4;
  // get the vector size
  if (!self.is_non_overlapping_and_dense() || !ret.is_non_overlapping_and_dense() || !mask.is_non_overlapping_and_dense()) {
    vec_size = 1;
  } else {
    vec_size = memory::can_vectorize_up_to<scalar_t>((char*)self.data_ptr());
  }

  // check that we'd have no remainders - prefer a smaller vector size with no remainders over a larger vector and remainder.
  bool can_vectorize = true;
  do {
    can_vectorize = self.numel() % vec_size == 0 && ret.numel() % vec_size == 0 && mask.numel() % vec_size == 0;
    if (!can_vectorize) vec_size /= 2;
  } while (vec_size > 1 && !can_vectorize);
  return can_vectorize ? vec_size : 1;
}

template <typename index_type>
inline void launcher(
    const Tensor& self,
    Tensor& ret,
    Tensor& mask,
    double p,
    const int64_t nelem,
    const PhiloxCudaState rng_engine_inputs,
    dim3 grid,
    dim3 dim_block) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      self.scalar_type(),
      "fused_dropout",
      [&] {
        using accscalar_t = acc_type<scalar_t, true>;
        accscalar_t pa = (accscalar_t)(p);
        auto self_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(self);
        auto ret_info =
            cuda::detail::getTensorInfo<scalar_t, index_type>(ret);
        auto mask_info =
            cuda::detail::getTensorInfo<uint8_t, index_type>(mask);
        self_info.collapseDims();
        ret_info.collapseDims();
        mask_info.collapseDims(); // ret and mask are collapsed to 1d
                                  // contiguous tensor

        int vec_size = get_vector_size<scalar_t>(self, ret, mask);

        if (vec_size > 1) {
          switch (vec_size) {
            case 4:
              fused_dropout_kernel_vec<
                  scalar_t,
                  accscalar_t,
                  index_type,
                  1,
                  4>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            case 2:
              fused_dropout_kernel_vec<
                  scalar_t,
                  accscalar_t,
                  index_type,
                  1,
                  2>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
          }
        } else {
          switch (self_info.dims) {
            case 1:
              fused_dropout_kernel<scalar_t, accscalar_t, index_type, 1>
                  <<<grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
                      self_info,
                      ret_info,
                      mask_info,
                      nelem,
                      pa,
                      rng_engine_inputs);
              C10_CUDA_KERNEL_LAUNCH_CHECK();
              break;
            default:
              if (!self.is_contiguous() && ret.is_contiguous() &&
                  mask.is_contiguous()) {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1, 1>
                    <<<grid,
                        dim_block,
                        0,
                        at::cuda::getCurrentCUDAStream()>>>(
                        self_info,
                        ret_info,
                        mask_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              } else {
                fused_dropout_kernel<scalar_t, accscalar_t, index_type, -1>
                    <<<grid,
                        dim_block,
                        0,
                        at::cuda::getCurrentCUDAStream()>>>(
                        self_info,
                        ret_info,
                        mask_info,
                        nelem,
                        pa,
                        rng_engine_inputs);
                C10_CUDA_KERNEL_LAUNCH_CHECK();
              }
          }
        }
      });
}

} //anonymous namespace

std::tuple<Tensor,Tensor>
fused_dropout_cuda(const Tensor& self, double p, c10::optional<Generator> gen_){
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  Tensor ret = at::empty_like(self);
  Tensor mask = at::empty_like(self, self.options().dtype(kByte));
  const int64_t nelem = self.numel();
//empty tensors should not get here, but just in case, avoid FPE
  if (nelem==0) return std::tuple<Tensor,Tensor>(self, mask);
  const int64_t block_size = 256;
  unsigned int blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor/block_size;
  dim3 dim_block(block_size);
  dim3 grid((nelem + block_size -1)/block_size);
  grid.x = std::min((unsigned int)at::cuda::getCurrentDeviceProperties()->multiProcessorCount * blocks_per_sm, grid.x);
//number of times random will be generated per thread, to offset philox counter in thc random state
  int64_t counter_offset = ((nelem - 1)/(block_size*grid.x*UNROLL)+1)*UNROLL;
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(counter_offset);
  }
  if (cuda::detail::canUse32BitIndexMath(self)){
    launcher<unsigned int>(
        self, ret, mask, p, nelem, rng_engine_inputs, grid, dim_block);
  } else {
    launcher<uint64_t>(
        self, ret, mask, p, nelem, rng_engine_inputs, grid, dim_block);
  }
  return std::tuple<Tensor,Tensor>(ret, mask);
}

Tensor masked_scale_cuda(const Tensor& self, const Tensor& mask, double scale){
   Tensor ret = at::empty_like(self, self.suggest_memory_format());
   TORCH_CHECK(mask.scalar_type() == at::ScalarType::Byte, "mask should be torch.uint8 dtype");
   AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "masked_scale", [&] {
      using accscalar_t = acc_type<scalar_t, true>;
      accscalar_t pa = (accscalar_t)(scale);
      masked_scale_kernel<scalar_t>(ret, self, mask, pa);
  });
  return ret;
}

}
}
