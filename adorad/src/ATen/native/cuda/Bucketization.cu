#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/BucketizationUtils.h>
#include <THC/THC.h>

namespace at {
namespace native {

// Implement a TF like searchsorted and a bucketize function running on cuda
// See details in ATen/nativate/Bucketization.cpp

namespace {

template<typename input_t>
__device__ int64_t lower_bound(const input_t *data_ss, int64_t start, int64_t end, input_t val) {
  while (start < end) {
    int64_t mid = start + ((end - start) >> 1);
    if (!(data_ss[mid] >= val)) {
      start = mid + 1;
    }
    else {
      end = mid;
    }
  }
  return start;
}

template<typename input_t>
__device__ int64_t upper_bound(const input_t *data_ss, int64_t start, int64_t end, input_t val) {
  while (start < end) {
    int64_t mid = start + ((end - start) >> 1);
    if (!(data_ss[mid] > val)) {
      start = mid + 1;
    }
    else {
      end = mid;
    }
  }
  return start;
}

template<typename input_t, typename output_t>
__global__ void searchsorted_cuda_kernel(
  output_t *data_out,
  const input_t *data_in,
  const input_t *data_bd,
  int64_t idim_in,
  int64_t idim_bd,
  int64_t numel_in,
  bool right,
  bool is_1d_boundaries) {

  for (int64_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < numel_in; tid += blockDim.x * gridDim.x) {
    // If boundaries tensor is 1d, we always search the entire boundary tensor
    int64_t start_bd = is_1d_boundaries ? 0 : tid / idim_in * idim_bd;
    int64_t end_bd = start_bd + idim_bd;

    int64_t pos = !right ?
      lower_bound<input_t>(data_bd, start_bd, end_bd, data_in[tid]) - start_bd :
      upper_bound<input_t>(data_bd, start_bd, end_bd, data_in[tid]) - start_bd;

    // type conversion might happen here
    data_out[tid] = pos;
  }
}

template<typename input_t, typename output_t>
void searchsorted_cuda_contiguous(Tensor& result, const Tensor& input, const Tensor& boundaries, const bool& right) {
  int64_t numel_in = input.numel();
  bool is_scalar_input = input.dim() == 0 && numel_in == 1;
  // inner most dim size of input and boundaries
  int64_t idim_in = is_scalar_input ? 1 : input.sizes().back();
  int64_t idim_bd = boundaries.sizes().back();

  const input_t *data_in = input.data_ptr<input_t>();
  const input_t *data_bd = boundaries.data_ptr<input_t>();
  output_t *data_out = result.data_ptr<output_t>();

  int64_t maxThread = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t maxGrid = 1024;
  dim3 block = dim3(std::min(maxThread, numel_in));
  dim3 grid  = dim3(std::min(maxGrid, cuda::ATenCeilDiv<int64_t>(numel_in, block.x)));
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

  searchsorted_cuda_kernel<<<grid, block, 0, stream>>>(
    data_out, data_in, data_bd, idim_in, idim_bd, numel_in, right, boundaries.dim() == 1);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void dispatch(Tensor& result, const Tensor& input, const Tensor& boundaries, bool out_int32, bool right) {
  if (!out_int32) {
    AT_DISPATCH_ALL_TYPES(input.scalar_type(), "searchsorted_out_cuda", [&] {
      searchsorted_cuda_contiguous<scalar_t, int64_t>(result, input, boundaries, right);
    });
  }
  else {
    AT_DISPATCH_ALL_TYPES(input.scalar_type(), "searchsorted_out_cuda", [&] {
      searchsorted_cuda_contiguous<scalar_t, int>(result, input, boundaries, right);
    });
  }
}

}

Tensor& searchsorted_out_cuda(Tensor& result, const Tensor& sorted_sequence, const Tensor& self, bool out_int32, bool right) {
  searchsorted_pre_check(sorted_sequence, self, result, out_int32);
  if (result.numel() == 0) {
    result.resize_(self.sizes());
  }
  if (self.numel() == 0) {
    return result;
  }
  if (sorted_sequence.is_contiguous() && self.is_contiguous() && sorted_sequence.dtype() == self.dtype()) {
    dispatch(result, self, sorted_sequence, out_int32, right);
    return result;
  }

  Tensor trimmed_input;
  Tensor trimmed_boundaries;
  searchsorted_maybe_trim_input_tensors(trimmed_input, trimmed_boundaries, self, sorted_sequence);
  const Tensor& final_input = trimmed_input.defined() ? trimmed_input : self;
  const Tensor& final_boundaries = trimmed_boundaries.defined() ? trimmed_boundaries : sorted_sequence;
  dispatch(result, final_input, final_boundaries, out_int32, right);
  return result;
}

Tensor searchsorted_cuda(const Tensor& sorted_sequence, const Tensor& self, bool out_int32, bool right) {
  ScalarType scalar_type = out_int32 ? ScalarType::Int : ScalarType::Long;
  c10::TensorOptions options = TensorOptions().device(self.options().device()).dtype(scalar_type);
  Tensor result = at::empty({0}, options, MemoryFormat::Contiguous);
  searchsorted_out_cuda(result, sorted_sequence, self, out_int32, right);
  return result;
}

Tensor searchsorted_cuda(const Tensor& sorted_sequence, Scalar self, bool out_int32, bool right) {
  return searchsorted_cuda(sorted_sequence, searchsorted_scalar_tensor(self, sorted_sequence.device()), out_int32, right);
}

Tensor& bucketize_out_cuda(Tensor& result, const Tensor& self, const Tensor& boundaries, bool out_int32, bool right) {
  TORCH_CHECK(boundaries.dim() == 1, "boundaries tensor must be 1 dimension, but got dim(", boundaries.dim(), ")");
  searchsorted_out_cuda(result, boundaries, self, out_int32, right);
  return result;
}

Tensor bucketize_cuda(const Tensor& self, const Tensor& boundaries, bool out_int32, bool right) {
  ScalarType scalar_type = out_int32 ? ScalarType::Int : ScalarType::Long;
  c10::TensorOptions options = TensorOptions().device(self.options().device()).dtype(scalar_type);
  Tensor result = at::empty({0}, options, MemoryFormat::Contiguous);
  bucketize_out_cuda(result, self, boundaries, out_int32, right);
  return result;
}

Tensor bucketize_cuda(Scalar self, const Tensor& boundaries, bool out_int32, bool right) {
  return bucketize_cuda(searchsorted_scalar_tensor(self, boundaries.device()), boundaries, out_int32, right);
}

}} // namespace at::native
