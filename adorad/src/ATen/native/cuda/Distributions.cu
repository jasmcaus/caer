#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

/**
 * Note [Register spilling in hiprand call for CUDA < 10]
 * ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 * For CUDA < 10, hiprandStatePhilox4_32_10_t engine achieves poor performance (60% SOL bandwidth)
 * when called to generate one random number at a time. This is because the line
 *            unsigned ret = (&state->output.x)[state->STATE++];
 * in
 *            QUALIFIERS unsigned int hiprand(hiprandStatePhilox4_32_10_t *state)
 * in hiprand/hiprand_kernel.h dynamically indexes into state.output, preventing the compiler from ever
 * storing state.output in registers.
 *
 * CUDA 10 fixed this problem. However, for backwards compatibility, in the following kernels
 * we are using hiprand distributions that utilize curand4 call. curand4 call doesn't have the
 * register spilling problem.
 */

namespace {

template <typename scalar_t>
void poisson_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& lambda,
    at::PhiloxCudaState philox_args) {
  auto functor = [philox_args] __device__(
          scalar_t & ret_val, const scalar_t& lambda) {
        auto seeds = at::cuda::philox::unpack(philox_args);
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(std::get<0>(seeds),
                    blockIdx.x * blockDim.x + threadIdx.x,
                    std::get<1>(seeds),
                    &state);
        ret_val = static_cast<scalar_t>(hiprand_poisson(&state, lambda));
      };
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t, decltype(functor),
                               /*max_threads_per_block=*/512,
                               /*min_blocks_per_sm==*/2>(ret, lambda, functor);
}

struct curand_uniform_wrapper {
  hiprandStatePhilox4_32_10_t &state;
  __device__ curand_uniform_wrapper(hiprandStatePhilox4_32_10_t &state): state(state) {}
  __device__ float operator()() {

  uint32_t val = hiprand(&state); //need just bits
  constexpr auto MASK = static_cast<uint32_t>((static_cast<uint64_t>(1) << std::numeric_limits<float>::digits) - 1);
  constexpr auto DIVISOR = static_cast<float>(1) / (static_cast<uint32_t>(1) << std::numeric_limits<float>::digits);
    return (val & MASK) * DIVISOR;
  }
};

template <typename scalar_t>
void binomial_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& count,
    const at::Tensor& prob,
    at::PhiloxCudaState philox_args) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(count)
      .add_input(prob)
      .build();

  at::native::distribution_binary_kernel(iter, philox_args,
      [philox_args] GPU_LAMBDA (hiprandStatePhilox4_32_10_t& state, scalar_t count, scalar_t prob) {
        #if defined(__CUDA_ARCH__) || defined(__HIP_PLATFORM_HCC__)
        auto uniform_lambda = curand_uniform_wrapper(state);
        BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);
        auto sample = sample_binomial<scalar_t, accscalar_t, decltype(uniform_lambda)>(count, prob, standard_uniform);
        return static_cast<scalar_t>(sample);
        #else
        return count; // useless.
        #endif
      }
  );
}

template <typename scalar_t>
void gamma_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& alpha,
    at::PhiloxCudaState philox_args) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  auto functor = [philox_args] __device__(
          scalar_t & ret_val, const scalar_t& alpha) {
        auto seeds = at::cuda::philox::unpack(philox_args);
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(std::get<0>(seeds),
                    blockIdx.x * blockDim.x + threadIdx.x,
                    std::get<1>(seeds),
                    &state);

        auto uniform_lambda = [&state] __device__ () {
          return hiprand_uniform(&state);
        };
        BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);

        auto normal_lambda = [&state] __device__ () {
          return hiprand_normal(&state);
        };
        BaseSampler<accscalar_t, decltype(normal_lambda)> standard_normal(normal_lambda);
        auto sample = sample_gamma<scalar_t, accscalar_t, decltype(uniform_lambda), decltype(normal_lambda)>(alpha, standard_uniform, standard_normal);
        auto min_value = std::numeric_limits<scalar_t>::min();
        ret_val = (min_value > sample) ? min_value : sample;
      };
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t, decltype(functor),
                               /*max_threads_per_block=*/512,
                               /*min_blocks_per_sm==*/2>(ret, alpha, functor);
}

template<typename scalar_t>
void dirichlet_scalar_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& gamma) {
  auto gamma_sum = gamma.sum(-1, true);
  at::TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(gamma)
      .add_input(gamma_sum)
      .build();
  at::native::gpu_kernel(iter,
    [] GPU_LAMBDA (scalar_t gamma, scalar_t gamma_sum) {
      auto ret_val = gamma / gamma_sum;
      auto min_value = std::numeric_limits<scalar_t>::min();
      auto max_value = 1 - std::numeric_limits<scalar_t>::epsilon();
      ret_val = (min_value > ret_val) ? min_value : ret_val;
      ret_val = (max_value < ret_val) ? max_value : ret_val;
      return ret_val;
    });
}

} // namespace

namespace at { namespace native {

Tensor _s_poisson_cuda(const Tensor& lambda, c10::optional<Generator> gen_) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(20);
  }
  Tensor ret = at::empty(lambda.sizes(), lambda.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "poisson_cuda", [&] {
    poisson_cuda_kernel<scalar_t>(ret, lambda, rng_engine_inputs);
  });
  return ret;
}

Tensor _s_binomial_cuda(const Tensor& count, const Tensor& prob, c10::optional<Generator> gen_) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(42);
  }
  Tensor ret = at::empty(count.sizes(), count.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "binomial_cuda", [&] {
    binomial_cuda_kernel<scalar_t>(ret, count, prob, rng_engine_inputs);
  });
  return ret;
}

Tensor _s_gamma_cuda(const Tensor& alpha, c10::optional<Generator> gen_) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "gamma_cuda", [&] {
     gamma_cuda_kernel<scalar_t>(ret, alpha, rng_engine_inputs);
   });
  return ret;
}

Tensor _s_dirichlet_cuda(const Tensor& alpha, c10::optional<Generator> gen_) {
  auto gen = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  PhiloxCudaState rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_cuda_state(10);
  }
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, ret.scalar_type(), "dirichlet", [&] {
    Tensor gamma = at::empty(alpha.sizes(), alpha.options());
    gamma_cuda_kernel<scalar_t>(gamma, alpha, rng_engine_inputs);
    dirichlet_scalar_cuda_kernel<scalar_t>(ret, gamma);
  });
  return ret;
}

Tensor _standard_gamma_grad_cuda(const Tensor& self, const Tensor& output) {
  Tensor ret = at::empty(self.sizes(), self.options());
  TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(self)
      .add_input(output)
      .build();
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "_standard_gamma_grad_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    gpu_kernel(iter,
      [] GPU_LAMBDA (scalar_t self_val, scalar_t output_val) {
        return standard_gamma_grad_one<scalar_t, accscalar_t>(self_val, output_val);
      });
  });
  return ret;
}

Tensor _dirichlet_grad_cuda(const Tensor& x, const Tensor& alpha, const Tensor& total) {
  Tensor ret = at::empty(x.sizes(), x.options());
  TensorIterator iter = at::TensorIteratorConfig()
      .add_output(ret)
      .add_input(x)
      .add_input(alpha)
      .add_input(total)
      .build();
  AT_DISPATCH_FLOATING_TYPES(x.scalar_type(), "_dirichlet_grad_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    gpu_kernel(iter,
      [] GPU_LAMBDA (scalar_t x_val, scalar_t alpha_val, scalar_t total_val) -> scalar_t {
        return dirichlet_grad_one<scalar_t, accscalar_t>(x_val, alpha_val, total_val);
      });
  });
  return ret;
}

}} // namespace at::native
