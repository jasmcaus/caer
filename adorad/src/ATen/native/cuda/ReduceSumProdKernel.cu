#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/Dispatch.h>
#include <ATen/native/ReduceOps.h>

namespace at { namespace native {

template <typename scalar_t, typename acc_t = scalar_t, typename out_t = scalar_t>
struct sum_functor {
  void operator()(TensorIterator& iter) {
    gpu_reduce_kernel<scalar_t, out_t>(
        iter, func_wrapper<out_t>([] GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
          return a + b;
        }));
  }
};

template <typename scalar_t, typename acc_t = scalar_t, typename out_t = scalar_t>
struct nansum_functor {
  void operator()(TensorIterator& iter) {
    gpu_reduce_kernel<scalar_t, out_t>(
        iter, NanSumOps<acc_t, out_t>{});
  }
};

template <typename scalar_t, typename acc_t = scalar_t, typename out_t = scalar_t>
struct prod_functor {
  void operator()(TensorIterator& iter) {
    gpu_reduce_kernel<scalar_t, out_t>(
        iter, func_wrapper<out_t>([] GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
          return a * b;
        }), 1);
  }
};

// The function `reduce_dispatch` below dispatches to the kernel based
// on the type of `iter`. It takes care of the common logic
// for handling Half-Precision floating types.
// Otherwise the functor `op` is called to dispatch to the kernel
// of relevant type.
//
// Note: Functor `op` should take care of all the types to be supported
//       except for `at::Half` and `at::BFloat16`.
template <
    template <
        typename scalar_t,
        typename acc_t = scalar_t,
        typename out_t = scalar_t>
    typename OpFunctor,
    typename GeneralDispatcher>
static void reduce_dispatch(TensorIterator& iter, GeneralDispatcher op) {
  if (iter.dtype() == kHalf) {
    return OpFunctor<at::Half, float>{}(iter);
  } else if (iter.dtype(1) == kHalf && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return OpFunctor<at::Half, float, float>{}(iter);
  } else if (iter.dtype() == kBFloat16) {
    return OpFunctor<at::BFloat16, float>{}(iter);
  } else if (iter.dtype(1) == kBFloat16 && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return OpFunctor<at::BFloat16, float, float>{}(iter);
  }
  op(iter);
}

static void sum_kernel_cuda(TensorIterator& iter){
  auto general_dispatcher = [](TensorIterator& iter) {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(
        ScalarType::Bool, iter.dtype(), "sum_cuda", [&]() {
          sum_functor<scalar_t>{}(iter);
        });
  };

  reduce_dispatch<sum_functor>(iter, general_dispatcher);
}

static void nansum_kernel_cuda(TensorIterator& iter) {
  auto general_dispatcher = [](TensorIterator& iter) {
    AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "nansum_cuda", [&]() {
      nansum_functor<scalar_t>{}(iter);
    });
  };

  reduce_dispatch<nansum_functor>(iter, general_dispatcher);
}

static void prod_kernel_cuda(TensorIterator& iter) {
  auto general_dispatcher = [](TensorIterator& iter) {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX(iter.dtype(), "prod_cuda", [&]() {
      prod_functor<scalar_t>{}(iter);
    });
  };

  reduce_dispatch<prod_functor>(iter, general_dispatcher);
}

REGISTER_DISPATCH(sum_stub, &sum_kernel_cuda);
REGISTER_DISPATCH(nansum_stub, &nansum_kernel_cuda);
REGISTER_DISPATCH(prod_stub, &prod_kernel_cuda);

}} // namespace at::native
