#include <ATen/native/BinaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at {
namespace native {

void sigmoid_backward_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "sigmoid_backward_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a * (scalar_t(1.) - b) * b;
    });
  });
}

void logit_backward_kernel_cuda(TensorIterator& iter, Scalar eps_scalar) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "logit_cuda",
      [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        const T_ACC eps = eps_scalar.to<T_ACC>();
        if (eps < T_ACC(0)) {
          gpu_kernel(
              iter, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
                const T_ACC dy_acc = static_cast<T_ACC>(dy);
                const T_ACC x_acc = static_cast<T_ACC>(x);
                return (x_acc < T_ACC(0) || x_acc > T_ACC(1))
                    ? std::numeric_limits<T_ACC>::quiet_NaN()
                    : dy_acc / (x_acc * (T_ACC(1) - x_acc));
              });
        } else {
          const T_ACC lo = eps;
          const T_ACC hi = T_ACC(1) - eps;
          gpu_kernel(
              iter, [lo, hi] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
                const T_ACC dy_acc = static_cast<T_ACC>(dy);
                const T_ACC x_acc = static_cast<T_ACC>(x);
                return (x_acc < lo || x_acc > hi)
                    ? T_ACC(0)
                    : dy_acc / (x_acc * (T_ACC(1) - x_acc));
              });
        }
      });
}

void tanh_backward_kernel_cuda(TensorIterator& iter) {
  if(isComplexType(iter.dtype())) {
    AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "tanh_backward_complex_cuda", [&]() {
      gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a * std::conj(scalar_t{1.} - b * b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "tanh_backward_cuda", [&]() {
      gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return a * (scalar_t{1.} - b * b);
      });
    });
  }
}

REGISTER_DISPATCH(sigmoid_backward_stub, &sigmoid_backward_kernel_cuda);
REGISTER_DISPATCH(logit_backward_stub, &logit_backward_kernel_cuda);
REGISTER_DISPATCH(tanh_backward_stub, &tanh_backward_kernel_cuda);

} // namespace native
} // namespace at
