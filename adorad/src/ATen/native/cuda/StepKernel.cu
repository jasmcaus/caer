#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void nextafter_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "nextafter_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::nextafter(a, b);
    });
  });
}

void heaviside_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBool, kBFloat16, iter.dtype(), "heaviside_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a == 0 ? b : static_cast<scalar_t>(a > 0);
    });
  });
}

REGISTER_DISPATCH(nextafter_stub, &nextafter_kernel_cuda);
REGISTER_DISPATCH(heaviside_stub, &heaviside_kernel_cuda);

}} // namespace at::native
