#include "hip/hip_runtime.h"
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/Dispatch.h>
#include <ATen/native/ReduceOps.h>

namespace at { namespace native {

// This reduction accumulates results as the type `acc_t`. By default, when
// `scalar_t` is complex, `acc_t` is the downgraded real number type.
// Otherwise, `acc_t` and `scalar_t` are the same type.
template <typename scalar_t, typename acc_t=typename scalar_value_type<scalar_t>::type, typename out_t=typename scalar_value_type<scalar_t>::type>
void norm_kernel_cuda_impl(TensorIterator& iter, Scalar val) {
  double p;
  if (val.isIntegral(false)) {
     p = val.to<int64_t>();
  } else if (val.isFloatingPoint()) {
     p = val.to<double>();
  } else {
     AT_ERROR("norm_kernel_cuda_impl expects norm to be integer or float");
  }

  if (p == static_cast<double>(0)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormZeroOps<scalar_t, acc_t>(), 0);
  } else if (p == static_cast<double>(1)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormOneOps<scalar_t, acc_t>(), 0);
  } else if (p == static_cast<double>(2)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormTwoOps<scalar_t, acc_t>(), 0);
  } else if (p == static_cast<double>(INFINITY)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, AbsMaxOps<scalar_t, acc_t>(), std::numeric_limits<acc_t>::min());
  } else if (p == static_cast<double>(-INFINITY)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, AbsMinOps<scalar_t, acc_t>(), std::numeric_limits<acc_t>::max());
  } else {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormOps<scalar_t, acc_t>{ acc_t(p) }, 0);
  }
}

static void norm_kernel_cuda(TensorIterator& iter, Scalar p) {
  if (iter.input_dtype() == kHalf) {
    return norm_kernel_cuda_impl<at::Half, float>(iter, p);
  } else if (iter.dtype(1) == kHalf && iter.input_dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return norm_kernel_cuda_impl<at::Half, float, float>(iter, p);
  }
  else if(iter.input_dtype() == kBFloat16) {
    return norm_kernel_cuda_impl<at::BFloat16, float>(iter, p);
  } else if (iter.dtype(1) == kBFloat16 && iter.input_dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return norm_kernel_cuda_impl<at::BFloat16, float, float>(iter, p);
  }
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES(iter.input_dtype(), "norm_cuda", [&] {
    norm_kernel_cuda_impl<scalar_t>(iter, p);
  });
}

REGISTER_DISPATCH(norm_stub, &norm_kernel_cuda);

}} // namespace at::native
