#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Config.h>
#include <ATen/Dispatch.h>
#include <ATen/Utils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/detail/CUDAHooksInterface.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/SpectralOpsUtils.h>
#include <ATen/native/cuda/CuFFTUtils.h>
#include <ATen/native/cuda/CuFFTPlanCache.h>
#include <THC/THCTensorSort.cuh>
#include <THC/THCThrustAllocator.cuh>

#include <thrust/execution_policy.h>
#include <thrust/unique.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <vector>
#include <cmath>

namespace at { namespace native {

using namespace at::native::detail;

// Offset calculator for indexing in Hermitian mirrored order.
// In mirrored dims, maps linear index i to (n - i) % n
template <typename index_t>
struct HermitianSymmetryOffsetCalculator {
  using offset_type = at::detail::Array<index_t, 1>;
  using dim_type = std::remove_cv_t<decltype(MAX_DIMS)>;
  dim_type dims;
  IntDivider<index_t> sizes_[MAX_DIMS];
  index_t strides_[MAX_DIMS];
  uint32_t mirror_dim_;  // bit mask
  static_assert(MAX_DIMS < 32, "Need a bigger mask type");

  HermitianSymmetryOffsetCalculator(
      IntArrayRef sizes, IntArrayRef strides, IntArrayRef dim,
      const int64_t element_size){
    TORCH_INTERNAL_ASSERT(sizes.size() == strides.size());
    TORCH_INTERNAL_ASSERT(sizes.size() <= MAX_DIMS);
    dims = sizes.size();

    for (dim_type i = 0; i < MAX_DIMS; ++i) {
      if (i < dims) {
        sizes_[i] = IntDivider<index_t>(sizes[i]);
        strides_[i] = strides[i] / element_size;
      } else {
        sizes_[i] = IntDivider<index_t>(1);
        strides_[i] = 0;
      }
    }

    mirror_dim_ = 0;
    for (int64_t i = 0; i < dim.size(); ++i) {
      mirror_dim_ |= (uint32_t{1} << dim[i]);
    }
  }

  C10_HOST_DEVICE offset_type get(index_t linear_idx) const {
    index_t offset = 0;

    for (dim_type dim = 0; dim < dims; ++dim) {
      auto divmod = sizes_[dim].divmod(linear_idx);
      linear_idx = divmod.div;

      if ((mirror_dim_ & (uint32_t{1} << dim)) == 0) {
        offset += divmod.mod * strides_[dim];
      } else if (divmod.mod != 0) {
        offset += (sizes_[dim].divisor - divmod.mod) * strides_[dim];
      }
    }
    offset_type offsets;
    offsets[0] = offset;
    return offsets;
  }
};

// out[:] = conj(in[:]) where in and out ordering is generalized by offset calculators
template <typename scalar_t, typename inp_calc_t, typename out_calc_t>
C10_LAUNCH_BOUNDS_1(cuda::detail::CUDA_NUM_THREADS)
__global__ void _fft_conjugate_copy_kernel(
    int64_t numel, scalar_t * out_data, const scalar_t * in_data,
    inp_calc_t ic, out_calc_t oc) {
  CUDA_KERNEL_LOOP_TYPE(index, numel, int64_t) {
    auto in_offset = ic.get(index)[0];
    auto out_offset = oc.get(index)[0];
    out_data[out_offset] = std::conj(in_data[in_offset]);
  }
}

// In real-to-complex transform, cuFFT only fills half of the values due to
// conjugate symmetry. See native/SpectralUtils.h for more details.
// The following function fills in the other half with symmetry in
// case of real-to-complex transform with onesided=False flag.
// See NOTE [ Fourier Transform Conjugate Symmetry ] in native/SpectralOpsUtils.h.

// input should be a tensor of same size as full (twosided)
// signals, but only contains half (onesided) of the values.
// This function modifies inplace.
void _fft_fill_with_conjugate_symmetry_cuda_(
    ScalarType dtype, IntArrayRef mirror_dims, IntArrayRef signal_half_sizes,
    IntArrayRef in_strides, const void * in_data,
    IntArrayRef out_strides, void * out_data) {
  // Do the actual conjugate mirroring.
  // TODO: consider adding a 32bit indexed kernel for improved performance
  auto* in_strides_ptr = in_strides.data();
  const int ndim = in_strides.size();
  const int64_t element_size = scalarTypeToTypeMeta(dtype).itemsize();
  OffsetCalculator<1, int64_t> input_offset_calculator(
      ndim, signal_half_sizes.data(), &in_strides_ptr, &element_size);
  HermitianSymmetryOffsetCalculator<int64_t> output_offset_calculator(
      signal_half_sizes, out_strides, mirror_dims, element_size);

  const auto numel = at::prod_intlist(signal_half_sizes);
  AT_DISPATCH_COMPLEX_TYPES(dtype, "_fft_fill_with_conjugate_symmetry", [&] {
        using namespace cuda::detail;
        _fft_conjugate_copy_kernel<<<
          GET_BLOCKS(numel), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
              numel,
              static_cast<scalar_t*>(out_data),
              static_cast<const scalar_t*>(in_data),
              input_offset_calculator,
              output_offset_calculator);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

REGISTER_DISPATCH(fft_fill_with_conjugate_symmetry_stub, &_fft_fill_with_conjugate_symmetry_cuda_);

// Execute a pre-planned tranform
static void exec_cufft_plan(
    const CuFFTConfig &config, void* in_data, void* out_data, bool forward) {
  auto& plan = config.plan();
#ifdef __HIP_PLATFORM_HCC__
  auto value_type = config.data_type();
  if (value_type == kFloat) {
    switch (config.transform_type()) {
      case CuFFTTransformType::C2C: {
        CUFFT_CHECK(hipfftExecC2C(plan, static_cast<hipfftComplex*>(in_data),
                                  static_cast<hipfftComplex*>(out_data),
                                  forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case CuFFTTransformType::R2C: {
        CUFFT_CHECK(hipfftExecC2R(plan, static_cast<hipfftComplex*>(in_data),
                                  static_cast<hipfftReal*>(out_data)));
        return;
      }
      case CuFFTTransformType::C2R: {
        CUFFT_CHECK(hipfftExecR2C(plan, static_cast<hipfftReal*>(in_data),
                                  static_cast<hipfftComplex*>(out_data)));
        return;
      }
    }
  } else if (value_type == kDouble) {
    switch (config.transform_type()) {
      case CuFFTTransformType::C2C: {
        CUFFT_CHECK(hipfftExecZ2Z(plan, static_cast<hipfftDoubleComplex*>(in_data),
                                  static_cast<hipfftDoubleComplex*>(out_data),
                                  forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
        return;
      }
      case CuFFTTransformType::R2C: {
        CUFFT_CHECK(hipfftExecD2Z(plan, static_cast<hipfftDoubleReal*>(in_data),
                                  static_cast<hipfftDoubleComplex*>(out_data)));
        return;
      }
      case CuFFTTransformType::C2R: {
        CUFFT_CHECK(hipfftExecZ2D(plan, static_cast<hipfftDoubleComplex*>(in_data),
                                  static_cast<hipfftDoubleReal*>(out_data)));
        return;
      }
    }
  }
  TORCH_CHECK(false, "hipFFT doesn't support transforms on type: ", value_type);
#else
  CUFFT_CHECK(hipfftXtExec(plan, in_data, out_data,
                          forward ? HIPFFT_FORWARD : HIPFFT_BACKWARD));
#endif
}


// NOTE [ cuFFT Embedded Strides ]
//
// cuFFT supports a subset of arbitrary strides via their "advanced data layout"
// option (http://docs.nvidia.com/cuda/cufft/index.html#advanced-data-layout).
// Specifically, these are tensors that can be viewed as subtensors resulted
// from slicing a larger contiguous tensors. For such input tensors, let the
// sizes of the enclosing tensor be `inembed`, and we can have in 3d case:
//
//     input[x, y, z] = input[((x * inembed[1] + y) * inembed[2] + z)]
//
// Above is the simplified formula ignoring the batch dimension. In fact, the
// last dimension of the enclosing tensor doesn't have to be contiguous, i.e.,
// it can be greater than 1. Then one can set the base stride for the enclosing
// tensor with `istride`. Then we have
//
//     input[x, y, z] = input[((x * inembed[1] + y) * inembed[2] + z) * istride]
//
// For example, consider
//
//     enclosing = torch.zeros(6, 8, 10)  # contiguous
//     input = enclosing[:4, 2:6, 6:]
//     input.size()                       # [ 4,  4,  4]
//     input.stride()                     # [80, 10,  1]
//     # inembed = [6, 8, 10]
//     input[2, 1, 3] = input[((2 * 8) + 1) * 10 + 3]   # using above formula
//                    = input[173]
//                    = input[2 * 80 + 1 * 10 + 1 * 3]  # using strides directly
//
// Generally, the embedded strides can be computed as
//
//     embed[i] = stride[i - 1] / stride[i].
//
// Note that the value of embed[0] isn't used to compute indices and doesn't
// matter.
//
// Contrary to advanced data layout, simple layout means that *embeds have
// unit-strides. In particular, unit-stride refers to that the input and output
// tensors being contiguous, and that the strides at the innermost signal
// dimension being unit (1) w.r.t. the corresponding data type.

static inline Tensor _run_cufft(
    const CuFFTConfig &config, Tensor& input, int64_t signal_ndim,
    bool complex_input, bool complex_output, bool inverse,
    IntArrayRef checked_signal_sizes, fft_norm_mode norm, bool onesided,
    IntArrayRef output_sizes, bool input_was_cloned
) {
  if (config.should_clone_input() && !input_was_cloned) {
    input = input.clone(at::MemoryFormat::Contiguous);
  }

  auto& plan = config.plan();
  auto& ctx = at::globalContext();

  // set output
  auto output = at::empty(output_sizes, input.options());

  // set to current stream
  CUFFT_CHECK(hipfftSetStream(plan, at::cuda::getCurrentCUDAStream()));

  auto ws = at::empty({ config.workspace_size() }, at::device(at::kCUDA).dtype(at::kByte));
  CUFFT_CHECK(hipfftSetWorkArea(plan, ws.data_ptr()));

  // run
  exec_cufft_plan(config, input.data_ptr(), output.data_ptr(), !inverse);

  // rescale if requested
  auto size_last_signal_dim = checked_signal_sizes[signal_ndim - 1];
  if (norm != fft_norm_mode::none) {
    auto signal_numel = at::prod_intlist(checked_signal_sizes);
    double scale_denom;
    if (norm == fft_norm_mode::by_root_n) {
      scale_denom = std::sqrt(static_cast<double>(signal_numel));
    } else {
      scale_denom = static_cast<double>(signal_numel);
    }
    if (!complex_input && complex_output && !onesided) {
      auto end_data_slice = infer_ft_real_to_complex_onesided_size(size_last_signal_dim);
      output.narrow(signal_ndim, 0, end_data_slice).div_(scale_denom);
    } else {
      output.div_(scale_denom);
    }
  }

  // if needed, fill out the other half using conjugate symmetry
  if (!complex_input && complex_output && !onesided) {
    DimVector signal_dims(signal_ndim);
    std::iota(signal_dims.begin(), signal_dims.end(), 1);
    auto out_as_complex = at::view_as_complex(output);
    at::native::_fft_fill_with_conjugate_symmetry_(out_as_complex, signal_dims);
  }
  return output;
}

// The cuFFT plan cache
// unique_ptr for nullability and to avoid reference invalidation on vector resize
static std::vector<std::unique_ptr<CuFFTParamsLRUCache>> plan_caches;
static std::mutex plan_caches_mutex;

static inline
CuFFTParamsLRUCache &cufft_get_plan_cache(int64_t device_index) {
  std::lock_guard<std::mutex> guard(plan_caches_mutex);

  AT_ASSERT(device_index >= 0);

  if (device_index >= plan_caches.size()) {
    plan_caches.resize(device_index + 1);
  }

  if (!plan_caches[device_index]) {
    plan_caches[device_index] = std::make_unique<CuFFTParamsLRUCache>();
  }

  return *plan_caches[device_index];
}


namespace detail {

int64_t cufft_get_plan_cache_max_size_impl(int64_t device_index) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_get_plan_cache_max_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).max_size();
}

void cufft_set_plan_cache_max_size_impl(int64_t device_index, int64_t max_size) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_set_plan_cache_max_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).resize(max_size);
}

int64_t cufft_get_plan_cache_size_impl(int64_t device_index) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_get_plan_cache_size: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).size();
}

void cufft_clear_plan_cache_impl(int64_t device_index) {
  TORCH_CHECK(0 <= device_index && device_index < at::detail::getCUDAHooks().getNumGPUs(),
    "cufft_clear_plan_cache: expected 0 <= device_index < ",
    at::detail::getCUDAHooks().getNumGPUs(), "], but got device_index=",
    device_index);
  return cufft_get_plan_cache(device_index).clear();
}

} // namespace at::native::detail

namespace {
constexpr int64_t cufft_max_ndim = 3;

// Execute a general fft operation (can be c2c, onesided r2c or onesided c2r)
static Tensor& _exec_fft(Tensor& out, const Tensor& self, IntArrayRef out_sizes,
                         IntArrayRef dim, bool forward) {
  const auto ndim = self.dim();
  const int64_t signal_ndim = dim.size();
  const auto batch_dims = ndim - signal_ndim;

  // Permute dimensions so batch dimensions come first, and in stride order
  // This maximizes data locality when collapsing to a single batch dimension
  DimVector dim_permute(ndim);
  std::iota(dim_permute.begin(), dim_permute.end(), int64_t{0});

  c10::SmallVector<bool, kDimVectorStaticSize> is_transformed_dim(ndim);
  for (const auto& d : dim) {
    is_transformed_dim[d] = true;
  }
  auto batch_end = std::partition(dim_permute.begin(), dim_permute.end(),
                                  [&](int64_t d) {return !is_transformed_dim[d]; });
  auto self_strides = self.strides();
  std::sort(dim_permute.begin(), batch_end,
            [&](int64_t a, int64_t b) { return self_strides[a] > self_strides[b]; });
  std::copy(dim.cbegin(), dim.cend(), batch_end);
  auto input = self.permute(dim_permute);

  // Collapse batch dimensions into a single dimension
  DimVector batched_sizes(signal_ndim + 1);
  batched_sizes[0] = -1;
  std::copy(input.sizes().cbegin() + batch_dims, input.sizes().cend(), batched_sizes.begin() + 1);
  input = input.reshape(batched_sizes);

  const auto batch_size = input.sizes()[0];
  DimVector signal_size(signal_ndim + 1);
  signal_size[0] = batch_size;
  for (int64_t i = 0; i < signal_ndim; ++i) {
    auto in_size = input.sizes()[i + 1];
    auto out_size = out_sizes[dim[i]];
    signal_size[i + 1] = std::max(in_size, out_size);
    TORCH_INTERNAL_ASSERT(in_size == signal_size[i + 1] ||
                          in_size == (signal_size[i + 1] / 2) + 1);
    TORCH_INTERNAL_ASSERT(out_size == signal_size[i + 1] ||
                          out_size == (signal_size[i + 1] / 2) + 1);
  }

  batched_sizes[0] = batch_size;
  DimVector batched_out_sizes(batched_sizes.begin(), batched_sizes.end());
  for (size_t i = 0; i < dim.size(); ++i) {
    batched_out_sizes[i + 1] = out_sizes[dim[i]];
  }
  out.resize_(batched_out_sizes, MemoryFormat::Contiguous);

  // Create the transform plan (either from cache or locally)
  const auto value_type = c10::toValueType(input.scalar_type());
  auto fft_type = GetCuFFTTransformType(input.is_complex(), out.is_complex());
  CuFFTParams Params(input.strides(), out.strides(), signal_size, fft_type, value_type);
  CuFFTParamsLRUCache& plan_cache = cufft_get_plan_cache(input.device().index());
  std::unique_lock<std::mutex> guard(plan_cache.mutex, std::defer_lock);
  c10::optional<CuFFTConfig> uncached_plan;
  const CuFFTConfig * config = nullptr;

  if (plan_cache.max_size() > 0) {
    guard.lock();
    if (plan_cache.max_size() > 0) {  // check again after acquiring the lock
      config = &plan_cache.lookup(Params);
    }
  }

  if (config == nullptr) {
    uncached_plan.emplace(Params);
    config = &uncached_plan.value();
  }

  auto & plan = config->plan();

  if (config->should_clone_input()) {
    input = input.clone(MemoryFormat::Contiguous);
  }

  // prepare cufft for execution
  CUFFT_CHECK(hipfftSetStream(plan, at::cuda::getCurrentCUDAStream()));
  auto workspace = at::empty({ config->workspace_size() }, at::device(at::kCUDA).dtype(at::kByte));
  CUFFT_CHECK(hipfftSetWorkArea(plan, workspace.data_ptr()));

  // execute transform plan
  exec_cufft_plan(*config, input.data_ptr(), out.data_ptr(), forward);

  // Inplace reshaping to original batch shape and inverting the dimension permutation
  DimVector out_strides(ndim);
  int64_t batch_numel = 1;
  for (int64_t i = batch_dims - 1; i >= 0; --i) {
    out_strides[dim_permute[i]] = batch_numel * out.strides()[0];
    batch_numel *= out_sizes[dim_permute[i]];
  }
  for (int64_t i = batch_dims; i < ndim; ++i) {
    out_strides[dim_permute[i]] = out.strides()[1 + (i - batch_dims)];
  }
  return out.as_strided_(out_sizes, out_strides, out.storage_offset());
}

// Calculates the normalization constant and applies it in-place to self
// sizes is the sizes of a twosided tensor and dims are all transformed dims
void _fft_apply_normalization(const Tensor& self, int64_t normalization, IntArrayRef sizes, IntArrayRef dims) {
  auto norm = static_cast<fft_norm_mode>(normalization);
  if (norm == fft_norm_mode::none) {
    return;
  }

  int64_t signal_numel = 1;
  for (auto dim : dims) {
    signal_numel *= sizes[dim];
  }
  const double scale_denom = (norm == fft_norm_mode::by_root_n) ?
    std::sqrt(signal_numel) : static_cast<double>(signal_numel);
  self.div_(scale_denom);
}

}  // namespace (anonymous)

// n-dimensional real to complex FFT
Tensor _fft_r2c_cufft(const Tensor& self, IntArrayRef dim, int64_t normalization, bool onesided) {
  TORCH_CHECK(self.is_floating_point());
  auto input_sizes = self.sizes();
  DimVector onesided_sizes(input_sizes.begin(), input_sizes.end());
  auto last_dim = dim.back();
  auto last_dim_halfsize = (input_sizes[last_dim]) / 2 + 1;
  onesided_sizes[last_dim] = last_dim_halfsize;
  IntArrayRef out_sizes = onesided ? onesided_sizes : input_sizes;

  const auto out_options = self.options().dtype(c10::toComplexType(self.scalar_type()));
  auto output = at::empty(out_sizes, out_options);

  // CuFFT requires real input to be over-aligned, as if it were complex
  const auto complex_size = 2 * self.element_size();
  const bool complex_aligned = (
      reinterpret_cast<std::uintptr_t>(self.data_ptr()) % complex_size == 0);
  auto working_tensor = self;
  if (!complex_aligned) {
    working_tensor = self.movedim(last_dim, -1)
                         .clone(MemoryFormat::Contiguous)
                         .movedim(-1, last_dim);
  }

  // First do the R2C transform on the last dimension
  {
    auto target_sizes = dim.size() == 1 ? out_sizes : onesided_sizes;
    _exec_fft(output, working_tensor, target_sizes, last_dim, /*forward=*/true);
    if (dim.size() > 1) {
      working_tensor = at::empty(out_sizes, out_options);
    }
  }

  // Then any remaining C2C transforms
  DimVector sorted_dims(dim.begin(), dim.end() - 1);
  while (!sorted_dims.empty()) {
    std::swap(output, working_tensor);

    // Resort dimensions every time as _exec_fft re-strides the output
    auto strides = working_tensor.strides();
    std::sort(sorted_dims.begin(), sorted_dims.end(),
              [&](int64_t a, int64_t b) { return strides[a] > strides[b]; });

    const auto max_dims = std::min(static_cast<size_t>(cufft_max_ndim), sorted_dims.size());
    auto last_dims = IntArrayRef(sorted_dims).slice(sorted_dims.size() - max_dims, max_dims);

    // Intermediate results are always onesided
    _exec_fft(output, working_tensor, onesided_sizes, last_dims, /*forward=*/true);
    sorted_dims.resize(sorted_dims.size() - max_dims);
  }

  // Only need to normalize the onesided slice since data in the other half is overwritten
  auto out_slice = output.slice(last_dim, 0, last_dim_halfsize);
  _fft_apply_normalization(out_slice, normalization, input_sizes, dim);

  if (!onesided) {
    if (output.sizes()[last_dim] != out_sizes[last_dim]) {
      working_tensor.resize_(out_sizes, MemoryFormat::Contiguous);
      working_tensor.slice(last_dim, 0, last_dim_halfsize).copy_(output);
      output = std::move(working_tensor);
    }
    at::native::_fft_fill_with_conjugate_symmetry_(output, dim);
  }
  return output;
}

// n-dimensional complex to real IFFT
Tensor _fft_c2r_cufft(const Tensor& self, IntArrayRef dim, int64_t normalization, int64_t lastdim) {
  TORCH_CHECK(self.is_complex());
  auto in_sizes = self.sizes();
  DimVector out_sizes(in_sizes.begin(), in_sizes.end());
  out_sizes[dim.back()] = lastdim;

  // First complete any C2C transforms
  Tensor temp;
  if (dim.size() > 1) {
    temp = _fft_c2c_cufft(
        self, dim.slice(0, dim.size() - 1),
        static_cast<int64_t>(fft_norm_mode::none), /*forward=*/false);
  } else {
    // Complex to real FFTs may overwrite the input buffer, so must always clone (gh-34551)
    temp = self.clone(MemoryFormat::Contiguous);
  }

  // Finally, do a 1D C2R transform
  // TODO: could transform up to 2 other dims in the same cuFFT operation
  auto output = at::empty(out_sizes, self.options().dtype(c10::toValueType(self.scalar_type())));
  _exec_fft(output, temp, out_sizes, dim.back(), /*forward=*/false);
  _fft_apply_normalization(output, normalization, out_sizes, dim);
  return output;
}

// n-dimensional complex to complex FFT/IFFT
Tensor _fft_c2c_cufft(const Tensor& self, IntArrayRef dim, int64_t normalization, bool forward) {
  TORCH_CHECK(self.is_complex());
  if (dim.empty()) {
    return self.clone();
  }

  auto out_sizes = self.sizes();
  auto output = at::empty(out_sizes, self.options());

  // Perform any number of C2C transforms
  DimVector sorted_dims(dim.begin(), dim.end());
  auto self_strides = self.strides();
  auto working_tensor = self;
  while (true) {
    // Sort dimensions every time as _exec_fft re-strides the output
    auto strides = working_tensor.strides();
    std::sort(sorted_dims.begin(), sorted_dims.end(),
              [&](int64_t a, int64_t b) { return strides[a] > strides[b]; });

    const auto max_dims = std::min(static_cast<size_t>(cufft_max_ndim), sorted_dims.size());
    auto first_dims = IntArrayRef(sorted_dims).slice(sorted_dims.size() - max_dims, max_dims);

    _exec_fft(output, working_tensor, out_sizes, first_dims, forward);
    sorted_dims.resize(sorted_dims.size() - max_dims);

    if (sorted_dims.empty()) {
      break;
    }

    if (working_tensor.is_same(self)) {
      working_tensor = std::move(output);
      output = at::empty(out_sizes, self.options());
    } else {
      std::swap(output, working_tensor);
    }
  }

  _fft_apply_normalization(output, normalization, out_sizes, dim);
  return output;
}


}} // at::native
