#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>

namespace at {
namespace native {
namespace {

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bicubic2d_out_frame(
    const int num_elements,
    const accscalar_t height_scale,
    const accscalar_t width_scale,
    const bool align_corners,
    const PackedTensorAccessor64<scalar_t, 4> idata,
    PackedTensorAccessor64<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int input_height = idata.size(2);
  const int input_width = idata.size(3);
  const int output_height = odata.size(2);
  const int output_width = odata.size(3);

  if (index >= num_elements) {
    return;
  }

  // Special case: input and output are the same size, just copy
  const int output_x = index % output_width;
  const int output_y = index / output_width;

  if (input_height == output_height && input_width == output_width) {
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; c++) {
        const scalar_t val = idata[n][c][output_y][output_x];
        odata[n][c][output_y][output_x] = val;
      }
    }
    return;
  }

  // Interpolation kernel
  accscalar_t real_x = area_pixel_compute_source_index(
      width_scale, output_x, align_corners, /*cubic=*/true);
  int in_x = floorf(real_x);
  accscalar_t t_x = real_x - in_x;

  accscalar_t real_y = area_pixel_compute_source_index(
      height_scale, output_y, align_corners, /*cubic=*/true);
  int in_y = floorf(real_y);
  accscalar_t t_y = real_y - in_y;

  for (int n = 0; n < batchsize; n++) {
    for (int c = 0; c < channels; c++) {
      accscalar_t coefficients[4];

      for (int k = 0; k < 4; k++) {
        coefficients[k] = cubic_interp1d(
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x - 1),
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x + 0),
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x + 1),
            upsample_get_value_bounded<scalar_t>(
                idata, n, c, input_height, input_width, in_y - 1 + k, in_x + 2),
            t_x);
      }

      odata[n][c][output_y][output_x] = static_cast<scalar_t>(cubic_interp1d(
          coefficients[0],
          coefficients[1],
          coefficients[2],
          coefficients[3],
          t_y));
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bicubic2d_backward_out_frame(
    const int num_elements,
    const accscalar_t height_scale,
    const accscalar_t width_scale,
    const bool align_corners,
    PackedTensorAccessor64<scalar_t, 4> idata,
    const PackedTensorAccessor64<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int input_height = idata.size(2);
  const int input_width = idata.size(3);
  const int output_height = odata.size(2);
  const int output_width = odata.size(3);

  if (index >= num_elements) {
    return;
  }

  const int output_x = index % output_width;
  const int output_y = index / output_width;
  // special case: output_xust copy
  if (input_height == output_height && input_width == output_width) {
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const scalar_t val = odata[n][c][output_y][output_x];
        idata[n][c][output_y][output_x] = val;
      }
    }
    return;
  }

  accscalar_t real_x = area_pixel_compute_source_index(
      width_scale, output_x, align_corners, /*cubic=*/true);
  int input_x = floorf(real_x);
  accscalar_t t_x = real_x - input_x;

  accscalar_t real_y = area_pixel_compute_source_index(
      height_scale, output_y, align_corners, /*cubic=*/true);
  int input_y = floorf(real_y);
  accscalar_t t_y = real_y - input_y;

  accscalar_t x_coeffs[4];
  accscalar_t y_coeffs[4];

  get_cubic_upsampling_coefficients(x_coeffs, t_x);
  get_cubic_upsampling_coefficients(y_coeffs, t_y);

  for (int n = 0; n < batchsize; n++) {
    for (int c = 0; c < channels; ++c) {
      scalar_t out_value = odata[n][c][output_y][output_x];
      for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
          upsample_increment_value_bounded<scalar_t, accscalar_t>(
              idata,
              n,
              c,
              input_height,
              input_width,
              input_y - 1 + i,
              input_x - 1 + j,
              out_value * y_coeffs[i] * x_coeffs[j]);
        }
      }
    }
  }
}

static void upsample_bicubic2d_out_cuda_template(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_bicubic2d_out", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_height = input.size(2);
  int input_width = input.size(3);

  upsample_2d_shape_check(
      input,
      Tensor(),
      nbatch,
      channels,
      input_height,
      input_width,
      output_height,
      output_width);

  output.resize_({input.size(0), input.size(1), output_height, output_width});
  output.zero_();

  AT_ASSERT(
      input_height > 0 && input_width > 0 && output_height > 0 &&
      output_width > 0);

  const int num_output_elements = output_height * output_width;
  const int max_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);

  // Launch kernel
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_bicubic2d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<scalar_t, 4>();
        auto odata = output.packed_accessor64<scalar_t, 4>();

        // Get scaling factors
        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_bicubic2d_out_frame<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_output_elements, max_threads),
               max_threads,
               0,
               stream>>>(
                num_output_elements,
                rheight,
                rwidth,
                align_corners,
                idata,
                odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_bicubic2d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      "upsample_bicubic2d_backward_out_cuda",
      {grad_output_arg, grad_input_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 4,
      "It is expected input_size equals to 4, but got size ",
      input_size.size());

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_height = input_size[2];
  int input_width = input_size[3];

  upsample_2d_shape_check(
      Tensor(),
      grad_output_,
      nbatch,
      channels,
      input_height,
      input_width,
      output_height,
      output_width);

  Tensor grad_output = grad_output_.contiguous();

  grad_input.resize_({nbatch, channels, input_height, input_width});
  grad_input.zero_();

  const int num_kernels = output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "upsample_bicubic2d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.packed_accessor64<scalar_t, 4>();
        auto odata = grad_output.packed_accessor64<scalar_t, 4>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_bicubic2d_backward_out_frame<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(
                num_kernels, rheight, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

Tensor& upsample_bicubic2d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  upsample_bicubic2d_out_cuda_template(
      output, input, output_size, align_corners, scales_h, scales_w);
  return output;
}

Tensor upsample_bicubic2d_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_bicubic2d_out_cuda_template(
      output, input, output_size, align_corners, scales_h, scales_w);
  return output;
}

Tensor& upsample_bicubic2d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bicubic2d_backward_out_cuda");
  upsample_bicubic2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_h, scales_w);
  return grad_input;
}

Tensor upsample_bicubic2d_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bicubic2d_backward_cuda");
  Tensor grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_bicubic2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_h, scales_w);
  return grad_input;
}

using at::native::upsample::compute_output_size;
using at::native::upsample_cuda::get_scale_value;

Tensor upsample_bicubic2d_cuda(
    const Tensor& input,
    c10::optional<IntArrayRef> output_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto osize = compute_output_size(input.sizes(), output_size, scale_factors);
  auto scale_h = get_scale_value(scale_factors, 0);
  auto scale_w = get_scale_value(scale_factors, 1);
  upsample_bicubic2d_out_cuda_template(output, input, osize, align_corners, scale_h, scale_w);
  return output;
}

Tensor upsample_bicubic2d_backward_cuda(
    const Tensor& grad_output,
    c10::optional<IntArrayRef> output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bicubic2d_backward_cuda");
  auto osize = compute_output_size(input_size, output_size, scale_factors);
  auto scale_h = get_scale_value(scale_factors, 0);
  auto scale_w = get_scale_value(scale_factors, 1);
  auto grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_bicubic2d_backward_out_cuda_template(
      grad_input, grad_output, osize, input_size, align_corners, scale_h, scale_w);
  return grad_input;
}

} // namespace native
} // namespace at
