#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/TensorFactories.h>
#include <ATen/native/cuda/Resize.cuh>
#include <c10/util/Exception.h>

#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>

#include <algorithm>
#include <cstddef>
#include <cmath>

namespace at {
namespace native {

Tensor& eye_out_cuda(Tensor& result, int64_t n) {
  // the default value of `m` equals to `n`
  return at::native::eye_out_cuda(result, n, n);
}

Tensor& eye_out_cuda(Tensor& result, int64_t n, int64_t m) {
  TORCH_CHECK(n >= 0, "n must be greater or equal to 0, got ", n);
  TORCH_CHECK(m >= 0, "m must be greater or equal to 0, got ", m);

  result.resize_({n, m});
  result.zero_();

  int64_t sz = std::min<int64_t>(n, m);
  int64_t stride = result.stride(0) + result.stride(1);

  Tensor diag = result.as_strided({sz}, {stride});
  diag.fill_(1);
  return result;
}

Tensor empty_cuda(IntArrayRef size, c10::optional<ScalarType> dtype_opt, c10::optional<Layout> layout_opt, c10::optional<Device> device_opt, c10::optional<bool> pin_memory_opt, c10::optional<c10::MemoryFormat> memory_format_opt) {
  AT_ASSERT(device_or_default(device_opt).type() == at::DeviceType::CUDA);
  TORCH_CHECK(!pin_memory_opt.has_value() || !*pin_memory_opt, "Only dense CPU tensors can be pinned");
  check_size_nonnegative(size);

  auto* allocator = at::cuda::getCUDADeviceAllocator();
  int64_t nelements = prod_intlist(size);
  auto dtype = dtype_or_default(dtype_opt);
  auto dtype_meta = scalarTypeToTypeMeta(dtype);
  int64_t size_bytes = nelements * dtype_meta.itemsize();
  auto storage_impl = c10::make_intrusive<StorageImpl>(
      c10::StorageImpl::use_byte_size_t(),
      size_bytes,
      allocator->allocate(size_bytes),
      allocator,
      /*resizeable=*/true);

  auto tensor =
      detail::make_tensor<TensorImpl>(storage_impl, DispatchKey::CUDA, dtype_meta);
  // Default TensorImpl has size [0]
  if (size.size() != 1 || size[0] != 0) {
    tensor.unsafeGetTensorImpl()->set_sizes_contiguous(size);
  }

  auto memory_format = memory_format_opt.value_or(MemoryFormat::Contiguous);
  tensor.unsafeGetTensorImpl()->empty_tensor_restride(memory_format);
  return tensor;
}

Tensor empty_strided_cuda(IntArrayRef size, IntArrayRef stride, c10::optional<ScalarType> dtype_opt, c10::optional<Layout> layout_opt, c10::optional<Device> device_opt, c10::optional<bool> pin_memory_opt) {
  auto t = at::native::empty_cuda({0}, dtype_opt, layout_opt, device_opt, pin_memory_opt);
  at::native::resize_impl_cuda_(t.unsafeGetTensorImpl(), size, stride);
  return t;
}

Tensor& randperm_out_cuda(Tensor& result, int64_t n, c10::optional<Generator> generator) {
  TORCH_CHECK(n >= 0, "n must be non-negative, got", n);
  TORCH_CHECK(!generator.has_value() || (generator.has_value() && result.device() == generator->device()), "Expected a '", result.device(), "' generator device but found '", generator->device(), "'");
  check_supported_max_int_with_precision(n, result);

  result.resize_({n});

  if (n < 30000) {  // For small inputs, we offload it to CPU instead.
    auto result_cpu = at::empty({n}, result.options().device(kCPU));
    randperm_out(result_cpu, n, generator);
    return result.copy_(result_cpu);
  }

#if 0
  // This if condition should never be true because if n >= 30000 and the tensor has a Half type,
  // check_supported_max_int_with_precision should have reported an error. This snippet is commented out but left here
  // for the sake of clarity, because Half in thrust is spotty, and we do not want future change unaware of this.
  if (result.scalar_type() == at::ScalarType::Half) {  // Half in thrust is spotty. Avoid.
    auto result_float = at::empty({n}, initialTensorOptions().device(Device(DeviceType::CUDA)));
    return result.copy_(randperm_out_cuda(result_float, n, generator));
  }
#endif

  // Generate random values for the keys array
  AT_DISPATCH_ALL_TYPES(
    result.scalar_type(), "randperm_out_cuda", [&] {
      auto keys = at::empty(result.sizes(), result.options()).random_(generator);
      auto keys_data = thrust::device_ptr<scalar_t>(keys.data_ptr<scalar_t>());

      // shuffled_data points to the underlying data of the output tensor if the tensor is contiguous; otherwise it
      // points to a new tensor.
      Tensor shuffled;
      thrust::device_ptr<scalar_t> shuffled_data;
      if (result.is_contiguous()) {
        shuffled_data = thrust::device_ptr<scalar_t>(result.data_ptr<scalar_t>());
      } else {
        shuffled = at::empty(n, result.options());
        shuffled_data = thrust::device_ptr<scalar_t>(shuffled.data_ptr<scalar_t>());
      }

      auto state = globalContext().getTHCState();
      THCThrustAllocator thrustAlloc(state);
      auto policy = thrust::cuda::par(thrustAlloc).on(at::cuda::getCurrentCUDAStream());

      thrust::sequence(policy, shuffled_data, shuffled_data + n);

      // Use the sorted order of keys to rearrange the result array
      thrust::sort_by_key(policy, keys_data, keys_data + n, shuffled_data);

      if (!result.is_contiguous()) {
        result.copy_(shuffled);
      }
    }
  );

  return result;
}

// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ triangle ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

namespace {
// To find the max integer that does not exceed the root of an int64_t variable,
// we could use a loop to test one bit at a time, which takes up to 31
// iterations. This would give the accurate result, but is relatively slow and
// is an overkill for most cases where double's precision suffice.
//
// If we directly use sqrt to calculate the root, the conversion from int64_t
// to double would lose 11 bits precision.
//
// The following solution uses sqrt directly for most cases, and would only
// special handle it if there is indeed precision loss.
__device__
inline int64_t resolve_root_int(
    int64_t b, int64_t cX4, int64_t x, int32_t sign) {
  int64_t bXb_cX4 = b*b - cX4;
  // potential precision loss could occur here when casting int64_t (63 bits
  // precision) to double (52 bits precision)
  double sr = ::sqrt((double)bXb_cX4);
  int64_t res = ::__double2ll_rd((-b + sign * sr)/2);

  // have to cast double to int64_t, otherwise it would only compare up to the
  // precision of a double variable, ignoring the precision loss
  if (bXb_cX4 != (int64_t) (sr * sr)) {
    // handle precision loss by using binary search
    int64_t llsr = ::__double2ll_rd(sr);
    // Use the following math to reduce search space.
    // Suppose z is the accurate result of sqrt(bXb_cX4) without precision loss
    // let d = abs(bXb_cX4 - llsr * llsr), then we have:
    // z = sqrt(bXb_cX4) <= sqrt(llsr * llsr + d) <= llsr + sqrt(d)
    // z = sqrt(bXb_cX4) >= sqrt(llsr * llsr - d) >= llsr - sqrt(d)
    // Hence, it is sufficient to search range [llsr - sqrt(d), llsr + sqrt(d)).
    // And the true value of row would also be with in range,
    //            [res - sqrt(d), res + sqrt(d) + 1)
    // as the denominator would only reduce the precision penalty.
    int64_t diff =
      ::__double2ll_ru(::sqrt(::fabs((double)(bXb_cX4 - llsr * llsr))));
    // l never exceeds (could equal to) the target row index
    auto l = res > diff ? res - diff : 0;
    // r is always larger than the target row index
    auto r = res + diff + 1;

    // binary search for the correct answer
    x <<= 1; // the loop always compares with 2x, so do it once here
    while (l + 1 < r) {
      auto m = (l + r) >> 1;
      // for tril:
      //    b = 2f - 1, sign = 1, hence (2f + m - 1) * m / 2
      // for triu:
      //    b = -2f - 1, sign = -1, hence (2f - m + 1) * m / 2
      if (sign * (b + m) * m > x) {
        r = m;
      } else {
        l = m;
      }
    }
    res = l;
  }

  return res;
}

// f: the number of elements in the first row of the trapezoid.
// x: the index of the target coordinates ordered by row and then column.
//
// View the tril as a top trapezoid stacked on a bottom rectangle. Assume x
// corresponds to the coordinate (row, col) in the trapezoid, where the row and
// the col both start from 0, then we have:
//
//                   (f + f + row - 1) * row / 2 <= x                       [1]
//                 (f + f + row) * (row + 1) / 2  > x                       [2]
//
// Therefore, row is the maximum integer satisfying the following inequality:
//
//                       (row + 2f - 1)row <= 2x
//                  row^2 + (2f-1)row - 2x <= 0.                            [3]
//
// Based on inequality [3], we have the following coefficients for formula of
// root:
//                               a = 1
//                               b = 2f - 1
//                               c = -2x
// There are two roots, and we should use the largest integer that does not
// exceed the root on the right. Intuitively, it is because:
//  i)  the valid solution range of row is between two roots, as it is <= 0;
//  ii) as we count in more rows, the total # of elements should always
//      increase, hence so does the left-hand side row^2 + (2f-1)row - 2x.
//      Therefore, the valid range of row lies in between the nadir point and
//      the larger root on the right.
// Full proof can be derived from inequality [2]. So, we calculate the result
// coordinate as:
//
//                   row = floor((-b + sqrt(b^2 - 4c)) / 2)
//                   col = x - (f + f + row - 1) * row / 2
__device__
inline void get_coordinate_in_tril_trapezoid(
    int64_t f, int64_t x, int64_t & row, int64_t & col) {
  f <<= 1; // all statements use 2f, so only calculate it once here.
  auto b = f - 1;
  auto cX4 = - (x << 3); // 4 * c = 4 * (-2x) = -8x;
  row = resolve_root_int(b, cX4, x, 1);
  col = x - ((f + row - 1) * row >> 1);
}

// f: the number of elements in the first row of the bottom trapezoid.
// x: the index of the target coordinates ordered by row and then column.
//
// View the triu as a top rectangle stacked on a bottom trapezoid, where the
// trapezoid is upside down. Assume x corresponds to the coordinate (row, col)
// in the bottom trapezoid, where the row and the col start from 0, then we
// have:
//
//                   (f + f - row + 1) * row / 2 <= x                       [1]
//                 (f + f - row) * (row + 1) / 2  > x                       [2]
//
// Therefore, row is the maximum integer satisfying the following inequality:
//
//                       (-row + 2f + 1)row <= 2x
//                   row^2 - (2f+1)row + 2x >= 0.                           [3]
//
// Based on inequality [3], we have the following coefficients for formula of
// root:
//                               a = 1
//                               b = -1 - 2f
//                               c = 2x
// There are two roots, and we should use the largest integer that does not
// exceed the root on the left. Intuitively, it is because:
//  i)  the valid solution range of row is outside of the two roots, as it is <
//      > 0;
//  ii) as we count in more rows, the total # of elements should always
//      increase, hence so does the left-hand side row^2 - (2f+1)row + 2x.
//      Therefore, the valid range of row lies to the left of the smaller root
//      on the left.
// Full proof can be derived from inequality [2]. So, we calculate the result
// coordinate as:
//
//                   row = floor((-b - sqrt(b^2 - 4c)) / 2)
//                   col = x - (f + f - row + 1) * row / 2
__device__
inline void get_coordinate_in_triu_trapezoid(
    int64_t f, int64_t x, int64_t & row, int64_t & col) {
  f <<= 1; // all statements use 2f, so only calculate it once here.
  auto b = -1 - f;
  auto cX4 = x << 3; // 4 * c = 4 * (2x) = 8x;
  row = resolve_root_int(b, cX4, x, -1);
  col = x - ((f - row + 1) * row >> 1) + row;
}

} // namespace

template <typename scalar_t>
__global__
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
void tril_indices_kernel(scalar_t * tensor,
                         int64_t row_offset,
                         int64_t m_first_row,
                         int64_t col,
                         int64_t trapezoid_size,
                         int64_t tril_size) {
  int64_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (linear_index < tril_size) {
    int64_t r, c;
    if (linear_index < trapezoid_size) {
      // the coordinate is within the top trapezoid
      get_coordinate_in_tril_trapezoid(m_first_row, linear_index, r, c);
    } else {
      // the coordinate falls in the bottom rectangle
      auto surplus = linear_index - trapezoid_size;
      // add the height of trapezoid: m_last_row (col) - m_first_row + 1
      r = surplus / col + col - m_first_row + 1;
      c = surplus % col;
    }
    r += row_offset;

    tensor[linear_index] = r;
    tensor[linear_index + tril_size] = c;
  }
}

// Some Large test cases for the fallback binary search path is disabled by
// default to speed up CI tests and to avoid OOM error. When modifying the
// implementation, please enable them in test/test_cuda.py and make sure they
// pass on your local server.
Tensor tril_indices_cuda(
    int64_t row, int64_t col, int64_t offset, c10::optional<ScalarType> dtype_opt,
    c10::optional<Layout> layout_opt, c10::optional<Device> device_opt, c10::optional<bool> pin_memory_opt) {
  check_args(row, col, layout_opt);

  auto tril_size = get_tril_size(row, col, offset);
  auto tensor = empty_cuda({2, tril_size}, dtype_opt, layout_opt, device_opt, pin_memory_opt);

  if (tril_size > 0) {
    auto m_first_row = offset > 0 ?
      std::min<int64_t>(col, 1 + offset) : // upper bounded by col
      row + offset > 0; // either 0 or 1
    auto trapezoid_row_offset = std::max<int64_t>(0, -offset);
    auto rectangle_row_offset = trapezoid_row_offset + col - m_first_row + 1;
    int64_t rectangle_size = 0;
    if (rectangle_row_offset < row) {
      rectangle_size = (row - rectangle_row_offset) * col;
    }

    dim3 dim_block = cuda::getApplyBlock();
    dim3 dim_grid;
    // using tril_size instead of tensor.numel(), as each thread takes care of
    // two elements in the tensor.
    TORCH_CHECK(
      cuda::getApplyGrid(tril_size, dim_grid, tensor.get_device()),
      "unable to get dim grid");

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, tensor.scalar_type(), "tril_indices_cuda", [&] {
      tril_indices_kernel<<<
          dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        tensor.data_ptr<scalar_t>(),
        trapezoid_row_offset,
        m_first_row,
        col,
        tril_size - rectangle_size,
        tril_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }

  return tensor;
}

template <typename scalar_t>
__global__
void triu_indices_kernel(scalar_t * tensor,
                         int64_t col_offset,
                         int64_t m_first_row,
                         int64_t col,
                         int64_t rectangle_size,
                         int64_t triu_size) {
  int64_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (linear_index < triu_size) {
    int64_t r, c;
    if (linear_index < rectangle_size) {
      // the coordinate is within the top rectangle
      r = linear_index / col;
      c = linear_index % col;
    } else {
      // the coordinate falls in the bottom trapezoid
      get_coordinate_in_triu_trapezoid(
        m_first_row, linear_index - rectangle_size, r, c);
      r += rectangle_size / col;
    }

    c += col_offset;
    tensor[linear_index] = r;
    tensor[linear_index + triu_size] = c;
  }
}

// Some Large test cases for the fallback binary search path is disabled by
// default to speed up CI tests and to avoid OOM error. When modifying the
// implementation, please enable them in test/test_cuda.py and make sure they
// pass on your local server.
Tensor triu_indices_cuda(
    int64_t row, int64_t col, int64_t offset, c10::optional<ScalarType> dtype_opt,
    c10::optional<Layout> layout_opt, c10::optional<Device> device_opt, c10::optional<bool> pin_memory_opt) {
  check_args(row, col, layout_opt);

  auto triu_size = row * col - get_tril_size(row, col, offset - 1);
  auto tensor = empty_cuda({2, triu_size}, dtype_opt, layout_opt, device_opt, pin_memory_opt);

  if (triu_size > 0) {
    // # of triu elements in the first row
    auto m_first_row = offset > 0 ?
      std::max<int64_t>(col - offset, 0) : // upper bounded by col
      col;

    // size of the top rectangle
    int64_t rectangle_size = 0;
    if (offset < 0) {
      rectangle_size = std::min<int64_t>(row, -offset) * col;
    }

    dim3 dim_block = cuda::getApplyBlock();
    dim3 dim_grid;

    // using triu_size instead of tensor.numel(), as each thread takes care of
    // two elements in the tensor.
    TORCH_CHECK(
      cuda::getApplyGrid(triu_size, dim_grid, tensor.get_device()),
      "unable to get dim grid");

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, tensor.scalar_type(), "triu_indices_cuda", [&] {
      triu_indices_kernel<<<
          dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
        tensor.data_ptr<scalar_t>(),
        std::max<int64_t>(0, offset),
        m_first_row,
        col,
        rectangle_size,
        triu_size);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
  }

  return tensor;
}

}} // namespace at::native
