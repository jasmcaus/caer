#include "hip/hip_runtime.h"
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

#include <type_traits>

namespace at { namespace native {

void logical_not_kernel_cuda(TensorIterator& iter) {
  // error check -- this is just ensuring we don't dispatch on types that aren't in ALL_TYPES_AND_COMPLEX_AND3(...)
  // so we don't have to maintain a separate list or to do double dispatch.
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kBool, kHalf, kBFloat16, iter.dtype(0), "logical_not_cuda", [&]() {});

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kBool, kHalf, kBFloat16, iter.dtype(1), "logical_not_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> bool { return !a; });
  });
}

void neg_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "neg_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return -a;
    });
  });
}

void sign_kernel_cuda(TensorIterator& iter){
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a){
      return a;
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "sign_cuda", [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
            scalar_t zero = scalar_t(0);
            return (zero < a) - (a < zero);
        });
    });
  }
}

void signbit_kernel_cuda(TensorIterator& iter){
  AT_DISPATCH_ALL_TYPES_AND2(kBFloat16, ScalarType::Half, iter.input_dtype(), "signbit_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> bool { return !std::is_unsigned<scalar_t>::value && a < 0; });
  });
}

template<typename T>
__host__ __device__ static inline c10::complex<T> sgn_wrapper(c10::complex<T> z) {
  if (z == c10::complex<T>(0, 0)) {
    return c10::complex<T>(0, 0);
  } else {
    return z / std::abs(z);
  }
}

void sgn_kernel_cuda(TensorIterator& iter){
  AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "sgn_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return sgn_wrapper(a);
      });
  });
}
REGISTER_DISPATCH(logical_not_stub, &logical_not_kernel_cuda);
REGISTER_DISPATCH(neg_stub, &neg_kernel_cuda);
REGISTER_DISPATCH(sign_stub, &sign_kernel_cuda);
REGISTER_DISPATCH(signbit_stub, &signbit_kernel_cuda);
REGISTER_DISPATCH(sgn_stub, &sgn_kernel_cuda);

}} // namespace at::native
