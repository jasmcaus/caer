#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

constexpr float EPSILON = 1e-12;

namespace {

using namespace at;

void binary_cross_entropy_backward_out_kernel(Tensor& grad_input, const Tensor& grad, const Tensor& input, const Tensor& target) {
  at::TensorIterator iter = TensorIteratorConfig()
      .add_output(grad_input)
      .add_input(grad)
      .add_input(input)
      .add_input(target)
      .build();
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "binary_cross_entropy_backward_out_cuda", [&]() {
    at::native::gpu_kernel(iter, [] GPU_LAMBDA (
        scalar_t grad_val,
        scalar_t input_val,
        scalar_t target_val
      ) -> scalar_t {
        const scalar_t one = 1;
        const scalar_t epsilon = EPSILON;

        scalar_t grad_input_denominator = max(
          (one - input_val) * input_val,
          epsilon
        );

        return grad_val * (input_val - target_val) / grad_input_denominator;
      }
    );
  });
}

} // namespace

namespace at { namespace native {

Tensor kl_div_backward_cuda(const Tensor& grad, const Tensor& input, const Tensor& target, int64_t reduction, bool log_target) {
  auto grad_input = at::empty_like(input);
  if (!log_target) {
    TensorIterator iter = TensorIteratorConfig()
        .add_output(grad_input)
        .add_input(target)
        .add_input(grad)
        .build();
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "kl_div_backward_cuda", [&]() {
      scalar_t inv = (reduction == at::Reduction::Mean) ? scalar_t(1.0 / input.numel()) : scalar_t(1.0);
      gpu_kernel(iter,
        [inv] GPU_LAMBDA (scalar_t target_val, scalar_t grad_val) {
          return (target_val > 0) ? scalar_t(-target_val * grad_val * inv) : scalar_t(0.0);
        });
    });
  }
  else { 
    grad_input = -at::exp(target) * grad;
    if (reduction == at::Reduction::Mean) {
      grad_input /= input.numel();
    }
  }

  return grad_input;
}

Tensor binary_cross_entropy_cuda(const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
    Tensor loss = at::empty_like(input);
    return at::native::binary_cross_entropy_out_cuda(loss, input, target, weight, reduction);
}

Tensor& binary_cross_entropy_out_cuda(Tensor& loss, const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
  Tensor loss_squeezed = at::squeeze(loss);

  TensorIterator iter = TensorIteratorConfig()
      .add_output(loss_squeezed)
      .add_input(at::squeeze(input))
      .add_input(at::squeeze(target))
      .build();
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "binary_cross_entropy_out_cuda", [&]() {
    gpu_kernel(iter,
      [] GPU_LAMBDA (scalar_t input_val, scalar_t target_val) -> scalar_t {
        const scalar_t zero = 0;
        const scalar_t one = 1;
        const scalar_t neg_100 = -100;

        CUDA_KERNEL_ASSERT(input_val >= zero && input_val <= one);

        scalar_t log_input_val = std::log(input_val);
        scalar_t log_1_minus_input_val = std::log(one - input_val);

        log_input_val = std::max(log_input_val, neg_100);
        log_1_minus_input_val = std::max(log_1_minus_input_val, neg_100);

        return ((target_val - one) * log_1_minus_input_val) - (target_val * log_input_val);
      }
    );
  });
  if (weight.defined()) {
    loss.mul_(weight);
  }

  if (reduction != at::Reduction::None) {
    Tensor loss_reduced;
    if (reduction == at::Reduction::Mean) {
      loss_reduced = loss.mean();
    } else if (reduction == at::Reduction::Sum) {
      loss_reduced = loss.sum();
    }
    loss.resize_as_(loss_reduced).copy_(loss_reduced);
  }

  return loss;
}

Tensor binary_cross_entropy_backward_cuda(const Tensor& grad, const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
  Tensor grad_input = at::empty_like(input);
  return at::native::binary_cross_entropy_backward_out_cuda(grad_input, grad, input, target, weight, reduction);
}

Tensor& binary_cross_entropy_backward_out_cuda(Tensor& grad_input, const Tensor& grad, const Tensor& input, const Tensor& target, const Tensor& weight, int64_t reduction) {
  Tensor grad_expand = grad.expand_as(input);
  binary_cross_entropy_backward_out_kernel(grad_input, grad_expand, input, target);

  if (weight.defined()) {
    grad_input.mul_(weight);
  }
  if (reduction == at::Reduction::Mean) {
    grad_input.div_(input.numel());
  }
  return grad_input;
}

}}  // namespace at::native
