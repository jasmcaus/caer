#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void digamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "digamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_digamma(a);
    });
  });
}

void trigamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "trigamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_trigamma(a);
    });
  });
}

void polygamma_kernel_cuda(TensorIterator& iter, int64_t n) {
  if (n == 0) {
    digamma_kernel_cuda(iter);
  } else if (n == 1) {
    trigamma_kernel_cuda(iter);
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "polygamma_cuda", [&]() {
      gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_polygamma(int(n), a);
      });
    });
  }
}

void lgamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "lgamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::lgamma(a);
    });
  });
}

REGISTER_DISPATCH(digamma_stub, &digamma_kernel_cuda);
REGISTER_DISPATCH(polygamma_stub, &polygamma_kernel_cuda);
REGISTER_DISPATCH(lgamma_stub, &lgamma_kernel_cuda);

}} // namespace at::native
