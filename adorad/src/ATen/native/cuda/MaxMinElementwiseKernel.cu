#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void maximum_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a || b;
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "max_elementwise_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::max(a, b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "max_elementwise_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        if (a != a) {
          return a;
        } else if (b != b) {
          return b;
        } else {
          return ::max(a, b);
        }
      });
    });
  }
}

void minimum_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a && b;
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "minimum_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::min(a, b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "min_elementwise_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        if (a != a) {
          return a;
        } else if (b != b) {
          return b;
        } else {
          return ::min(a, b);
        }
      });
    });
  }
}

REGISTER_DISPATCH(maximum_stub, &maximum_kernel_cuda);
REGISTER_DISPATCH(minimum_stub, &minimum_kernel_cuda);

}} // namespace at::native
