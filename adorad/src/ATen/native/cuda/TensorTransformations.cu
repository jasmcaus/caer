#include "hip/hip_runtime.h"
#include <ATen/native/TensorTransformations.h>

#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <c10/macros/Macros.h>

#include <cstddef>
#include <vector>

namespace at {
namespace native {

constexpr uint32_t AT_APPLY_THREADS_PER_BLOCK = 512;
constexpr uint32_t AT_APPLY_BLOCKS_PER_SM = 4;

template <typename scalar_t, typename IndexType>
#if __CUDA_ARCH__ >= 350 || defined __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(AT_APPLY_THREADS_PER_BLOCK, AT_APPLY_BLOCKS_PER_SM)
#endif
__global__ void
kernel_pointwise_flip_apply2(const cuda::detail::TensorInfo<scalar_t, IndexType> in_tensor_info,
                          cuda::detail::TensorInfo<scalar_t, IndexType> out_tensor_info,
                          IndexType N,
                          int flip_dim,
                          IndexType total_dims) {
  for (IndexType linear_index = blockIdx.x * blockDim.x + threadIdx.x; linear_index < N; linear_index += gridDim.x * blockDim.x) {
    IndexType dst_offset = 0;
    if (flip_dim == 0) {
      // flip 1st dim
      dst_offset = (in_tensor_info.sizes[0] - 1 - linear_index / in_tensor_info.strides[0]) * in_tensor_info.strides[0] + linear_index % in_tensor_info.strides[0];
    }
    else {
      // flip last dim
      IndexType i = total_dims - 1;
      dst_offset = linear_index / in_tensor_info.strides[0] * in_tensor_info.strides[0] + (in_tensor_info.sizes[i] - 1 - linear_index % in_tensor_info.strides[0]);
    }
    out_tensor_info.data[dst_offset] = in_tensor_info.data[linear_index];
  }
}

template <typename scalar_t>
__global__
void flip_cuda_kernel(scalar_t* in_tensor, scalar_t* out_tensor, int64_t N, int64_t* flip_dims, int64_t flip_dims_size,
                      int64_t* strides, int64_t* strides_contiguous, int64_t* shape, int64_t total_dims) {

  int64_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (linear_index >= N) {
    return;
  }

  int64_t cur_indices = linear_index, rem = 0, dst_offset = 0;
  for (int64_t i = 0; i < total_dims; i++) {
    int64_t temp = cur_indices;
    cur_indices = cur_indices / strides_contiguous[i];
    rem = temp - cur_indices * strides_contiguous[i];
    // flip the indices if it is in flip_dims
    for (int64_t j = 0; j < flip_dims_size; j++) {
      if (i == flip_dims[j]) {
        cur_indices = shape[i] - 1 - cur_indices;
      }
    }
    dst_offset += cur_indices * strides[i];
    cur_indices = rem;
  }
  out_tensor[linear_index] = in_tensor[dst_offset];
}

// Flip tensor given a list of dims
Tensor flip_cuda(const Tensor& self, IntArrayRef dims) {
  auto in_tensor = self;
  const int64_t flip_dims_size = dims.size(), total_dims = in_tensor.dim(), N = in_tensor.numel();
  flip_check_errors(total_dims, flip_dims_size, dims);

  int64_t block_size = 512;
  dim3 dim_block(block_size);
  dim3 dim_grid((N + block_size - 1) / block_size);

  auto out_tensor = at::empty_like(in_tensor, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  if (out_tensor.numel() == 0) {
    return out_tensor;
  }

  auto flip_dims = dims.vec();
  wrap_all_dims(flip_dims, total_dims);

  // use kernel_pointwise_flip_apply2 only when to-flip dim is the 1st or last dim, where collapseDims can reduce the amount of work
  if (flip_dims_size == 1 && in_tensor.is_contiguous() && (flip_dims[0] == 0 || flip_dims[0] == total_dims - 1)) {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(at::ScalarType::Half, at::ScalarType::Bool, in_tensor.scalar_type(), "flip_cuda", [&] {
      auto in_tensor_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(in_tensor);
      auto out_tensor_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(out_tensor);
      int flip_dim = in_tensor_info.collapseDims(flip_dims[0]);
      out_tensor_info.collapseDims(flip_dims[0]);
      kernel_pointwise_flip_apply2<scalar_t, int64_t>
        <<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
          in_tensor_info, out_tensor_info, N, flip_dim, total_dims);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
    return out_tensor;
  }

  auto flip_dims_t = at::from_blob(
      flip_dims.data(), {static_cast<int64_t>(flip_dims.size())}, at::device(kCPU).dtype(kLong));

  auto shape = in_tensor.sizes().vec();
  auto shape_t = at::from_blob(
      shape.data(), {static_cast<int64_t>(shape.size())}, at::device(kCPU).dtype(kLong));

  auto strides = in_tensor.strides().vec();
  auto strides_t = at::from_blob(
      strides.data(), {static_cast<int64_t>(strides.size())}, at::device(kCPU).dtype(kLong));

  // stride_contiguous is the stride of non-contiguous tensor after calling contiguous(),
  // it is used to compute indices for each element in non-contiguous tensor
  Tensor stride_contiguous = at::zeros({total_dims}, kLong);
  int64_t* stride_contiguous_d = stride_contiguous.data_ptr<int64_t>();
  for (int64_t i = total_dims - 1; i >= 0; i--) {
    if (i == total_dims - 1) {
      stride_contiguous_d[i] = 1;
    } else {
      stride_contiguous_d[i] = std::max<int64_t>(shape[i+1], 1) * stride_contiguous_d[i + 1];
    }
  }

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND(at::ScalarType::Half, in_tensor.scalar_type(), "flip_cuda", [&] {
    flip_cuda_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
      in_tensor.data_ptr<scalar_t>(), out_tensor.data_ptr<scalar_t>(), N,
      flip_dims_t.cuda().data_ptr<int64_t>(),
      flip_dims_size,
      strides_t.cuda().data_ptr<int64_t>(),
      stride_contiguous.cuda().data_ptr<int64_t>(),
      shape_t.cuda().data_ptr<int64_t>(),
      total_dims);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });

  return out_tensor;
}

template <typename scalar_t>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__
void roll_cuda_kernel(scalar_t* in_tensor, scalar_t* out_tensor, int64_t N,
                      int64_t roll_dim, int64_t start,
                      int64_t size, int64_t stride, int64_t total_dims) {
  int64_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (linear_index >= N) {
    return;
  }
  // roll dim idx is the index of linear_index along the rolling dimension.
  int64_t roll_dim_idx = linear_index % (stride * size) / stride;
  // index into the source data to find appropriate value.
  int64_t source_idx = 0;
  if( roll_dim_idx >= (size - start) ) {
    source_idx = linear_index - ((size - start) * stride);
  } else {
    source_idx = linear_index + (start * stride);
  }
  out_tensor[linear_index] = in_tensor[source_idx];
}

// Roll a tensor along a dimension
Tensor roll_cuda(const Tensor& self, IntArrayRef shifts, IntArrayRef dims) {
  if (dims.size() != 1 || shifts.size() != 1) {
    return roll_common(self, shifts, dims);
  }

  auto in_tensor = self;
  if(!self.is_contiguous()) {
    in_tensor = self.contiguous();
  }
  auto out_tensor = at::empty_like(in_tensor, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  if (out_tensor.numel() == 0) {
    return out_tensor;
  }
  const int64_t N = in_tensor.numel();
  const int64_t dim = dims[0];
  const int64_t size = in_tensor.size(dim);
  int64_t start = (size - shifts[0]) % size;
  // Behavior of % is different in C++ vs Python for negative numbers. This
  // corrects the difference.
  if( start < 0 ) start = start + size;

  dim3 dim_block = cuda::getApplyBlock();
  dim3 dim_grid;
  TORCH_CHECK(cuda::getApplyGrid(N, dim_grid, in_tensor.get_device()), "unable to get dim grid");

  auto total_dims = in_tensor.dim();

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(at::ScalarType::Half, at::ScalarType::Bool, in_tensor.scalar_type(), "roll_cuda", [&] {
    roll_cuda_kernel<<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
      in_tensor.data_ptr<scalar_t>(), out_tensor.data_ptr<scalar_t>(), N,
      dim, start,
      size,
      in_tensor.stride(dim),
      total_dims);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });

  return out_tensor;
}

}} // namespace at::native
