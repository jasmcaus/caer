#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>

#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <ATen/native/cuda/vol2col.cuh>

namespace at {
namespace native {
namespace {

static inline void slow_conv_transpose3d_shape_check(
    const Tensor& input,
    const Tensor& grad_output,
    const Tensor& weight,
    const Tensor& bias,
    int kernel_depth,
    int kernel_width,
    int kernel_height,
    int stride_depth,
    int stride_width,
    int stride_height,
    int padding_depth,
    int padding_width,
    int padding_height,
    int dilation_depth,
    int dilation_width,
    int dilation_height,
    int output_padding_depth,
    int output_padding_width,
    int output_padding_height,
    int weight_nullable) {
  TORCH_CHECK(
      input.numel() != 0 && (input.dim() == 4 || input.dim() == 5),
      "non-empty 4D or 5D (batch mode) tensor expected for input, but got: ",
      input.sizes());
  TORCH_CHECK(
      stride_depth > 0 && stride_width > 0 && stride_height > 0,
      "stride should be greater than zero, but got stride_depth: ",
      stride_depth,
      " stride_height: ",
      stride_height,
      " stride_width: ",
      stride_width);
  TORCH_CHECK(
      dilation_depth > 0 && dilation_width > 0 && dilation_height > 0,
      "dilation should be greater than zero, but got dilation_depth: ",
      dilation_depth,
      ", dilation_height: ",
      dilation_height,
      ", dilation_width: ",
      dilation_width);
  TORCH_CHECK(
      (output_padding_depth < stride_depth ||
       output_padding_depth < dilation_depth) &&
          (output_padding_width < stride_width ||
           output_padding_width < dilation_width) &&
          (output_padding_height < stride_height ||
           output_padding_height < dilation_height),
      "output padding must be smaller than either stride or dilation,",
      " but got output_padding_depth: ",
      output_padding_depth,
      " output_padding_height: ",
      output_padding_height,
      " output_padding_width: ",
      output_padding_width,
      " stride_depth: ",
      stride_depth,
      " stride_height: ",
      stride_height,
      " stride_width: ",
      stride_width,
      " dilation_depth: ",
      dilation_depth,
      " dilation_height: ",
      dilation_height,
      " dilation_width: ",
      dilation_width);

  // number of input & output planes and kernel size is indirectly defined by
  // the weight tensor
  if (weight.defined()) {
    TORCH_CHECK(
        weight.numel() != 0 && weight.dim() == 5,
        "non-empty 5D (n_output_plane x n_input_plane ",
        "x kernel_depth x kernel_height x kernel_width) tensor ",
        "expected for weight, but got: ",
        weight.sizes());
    if (bias.defined()) {
      check_dim_size(bias, 1, 0, weight.size(1));
    }
  } else if (!weight_nullable) {
    AT_ERROR("weight tensor is expected to be non-nullable");
  }

  int ndim = input.dim();
  int dimf = 0;
  int dimd = 1;
  int dimh = 2;
  int dimw = 3;

  if (ndim == 5) {
    dimf++;
    dimd++;
    dimh++;
    dimw++;
  }

  if (weight.defined()) {
    const int64_t n_input_plane = weight.size(0);
    check_dim_size(input, ndim, dimf, n_input_plane);
  }

  int64_t input_width = input.size(dimw);
  int64_t input_height = input.size(dimh);
  int64_t input_depth = input.size(dimd);

  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  if (output_depth < 1 || output_width < 1 || output_height < 1) {
    AT_ERROR(
        "Given input size per channel: (",
        input_depth,
        " x ",
        input_height,
        " x ",
        input_width,
        "). Calculated output size per channel: (",
        output_depth,
        " x ",
        output_height,
        " x ",
        output_width,
        "). Output size is too small");
  }

  if (grad_output.defined()) {
    if (weight.defined()) {
      const int64_t n_output_plane = weight.size(1);
      check_dim_size(grad_output, ndim, dimf, n_output_plane);
    } else if (bias.defined()) {
      const int64_t n_output_plane = bias.size(0);
      check_dim_size(grad_output, ndim, dimf, n_output_plane);
    }
    check_dim_size(grad_output, ndim, dimd, output_depth);
    check_dim_size(grad_output, ndim, dimh, output_height);
    check_dim_size(grad_output, ndim, dimw, output_width);
  }
}

void slow_conv_transpose3d_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    const Tensor& weight_,
    IntArrayRef kernel_size,
    const Tensor& bias,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    Tensor& finput,
    Tensor& fgrad_input) {
  TORCH_CHECK(
      kernel_size.size() == 3,
      "It is expected kernel_size equals to 3, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 3,
      "It is expected dilation equals to 3, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 3,
      "It is expected padding equals to 3, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 3,
      "It is expected stride equals to 3, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 3,
      "It is expected stride equals to 3, but got size ",
      output_padding.size());

  int64_t kernel_depth = kernel_size[0];
  int64_t kernel_height = kernel_size[1];
  int64_t kernel_width = kernel_size[2];
  int64_t dilation_depth = dilation[0];
  int64_t dilation_height = dilation[1];
  int64_t dilation_width = dilation[2];
  int64_t padding_depth = padding[0];
  int64_t padding_height = padding[1];
  int64_t padding_width = padding[2];
  int64_t stride_depth = stride[0];
  int64_t stride_height = stride[1];
  int64_t stride_width = stride[2];
  int64_t output_padding_depth = output_padding[0];
  int64_t output_padding_height = output_padding[1];
  int64_t output_padding_width = output_padding[2];

  Tensor columns = finput;
  Tensor ones = fgrad_input;

  int n_input_plane = weight_.size(0);
  int n_output_plane = weight_.size(1);

  TensorArg input_arg{input_, "input", 1}, output_arg{output, "output", 2},
      weight_arg{weight_, "weight", 3}, bias_arg{bias, "bias", 4},
      columns_arg{columns, "columns", 5}, ones_arg{ones, "ones", 6};

  checkAllSameGPU(
      "slow_conv_transpose3d_out_cuda",
      {input_arg, output_arg, weight_arg, bias_arg, columns_arg, ones_arg});

  slow_conv_transpose3d_shape_check(
      input_,
      Tensor(),
      weight_,
      bias,
      kernel_depth,
      kernel_width,
      kernel_height,
      stride_depth,
      stride_width,
      stride_height,
      padding_depth,
      padding_width,
      padding_height,
      dilation_depth,
      dilation_width,
      dilation_height,
      output_padding_depth,
      output_padding_width,
      output_padding_height,
      0);

  TORCH_CHECK(
      !bias.defined() || bias.is_contiguous(),
      "bias tensor has to be contiguous");

  Tensor input = input_.contiguous();
  Tensor weight = weight_.contiguous();

  int is_batch = false;
  if (input.dim() == 4) {
    // Force batch
    is_batch = true;
    input.resize_(
        {1, input.size(0), input.size(1), input.size(2), input.size(3)});
  }

  int64_t input_width = input.size(4);
  int64_t input_height = input.size(3);
  int64_t input_depth = input.size(2);

  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Resize output
  output.resize_(
      {batch_size, n_output_plane, output_depth, output_height, output_width});

  // Resize temporary columns
  columns.resize_({n_output_plane * kernel_width * kernel_height * kernel_depth,
                   input_depth * input_height * input_width});

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets
  // increased, and always contains ones.
  if (ones.dim() != 3 ||
      ones.size(0) * ones.size(1) * ones.size(2) <
          output_depth * output_height * output_width) {
    // Resize plane and fill with ones...
    ones.resize_({output_depth, output_height, output_width});
    ones.fill_(1);
  }

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "slow_conv_transpose3d_out_cuda", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        // Helpers
        Tensor input_n;
        Tensor output_n;

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix mulitply per output:
          input_n = input.select(0, elt);
          output_n = output.select(0, elt);

          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m =
              weight.size(1) * weight.size(2) * weight.size(3) * weight.size(4);
          int64_t n = columns.size(1);
          int64_t k = weight.size(0);

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          at::cuda::blas::gemm<scalar_t>(
              'n',
              't',
              n,
              m,
              k,
              static_cast<scalar_t>(1),
              input_n.data_ptr<scalar_t>(),
              n,
              weight.data_ptr<scalar_t>(),
              m,
              static_cast<scalar_t>(0),
              columns.data_ptr<scalar_t>(),
              n);

          // Unpack columns back into input:
          at::native::col2vol<scalar_t, accscalar_t>(
              at::cuda::getCurrentCUDAStream(),
              columns.data_ptr<scalar_t>(),
              n_output_plane,
              output_depth,
              output_height,
              output_width,
              input_depth,
              input_height,
              input_width,
              kernel_depth,
              kernel_height,
              kernel_width,
              padding_depth,
              padding_height,
              padding_width,
              stride_depth,
              stride_height,
              stride_width,
              dilation_depth,
              dilation_height,
              dilation_width,
              output_n.data_ptr<scalar_t>());

          // Do Bias after:
          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m_ = n_output_plane;
          int64_t n_ = output_depth * output_height * output_width;
          int64_t k_ = 1;

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          if (bias.defined()) {
            at::cuda::blas::gemm<scalar_t>(
                't',
                'n',
                n_,
                m_,
                k_,
                static_cast<scalar_t>(1),
                ones.data_ptr<scalar_t>(),
                k_,
                bias.data_ptr<scalar_t>(),
                k_,
                static_cast<scalar_t>(1),
                output_n.data_ptr<scalar_t>(),
                n_);
          }
        }

        // Resize output
        if (is_batch) {
          output.resize_(
              {n_output_plane, output_depth, output_height, output_width});
          input.resize_(
              {n_input_plane, input_depth, input_height, input_width});
        }
      });
}

void slow_conv_transpose3d_backward_out_cuda_template(
    const Tensor& input_,
    const Tensor& grad_output_,
    Tensor& grad_input,
    const Tensor& weight_,
    const Tensor& finput,
    const Tensor& fgrad_input,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  TORCH_CHECK(
      kernel_size.size() == 3,
      "It is expected kernel_size equals to 3, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 3,
      "It is expected dilation equals to 3, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 3,
      "It is expected padding equals to 3, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 3,
      "It is expected stride equals to 3, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 3,
      "It is expected stride equals to 3, but got size ",
      output_padding.size());

  Tensor grad_columns = finput;

  int n_input_plane = weight_.size(0);
  int n_output_plane = weight_.size(1);

  int64_t kernel_depth = kernel_size[0];
  int64_t kernel_height = kernel_size[1];
  int64_t kernel_width = kernel_size[2];
  int64_t dilation_depth = dilation[0];
  int64_t dilation_height = dilation[1];
  int64_t dilation_width = dilation[2];
  int64_t padding_depth = padding[0];
  int64_t padding_height = padding[1];
  int64_t padding_width = padding[2];
  int64_t stride_depth = stride[0];
  int64_t stride_height = stride[1];
  int64_t stride_width = stride[2];
  int64_t output_padding_depth = output_padding[0];
  int64_t output_padding_height = output_padding[1];
  int64_t output_padding_width = output_padding[2];

  TensorArg input_arg{input_, "input", 1},
      grad_output_arg{grad_output_, "grad_output", 2},
      weight_arg{weight_, "weight", 3},
      grad_columns_arg{grad_columns, "grad_columns", 4},
      grad_input_arg{grad_input, "grad_input", 5};

  checkAllSameGPU(
      "slow_conv_transpose3d_backward_out_cuda",
      {input_arg,
       grad_output_arg,
       weight_arg,
       grad_columns_arg,
       grad_input_arg});

  slow_conv_transpose3d_shape_check(
      input_,
      grad_output_,
      weight_,
      Tensor(),
      kernel_depth,
      kernel_width,
      kernel_height,
      stride_depth,
      stride_width,
      stride_height,
      padding_depth,
      padding_width,
      padding_height,
      dilation_depth,
      dilation_width,
      dilation_height,
      output_padding_depth,
      output_padding_width,
      output_padding_height,
      0);

  Tensor input = input_.contiguous();
  Tensor grad_output = grad_output_.contiguous();
  Tensor weight = weight_.contiguous();

  bool is_batch = false;
  if (input.dim() == 4) {
    // Force batch
    is_batch = true;
    input.resize_(
        {1, input.size(0), input.size(1), input.size(2), input.size(3)});
    grad_output.resize_({1,
                         grad_output.size(0),
                         grad_output.size(1),
                         grad_output.size(2),
                         grad_output.size(3)});
  }

  int64_t input_width = input.size(4);
  int64_t input_height = input.size(3);
  int64_t input_depth = input.size(2);
  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Resize output
  grad_input.resize_(
      {batch_size, n_input_plane, input_depth, input_height, input_width});

  // Resize temporary columns
  grad_columns.resize_(
      {n_output_plane * kernel_width * kernel_height * kernel_depth,
       input_depth * input_height * input_width});

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(), "slow_conv_transpose3d_backward_out_cuda", [&] {
        // Helpers
        Tensor grad_input_n;
        Tensor grad_output_n;

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix mulitply per sample:
          grad_input_n = grad_input.select(0, elt);
          grad_output_n = grad_output.select(0, elt);

          if (kernel_depth != 1 || kernel_height != 1 || kernel_width != 1) {
            // Extract columns:
            at::native::vol2col<scalar_t>(
                at::cuda::getCurrentCUDAStream(),
                grad_output_n.data_ptr<scalar_t>(),
                n_output_plane,
                output_depth,
                output_height,
                output_width,
                input_depth,
                input_height,
                input_width,
                kernel_depth,
                kernel_height,
                kernel_width,
                padding_depth,
                padding_height,
                padding_width,
                stride_depth,
                stride_height,
                stride_width,
                dilation_depth,
                dilation_height,
                dilation_width,
                grad_columns.data_ptr<scalar_t>());
          }

          // M,N,K are dims of matrix A and B
          // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
          int64_t m = weight.size(0);
          int64_t n = grad_columns.size(1);
          int64_t k =
              weight.size(1) * weight.size(2) * weight.size(3) * weight.size(4);

          // Do GEMM (note: this is a bit confusing because gemm assumes
          // column-major matrices)
          auto gemm_in_ptr = (kernel_depth != 1 || kernel_height != 1 || kernel_width != 1) ?
              grad_columns.data_ptr<scalar_t>() : grad_output_n.data_ptr<scalar_t>();
          at::cuda::blas::gemm<scalar_t>(
              'n',
              'n',
              n,
              m,
              k,
              static_cast<scalar_t>(1),
              gemm_in_ptr,
              n,
              weight.data_ptr<scalar_t>(),
              k,
              static_cast<scalar_t>(0),
              grad_input_n.data_ptr<scalar_t>(),
              n);
        }

        // Resize output
        if (is_batch) {
          grad_output.resize_(
              {n_output_plane, output_depth, output_height, output_width});
          input.resize_(
              {n_input_plane, input_depth, input_height, input_width});
          grad_input.resize_(
              {n_input_plane, input_depth, input_height, input_width});
        }
      });
}

void slow_conv_transpose3d_acc_grad_parameters_cuda(
    const Tensor& input_,
    const Tensor& grad_output_,
    Tensor& grad_weight,
    Tensor& grad_bias,
    const Tensor& finput,
    const Tensor& fgrad_input,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    int scale_) {
  TORCH_CHECK(
      kernel_size.size() == 3,
      "It is expected kernel_size equals to 3, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 3,
      "It is expected dilation equals to 3, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 3,
      "It is expected padding equals to 3, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 3,
      "It is expected stride equals to 3, but got size ",
      stride.size());

  TORCH_CHECK(
      output_padding.size() == 3,
      "It is expected stride equals to 3, but got size ",
      output_padding.size());

  int64_t kernel_depth = kernel_size[0];
  int64_t kernel_height = kernel_size[1];
  int64_t kernel_width = kernel_size[2];
  int64_t dilation_depth = dilation[0];
  int64_t dilation_height = dilation[1];
  int64_t dilation_width = dilation[2];
  int64_t padding_depth = padding[0];
  int64_t padding_height = padding[1];
  int64_t padding_width = padding[2];
  int64_t stride_depth = stride[0];
  int64_t stride_height = stride[1];
  int64_t stride_width = stride[2];
  int64_t output_padding_depth = output_padding[0];
  int64_t output_padding_height = output_padding[1];
  int64_t output_padding_width = output_padding[2];

  Tensor columns = finput;
  Tensor ones = fgrad_input;

  TensorArg input_arg{input_, "input", 1},
      grad_output_arg{grad_output_, "grad_output", 2},
      grad_weight_arg{grad_weight, "grad_weight", 3},
      grad_bias_arg{grad_bias, "grad_bias", 4},
      columns_arg{columns, "columns", 5}, ones_arg{ones, "ones", 6};

  checkAllSameGPU(
      "slow_conv_transpose3d_acc_grad_parameters_cuda",
      {input_arg,
       grad_output_arg,
       grad_weight_arg,
       grad_bias_arg,
       columns_arg,
       ones_arg});

  slow_conv_transpose3d_shape_check(
      input_,
      grad_output_,
      grad_weight,
      grad_bias,
      kernel_depth,
      kernel_width,
      kernel_height,
      stride_depth,
      stride_width,
      stride_height,
      padding_depth,
      padding_width,
      padding_height,
      dilation_depth,
      dilation_width,
      dilation_height,
      output_padding_depth,
      output_padding_width,
      output_padding_height,
      1);

  int n_output_plane;
  if (grad_weight.defined()) {
    n_output_plane = grad_weight.size(1);
  } else if (grad_bias.defined()) {
    n_output_plane = grad_bias.size(0);
  } else {
    return;
  }

  if (grad_weight.defined()) {
    TORCH_CHECK(
        grad_weight.is_contiguous(), "grad_weight needs to be contiguous");
  }
  if (grad_bias.defined()) {
    TORCH_CHECK(grad_bias.is_contiguous(), "grad_bias needs to be contiguous");
    TORCH_CHECK(ones.is_contiguous(), "ones needs to be contiguous");
  }

  Tensor input = input_.contiguous();
  Tensor grad_output = grad_output_.contiguous();

  bool is_batch = false;
  if (input.dim() == 4) {
    // Force batch
    is_batch = true;
    input.resize_(
        {1, input.size(0), input.size(1), input.size(2), input.size(3)});
    grad_output.resize_({1,
                         grad_output.size(0),
                         grad_output.size(1),
                         grad_output.size(2),
                         grad_output.size(3)});
  }

  int64_t input_width = input.size(4);
  int64_t input_height = input.size(3);
  int64_t input_depth = input.size(2);

  int64_t output_depth = (input_depth - 1) * stride_depth - 2 * padding_depth +
      (dilation_depth * (kernel_depth - 1) + 1) + output_padding_depth;
  int64_t output_height = (input_height - 1) * stride_height -
      2 * padding_height + (dilation_height * (kernel_height - 1) + 1) +
      output_padding_height;
  int64_t output_width = (input_width - 1) * stride_width - 2 * padding_width +
      (dilation_width * (kernel_width - 1) + 1) + output_padding_width;

  // Batch size + input planes
  int64_t batch_size = input.size(0);

  // Define a buffer of ones, for bias accumulation
  if (ones.dim() != 3 ||
      ones.size(0) * ones.size(1) * ones.size(2) <
          output_depth * output_height * output_width) {
    // Resize plane and fill with ones...
    ones.resize_({output_depth, output_height, output_width});
    ones.fill_(1);
  }

  // Resize temporary columns
  columns.resize_({n_output_plane * kernel_width * kernel_height * kernel_depth,
                   input_depth * input_height * input_width});

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16,
      input.scalar_type(),
      "slow_conv_transpose3d_acc_grad_parameters_cuda",
      [&] {
        // Helpers
        Tensor input_n;
        Tensor grad_output_n;

        scalar_t scale = static_cast<scalar_t>(scale_);

        // For each elt in batch, do:
        for (int elt = 0; elt < batch_size; elt++) {
          // Matrix mulitply per output:
          grad_output_n = grad_output.select(0, elt);

          // Do Weight:
          if (grad_weight.defined()) {
            // Matrix mulitply per output:
            input_n = input.select(0, elt);

            if (kernel_depth != 1 || kernel_height != 1 || kernel_width != 1) {
              // Extract columns:
              at::native::vol2col<scalar_t>(
                  at::cuda::getCurrentCUDAStream(),
                  grad_output_n.data_ptr<scalar_t>(),
                  n_output_plane,
                  output_depth,
                  output_height,
                  output_width,
                  input_depth,
                  input_height,
                  input_width,
                  kernel_depth,
                  kernel_height,
                  kernel_width,
                  padding_depth,
                  padding_height,
                  padding_width,
                  stride_depth,
                  stride_height,
                  stride_width,
                  dilation_depth,
                  dilation_height,
                  dilation_width,
                  columns.data_ptr<scalar_t>());
            }

            // M,N,K are dims of matrix A and B
            // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
            int64_t n = columns.size(0); // n_output_plane * kt * kh * kw
            int64_t m = input_n.size(0); // n_input_plane
            int64_t k = columns.size(1); // input_height * input_width

            // Do GEMM (note: this is a bit confusing because gemm assumes
            // column-major matrices)
            auto gemm_in_ptr = (kernel_depth != 1 || kernel_height != 1 || kernel_width != 1) ?
                columns.data_ptr<scalar_t>() : grad_output_n.data_ptr<scalar_t>();
            at::cuda::blas::gemm<scalar_t>(
                't',
                'n',
                n,
                m,
                k,
                scale,
                gemm_in_ptr,
                k,
                input_n.data_ptr<scalar_t>(),
                k,
                static_cast<scalar_t>(1),
                grad_weight.data_ptr<scalar_t>(),
                n);
          }

          // Do Bias:
          if (grad_bias.defined()) {
            // M,N,K are dims of matrix A and B
            // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
            int64_t m_ = n_output_plane;
            int64_t k_ = output_depth * output_height * output_width;

            // Do GEMV (note: this is a bit confusing because gemv assumes
            // column-major matrices)
            at::cuda::blas::gemv<scalar_t>(
                't',
                k_,
                m_,
                scale,
                grad_output_n.data_ptr<scalar_t>(),
                k_,
                ones.data_ptr<scalar_t>(),
                1,
                static_cast<scalar_t>(1),
                grad_bias.data_ptr<scalar_t>(),
                1);
          }
        }

        // Resize
        if (is_batch) {
          grad_output.resize_(
              {n_output_plane, output_depth, output_height, output_width});
          input.resize_(
              {input.size(1), input_depth, input_height, input_width});
        }
      });
}

} // namespace

Tensor& slow_conv_transpose3d_out_cuda(
    Tensor& output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    const Tensor& bias,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  Tensor finput = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor fgrad = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  slow_conv_transpose3d_out_cuda_template(
      output,
      input,
      weight,
      kernel_size,
      bias,
      stride,
      padding,
      output_padding,
      dilation,
      finput,
      fgrad);

  return output;
}

Tensor slow_conv_transpose3d_cuda(
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    const Tensor& bias,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor finput = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  Tensor fgrad = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  slow_conv_transpose3d_out_cuda_template(
      output,
      input,
      weight,
      kernel_size,
      bias,
      stride,
      padding,
      output_padding,
      dilation,
      finput,
      fgrad);

  return output;
}

std::tuple<Tensor&, Tensor&, Tensor&> slow_conv_transpose3d_backward_out_cuda(
    Tensor& grad_input,
    Tensor& grad_weight,
    Tensor& grad_bias,
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    const Tensor& finput,
    const Tensor& fgrad) {
  if (grad_input.defined()) {
    slow_conv_transpose3d_backward_out_cuda_template(
        input,
        grad_output,
        grad_input,
        weight,
        finput,
        fgrad,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation);
  }

  if (grad_weight.defined()) {
    grad_weight.resize_(weight.sizes());
    grad_weight.zero_();
  }

  if (grad_bias.defined()) {
    grad_bias.resize_({weight.size(1)});
    grad_bias.zero_();
  }

  if (grad_weight.defined() || grad_bias.defined()) {
    slow_conv_transpose3d_acc_grad_parameters_cuda(
        input,
        grad_output,
        grad_weight,
        grad_bias,
        finput,
        fgrad,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation,
        1);
  }

  return std::tuple<Tensor&, Tensor&, Tensor&>(
      grad_input, grad_weight, grad_bias);
}

std::tuple<Tensor, Tensor, Tensor> slow_conv_transpose3d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& input,
    const Tensor& weight,
    IntArrayRef kernel_size,
    IntArrayRef stride,
    IntArrayRef padding,
    IntArrayRef output_padding,
    IntArrayRef dilation,
    const Tensor& finput,
    const Tensor& fgrad,
    std::array<bool, 3> output_mask) {
  Tensor grad_input;
  Tensor grad_weight;
  Tensor grad_bias;

  if (output_mask[0]) {
    grad_input = at::empty({0}, grad_output.options());
  } else {
    grad_input = Tensor();
  }

  if (output_mask[1]) {
    grad_weight = at::empty({0}, grad_output.options());
  } else {
    grad_weight = Tensor();
  }

  if (output_mask[2]) {
    grad_bias = at::empty({0}, grad_output.options());
  } else {
    grad_bias = Tensor();
  }

  if (grad_input.defined()) {
    slow_conv_transpose3d_backward_out_cuda_template(
        input,
        grad_output,
        grad_input,
        weight,
        finput,
        fgrad,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation);
  }

  if (grad_weight.defined()) {
    grad_weight.resize_(weight.sizes());
    grad_weight.zero_();
  }

  if (grad_bias.defined()) {
    grad_bias.resize_({weight.size(1)});
    grad_bias.zero_();
  }

  if (grad_weight.defined() || grad_bias.defined()) {
    slow_conv_transpose3d_acc_grad_parameters_cuda(
        input,
        grad_output,
        grad_weight,
        grad_bias,
        finput,
        fgrad,
        kernel_size,
        stride,
        padding,
        output_padding,
        dilation,
        1);
  }

  return std::tuple<Tensor, Tensor, Tensor>(grad_input, grad_weight, grad_bias);
}

} // namespace native
} // namespace at
