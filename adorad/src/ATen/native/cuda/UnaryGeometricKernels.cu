#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void acos_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "acos_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::acos(a);
    });
  });
}

void asin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "asin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::asin(a);
    });
  });
}

void atan_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "atan_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::atan(a);
    });
  });
}

void sin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "sin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sin(a);
    });
  });
}

void cos_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "cos_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::cos(a);
    });
  });
}

void sinh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "sinh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sinh(a);
    });
  });
}

void cosh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "cosh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::cosh(a);
    });
  });
}

void tanh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "tanh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::tanh(a);
    });
  });
}

void acosh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "acosh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::acosh(a);
    });
  });
}

void asinh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "asinh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::asinh(a);
    });
  });
}

void atanh_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "atanh_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
            return ::atanh(a);
    });
  });
}

void tan_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "tan_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::tan(a);
    });
  });
}

REGISTER_DISPATCH(acos_stub, &acos_kernel_cuda);
REGISTER_DISPATCH(acosh_stub, &acosh_kernel_cuda);
REGISTER_DISPATCH(asinh_stub, &asinh_kernel_cuda);
REGISTER_DISPATCH(atanh_stub, &atanh_kernel_cuda);
REGISTER_DISPATCH(asin_stub, &asin_kernel_cuda);
REGISTER_DISPATCH(atan_stub, &atan_kernel_cuda);
REGISTER_DISPATCH(sin_stub, &sin_kernel_cuda);
REGISTER_DISPATCH(cos_stub, &cos_kernel_cuda);
REGISTER_DISPATCH(sinh_stub, &sinh_kernel_cuda);
REGISTER_DISPATCH(cosh_stub, &cosh_kernel_cuda);
REGISTER_DISPATCH(tanh_stub, &tanh_kernel_cuda);
REGISTER_DISPATCH(tan_stub, &tan_kernel_cuda);

}} // namespace at::native
