#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/div_rtn.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <ATen/native/cuda/im2col.cuh>
#include <ATen/native/im2col_shape_check.h>

namespace at {
namespace native {
namespace {

void col2im_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    IntArrayRef output_size,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  TensorArg input_arg{input_, "input", 1};
  TensorArg output_arg{output, "output", 2};
  checkAllSameGPU("col2im_out_cuda", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  TORCH_CHECK(
      kernel_size.size() == 2,
      "It is expected kernel_size equals to 2, but got size ",
      kernel_size.size());

  TORCH_CHECK(
      dilation.size() == 2,
      "It is expected dilation equals to 2, but got size ",
      dilation.size());

  TORCH_CHECK(
      padding.size() == 2,
      "It is expected padding equals to 2, but got size ",
      padding.size());

  TORCH_CHECK(
      stride.size() == 2,
      "It is expected stride equals to 2, but got size ",
      stride.size());

  int64_t output_height = output_size[0];
  int64_t output_width = output_size[1];
  int64_t kernel_height = kernel_size[0];
  int64_t kernel_width = kernel_size[1];
  int64_t dilation_height = dilation[0];
  int64_t dilation_width = dilation[1];
  int64_t pad_height = padding[0];
  int64_t pad_width = padding[1];
  int64_t stride_height = stride[0];
  int64_t stride_width = stride[1];

  col2im_shape_check(
      input_,
      Tensor(),
      output_height,
      output_width,
      kernel_height,
      kernel_width,
      dilation_height,
      dilation_width,
      pad_height,
      pad_width,
      stride_height,
      stride_width);

  Tensor input = input_.contiguous();

  bool batched_input = true;
  if (input.dim() == 2) {
    // Force batch
    batched_input = false;
    input.resize_({1, input.size(0), input.size(1)});
  }

  int64_t batch_size = input.size(0);
  int64_t n_input_plane = input.size(1);
  int64_t n_output_plane = n_input_plane / (kernel_width * kernel_height);

  output.resize_({batch_size, n_output_plane, output_height, output_width});
  output.zero_();

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(kHalf,
      input.scalar_type(), "col2im_out_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;

    Tensor input_n;
    Tensor output_n;

    int64_t height_col = (output_height + 2 * pad_height -
                          (dilation_height * (kernel_height - 1) + 1)) /
            stride_height +
        1;
    int64_t width_col = (output_width + 2 * pad_width -
                         (dilation_width * (kernel_width - 1) + 1)) /
            stride_width +
        1;

    for (int64_t elt = 0; elt < batch_size; elt++) {
      input_n = input.select(0, elt);
      output_n = output.select(0, elt);

      col2im<scalar_t, accscalar_t>(
          at::cuda::getCurrentCUDAStream(),
          input_n.data_ptr<scalar_t>(),
          n_output_plane,
          output_height,
          output_width,
          height_col,
          width_col,
          kernel_height,
          kernel_width,
          pad_height,
          pad_width,
          stride_height,
          stride_width,
          dilation_height,
          dilation_width,
          output_n.data_ptr<scalar_t>());
    }

    if (!batched_input) {
      output.resize_({n_output_plane, output_height, output_width});
    }
  });
}

void col2im_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  // im2col_out_cuda checks size of kernel_size, dilation, padding and stride
  im2col_out_cuda(
      grad_input, grad_output, kernel_size, dilation, padding, stride);
}

} // namespace

Tensor& col2im_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  col2im_out_cuda_template(
      output, input, output_size, kernel_size, dilation, padding, stride);
  return output;
}

Tensor col2im_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  col2im_out_cuda_template(
      output, input, output_size, kernel_size, dilation, padding, stride);
  return output;
}

Tensor& col2im_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  col2im_backward_out_cuda_template(
      grad_input, grad_output, kernel_size, dilation, padding, stride);
  return grad_input;
}

Tensor col2im_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef kernel_size,
    IntArrayRef dilation,
    IntArrayRef padding,
    IntArrayRef stride) {
  Tensor grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  col2im_backward_out_cuda_template(
      grad_input, grad_output, kernel_size, dilation, padding, stride);
  return grad_input;
}

} // namespace native
} // namespace at
