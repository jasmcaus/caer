#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/native/cuda/GridSampler.cuh>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/macros/Macros.h>

namespace at { namespace native {

using namespace at::cuda::detail;

using at::native::detail::GridSamplerInterpolation;
using at::native::detail::GridSamplerPadding;

namespace {
  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_2d_kernel(
      const index_t nthreads,
      TensorInfo<scalar_t, index_t> input,
      TensorInfo<scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> output,
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {
    index_t C = input.sizes[1];
    index_t inp_H = input.sizes[2];
    index_t inp_W = input.sizes[3];
    index_t out_H = grid.sizes[1];
    index_t out_W = grid.sizes[2];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sH = input.strides[2];
    index_t inp_sW = input.strides[3];
    index_t grid_sN = grid.strides[0];
    index_t grid_sH = grid.strides[1];
    index_t grid_sW = grid.strides[2];
    index_t grid_sCoor = grid.strides[3];
    index_t out_sN = output.strides[0];
    index_t out_sC = output.strides[1];
    index_t out_sH = output.strides[2];
    index_t out_sW = output.strides[3];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t n = index / (out_H * out_W);
      const index_t grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y co-ordinates from grid
      scalar_t x = grid.data[grid_offset];
      scalar_t y = grid.data[grid_offset + grid_sCoor];

      scalar_t ix = grid_sampler_compute_source_index(x, inp_W, padding_mode, align_corners);
      scalar_t iy = grid_sampler_compute_source_index(y, inp_H, padding_mode, align_corners);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get NE, NW, SE, SW pixel values from (x, y)
        index_t ix_nw = static_cast<index_t>(::floor(ix));
        index_t iy_nw = static_cast<index_t>(::floor(iy));
        index_t ix_ne = ix_nw + 1;
        index_t iy_ne = iy_nw;
        index_t ix_sw = ix_nw;
        index_t iy_sw = iy_nw + 1;
        index_t ix_se = ix_nw + 1;
        index_t iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        scalar_t nw = (ix_se - ix)    * (iy_se - iy);
        scalar_t ne = (ix    - ix_sw) * (iy_sw - iy);
        scalar_t sw = (ix_ne - ix)    * (iy    - iy_ne);
        scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);

        // calculate bilinear weighted pixel value and set output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          *out_ptr_NCHW = static_cast<scalar_t>(0);
          if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
          }
          if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
          }
          if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
          }
          if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
            *out_ptr_NCHW += inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW] * se;
          }
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        index_t ix_nearest = static_cast<index_t>(::round(ix));
        index_t iy_nearest = static_cast<index_t>(::round(iy));

        // assign nearest neighor pixel value to output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          if (within_bounds_2d(iy_nearest, ix_nearest, inp_H, inp_W)) {
            *out_ptr_NCHW = inp_ptr_NC[iy_nearest * inp_sH + ix_nearest * inp_sW];
          } else {
            *out_ptr_NCHW = static_cast<scalar_t>(0);
          }
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Bicubic) {

        ix = grid_sampler_unnormalize(x, inp_W, align_corners);
        iy = grid_sampler_unnormalize(y, inp_H, align_corners);

        scalar_t ix_nw = ::floor(ix);
        scalar_t iy_nw = ::floor(iy);

        const scalar_t tx = ix - ix_nw;
        const scalar_t ty = iy - iy_nw;

        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCHW = output.data + n * out_sN + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
          scalar_t coefficients[4];

          for (index_t i = 0; i < 4; ++i) {
            coefficients[i] = cubic_interp1d(
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw - 1, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw + 0, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw + 1, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw + 2, iy_nw - 1 + i, inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners),
              tx);
          }

          *out_ptr_NCHW = cubic_interp1d(
            coefficients[0],
            coefficients[1],
            coefficients[2],
            coefficients[3],
            ty);
        }
      }
    }
  }

  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_3d_kernel(
      const index_t nthreads,
      TensorInfo<scalar_t, index_t> input,
      TensorInfo<scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> output,
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    index_t C = input.sizes[1];
    index_t inp_D = input.sizes[2];
    index_t inp_H = input.sizes[3];
    index_t inp_W = input.sizes[4];
    index_t out_D = grid.sizes[1];
    index_t out_H = grid.sizes[2];
    index_t out_W = grid.sizes[3];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sD = input.strides[2];
    index_t inp_sH = input.strides[3];
    index_t inp_sW = input.strides[4];
    index_t grid_sN = grid.strides[0];
    index_t grid_sD = grid.strides[1];
    index_t grid_sH = grid.strides[2];
    index_t grid_sW = grid.strides[3];
    index_t grid_sCoor = grid.strides[4];
    index_t out_sN = output.strides[0];
    index_t out_sC = output.strides[1];
    index_t out_sD = output.strides[2];
    index_t out_sH = output.strides[3];
    index_t out_sW = output.strides[4];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t d = (index / (out_H * out_W)) % out_D;
      const index_t n = index / (out_D * out_H * out_W);
      const index_t grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y, z co-ordinates from grid
      scalar_t ix = grid.data[grid_offset];
      scalar_t iy = grid.data[grid_offset + grid_sCoor];
      scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

      ix = grid_sampler_compute_source_index(ix, inp_W, padding_mode, align_corners);
      iy = grid_sampler_compute_source_index(iy, inp_H, padding_mode, align_corners);
      iz = grid_sampler_compute_source_index(iz, inp_D, padding_mode, align_corners);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get corner pixel values from (x, y, z)
        // for 4d, we used north-east-south-west
        // for 5d, we add top-bottom
        index_t ix_tnw = static_cast<index_t>(::floor(ix));
        index_t iy_tnw = static_cast<index_t>(::floor(iy));
        index_t iz_tnw = static_cast<index_t>(::floor(iz));

        index_t ix_tne = ix_tnw + 1;
        index_t iy_tne = iy_tnw;
        index_t iz_tne = iz_tnw;

        index_t ix_tsw = ix_tnw;
        index_t iy_tsw = iy_tnw + 1;
        index_t iz_tsw = iz_tnw;

        index_t ix_tse = ix_tnw + 1;
        index_t iy_tse = iy_tnw + 1;
        index_t iz_tse = iz_tnw;

        index_t ix_bnw = ix_tnw;
        index_t iy_bnw = iy_tnw;
        index_t iz_bnw = iz_tnw + 1;

        index_t ix_bne = ix_tnw + 1;
        index_t iy_bne = iy_tnw;
        index_t iz_bne = iz_tnw + 1;

        index_t ix_bsw = ix_tnw;
        index_t iy_bsw = iy_tnw + 1;
        index_t iz_bsw = iz_tnw + 1;

        index_t ix_bse = ix_tnw + 1;
        index_t iy_bse = iy_tnw + 1;
        index_t iz_bse = iz_tnw + 1;

        // get surfaces to each neighbor:
        scalar_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
        scalar_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
        scalar_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
        scalar_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
        scalar_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
        scalar_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
        scalar_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
        scalar_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCDHW = output.data + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
          //   (c, iz_tnw, iy_tnw, ix_tnw) * tnw + (c, iz_tne, iy_tne, ix_tne) * tne
          // + (c, iz_tsw, iy_tsw, ix_tsw) * tsw + (c, iz_tse, iy_tse, ix_tse) * tse
          // + (c, iz_bnw, iy_bnw, ix_bnw) * bnw + (c, iz_bne, iy_bne, ix_bne) * bne
          // + (c, iz_bsw, iy_bsw, ix_bsw) * bsw + (c, iz_bse, iy_bse, ix_bse) * bse
          *out_ptr_NCDHW = static_cast<scalar_t>(0);
          if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW] * tnw;
          }
          if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW] * tne;
          }
          if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW] * tsw;
          }
          if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW] * tse;
          }
          if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW] * bnw;
          }
          if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW] * bne;
          }
          if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW] * bsw;
          }
          if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW += inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW] * bse;
          }
        }
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        index_t ix_nearest = static_cast<index_t>(::round(ix));
        index_t iy_nearest = static_cast<index_t>(::round(iy));
        index_t iz_nearest = static_cast<index_t>(::round(iz));

        // assign nearest neighor pixel value to output pixel
        auto inp_ptr_NC = input.data + n * inp_sN;
        auto out_ptr_NCDHW = output.data + n * out_sN + d * out_sD + h * out_sH + w * out_sW;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
          if (within_bounds_3d(iz_nearest, iy_nearest, ix_nearest, inp_D, inp_H, inp_W)) {
            *out_ptr_NCDHW = inp_ptr_NC[iz_nearest * inp_sD + iy_nearest * inp_sH + ix_nearest * inp_sW];
          } else {
            *out_ptr_NCDHW = static_cast<scalar_t>(0);
          }
        }
      }
    }
  }

  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_2d_backward_kernel(
      const index_t nthreads,
      TensorInfo<scalar_t, index_t> grad_output,
      TensorInfo<scalar_t, index_t> input,
      TensorInfo<scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> grad_input,  // initialized to zeros
      TensorInfo<scalar_t, index_t> grad_grid,   // initialized to empty
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    index_t C = input.sizes[1];
    index_t inp_H = input.sizes[2];
    index_t inp_W = input.sizes[3];
    index_t out_H = grid.sizes[1];
    index_t out_W = grid.sizes[2];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sH = input.strides[2];
    index_t inp_sW = input.strides[3];
    index_t grid_sN = grid.strides[0];
    index_t grid_sH = grid.strides[1];
    index_t grid_sW = grid.strides[2];
    index_t grid_sCoor = grid.strides[3];
    index_t gOut_sN = grad_output.strides[0];
    index_t gOut_sC = grad_output.strides[1];
    index_t gOut_sH = grad_output.strides[2];
    index_t gOut_sW = grad_output.strides[3];
    index_t gInp_sN = grad_input.strides[0];
    index_t gInp_sC = grad_input.strides[1];
    index_t gInp_sH = grad_input.strides[2];
    index_t gInp_sW = grad_input.strides[3];
    index_t gGrid_sW = grad_grid.strides[2];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t n = index / (out_H * out_W);
      const auto grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y co-ordinates from grid
      scalar_t x = grid.data[grid_offset];
      scalar_t y = grid.data[grid_offset + grid_sCoor];

      // multipliers for gradients on ix and iy
      scalar_t gix_mult, giy_mult;
      scalar_t ix = grid_sampler_compute_source_index_set_grad(x, inp_W, padding_mode, align_corners, &gix_mult);
      scalar_t iy = grid_sampler_compute_source_index_set_grad(y, inp_H, padding_mode, align_corners, &giy_mult);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get NE, NW, SE, SW pixel values from (x, y)
        index_t ix_nw = static_cast<index_t>(::floor(ix));
        index_t iy_nw = static_cast<index_t>(::floor(iy));
        index_t ix_ne = ix_nw + 1;
        index_t iy_ne = iy_nw;
        index_t ix_sw = ix_nw;
        index_t iy_sw = iy_nw + 1;
        index_t ix_se = ix_nw + 1;
        index_t iy_se = iy_nw + 1;

        // get surfaces to each neighbor:
        scalar_t nw = (ix_se - ix)    * (iy_se - iy);
        scalar_t ne = (ix    - ix_sw) * (iy_sw - iy);
        scalar_t sw = (ix_ne - ix)    * (iy    - iy_ne);
        scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);

        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0);
        scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        scalar_t *inp_ptr_NC = input.data + n * inp_sN;
        for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, gInp_ptr_NC += gInp_sC, gOut_ptr_NCHW += gOut_sC) {
          scalar_t gOut = *gOut_ptr_NCHW;

          // calculate and set grad_input
          safe_add_2d(gInp_ptr_NC, iy_nw, ix_nw, gInp_sH, gInp_sW, inp_H, inp_W, nw * gOut);
          safe_add_2d(gInp_ptr_NC, iy_ne, ix_ne, gInp_sH, gInp_sW, inp_H, inp_W, ne * gOut);
          safe_add_2d(gInp_ptr_NC, iy_sw, ix_sw, gInp_sH, gInp_sW, inp_H, inp_W, sw * gOut);
          safe_add_2d(gInp_ptr_NC, iy_se, ix_se, gInp_sH, gInp_sW, inp_H, inp_W, se * gOut);

          // calculate grad_grid
          if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
            scalar_t nw_val = inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW];
            gix -= nw_val * (iy_se - iy) * gOut;
            giy -= nw_val * (ix_se - ix) * gOut;
          }
          if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
            scalar_t ne_val = inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW];
            gix += ne_val * (iy_sw - iy) * gOut;
            giy -= ne_val * (ix - ix_sw) * gOut;
          }
          if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
            scalar_t sw_val = inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW];
            gix -= sw_val * (iy - iy_ne) * gOut;
            giy += sw_val * (ix_ne - ix) * gOut;
          }
          if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
            scalar_t se_val = inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW];
            gix += se_val * (iy - iy_nw) * gOut;
            giy += se_val * (ix - ix_nw) * gOut;
          }
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NHW
        //   2. directly assign to gGrid_ptr_NHW[0], gGrid_ptr_NHW[1]
        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = gix_mult * gix;
        gGrid_ptr_NHW[1] = giy_mult * giy;
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        index_t ix_nearest = static_cast<index_t>(::round(ix));
        index_t iy_nearest = static_cast<index_t>(::round(iy));

        // assign nearest neighor pixel value to output pixel
        scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        for (index_t c = 0; c < C; ++c, gInp_ptr_NC += gInp_sC, gOut_ptr_NCHW += gOut_sC) {
          // calculate and set grad_input
          safe_add_2d(gInp_ptr_NC, iy_nearest, ix_nearest, gInp_sH, gInp_sW, inp_H, inp_W, *gOut_ptr_NCHW);
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NHW
        //   2. directly assign to gGrid_ptr_NHW[0], gGrid_ptr_NHW[1]
        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = static_cast<scalar_t>(0);
        gGrid_ptr_NHW[1] = static_cast<scalar_t>(0);
      } else if (interpolation_mode == GridSamplerInterpolation::Bicubic) {

        ix = grid_sampler_unnormalize_set_grad(x, inp_W, align_corners, &gix_mult);
        iy = grid_sampler_unnormalize_set_grad(y, inp_H, align_corners, &giy_mult);

        scalar_t ix_nw = ::floor(ix);
        scalar_t iy_nw = ::floor(iy);

        const scalar_t tx = ix - ix_nw;
        const scalar_t ty = iy - iy_nw;

        scalar_t x_coeffs[4];
        scalar_t y_coeffs[4];
        scalar_t x_coeffs_grad[4];
        scalar_t y_coeffs_grad[4];

        get_cubic_upsampling_coefficients<scalar_t>(x_coeffs, tx);
        get_cubic_upsampling_coefficients<scalar_t>(y_coeffs, ty);
        get_cubic_coefficients_grad<scalar_t>(x_coeffs_grad, tx);
        get_cubic_coefficients_grad<scalar_t>(y_coeffs_grad, ty);

        scalar_t gix = static_cast<scalar_t>(0);
        scalar_t giy = static_cast<scalar_t>(0);

        scalar_t *gOut_ptr_NCHW = grad_output.data + n * gOut_sN + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        scalar_t *inp_ptr_NC = input.data + n * inp_sN;

        for (index_t c = 0; c < C; ++c, gOut_ptr_NCHW += gOut_sC, gInp_ptr_NC += gInp_sC, inp_ptr_NC+= inp_sC) {
          scalar_t gOut = *gOut_ptr_NCHW;

          for (index_t i = 0; i < 4; ++i) {
            for (index_t j = 0; j < 4; ++j) {

              // set input gradient
              add_value_bounded<scalar_t>(gInp_ptr_NC, ix_nw - 1 + i, iy_nw - 1 + j, inp_W, inp_H,
                gInp_sW, gInp_sH, gOut * x_coeffs[i] * y_coeffs[j], padding_mode, align_corners);

              // set grid gradient
              scalar_t val = get_value_bounded<scalar_t>(inp_ptr_NC, ix_nw - 1 + i, iy_nw - 1 + j,
                inp_W, inp_H, inp_sW, inp_sH, padding_mode, align_corners);

              gix -= val * x_coeffs_grad[i] * y_coeffs[j] * gOut;
              giy -= val * y_coeffs_grad[j] * x_coeffs[i] * gOut;
            }
          }
        }

        scalar_t *gGrid_ptr_NHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NHW[0] = gix_mult * gix;
        gGrid_ptr_NHW[1] = giy_mult * giy;
      }
    }
  }

  template <typename scalar_t, typename index_t>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void grid_sampler_3d_backward_kernel(
      const index_t nthreads,
      TensorInfo<scalar_t, index_t> grad_output,
      TensorInfo<scalar_t, index_t> input,
      TensorInfo<scalar_t, index_t> grid,
      TensorInfo<scalar_t, index_t> grad_input,  // initialized to zeros
      TensorInfo<scalar_t, index_t> grad_grid,   // initialized to empty
      const GridSamplerInterpolation interpolation_mode,
      const GridSamplerPadding padding_mode,
      bool align_corners) {

    index_t C = input.sizes[1];
    index_t inp_D = input.sizes[2];
    index_t inp_H = input.sizes[3];
    index_t inp_W = input.sizes[4];
    index_t out_D = grid.sizes[1];
    index_t out_H = grid.sizes[2];
    index_t out_W = grid.sizes[3];
    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sD = input.strides[2];
    index_t inp_sH = input.strides[3];
    index_t inp_sW = input.strides[4];
    index_t grid_sN = grid.strides[0];
    index_t grid_sD = grid.strides[1];
    index_t grid_sH = grid.strides[2];
    index_t grid_sW = grid.strides[3];
    index_t grid_sCoor = grid.strides[4];
    index_t gOut_sN = grad_output.strides[0];
    index_t gOut_sC = grad_output.strides[1];
    index_t gOut_sD = grad_output.strides[2];
    index_t gOut_sH = grad_output.strides[3];
    index_t gOut_sW = grad_output.strides[4];
    index_t gInp_sN = grad_input.strides[0];
    index_t gInp_sC = grad_input.strides[1];
    index_t gInp_sD = grad_input.strides[2];
    index_t gInp_sH = grad_input.strides[3];
    index_t gInp_sW = grad_input.strides[4];
    index_t gGrid_sW = grad_grid.strides[3];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
      const index_t w = index % out_W;
      const index_t h = (index / out_W) % out_H;
      const index_t d = (index / (out_H * out_W)) % out_D;
      const index_t n = index / (out_D * out_H * out_W);
      const auto grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

      // get the corresponding input x, y, z co-ordinates from grid
      scalar_t ix = grid.data[grid_offset];
      scalar_t iy = grid.data[grid_offset + grid_sCoor];
      scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

      // multipliers for gradients on ix, iy, and iz
      scalar_t gix_mult, giy_mult, giz_mult;
      ix = grid_sampler_compute_source_index_set_grad(ix, inp_W, padding_mode, align_corners, &gix_mult);
      iy = grid_sampler_compute_source_index_set_grad(iy, inp_H, padding_mode, align_corners, &giy_mult);
      iz = grid_sampler_compute_source_index_set_grad(iz, inp_D, padding_mode, align_corners, &giz_mult);

      if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
        // get corner pixel values from (x, y, z)
        // for 4d, we used north-east-south-west
        // for 5d, we add top-bottom
        index_t ix_tnw = static_cast<index_t>(::floor(ix));
        index_t iy_tnw = static_cast<index_t>(::floor(iy));
        index_t iz_tnw = static_cast<index_t>(::floor(iz));

        index_t ix_tne = ix_tnw + 1;
        index_t iy_tne = iy_tnw;
        index_t iz_tne = iz_tnw;

        index_t ix_tsw = ix_tnw;
        index_t iy_tsw = iy_tnw + 1;
        index_t iz_tsw = iz_tnw;

        index_t ix_tse = ix_tnw + 1;
        index_t iy_tse = iy_tnw + 1;
        index_t iz_tse = iz_tnw;

        index_t ix_bnw = ix_tnw;
        index_t iy_bnw = iy_tnw;
        index_t iz_bnw = iz_tnw + 1;

        index_t ix_bne = ix_tnw + 1;
        index_t iy_bne = iy_tnw;
        index_t iz_bne = iz_tnw + 1;

        index_t ix_bsw = ix_tnw;
        index_t iy_bsw = iy_tnw + 1;
        index_t iz_bsw = iz_tnw + 1;

        index_t ix_bse = ix_tnw + 1;
        index_t iy_bse = iy_tnw + 1;
        index_t iz_bse = iz_tnw + 1;

        // get surfaces to each neighbor:
        scalar_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
        scalar_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
        scalar_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
        scalar_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
        scalar_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
        scalar_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
        scalar_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
        scalar_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

        scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0), giz = static_cast<scalar_t>(0);
        scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        scalar_t *inp_ptr_NC = input.data + n * inp_sN;
        // calculate bilinear weighted pixel value and set output pixel
        for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, gInp_ptr_NC += gInp_sC, inp_ptr_NC += inp_sC) {
          scalar_t gOut = *gOut_ptr_NCDHW;

          // calculate and set grad_input
          safe_add_3d(gInp_ptr_NC, iz_tnw, iy_tnw, ix_tnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tnw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_tne, iy_tne, ix_tne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tne * gOut);
          safe_add_3d(gInp_ptr_NC, iz_tsw, iy_tsw, ix_tsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tsw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_tse, iy_tse, ix_tse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tse * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bnw, iy_bnw, ix_bnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bnw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bne, iy_bne, ix_bne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bne * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bsw, iy_bsw, ix_bsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bsw * gOut);
          safe_add_3d(gInp_ptr_NC, iz_bse, iy_bse, ix_bse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bse * gOut);

          // calculate grad_grid
          if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
            scalar_t tnw_val = inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW];
            gix -= tnw_val * (iy_bse - iy)    * (iz_bse - iz)    * gOut;
            giy -= tnw_val * (ix_bse - ix)    * (iz_bse - iz)    * gOut;
            giz -= tnw_val * (ix_bse - ix)    * (iy_bse - iy)    * gOut;
          }
          if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
            scalar_t tne_val = inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW];
            gix += tne_val * (iy_bsw - iy)    * (iz_bsw - iz)    * gOut;
            giy -= tne_val * (ix    - ix_bsw) * (iz_bsw - iz)    * gOut;
            giz -= tne_val * (ix    - ix_bsw) * (iy_bsw - iy)    * gOut;
          }
          if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
            scalar_t tsw_val = inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW];
            gix -= tsw_val * (iy - iy_bne)    * (iz_bne - iz)    * gOut;
            giy += tsw_val * (ix_bne - ix)    * (iz_bne - iz)    * gOut;
            giz -= tsw_val * (ix_bne - ix)    * (iy    - iy_bne) * gOut;
          }
          if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
            scalar_t tse_val = inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW];
            gix += tse_val * (iy - iy_bnw)    * (iz_bnw - iz)    * gOut;
            giy += tse_val * (ix    - ix_bnw) * (iz_bnw - iz)    * gOut;
            giz -= tse_val * (ix    - ix_bnw) * (iy    - iy_bnw) * gOut;
          }
          if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
            scalar_t bnw_val = inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW];
            gix -= bnw_val * (iy_tse - iy)    * (iz - iz_tse)    * gOut;
            giy -= bnw_val * (ix_tse - ix)    * (iz - iz_tse)    * gOut;
            giz += bnw_val * (ix_tse - ix)    * (iy_tse - iy)    * gOut;
          }
          if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
            scalar_t bne_val = inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW];
            gix += bne_val * (iy_tsw - iy)    * (iz - iz_tsw)    * gOut;
            giy -= bne_val * (ix    - ix_tsw) * (iz - iz_tsw)    * gOut;
            giz += bne_val * (ix    - ix_tsw) * (iy_tsw - iy)    * gOut;
          }
          if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
            scalar_t bsw_val = inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW];
            gix -= bsw_val * (iy - iy_tne)    * (iz - iz_tne)    * gOut;
            giy += bsw_val * (ix_tne - ix)    * (iz - iz_tne)    * gOut;
            giz += bsw_val * (ix_tne - ix)    * (iy    - iy_tne) * gOut;
          }
          if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
            scalar_t bse_val = inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW];
            gix += bse_val * (iy - iy_tnw)    * (iz - iz_tnw)    * gOut;
            giy += bse_val * (ix    - ix_tnw) * (iz - iz_tnw)    * gOut;
            giz += bse_val * (ix    - ix_tnw) * (iy    - iy_tnw) * gOut;
          }
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
        //   2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
        scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NDHW[0] = gix_mult * gix;
        gGrid_ptr_NDHW[1] = giy_mult * giy;
        gGrid_ptr_NDHW[2] = giz_mult * giz;
      } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
        auto ix_nearest = static_cast<index_t>(::round(ix));
        auto iy_nearest = static_cast<index_t>(::round(iy));
        auto iz_nearest = static_cast<index_t>(::round(iz));

        // assign nearest neighor pixel value to output pixel
        scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
        scalar_t *gInp_ptr_NC = grad_input.data + n * gInp_sN;
        for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, gInp_ptr_NC += gInp_sC) {
          // calculate and set grad_input
          safe_add_3d(gInp_ptr_NC, iz_nearest, iy_nearest, ix_nearest,
                      gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, *gOut_ptr_NCDHW);
        }

        // assuming grad_grid is contiguous
        // thus we can
        //   1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
        //   2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
        scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
        gGrid_ptr_NDHW[0] = static_cast<scalar_t>(0);
        gGrid_ptr_NDHW[1] = static_cast<scalar_t>(0);
        gGrid_ptr_NDHW[2] = static_cast<scalar_t>(0);
      }
    }
  }
}  // namespace

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
Tensor grid_sampler_2d_cuda(const Tensor& input, const Tensor& grid,
                            int64_t interpolation_mode, int64_t padding_mode,
                            bool align_corners) {
  auto N = input.size(0);
  auto C = input.size(1);
  auto H = grid.size(1);
  auto W = grid.size(2);
  auto output = at::empty({N, C, H, W}, input.options());
  int64_t count = N * H * W;
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_2d_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(output)) {
        grid_sampler_2d_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<scalar_t, int>(input),
            getTensorInfo<scalar_t, int>(grid),
            getTensorInfo<scalar_t, int>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_2d_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<scalar_t, int64_t>(input),
            getTensorInfo<scalar_t, int64_t>(grid),
            getTensorInfo<scalar_t, int64_t>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
  return output;
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
Tensor grid_sampler_3d_cuda(const Tensor& input, const Tensor& grid,
                            int64_t interpolation_mode, int64_t padding_mode,
                            bool align_corners) {
  auto N = input.size(0);
  auto D = grid.size(1);
  auto H = grid.size(2);
  auto W = grid.size(3);
  auto output = at::empty({N, input.size(1), D, H, W}, input.options());
  int64_t count = N * D * H * W;
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_2d_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(output)) {
        grid_sampler_3d_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<scalar_t, int>(input),
            getTensorInfo<scalar_t, int>(grid),
            getTensorInfo<scalar_t, int>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_3d_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<scalar_t, int64_t>(input),
            getTensorInfo<scalar_t, int64_t>(grid),
            getTensorInfo<scalar_t, int64_t>(output),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
  return output;
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
std::tuple<Tensor, Tensor>
grid_sampler_2d_backward_cuda(const Tensor& grad_output, const Tensor& input,
                              const Tensor& grid, int64_t interpolation_mode,
                              int64_t padding_mode, bool align_corners) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("grid_sampler_2d_backward_cuda");
  auto N = input.size(0);
  auto H = grid.size(1);
  auto W = grid.size(2);
  auto grad_input = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto grad_grid = at::empty_like(grid, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  int64_t count = N * H * W;
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_2d_backward_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(grad_output)) {
        grid_sampler_2d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<scalar_t, int>(grad_output),
            getTensorInfo<scalar_t, int>(input),
            getTensorInfo<scalar_t, int>(grid),
            getTensorInfo<scalar_t, int>(grad_input),
            getTensorInfo<scalar_t, int>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_2d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<scalar_t, int64_t>(grad_output),
            getTensorInfo<scalar_t, int64_t>(input),
            getTensorInfo<scalar_t, int64_t>(grid),
            getTensorInfo<scalar_t, int64_t>(grad_input),
            getTensorInfo<scalar_t, int64_t>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
  return std::make_tuple(grad_input, grad_grid);
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
std::tuple<Tensor, Tensor>
grid_sampler_3d_backward_cuda(const Tensor& grad_output, const Tensor& input,
                              const Tensor& grid, int64_t interpolation_mode, int64_t padding_mode,
                              bool align_corners) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("grid_sampler_3d_backward_cuda");
  auto N = input.size(0);
  auto D = grid.size(1);
  auto H = grid.size(2);
  auto W = grid.size(3);
  auto grad_input = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto grad_grid = at::empty_like(grid, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  int64_t count = N * D * H * W;
  if (count > 0) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "grid_sampler_3d_backward_cuda", [&] {
      if (canUse32BitIndexMath(input) && canUse32BitIndexMath(grid) &&
          canUse32BitIndexMath(grad_output)) {
        grid_sampler_3d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            static_cast<int>(count),
            getTensorInfo<scalar_t, int>(grad_output),
            getTensorInfo<scalar_t, int>(input),
            getTensorInfo<scalar_t, int>(grid),
            getTensorInfo<scalar_t, int>(grad_input),
            getTensorInfo<scalar_t, int>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      } else {
        grid_sampler_3d_backward_kernel<scalar_t>
          <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
            getTensorInfo<scalar_t, int64_t>(grad_output),
            getTensorInfo<scalar_t, int64_t>(input),
            getTensorInfo<scalar_t, int64_t>(grid),
            getTensorInfo<scalar_t, int64_t>(grad_input),
            getTensorInfo<scalar_t, int64_t>(grad_grid),
            static_cast<GridSamplerInterpolation>(interpolation_mode),
            static_cast<GridSamplerPadding>(padding_mode),
            align_corners);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      }
    });
  }
  return std::make_tuple(grad_input, grad_grid);
}

}}  // namespace at::native
