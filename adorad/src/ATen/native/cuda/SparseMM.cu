#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <c10/util/Exception.h>

namespace at { namespace native {
// sparse, sparse, sparse, dense, real, real -> sparse
Tensor& _sspaddmm_out_only_sparse_cuda(Tensor& result, const Tensor& self,
    const Tensor& mat1, const Tensor& mat2, Scalar beta, Scalar alpha) {
  AT_ERROR("tensor.sspaddmm(...) can only be called on sparse tensors");
}
Tensor& _sspaddmm_out_cuda(Tensor& result, const Tensor& self,
    const Tensor& mat1, const Tensor& mat2, Scalar beta, Scalar alpha) {
  AT_ERROR("NYI: CUDA sspaddmm is not implemented");
}
}} // namespace at::native
