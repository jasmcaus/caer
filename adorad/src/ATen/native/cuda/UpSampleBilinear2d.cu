#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>

namespace at {
namespace native {
namespace {

__device__ __forceinline__ size_t
idx(const size_t nc,
    const size_t height,
    const size_t width,
    const size_t y,
    const size_t x) {
  return (nc * height + y) * width + x;
}

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bilinear2d_out_frame(
    const int n,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor<scalar_t, 4> idata,
    PackedTensorAccessor<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int height1 = idata.size(2);
  const int width1 = idata.size(3);
  const int height2 = odata.size(2);
  const int width2 = odata.size(3);

  if (index < n) {
    const int w2 = index % width2; // 0:width2-1
    const int h2 = index / width2; // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][h1][w1];
          odata[n][c][h2][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val = h0lambda *
                (w0lambda * idata[n][c][h1][w1] +
                 w1lambda * idata[n][c][h1][w1 + w1p]) +
            h1lambda *
                (w0lambda * idata[n][c][h1 + h1p][w1] +
                 w1lambda * idata[n][c][h1 + h1p][w1 + w1p]);
        odata[n][c][h2][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bilinear2d_backward_out_frame(
    const size_t nc,
    const int height1,
    const int width1,
    const int height2,
    const int width2,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    scalar_t* __restrict__ idata,
    const scalar_t* __restrict__ odata) {
  const size_t o_numel = nc * width2 * height2;
  const size_t i_numel = nc * width1 * height1;
  for (size_t index = blockDim.x * blockIdx.x + threadIdx.x; index < o_numel;
       index += blockDim.x * gridDim.x) {
    size_t index_temp = index;
    const int w2 = index_temp % width2; // 0:width2-1
    index_temp /= width2;
    const int h2 = index_temp % height2; // 0:height2-1
    const size_t nc = index_temp / height2;
    //
    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    const scalar_t d2val = odata[index];
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1, w1),
        i_numel,
        static_cast<scalar_t>(h0lambda * w0lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1, w1 + w1p),
        i_numel,
        static_cast<scalar_t>(h0lambda * w1lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1 + h1p, w1),
        i_numel,
        static_cast<scalar_t>(h1lambda * w0lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1 + h1p, w1 + w1p),
        i_numel,
        static_cast<scalar_t>(h1lambda * w1lambda * d2val),
        true);
  }
}

static void upsample_bilinear2d_out_cuda_template(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_bilinear2d_out_cuda", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_height = input.size(2);
  int input_width = input.size(3);

  upsample_2d_shape_check(
      input,
      Tensor(),
      nbatch,
      channels,
      input_height,
      input_width,
      output_height,
      output_width);

  output.resize_({input.size(0), input.size(1), output_height, output_width});

  AT_ASSERT(
      input_height > 0 && input_width > 0 && output_height > 0 &&
      output_width > 0);

  const int num_kernels = output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_bilinear2d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<scalar_t, 4>();
        auto odata = output.packed_accessor64<scalar_t, 4>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_bilinear2d_out_frame<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(
                num_kernels, rheight, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_bilinear2d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      "upsample_bilinear2d_backward_out_cuda",
      {grad_output_arg, grad_input_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 4,
      "It is expected input_size equals to 4, but got size ",
      input_size.size());

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_height = input_size[2];
  int input_width = input_size[3];

  upsample_2d_shape_check(
      Tensor(),
      grad_output_,
      nbatch,
      channels,
      input_height,
      input_width,
      output_height,
      output_width);

  Tensor grad_output = grad_output_.contiguous();

  grad_input.resize_({nbatch, channels, input_height, input_width});
  if (grad_input.numel() == 0) {
    return;
  }

  // A contiguous tensor is required for the kernel launch config
  grad_input.contiguous();
  // initialization to zero is required here. As we launch one thread per output
  // element, and atomicAdd to input gradient. Given a sparse sampling case, our
  // threads are not covering the whole input tensor.
  grad_input.zero_();

  const size_t num_kernels = nbatch * channels * output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "upsample_bilinear2d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.data_ptr<scalar_t>();
        auto odata = grad_output.data_ptr<scalar_t>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_bilinear2d_backward_out_frame<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_kernels, static_cast<size_t>(num_threads)),
               num_threads,
               0,
               stream>>>(
                nbatch * channels,
                input_height,
                input_width,
                output_height,
                output_width,
                rheight,
                rwidth,
                align_corners,
                idata,
                odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

Tensor& upsample_bilinear2d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  upsample_bilinear2d_out_cuda_template(
      output, input, output_size, align_corners, scales_h, scales_w);
  return output;
}

Tensor upsample_bilinear2d_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_bilinear2d_out_cuda_template(
      output, input, output_size, align_corners, scales_h, scales_w);
  return output;
}

Tensor& upsample_bilinear2d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bilinear2d_backward_out_cuda");
  upsample_bilinear2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_h, scales_w);
  return grad_input;
}

Tensor upsample_bilinear2d_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bilinear2d_backward_cuda");
  Tensor grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_bilinear2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_h, scales_w);
  return grad_input;
}

using at::native::upsample::compute_output_size;
using at::native::upsample_cuda::get_scale_value;

Tensor upsample_bilinear2d_cuda(
    const Tensor& input,
    c10::optional<IntArrayRef> output_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto osize = compute_output_size(input.sizes(), output_size, scale_factors);
  auto scale_h = get_scale_value(scale_factors, 0);
  auto scale_w = get_scale_value(scale_factors, 1);
  upsample_bilinear2d_out_cuda_template(output, input, osize, align_corners, scale_h, scale_w);
  return output;
}

Tensor upsample_bilinear2d_backward_cuda(
    const Tensor& grad_output,
    c10::optional<IntArrayRef> output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bilinear2d_backward_cuda");
  auto osize = compute_output_size(input_size, output_size, scale_factors);
  auto scale_h = get_scale_value(scale_factors, 0);
  auto scale_w = get_scale_value(scale_factors, 1);
  auto grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_bilinear2d_backward_out_cuda_template(
      grad_input, grad_output, osize, input_size, align_corners, scale_h, scale_w);
  return grad_input;
}

} // namespace native
} // namespace at
