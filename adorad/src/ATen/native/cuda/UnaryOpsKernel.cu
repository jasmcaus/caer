#include "hip/hip_runtime.h"
#include <ATen/native/UnaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorFactories.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/NumericUtils.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <ATen/NumericUtils.h>
#include <c10/util/complex.h>

namespace at {
namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void exp_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "exp_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::exp(a);
    });
  });
}

void exp2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "exp2_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::exp2(a);
    });
  });
}

void expm1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "expm1_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::expm1(a);
    });
  });
}

void i0_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "i0_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_i0(a);
    });
  });
}

// We manually overload rsqrt because std::rsqrt does not work with complex types.
template<typename scalar_t>
__host__ __device__ static inline scalar_t rsqrt_wrapper(scalar_t v) {
  return ::rsqrt(v);
}

template<typename T>
__host__ __device__ static inline c10::complex<T> rsqrt_wrapper(c10::complex<T> v) {
  const c10::complex<T> one = c10::complex<T>(1.0, 0);
  // std::sqrt for c10::complex is overloaded in c10/util/complex_math.h
  return one / ::sqrt(v);
}

void rsqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "rsqrt_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      // In CUDA, ::rsqrt is overloaded for float and at::Half here is implicitly cast to float.
      return rsqrt_wrapper(a);
    });
  });
}

void sqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "sqrt_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sqrt(a);
    });
  });
}

void sigmoid_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "sigmoid_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      scalar_t one = scalar_t(1);
      return  one / (one + std::exp(- a));
    });
  });
}

void logit_kernel_cuda(TensorIterator& iter, Scalar eps_scalar) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "logit_cuda",
      [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        const T_ACC eps = eps_scalar.to<T_ACC>();
        if (eps < T_ACC(0)) {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
            const T_ACC x_acc = static_cast<T_ACC>(x);
            return c10::cuda::compat::log(x_acc / (T_ACC(1) - x_acc));
          });
        } else {
          const T_ACC lo = eps;
          const T_ACC hi = T_ACC(1) - eps;
          gpu_kernel(
              iter, [lo, hi] GPU_LAMBDA(scalar_t x) -> scalar_t {
                const T_ACC x_acc = static_cast<T_ACC>(x);
                T_ACC z = x_acc < lo ? lo : (x_acc > hi ? hi : x_acc);
                return c10::cuda::compat::log(z / (T_ACC(1) - z));
              });
        }
      });
}

void erf_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "erf_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erf(a);
    });
  });
}

void erfc_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "erfc_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfc(a);
    });
  });
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

void clamp_kernel_cuda(TensorIterator& iter, Scalar min_value, Scalar max_value) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "clamp_cuda", [&]() {
    auto lower = min_value.to<scalar_t>();
    auto upper = max_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::min(::max(v, lower), upper);
      }
    });
  });
}

void clamp_min_kernel_cuda(TensorIterator& iter, Scalar min_value) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "clamp_min_cuda", [&]() {
    auto lower = min_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::max(v, lower);
      }
    });
  });
}

void clamp_max_kernel_cuda(TensorIterator& iter, Scalar max_value) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "clamp_max_cuda", [&]() {
    auto upper = max_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::min(v, upper);
      }
    });
  });
}

void nan_to_num_kernel_cuda(
    TensorIterator& iter,
    c10::optional<double> nan,
    c10::optional<double> pos_inf,
    c10::optional<double> neg_inf) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "nan_to_num_cuda", [&]() {
    scalar_t nan_replacement = static_cast<scalar_t>(nan.value_or(0.));
    scalar_t pos_inf_replacement = pos_inf.has_value()
        ? static_cast<scalar_t>(pos_inf.value())
        : std::numeric_limits<scalar_t>::max();
    scalar_t neg_inf_replacement = neg_inf.has_value()
        ? static_cast<scalar_t>(neg_inf.value())
        : std::numeric_limits<scalar_t>::lowest();
    gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (
          at::_isnan(a)
              ? nan_replacement
              : (a == std::numeric_limits<scalar_t>::infinity()
                     ? pos_inf_replacement
                     : (a == -std::numeric_limits<scalar_t>::infinity()
                            ? neg_inf_replacement
                            : a)));
    });
  });
}

void kaiser_window_kernel_cuda(TensorIterator& iter, int64_t window_length, double beta_){
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "kaiser_window_cuda", [&](){
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC inv_alpha = static_cast<T_ACC>(2.0 / (window_length - 1));
    const T_ACC beta = static_cast<T_ACC>(beta_);
    const T_ACC inv_i0_beta = 1.0 / calc_i0(beta);
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t a) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(a) * inv_alpha - 1;
      T_ACC y = std::max<T_ACC>(0, 1 - x * x);
      return calc_i0(beta * ::sqrt(y)) * inv_i0_beta;
    });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(exp_stub, &exp_kernel_cuda);
REGISTER_DISPATCH(exp2_stub, &exp2_kernel_cuda);
REGISTER_DISPATCH(expm1_stub, &expm1_kernel_cuda);
REGISTER_DISPATCH(i0_stub, &i0_kernel_cuda);
REGISTER_DISPATCH(rsqrt_stub, &rsqrt_kernel_cuda);
REGISTER_DISPATCH(sqrt_stub, &sqrt_kernel_cuda);
REGISTER_DISPATCH(sigmoid_stub, &sigmoid_kernel_cuda);
REGISTER_DISPATCH(logit_stub, &logit_kernel_cuda);
REGISTER_DISPATCH(erf_stub, &erf_kernel_cuda);
REGISTER_DISPATCH(erfc_stub, &erfc_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(clamp_stub, &clamp_kernel_cuda);
REGISTER_DISPATCH(clamp_min_stub, &clamp_min_kernel_cuda);
REGISTER_DISPATCH(clamp_max_stub, &clamp_max_kernel_cuda);
REGISTER_DISPATCH(nan_to_num_stub, &nan_to_num_kernel_cuda);
REGISTER_DISPATCH(kaiser_window_stub, &kaiser_window_kernel_cuda);

} // namespace native
} // namespace at
