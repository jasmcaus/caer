#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/TensorUtils.h>
#include <c10/util/Exception.h>

#include <ATen/cuda/HIPContext.h>
#include <THC/THCDeviceUtils.cuh>
#include <THC/THCTensorMathReduce.cuh>

namespace at {
namespace native {
namespace {

// Block size for weight_norm_*_first_dim_kernel.
// Currently, kernels are non-persistent.
// Dialing up the block size to, say 1024, can improve performance by
// increase the amount of cache available per block, which can improve cache hit rate.
// However, this is less efficient for short rows.  256 is pretty versatile.
// May be worth implementing heuristics later.
#define BLOCK 256

// Block size for weight_norm_*_last_dim_kernel.
// This is tricker than the first_dim case because we must make blocks
// at least 16 fast elements wide to ensure fully-coalesced half-precision accesses.
// Since output-element parallelism is along the fast dimension, this reduces the number of
// blocks we can launch by 16X.
#define TILE_W 16
// Somewhat versatile strategy: max out intra-block parallelism by extending
// blocks across the slow dimension up to the hardware-max block size of 1024.
#define TILE_H 64

template<typename T, typename ReduceOp>
__device__ __forceinline__ void reduce_block_into_lanes
  (T *x,
   T val,
   int lanes, // lanes is intended to be <= 32.
   ReduceOp reduceOp)
{
  int tid = threadIdx.x + threadIdx.y*blockDim.x;
  int blockSize = blockDim.x*blockDim.y; // blockSize is intended to be a multiple of 32.

  if(blockSize >= 64)
  {
    x[tid] = val;
    __syncthreads();
  }

#ifndef __HIP_PLATFORM_HCC__
  #pragma unroll
#endif
  for(int i = (blockSize >> 1); i >= 64; i >>= 1)
  {
    if(tid < i)
      x[tid] = reduceOp(x[tid], x[tid+i]);
    __syncthreads();
  }

  if(tid < 32)
  {
    T final;
    if(blockSize >= 64)
      final = reduceOp(x[tid], x[tid+32]);
    else
      final = val;
    // __SYNCWARP();

#ifndef __HIP_PLATFORM_HCC__
    #pragma unroll
#endif
    for(int i = 16; i >= lanes; i >>= 1)
      final = reduceOp(final, WARP_SHFL_DOWN(final, i));

    if(tid < lanes)
      x[tid] = final; // EpilogueOp
  }

  // Make sure the smem result is visible to all warps.
  __syncthreads();
}

template
  <typename scalar_t,
   typename accscalar_t>
__global__ void weight_norm_fwd_first_dim_kernel
  (scalar_t* __restrict__ w,
   accscalar_t* __restrict__ norms,
   const scalar_t* __restrict__ v,
   const scalar_t* __restrict__ g,
   const int rowSize)
{
  // We are norming each slowest-dim row of the tensor separately.
  // For now, assign one block to each row.
  const int tid = threadIdx.x;
  const int row = blockIdx.x;
  const int stride = blockDim.x;

  // Logical index offset for this flattened row
  const int rowStart = row*rowSize;

  // Hack to get around nvcc complaining when an smem array is declared with the same name
  // but different types in different kernels (in this case different instantiations)
  // extern __shared__ accscalar_t s[]; // error: declaration is incompatible with previous "s"
  extern __shared__ char buf[];
  accscalar_t* s = (accscalar_t*)buf;

  accscalar_t thread_sum = 0.f;
  for(int i = tid; i < rowSize; i += stride )
  {
    accscalar_t val_f = scalar_cast<accscalar_t>(v[i+rowStart]);
    thread_sum += val_f*val_f; // AccumOp, could do Kahan here
  }

  reduce_block_into_lanes(s, thread_sum, 1, ReduceAdd<accscalar_t>());
  accscalar_t result = s[0];

  result = sqrtf(result);

  if(tid == 0)
    norms[row] = result;

  // Broadcast load, could use shared memory instead.
  accscalar_t g_this_row = scalar_cast<accscalar_t>(g[row]);

  accscalar_t rnorm = 1.f/result; // for consistency with backward kernel

  // Write data to output
  for(int i = tid; i < rowSize; i += stride )
  {
    accscalar_t val_f = scalar_cast<accscalar_t>(v[i+rowStart]);
    w[i+rowStart] = scalar_cast<scalar_t>(g_this_row*val_f*rnorm);
  }
}

template
  <typename scalar_t,
   typename accscalar_t>
__global__ void weight_norm_fwd_last_dim_kernel
(
  scalar_t* __restrict__ w,
  accscalar_t* __restrict__ norms,
  const scalar_t* __restrict__ v,
  const scalar_t* __restrict__ g,
  const int fast_dim_size,
  const int slower_dims_size
)
{
  const int fast_dim_location = threadIdx.x + blockIdx.x*blockDim.x;

  extern __shared__ char buf[];
  accscalar_t* alloc = (accscalar_t*)buf;
  accscalar_t* s = &alloc[0];
  accscalar_t* rnorms_this_block = &alloc[blockDim.x*blockDim.y];

  accscalar_t thread_sum = 0.f;

  int slower_dims_location = threadIdx.y;
  int currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      accscalar_t val_f = scalar_cast<accscalar_t>(v[currentIdx]);
      thread_sum += val_f*val_f; // AccumOp, could do Kahan here
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y;
    }

  reduce_block_into_lanes(s, thread_sum, blockDim.x, ReduceAdd<accscalar_t>());

  // Better to pass an EpilogueOp to reduce_block_into_lanes?
  if(threadIdx.y == 0)
  {
    accscalar_t result = s[threadIdx.x];
    accscalar_t norm_this_col = sqrtf(result);
    norms[fast_dim_location] = norm_this_col;
    rnorms_this_block[threadIdx.x] = 1.f/norm_this_col;
  }

  __syncthreads();

  accscalar_t g_this_col = scalar_cast<accscalar_t>(g[fast_dim_location]);
  accscalar_t rnorm = rnorms_this_block[threadIdx.x];

  slower_dims_location = threadIdx.y;
  currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      accscalar_t val_f = scalar_cast<accscalar_t>(v[currentIdx]);
      w[currentIdx] = scalar_cast<scalar_t>(g_this_col*val_f*rnorm);
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y;
    }
}

template
  <typename scalar_t,
   typename accscalar_t>
__global__ void weight_norm_bwd_first_dim_kernel
  (scalar_t* __restrict__ grad_v,
   scalar_t* __restrict__ grad_g,
   const scalar_t* __restrict__ grad_w,
   const scalar_t* __restrict__ saved_v,
   const scalar_t* __restrict__ saved_g,
   const accscalar_t* __restrict__ saved_norms,
   const int rowSize)
{
  // For now, assign one block to each row.
  const int tid = threadIdx.x;
  const int row = blockIdx.x;
  const int stride = blockDim.x;

  // Logical index offset for this flattened row
  const int rowStart = row*rowSize;

  // Hack to get around nvcc complaining when an smem array is declared with the same name
  // but different types in different kernels (in this case different instantiations)
  // extern __shared__ accscalar_t s[]; // error: declaration is incompatible with previous "s"
  extern __shared__ char buf[];
  accscalar_t* s = (accscalar_t*)buf;

  accscalar_t thread_sum = 0.f;
  for(int i = tid; i < rowSize; i += stride )
  {
    accscalar_t grad_wi = scalar_cast<accscalar_t>(grad_w[i+rowStart]);
    accscalar_t saved_vi = scalar_cast<accscalar_t>(saved_v[i+rowStart]);
    thread_sum += grad_wi*saved_vi; // AccumOp, could do Kahan here
  }

  reduce_block_into_lanes(s, thread_sum, 1, ReduceAdd<accscalar_t>());
  accscalar_t result = s[0];

  // Could choose to save reciprocal of norm instead I suppose, but norms is probably
  // more handy to keep around.
  // Broadcast load; could use shared memory instead.
  accscalar_t rnorm = 1.f/saved_norms[row];
  accscalar_t rnorm3 = rnorm*rnorm*rnorm;

  // Write g gradients.
  if(tid == 0)
    grad_g[row] = scalar_cast<scalar_t>(result*rnorm);

  // Broadcast load, could use shared memory instead.
  accscalar_t g_this_row = scalar_cast<accscalar_t>(saved_g[row]);

  // Write v gradients.  We are reusing values that were loaded earlier, so there
  // is an optimization opportunity here (store values persistently).
  for(int j = tid; j < rowSize; j += stride )
  {
    accscalar_t grad_wj = scalar_cast<accscalar_t>(grad_w[j+rowStart]);
    accscalar_t saved_vj = scalar_cast<accscalar_t>(saved_v[j+rowStart]);
    accscalar_t grad_vj = g_this_row*(rnorm*grad_wj - rnorm3*saved_vj*result);
    grad_v[j+rowStart] = scalar_cast<scalar_t>(grad_vj);
  }
}

template
  <typename scalar_t,
   typename accscalar_t>
__global__ void weight_norm_bwd_last_dim_kernel
  (scalar_t* __restrict__ grad_v,
   scalar_t* __restrict__ grad_g,
   const scalar_t* __restrict__ grad_w,
   const scalar_t* __restrict__ saved_v,
   const scalar_t* __restrict__ saved_g,
   const accscalar_t* __restrict__ saved_norms,
   const int fast_dim_size,
   const int slower_dims_size)
{
  const int fast_dim_location = threadIdx.x + blockIdx.x*blockDim.x;

  extern __shared__ char buf[];
  accscalar_t* s = (accscalar_t*)buf;

  accscalar_t thread_sum = 0.f;

  int slower_dims_location = threadIdx.y;
  int currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      accscalar_t grad_wi = scalar_cast<accscalar_t>(grad_w[currentIdx]);
      accscalar_t saved_vi = scalar_cast<accscalar_t>(saved_v[currentIdx]);
      thread_sum += grad_wi*saved_vi; // AccumOp, could do Kahan here
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y;
    }

  reduce_block_into_lanes(s, thread_sum, blockDim.x, ReduceAdd<accscalar_t>());
  accscalar_t result = s[threadIdx.x];

  // Broadcast load; could use shared memory instead.
  accscalar_t rnorm = 1.f/saved_norms[fast_dim_location];
  accscalar_t rnorm3 = rnorm*rnorm*rnorm;

  // Write g gradients.
  if(threadIdx.y == 0)
    grad_g[fast_dim_location] = scalar_cast<scalar_t>(result*rnorm);

  // Entire block pulls these values, could use shared memory instead.
  accscalar_t g_this_col = scalar_cast<accscalar_t>(saved_g[fast_dim_location]);

  // Write v gradients.
  slower_dims_location = threadIdx.y;
  currentIdx = fast_dim_location + fast_dim_size*slower_dims_location;
  if(fast_dim_location < fast_dim_size)
    while(slower_dims_location < slower_dims_size)
    {
      accscalar_t grad_wj = scalar_cast<accscalar_t>(grad_w[currentIdx]);
      accscalar_t saved_vj = scalar_cast<accscalar_t>(saved_v[currentIdx]);
      accscalar_t grad_vj = g_this_col*(rnorm*grad_wj - rnorm3*saved_vj*result);
      grad_v[currentIdx] = scalar_cast<scalar_t>(grad_vj);
      currentIdx += blockDim.y*fast_dim_size;
      slower_dims_location += blockDim.y;
    }
}

} // anonymous namespace

std::tuple<Tensor,Tensor> weight_norm_cuda
  (const Tensor & v,
   const Tensor & g,
   int64_t dim)
{
  auto w = at::empty_like(v, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  // weight_norm_fused does have a derivative defined in derivatives.yaml, therefore, VariableType.cpp
  // sends the unpacked g.data() as the argument.  In other words, we expect "g" is a bare Tensor here.

  // norms is only needed to stash for backward.
  // g.scalar_type() may be at::ScalarType::Double, Float, or Half.
  // If Half, stash norms as float.
  at::ScalarType AccType = g.scalar_type() == at::ScalarType::Half ?
                           at::ScalarType::Float : g.scalar_type();
  // Will this create norms on the same device as g, regardless of what the thread's default
  // current device is?  I believe so, because Type::* functions are DeviceGuard()ed.
  auto norms = at::empty_strided(g.sizes(), g.strides(), g.options().dtype(AccType));

  const int ndims = v.dim();

  if(dim == 0)
  {
    // Find logical size of each flattened slowest-dim row
    int rowSize = 1;
    for(int i = ndims - 1; i > 0; i--)
      rowSize *= v.size(i);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF
      (v.scalar_type(),
       "weight_norm_fwd_first_dim_kernel",
       [&]
       {
         using accscalar_t = acc_type<scalar_t, true>;

         weight_norm_fwd_first_dim_kernel<scalar_t, accscalar_t>
           <<<v.size(0),
              BLOCK,
              BLOCK*sizeof(accscalar_t),
              stream>>>
           (w.data_ptr<scalar_t>(),
            norms.data_ptr<accscalar_t>(),
            v.data_ptr<scalar_t>(),
            g.data_ptr<scalar_t>(),
            rowSize);
       });
  }
  else if(dim == ndims - 1)
  {
    // Precompute slower_dims_size and fast_dim_size
    int slower_dims_size = 1;
    for(int i = 0; i < ndims - 1; i++)
      slower_dims_size *= v.size(i);

    int fast_dim_size = v.size(ndims-1);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF
      (v.scalar_type(),
       "weight_norm_fwd_last_dim_kernel",
       [&]
       {
         using accscalar_t = acc_type<scalar_t, true>;

         weight_norm_fwd_last_dim_kernel<scalar_t, accscalar_t>
           <<<(fast_dim_size+TILE_W-1)/TILE_W,
              dim3(TILE_W,TILE_H),
              (TILE_W*TILE_H + TILE_W)*sizeof(accscalar_t),
              stream>>>
           (w.data_ptr<scalar_t>(),
            norms.data_ptr<accscalar_t>(),
            v.data_ptr<scalar_t>(),
            g.data_ptr<scalar_t>(),
            fast_dim_size,
            slower_dims_size);
         C10_CUDA_KERNEL_LAUNCH_CHECK();
       });
  }

  // The kernel execution is asynchronous, so this will only catch errors on the kernel launch,
  // not the kernel's execution.  Errors in kernel execution aren't guaranteed to be caught
  // until a later error check on a synchronizing CUDA call.  Unfortunately, without manually
  // synchronizing here, the foregoing is the best we can do.

  return std::tuple<Tensor, Tensor>{w, norms};
}

std::tuple<Tensor, Tensor> weight_norm_cuda_backward
  (const Tensor & grad_w,
   const Tensor & saved_v,
   const Tensor & saved_g,
   const Tensor & saved_norms,
   int64_t dim)
{
  // These checks should always succeed, because weight_norm_fused_backward should only
  // ever be recorded in the autograd graph via weight_norm, which passes contiguous v and g.
  TORCH_CHECK(saved_v.is_contiguous(), "saved_v must be contiguous");
  TORCH_CHECK(saved_g.is_contiguous(), "saved_g must be contiguous");
  TORCH_CHECK(saved_norms.is_contiguous(), "saved_norms must be contiguous");
  TORCH_CHECK(dim == 0 || dim == saved_v.dim() - 1, "fused kernels can only be applied for first or last dim")

  auto grad_v = at::empty_like(saved_v, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto grad_g = at::empty_like(saved_g, LEGACY_CONTIGUOUS_MEMORY_FORMAT);

  const int ndims = saved_v.dim();

  if(dim == 0)
  {
    // Find logical size of each flattened slowest-dim row
    int rowSize = 1;
    for(int i = ndims - 1; i > 0; i--)
      rowSize *= saved_v.size(i);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF
      (saved_v.scalar_type(),
       "weight_norm_bwd_first_dim_kernel",
       [&]
       {
         using accscalar_t = acc_type<scalar_t, true>;

         weight_norm_bwd_first_dim_kernel<scalar_t, accscalar_t>
           <<<grad_w.size(0),
              BLOCK,
              BLOCK*sizeof(accscalar_t),
              stream>>>
           (grad_v.data_ptr<scalar_t>(),
            grad_g.data_ptr<scalar_t>(),
            grad_w.data_ptr<scalar_t>(),
            saved_v.data_ptr<scalar_t>(),
            saved_g.data_ptr<scalar_t>(),
            saved_norms.data_ptr<accscalar_t>(),
            rowSize);
       });
  }
  else if(dim == ndims - 1)
  {
    // Precompute slower_dims_size and fast_dim_size because they involve dynamically indexing an array.
    int slower_dims_size = 1;
    for(int i = 0; i < ndims - 1; i++)
      slower_dims_size *= saved_v.size(i);

    int fast_dim_size = saved_v.size(ndims-1);

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF
      (saved_v.scalar_type(),
       "weight_norm_bwd_last_dim_kernel",
       [&]
       {
         using accscalar_t = acc_type<scalar_t, true>;

         weight_norm_bwd_last_dim_kernel<scalar_t, accscalar_t>
           <<<(fast_dim_size+TILE_W-1)/TILE_W,
              dim3(TILE_W,TILE_H),
              (TILE_W*TILE_H + TILE_W)*sizeof(accscalar_t),
              stream>>>
           (grad_v.data_ptr<scalar_t>(),
            grad_g.data_ptr<scalar_t>(),
            grad_w.data_ptr<scalar_t>(),
            saved_v.data_ptr<scalar_t>(),
            saved_g.data_ptr<scalar_t>(),
            saved_norms.data_ptr<accscalar_t>(),
            fast_dim_size,
            slower_dims_size);
         C10_CUDA_KERNEL_LAUNCH_CHECK();
       });
  }

  // The kernel execution is asynchronous, so this will only catch errors on the kernel launch,
  // not the kernel's execution.  Errors in kernel execution aren't guaranteed to be caught
  // until a later error check on a synchronizing CUDA call.  Unfortunately, without manually
  // synchronizing here, the foregoing is the best we can do.

  return std::tuple<Tensor, Tensor>{grad_v, grad_g};
}

#undef BLOCK
#undef TILE_W
#undef TILE_H

} // namespace native
} // namespace at
