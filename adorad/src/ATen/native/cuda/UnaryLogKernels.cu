#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void log_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "log_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::log(a);
    });
  });
}

void log10_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "log10_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::log10(a);
    });
  });
}

void log1p_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "log1p_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::log1p(a);
    });
  });
}

void log2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "log2_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::log2(a);
    });
  });
}

REGISTER_DISPATCH(log_stub, &log_kernel_cuda);
REGISTER_DISPATCH(log10_stub, &log10_kernel_cuda);
REGISTER_DISPATCH(log2_stub, &log2_kernel_cuda);
REGISTER_DISPATCH(log1p_stub, &log1p_kernel_cuda);

}} // namespace at::native
