#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Pow.h>

namespace at { namespace native {

namespace {


// SFINAE doesn't work well with NVCC under Windows for math functions like pow and sqrt.
// So we need to define the functions with the explicit function signatures.
// As for pow, the following signatures are defined as the device function:
//   pow(float, int)
//   pow(double, int)
//   pow(float, float)
//   pow(double, double)
// As for sqrt, the following signatures are defined as the device function:
//   sqrt(float)
//   sqrt(double)
// As for inverse sqrt, we must define it explicitly in MSVC, otherwise the static cast will be
// applied to the result of the inline function, and thus the result is incorrect.
//   e.g. if we use 1.0 / sqrt(2) for 2 ^ (-0.5) in MSVC, we get
//          int(2 ^ (-0.5)) = int(1.0 / sqrt(2)) = int(1.0 / int(1.414)) = int(1.0 / 1) = 1
//        However, the correct result is 
//          int(2 ^ (-0.5)) = int(1.0 / 1.414) = 0
#ifdef _MSC_VER
// Functions for pow
// pow for at::Half
static inline __host__ __device__ at::Half pow_(at::Half base, at::Half exp) {
  return static_cast<at::Half>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow for at::BFloat16
static inline __host__ __device__ at::BFloat16 pow_(at::BFloat16 base, at::BFloat16 exp) {
  return static_cast<at::BFloat16>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow (floating, floating/int)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<std::is_floating_point<Base_type>::value && (std::is_same<Base_type, Exp_type>::value || std::is_same<Exp_type, int>::value), Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return std::pow(base, exp);
}
// pow (integral, integral)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<std::is_integral<Base_type>::value && std::is_same<Base_type, Exp_type>::value, Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return native::powi(base, exp);
}
// pow (Otherwise)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<!std::is_same<Base_type, Exp_type>::value && !std::is_same<Exp_type, int>::value, Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return static_cast<Base_type>(std::pow(static_cast<double>(base), static_cast<double>(exp)));
}
// pow (Complex)
template<typename B, typename E>
static inline __host__ __device__ B complex_pow_(B base, E exp) {
  return std::pow(base, exp);
}
// Functions for sqrt
// sqrt (floating)
template <typename T>
static inline __host__ __device__ typename std::enable_if<std::is_floating_point<T>::value, T>::type sqrt_(T x) {
  return std::sqrt(x);
}
// sqrt (integral)
template <typename T>
static inline __host__ __device__ typename std::enable_if<!std::is_floating_point<T>::value, T>::type sqrt_(T x) {
  return static_cast<T>(std::sqrt(static_cast<double>(x)));
}
// Function for inverse sqrt
// invsqrt (floating)
template <typename T>
static inline __host__ __device__ typename std::enable_if<std::is_floating_point<T>::value, T>::type invsqrt_(T x) {
  return 1.0 / std::sqrt(x);
}
// invsqrt (integral)
template <typename T>
static inline __host__ __device__ typename std::enable_if<!std::is_floating_point<T>::value, T>::type invsqrt_(T x) {
  return static_cast<T>(1.0 / std::sqrt(static_cast<double>(x)));
}
#else
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ Base_type pow_(Base_type base, Exp_type exp) {
  return ::pow(base, exp);
}
template <typename T>
static inline __host__ __device__ T sqrt_(T x) {
  return ::sqrt(x);
}
template <typename T>
static inline __host__ __device__ T invsqrt_(T x) {
  return 1.0 / ::sqrt(x);
}
// pow (Otherwise)
template<typename B, typename E>
static inline __host__ __device__ B complex_pow_(B base, E exp) {
  return std::pow(base, exp);
}
#endif

void pow_tensor_tensor_kernel(TensorIterator& iter) {
  if (isComplexType(iter.dtype())) {
    AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "pow_cuda", [&]() {
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return complex_pow_(base, exp);
      });
    });
  } else if (isFloatingType(iter.dtype())) {
    AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "pow_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return pow_(base, exp);
      });
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "pow_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return native::powi(base, exp);
      });
    });
  }
}

template<typename Base_type, typename Exp_type>
void pow_tensor_scalar_kernel_impl(TensorIterator& iter,
                                                 Exp_type exp) {
  const auto d_exp = static_cast<double>(exp);
  if (d_exp == 0.5) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return sqrt_(base);
    });
  } else if (d_exp == 2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base;
    });
  } else if (d_exp == 3) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base * base;
    });
  } else if (d_exp == -0.5) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return invsqrt_(base);
    });
  } else if (d_exp == -1) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / base;
    });
  } else if (d_exp == -2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / (base * base);
    });
  } else {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return pow_(base, exp);
    });
  }
}

void pow_tensor_scalar_kernel(TensorIterator& iter, Scalar exp_scalar) {
  if (isComplexType(iter.dtype()) || exp_scalar.isComplex()) {
    AT_DISPATCH_COMPLEX_TYPES(iter.dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base) -> scalar_t {
        return complex_pow_(base, exp);
      });
    });
  } else if (isFloatingType(iter.dtype()) || exp_scalar.isIntegral(false)) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  } else {
    const auto exp = exp_scalar.to<float>();
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "pow_cuda", [&]() {
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  }
}

} // anonymous namespace

REGISTER_DISPATCH(pow_tensor_tensor_stub, &pow_tensor_tensor_kernel);
REGISTER_DISPATCH(pow_tensor_scalar_stub, &pow_tensor_scalar_kernel);

}} // namespace at::native
