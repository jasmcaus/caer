#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

// We manually overload ceil because std::ceil does not work with std::complex types.
template <typename scalar_t>
__host__ __device__ static inline scalar_t ceil_wrapper(scalar_t a) {
  return std::ceil(a);
}

template<typename T>
__host__ __device__ static inline std::complex<T> ceil_wrapper(std::complex<T> v) {
  return std::complex<T>(std::ceil(v.real()), std::ceil(v.imag()));
}

void ceil_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(ScalarType::Half, iter.dtype(), "ceil_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ceil_wrapper(a);
    });
  });
}

void frac_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "frac_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a - ::trunc(a);
    });
  });
}

// We manually overload floor because std::floor does not work with std::complex types.
template <typename scalar_t>
__host__ __device__ static inline scalar_t floor_wrapper(scalar_t a) {
  return std::floor(a);
}

template<typename T>
__host__ __device__ static inline std::complex<T> floor_wrapper(std::complex<T> v) {
  return std::complex<T>(std::floor(v.real()), std::floor(v.imag()));
}

void floor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(ScalarType::Half, iter.dtype(), "floor_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return floor_wrapper(a);
    });
  });
}

template <typename scalar_t>
__host__ __device__ static inline scalar_t reciprocal_wrapper(scalar_t a) {
  return static_cast<scalar_t>(1)/a;
}

template<typename T>
__host__ __device__ static inline c10::complex<T> reciprocal_wrapper(c10::complex<T> v) {
  // Handle extreme cases for numpy compatibility
  auto both_inf = [](T real, T imag) {
    return (::isinf(real) && ::isinf(imag));
  };

  auto either_inf = [](T real, T imag) {
    return ::isinf(real) || ::isinf(imag);
  };

  auto either_nan = [](T real, T imag) {
    return ::isnan(real) || ::isnan(imag);
  };

  if (either_nan(v.real(), v.imag()) || both_inf(v.real(), v.imag())) {
    // If either is Nan or both are infinite, return {nan, nan}
    return {std::numeric_limits<T>::quiet_NaN(), std::numeric_limits<T>::quiet_NaN()};
  } else if (either_inf(v.real(), v.imag())) {
    // If either is Inf, return {0, 0}
    return {0, 0};
  }
  const c10::complex<T> one = c10::complex<T>(1.0, 0);
  return one/v;
}

void reciprocal_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "reciprocal_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return reciprocal_wrapper(a);
    });
  });
}

// We manually overload nearbyint because std::nearbyint does not work with std::complex types and ROCm.
template <typename scalar_t>
__host__ __device__ static inline scalar_t nearbyint_wrapper(scalar_t a) {
  return static_cast<scalar_t>(::nearbyintf(static_cast<float>(a)));
}

__host__ __device__ static inline double nearbyint_wrapper(double a) {
  return ::nearbyint(a);
}

__host__ __device__ static inline c10::complex<float> nearbyint_wrapper(c10::complex<float> a) {
  return c10::complex<float>(::nearbyintf(static_cast<float>(a.real())), ::nearbyintf(static_cast<float>(a.imag())));
}

__host__ __device__ static inline c10::complex<double> nearbyint_wrapper(c10::complex<double> a) {
  return c10::complex<double>(::nearbyint(static_cast<double>(a.real())), ::nearbyint(static_cast<double>(a.imag())));
}

void round_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(ScalarType::Half, iter.dtype(), "round_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      // We do not use std::round because we would like to round midway numbers to the nearest even integer.
      return nearbyint_wrapper(a);
    });
  });
}

// We manually overload trunc because std::trunc does not work with std::complex types and ROCm.
template <typename scalar_t>
__host__ __device__ static inline scalar_t trunc_wrapper(scalar_t a) {
  return static_cast<scalar_t>(::truncf(static_cast<float>(a)));
}

__host__ __device__ static inline double trunc_wrapper(double a) {
  return ::trunc(a);
}

__host__ __device__ static inline c10::complex<float> trunc_wrapper(c10::complex<float> a) {
  return c10::complex<float>(::truncf(static_cast<float>(a.real())), ::truncf(static_cast<float>(a.imag())));
}

__host__ __device__ static inline c10::complex<double> trunc_wrapper(c10::complex<double> a) {
  return c10::complex<double>(::trunc(static_cast<double>(a.real())), ::trunc(static_cast<double>(a.imag())));
}

void trunc_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(ScalarType::Half, iter.dtype(), "trunc_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return trunc_wrapper(a);
    });
  });
}

REGISTER_DISPATCH(ceil_stub, &ceil_kernel_cuda);
REGISTER_DISPATCH(frac_stub, &frac_kernel_cuda);
REGISTER_DISPATCH(floor_stub, &floor_kernel_cuda);
REGISTER_DISPATCH(reciprocal_stub, &reciprocal_kernel_cuda);
REGISTER_DISPATCH(round_stub, &round_kernel_cuda);
REGISTER_DISPATCH(trunc_stub, &trunc_kernel_cuda);

}} // namespace at::native
