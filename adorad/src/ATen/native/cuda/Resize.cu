#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/library.h>
#include <ATen/native/cuda/Resize.cuh>
#include <ATen/native/ResizeCommon.h>

namespace at {
namespace native {

Tensor& resize_cuda_(
    Tensor& self,
    IntArrayRef size,
    c10::optional<MemoryFormat> optional_memory_format) {
  if (self.has_names()) {
    return resize_named_tensor_(self, size, optional_memory_format);
  }
  auto* self_ = self.unsafeGetTensorImpl();
  resize_impl_cuda_(self_, size, /*strides=*/c10::nullopt);
  if (optional_memory_format.has_value()) {
    auto memory_format =
        optional_memory_format.value();
    TORCH_CHECK(
        memory_format != MemoryFormat::Preserve,
        "Unsupported memory format",
        memory_format);
    self_->empty_tensor_restride(memory_format);
  }
  return self;
}
} // namespace native
} // namespace at
