#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void logaddexp_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "logaddexp_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      if (::isinf(a) && a == b) {
        return a;
      }
      else {
        scalar_t m = ::max(a, b);
        return m + ::log((scalar_t)(1.0) + ::exp(-::abs(a - b)));
      }
    });
  });
}

void logaddexp2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "logaddexp2_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      if (::isinf(a) && a == b) {
        return a;
      }
      else {
        scalar_t m = ::max(a, b);
        return m + ::log2((scalar_t)(1.0) + ::pow((scalar_t)(2.0), -::abs(a - b)));
      }
    });
  });
}

REGISTER_DISPATCH(logaddexp_stub, &logaddexp_kernel_cuda);
REGISTER_DISPATCH(logaddexp2_stub, &logaddexp2_kernel_cuda);

}} // namespace at::native
