#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/LaunchUtils.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>

namespace at {
namespace native {
namespace {

#define MAX_THREADS 512

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest2d_out_frame(
    const scalar_t* idata,
    scalar_t* odata,
    const size_t nc,
    const size_t height1,
    const size_t width1,
    const size_t height2,
    const size_t width2,
    float height_scale,
    float width_scale) {
  size_t nc_iter = threadIdx.z + blockIdx.z * blockDim.z;
  int w2 = threadIdx.x + blockIdx.x * blockDim.x;
  int h2 = threadIdx.y + blockIdx.y * blockDim.y;

  if (w2 >= width2 || h2 >= height2) {
    return;
  }

  int nc_stride = blockDim.z * gridDim.z;

  const size_t h1 = height1 == height2
      ? h2
      : nearest_neighbor_compute_source_index(height_scale, h2, height1);
  const size_t w1 = width1 == width2
      ? w2
      : nearest_neighbor_compute_source_index(width_scale, w2, width1);

  size_t src_index = (nc_iter * height1 + h1) * width1 + w1;
  size_t src_index_stride = nc_stride * width1 * height1;
  size_t dst_index = (nc_iter * height2 + h2) * width2 + w2;
  size_t dst_index_stride = nc_stride * width2 * height2;

  // iterating over
  while (nc_iter < nc) {
    odata[dst_index] = idata[src_index];
    dst_index += dst_index_stride;
    src_index += src_index_stride;
    nc_iter += nc_stride;
  }
}

// see NOTE [ Nearest neighbor upsampling kernel implementation ]
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_nearest2d_backward_out_frame(
    const scalar_t* grad_o,
    size_t dim_b,
    size_t dim_c,
    size_t src_dim_h,
    size_t src_dim_w,
    size_t dst_dim_h,
    size_t dst_dim_w,
    scalar_t* grad_i,
    float height_scale,
    float width_scale) {
  int dst_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dst_idx >= dim_c * dst_dim_h * dst_dim_w)
    return;

  int dst_c_stride = dst_dim_h * dst_dim_w;
  int src_c_stride = src_dim_h * src_dim_w;

  int c = (dst_idx / (dst_c_stride)) % dim_c;

  int dst_y = (dst_idx / dst_dim_w) % dst_dim_h;
  int src_y =
      nearest_neighbor_bw_compute_source_index(height_scale, dst_y, src_dim_h);
  int src_y_up = nearest_neighbor_bw_compute_source_index(
      height_scale, dst_y + 1, src_dim_h + 1);

  int dst_x = dst_idx % dst_dim_w;
  int src_x =
      nearest_neighbor_bw_compute_source_index(width_scale, dst_x, src_dim_w);
  int src_x_up = nearest_neighbor_bw_compute_source_index(
      width_scale, dst_x + 1, src_dim_w + 1);

  for (int b = 0; b < dim_b; b++) {
    accscalar_t grad = 0;
    for (int y = src_y; y < src_y_up; y++) {
      for (int x = src_x; x < src_x_up; x++) {
        int src_idx =
            b * dim_c * src_c_stride + c * src_c_stride + y * src_dim_w + x;
        grad += grad_o[src_idx];
      }
    }
    grad_i[dst_idx] = grad;
    dst_idx += dim_c * dst_c_stride;
  }
}

static void upsample_nearest2d_out_cuda_template(
    Tensor& output,
    const Tensor& input_,
    IntArrayRef output_size,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg input_arg{input_, "input_", 1}, output_arg{output, "output", 2};
  checkAllSameGPU(
      "upsample_nearest2d_out_cuda_template", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_.size(0);
  int channels = input_.size(1);
  int input_height = input_.size(2);
  int input_width = input_.size(3);

  upsample_2d_shape_check(
      input_,
      Tensor(),
      nbatch,
      channels,
      input_height,
      input_width,
      output_height,
      output_width);

  AT_ASSERT(
      input_height > 0 && input_width > 0 && output_height > 0 &&
      output_width > 0);

  Tensor input = input_.contiguous();
  output.resize_({nbatch, channels, output_height, output_width});

  if (input.numel() == 0) {
    return;
  }

  int nc = nbatch * channels;

  const int max_threads = std::min<int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS);

  int* maxThreadsDim = at::cuda::getCurrentDeviceProperties()->maxThreadsDim;
  int* maxGridSize = at::cuda::getCurrentDeviceProperties()->maxGridSize;

  // upsample_2d_shape_check makes sure input/output tensor is not empty;
  int block_x = std::min<int>(
      maxThreadsDim[0], std::min<int>(lastPow2(output_width), max_threads));
  int block_y = std::min<int>(
      maxThreadsDim[1],
      std::min<int>(lastPow2(output_height), max_threads / block_x));
  int block_z = std::min<int>(
      maxThreadsDim[2], std::min<int>(nc, max_threads / block_x / block_y));
  const dim3 block(block_x, block_y, block_z);

  int grid_x = cuda::ATenCeilDiv(output_width, block_x);
  int grid_y = cuda::ATenCeilDiv(output_height, block_y);
  int grid_z = std::min<int>(
      maxGridSize[2], cuda::ATenCeilDiv(nc, block_z * 4));
  const dim3 grid(grid_x, grid_y, grid_z);
  // Error out on cases where grid_x & grid_y exceeds limit of launch config, as
  // the current kernel implementation doesn't loop over the two dimensions.
  // This is unlikely to happen.
  // TODO: kernel implementation could stride on spatial dimension. We probably
  //       need to overhaul the kernel.
  TORCH_CHECK(
      grid_x <= maxGridSize[0] && grid_y <= maxGridSize[1],
      "input tensor has spatial dimension larger than the kernel capacity");

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::Byte, input.scalar_type(), "upsample_nearest2d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.data_ptr<scalar_t>();
        auto odata = output.data_ptr<scalar_t>();

        const float height_scale = compute_scales_value<float>(scales_h, input_height, output_height);
        const float width_scale = compute_scales_value<float>(scales_w, input_width, output_width);

        upsample_nearest2d_out_frame<scalar_t, accscalar_t>
            <<<grid, block, 0, stream>>>(
                idata,
                odata,
                nc,
                input_height,
                input_width,
                output_height,
                output_width,
                height_scale,
                width_scale);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_nearest2d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      "upsample_nearest2d_backward_out_cuda",
      {grad_output_arg, grad_input_arg});

  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 4,
      "It is expected input_size equals to 4, but got size ",
      input_size.size());

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_height = input_size[2];
  int input_width = input_size[3];

  upsample_2d_shape_check(
      Tensor(),
      grad_output_,
      nbatch,
      channels,
      input_height,
      input_width,
      output_height,
      output_width);

  Tensor grad_output = grad_output_.contiguous();
  grad_input.resize_({nbatch, channels, input_height, input_width});

  if (grad_input.numel() == 0) {
    return;
  }

  // upsample_2d_shape_check makes sure `nbatch != 0`
  unsigned int n = grad_input.numel() / nbatch;
  dim3 bdim{std::min<unsigned int>(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, MAX_THREADS)};
  dim3 gdim{cuda::ATenCeilDiv(n, bdim.x)};
  // safe check for int32 indexing; implicitly restrict launch config for kernel
  TORCH_CHECK(grad_input.numel() <= std::numeric_limits<int32_t>::max());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::Byte, grad_output.scalar_type(), "upsample_nearest2d_backward_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.data_ptr<scalar_t>();
        auto odata = grad_output.data_ptr<scalar_t>();

        const float height_scale = compute_scales_value_backwards<float>(scales_h, output_height, input_height);
        const float width_scale = compute_scales_value_backwards<float>(scales_w, output_width, input_width);

        upsample_nearest2d_backward_out_frame<scalar_t, accscalar_t>
            <<<gdim, bdim, 0, stream>>>(
                odata,
                nbatch,
                channels,
                output_height,
                output_width,
                input_height,
                input_width,
                idata,
                height_scale,
                width_scale);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace

Tensor& upsample_nearest2d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  upsample_nearest2d_out_cuda_template(output, input, output_size, scales_h, scales_w);
  return output;
}

Tensor upsample_nearest2d_cuda(const Tensor& input, IntArrayRef output_size, c10::optional<double> scales_h, c10::optional<double> scales_w) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_nearest2d_out_cuda_template(output, input, output_size, scales_h, scales_w);
  return output;
}

Tensor& upsample_nearest2d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  upsample_nearest2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, scales_h, scales_w);
  return grad_input;
}

Tensor upsample_nearest2d_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  Tensor grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_nearest2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, scales_h, scales_w);
  return grad_input;
}

using at::native::upsample::compute_output_size;
using at::native::upsample_cuda::get_scale_value;

Tensor upsample_nearest2d_cuda(
    const Tensor& input,
    c10::optional<IntArrayRef> output_size,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto osize = compute_output_size(input.sizes(), output_size, scale_factors);
  auto scale_h = get_scale_value(scale_factors, 0);
  auto scale_w = get_scale_value(scale_factors, 1);
  upsample_nearest2d_out_cuda_template(output, input, osize, scale_h, scale_w);
  return output;
}

Tensor upsample_nearest2d_backward_cuda(
    const Tensor& grad_output,
    c10::optional<IntArrayRef> output_size,
    IntArrayRef input_size,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto osize = compute_output_size(input_size, output_size, scale_factors);
  auto scale_h = get_scale_value(scale_factors, 0);
  auto scale_w = get_scale_value(scale_factors, 1);
  auto grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_nearest2d_backward_out_cuda_template(
      grad_input, grad_output, osize, input_size, scale_h, scale_w);
  return grad_input;
}

} // namespace native
} // namespace at
