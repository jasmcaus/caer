#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

namespace at { namespace native {

void bernoulli_tensor_kernel(Tensor& self, const Tensor& p_, c10::optional<Generator> gen_) {
  auto generator = get_generator_or_default<CUDAGeneratorImpl>(gen_, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::bernoulli_kernel(self, p_, generator);
}

void bernoulli_scalar_kernel(Tensor& self, double p, c10::optional<Generator> gen) {
  auto iter = TensorIterator::nullary_op(self);
  auto generator = get_generator_or_default<CUDAGeneratorImpl>(gen, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::bernoulli_kernel(iter, p, generator);
}

REGISTER_DISPATCH(bernoulli_tensor_stub, &bernoulli_tensor_kernel);
REGISTER_DISPATCH(bernoulli_scalar_stub, &bernoulli_scalar_kernel);

}} // namespace at::native
