#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

namespace at { namespace native {

void log_normal_kernel(TensorIterator& iter, double mean, double std, c10::optional<Generator> gen) {
  auto generator = get_generator_or_default<CUDAGeneratorImpl>(gen, cuda::detail::getDefaultCUDAGenerator());
  at::native::templates::cuda::log_normal_kernel(iter, mean, std, generator);
}

REGISTER_DISPATCH(log_normal_stub, &log_normal_kernel);

}} // namespace at::native
