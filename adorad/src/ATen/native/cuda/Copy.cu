#include <ATen/ATen.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/native/Copy.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <THC/THC.h>

#ifdef __HIP_PLATFORM_HCC__
#include <hip/hip_version.h>
#endif

namespace at {
namespace native {

using namespace at::cuda;

// device-to-device copy, does type conversion
void copy_device_to_device(TensorIterator& iter, bool non_blocking) {
  int64_t numel = iter.numel();

  // We can memcpy the memory if both tensors have the same type AND both
  // tensors are contiguous after dimension coalescing and reordering.
  bool same_type = iter.dtype(0) == iter.dtype(1);
  bool memcpy_eligible = same_type && iter.is_contiguous();

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  CUDAGuard device_guard(src_device);

  // We always perform the copy on the source device, using the current stream
  // on the source device, and we fully synchronize on both src and dst's
  // current streams for completion of the copy. We have to explicitly do this
  // for non-contig copies. This mimics the behavior of cross-device
  // hipMemcpyAsync on the default stream.
  CUDAStream copy_stream = getCurrentCUDAStream(src_device.index());
  if (src_device != dst_device) {
    // This is a cross-device copy on the src current stream and dst current
    // stream. We perform a two-way barrier between both devices' streams
    // before the copy. This ensures that any write-after-write and
    // write-after-read dependencies on the destination side are handled, so
    // that no one is operating on the dst memory when we perform the copy.
    // src waits on dst barrier (src already waits on src)
    CUDAEvent dst_ready;
    device_guard.set_device(dst_device);
    dst_ready.record(getCurrentCUDAStream(dst_device.index()));

    device_guard.set_device(src_device);
    dst_ready.block(copy_stream);
  }

  if (memcpy_eligible) {
    void *dst = iter.data_ptr(0);
    void *src = iter.data_ptr(1);
    size_t size = numel * iter.element_size(0);
    if (src != dst || src_device != dst_device) {
      // Perform the copy
      AT_CUDA_CHECK(hipMemcpyAsync(
          dst, src, size,
          hipMemcpyDeviceToDevice,
          copy_stream));
    }
  } else {
    auto dtype = iter.dtype(0);
    if (isQIntType(dtype)) {
      AT_DISPATCH_QINT_TYPES(dtype, "copy_", [&] {
        gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
      });
    } else {
      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
          kHalf, kBool, kBFloat16, dtype, "copy_", [&] {
            gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) { return x; });
          });
    }
  }

  if (src_device != dst_device) {
    // dst waits on src barrier (dst already waits on dst). We cannot
    // operate on dst's copy until the copy is complete.

    // Still on src_device, record stream event
    CUDAEvent src_ready;
    src_ready.record(copy_stream);

    device_guard.set_device(dst_device);
    src_ready.block(getCurrentCUDAStream(dst_device.index()));
  }

  AT_CUDA_CHECK(hipGetLastError());
}

static bool copy_requires_temporaries(TensorIterator& iter, bool p2p_enabled) {
  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  if (dst_device == src_device) {
    // We never require temporaries for copies on the same GPU.
    TORCH_INTERNAL_ASSERT(dst_device.is_cuda() && src_device.is_cuda());
    return false;
  }

  bool same_dtype = iter.dtype(0) == iter.dtype(1);
  if (same_dtype && iter.is_contiguous()) {
    // Contiguous same-dtype copies can always use hipMemcpyAsync
    return false;
  } else if (dst_device.is_cuda() && src_device.is_cuda()) {
    // Copies between GPUs can use the copy kernel if P2P is supported
    return !p2p_enabled;
  } else {
    // The remaining cases require temporaries. For example, this includes
    // non-contiguous copies between CPU and GPU.
    return true;
  }
}

static bool maybe_enable_p2p_access(Device dst_device, Device src_device) {
  if (dst_device.is_cpu() || src_device.is_cpu()) {
    return false;
  }
  return THCState_getPeerToPeerAccess(
        globalContext().getTHCState(), src_device.index(), dst_device.index());
}

static void copy_kernel_cuda(TensorIterator& iter, bool non_blocking) {
  AT_ASSERT(iter.ntensors() == 2);

  Device dst_device = iter.device(0);
  Device src_device = iter.device(1);

  // Enable p2p access between devices. (No-op if it involves the CPU)
  bool p2p_enabled = maybe_enable_p2p_access(dst_device, src_device);

  if (copy_requires_temporaries(iter, p2p_enabled)) {
    // NB: this involves recursive calls to copy. Be careful that those copies
    // don't require temporaries or you will cause an infinite recursion!
    auto& dst = iter.tensor(0);
    Tensor dst_contig;
    Tensor src_contig;

    // Type conversions are performed on the CPU for CPU-GPU copies and on
    // the src device for GPU-GPU copies.
    if (iter.device_type(0) == kCUDA) {
      dst_contig = dst.is_contiguous() ? dst : at::empty_like(dst, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).to(iter.dtype(0)).expand_as(dst).contiguous();
    } else {
      bool same_type = iter.dtype(0) == iter.dtype(1);
      dst_contig = (dst.is_contiguous() && same_type) ? dst : at::empty_like(dst, iter.dtype(1), LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      src_contig = iter.tensor(1).expand_as(dst).contiguous();
    }

    // perform a same-dtype copy on contiguous tensors
    TORCH_INTERNAL_ASSERT(dst_contig.sizes().equals(src_contig.sizes()));
    TORCH_INTERNAL_ASSERT(dst_contig.scalar_type() == src_contig.scalar_type());
    dst_contig.copy_(src_contig, non_blocking);

    // if necessary, copy back into dst
    if (!dst_contig.is_same(dst)) {
      TORCH_INTERNAL_ASSERT(dst_contig.device() == dst.device());
      dst.copy_(dst_contig, non_blocking);
    }
    return;
  }

  // Copy on GPU (or between GPUs)
  if (dst_device.is_cuda() && src_device.is_cuda()) {
    copy_device_to_device(iter, non_blocking);
    return;
  }

  // Copy between CPU and GPU
  cuda::OptionalCUDAGuard device_guard;
  hipMemcpyKind kind;
  if (dst_device.is_cuda() && src_device.is_cpu()) {
    device_guard.set_device(dst_device);
    kind = hipMemcpyHostToDevice;
  } else if (dst_device.is_cpu() && src_device.is_cuda()) {
    device_guard.set_device(src_device);
    kind = hipMemcpyDeviceToHost;
  } else {
    TORCH_INTERNAL_ASSERT(false, "unsupported devices in GPU copy_()");
  }

  void* dst = iter.data_ptr(0);
  void* src = iter.data_ptr(1);
  int64_t nbytes = iter.numel() * iter.element_size(0);
  CUDAStream stream = getCurrentCUDAStream();

  if (non_blocking) {
    AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));
    void* ptr = (dst_device == kCPU ? dst : src);
    AT_CUDA_CHECK(THCCachingHostAllocator_recordEvent(ptr, stream));
  } else {
#if HIP_VERSION >= 301
    AT_CUDA_CHECK(hipMemcpyWithStream(dst, src, nbytes, kind, stream));
#else
    AT_CUDA_CHECK(hipMemcpyAsync(dst, src, nbytes, kind, stream));
    AT_CUDA_CHECK(hipStreamSynchronize(stream));
#endif
  }
}

REGISTER_DISPATCH(copy_stub, &copy_kernel_cuda);

} // namespace native
} // namespace at
