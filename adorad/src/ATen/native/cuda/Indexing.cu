#include "hip/hip_runtime.h"
#include <ATen/native/TensorAdvancedIndexing.h>
#include <ATen/native/IndexingUtils.h>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/ExpandUtils.h>
#include <ATen/MemoryOverlap.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDAUtils.h>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCGeneral.h>
#include <THC/THCTensorSort.cuh>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <THC/THCAtomics.cuh>

#include <hipcub/hipcub.hpp>

#include <c10/macros/Macros.h>

namespace {

template <typename scalar_t, int SZ>
__global__ void indexing_backward_kernel(
  int64_t* sorted_indices, int64_t* indices, scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim) {
//numel is total number of flattened indices, not expanded to dimensions that are not indexed.
//stride is the cumulative size of the not-indexed last dimensions
//stride_before is the stride of the dimension immediately preceding first indexed dimension
//if indexing starts from the 0th dimension, stride_before does not matter because blockIdx.z will be 0 in this case
//outer_dim is number of elements in the first unindexed dimensions
  using accscalar_t = at::acc_type<scalar_t, true>;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same destination index as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values processed by each thread (grain size)
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int64_t idx = blockIdx.x * blockDim.y + threadIdx.y;
    if (idx < numel
        && (idx == 0 || sorted_indices[idx] != sorted_indices[idx - 1])){
      do {
        int64_t start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        const int64_t weight_row = ((int64_t) sorted_indices[idx]) * stride + z * stride_before;
        const int64_t grad_row = ((int64_t) indices[idx]) * stride + z * numel * stride;
        const accscalar_t scale = (accscalar_t)1.0;

        accscalar_t gradient[SZ];
        accscalar_t weight[SZ];

        while (start_feature < stride) {
          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
              gradient[ii] = static_cast<accscalar_t>(grad_output[grad_row + feature_dim]);
              weight[ii] = static_cast<accscalar_t>(grad_weight[weight_row + feature_dim]);
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            weight[ii] += gradient[ii] * scale;
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
                grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(weight[ii]);
            }
          }
          start_feature += gridDim.y * blockDim.x * SZ;
        }

        idx++;
      } while (idx < numel && sorted_indices[idx] == sorted_indices[idx - 1]);
    }
  }
}


}


namespace at { namespace native {

static Tensor wrapIndexOnce(const Tensor & index, int64_t dim, int64_t dim_size, bool check_range=true) {
//we don't need to check range in backward - if there were out of bounds indices forward should already have errored out
  if (index.numel() != 0 && check_range) {
    auto max_idx = index.max().item<int64_t>();
    auto min_idx = index.min().item<int64_t>();
    if (max_idx >= dim_size) {
      TORCH_CHECK_INDEX(false, "index ", max_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
    if (min_idx < -dim_size) {
      TORCH_CHECK_INDEX(false, "index ", min_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
  }
  return index.remainder(dim_size);
}

static std::vector<int64_t> computeLinearStride(const Tensor & tensor) {
  // computes the stride as if tensor were contiguous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1, std::multiplies<int64_t>());
  return stride;
}

static std::tuple<Tensor, int64_t, int64_t, int64_t>
computeLinearIndex(const Tensor & src, TensorList indices, bool check_range) {
  auto strides = computeLinearStride(src);
  const auto& backend = src.type().backend();

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1, strideBefore =0;
  for (auto i = decltype(src.dim()){0}; i < src.dim(); i++) {
    if (indices[i].defined()) {
      // Cast index to the longType matching src's backend
      // This allows us to support ie indexing a cuda tensor with a cpu tensor
      Tensor index = (wrapIndexOnce(indices[i], i, src.size(i), check_range) * strides[i]).toBackend(backend);
      if (linearIndex.defined()) {
        linearIndex += index;
      } else {
        linearIndex = index;
        if (i>0) {
           strideBefore = src.stride(i-1); // stride after undefined dimensions
        }
      }
    } else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= src.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= src.size(i);
    }
  }

  return std::make_tuple(std::move(linearIndex), nElemBefore, strideBefore, nElemAfter);
}


static std::tuple<Tensor, Tensor, int64_t, int64_t, int64_t, std::vector<int64_t>> makeLinearIndex(Tensor self, TensorList orig, bool check_range) {
  checkIndexTensorTypes(orig);
  // first expand BoolTensor (masks) or ByteTensor (masks) into 1 or more LongTensors
  auto indices = expandTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t)self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  std::vector<int64_t> inversePerm;
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices, inversePerm) = transposeToFrontAndInvPerm(self, indices);
  }
  int64_t nElemBefore, strideBefore, nElemAfter;
  Tensor linearIndex;
  std::tie(linearIndex, nElemBefore, strideBefore, nElemAfter) = computeLinearIndex(self, indices, check_range);
  return std::make_tuple(linearIndex, self, nElemBefore, strideBefore, nElemAfter, inversePerm);
}


namespace {
void index_put_accum_kernel(Tensor & self, TensorList indices, const Tensor & value, bool unsafe) {
  if (indices.size() > (size_t)self.dim()) {
    TORCH_CHECK_INDEX(false, "too many indices for tensor of dimension ", self.dim(), " (got ", indices.size(), ")");
  }
  auto value_ = value.contiguous();
  Tensor linearIndex, expandedValue, src;
  int64_t nElemBefore, strideBefore, sliceSize;
  std::vector<int64_t> inversePerm;
  std::tie(linearIndex, src, nElemBefore, strideBefore, sliceSize, inversePerm) = makeLinearIndex(self, indices, !unsafe);
  int64_t num_indices = linearIndex.numel();
  if (num_indices > 0 && sliceSize > 0) {
      const bool permuted = !src.is_contiguous();
      auto src_ = permuted ? src.contiguous() : src;
      linearIndex = linearIndex.reshape(-1);
      auto sorted_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      auto orig_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      using device_ptr = thrust::device_ptr<int64_t>;
      const hipStream_t stream = at::cuda::getCurrentCUDAStream();

      linearIndex.floor_divide_(sliceSize);
      {
      sorted_indices.copy_(linearIndex);
      auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
      auto policy = thrust::cuda::par(allocator).on(stream);

      // Fill sortedOrigIndices with sequential indices
      const auto count_iter = thrust::counting_iterator<int64_t>(0);
      auto orig_data = device_ptr(orig_indices.data_ptr<int64_t>());
      thrust::copy(policy, count_iter, count_iter + num_indices, orig_data);

      // Sort the inputs into sorted with the corresponding indices; we
      // don't need a stable or multidimensional sort, so just use Thrust
      // directly
      // Sort; a stable sort is not required
      // NB - not passing comparator causes thrust to use radix sort, and it hurts perf A LOT, at least for medium (few K) sized indices
      auto sorted_data = device_ptr(sorted_indices.data_ptr<int64_t>());
      thrust::sort_by_key(policy, sorted_data, sorted_data + num_indices, orig_data, ThrustLTOp<int64_t>());
      }
      TORCH_INTERNAL_ASSERT(linearIndex.numel()*sliceSize*nElemBefore == value.numel(), "number of flattened indices did not match number of elements in the value tensor", linearIndex.numel()*sliceSize*nElemBefore, value.numel());
      const int UNROLL = 4;
      const int indices_per_block = 4;
      dim3 grid(THCCeilDiv(num_indices, (int64_t) indices_per_block),
           std::min<int>(at::cuda::getCurrentDeviceProperties()->maxGridSize[1], THCCeilDiv(sliceSize, (int64_t) (C10_WARP_SIZE*UNROLL))),
           std::min(std::max<int>(1,nElemBefore), at::cuda::getCurrentDeviceProperties()->maxGridSize[2]));
      dim3 block(C10_WARP_SIZE, indices_per_block);

      AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      value_.scalar_type(), "indexing_backward", [&] {
      indexing_backward_kernel<scalar_t, UNROLL><<<grid, block, 0, stream>>>(
        sorted_indices.data_ptr<int64_t>(),
        orig_indices.data_ptr<int64_t>(),
        value_.data_ptr<scalar_t>(),
        src_.data_ptr<scalar_t>(),
        num_indices,
        sliceSize,
        strideBefore,
        nElemBefore);
      });
      C10_CUDA_KERNEL_LAUNCH_CHECK();
      if (permuted)
          self.copy_(src_.permute(inversePerm));
  }
}

REGISTER_CUDA_DISPATCH(index_put_accum_stub, &index_put_accum_kernel);
} //anonymous


// Check tensor dimensions for index operations, and return the slice size.
static ptrdiff_t getSliceSize(const Tensor & dst,
                              int dim,
                              const Tensor & index,
                              const Tensor & src)
{
  int dstDims = dst.dim();
  int srcDims = src.dim();

  TORCH_CHECK(index.dim() <= 1, "Index must be vector or scalar");

  ptrdiff_t dstSliceSize = 1;
  TORCH_CHECK(dim >= 0 && dim < dstDims, "Indexing dim ", dim, " is out of bounds");
  for (int d = 0; d < dstDims; d++) {
    if (d != dim) {
      dstSliceSize *= dst.size(d);
    }
  }

  TORCH_CHECK(dim < srcDims, "Indexing dim ", dim, " is out of bounds");
  TORCH_CHECK(index.numel() == src.size(dim),
             "length of src.size[dim] is not equal to length of indices");

  ptrdiff_t srcSliceSize = 1;
  bool mismatch = false;

  if (dstDims != srcDims) mismatch = true;

  for (int d = 0; d < srcDims; d++) {
    if (d != dim) {
      srcSliceSize *= src.size(d);
      if (!mismatch && dst.size(d) != src.size(d)) mismatch = true;
    }
  }

  TORCH_CHECK(dstSliceSize == srcSliceSize,
             "Source/destination tensor have different slice sizes (%ld vs %ld)",
             dstSliceSize, srcSliceSize);

  if (mismatch) {
    TORCH_WARN_ONCE(
        "Warning: source/destination slices have same size but different "
        "shape for an index operation.  This behavior is deprecated.\n");
  }

  return dstSliceSize;
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexAddLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexAddSmallIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                   cuda::detail::TensorInfo<T, IndexType> src,
                                   cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType innerSize,
                                   int64_t dstAddDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType srcIndex = 0; srcIndex < indices.sizes[0]; ++srcIndex) {
    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
          cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstAddDim];

      IndexType srcOffset =
          cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcAddDim];

      gpuAtomicAdd(&dst.data[dstOffset], src.data[srcOffset]);
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexAddSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexAddLargeIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                   cuda::detail::TensorInfo<T, IndexType> src,
                                   cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType totalSize,
                                   IndexType innerSize,
                                   int64_t dstAddDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType srcIndex, elementInSlice;
    if (IndexIsMajor) {
      srcIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      srcIndex = linearIndex % innerSize;
    }

    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    IndexType dstOffset =
      cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstAddDim];

    IndexType srcOffset =
      cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcAddDim];

    gpuAtomicAdd(&dst.data[dstOffset], src.data[srcOffset]);
  }
}

// Compare the stride between adjacent slices (sliceStride) with strides in the
// other dimensions (i.e., strides *inside* each slice).
//
// - Returns true if some dimension inside the slice has lower stride than
//   sliceStride.  The simplest example is a 2-D contiguous tensor with sliceDim
//   == 0 (that is, each slice is a row).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "index-major order".  For example, if thread count equals slice size, then
//   all threads process slice #0 in lockstep, and then slice #1, and so on.
//
// - Otherwise (i.e., sliceStride has the lowest value), this function returns
//   false.  The simplest example is a 2-D contiguous tensor with sliceDim == 1
//   (each slice is a column).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "elementInSlice-major order".  For example, each thread can process element
//   #0 of every slice, and then element #1 of every slice, and so on.
template <typename scalar_t>
bool indexShouldBeMajor(cuda::detail::TensorInfo<scalar_t, unsigned int> &info,
                                    int sliceDim)
{
  // The stride between adjacent slices (e.g., between element #0 of slice #100
  // and element #0 of slice #101).
  unsigned int sliceStride = info.strides[sliceDim];

  for (int i = 0; i < info.dims; ++i) {
    if (i != sliceDim && info.sizes[i] > 1 && info.strides[i] < sliceStride) {
      return true;
    }
  }

  return false;
}

Tensor& index_add_cuda_(Tensor & self, int64_t dim, const Tensor & index, const Tensor & source) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("index_add_cuda_");
  dim = maybe_wrap_dim(dim, self.dim());

  TensorArg self_arg{self, "self", 1}, index_arg{index, "index", 3}, source_arg{source, "source", 4};
  checkAllSameGPU("index_add", {self_arg, index_arg, source_arg});

  TORCH_CHECK_INDEX(index.dim() <= 1, "index_add_(): Index is supposed to be a vector");
  TORCH_CHECK(index.scalar_type() == ScalarType::Long || index.scalar_type() == ScalarType::Int, "index_add_(): Expected dtype int32/int64 for index");
  TORCH_CHECK(self.scalar_type() == source.scalar_type(),
              "index_add_(): self and source must have the same scalar type");
  TORCH_CHECK(dim == 0 || dim < source.dim(),
              "index_add_(): Indexing dim ", dim, " is out of bounds of tensor");
  TORCH_CHECK(index.numel() == (source.dim() == 0 ? 1 : source.size(dim)),
              "index_add_(): Number of indices should be equal to self.size(dim)");

  at::assert_no_internal_overlap(self);
  at::assert_no_overlap(self, index);
  at::assert_no_overlap(self, source);

  // Scalars are treated as 1-d tensor
  Tensor self_ = (self.dim() == 0) ? self.view(1) : self;
  Tensor source_ = (source.dim() == 0) ? source.view(1) : source;

  TORCH_CHECK(self.dim() <= MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);
  TORCH_CHECK(source.dim() <= MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);
  TORCH_CHECK(index.dim() <= MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);

  at::assert_no_internal_overlap(self);
  at::assert_no_partial_overlap(self, index);
  at::assert_no_partial_overlap(self, source);

  // The `source` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of index we are choosing, which is the total size
  // of the tensor `index`.
  ptrdiff_t sliceSize = getSliceSize(self_, dim, index, source_);
  ptrdiff_t sourceTotalSize = source.numel();
  int64_t selfAddDimSize = self_.size(dim);
  ptrdiff_t numIndex = index.numel();

  if (sliceSize == 0) {
    return self;
  }
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  bool indContig = index.is_contiguous();

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM)  \
  indexAddSmallIndex<TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                                \
      selfInfo, sourceInfo, indexInfo,                                               \
      selfAddDim, sourceAddDim, sliceSize, selfAddDimSize);                          \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE,                        \
                    SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR)            \
  indexAddLargeIndex<TENSOR_TYPE, INDICES_TYPE, TYPE,                       \
                     SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR>           \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                       \
      selfInfo, sourceInfo, indexInfo,                                      \
      selfAddDim, sourceAddDim, sourceTotalSize,                            \
      (IDX_IS_MAJOR) ? sliceSize : numIndex,                                \
      selfAddDimSize);                                                      \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(sourceTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(sourceTotalSize, (ptrdiff_t)128));

  if (cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(source) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_add", [&] {
      cuda::detail::TensorInfo<scalar_t, unsigned int> selfInfo =
          cuda::detail::getTensorInfo<scalar_t, unsigned int>(self_);
      int selfAddDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfAddDim);
      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_add_cuda_", [&] () {
        auto sourceInfo =
          cuda::detail::getTensorInfo<scalar_t, unsigned int>(source_);
        int sourceAddDim = sourceInfo.collapseDims(dim);
        sourceInfo.reduceDim(sourceAddDim);

        auto indexInfo =
        cuda::detail::getTensorInfo<index_t, unsigned int>(index);
        indexInfo.collapseDims();

        // A reasonable choice for when to have each thread iterate over
        // index to choose
        if (numIndex <= 16) {
          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2);
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2);
          } else {
            SMALL_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1);
          }
        } else {
          bool indexIsMajor = indexShouldBeMajor(selfInfo, selfAddDim);

          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2, true);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, false);
            }
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, false);
            }
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, true);
          }
        }
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_add", [&] {
      cuda::detail::TensorInfo<scalar_t, uint64_t> selfInfo =
        cuda::detail::getTensorInfo<scalar_t, uint64_t>(self_);
      int selfAddDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfAddDim);

      cuda::detail::TensorInfo<scalar_t, uint64_t> sourceInfo =
        cuda::detail::getTensorInfo<scalar_t, uint64_t>(source_);
      int sourceAddDim = sourceInfo.collapseDims(dim);
      sourceInfo.reduceDim(sourceAddDim);

      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_add_cuda_", [&] () {
        cuda::detail::TensorInfo<index_t, uint64_t> indexInfo =
          cuda::detail::getTensorInfo<index_t, uint64_t>(index);
        indexInfo.collapseDims();

        LARGE_INDEX(scalar_t, index_t, uint64_t, -1, -1, -1, true);
      });
    });
  }

  return self;
#undef SMALL_INDEX
#undef LARGE_INDEX
}

namespace {
// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexSelectLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexSelectSmallIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                      cuda::detail::TensorInfo<T, IndexType> src,
                                      cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType innerSize,
                                      int64_t srcSelectDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType dstIndex = 0; dstIndex < indices.sizes[0]; ++dstIndex) {
    IndexType srcIndex =
      indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(srcIndex < srcSelectDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
        cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstSelectDim];

      IndexType srcOffset =
        cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcSelectDim];

      dst.data[dstOffset] = src.data[srcOffset];
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexSelectSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexSelectLargeIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                      cuda::detail::TensorInfo<T, IndexType> src,
                                      cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType totalSize,
                                      IndexType innerSize,
                                      int64_t srcSelectDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType dstIndex, elementInSlice;
    if (IndexIsMajor) {
      dstIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      dstIndex = linearIndex % innerSize;
    }

    IndexType srcIndex =
      indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(srcIndex < srcSelectDimSize);

    IndexType dstOffset =
      cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstSelectDim];

    IndexType srcOffset =
      cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcSelectDim];

    dst.data[dstOffset] = src.data[srcOffset];
  }
}

namespace {

// When using a 0-dim scalar tensor, we need the legacy (THC) semantics of
// TensorInfo: Pretend that the scalar tensor is in fact a one-element vector.
template <typename T, typename IndexType>
cuda::detail::TensorInfo<T, IndexType>
tensorInfoLegacyIfScalar(cuda::detail::TensorInfo<T, IndexType> ti) {
  if (ti.dims == 0) {
    ti.dims = 1;
    ti.sizes[0] = 1;
    ti.strides[0] = 1;
  }
  return ti;
}

}

template<typename scalar_t>
void index_select_out_cuda_impl(Tensor& out, const Tensor& self, long dim,
                                const Tensor& index) {
  ptrdiff_t numIndices = index.numel();

  int selfDims = self.dim() == 0 ? 1 : self.dim();

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(index.dim() <= 1,
             "Index is supposed to be an empty tensor or a vector");
  TORCH_CHECK(dim < selfDims, "Indexing dim is out of bounds");

  std::vector<int64_t> newSize = self.sizes().vec();
  if (self.dim() > 0) {
    newSize[dim] = numIndices;
  }
  at::native::resize_(out, newSize, {});

  ptrdiff_t outTotalSize = out.numel();
  if (outTotalSize == 0) {
    return;
  }

  bool indContig = index.is_contiguous();

  // The `self` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  int64_t selfSelectDimSize = self.dim() == 0 ? 1 : self.size(dim);
  ptrdiff_t sliceSize = outTotalSize / numIndices;

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)         \
  indexSelectSmallIndex<TENSOR_TYPE, INDICES_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>     \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                                   \
      outInfo, selfInfo, indicesInfo,                                                   \
      outSelectDim, selfSelectDim, static_cast<TYPE>(sliceSize),                        \
      selfSelectDimSize);                                                               \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE,                           \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)                   \
  indexSelectLargeIndex<TENSOR_TYPE, INDICES_TYPE, TYPE,                       \
                        DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR>               \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                          \
      outInfo, selfInfo, indicesInfo,                                          \
      outSelectDim, selfSelectDim, static_cast<TYPE>(outTotalSize),            \
      static_cast<TYPE>((IDX_IS_MAJOR) ? sliceSize : numIndices),              \
      selfSelectDimSize);                                                      \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(outTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(outTotalSize, (ptrdiff_t)128));
  if (cuda::detail::canUse32BitIndexMath(out) &&
      cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    auto outInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, unsigned int>(out));
    int outSelectDim = outInfo.collapseDims(dim);
    outInfo.reduceDim(outSelectDim);

    auto  selfInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, unsigned int>(self));
    int selfSelectDim = selfInfo.collapseDims(dim);
    selfInfo.reduceDim(selfSelectDim);

    AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_select_out_cuda_impl", [&] () {
      auto indicesInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<index_t, unsigned int>(index));
      indicesInfo.collapseDims();

      // A reasonable choice for when to have each thread iterate over
      // indices to choose
      if (numIndices <= 16) {
        if (outInfo.dims == 1 && selfInfo.dims == 1 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2);
        } else if (outInfo.dims == 2 && selfInfo.dims == 2 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2);
        } else if (outInfo.dims == 3 && selfInfo.dims == 3 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2);
        } else {
          SMALL_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1);
        }
      } else {
        bool indexIsMajor = indexShouldBeMajor(outInfo, outSelectDim);

        if (outInfo.dims == 1 && selfInfo.dims == 1 && indContig) {
          LARGE_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2, true);
        } else if (outInfo.dims == 2 && selfInfo.dims == 2 && indContig) {
          if (indexIsMajor) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, true);
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, false);
          }
        } else if (outInfo.dims == 3 && selfInfo.dims == 3 && indContig) {
          if (indexIsMajor) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, true);
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, false);
          }
        } else {
          LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, true);
        }
      }
    });
  } else {
    auto outInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, uint64_t>(out));
    int outSelectDim = outInfo.collapseDims(dim);
    outInfo.reduceDim(outSelectDim);

    auto selfInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, uint64_t>(self));
    int selfSelectDim = selfInfo.collapseDims(dim);
    selfInfo.reduceDim(selfSelectDim);
    AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_select_out_cuda_impl", [&] () {
      auto indicesInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<index_t, uint64_t>(index));
      indicesInfo.collapseDims();

      LARGE_INDEX(scalar_t, index_t, uint64_t, -1, -1, -1, true);
    });
  }
#undef SMALL_INDEX
#undef LARGE_INDEX
}
} // anonymous namespace

Tensor& index_select_out_cuda(Tensor& out, const Tensor& self, int64_t dim,
                              const Tensor& index) {
  static constexpr string_view DIM_WARNING =
    "Tensor too large or too many (> 25) dimensions";

  TORCH_CHECK(at::cuda::check_device({out, self, index}),
              "Input, output and indices must be on the current device");
  at::assert_no_internal_overlap(out);
  at::assert_no_overlap(out, self);
  at::assert_no_overlap(out, index);

  dim = at::maybe_wrap_dim(dim, self);
  TORCH_CHECK(self.dim() <= MAX_TENSORINFO_DIMS, DIM_WARNING);
  TORCH_CHECK(index.dim() <= MAX_TENSORINFO_DIMS, DIM_WARNING);

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      out.scalar_type(), "index_select_cuda",
      [&] { index_select_out_cuda_impl<scalar_t>(out, self, dim, index); });

  return out;
}

Tensor index_select_cuda(const Tensor& self, int64_t dim, const Tensor& index) {
  Tensor out = at::empty({0}, self.options());
  index_select_out_cuda(out, self, dim, index);
  return out;
}

template<typename T>
struct NonZeroOp
{
    __host__ __device__ __forceinline__ bool operator()(const T& a) const {
      return (a!=T(0));
    }
};

template<typename scalar_t>
void nonzero_cuda_out_impl(const Tensor& self, Tensor& out){
  Tensor self_ = self.contiguous();
  int N = self_.numel();
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
// compute number of nonzero elements
  size_t temp_storage_bytes=0;
  auto& allocator = *c10::cuda::CUDACachingAllocator::get();
  auto num_nonzeros = allocator.allocate(sizeof(int));
  hipcub::TransformInputIterator<bool, NonZeroOp<scalar_t>, scalar_t*> itr(self_.data_ptr<scalar_t>(), NonZeroOp<scalar_t>());
  hipcub::DeviceReduce::Sum(nullptr, temp_storage_bytes, itr, (int*)num_nonzeros.get(), N, stream);
  auto temp_storage = allocator.allocate(temp_storage_bytes);
  hipcub::DeviceReduce::Sum(temp_storage.get(), temp_storage_bytes, itr, (int*)num_nonzeros.get(), N, stream);
  int num_nonzeros_h;
  C10_CUDA_CHECK(hipMemcpyAsync(&num_nonzeros_h, num_nonzeros.get(), sizeof(int), hipMemcpyDeviceToHost, stream));
  //need to synchronize to make sure data is available on the host
  C10_CUDA_CHECK(hipStreamSynchronize(stream));
  //expected output size is num_nonzeros x ndim
  //we are producing output with size {num_nonzeros, ndim} and strides {num_nonzeros, 1} (that is, transposed ndim x num_nonzeros output)
  //we are able to directly use passed output with this size and strides, and we can also (per contract)
  //resize passed output with incorrect sizes anyway we want.
  //However, out with correct sizes and incorrect strides will have to be copied to from the intermediate we've produced.
  bool need_to_copy = out.dim() == 2 && out.sizes()[0] == num_nonzeros_h && out.sizes()[1] == self.dim() && !out.t().is_contiguous();
  at::Tensor out_temp = need_to_copy ?
    at::native::empty_cuda({self.dim(), num_nonzeros_h}, optTypeMetaToScalarType(out.options().dtype_opt()),
                           out.options().layout_opt(), out.options().device_opt(), out.options().pinned_memory_opt()) :
    out.resize_({self.dim(), num_nonzeros_h});
  //Scalars are expected to produce output of size (1,0), so we can't write to it
  if (self.dim() > 0) {
    hipcub::CountingInputIterator<int64_t> counting_itr(0);
    temp_storage_bytes = 0;
    hipcub::DeviceSelect::Flagged(nullptr, temp_storage_bytes, counting_itr, itr,
      out_temp.data_ptr<int64_t>(), (int*)num_nonzeros.get(), N, stream);
    temp_storage = allocator.allocate(temp_storage_bytes);
    hipcub::DeviceSelect::Flagged(temp_storage.get(), temp_storage_bytes, counting_itr, itr,
      out_temp.data_ptr<int64_t>(), (int*)num_nonzeros.get(), N, stream);
    if (num_nonzeros_h > 0 && self.dim() > 1){
        int64_t div = 1;
        auto thrust_allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
        for (int dim = self.dim()-1; dim >= 0; dim--){
            int64_t dim_size = self.sizes()[dim];
            thrust::transform(
              thrust::cuda::par(thrust_allocator).on(stream),
              thrust::device_ptr<int64_t>(out_temp.data_ptr<int64_t>()),
              thrust::device_ptr<int64_t>(out_temp.data_ptr<int64_t>()) + num_nonzeros_h,
              thrust::device_ptr<int64_t>(out_temp.data_ptr<int64_t>()) + num_nonzeros_h * dim,
              [=] C10_HOST_DEVICE (const int64_t val) {return (val/div) % dim_size;}
            );
            div *= dim_size;
        }
    }
  }
  if (need_to_copy) {
    out.copy_(out_temp.t());
  } else {
    //transpose out so it is correct size
    Tensor out_ = out_temp.t();
    out.set_(out_);
  }
}

Tensor& nonzero_out_cuda(Tensor& out, const Tensor& self){
  TORCH_CHECK(self.numel() < std::numeric_limits<int>::max(), "nonzero is not supported for tensors with more than INT_MAX elements, \
  file a support request");
  TORCH_CHECK(out.dtype() == at::kLong, "Expected object of scalar type ", at::kLong, " as out, but got ", out.dtype());
  TORCH_CHECK(self.device() == out.device(), "expected self and out to be on the same device, but got out on ",
  out.device(), " and self on ", self.device());
  AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Bool, at::ScalarType::BFloat16, at::ScalarType::Half,
    self.scalar_type(), "nonzero_cuda",
    [&] {nonzero_cuda_out_impl<scalar_t>(self, out);});
  return out;
}

Tensor nonzero_cuda(const Tensor& self){
  Tensor out = at::native::empty_cuda({0}, kLong, self.options().layout_opt(), self.options().device_opt(), self.options().pinned_memory_opt());
  return nonzero_out_cuda(out, self);
}


} // native
} // at
