#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void smooth_l1_kernel_cuda(TensorIterator& iter, double beta) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "smooth_l1_cuda", [&iter, beta]() {
    scalar_t beta_val(beta);
    gpu_kernel(iter, [beta_val] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      auto z = ::abs(a - b);
      return z < beta_val ? scalar_t(0.5) * z * z / beta_val : z - scalar_t(0.5) * beta_val;
    });
  });
}


void mse_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "mse_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto diff = a - b;
      return diff * diff;
    });
  });
}

REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda);

// DO NOT ADD ANY NEW KERNELS HERE
// CUDA compilation times grow quickly.  It's perfectly acceptable to have a file per kernel.

}} // namespace at::native
