#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
// keeping THC headers for gpuAtomicAdd
#include <THC/THCAtomics.cuh>

#include <thrust/pair.h>

namespace at {
namespace native {
namespace {

using at::cuda::detail::canUse32BitIndexMath;

__device__
inline thrust::pair<int64_t, int64_t> get_index_mapping1d(
    int64_t input_w, int64_t output_w,
    int64_t output_x,
    int64_t pad_l) {
  // 3D grid of 1D blocks
  auto input_offset =
    (blockIdx.y + blockIdx.z * gridDim.y) * input_w;
  auto output_offset =
    (blockIdx.y + blockIdx.z * gridDim.y) * output_w;

  auto i_start_x = ::max(int64_t(0), -pad_l);
  auto o_start_x = ::max(int64_t(0), pad_l);

  int64_t input_x = ::abs(output_x - pad_l)
                    - ::abs(output_x - (input_w + pad_l - 1))
                    - output_x
                    + 2 * pad_l + input_w - 1
                    - o_start_x + i_start_x;

  return thrust::make_pair<int64_t, int64_t>(
    input_offset + input_x, output_offset + output_x);
}


__device__
inline thrust::pair<int64_t, int64_t>  get_index_mapping2d(
    int64_t input_dim_x, int64_t input_dim_y,
    int64_t output_dim_x, int64_t output_dim_y,
    int64_t pad_l, int64_t pad_t,
    int64_t output_xy) {
  // 3D grid of 1D blocks
  auto input_offset =
    (blockIdx.y + blockIdx.z * gridDim.y) * input_dim_x * input_dim_y;
  auto output_offset =
    (blockIdx.y + blockIdx.z * gridDim.y) * output_dim_x * output_dim_y;

  auto output_x = output_xy % output_dim_x;
  auto output_y = output_xy / output_dim_x;

  auto i_start_x = ::max(int64_t(0), -pad_l);
  auto i_start_y = ::max(int64_t(0), -pad_t);
  auto o_start_x = ::max(int64_t(0), pad_l);
  auto o_start_y = ::max(int64_t(0), pad_t);

  auto input_x = ::abs(output_x - pad_l)
                 - ::abs(output_x - (input_dim_x + pad_l - 1))
                 - output_x
                 + 2 * pad_l + input_dim_x - 1
                 - o_start_x + i_start_x;

  auto input_y = ::abs(output_y - pad_t)
                 - ::abs(output_y - (input_dim_y + pad_t - 1))
                 - output_y
                 + 2 * pad_t + input_dim_y - 1
                 - o_start_y + i_start_y;

  return thrust::make_pair<int64_t, int64_t>(
    input_offset + input_y * input_dim_x + input_x,
    output_offset + output_y * output_dim_x + output_x);
}

template<typename scalar_t>
__global__ void reflection_pad1d_out_kernel(
    scalar_t * input, scalar_t * output,
    int64_t input_w,
    int64_t pad_l, int64_t pad_r) {
  auto output_x = threadIdx.x + blockIdx.x * blockDim.x;
  auto output_w = input_w + pad_l + pad_r;

  if (output_x < output_w) {
    auto index_pair = get_index_mapping1d(input_w, output_w, output_x, pad_l);
    output[index_pair.second] = input[index_pair.first];
  }
}

template <typename scalar_t>
__global__ void reflection_pad1d_backward_out_kernel(
    scalar_t * grad_input, scalar_t * grad_output,
    int64_t input_w,
    int64_t pad_l, int64_t pad_r) {
  auto output_x = threadIdx.x + blockIdx.x * blockDim.x;
  auto output_w = input_w + pad_l + pad_r;

  if (output_x < output_w) {
    auto index_pair = get_index_mapping1d(input_w, output_w, output_x, pad_l);
    gpuAtomicAdd(
      &grad_input[index_pair.first], grad_output[index_pair.second]);
  }
}

template<typename scalar_t>
__global__ void reflection_pad2d_out_kernel(
    scalar_t * input, scalar_t * output,
    int64_t input_dim_x, int64_t input_dim_y,
    int pad_t, int pad_b, int pad_l, int pad_r) {
  auto output_xy = threadIdx.x + blockIdx.x * blockDim.x;
  auto output_dim_x = input_dim_x + pad_l + pad_r;
  auto output_dim_y = input_dim_y + pad_t + pad_b;

  if (output_xy < output_dim_x * output_dim_y) {
    auto index_pair = get_index_mapping2d(
      input_dim_x, input_dim_y,
      output_dim_x, output_dim_y,
      pad_l, pad_t,
      output_xy);

    output[index_pair.second] = input[index_pair.first];
  }
}

template <typename scalar_t>
__global__ void reflection_pad2d_backward_out_kernel(
    scalar_t * grad_input, scalar_t * grad_output,
    int64_t input_dim_x, int64_t input_dim_y,
    int pad_t, int pad_b, int pad_l, int pad_r) {
  auto output_xy = threadIdx.x + blockIdx.x * blockDim.x;
  auto output_dim_x = input_dim_x + pad_l + pad_r;
  auto output_dim_y = input_dim_y + pad_t + pad_b;

  if (output_xy < output_dim_x * output_dim_y) {
    auto index_pair = get_index_mapping2d(
      input_dim_x, input_dim_y,
      output_dim_x, output_dim_y,
      pad_l, pad_t,
      output_xy);

    gpuAtomicAdd(&grad_input[index_pair.first], grad_output[index_pair.second]);
  }
}

void reflection_pad1d_out_template(
    Tensor &output, const Tensor &input_, IntArrayRef padding) {
  TORCH_CHECK(canUse32BitIndexMath(input_),
    "input tensor must fit into 32-bit index math");

  int64_t dim_plane = 0;
  int64_t dim_w = 1;
  int64_t nbatch = 1;

  TORCH_CHECK(
      (input_.ndimension() == 2 && input_.size(1) != 0) ||
      (input_.ndimension() == 3 && input_.size(1) != 0 && input_.size(2) != 0),
      "2D or 3D (batch mode) tensor expected for input, but got: ", input_);

  if (input_.ndimension() == 3) {
    nbatch = input_.size(0);
    dim_plane++;
    dim_w++;
  }

  int64_t pad_l = padding[0];
  int64_t pad_r = padding[1];

  int64_t nplane = input_.size(dim_plane);
  int64_t input_w = input_.size(dim_w);
  int64_t output_w  = input_w + pad_l + pad_r;

  TORCH_CHECK(pad_l < input_w && pad_r < input_w, "Padding size should be less "
    "than the corresponding input dimension, but got: padding (",  pad_l, ", ",
    pad_r, ") at dimension ", dim_w, " of input ", input_);

  TORCH_CHECK(output_w >= 1,
    "input (W: ", input_w, ")is too small. Calculated output W: ", output_w);

  if (input_.ndimension() == 2) {
    output.resize_({nplane, output_w});
  } else {
    output.resize_({nbatch, nplane, output_w});
  }
  if (output.numel() == 0) {
    return;
  }

  dim3 block_size(output_w > 256 ? 256 : output_w);
  dim3 grid_size((int) ::ceil(output_w / 256.0), nplane, nbatch);

  Tensor input = input_.contiguous();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    input.scalar_type(), "reflection_pad1d_out_template", [&] {
      reflection_pad1d_out_kernel<<<
        grid_size, block_size, 0, at::cuda::getCurrentCUDAStream()>>>(
          input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
          input_w, pad_l, pad_r);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  );
}

void reflection_pad1d_backward_out_template(
    Tensor & grad_input, const Tensor & grad_output_,
    const Tensor & input, IntArrayRef padding) {

  if (grad_input.numel() == 0) {
    return;
  }

  TORCH_CHECK(canUse32BitIndexMath(input),
    "input tensor must fit into 32-bit index math");

  TORCH_CHECK(canUse32BitIndexMath(grad_output_),
    "input tensor must fit into 32-bit index math");

  int64_t dim_plane = 0;
  int64_t dim_w = 1;
  int64_t nbatch = 1;

  if (input.ndimension() == 3) {
    nbatch = input.size(0);
    dim_plane++;
    dim_w++;
  }

  int64_t pad_l = padding[0];
  int64_t pad_r = padding[1];

  int64_t nplane = input.size(dim_plane);
  int64_t input_w = input.size(dim_w);
  int64_t output_w  = input_w + pad_l + pad_r;

  Tensor grad_output = grad_output_.contiguous();

  TORCH_CHECK(output_w == grad_output.size(dim_w),
    "gradOutput width unexpected. Expected: ", output_w, ", Got: ",
    grad_output.size(dim_w));

  dim3 block_size(output_w > 256 ? 256 : output_w);
  dim3 grid_size((int) ::ceil(output_w / 256.0), nplane, nbatch);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    grad_input.scalar_type(), "reflection_pad1d_backward_out_template", [&] {
      reflection_pad1d_backward_out_kernel<<<
        grid_size, block_size, 0, at::cuda::getCurrentCUDAStream()>>>(
          grad_input.data_ptr<scalar_t>(), grad_output.data_ptr<scalar_t>(),
          input_w, pad_l, pad_r);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  );
}

void reflection_pad2d_out_template(
    Tensor &output, const Tensor &input_, IntArrayRef padding) {

  TORCH_CHECK(canUse32BitIndexMath(input_),
    "input tensor must fit into 32-bit index math");

  int plane_dim = 0;
  int dim_h = 1;
  int dim_w = 2;
  int nbatch = 1;

  bool valid_dims = input_.size(1) != 0 && input_.size(2) != 0;
  TORCH_CHECK(
      (input_.ndimension() == 3 && valid_dims) ||
      (input_.ndimension() == 4 && valid_dims && input_.size(3) != 0),
      "3D or 4D (batch mode) tensor expected for input, but got: ", input_);

  if (input_.ndimension() == 4) {
    nbatch = input_.size(0);
    plane_dim++;
    dim_h++;
    dim_w++;
  }

  int64_t pad_l = padding[0];
  int64_t pad_r = padding[1];
  int64_t pad_t = padding[2];
  int64_t pad_b = padding[3];

  int nplane = input_.size(plane_dim);
  int input_h = input_.size(dim_h);
  int input_w = input_.size(dim_w);

  TORCH_CHECK(pad_l < input_w && pad_r < input_w,
    "Padding size should be less than the corresponding input dimension, but "
    "got: padding (", pad_l, ", ", pad_r, ") at dimension ", dim_w,
    " of input ", input_.sizes());

  TORCH_CHECK(pad_t < input_h && pad_b < input_h,
    "Padding size should be less than the corresponding input dimension, but "
    "got: padding (", pad_t, ", ", pad_b, ") at dimension ", dim_h,
    " of input ", input_.sizes());

  int output_h = input_h + pad_t + pad_b;
  int output_w  = input_w + pad_l + pad_r;

  TORCH_CHECK(output_w >= 1 || output_h >= 1,
    "input (H: ", input_h, ", W: ", input_w, ")is too small.  Calculated "
    "output H: ", output_h, " W: ", output_w);

  if (input_.ndimension() == 3) {
    output.resize_({nplane, output_h, output_w});
  } else {
    output.resize_({nbatch, nplane, output_h, output_w});
  }
  if (output.numel() == 0) {
    return;
  }

  Tensor input = input_.contiguous();

  int output_plane_size = output_h * output_w;
  dim3 block_size(output_plane_size > 256 ? 256 : output_plane_size);
  dim3 grid_size(
    (int) std::ceil(output_plane_size/256.0), nplane, nbatch);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    input.scalar_type(), "reflection_pad2d_out_template", [&] {
      reflection_pad2d_out_kernel<<<
        grid_size, block_size, 0, at::cuda::getCurrentCUDAStream()>>>(
          input.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
          input_w, input_h,
          pad_t, pad_b, pad_l, pad_r);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  );
}

void reflection_pad2d_backward_out_template(
    Tensor &grad_input, const Tensor &grad_output_,
    const Tensor &input, IntArrayRef padding) {

  if (grad_input.numel() == 0) {
    return;
  }

  TORCH_CHECK(canUse32BitIndexMath(input),
    "input tensor must fit into 32-bit index math");
  TORCH_CHECK(canUse32BitIndexMath(grad_output_),
    "output gradient tensor must fit into 32-bit index math");

  int plane_dim = 0;
  int dim_h = 1;
  int dim_w = 2;
  int nbatch = 1;

  if (input.ndimension() == 4) {
    nbatch = input.size(0);
    plane_dim++;
    dim_h++;
    dim_w++;
  }

  int64_t pad_l = padding[0];
  int64_t pad_r = padding[1];
  int64_t pad_t = padding[2];
  int64_t pad_b = padding[3];

  int nplane = input.size(plane_dim);
  int input_h = input.size(dim_h);
  int input_w = input.size(dim_w);

  int output_h = input_h + pad_t + pad_b;
  int output_w  = input_w + pad_l + pad_r;

  TORCH_CHECK(output_w == grad_output_.size(dim_w), "grad_output width "
    "unexpected. Expected: ", output_w, ", Got: ", grad_output_.size(dim_w));
  TORCH_CHECK(output_h == grad_output_.size(dim_h), "grad_output height "
    "unexpected. Expected: ", output_h, ", Got: ", grad_output_.size(dim_h));

  Tensor grad_output = grad_output_.contiguous();

  int output_plane_size = output_h * output_w;
  dim3 block_size(output_plane_size > 256 ? 256 : output_plane_size);
  dim3 grid_size(
    (int) std::ceil(output_plane_size/256.0), nplane, nbatch);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    input.scalar_type(), "reflection_pad2d_backward_out_template", [&] {
      reflection_pad2d_backward_out_kernel<<<
        grid_size, block_size, 0, at::cuda::getCurrentCUDAStream()>>>(
          grad_input.data_ptr<scalar_t>(), grad_output.data_ptr<scalar_t>(),
          input_w, input_h,
          pad_t, pad_b, pad_l, pad_r);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  );
}

} // namespace


Tensor& reflection_pad1d_out_cuda(
    Tensor& output, const Tensor& input, IntArrayRef padding) {
  reflection_pad1d_out_template(output, input, padding);
  return output;
}

Tensor reflection_pad1d_cuda(const Tensor& input, IntArrayRef padding) {
  auto output = at::empty({0}, input.options());
  reflection_pad1d_out_template(output, input, padding);
  return output;
}

Tensor& reflection_pad1d_backward_out_cuda(
    Tensor& grad_input, const Tensor& grad_output,
    const Tensor& input,
    IntArrayRef padding) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("reflection_pad1d_backward_out_cuda");
  grad_input.resize_as_(input);
  grad_input.zero_();
  reflection_pad1d_backward_out_template(
    grad_input, grad_output, input, padding);
  return grad_input;
}

Tensor reflection_pad1d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& input,
    IntArrayRef padding) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("reflection_pad1d_backward_cuda");
  auto grad_input = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  reflection_pad1d_backward_out_template(
    grad_input, grad_output, input, padding);
  return grad_input;
}

Tensor& reflection_pad2d_out_cuda(
    Tensor& output, const Tensor& input, IntArrayRef padding) {
  reflection_pad2d_out_template(output, input, padding);
  return output;
}

Tensor reflection_pad2d_cuda(const Tensor& input, IntArrayRef padding) {
  auto output = at::empty({0}, input.options());
  reflection_pad2d_out_template(output, input, padding);
  return output;
}

Tensor& reflection_pad2d_backward_out_cuda(
    Tensor& grad_input, const Tensor& grad_output,
    const Tensor& input,
    IntArrayRef padding) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("reflection_pad2d_backward_out_cuda");
  grad_input.resize_as_(input);
  grad_input.zero_();
  reflection_pad2d_backward_out_template(
    grad_input, grad_output, input, padding);
  return grad_input;
}

Tensor reflection_pad2d_backward_cuda(
    const Tensor& grad_output,
    const Tensor& input,
    IntArrayRef padding) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("reflection_pad2d_backward_cuda");
  auto grad_input = at::zeros_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  reflection_pad2d_backward_out_template(
    grad_input, grad_output, input, padding);
  return grad_input;
}

} // namespace native
} // namespace at
