#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/LegacyTHFunctionsCUDA.h>
#include <ATen/NamedTensorUtils.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/Dispatch.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/LinearAlgebra.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>

namespace at { namespace native {

Tensor prepare_matrix_for_cublas(Tensor& tensor, bool& transpose_tensor) {
  Tensor tensor_;
  IntArrayRef tensor_strides = tensor.strides();
  IntArrayRef tensor_sizes = tensor.sizes();

  if ((tensor_strides[0] == 1) && (tensor_strides[1] >= std::max<int64_t>(1, tensor_sizes[0]))) {
    tensor_ = tensor;
    transpose_tensor = false;
  } else if ((tensor_strides[1] == 1) && (tensor_strides[0] >= std::max<int64_t>(1, tensor_sizes[1]))) {
    tensor_ = tensor;
    transpose_tensor = true;
  } else {
    transpose_tensor = true;
    tensor_ = tensor.clone(at::MemoryFormat::Contiguous);
  }

  return tensor_;
}

Tensor prepare_batch_matrix_for_cublas(const Tensor& tensor, bool& transpose_tensor, int64_t& ld_tensor, bool transpose_result, int64_t m, int64_t n) {
  IntArrayRef tensor_strides = tensor.strides();
  Tensor tensor_;
  int fast_dim = transpose_result ? 2 : 1;
  int leading_dim = transpose_result ? 1 : 2;

  if (tensor_strides[fast_dim] == 1 &&
    (tensor_strides[leading_dim] >= std::max<int64_t>(1, m))) {
    transpose_tensor = false;
    tensor_ = tensor;
    ld_tensor = tensor_strides[leading_dim];
  } else if ((tensor_strides[leading_dim] == 1) &&
    (tensor_strides[fast_dim] >= std::max<int64_t>(1, n))) {
    transpose_tensor = true;
    tensor_ = tensor;
    ld_tensor = tensor_strides[fast_dim];
  } else {
    transpose_tensor = !transpose_result;
    if (tensor.is_contiguous()) {
      tensor_ = tensor;
    } else {
      tensor_ = tensor.clone(at::MemoryFormat::Contiguous);
    }
    ld_tensor = tensor_.stride(1);
  }

  return tensor_;
}

namespace {

Tensor& addmm_out_cuda_impl(Tensor& result, const Tensor& self, const Tensor& mat1, const Tensor& mat2, Scalar beta, Scalar alpha) {
  TORCH_CHECK(mat1.dim() == 2 && mat2.dim() == 2, "tensors must be 2-D");

  TensorArg args[]{{result, "out", 0}, {self, "self", 1}, {mat1, "mat1", 2}, {mat2, "mat2", 3}};
  checkAllSameGPU("addmm", args);

  Tensor self_;
  if (&result != &self) {
    std::tie(self_) = expand_size(self, {mat1.size(0), mat2.size(1)}, "addmm");
  } else {
    self_ = self;
  }

  IntArrayRef mat1_sizes = mat1.sizes();
  IntArrayRef mat2_sizes = mat2.sizes();
  IntArrayRef self__sizes = self_.sizes();
  TORCH_CHECK(mat1_sizes[1] == mat2_sizes[0], "mat1 dim 1 must match mat2 dim 0");
  TORCH_CHECK(self__sizes[0] == mat1_sizes[0], "self_ dim 0 must match mat1 dim 0");
  TORCH_CHECK(self__sizes[1] == mat2_sizes[1], "self_ dim 1 must match mat2 dim 1");

  if (&result != &self) {
    at::native::resize_as_(result, self_);
    if (beta.toComplexDouble() != 0.0) {
      at::native::copy_(result, self_);
    }
  }

  TORCH_CHECK(result.dim() == 2 && self_.dim() == 2, "tensors must be 2-D");

  IntArrayRef result_sizes = result.sizes();
  if ((result_sizes[0] == 0) || (result_sizes[1] == 0)) {
    return result;
  }

  bool transpose_result;
  Tensor result_ = prepare_matrix_for_cublas(result, transpose_result);
  bool transpose_mat1;
  bool transpose_mat2;
  Tensor mat1_ = transpose_result ? mat2 : mat1;
  Tensor mat2_ = transpose_result ? mat1 : mat2;
  mat1_ = prepare_matrix_for_cublas(mat1_, transpose_mat1);
  mat2_ = prepare_matrix_for_cublas(mat2_, transpose_mat2);

  if (transpose_result) {
    transpose_mat1 = !transpose_mat1;
    transpose_mat2 = !transpose_mat2;
    mat1_sizes = mat1_.sizes();
    mat2_sizes = mat2_.sizes();
  }

  int64_t m = mat1_sizes[transpose_result ? 1 : 0];
  int64_t k = mat1_sizes[transpose_result ? 0 : 1];
  int64_t n = mat2_sizes[transpose_result ? 0 : 1];
  int64_t mat1_ld = mat1_.stride((transpose_mat1 == transpose_result) ? 1 : 0);
  int64_t mat2_ld = mat2_.stride((transpose_mat2 == transpose_result) ? 1 : 0);
  int64_t result_ld = result_.stride(transpose_result ? 0 : 1);
  at::ScalarType scalar_type = self_.scalar_type();

  if (mat1.numel() == 0) {
    // By definition, when beta==0, values in self should be ignored. nans and infs
    // should not propagate
    if (beta.toComplexDouble() == 0.) {
      return result.zero_();
    }
    return at::native::mul_out(result, self, at::native::scalar_tensor(beta, at::device(at::kCPU).dtype(self.scalar_type())));
  }

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, scalar_type, "addmm_cuda", [&] {
    scalar_t alpha_val = alpha.to<scalar_t>();
    scalar_t beta_val = beta.to<scalar_t>();
    scalar_t* mat1_ptr = mat1_.data_ptr<scalar_t>();
    scalar_t* mat2_ptr = mat2_.data_ptr<scalar_t>();
    scalar_t* result_ptr = result_.data_ptr<scalar_t>();
    at::cuda::blas::gemm<scalar_t>(
      transpose_mat1 ? 't' : 'n',
      transpose_mat2 ? 't' : 'n',
      m, n, k,
      alpha_val,
      mat1_ptr, mat1_ld,
      mat2_ptr, mat2_ld,
      beta_val,
      result_ptr, result_ld
    );
  });
  if (result.data_ptr() != result_.data_ptr()) {
    result.copy_(result_);
  }
  return result;
}

Tensor& baddbmm_out_cuda_impl(Tensor& result, const Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  TORCH_CHECK(self.dim() == 3, "self must be a 3D tensor");
  TORCH_CHECK(batch1.dim() == 3, "batch1 must be a 3D tensor");
  TORCH_CHECK(batch2.dim() == 3, "batch2 must be a 3D tensor");

  TensorArg args[]{{result, "out", 0}, {self, "self", 1}, {batch1, "batch1", 2}, {batch2, "batch2", 3}};
  checkAllSameGPU("baddbmm", args);

  IntArrayRef batch1_sizes = batch1.sizes();
  IntArrayRef batch2_sizes = batch2.sizes();
  IntArrayRef self_sizes = self.sizes();

  TORCH_CHECK(self_sizes[0] == batch1_sizes[0], "self dim 0 must match batch1 dim 0");
  TORCH_CHECK(self_sizes[0] == batch2_sizes[0], "self dim 0 must match batch2 dim 0");
  TORCH_CHECK(self_sizes[1] == batch1_sizes[1], "self dim 1 must match batch1 dim 1");
  TORCH_CHECK(self_sizes[2] == batch2_sizes[2], "self dim 2 must match batch2 dim 2");
  TORCH_CHECK(batch1_sizes[2] == batch2_sizes[1], "batch1 dim 2 must match batch2 dim 1");

  if (!result.is_same(self)) {
    result.resize_as_(self);
    if (beta.to<c10::complex<double>>() != 0.0) {
      result.copy_(self);
    }
  }

  bool transpose_result = false;
  Tensor result_;
  IntArrayRef result_strides = result.strides();
  IntArrayRef result_sizes = result.sizes();

  if ((result_strides[1] == 1) &&
      ((result_sizes[2] == 1) || (result_strides[2] >= std::max<int64_t>(1, result_sizes[1])))) {
    result_ = result;
  } else if ((result_strides[2] == 1) &&
    (result_sizes[1] == 1 || (result_strides[1] >= std::max<int64_t>(1, result_sizes[2])))) {
    transpose_result = true;
    result_ = result;
  } else {
    result_ = result.transpose(1, 2).clone(at::MemoryFormat::Contiguous);
    result_ = result_.transpose(1, 2);
  }

  int leading_dim = transpose_result ? 1 : 2;

  Tensor batch1_ = transpose_result ? batch2 : batch1;
  Tensor batch2_ = transpose_result ? batch1 : batch2;
  int64_t m = result_sizes[transpose_result ? 2 : 1];
  int64_t n = result_sizes[leading_dim];
  int64_t k = batch1_.size(leading_dim);

  int64_t lda, ldb, ldc;
  bool transpose_batch1, transpose_batch2;
  batch1_ = prepare_batch_matrix_for_cublas(batch1_, transpose_batch1, lda, transpose_result, m, k);
  batch2_ = prepare_batch_matrix_for_cublas(batch2_, transpose_batch2, ldb, transpose_result, k, n);

  ldc = result_.stride(leading_dim);
  int64_t num_batches = result_.size(0);

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "baddbmm_cuda", [&] {
    scalar_t alpha_val = alpha.to<scalar_t>();
    scalar_t beta_val = beta.to<scalar_t>();
    scalar_t* batch1_ptr = batch1_.data_ptr<scalar_t>();
    scalar_t* batch2_ptr = batch2_.data_ptr<scalar_t>();
    scalar_t* result_ptr = result_.data_ptr<scalar_t>();
    at::cuda::blas::bgemm<scalar_t>(
      transpose_batch1 ? 't' : 'n',
      transpose_batch2 ? 't' : 'n',
      m, n, k,
      alpha_val,
      batch1_ptr, lda, batch1_.stride(0),
      batch2_ptr, ldb, batch2_.stride(0),
      beta_val,
      result_ptr, ldc, result_.stride(0),
      num_batches
    );
  });
  if (!result.is_same(result_)) {
    result.copy_(result_);
  }
  return result;
}

} // anonymous namespace

Tensor& mm_out_cuda(Tensor& result, const Tensor& self, const Tensor& mat2) {
  result.resize_({ self.size(0), mat2.size(1) });
  return addmm_out_cuda_impl(result, result, self, mat2, 0, 1);
}

Tensor mm_cuda(const Tensor& self, const Tensor& mat2) {
  Tensor result = at::empty({ self.size(0), mat2.size(1) }, self.options());
  return addmm_out_cuda_impl(result, result, self, mat2, 0, 1);
}

Tensor& addmm_out_cuda(Tensor &out, const Tensor &self,
                        const Tensor &mat1, const Tensor &mat2,
                        Scalar beta, Scalar alpha) {
  {
    at::NoNamesGuard guard;
    Tensor& result = addmm_out_cuda_impl(out, self, mat1, mat2, beta, alpha);
  }
  at::namedinference::propagate_names_for_addmm(out, mat1, mat2, self);
  return out;
}

Tensor addmm_cuda(const Tensor& self, const Tensor& mat1, const Tensor& mat2,
                  Scalar beta, Scalar alpha) {
  Tensor out = at::empty({0}, self.options());
  addmm_out_cuda(out, self, mat1, mat2, beta, alpha);
  return out;
}

Tensor& addmm__cuda(Tensor& self, const Tensor& mat1, const Tensor& mat2,
                    Scalar beta, Scalar alpha) {
  addmm_out_cuda(self, self, mat1, mat2, beta, alpha);
  return self;
}

Tensor& baddbmm_out_cuda(Tensor &result, const Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  Tensor self_;
  if (&result != &self) {
    std::tie(self_) = expand_size(self, {batch1.size(0), batch1.size(1), batch2.size(2)}, "baddbmm");
  } else {
   self_ = self;
  }
  {
    at::NoNamesGuard guard;
    baddbmm_out_cuda_impl(result, self_, batch1, batch2, beta, alpha);
  }
  namedinference::propagate_names_if_nonempty(
       result,
       namedinference::compute_baddbmm_outnames(result, batch1, batch2, self));
  return result;
}

Tensor baddbmm_cuda(const Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  Tensor out = at::empty({0}, self.options());
  return baddbmm_out_cuda(out, self, batch1, batch2, beta, alpha);
}

Tensor& baddbmm__cuda(Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  return baddbmm_out_cuda(self, self, batch1, batch2, beta, alpha);
}

Tensor& bmm_out_cuda(Tensor &result, const Tensor& batch1, const Tensor& batch2) {
  result.resize_({ batch1.size(0), batch1.size(1), batch2.size(2) });
  Scalar beta(0.0);
  Scalar alpha(1.0);
  {
    NoNamesGuard guard;
    baddbmm_out_cuda_impl(result, result, batch1, batch2, beta, alpha);
  }
  namedinference::propagate_names_if_nonempty(
      result,
      namedinference::compute_bmm_outnames(result, batch1, batch2));
  return result;
}

Tensor bmm_cuda(const Tensor& self, const Tensor& mat2) {
  Tensor result = at::empty({0}, self.options());
  return native::bmm_out_cuda(result, self, mat2);
}

Tensor& addbmm_out_cuda(Tensor& out, const Tensor& self,
                        const Tensor& batch1, const Tensor& batch2,
                        Scalar beta, Scalar alpha) {
  TORCH_CHECK(batch1.dim() == 3 && batch2.dim() == 3,
              "Batch tensors should be 3D, got dimensions ", batch1.dim(),
              " and ", batch2.dim());

  Tensor self_;
  if (&out != &self) {
    std::tie(self_) = expand_size(self, {batch1.size(1), batch2.size(2)}, "addbmm");
  } else {
    self_ = self;
  }

  TORCH_CHECK(out.device() == self_.device() &&
              out.device() == batch1.device() &&
              out.device() == batch2.device(),
              "Expected all tensors to be on the same device. Found: ",
              out.device(), ", ", self_.device(), ", ",
              batch1.device(), " and ", batch2.device());
  TORCH_CHECK(self_.dim() == 2,
              "2D tensor expected, got ", self_.dim(), "D tensor for input");
  int64_t batchnum = batch1.size(0);
  int64_t m1d1 = batch1.size(1);
  int64_t innerdim = batch1.size(2);
  int64_t m2d2 = batch2.size(2);
  TORCH_CHECK(batchnum == batch2.size(0),
              "equal number of batches expected");
  TORCH_CHECK(m1d1 == self_.size(0),
              "first dimension of batch1  must match first dimension of input");
  TORCH_CHECK(m2d2 == self_.size(1),
              "second dimension of batch2 must match second dimension of input");
  TORCH_CHECK(innerdim == batch2.size(1),
              "second dimension of batch1 must match first dimension of batch2");

  if (&out != &self) {
    at::native::resize_as_(out, self_);
    if (beta.to<c10::complex<double>>() != 0.0) {
      at::native::copy_(out, self_);
    }
  }

  for (int64_t i=0; i<batchnum; i++) {
    addmm_out_cuda(out, out, batch1[i], batch2[i], beta, alpha);
    beta = 1;
  }
  return out;
}

Tensor& addbmm__cuda(Tensor& self,
                     const Tensor& batch1, const Tensor& batch2,
                     Scalar beta, Scalar alpha) {
  addbmm_out_cuda(self, self, batch1, batch2, beta, alpha);
  return self;
}

Tensor addbmm_cuda(const Tensor& self,
                   const Tensor& batch1, const Tensor& batch2,
                   Scalar beta, Scalar alpha)
{
  Tensor out = at::empty({0}, self.options());
  addbmm_out_cuda(out, self, batch1, batch2, beta, alpha);
  return out;
}

namespace {

inline void dot_check(const Tensor& self, const Tensor& other) {
  TORCH_CHECK(
      self.dim() == 1 && other.dim() == 1,
      "1D tensors expected, but got ",
      self.dim(),
      "D and ",
      other.dim(),
      "D tensors");
  TORCH_CHECK(
      self.scalar_type() == other.scalar_type(),
      "dot : expected both vectors to have same dtype, but found ",
      self.scalar_type(),
      " and ",
      other.scalar_type());
  TORCH_CHECK(
      self.numel() == other.numel(),
      "inconsistent tensor size, expected tensor [",
      self.numel(),
      "] and src [",
      other.numel(),
      "] to have the same number of elements, but got ",
      self.numel(),
      " and ",
      other.numel(),
      " elements respectively");
  TORCH_CHECK(
      self.device() == other.device(),
      "expected all tensors to be on the same device. Found: ",
      self.device(),
      ", ",
      other.device());
  TORCH_CHECK(
      (self.numel() <= INT_MAX) && (self.stride(0) <= INT_MAX) &&
          (other.stride(0) <= INT_MAX),
      "dot only supports n, incx, incy with the bound [val] <= %d",
      INT_MAX);
}

} // anonymous namespace

Tensor dot_cuda(const Tensor& self, const Tensor& other) {
  at::NoNamesGuard guard;

  dot_check(self, other);

  const int n = static_cast<int>(self.numel());
  int incx = static_cast<int>(self.stride(0));
  int incy = static_cast<int>(other.stride(0));
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  return AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, self.scalar_type(), "dot", [&] {
    Tensor result = at::empty({}, self.options());

    auto handle = at::cuda::getCurrentCUDABlasHandle();
    at::cuda::blas::PointerModeGuard pointerModeGuard(handle, HIPBLAS_POINTER_MODE_DEVICE);
    at::cuda::blas::dot<scalar_t>(
        handle,
        n,
        self.data_ptr<scalar_t>(),
        incx,
        other.data_ptr<scalar_t>(),
        incy,
        result.data_ptr<scalar_t>());

    return result;
  });
}

Tensor vdot_cuda(const Tensor& self, const Tensor& other) {
  if (!self.is_complex()) {
    return dot_cuda(self, other);
  }

  at::NoNamesGuard guard;
  dot_check(self, other);

  const int n = static_cast<int>(self.numel());
  int incx = static_cast<int>(self.stride(0));
  int incy = static_cast<int>(other.stride(0));
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  return AT_DISPATCH_COMPLEX_TYPES(self.scalar_type(), "vdot", [&] {
    Tensor result = at::empty({}, self.options());

    auto handle = at::cuda::getCurrentCUDABlasHandle();
    at::cuda::blas::PointerModeGuard pointerModeGuard(
        handle, HIPBLAS_POINTER_MODE_DEVICE);
    at::cuda::blas::vdot<scalar_t>(
        handle,
        n,
        self.data_ptr<scalar_t>(),
        incx,
        other.data_ptr<scalar_t>(),
        incy,
        result.data_ptr<scalar_t>());

    return result;
  });
}

namespace {

void addr_kernel_cuda(TensorIterator &iter, Scalar beta, Scalar alpha) {
  if (iter.dtype() == ScalarType::Bool) {
    using scalar_t = bool;
    auto beta_val = beta.to<scalar_t>();
    auto alpha_val = alpha.to<scalar_t>();

    // when beta is false, values in self should be ignored,
    // nans and infs in self should not propagate.
    if (beta_val == false) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return alpha_val && vec1_val && vec2_val;
        }
      );
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return (beta_val && self_val) || (alpha_val && vec1_val && vec2_val);
        }
      );
    }
    return;
  }

  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kBFloat16, kHalf,
                                         iter.dtype(), "addr_cuda", [&] {
    auto beta_val = beta.to<scalar_t>();
    auto alpha_val = alpha.to<scalar_t>();

    scalar_t zero_val(0);
    // when beta==0, values in self should be ignored,
    // nans and infs in self should not propagate.
    if (beta_val == zero_val) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return alpha_val * vec1_val * vec2_val;
        }
      );
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (scalar_t self_val,
                        scalar_t vec1_val, scalar_t vec2_val) -> scalar_t {
          return beta_val * self_val + alpha_val * vec1_val * vec2_val;
        }
      );
    }
  });
}

} // anonymous namespace

REGISTER_DISPATCH(addr_stub, &addr_kernel_cuda);

}}
