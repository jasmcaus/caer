#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void logical_and_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBool, ScalarType::BFloat16,
                                         iter.common_dtype(), "logical_and_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
      return a && b;
    });
  });
}

void logical_or_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBool, ScalarType::BFloat16,
                                         iter.common_dtype(), "logical_or_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
      return a || b;
    });
  });
}

void logical_xor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBool, ScalarType::BFloat16,
                                         iter.common_dtype(), "logical_xor_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> bool {
      return bool(a) != bool(b);
    });
  });
}

REGISTER_DISPATCH(logical_and_stub, &logical_and_kernel_cuda);
REGISTER_DISPATCH(logical_or_stub, &logical_or_kernel_cuda);
REGISTER_DISPATCH(logical_xor_stub, &logical_xor_kernel_cuda);


}} // namespace at::native
