#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/PointwiseOps.h>
#include <THC/THCNumerics.cuh>

namespace at { namespace native {

void addcmul_cuda_kernel(TensorIterator& iter, Scalar value) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.dtype(), "addcmul_cuda", [&]() {
    auto alpha = value.to<scalar_t>();
    gpu_kernel(iter, [alpha]GPU_LAMBDA(scalar_t a, scalar_t b, scalar_t c) -> scalar_t {
      return a + alpha * b * c;
    });
  });
}

void addcdiv_cuda_kernel(TensorIterator& iter, Scalar value) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.dtype(), "addcdiv_cuda", [&]() {
    auto alpha = value.to<scalar_t>();
    gpu_kernel(iter, [alpha]GPU_LAMBDA(scalar_t a, scalar_t b, scalar_t c) -> scalar_t {
      return a + alpha * (b / c);
    });
  });
}

void smooth_l1_backward_cuda_kernel(TensorIterator& iter, Scalar norm, double beta) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "smooth_l1_backward_cuda", [&iter, &norm, beta] {
      auto norm_val = norm.to<scalar_t>();
      scalar_t beta_val(beta);
      gpu_kernel(iter, [norm_val, beta_val]GPU_LAMBDA(scalar_t input, scalar_t target, scalar_t grad_output) -> scalar_t {
        const auto x = input - target;
        if (x < -beta_val)
          return -norm_val * grad_output;
        else if (x > beta_val)
          return norm_val * grad_output;
        else
          return norm_val * x * grad_output / beta_val;
    });
  });
}

void mse_backward_cuda_kernel(TensorIterator& iter, Scalar value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "mse_backward_cuda", [&]() {
    auto alpha = value.to<scalar_t>();
    gpu_kernel(iter, [alpha]GPU_LAMBDA(scalar_t a, scalar_t b, scalar_t c) -> scalar_t {
      return alpha * (a - b) * c;
    });
  });
}

REGISTER_DISPATCH(addcdiv_stub, &addcdiv_cuda_kernel);
REGISTER_DISPATCH(addcmul_stub, &addcmul_cuda_kernel);
REGISTER_DISPATCH(smooth_l1_backward_stub, &smooth_l1_backward_cuda_kernel);
REGISTER_DISPATCH(mse_backward_stub, &mse_backward_cuda_kernel);
}} // namespace at::native
