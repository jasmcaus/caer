#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <THC/THCAtomics.cuh>

namespace at {
namespace native {
namespace {

__device__ __forceinline__ size_t
idx_3d(const size_t nc,
    const size_t depth,
    const size_t height,
    const size_t width,
    const size_t z,
    const size_t y,
    const size_t x) {
  return ((nc * depth + z) * height + y) * width + x;
}

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_trilinear3d_out_frame(
    const int n,
    const accscalar_t rdepth,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor64<scalar_t, 5> idata,
    PackedTensorAccessor64<scalar_t, 5> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int depth1 = idata.size(2);
  const int height1 = idata.size(3);
  const int width1 = idata.size(4);
  const int depth2 = odata.size(2);
  const int height2 = odata.size(3);
  const int width2 = odata.size(4);

  if (index < n) {
    const int w2 = (index % (height2 * width2)) % width2; // 0:width2-1
    const int h2 = (index % (height2 * width2)) / width2; // 0:height2-1
    const int t2 = index / (height2 * width2); // 0:depth2-1
    // special case: just copy
    if (depth1 == depth2 && height1 == height2 && width1 == width2) {
      const int t1 = t2;
      const int h1 = h2;
      const int w1 = w2;

      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][t1][h1][w1];
          odata[n][c][t2][h2][w2] = val;
        }
      }
      return;
    }
    //
    const accscalar_t t1r = area_pixel_compute_source_index<accscalar_t>(
        rdepth, t2, align_corners, /*cubic=*/false);
    const int t1 = t1r;
    const int t1p = (t1 < depth1 - 1) ? 1 : 0;
    const accscalar_t t1lambda = t1r - t1;
    const accscalar_t t0lambda = static_cast<accscalar_t>(1) - t1lambda;
    //
    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val = t0lambda *
                (h0lambda *
                     (w0lambda * idata[n][c][t1][h1][w1] +
                      w1lambda * idata[n][c][t1][h1][w1 + w1p]) +
                 h1lambda *
                     (w0lambda * idata[n][c][t1][h1 + h1p][w1] +
                      w1lambda * idata[n][c][t1][h1 + h1p][w1 + w1p])) +
            t1lambda *
                (h0lambda *
                     (w0lambda * idata[n][c][t1 + t1p][h1][w1] +
                      w1lambda * idata[n][c][t1 + t1p][h1][w1 + w1p]) +
                 h1lambda *
                     (w0lambda * idata[n][c][t1 + t1p][h1 + h1p][w1] +
                      w1lambda * idata[n][c][t1 + t1p][h1 + h1p][w1 + w1p]));
        odata[n][c][t2][h2][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_trilinear3d_backward_out_frame(
    const size_t nc_,
    const int depth1,
    const int height1,
    const int width1,
    const int depth2,
    const int height2,
    const int width2,
    const accscalar_t rdepth,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    scalar_t* __restrict__ idata,
    const scalar_t* __restrict__ odata) {
  const size_t i_numel = nc_ * depth1 * height1 * width1;
  const size_t o_numel = nc_ * depth2 * height2 * width2;

  for (size_t index = blockDim.x * blockIdx.x + threadIdx.x; index < o_numel; index += blockDim.x * gridDim.x) {
    size_t index_temp = index;
    const int w2 = index_temp % width2;   // 0:width2-1
    index_temp /= width2;
    const int h2 = index_temp % height2;  // 0:height2-1
    index_temp /= height2;
    const int t2 = index_temp % depth2;   // 0:depth2-1
    const int nc = index_temp / depth2;

    const accscalar_t t1r = area_pixel_compute_source_index<accscalar_t>(
        rdepth, t2, align_corners, /*cubic=*/false);
    const int t1 = t1r;
    const int t1p = (t1 < depth1 - 1) ? 1 : 0;
    const accscalar_t t1lambda = t1r - t1;
    const accscalar_t t0lambda = static_cast<accscalar_t>(1) - t1lambda;
    //
    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    const scalar_t d2val = odata[index];
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, height1, width1, t1, h1, w1),
      i_numel,
      static_cast<scalar_t>(t0lambda * h0lambda * w0lambda * d2val),
      true);
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, width1, height1, t1, h1, w1 + w1p),
      i_numel,
      static_cast<scalar_t>(t0lambda * h0lambda * w1lambda * d2val),
      true);
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, width1, height1, t1, h1 + h1p, w1),
      i_numel,
      static_cast<scalar_t>(t0lambda * h1lambda * w0lambda * d2val),
      true);
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, width1, height1, t1, h1 + h1p, w1 + w1p),
      i_numel,
      static_cast<scalar_t>(t0lambda * h1lambda * w1lambda * d2val),
      true);
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, width1, height1, t1 + t1p, h1, w1),
      i_numel,
      static_cast<scalar_t>(t1lambda * h0lambda * w0lambda * d2val),
      true);
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, width1, height1, t1 + t1p, h1, w1 + w1p),
      i_numel,
      static_cast<scalar_t>(t1lambda * h0lambda * w1lambda * d2val),
      true);
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, width1, height1, t1 + t1p, h1 + h1p, w1),
      i_numel,
      static_cast<scalar_t>(t1lambda * h1lambda * w0lambda * d2val),
      true);
    fastAtomicAdd(
      idata,
      idx_3d(nc, depth1, width1, height1, t1 + t1p, h1 + h1p, w1 + w1p),
      i_numel,
      static_cast<scalar_t>(t1lambda * h1lambda * w1lambda * d2val),
      true);
  }
}

static void upsample_trilinear3d_out_cuda_template(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_d,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_trilinear3d_out_cuda", {input_arg, output_arg});

  TORCH_CHECK(
      output_size.size() == 3,
      "It is expected output_size equals to 3, but got size ",
      output_size.size());

  int output_depth = output_size[0];
  int output_height = output_size[1];
  int output_width = output_size[2];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_depth = input.size(2);
  int input_height = input.size(3);
  int input_width = input.size(4);

  upsample_3d_shape_check(
      input,
      Tensor(),
      nbatch,
      channels,
      input_depth,
      input_height,
      input_width,
      output_depth,
      output_height,
      output_width);

  output.resize_({input.size(0),
                  input.size(1),
                  output_depth,
                  output_height,
                  output_width});

  AT_ASSERT(
      input_depth > 0 && input_height > 0 && input_width > 0 &&
      output_depth > 0 && output_height > 0 && output_width > 0);

  const int num_kernels = output_depth * output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_trilinear3d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<scalar_t, 5>();
        auto odata = output.packed_accessor64<scalar_t, 5>();

        const accscalar_t rdepth = area_pixel_compute_scale<accscalar_t>(
            input_depth, output_depth, align_corners, scales_d);
        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_trilinear3d_out_frame<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(
                num_kernels,
                rdepth,
                rheight,
                rwidth,
                align_corners,
                idata,
                odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

static void upsample_trilinear3d_backward_out_cuda_template(
    Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_d,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(
      "upsample_trilinear3d_backward_out_cuda",
      {grad_output_arg, grad_input_arg});

  TORCH_CHECK(
      output_size.size() == 3,
      "It is expected output_size equals to 3, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 5,
      "It is expected input_size equals to 5, but got size ",
      input_size.size());

  int output_depth = output_size[0];
  int output_height = output_size[1];
  int output_width = output_size[2];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_depth = input_size[2];
  int input_height = input_size[3];
  int input_width = input_size[4];

  upsample_3d_shape_check(
      Tensor(),
      grad_output_,
      nbatch,
      channels,
      input_depth,
      input_height,
      input_width,
      output_depth,
      output_height,
      output_width);
  Tensor grad_output = grad_output_.contiguous();

  grad_input.resize_(
      {nbatch, channels, input_depth, input_height, input_width});
  // A contiguous tensor is required for the kernel launch config
  grad_input.contiguous();
  // Numbers are added atomically to grad_input tensor from multiple threads,
  // so it has to be initialized to zero.
  grad_input.zero_();

  // const size_t num_kernels = nbatch * channels * output_depth * output_height * output_width;
  const size_t num_kernels = grad_output.numel();
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (num_kernels > 0) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(),
      "upsample_trilinear3d_backward_out_frame",
      [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = grad_input.data_ptr<scalar_t>();
        auto odata = grad_output.data_ptr<scalar_t>();

        const accscalar_t rdepth = area_pixel_compute_scale<accscalar_t>(
            input_depth, output_depth, align_corners, scales_d);
        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        upsample_trilinear3d_backward_out_frame<scalar_t, accscalar_t>
            <<<cuda::ATenCeilDiv(num_kernels, static_cast<size_t>(num_threads)),
               num_threads,
               0,
               stream>>>(
                nbatch * channels,
                input_depth,
                input_height,
                input_width,
                output_depth,
                output_height,
                output_width,
                rdepth,
                rheight,
                rwidth,
                align_corners,
                idata,
                odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  }
}

} // namespace

Tensor& upsample_trilinear3d_out_cuda(
    Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_d,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  upsample_trilinear3d_out_cuda_template(
      output, input, output_size, align_corners, scales_d, scales_h, scales_w);
  return output;
}

Tensor upsample_trilinear3d_cuda(
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_d,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  Tensor output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_trilinear3d_out_cuda_template(
      output, input, output_size, align_corners, scales_d, scales_h, scales_w);
  return output;
}

Tensor& upsample_trilinear3d_backward_out_cuda(
    Tensor& grad_input,
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_d,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_trilinear3d_backward_out_cuda");
  upsample_trilinear3d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_d, scales_h, scales_w);
  return grad_input;
}

Tensor upsample_trilinear3d_backward_cuda(
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_d,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_trilinear3d_backward_cuda");
  Tensor grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_trilinear3d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_d, scales_h, scales_w);
  return grad_input;
}

using at::native::upsample::compute_output_size;
using at::native::upsample_cuda::get_scale_value;

Tensor upsample_trilinear3d_cuda(
    const Tensor& input,
    c10::optional<IntArrayRef> output_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  auto output = at::empty_like(input, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto osize = compute_output_size(input.sizes(), output_size, scale_factors);
  auto scale_d = get_scale_value(scale_factors, 0);
  auto scale_h = get_scale_value(scale_factors, 1);
  auto scale_w = get_scale_value(scale_factors, 2);
  upsample_trilinear3d_out_cuda_template(output, input, osize, align_corners, scale_d, scale_h, scale_w);
  return output;
}

Tensor upsample_trilinear3d_backward_cuda(
    const Tensor& grad_output,
    c10::optional<IntArrayRef> output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<ArrayRef<double>> scale_factors) {
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_trilinear3d_backward_cuda");
  auto osize = compute_output_size(input_size, output_size, scale_factors);
  auto scale_d = get_scale_value(scale_factors, 0);
  auto scale_h = get_scale_value(scale_factors, 1);
  auto scale_w = get_scale_value(scale_factors, 2);
  auto grad_input = at::empty_like(grad_output, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  upsample_trilinear3d_backward_out_cuda_template(
      grad_input, grad_output, osize, input_size, align_corners, scale_d, scale_h, scale_w);
  return grad_input;
}

} // namespace native
} // namespace at
