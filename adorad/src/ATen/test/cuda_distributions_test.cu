#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

__global__ void expected_uniforms(float* x, uint64_t counter_offset) {
  for(int i=0; i < 4; i++) {
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(
            123,
            i,
            counter_offset,
            &state);
    auto ret = hiprand_uniform4(&state);
    x[i] = ret.x;
  }
}

/**
 * Helper function that asserts call to uniform_ starts from the correct
 * philox offset.
 *   - Get 4 randoms with counter_offset for thread {0,1,2,3} from expected_uniforms
 *     kernel above.
 *   - Now get 4 more randoms from uniform_ (note thread {0,1,2,3} for this call should
 *     start from a counter_offset value)
 *   - the 4 randoms from expected_uniforms kernel and the 4 randoms from the previous call
 *     of uniform_ should match, signifying that the philox offset was
 *     incremented properly and no randoms are being reused from previous calls
 */
void assert_with_expected_uniforms(uint64_t counter_offset) {
  // allocate 4 float on host memory
  float *x;
  hipMallocManaged(&x, 4*sizeof(float));

  // launch kernel to get expected randoms
  expected_uniforms<<<1, 1>>>(x, counter_offset);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // get 4 new float from uniform_()
  auto self = at::empty({4}, at::TensorOptions(at::kCUDA));
  self.uniform_();

  // check randoms from expected_uniforms kernel are equal to the randoms from the second
  // call of uniform_()
  for (int i = 0; i < 4; i++) {
    ASSERT_EQ(self[i].item().to<float>(), x[i]);
  }

  // Free memory
  hipFree(x);
}

TEST(DistributionsTest, TestPhiloxIncrementSmallUniformTensor) {
  // Test Description:
  //   In Distributions.cu we mentioned that philox increment
  //   should be at least the number of hiprand() random numbers used in
  //   each thread. In this test, we make sure that uniform_ correctly
  //   increments philox and doesn't reuse randoms from previous calls
  //   for a small tensor size of 4.
  //    - We check that by first getting 4 randoms from uniform_.
  //      Once we get these 4 randoms, that would mean that philox counter for
  //      thread 0, 1, 2 and 3, was incremented by 4 (check calc_execution_policy
  //      function for details).
  //    - assert the call to uniform_ will start from counter_offset of 4

  // if cuda not available, return
  if (!at::cuda::is_available()) return;

  // manual seed to 123
  at::manual_seed(123);

  // get 4 randoms from uniform_(), philox offset is now incremented to 4 by this call
  at::empty({4}, at::TensorOptions(at::kCUDA)).uniform_();
  
  // expected uniforms will start from counter offset of 4
  assert_with_expected_uniforms(4);
}

TEST(DistributionsTest, TestPhiloxIncrementBigUniformTensor) {
  // Test Description:
  //   In Distributions.cu we mentioned that philox increment
  //   should be at least the number of hiprand() random numbers used in
  //   each thread. In this test, we make sure that uniform_ correctly
  //   increments philox and doesn't reuse randoms from previous calls
  //   for a big size tensor.
  //    - First of all, we come up with what the size of the big tensor
  //      should be for this test. Our goal is to show that when the uniform_
  //      kernel runs at full occupancy (i.e. when the number of elements is
  //      greater the number of threads launched), it hits the unroll loop in
  //      the uniform_ kernel.
  //    - Hence, we set the size of the tensor in this test to be 8 times the
  //      maximum number of threads we can launch. This means that, each thread will
  //      be yielding 8 elements, and as a result, hiprand_uniform4 will be called twice
  //      and all the 8 elements in a thread will consume all the float4 from the
  //      two calls of curand_unfiorm4 as a result of the unroll loop. Therefore,
  //      after this call to the unform_, counter_offset for the next call to uniform_
  //      will start from 8. This is what we test next.
  //    - assert that call to uniform_ will start from counter_offset of 8

  // if cuda not available, return
  if (!at::cuda::is_available()) return;

  // manual seed to 123
  at::manual_seed(123);

  // calculate maximum number of threads that can be launched
  // and set the numel to be 8 times that
  const int block_size = 256;
  dim3 dim_block(block_size);
  uint32_t blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor / block_size;
  dim3 grid(static_cast<uint32_t>(at::cuda::getCurrentDeviceProperties()->multiProcessorCount) * blocks_per_sm);
  auto numel = block_size * grid.x * 8;

  // get numel randoms from uniform_(), philox offset is now incremented to 8 by this call
  at::empty({numel}, at::TensorOptions(at::kCUDA)).uniform_();
  
  // expected uniforms will start from counter offset of 8
  assert_with_expected_uniforms(8);
}

TEST(DistributionsTest, TestPhiloxIncrementSmallMultinomialTensor) {
  // Test Description:
  //   Same concept as TestPhiloxIncrementSmallUniformTensor.
  //   Multinomial increments offset by 4. Tests if uniform starts from the correct offset.

  // if cuda not available, return
  if (!at::cuda::is_available()) return;

  // manual seed to 123
  at::manual_seed(123);

  // get some multinomial samples
  // this will trigger torch.multinomial without replacement
  // which utilizes uniform which increments counter by 4.
  // num_samples in the following call is 4.
  at::ones({4}, at::TensorOptions(at::kCUDA)).multinomial(4);

  // expected uniforms will start from counter offset of 4
  assert_with_expected_uniforms(4);
}
