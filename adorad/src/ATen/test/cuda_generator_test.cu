#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <c10/cuda/CUDAFunctions.h>
#include <ATen/core/PhiloxRNGEngine.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <assert.h>
#include <thread>

using namespace at;

/*
* Philox Engine Tests
*/

__global__ void testEngineReproducibility(){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  at::Philox4_32_10 engine1(0, idx, 4);
  at::Philox4_32_10 engine2(0, idx, 4);
  assert(engine1() == engine2());
}

void test_engine_reproducibility(){
  testEngineReproducibility<<<1, 1>>>();
}

TEST(CUDAGeneratorImpl, TestPhiloxEngineReproducibility) {
  // Test Description:
  //   Tests if same inputs give same results.
  //   launch one thread and create two engines.
  //   Given same seed, idx and offset, assert that the engines
  //   should be aligned and have the same sequence.
  if (!at::cuda::is_available()) return;
  test_engine_reproducibility();
  hipError_t err = hipDeviceSynchronize();
  bool isEQ = err == hipSuccess;
  ASSERT_TRUE(isEQ);
}

__global__ void testEngineOffset1(){
  at::Philox4_32_10 engine1(123, 1, 0);
  // Note: offset is a multiple of 4.
  // So if you want to skip 8 values, offset would
  // be 2, since 2*4=8.
  at::Philox4_32_10 engine2(123, 1, 2);
  for(int i = 0; i < 8; i++){
    // Note: instead of using the engine() call 8 times
    // we could have achieved the same functionality by
    // calling the incr() function twice.
    engine1();
  }
  assert(engine1() == engine2());
}

void test_engine_offset1(){
  testEngineOffset1<<<1, 1>>>();
}

TEST(CUDAGeneratorImpl, TestPhiloxEngineOffset1) {
  // Test Description:
  //   Tests offsetting in same thread.
  //   launch one thread and create two engines.
  //   make one engine skip the first 8 values and
  //   make another engine increment to until the
  //   first 8 values. Assert that the first call
  //   of engine2 and the 9th call of engine1 are equal.
  if (!at::cuda::is_available()) return;
  test_engine_offset1();
  hipError_t err = hipDeviceSynchronize();
  bool isEQ = err == hipSuccess;
  ASSERT_TRUE(isEQ);
}

__global__ void testEngineOffset2(){
  unsigned long long increment_val = ::ldexp(1.0, 64);
  at::Philox4_32_10 engine1(123, 0, increment_val);
  at::Philox4_32_10 engine2(123, increment_val, increment_val);
  
  engine2.incr_n(increment_val);
  engine2.incr();
  assert(engine1() == engine2());
}

void test_engine_offset2(){
  testEngineOffset2<<<1, 1>>>();
}

TEST(CUDAGeneratorImpl, TestPhiloxEngineOffset2) {
  // Test Description:
  //   Tests edge case at the end of the 2^190th value of the generator.
  //   launch one thread and create two engines
  //   make engine1 skip to the 2^64th 128 bit while being at thread 0
  //   make engine2 skip to the 2^64th 128 bit while being at 2^64th thread
  //   Assert that engine2 should be increment_val+1 steps behind engine1.
  if (!at::cuda::is_available()) return;
  test_engine_offset2();
  hipDeviceSynchronize();
  bool isEQ = hipGetLastError() == hipSuccess;
  ASSERT_TRUE(isEQ);
}

__global__ void testEngineOffset3(){
  unsigned long long increment_val = ::ldexp(1.0, 64);
  at::Philox4_32_10 engine1(123, 0, increment_val);
  at::Philox4_32_10 engine2(123, 1, 0);
  engine1.incr();
  assert(engine1() == engine2());
}

void test_engine_offset3(){
  testEngineOffset2<<<1, 1>>>();
}

TEST(CUDAGeneratorImpl, TestPhiloxEngineOffset3) {
  // Test Description:
  //   Tests edge case in between threads.
  //   launch one thread and create two engines
  //   make engine1 skip to the 2^64th 128 bit while being at thread 0
  //   start engine2 at thread 1, with offset 0
  //   Assert that engine1 is 1 step behind engine2.
  if (!at::cuda::is_available()) return;
  test_engine_offset3();
  hipDeviceSynchronize();
  bool isEQ = hipGetLastError() == hipSuccess;
  ASSERT_TRUE(isEQ);
}

__global__ void testEngineThreadIndex(){
  at::Philox4_32_10 engine1(123456, 0, 4);
  at::Philox4_32_10 engine2(123456, 1, 4);
  assert(engine1() != engine2());
}

void test_engine_thread_index(){
  testEngineThreadIndex<<<1, 1>>>();
}

TEST(CUDAGeneratorImpl, TestPhiloxEngineIndex) {
  // Test Description:
  //   Tests if thread indexing is working properly.
  //   launch one thread and create two engines
  //   with different thread index but same offset.
  //   Assert that the engines have different sequences.
  if (!at::cuda::is_available()) return;
  test_engine_thread_index();
  hipDeviceSynchronize();
  bool isEQ = hipGetLastError() == hipSuccess;
  ASSERT_TRUE(isEQ);
}

/*
* CUDA Generator Tests
*/

TEST(CUDAGeneratorImpl, TestGeneratorDynamicCast) {
  //  Test Description: Check dynamic cast for CUDA
  if (!at::cuda::is_available()) return;
  auto foo = at::cuda::detail::createCUDAGenerator();
  auto result = foo.get<CUDAGeneratorImpl>();
  ASSERT_EQ(typeid(at::CUDAGeneratorImpl*).hash_code(), typeid(result).hash_code());
}

TEST(CUDAGeneratorImpl, TestDefaultGenerator) {
  // Test Description: 
  // Check if default generator state is created only once
  // address of generator should be same in all calls
  if (!at::cuda::is_available()) return;
  auto foo = at::cuda::detail::getDefaultCUDAGenerator();
  auto bar = at::cuda::detail::getDefaultCUDAGenerator();
  ASSERT_EQ(foo, bar);

  if (c10::cuda::device_count() >= 2) {
    foo = at::cuda::detail::getDefaultCUDAGenerator(1);
    bar = at::cuda::detail::getDefaultCUDAGenerator(1);
    ASSERT_EQ(foo, bar);

    foo = at::cuda::detail::getDefaultCUDAGenerator(0);
    bar = at::cuda::detail::getDefaultCUDAGenerator(1);
    ASSERT_NE(foo, bar);
  }
}

TEST(CUDAGeneratorImpl, TestCloning) {
  // Test Description: 
  // Check cloning of new generators.
  // Note that we don't allow cloning of other
  // generator states into default generators.
  if (!at::cuda::is_available()) return;
  auto gen1 = at::cuda::detail::createCUDAGenerator();
  gen1.set_current_seed(123); // modify gen1 state
  auto cuda_gen1 = check_generator<CUDAGeneratorImpl>(gen1);
  cuda_gen1->set_philox_offset_per_thread(4);
  auto gen2 = at::cuda::detail::createCUDAGenerator();
  gen2 = gen1.clone();
  auto cuda_gen2 = check_generator<CUDAGeneratorImpl>(gen2);
  ASSERT_EQ(gen1.current_seed(), gen2.current_seed());
  ASSERT_EQ(
    cuda_gen1->philox_offset_per_thread(),
    cuda_gen2->philox_offset_per_thread()
  );
}

void thread_func_get_set_current_seed(Generator generator) {
  std::lock_guard<std::mutex> lock(generator.mutex());
  auto current_seed = generator.current_seed();
  current_seed++;
  generator.set_current_seed(current_seed);
}
  
TEST(CUDAGeneratorImpl, TestMultithreadingGetSetCurrentSeed) {
  // Test Description: 
  // Test current seed getter and setter are thread safe
  // See Note [Acquire lock when using random generators]
  if (!at::cuda::is_available()) return;
  auto gen1 = at::cuda::detail::getDefaultCUDAGenerator();
  auto initial_seed = gen1.current_seed();
  std::thread t0{thread_func_get_set_current_seed, gen1};
  std::thread t1{thread_func_get_set_current_seed, gen1};
  std::thread t2{thread_func_get_set_current_seed, gen1};
  t0.join();
  t1.join();
  t2.join();
  ASSERT_EQ(gen1.current_seed(), initial_seed+3);
}

TEST(CUDAGeneratorImpl, TestRNGForking) {
  // Test Description: 
  // Test that state of a generator can be frozen and
  // restored
  // See Note [Acquire lock when using random generators]
  if (!at::cuda::is_available()) return;
  auto default_gen = at::cuda::detail::getDefaultCUDAGenerator();
  auto current_gen = at::cuda::detail::createCUDAGenerator();
  {
    std::lock_guard<std::mutex> lock(default_gen.mutex());
    current_gen = default_gen.clone(); // capture the current state of default generator
  }
  auto target_value = at::randn({1000}, at::kCUDA);
  // Dramatically alter the internal state of the main generator
  auto x = at::randn({100000}, at::kCUDA);
  auto forked_value = at::randn({1000}, current_gen, at::kCUDA);
  ASSERT_EQ(target_value.sum().item<double>(), forked_value.sum().item<double>());
}

void makeRandomNumber() {
  hipSetDevice(std::rand() % 2);
  auto x = at::randn({1000});
}

void testCudaRNGMultithread() {
  auto threads = std::vector<std::thread>();
  for (auto i = 0; i < 1000; i++) {
    threads.emplace_back(makeRandomNumber);
  }
  for (auto& t : threads) {
    t.join();
  }
};

TEST(CUDAGeneratorImpl, TestMultithreadRNG) {
  if (!at::cuda::is_available()) return;
  testCudaRNGMultithread();
}
