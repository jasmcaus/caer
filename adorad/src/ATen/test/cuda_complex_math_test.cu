#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

int safeDeviceCount() {
  int count;
  hipError_t err = hipGetDeviceCount(&count);
  if (err == hipErrorInsufficientDriver || err == hipErrorNoDevice) {
    return 0;
  }
  return count;
}

#define SKIP_IF_NO_GPU()                    \
  do {                                      \
    if (safeDeviceCount() == 0) {           \
      return;                               \
    }                                       \
  } while(0)

#define C10_ASSERT_NEAR(a, b, tol) assert(abs(a - b) < tol)
#define C10_DEFINE_TEST(a, b)                       \
__global__ void CUDA##a##b();                       \
TEST(a##Device, b) {                                \
  SKIP_IF_NO_GPU();                                 \
  hipDeviceSynchronize();                          \
  CUDA##a##b<<<1, 1>>>();                           \
  hipDeviceSynchronize();                          \
  ASSERT_EQ(hipGetLastError(), hipSuccess);       \
}                                                   \
__global__ void CUDA##a##b()
#include <c10/test/util/complex_math_test_common.h>


#undef C10_DEFINE_TEST
#undef C10_ASSERT_NEAR
#define C10_DEFINE_TEST(a, b) TEST(a##Host, b)
#define C10_ASSERT_NEAR(a, b, tol) ASSERT_NEAR(a, b, tol)
#include <c10/test/util/complex_math_test_common.h>

