#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/core/Array.h>

using namespace at::native;
using namespace at::native::memory;

__managed__ double4 buffer1[1024];
__managed__ double4 buffer2[1024];

void reset_buffers() {
  for (int i = 0; i < 1024; i++) {
    buffer1[i].x = i;
    buffer1[i].y = i + 0.1;
    buffer1[i].z = i + 0.2;
    buffer1[i].w = i + 0.3;

    buffer2[2].x = -i;
    buffer2[2].y = -(i + 0.1);
    buffer2[2].z = -(i + 0.2);
    buffer2[2].w = -(i + 0.3);
  }
}

#ifdef __HIP_PLATFORM_HCC__
TEST(TestLoops, HasSameArgTypes) {
  // This is a compile-time unit test. If this file compiles without error,
  // then the test passes and during runtime, we just need to return.
  using namespace at::native::modern::detail;
  using func1_t = int (*)(float, float);
  using func2_t = int (*)(bool, float, float);
  using func3_t = int (*)(float);
  using func4_t = int (*)();
  static_assert(has_same_arg_types<func1_t>::value, "func1_t has the same argument types");
  static_assert(!has_same_arg_types<func2_t>::value, "func2_t does not have the same argument types");
  static_assert(has_same_arg_types<func3_t>::value, "func3_t has the same argument types");
  static_assert(has_same_arg_types<func4_t>::value, "func4_t has the same argument types");
  return;
}
#endif

TEST(TestVectorizedMemoryAccess, CanVectorizeUpTo) {
  char *ptr = reinterpret_cast<char *>(buffer1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int>(ptr), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int64_t>(ptr), 4);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 1), 1);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 1), 1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 2), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 2), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr + 2), 1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 4), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 4), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr + 4), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int>(ptr + 4), 1);

  ASSERT_EQ(memory::can_vectorize_up_to<bool>(ptr + 8), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int8_t>(ptr + 8), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int16_t>(ptr + 8), 4);
  ASSERT_EQ(memory::can_vectorize_up_to<int>(ptr + 8), 2);
  ASSERT_EQ(memory::can_vectorize_up_to<int64_t>(ptr + 8), 1);
}

// The following kernel copy values by using vectorized policies
// defined in `ATen/native/cuda/MemoryAccess.cuh`
template <typename scalar_t, int vec_size>
__global__ void vectorized_copy(scalar_t *dst, scalar_t *src) {
  using array_t = at::detail::Array<char*, 2>;
  array_t data;
  data[0] = reinterpret_cast<char *>(dst);
  data[1] = reinterpret_cast<char *>(src);
  int idx = blockIdx.x;
  using vectorized = policies::vectorized<vec_size, array_t>;
  auto policy = vectorized(data);
  scalar_t buf[thread_work_size];
  auto accessor = [&](int index) -> scalar_t & { return buf[index]; };
  policy.load_single_arg(accessor, src + 256 * blockIdx.x);
  policy.store(buf, idx);
}

TEST(TestVectorizedMemoryAccess, CopyKernel) {
  if (!at::cuda::is_available()) {
    return;
  }

  double *b1 = reinterpret_cast<double *>(buffer1);
  double *b2 = reinterpret_cast<double *>(buffer2);

  // vec4 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 4><<<16, 64>>>(b2, b1);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // vec2 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 2><<<16, 64>>>(b2, b1);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // vec1 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 1><<<16, 64>>>(b2, b1);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // unaligned
  for (int i = 0; i < 16; i++) {
    for (int j = 0; j < 16; j++) {
      b1 = reinterpret_cast<double *>(reinterpret_cast<char *>(buffer1) + i);
      b2 = reinterpret_cast<double *>(reinterpret_cast<char *>(buffer2) + j);
      hipGetLastError();
      hipDeviceSynchronize();
      vectorized_copy<double, 4><<<1, 64>>>(b2, b1);
      hipDeviceSynchronize();
      auto err = hipGetLastError();
      if (i % 16 == 0 && j % 16 == 0) {
        ASSERT_EQ(err, hipSuccess);
      } else {
        ASSERT_EQ(err, cudaErrorMisalignedAddress);
      }
    }
  }
}
