#include <gtest/gtest.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/util/Optional.h>

#include <assert.h>

using namespace at;

// optional in cuda files
TEST(OptionalTest, OptionalTestCUDA) {
  if (!at::cuda::is_available()) return;
  c10::optional<int64_t> trivially_destructible;
  c10::optional<std::vector<int64_t>> non_trivially_destructible;
  ASSERT_FALSE(trivially_destructible.has_value());
  ASSERT_FALSE(non_trivially_destructible.has_value());

  trivially_destructible = {5};
  non_trivially_destructible = std::vector<int64_t>{5, 10};
  ASSERT_TRUE(trivially_destructible.has_value());
  ASSERT_TRUE(non_trivially_destructible.has_value());
}
