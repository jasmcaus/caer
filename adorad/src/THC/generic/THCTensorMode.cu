
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMode.cu"
#else

#include <c10/cuda/CUDAException.h>
#include <thrust/iterator/constant_iterator.h>

void THCTensor_(calculateMode)(THCState *state,
                               THCTensor *values,
                               THCudaLongTensor *indices,
                               THCTensor *input,
                               THCudaLongStorage *sortBuffer,
                               int dimension,
                               THLongStorage *position) {
  THAssert(THCTensor_(isContiguous)(state, input));

  // Because the input is contiguous, we want to get a reference to the
  // location of the buffer at the innermost dimension that we are going
  // to calculate the mode for --> we do this by manually doing the stride
  // calculations to get an offset
  scalar_t *data = THCTensor_(data)(state, input);
  for (int i = 0; i < (position->nbytes() / sizeof(int64_t)); ++i) {
    data += THLongStorage_data(position)[i] * THTensor_strideLegacyNoScalars(input, i);
  }

  int64_t nElement = THCTensor_(sizeLegacyNoScalars)(state, input, THCTensor_(nDimensionLegacyAll)(state, input) - 1);
  THCThrustAllocator thrustAlloc(state);

  // Wrap input data, sortBuffer, in Thrust device vectors
  thrust::device_ptr<scalar_t> vecPtr = thrust::device_pointer_cast(data);
  thrust::device_vector<scalar_t> iter(vecPtr, vecPtr + nElement);
  thrust::device_ptr<int64_t> sbPtr = thrust::device_pointer_cast(THCudaLongStorage_data(state, sortBuffer));
  thrust::device_vector<int64_t> seq(sbPtr, sbPtr + nElement);

  // Fill sortBuffer with [0, 1, 2, ... nElement - 1]
  thrust::sequence(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#else
    thrust::device,
#endif
    seq.begin(), seq.end());

  // Sort the input data. The original indices of the data are stored in seq
  thrust::sort_by_key(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(), seq.begin()
#if defined(THC_REAL_IS_HALF)
    , ThrustHalfLess()
#endif
  );

  // Count # of unique elements via an inner product between adjacent elements.
  // Add 1 if two neighboring element are not equal.
  int unique = 1 + thrust::inner_product(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#else
    thrust::device,
#endif
    iter.begin(), iter.end() - 1, iter.begin() + 1, 0, thrust::plus<int>(),
#if defined(THC_REAL_IS_HALF)
    ThrustHalfNotEqualTo()
#else
    thrust::not_equal_to<scalar_t>()
#endif
  );

  // Count frequency of each element
  thrust::device_vector<scalar_t> keys(unique);
  thrust::device_vector<int> counts(unique);
  thrust::reduce_by_key(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(),
    thrust::constant_iterator<int>(1), keys.begin(), counts.begin()
#if defined(THC_REAL_IS_HALF)
    , ThrustHalfEqualTo()
#endif
  );

  // Find index of maximum count
  thrust::device_vector<int>::iterator it = thrust::max_element(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#else
    thrust::device,
#endif
    counts.begin(), counts.end());
  scalar_t mode = keys[it - counts.begin()];

  // Find first index within which it occurs
#if defined(THC_REAL_IS_HALF)
  thrust::device_vector<scalar_t>::iterator positionIter = thrust::find_if(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(), ThrustHalfEqualToPredicate(mode));
#else
  thrust::device_vector<scalar_t>::iterator positionIter = thrust::find(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#else
    thrust::device,
#endif
    iter.begin(), iter.end(), mode);
#endif

  THAssert(positionIter != iter.end());
  int64_t index = seq[positionIter - iter.begin()];

  // Place mode, index in output
  ptrdiff_t valuesOffset = THCTensor_(storageOffset)(state, values);
  int64_t indicesOffset = THCudaLongTensor_storageOffset(state, indices);

  for (int i = 0; i < (position->nbytes() / sizeof(int64_t)); ++i) {
    int64_t pos = THLongStorage_data(position)[i];
    valuesOffset += THTensor_strideLegacyNoScalars(values, i) * pos;
    indicesOffset += THTensor_strideLegacyNoScalars(indices, i) * pos;
  }
  THCStorage_(set)(state, THCTensor_(storage)(state, values), valuesOffset, mode);
  THCudaLongStorage_set(state, THCudaLongTensor_storage(state, indices), indicesOffset, index);
}

// this probably could be a loop, not a recursive algorithm
void THCTensor_(dimApplyMode)(THCState *state,
                              THCTensor *values,
                              THCudaLongTensor *indices,
                              THCTensor *input,
                              THCudaLongStorage *sortBuffer,
                              int dimension,
                              THLongStorage *position,
                              int curDim) {
  int64_t ndim = THCTensor_(nDimensionLegacyAll)(state, input);

  // Because we have transposed the Tensor, the data for the dimension we are mode'ing along
  // is always in the innermost dimension
  if (curDim == ndim - 1) {
    THCTensor_(calculateMode)(state, values, indices, input, sortBuffer, dimension, position);
  } else {
    // Loop through the values and recurse
    for (int i = 0; i < THCTensor_(sizeLegacyNoScalars)(state, input, curDim); ++i) {
      THLongStorage_data(position)[curDim] = i;
      THCTensor_(dimApplyMode)(state, values, indices, input, sortBuffer, dimension, position, curDim + 1);
    }
  }
}

#define MAX_GRID_SIZE  65535
#define MAX_BLOCK_SIZE 1024

void THCTensor_(mode)(THCState *state,
                      THCTensor *values,
                      THCudaLongTensor *indices,
                      THCTensor *input,
                      int dimension,
                      int keepdim) {
  THCTensor *transposed, *contiguous, *valuesTransposed;
  THLongStorage *position;
  THCudaLongStorage *sortBuffer;
  THCudaLongTensor *indicesTransposed;
  int64_t ndim, sliceSize, slices;


  THAssert(THCTensor_(checkGPU)(state, 1, values));

  // Verify they are asking for a valid dimension
  ndim = THCTensor_(nDimensionLegacyAll)(state, input);
  THArgCheck(dimension >= 0 && dimension < ndim, 4, "Dimension of out bounds");

  sliceSize = THCTensor_(sizeLegacyNoScalars)(state, input, dimension);
  slices = THCTensor_(nElement)(state, input) / sliceSize;

  // Resize output value, index Tensors to appropriate sizes (i.e. the same as
  // the input Tensor, except at dim=dimension, the size is 1)
  THCTensor_preserveReduceDimSemantics(
      state, values, ndim, dimension, keepdim);
  THCTensor_preserveReduceDimSemantics(
      state, indices, ndim, dimension, keepdim);
  std::vector<int64_t> dim = THTensor_sizesLegacyNoScalars(input);
  dim[dimension] = 1;
  THCTensor_(resize)(state, values, dim, {});
  THCudaLongTensor_resize(state, indices, dim, {});

  // If sliceSize is 1, copy input to values and set indices
  if (sliceSize == 1) {
    THCTensor_(copy)(state, values, input);
    THCudaLongTensor_fill(state, indices, 0);
    if (!keepdim) {
      THCTensor_(squeeze1d)(state, values, values, dimension);
      THCudaLongTensor_squeeze1d(state, indices, indices, dimension);
    }
    return;
  }

  // Requirements for fused kernel implementation:
  //
  // 1. sliceSize <= 2 * max threads per block
  // 2. uses one block per slice, so number of slices must be less than the maximum number of blocks for
  // a kernel launch
  // 3. Can use 32-bit index math for indexing (mainly just for implementation conciseness, could be changed)
  if (sliceSize <= MAX_BLOCK_SIZE &&
      slices <= MAX_GRID_SIZE &&
      THCTensor_canUse32BitIndexMath(state, input)) {
    // Beginning our optimized implementation. First thing we want to do is to transpose
    // the input Tensor along the sort dimension, and then make it contiguous
    transposed = THCTensor_(newTranspose)(state, input, dimension, ndim - 1);
    contiguous = THCTensor_(newContiguous)(state, transposed);

    // We also need to view the values and indices Tensors as transposed in order to
    // properly determine the offset into the underlying storage in which to place the
    // mode and index for a particular set of dimension values
    valuesTransposed = THCTensor_(newTranspose)(state, values, dimension, ndim-1);
    indicesTransposed = THCudaLongTensor_newTranspose(state, indices, dimension, ndim-1);

    // Set-up TensorInfo structs for passing to kernel
    TensorInfo<scalar_t, unsigned int> tiValues = getTensorInfo<scalar_t, THCTensor, unsigned int>(state, valuesTransposed);
    TensorInfo<int64_t, unsigned int> tiIndices = getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, indicesTransposed);

    // The number of blocks is the number of slices that we need to calculate the mode for. Each block
    // is responsible for computing a single mode
    dim3 grid;
    THC_getGridFromTiles(slices, grid);

    // The blocksize is two elements per thread, rounded up to the nearest power of 2
    int64_t ceilPowerOf2 = nextHighestPowerOf2(sliceSize);

    // Macro that calls kernel --> note that we set the block dimensions here, and
    // the amount of shared memory
  #define HANDLE_MODE(SIZE)                                                             \
  {                                                                                     \
    const dim3 blockSize(SIZE / 2);                                                     \
    const auto memsize = (sizeof(scalar_t) * SIZE) + (2 * SIZE * sizeof(unsigned int)); \
    computeMode<scalar_t, SIZE>                                                         \
      <<<grid, blockSize, memsize, c10::cuda::getCurrentCUDAStream()>>>(                \
        THCTensor_(data)(state, contiguous), tiValues, tiIndices, sliceSize);           \
    C10_CUDA_KERNEL_LAUNCH_CHECK();                                                     \
  }

    // Tradeoff between compilation time and the number of specializations. Ideally we would have
    // one HANDLE_MODE for each power of 2
    switch(ceilPowerOf2) {
      case 2048:
        HANDLE_MODE(2048)
        break;
      case 1024:
      case 512:
      case 256:
        HANDLE_MODE(1024)
        break;
      case 128:
      case 64:
        HANDLE_MODE(128)
        break;
      case 32:
      case 16:
      case 8:
      case 4:
      case 2:
        HANDLE_MODE(32)
        break;
      case 1:
      default:
        TORCH_INTERNAL_ASSERT(false);
    }
    THCudaCheck(cudaGetLastError());

    THCTensor_(free)(state, transposed);
    THCTensor_(free)(state, contiguous);
    THCTensor_(free)(state, valuesTransposed);
    THCudaLongTensor_free(state, indicesTransposed);
  } else {
    // Beginning our naive implementation: We don't want to mutate the input Tensor, but
    // we need to be able to sort the inputs along the dimension in order to calculate the
    // mode. Additionally, its ideal if the data along the dimension is contiguous. So
    // we transpose the dimension with the innermost dimension and make a new contiguous
    // version that we can use.
    transposed = THCTensor_(newClone)(state, input);
    THCTensor_(transpose)(state, transposed, NULL, dimension, ndim - 1);
    contiguous = THCTensor_(newContiguous)(state, transposed);
    THCTensor_(free)(state, transposed);

    // We also need to view the values and indices Tensors as transposed in order to
    // properly determine the offset into the underlying storage in which to place the
    // mode and index for a particular set of dimension values
    valuesTransposed = THCTensor_(newTranspose)(state, values, dimension, ndim - 1);
    indicesTransposed = THCudaLongTensor_newTranspose(state, indices, dimension, ndim - 1);

    // Position is a Storage that will store the dimension values we are processing
    position = THLongStorage_newWithSize(ndim - 1);

    // Sort Buffer is a Storage that will be used in the internal sort required to calculate
    // the mode efficiently
    sortBuffer = THCudaLongStorage_newWithSize(state, sliceSize);

    // Call mode
    THCTensor_(dimApplyMode)(state, valuesTransposed, indicesTransposed, contiguous, sortBuffer, dimension, position, 0);

    THCTensor_(free)(state, contiguous);
    THLongStorage_free(position);
    THCTensor_(free)(state, valuesTransposed);
    THCudaLongTensor_free(state, indicesTransposed);
    THCudaLongStorage_free(state, sortBuffer);
  }

  if (!keepdim) {
    THCTensor_(squeeze1d)(state, values, values, dimension);
    THCudaLongTensor_squeeze1d(state, indices, indices, dimension);
  }
}

#undef MAX_GRID_SIZE
#undef MAX_BLOCK_SIZE

#endif
