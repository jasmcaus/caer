
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPointwise.cu"
#else

#include <ATen/MemoryOverlap.h>
#include <ATen/NamedTensorUtils.h>

#if !defined(THC_REAL_IS_BOOL)

static void propagate_names_if_named_tensor_enabled(THCTensor* result, THCTensor* src) {
  at::namedinference::propagate_names(result, src);
}

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)             \
  struct Tensor_##NAME##_##REAL##_Op {                                  \
    __device__ __forceinline__ void operator()(scalar_t* out, scalar_t* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(scalar_t* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));       \
    at::assert_no_internal_overlap(self_);                              \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1<scalar_t>(state, self_, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
    propagate_names_if_named_tensor_enabled(self_, src);                \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, REAL) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<scalar_t>::sqrt,  Real)

#endif
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(crossKernel)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, x, y));

  int64_t sx = THCTensor_(stride)(state, x, dimension);
  int64_t sy = THCTensor_(stride)(state, y, dimension);
  int64_t so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, nself, nx, ny, TensorCrossOp<scalar_t>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}

namespace {
c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl> retainTensorImpl(THCTensor* self) {
  c10::raw::intrusive_ptr::incref(self);
  return c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl>::reclaim(self);
}
}

void THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
  at::mul_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)));
}

#endif
#endif
