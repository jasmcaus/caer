
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorTopK.cu"
#else

#include <c10/macros/Macros.h>
#include <c10/cuda/CUDAException.h>

void THCTensor_(topk)(THCState* state,
                      THCTensor *topK,
                      THCudaLongTensor *indices,
                      THCTensor *input_,
                      int64_t k, int dim, int dir, int sorted) {
  THAssert(topK != NULL && indices != NULL && input_ != NULL);
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, topK, indices, input_));
  dim = at::maybe_wrap_dim(dim, input_);
  THArgCheck(THCTensor_(nDimension)(state, topK) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimension(state, indices) <= MAX_CUTORCH_DIMS, 3, CUTORCH_DIM_WARNING);
  int numDims = THCTensor_(nDimensionLegacyNoScalars)(state, input_);
  THArgCheck(numDims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  THArgCheck(dim >= 0 && dim < numDims, 6, "dim not in range");

  int64_t sliceSize = THCTensor_(sizeLegacyNoScalars)(state, input_, dim);
  THArgCheck(k >= 0 && k <= sliceSize, 5, "k not in range for dimension");

  THCTensor *input = THCTensor_(newContiguous)(state, input_);

  // Build the output size, which is the dim being selected set to
  // size k
  std::vector<int64_t> topKSize = input->sizes().vec();
  if (topKSize.size() > 0) {
    topKSize[dim] = k;
  }
  THCTensor_(resize)(state, topK, topKSize, {});
  THCudaLongTensor_resize(state, indices, topKSize, {});

  // static_cast is required to ensure that the correct type (INDEX_T)
  // is provided to the kernel for the arguments.

#define RUN_K(INDEX_T, DIM, DIR)                                        \
  gatherTopK<scalar_t, INDEX_T, DIM, DIR>                               \
    <<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(            \
      inputInfo,                                                        \
      static_cast<INDEX_T>(sliceSize),                                  \
      static_cast<INDEX_T>(k),                                          \
      static_cast<INDEX_T>(inputSlices),                                \
      /* The actual dimension that the k-selection is running in */     \
      /* may have changed from collapseDims() */                        \
      static_cast<INDEX_T>(inputInfo.strides[collapseInputDim]),        \
      topKInfo,                                                         \
      static_cast<INDEX_T>(topKSlices),                                 \
      static_cast<INDEX_T>(topKInfo.strides[collapseTopKDim]),          \
      indicesInfo,                                                      \
      static_cast<INDEX_T>(indicesInfo.strides[collapseIndicesDim]));   \
  C10_CUDA_KERNEL_LAUNCH_CHECK()

#define RUN_DIR(INDEX_T, DIM)                   \
  if (dir) {                                    \
    RUN_K(INDEX_T, DIM, true);                  \
  } else {                                      \
    RUN_K(INDEX_T, DIM, false);                 \
  }

#define RUN_DIM(INDEX_T)                        \
  if (allDims == 1) {                           \
    RUN_DIR(INDEX_T, 1);                        \
  } else if (allDims == 2) {                    \
    RUN_DIR(INDEX_T, 2);                        \
  } else if (allDims == 3) {                    \
    RUN_DIR(INDEX_T, 3);                        \
  } else {                                      \
    RUN_DIR(INDEX_T, -1);                       \
  }

#define RUN_T(INDEX_T)                                                  \
  TensorInfo<scalar_t, INDEX_T> inputInfo =                             \
    getTensorInfo<scalar_t, THCTensor, INDEX_T>(state, input);          \
  TensorInfo<scalar_t, INDEX_T> topKInfo =                              \
    getTensorInfo<scalar_t, THCTensor, INDEX_T>(state, topK);           \
  TensorInfo<int64_t, INDEX_T> indicesInfo =                            \
    getTensorInfo<int64_t, THCudaLongTensor, INDEX_T>(state, indices);  \
                                                                        \
  /* We use these structures solely to find the offset to */            \
  /* each slice we are operating on */                                  \
  inputInfo.sizes[dim] = 1;                                             \
  topKInfo.sizes[dim] = 1;                                              \
  indicesInfo.sizes[dim] = 1;                                           \
                                                                        \
  /* Collapse all other dims */                                         \
  int collapseInputDim = inputInfo.collapseDims(dim);                   \
  int collapseTopKDim = topKInfo.collapseDims(dim);                     \
  int collapseIndicesDim = indicesInfo.collapseDims(dim);               \
                                                                        \
  int64_t inputSlices = 1;                                              \
  for (int i = 0; i < inputInfo.dims; ++i) {                            \
    inputSlices *= inputInfo.sizes[i];                                  \
  }                                                                     \
  int64_t topKSlices = 1;                                               \
  for (int i = 0; i < topKInfo.dims; ++i) {                             \
    topKSlices *= topKInfo.sizes[i];                                    \
  }                                                                     \
                                                                        \
  dim3 grid;                                                            \
  if (!THC_getGridFromTiles(inputSlices, grid)) {                       \
    THError("Slice to sort is too large");                              \
  }                                                                     \
                                                                        \
  dim3 block(std::min(THCRoundUp(sliceSize, (int64_t) C10_WARP_SIZE), (int64_t) 1024)); \
                                                                        \
  /* This is used as a template parameter to calculate indices. */      \
  /* We only specialize it if all collapsed dim sizes are the */        \
  /* same; otherwise, we use -1 which is the specialization */          \
  /* parameter for arbitrary dimensions */                              \
  int allDims = inputInfo.dims;                                         \
  if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {        \
    allDims = -1;                                                       \
  }                                                                     \
                                                                        \
  RUN_DIM(INDEX_T);

  // the below is safe with 0-dimensional tensors because it is based on
  // THCTensorInfo which implicitly expands to 1-dimensional.
  if (THCTensor_nElement(state, input) > 0) {
    // Based on required index size, run the algorithm with the
    // appropriate index type
    if (THCTensor_canUse32BitIndexMath(state, input) &&
        THCTensor_canUse32BitIndexMath(state, topK) &&
        THCTensor_canUse32BitIndexMath(state, indices)) {
      RUN_T(uint32_t);
    } else {
      RUN_T(uint64_t);
    }
  }
#undef RUN_T
#undef RUN_DIM
#undef RUN_DIR
#undef RUN_K

  // Sort the results if the user wants them sorted, since our
  // selection routine does not ensure sorting
  if (sorted && THCTensor_(numel)(state, topK) > 1) {
    // FIXME: the k/v inplace sort along slice only works for size <=
    // 2048 at the moment
    // Workaround:
    // CUDA 8 uses more shared memory than 7.5 for bitonicSortKVInPlace,
    // and so for the double word types,
    // we get "too many resources requested for launch" in the 2048 case
#if CUDA_VERSION >= 8000
#if defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_LONG)
    int maxSliceSize = 1024;
#else
    int maxSliceSize = 2048;
#endif
#else
    int maxSliceSize = 2048;
#endif
    if (sliceSize <= maxSliceSize) {
      // This avoids any memory allocations and performs all sorting
      // work inplace along the slice
      THCTensor_(sortKeyValueInplace)(state, topK, indices, dim, dir);
    } else {
      // Depend upon the backup sort that returns indices, which we
      // can use in conjunction with gather to produce the original
      // indices.
      // This is not the most efficient implementation, especially since
      // there are memory allocations performed here. If the user desires
      // greater performance, they should torch.gather() the results
      // themselves using the reported indices, providing previously
      // allocated tensors to receive the results.
      THCTensor* sortedTopK = THCTensor_(new)(state);
      THCudaLongTensor* sortedIndices = THCudaLongTensor_new(state);
      THCTensor_(sort)(state, sortedTopK, sortedIndices, topK, dim, dir);

      THCudaLongTensor* sortedTopKIndices = THCudaLongTensor_new(state);

      THCudaLongTensor_resizeAs(state, sortedTopKIndices, indices);
      THCudaLongTensor_gather(state, sortedTopKIndices, indices, dim, sortedIndices);

      THCTensor_(freeCopyTo)(state, sortedTopK, topK);
      THCudaLongTensor_freeCopyTo(state, sortedTopKIndices, indices);
      THCudaLongTensor_free(state, sortedIndices);
    }
  }

  THCudaLongTensor_free(state, input);

  THCudaCheck(cudaGetLastError());
}

#endif // THC_GENERIC_FILE
