
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorScatterGather.cu"
#else

#include <c10/cuda/CUDAException.h>

#define RUN(TYPE, DIMS, REAL)                                           \
  THCudaTensor_gatherKernel<TYPE, REAL, DIMS>                           \
  <<<grid, block, 0, c10::cuda::getCurrentCUDAStream(curDevice)>>>(     \
    tensorInfo, srcInfo, indexInfo, dim, (TYPE)totalElements);          \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

void THCTensor_(gather)(THCState* state, THCTensor *tensor,
                         THCTensor *src, int dim, THCudaLongTensor *index) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, tensor, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  dim = at::maybe_wrap_dim(dim, src);
  THArgCheck(THCudaLongTensor_nDimensionLegacyNoScalars(state, index) == THCTensor_(nDimensionLegacyNoScalars)(state, src), 4,
             "Index tensor must have same dimensions as input tensor");
  THArgCheck(tensor->sizes().equals(index->sizes()), 4,
             "Index tensor must have the same size as output tensor.");
  THArgCheck(dim >= 0 && dim < THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 3,
             "Index dimension is out of bounds");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) == THCTensor_(nDimensionLegacyNoScalars)(state, tensor), 2,
             "Input tensor must have same dimensions as output tensor");

  for (int d = 0; d < THCTensor_(nDimensionLegacyNoScalars)(state, tensor); d++) {
    if (d != dim) {
      THArgCheck(THCTensor_(sizeLegacyNoScalars)(state, tensor, d) == THCTensor_(sizeLegacyNoScalars)(state, src, d), 2,
                 "Input tensor must have same size as output tensor apart from the specified dimension");
    }
  }

  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, tensor) <= MAX_CUTORCH_DIMS,
             1, CUTORCH_DIM_WARNING);


  const ptrdiff_t totalElements = THCudaLongTensor_nElement(state, index);
  const dim3 block = getApplyBlock();
  dim3 grid;
  int curDevice = -1;
  cudaGetDevice(&curDevice);
  THArgCheck(getApplyGrid(state, totalElements, grid, curDevice), 1, CUTORCH_DIM_WARNING);

  THCTensor* oldTensor = NULL;
  if (THCTensor_maybeOverlappingIndices(state, tensor)) {
    oldTensor = tensor;
    tensor = THCTensor_(newContiguous)(state, tensor);
  }

  if (totalElements > 0) {
    if (THCTensor_canUse32BitIndexMath(state, tensor) &&
        THCTensor_canUse32BitIndexMath(state, src) &&
        THCTensor_canUse32BitIndexMath(state, index)) {
      TensorInfo<scalar_t, unsigned int> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, tensor);
      TensorInfo<scalar_t, unsigned int> srcInfo =
        getTensorInfo<scalar_t, THCTensor, unsigned int>(state, src);
      TensorInfo<int64_t, unsigned int> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, index);

      // Specialize for a small number of dimensions.
      switch (indexInfo.dims) {
        case 1:
          RUN(unsigned int, 1, scalar_t);
          break;
        case 2:
          RUN(unsigned int, 2, scalar_t);
          break;
        case 3:
          RUN(unsigned int, 3, scalar_t);
          break;
        default:
          RUN(unsigned int, -1, scalar_t);
          break;
      }
    } else {
      TensorInfo<scalar_t, uint64_t> tensorInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, tensor);
      TensorInfo<scalar_t, uint64_t> srcInfo =
        getTensorInfo<scalar_t, THCTensor, uint64_t>(state, src);
      TensorInfo<int64_t, uint64_t> indexInfo =
        getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, index);
      RUN(uint64_t, -1, scalar_t);
    }
  }

  if (oldTensor) {
    THCTensor_copyIgnoringOverlaps<scalar_t>(state, oldTensor, tensor);
    THCTensor_(free)(state, tensor);
    tensor = oldTensor;
  }
  THCudaCheck(cudaGetLastError());
}

#undef RUN

#endif
