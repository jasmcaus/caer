
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorIndex.cu"
#else

#include <ATen/cuda/CUDAContext.h>
#include <ATen/MemoryOverlap.h>
#include <c10/cuda/CUDAException.h>

// Check tensor dimensions for index operations, and return the slice size.
// src can be nullptr in case of indexFill: in that case it is ignored.
static ptrdiff_t THCTensor_(getSliceSize)(THCState *state, THCTensor *dst,
                                          int dim,
                                          THCudaLongTensor *index,
                                          THCTensor *src)
{
  int dstDims = THCTensor_(nDimensionLegacyNoScalars)(state, dst);
  int srcDims = (src == nullptr) ? dstDims : THCTensor_(nDimensionLegacyNoScalars)(state, src);

  THArgCheck(THCudaLongTensor_nDimensionLegacyNoScalars(state, index) == 1, 4,
             "expecting vector of indices");
  THArgCheck(dim >= 0 && dim < dstDims, 2, "Indexing dim is out of bounds");

  ptrdiff_t dstSliceSize = 1;
  for (int d = 0; d < dstDims; d++) {
    if (d != dim) {
      dstSliceSize *= THTensor_sizeLegacyNoScalars(dst, d);
    }
  }

  if (src == nullptr) return dstSliceSize;

  THArgCheck(dim < srcDims, 3, "Indexing dim is out of bounds");
  THArgCheck(THCudaLongTensor_nElement(state, index) == THTensor_sizeLegacyNoScalars(src, dim), 4,
             "length of src.size[dim] is not equal to length of indices");

  ptrdiff_t srcSliceSize = 1;
  bool mismatch = false;

  if (dstDims != srcDims) mismatch = true;

  for (int d = 0; d < srcDims; d++) {
    if (d != dim) {
      srcSliceSize *= THTensor_sizeLegacyNoScalars(src, d);
      if (!mismatch && THTensor_sizeLegacyNoScalars(dst, d) != THTensor_sizeLegacyNoScalars(src, d)) mismatch = true;
    }
  }

  THArgCheck(dstSliceSize == srcSliceSize, 2,
             "Source/destination tensor have different slice sizes (%ld vs %ld)",
             dstSliceSize, srcSliceSize);

  if (mismatch) {
    static bool warningShown = false;
    if (!warningShown) {
      warningShown = true;
      fprintf(stderr,
              "Warning: source/destination slices have same size but different "
              "shape for an index operation.  This behavior is deprecated.\n");
    }
  }

  return dstSliceSize;
}

// Compare the stride between adjacent slices (sliceStride) with strides in the
// other dimensions (i.e., strides *inside* each slice).
//
// - Returns true if some dimension inside the slice has lower stride than
//   sliceStride.  The simplest example is a 2-D contiguous tensor with sliceDim
//   == 0 (that is, each slice is a row).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "index-major order".  For example, if thread count equals slice size, then
//   all threads process slice #0 in lockstep, and then slice #1, and so on.
//
// - Otherwise (i.e., sliceStride has the lowest value), this function returns
//   false.  The simplest example is a 2-D contiguous tensor with sliceDim == 1
//   (each slice is a column).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "elementInSlice-major order".  For example, each thread can process element
//   #0 of every slice, and then element #1 of every slice, and so on.
bool THCTensor_(indexShouldBeMajor)(TensorInfo<scalar_t, unsigned int> &info,
                                    int sliceDim)
{
  // The stride between adjacent slices (e.g., between element #0 of slice #100
  // and element #0 of slice #101).
  unsigned int sliceStride = info.strides[sliceDim];

  for (int i = 0; i < info.dims; ++i) {
    if (i != sliceDim && info.sizes[i] > 1 && info.strides[i] < sliceStride) {
      return true;
    }
  }

  return false;
}

void THCTensor_(indexCopy)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, THCTensor *src)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));

  dim  = at::maybe_wrap_dim(dim, dst);
  int dims = THCTensor_(nDimensionLegacyNoScalars)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCTensor_(nDimensionLegacyNoScalars)(state, src);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 5, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimensionLegacyNoScalars(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize = THCTensor_(getSliceSize)(state, dst, dim, indices, src);
  ptrdiff_t srcTotalSize = THCTensor_(nElement)(state, src);
  int64_t dstCopyDimSize = THCTensor_(sizeLegacyNoScalars)(state, dst, dim);
  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  if (sliceSize == 0) {
    return;
  }

  cudaStream_t stream = c10::cuda::getCurrentCUDAStream();
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)    \
  indexCopySmallIndex<TENSOR_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>  \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                \
      dstInfo, srcInfo, indicesInfo,                                 \
      dstCopyDim, srcCopyDim, sliceSize, dstCopyDimSize);            \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                         \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)   \
  indexCopyLargeIndex<TENSOR_TYPE, TYPE,                       \
                      DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(          \
      dstInfo, srcInfo, indicesInfo,                           \
      dstCopyDim, srcCopyDim, srcTotalSize,                    \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                 \
      dstCopyDimSize);                                         \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(srcTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(srcTotalSize, (ptrdiff_t)128));

  if (THCTensor_canUse32BitIndexMath(state, dst) &&
      THCTensor_canUse32BitIndexMath(state, src) &&
      THCTensor_canUse32BitIndexMath(state, indices)) {
    TensorInfo<scalar_t, unsigned int> dstInfo =
      getTensorInfo<scalar_t, THCTensor, unsigned int>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<scalar_t, unsigned int> srcInfo =
      getTensorInfo<scalar_t, THCTensor, unsigned int>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        SMALL_INDEX(scalar_t, unsigned int, 1, 1, -2);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        SMALL_INDEX(scalar_t, unsigned int, 2, 2, -2);
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        SMALL_INDEX(scalar_t, unsigned int, 3, 3, -2);
      } else {
        SMALL_INDEX(scalar_t, unsigned int, -1, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstCopyDim);

      if (dstInfo.dims == 1 && srcInfo.dims == 1 && indContig) {
        LARGE_INDEX(scalar_t, unsigned int, 1, 1, -2, true);
      } else if (dstInfo.dims == 2 && srcInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(scalar_t, unsigned int, 2, 2, -2, true);
        } else {
          LARGE_INDEX(scalar_t, unsigned int, 2, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && srcInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(scalar_t, unsigned int, 3, 3, -2, true);
        } else {
          LARGE_INDEX(scalar_t, unsigned int, 3, 3, -2, false);
        }
      } else {
        LARGE_INDEX(scalar_t, unsigned int, -1, -1, -1, true);
      }
    }
  } else {
    TensorInfo<scalar_t, uint64_t> dstInfo =
      getTensorInfo<scalar_t, THCTensor, uint64_t>(state, dst);
    int dstCopyDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstCopyDim);

    TensorInfo<scalar_t, uint64_t> srcInfo =
      getTensorInfo<scalar_t, THCTensor, uint64_t>(state, src);
    int srcCopyDim = srcInfo.collapseDims(dim);
    srcInfo.reduceDim(srcCopyDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(scalar_t, uint64_t, -1, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}

static void THCTensor_(sort_indices)(THCState *state, THCudaLongTensor *index, THCTensor *src) {
  THCThrustAllocator thrustAlloc(state);

  auto index_iter = thrust::device_ptr<int64_t>(THCudaLongTensor_data(state, index));
  auto src_iter = thrust::device_ptr<scalar_t>(THCTensor_(data)(state, src));
  auto numel = THCTensor_(numel)(state, src);

  thrust::sort_by_key(
    thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
    index_iter, index_iter + numel,
    src_iter, ThrustLTOp<int64_t>());
}

void THCTensor_(put)(THCState *state, THCTensor *dst, THCudaLongTensor *index, THCTensor *src, int accumulate)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, dst, src));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, index));

  ptrdiff_t dstSize = THCTensor_(nElement)(state, dst);
  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, index);
  THArgCheck(THCTensor_(nElement)(state, src) == numIndices,
    3, "src should have the same number of elements as index");

  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, dst) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  THArgCheck(THCudaLongTensor_nDimensionLegacyNoScalars(state, index) <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  if (numIndices == 0) {
    return;
  }

  if (accumulate) {
    // wrap indices so to replace negative indices
    THCudaLongTensor* sorted_index = THCudaLongTensor_new(state);
    THCudaLongTensor_resizeAs(state, sorted_index, index);
    THC_pointwiseApply2<int64_t, int64_t>(state, sorted_index, index, WrapIndexOp(dstSize));

    THCTensor* sorted_src = THCTensor_(newClone)(state, src);

    THCTensor_(sort_indices)(state, sorted_index, sorted_src);
    dispatchTakePut<scalar_t, TensorPutAccumulateOp>(state, dst, sorted_src, sorted_index);

    THCTensor_(free)(state, sorted_src);
    THCudaLongTensor_free(state, sorted_index);
  } else {
    dispatchTakePut<scalar_t, TensorPutOp>(state, dst, src, index);
  }
}

void THCTensor_(indexFill)(THCState *state, THCTensor *dst, int dim, THCudaLongTensor *indices, scalar_t val)
{
  at::NoNamesGuard guard;
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, dst));
  THCAssertSameGPU(THCudaLongTensor_checkGPU(state, 1, indices));
  dim = at::maybe_wrap_dim(dim, dst);
  int dims = THCTensor_(nDimensionLegacyNoScalars)(state, dst);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);
  dims = THCudaLongTensor_nDimensionLegacyNoScalars(state, indices);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 4, CUTORCH_DIM_WARNING);
  at::assert_no_overlap(dst, indices);
  if (at::has_internal_overlap(dst) == at::MemOverlap::YES) {
    TORCH_WARN(
      "Use of index_fill_ on expanded tensors is deprecated. "
      "Please clone() the tensor before performing this operation. "
      "This also applies to advanced indexing e.g. tensor[mask] = scalar");
  }

  // The `src` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  ptrdiff_t sliceSize =
    THCTensor_(getSliceSize)(state, dst, dim, indices, nullptr);
  ptrdiff_t dstTotalSize = THCTensor_(nElement)(state, dst);
  int64_t dstFillDimSize = THCTensor_(sizeLegacyNoScalars)(state, dst, dim);
  ptrdiff_t numIndices = THCudaLongTensor_nElement(state, indices);

  if (sliceSize == 0) {
    return;
  }
  cudaStream_t stream = c10::cuda::getCurrentCUDAStream();
  int indContig = THCudaLongTensor_isContiguous(state, indices);

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM)   \
  indexFillSmallIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(      \
      dstInfo, indicesInfo,                                \
      dstFillDim, sliceSize, dstFillDimSize, val);         \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR)   \
  indexFillLargeIndex<TENSOR_TYPE, TYPE, DST_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                    \
      dstInfo, indicesInfo,                                              \
      dstFillDim, sliceSize * numIndices,                                \
      (IDX_IS_MAJOR) ? sliceSize : numIndices,                           \
      dstFillDimSize, val);                                              \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(dstTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(dstTotalSize, (ptrdiff_t)128));

  if (THCTensor_canUse32BitIndexMath(state, dst) &&
      THCTensor_canUse32BitIndexMath(state, indices)) {
    TensorInfo<scalar_t, unsigned int> dstInfo =
      getTensorInfo<scalar_t, THCTensor, unsigned int>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, unsigned int> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, indices);
    indicesInfo.collapseDims();

    // A reasonable choice for when to have each thread iterate over
    // indices to choose
    if (numIndices <= 16) {
      if (dstInfo.dims == 1 && indContig) {
        SMALL_INDEX(scalar_t, unsigned int, 1, -2);
      } else if (dstInfo.dims == 2 && indContig) {
        SMALL_INDEX(scalar_t, unsigned int, 2, -2);
      } else if (dstInfo.dims == 3 && indContig) {
        SMALL_INDEX(scalar_t, unsigned int, 3, -2);
      } else {
        SMALL_INDEX(scalar_t, unsigned int, -1, -1);
      }
    } else {
      bool indexIsMajor = THCTensor_(indexShouldBeMajor)(dstInfo, dstFillDim);

      if (dstInfo.dims == 1 && indContig) {
        LARGE_INDEX(scalar_t, unsigned int, 1, -2, true);
      } else if (dstInfo.dims == 2 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(scalar_t, unsigned int, 2, -2, true);
        } else {
          LARGE_INDEX(scalar_t, unsigned int, 2, -2, false);
        }
      } else if (dstInfo.dims == 3 && indContig) {
        if (indexIsMajor) {
          LARGE_INDEX(scalar_t, unsigned int, 3, -2, true);
        } else {
          LARGE_INDEX(scalar_t, unsigned int, 3, -2, false);
        }
      } else {
        LARGE_INDEX(scalar_t, unsigned int, -1, -1, true);
      }
    }
  } else {
    TensorInfo<scalar_t, uint64_t> dstInfo =
      getTensorInfo<scalar_t, THCTensor, uint64_t>(state, dst);
    int dstFillDim = dstInfo.collapseDims(dim);
    dstInfo.reduceDim(dstFillDim);

    TensorInfo<int64_t, uint64_t> indicesInfo =
      getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, indices);
    indicesInfo.collapseDims();

    LARGE_INDEX(scalar_t, uint64_t, -1, -1, true);
  }

#undef SMALL_INDEX
#undef LARGE_INDEX
}


#endif
