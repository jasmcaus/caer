
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCStorage.cu"
#else

void THCStorage_(fill)(THCState *state, THCStorage *self, scalar_t value)
{
  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<scalar_t> self_data(THCStorage_(data)(state, self));
  thrust::fill(
#if CUDA_VERSION >= 7000 || defined __HIP_PLATFORM_HCC__
      thrust::cuda::par(thrustAlloc).on(c10::cuda::getCurrentCUDAStream()),
#endif
      self_data,
      self_data + (self->nbytes() / sizeof(scalar_t)),
      value);
}

void THCStorage_(
    resizeBytes)(THCState* state, THCStorage* self, ptrdiff_t size_bytes) {
  THCStorage_resizeBytes(state, self, size_bytes);
}

int THCStorage_(getDevice)(THCState* state, const THCStorage* storage) {
  return THCStorage_getDevice(state, storage);
}

#endif
