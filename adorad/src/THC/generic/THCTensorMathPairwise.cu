
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPairwise.cu"
#else

#include <ATen/NamedTensorUtils.h>

#if !defined(THC_REAL_IS_BOOL)

void THCTensor_(mul)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorMulConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorMulConstantOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(fmod)(THCState *state, THCTensor *self_, THCTensor *src_, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src_));
  if (self_ == src_) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorFmodOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src_);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src_, TensorFmodOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

#endif

#endif
