
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathReduce.cu"
#else

#if !defined(THC_REAL_IS_BOOL)

void THCTensor_(prod)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(1),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(renorm)(THCState *state, THCTensor* self, THCTensor* src, scalar_t value, int dimension, scalar_t maxnorm)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  dimension = at::maybe_wrap_dim(dimension, src);
  THArgCheck(dimension >= 0 && dimension < THCTensor_(nDimensionLegacyNoScalars)(state, src), 3, "invalid dimension");
  THArgCheck(THCNumerics<scalar_t>::gt(value, scalar_cast<scalar_t>(0)), 2, "non-positive-norm not supported");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) > 1, 1, "need at least 2 dimensions");

  THCTensor *self_;
  THCTensor *src_ = THCTensor_(newTranspose)(state, src, dimension, 0);
  THCTensor *data = THCTensor_(newClone)(state, src_);
  int64_t numel = THCTensor_(nElement)(state, data);

  if (numel > 0) {
    ptrdiff_t size = numel / THTensor_sizeLegacyNoScalars(data, 0);
    dim3 grid( THTensor_sizeLegacyNoScalars(data, 0));
    // NOTE: only with this specific number of threads can this work on GPUs with a warp size != 32 (such as AMD). Do not alter w/o changing buffer size in kernel.
    dim3 threads(32);

    THCTensor_kernel_renorm<scalar_t, accreal>
      <<<grid, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(THCTensor_(data)(state, data),
        scalar_cast<accreal>(value), size, scalar_cast<accreal>(maxnorm));

    // Do not replace with C10_CUDA_KERNEL_LAUNCH_CHECK() yet as it exhibits different behaviour from THError().
    // THError() calls the an error handler, or throws std::runtime_error if a custom handler hasn't been registered.
    cudaError_t errcode = cudaGetLastError();
    if(errcode != cudaSuccess)
      THError(cudaGetErrorString(errcode));
  }

  THCTensor_(free)(state, src_);
  self_ = THCTensor_(newTranspose)(state, data, dimension, 0);
  THCTensor_(resizeAs)(state, self, self_);
  THCTensor_(freeCopyTo)(state, self_, self);
  THCTensor_(free)(state, data);
}

#endif

#endif

#endif
