#include <THC/THCBlas.h>
#include <THC/THCGeneral.h>
#include <TH/THHalf.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDABlas.h>

#include <algorithm>
#include <mutex>

#ifdef __HIP_PLATFORM_HCC__
#include <hip/hip_version.h>
#endif

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, float alpha, float *a, int64_t lda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  at::cuda::blas::gemm<float>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::Half alpha, at::Half *a, int64_t lda, at::Half *b, int64_t ldb, at::Half beta, at::Half *c, int64_t ldc)
{
  at::cuda::blas::gemm<at::Half>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

void THCudaBlas_Bgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::BFloat16 alpha, at::BFloat16 *a, int64_t lda, at::BFloat16 *b, int64_t ldb, at::BFloat16 beta, at::BFloat16 *c, int64_t ldc)
{
  at::cuda::blas::gemm<at::BFloat16>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, double alpha, double *a, int64_t lda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  at::cuda::blas::gemm<double>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}
