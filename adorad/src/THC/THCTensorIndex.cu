#include "hip/hip_runtime.h"
#include <THC/THC.h>
#include <THC/THCTensorMath.h>
#include <THC/THCGeneral.h>
#include <THC/THCBlas.h>
#include <THC/THCTensorCopy.h>
#include <TH/THHalf.h>
#include <THC/THCApply.cuh>
#include <THC/THCReduce.cuh>
#include <THC/THCDeviceUtils.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCAtomics.cuh>
#include <THC/THCThrustAllocator.cuh>
#include <THC/THCTensorSort.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <algorithm> // for std::min
#include <c10/macros/Macros.h>
#include <ATen/WrapDimUtils.h>

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexCopyLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexCopySmallIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<T, IndexType> src,
                                    TensorInfo<int64_t, IndexType> indices,
                                    int dstCopyDim,
                                    int srcCopyDim,
                                    IndexType innerSize,
                                    int64_t dstCopyDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType srcIndex = 0; srcIndex < indices.sizes[0]; ++srcIndex) {
    // Lua indices begin at 1
    IndexType dstIndex =
      indices.data[IndexToOffset<int64_t, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstCopyDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
        IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);

      dstOffset += dstIndex * dst.strides[dstCopyDim];

      IndexType srcOffset =
        IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcCopyDim];

      dst.data[dstOffset] = src.data[srcOffset];
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexCopySmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexCopyLargeIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<T, IndexType> src,
                                    TensorInfo<int64_t, IndexType> indices,
                                    int dstCopyDim,
                                    int srcCopyDim,
                                    IndexType totalSize,
                                    IndexType innerSize,
                                    int64_t dstCopyDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType srcIndex, elementInSlice;
    if (IndexIsMajor) {
      srcIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      srcIndex = linearIndex % innerSize;
    }

    // Lua indices begin at 1
    IndexType dstIndex =
      indices.data[IndexToOffset<int64_t, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstCopyDimSize);

    IndexType dstOffset =
      IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstCopyDim];

    IndexType srcOffset =
      IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcCopyDim];

    dst.data[dstOffset] = src.data[srcOffset];
  }
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexFillLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndexType, int DstDim, int IdxDim>
__global__ void indexFillSmallIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<int64_t, IndexType> indices,
                                    int dstFillDim,
                                    IndexType innerSize,
                                    int64_t dstFillDimSize,
                                    T val) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType dstIndex = 0; dstIndex < indices.sizes[0]; ++dstIndex) {
    // Lua indices begin at 1
    IndexType dstIndex_ =
      indices.data[IndexToOffset<int64_t, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex_ < dstFillDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
          IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex_ * dst.strides[dstFillDim];

      dst.data[dstOffset] = val;
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexFillSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndexType, int DstDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexFillLargeIndex(TensorInfo<T, IndexType> dst,
                                    TensorInfo<int64_t, IndexType> indices,
                                    int dstFillDim,
                                    IndexType totalSize,
                                    IndexType innerSize,
                                    int64_t dstFillDimSize,
                                    T val) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType dstIndex, elementInSlice;
    if (IndexIsMajor) {
      dstIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      dstIndex = linearIndex % innerSize;
    }

    // Lua indices begin at 1
    IndexType dstIndex_ =
      indices.data[IndexToOffset<int64_t, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex_ < dstFillDimSize);

    IndexType dstOffset =
      IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex_ * dst.strides[dstFillDim];

    dst.data[dstOffset] = val;
  }
}

template <int Dims, typename T, typename IndexType>
__device__ __forceinline__ IndexType indexToOffset(
    const TensorInfo<T, IndexType>& info,
    int64_t index,
    IndexType size)
{
  IndexType linearIndex = static_cast<IndexType>(index);
  CUDA_KERNEL_ASSERT(linearIndex < size && linearIndex >= -size);
  if (linearIndex < 0) {
    linearIndex += size;
  }
  return IndexToOffset<T, IndexType, Dims>::get(linearIndex, info);
}

struct WrapIndexOp {
  WrapIndexOp(int64_t size) : size(size) {}

  __device__ __forceinline__ void operator()(int64_t* out, int64_t* in) {
    auto idx = *in;
    CUDA_KERNEL_ASSERT(idx < size && idx >= -size);
    *out = idx < 0 ? idx + size : idx;
  }

  int64_t size;
};

template <typename T, typename IndexType, int Dims>
struct TensorPutOp {
  TensorPutOp(TensorInfo<T, IndexType> info, IndexType numel, int64_t*, int64_t*)
    : info(info), numel(numel) {}

  __device__ __forceinline__ void operator()(T* value, int64_t* index) {
    auto offset = indexToOffset<Dims>(info, *index, numel);
    info.data[offset] = *value;
  }

  const TensorInfo<T, IndexType> info;
  IndexType numel;
};

template <typename T, typename IndexType, int Dims>
struct TensorPutAccumulateOp {
  TensorPutAccumulateOp(TensorInfo<T, IndexType> info, IndexType numel, int64_t* start, int64_t* end)
    : info(info), numel(numel), start(start), end(end) {}

  __device__ __forceinline__ void operator()(T* value, int64_t* index) {
    if (index == start || *index != *(index - 1)) {
      int64_t linear_index = *index;
      auto offset = indexToOffset<Dims>(info, linear_index, numel);
      do {
        info.data[offset] = THCNumerics<T>::add(info.data[offset], *value);
        index++;
        value++;
      } while (index != end && *index == linear_index);
    }
  }

  const TensorInfo<T, IndexType> info;
  IndexType numel;
  int64_t* start;
  int64_t* end;
};


template<typename IndexType, typename T, template<class, class, int> class Op, typename TensorType>
void dispatchTakePutImpl(THCState *state, TensorType *a, TensorType *b, THCudaLongTensor *index) {
  // These are only valid if index is contiguous
  auto start = THCudaLongTensor_data(state, index);
  auto end = start + THCudaLongTensor_numel(state, index);

  auto aInfo = getTensorInfo<T, TensorType, IndexType>(state, a);
  aInfo.collapseDims();
  auto numel = THCTensor_nElement(state, a);
  if (aInfo.isContiguous()) {
    auto op = Op<T, IndexType, -2>(aInfo, numel, start, end);
    THC_pointwiseApply2<T, int64_t>(state, b, index, op);
  } else {
    auto op = Op<T, IndexType, -1>(aInfo, numel, start, end);
    THC_pointwiseApply2<T, int64_t>(state, b, index, op);
  }
}

template<typename T, template<class, class, int> class Op, typename TensorType>
void dispatchTakePut(THCState *state, TensorType *a, TensorType *b, THCudaLongTensor *index) {
  if (THCTensor_canUse32BitIndexMath(state, a, INT_MAX)) {
    dispatchTakePutImpl<int32_t, T, Op>(state, a, b, index);
  } else {
    dispatchTakePutImpl<int64_t, T, Op>(state, a, b, index);
  }
}

#include <THC/generic/THCTensorIndex.cu>
#include <THC/THCGenerateAllTypes.h>

#include <THC/generic/THCTensorIndex.cu>
#include <THC/THCGenerateBoolType.h>

#include <THC/generic/THCTensorIndex.cu>
#include <THC/THCGenerateBFloat16Type.h>
