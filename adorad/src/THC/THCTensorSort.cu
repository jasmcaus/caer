#include <THC/THCTensorSort.cuh>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAException.h>

void THCudaLongTensor_fillSliceWithIndex(THCState* state,
                                         THCudaLongTensor* t,
                                         int dim) {
  int64_t dims = THCudaLongTensor_nDimensionLegacyNoScalars(state, t);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  ptrdiff_t inElements = THCudaLongTensor_nElement(state, t);
  if (inElements > 0) {
    int64_t sliceSize = THCudaLongTensor_sizeLegacyNoScalars(state, t, dim);
    ptrdiff_t numSlices = inElements / sliceSize;

    dim3 grid;
    if (!THC_getGridFromTiles(numSlices, grid)) {
      THError("Slice to fill with indices is too large");
    }

    int64_t maxThreads =
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
    int64_t numThreads = sliceSize;
    if (numThreads > maxThreads) {
      numThreads = maxThreads;
    }

    dim3 block(numThreads);

#define FILL_INDEX(T, DIM)                                         \
    fillSliceWithIndex<T, DIM>                                     \
      <<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(     \
        info, numSlices, sliceSize, info.strides[collapseDim]);    \
    C10_CUDA_KERNEL_LAUNCH_CHECK()


    if (THCTensor_canUse32BitIndexMath(state, t)) {
      TensorInfo<int64_t, uint32_t> info =
        getTensorInfo<int64_t, THCudaLongTensor, unsigned int>(state, t);
      info.reduceDim(dim);
      int collapseDim = info.collapseDims(dim);

      if (info.isContiguous()) {
        FILL_INDEX(unsigned int, -2);
      } else {
        if (info.dims == 1) {
          FILL_INDEX(unsigned int, 1);
        } else if (info.dims == 2) {
          FILL_INDEX(unsigned int, 2);
        } else {
          FILL_INDEX(unsigned int, -1);
        }
      }
    } else {
      TensorInfo<int64_t, uint64_t> info =
        getTensorInfo<int64_t, THCudaLongTensor, uint64_t>(state, t);
      info.reduceDim(dim);
      int collapseDim = info.collapseDims(dim);

      // catch-all implementation
      FILL_INDEX(uint64_t, -1);
    }

#undef FILL_INDEX
  }
}
