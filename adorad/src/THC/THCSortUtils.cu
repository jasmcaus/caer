#include "hip/hip_runtime.h"
#include <THC/THCSortUtils.cuh>

// Returns 2^(ceil(lg(n)) from Stanford bit twiddling hacks
uint64_t nextHighestPowerOf2(uint64_t n) {
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
#ifndef _MSC_VER
  n |= n >> 32;
#endif
  n++;

  return n;
}
