#include "hip/hip_runtime.h"
#include <THC/THCSleep.h>


__global__ void spin_kernel(int64_t cycles)
{
  // see concurrentKernels CUDA sampl
  int64_t start_clock = clock64();
  int64_t clock_offset = 0;
  while (clock_offset < cycles)
  {
    clock_offset = clock64() - start_clock;
  }
}

void THC_sleep(THCState* state, int64_t cycles)
{
  dim3 grid(1);
  dim3 block(1);
  spin_kernel<<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(cycles);
  THCudaCheck(hipGetLastError());
}
