#include <THC/THCGeneral.h>
#include <THC/THCTensorMath.h>
#include <THC/THCTensorCopy.h>
#include <THC/THCTensorMathMagma.cuh>
#include <THC/THCTensor.hpp>
#include <THC/THCStorage.hpp>
#include <algorithm>
#include <ATen/native/cuda/MiscUtils.h>

#ifdef USE_MAGMA
#include <magma.h>
#else
#include <THC/THCBlas.h>
#endif

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

#define NoMagma(name) "No CUDA implementation of '" #name "'. Install MAGMA and rebuild cutorch (http://icl.cs.utk.edu/magma/)"

void THCMagma_init(THCState *state)
{
#ifdef USE_MAGMA
  magma_init();
#endif
}

#include <THC/generic/THCTensorMathMagma.cu>
#include <THC/THCGenerateAllTypes.h>
